/* Include files */

#include "PD_rbdl_sfun.h"
#include "c1_PD_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance);
static void initialize_params_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct
  *chartInstance);
static void enable_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance);
static void disable_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance);
static void c1_do_animation_call_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct
  *chartInstance);
static void ext_mode_exec_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct
  *chartInstance);
static void set_sim_state_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c1_st);
static void sf_gateway_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance);
static void mdl_start_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct
  *chartInstance);
static void mdl_cleanup_runtime_resources_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *
  chartInstance);
static void initSimStructsc1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance);
static void c1_eML_blk_kernel(SFc1_PD_rbdlInstanceStruct *chartInstance, real_T
  c1_b_u[6], real_T c1_b_y[6]);
static void c1_emlrt_marshallIn(SFc1_PD_rbdlInstanceStruct *chartInstance, const
  mxArray *c1_b_y, const char_T *c1_identifier, real_T c1_c_y[6]);
static void c1_b_emlrt_marshallIn(SFc1_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c1_b_u, const emlrtMsgIdentifier *c1_parentId, real_T c1_b_y[6]);
static uint8_T c1_c_emlrt_marshallIn(SFc1_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c1_b_is_active_c1_PD_rbdl, const char_T *c1_identifier);
static uint8_T c1_d_emlrt_marshallIn(SFc1_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c1_b_u, const emlrtMsgIdentifier *c1_parentId);
static void init_dsm_address_info(SFc1_PD_rbdlInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc1_PD_rbdlInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c1_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c1_is_active_c1_PD_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct
  *chartInstance)
{
}

static void enable_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c1_do_animation_call_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct
  *chartInstance)
{
  const mxArray *c1_b_y = NULL;
  const mxArray *c1_c_y = NULL;
  const mxArray *c1_d_y = NULL;
  const mxArray *c1_st;
  c1_st = NULL;
  c1_b_y = NULL;
  sf_mex_assign(&c1_b_y, sf_mex_createcellmatrix(2, 1), false);
  c1_c_y = NULL;
  sf_mex_assign(&c1_c_y, sf_mex_create("y", *chartInstance->c1_y, 0, 0U, 1U, 0U,
    1, 6), false);
  sf_mex_setcell(c1_b_y, 0, c1_c_y);
  c1_d_y = NULL;
  sf_mex_assign(&c1_d_y, sf_mex_create("y",
    &chartInstance->c1_is_active_c1_PD_rbdl, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c1_b_y, 1, c1_d_y);
  sf_mex_assign(&c1_st, c1_b_y, false);
  return c1_st;
}

static void set_sim_state_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c1_st)
{
  const mxArray *c1_b_u;
  c1_b_u = sf_mex_dup(c1_st);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_b_u, 0)), "y",
                      *chartInstance->c1_y);
  chartInstance->c1_is_active_c1_PD_rbdl = c1_c_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c1_b_u, 1)), "is_active_c1_PD_rbdl");
  sf_mex_destroy(&c1_b_u);
  sf_mex_destroy(&c1_st);
}

static void sf_gateway_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c1_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c1_eML_blk_kernel(chartInstance, *chartInstance->c1_u, *chartInstance->c1_y);
  c1_do_animation_call_c1_PD_rbdl(chartInstance);
}

static void mdl_start_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
  hipError_t c1_errCode;
  c1_errCode = hipGetLastError();
  if (c1_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c1_errCode, (char_T *)hipGetErrorName
                       (c1_errCode), (char_T *)hipGetErrorString(c1_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c1_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct
  *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *
  chartInstance)
{
}

static void initSimStructsc1_PD_rbdl(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c1_PD_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c1_nameCaptureInfo = NULL;
  c1_nameCaptureInfo = NULL;
  sf_mex_assign(&c1_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c1_nameCaptureInfo;
}

static void c1_eML_blk_kernel(SFc1_PD_rbdlInstanceStruct *chartInstance, real_T
  c1_b_u[6], real_T c1_b_y[6])
{
  c1_b_y[0] = c1_b_u[1];
  c1_b_y[1] = c1_b_u[2];
  c1_b_y[2] = c1_b_u[3];
  c1_b_y[3] = c1_b_u[0];
  c1_b_y[4] = c1_b_u[4];
  c1_b_y[5] = c1_b_u[5];
}

static void c1_emlrt_marshallIn(SFc1_PD_rbdlInstanceStruct *chartInstance, const
  mxArray *c1_b_y, const char_T *c1_identifier, real_T c1_c_y[6])
{
  emlrtMsgIdentifier c1_thisId;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_b_y), &c1_thisId, c1_c_y);
  sf_mex_destroy(&c1_b_y);
}

static void c1_b_emlrt_marshallIn(SFc1_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c1_b_u, const emlrtMsgIdentifier *c1_parentId, real_T c1_b_y[6])
{
  real_T c1_dv[6];
  int32_T c1_i;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_b_u), c1_dv, 1, 0, 0U, 1, 0U, 1, 6);
  for (c1_i = 0; c1_i < 6; c1_i++) {
    c1_b_y[c1_i] = c1_dv[c1_i];
  }

  sf_mex_destroy(&c1_b_u);
}

static uint8_T c1_c_emlrt_marshallIn(SFc1_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c1_b_is_active_c1_PD_rbdl, const char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  uint8_T c1_b_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_b_y = c1_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c1_b_is_active_c1_PD_rbdl), &c1_thisId);
  sf_mex_destroy(&c1_b_is_active_c1_PD_rbdl);
  return c1_b_y;
}

static uint8_T c1_d_emlrt_marshallIn(SFc1_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c1_b_u, const emlrtMsgIdentifier *c1_parentId)
{
  uint8_T c1_b_y;
  uint8_T c1_c_u;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_b_u), &c1_c_u, 1, 3, 0U, 0, 0U, 0);
  c1_b_y = c1_c_u;
  sf_mex_destroy(&c1_b_u);
  return c1_b_y;
}

static void init_dsm_address_info(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc1_PD_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c1_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c1_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c1_y = (real_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c1_PD_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1797144223U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(73479396U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1318086020U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3398444244U);
}

mxArray *sf_c1_PD_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c1_PD_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c1_PD_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c1_PD_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiD+wMjAwAakOYCYiQECWKF8RqgYI1ycBS6uAMQllQWpIPHiomTPFCCdl5gL5ie"
    "WVnjmpeWDzbdgQJjPhsV8RiTzOaHiEPDBnjL9Ig4g/QZI+lkI6BcAsiqh4QILH/LtV3CgTD/Efg"
    "8C7hdBcT+En1kcn5hcklmWGp9sGB/gEl+UlJLDgOQvAEWbF0A="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c1_PD_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sz4RAlAineuVo8CCV5Hr2sG";
}

static void sf_opaque_initialize_c1_PD_rbdl(void *chartInstanceVar)
{
  initialize_params_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c1_PD_rbdl(void *chartInstanceVar)
{
  enable_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c1_PD_rbdl(void *chartInstanceVar)
{
  disable_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c1_PD_rbdl(void *chartInstanceVar)
{
  sf_gateway_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c1_PD_rbdl(SimStruct* S)
{
  return get_sim_state_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c1_PD_rbdl(SimStruct* S, const mxArray *st)
{
  set_sim_state_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c1_PD_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_PD_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*)
      chartInstanceVar);
    ((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc1_PD_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c1_PD_rbdl(void *chartInstanceVar)
{
  mdl_start_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c1_PD_rbdl(void *chartInstanceVar)
{
  mdl_terminate_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c1_PD_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc1_PD_rbdl((SFc1_PD_rbdlInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c1_PD_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV01v00AQdaKCAEFVoUpwQKI3OAICqVygrZNApJZWdVqO1daexKusd939SBskuHHkP3Dhyt+",
    "BOxdOqBIHrsw6bhoc22mJWgHCkuus/ebtzLzZ2a1Taa45eM3ife+641zE5yW8q87gupCOKyP34P",
    "2Mcycdv0MjbqINIkmknNKLkwg2QQlmNBW8ydsiF0Z5GyRwH7GxkLqITdHIMMq7DcN9y6dehtQPv",
    "VAYFqygLQnWOesjW2z0BvLUqARfNwACHUphOmGDkc7QY6n33RD8rjJRWQgKtGdi65ZaM0zTmEH9",
    "APwmV5qgx+rYN08TDa4+KAzTRqq8I6CIYkYJz402JMqDGBOsYSsO8O+60RhUFuaHROoVCEkP1Cr",
    "tJpyCQ5aTKvywSznRQlLC6hFzreG4bxsM/VkTAbCShKBvKxJINxaU62L9vQZGWudkl0ENdk2nmM",
    "2DPWPF36awD7Iwb21X9ECSDqzzwkmThNQPErWGVTIO0zSCbSKXfdRPQVBYvVg5yiOoE7TQoggGS",
    "ZBN1ZK0h+ktZDNR01bmpCVjooHYahIsYav3oEyFIVvD5y5hTBXCWiJehR6whLVGNCmHDVjzcUrR",
    "oCUwwba8i1eD4RSFT2Gu4AHNlauXASR95wU2ll+RvlFaRC4Wb211dfzzOKzJNcg28SGvC0hCFWD",
    "OkvQWswVUWe0RiF7pxL088KBCJqEc1Ta8ti9kF3NS0kSOQ7CKFgIj1UEtcSVsKVw0ZTCr5SScT/",
    "wQAttgKIM1XDaIzcmJsq1tGdddj+p+DZQvaZyjqsFVh22obguqH8MW73KxzxtSRF7a4wfpBcBqI",
    "5JT3lnB/iX7DZw9T4pkP3OO97OrJ9jPjuyyz7sjPJUcHmfkmZ33crV83ir+qqR2SyN21zLzzGTs",
    "LG4O7x8f9j5+uXnl67e3rz99f/ymNc3876un2/9n0/Gto0Y7XDi9sXq12Ocjfs3k8N8Y4Z9Lx+r",
    "Vw81ltkw5mG2x6Lrbj57LB+pZwndYKfe3mvH36P2C7fhYZUl9Sr8ZpAcTOyZmsF1b/sURfy9OyM",
    "fl9P3gOnw6nf38UlbHmQn2c/irn6nb359/YWk6+5PpPZ/Rez7Z13eI7Raw49/f2ajtyN2AZdfj7",
    "9b3ae2cc7b7W/z8b3f2up9kvzpvO+ec7aaN77T79p+GL+v/TgY/9wfHMe156qzxn53TnXtup+Mn",
    "w39t3JCyIOeUm37GI2s77+s/UKc/Af73r2c=",
    ""
  };

  static char newstr [1237] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c1_PD_rbdl(SimStruct *S)
{
  const char* newstr = sf_c1_PD_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2926683639U));
  ssSetChecksum1(S,(3892975843U));
  ssSetChecksum2(S,(3615327726U));
  ssSetChecksum3(S,(1417558515U));
}

static void mdlRTW_c1_PD_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c1_PD_rbdl(SimStruct *S)
{
  SFc1_PD_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc1_PD_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc1_PD_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc1_PD_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc1_PD_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c1_PD_rbdl;
  chartInstance->chartInfo.initializeChart = sf_opaque_initialize_c1_PD_rbdl;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c1_PD_rbdl;
  chartInstance->chartInfo.mdlTerminate = sf_opaque_mdl_terminate_c1_PD_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c1_PD_rbdl;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c1_PD_rbdl;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c1_PD_rbdl;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c1_PD_rbdl;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c1_PD_rbdl;
  chartInstance->chartInfo.getSimStateInfo = sf_get_sim_state_info_c1_PD_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c1_PD_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c1_PD_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c1_JITStateAnimation,
    chartInstance->c1_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c1_PD_rbdl(chartInstance);
}

void c1_PD_rbdl_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c1_PD_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c1_PD_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c1_PD_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c1_PD_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
