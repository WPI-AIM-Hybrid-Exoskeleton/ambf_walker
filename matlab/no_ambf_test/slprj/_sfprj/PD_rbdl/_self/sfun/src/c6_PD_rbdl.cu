/* Include files */

#include "PD_rbdl_sfun.h"
#include "c6_PD_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance);
static void initialize_params_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct
  *chartInstance);
static void enable_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance);
static void disable_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance);
static void c6_do_animation_call_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct
  *chartInstance);
static void ext_mode_exec_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct
  *chartInstance);
static void set_sim_state_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c6_st);
static void sf_gateway_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance);
static void mdl_start_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct
  *chartInstance);
static void mdl_cleanup_runtime_resources_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *
  chartInstance);
static void initSimStructsc6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance);
static void c6_eML_blk_kernel(SFc6_PD_rbdlInstanceStruct *chartInstance, real_T
  c6_b_u[6], real_T c6_b_y[6]);
static void c6_emlrt_marshallIn(SFc6_PD_rbdlInstanceStruct *chartInstance, const
  mxArray *c6_b_y, const char_T *c6_identifier, real_T c6_c_y[6]);
static void c6_b_emlrt_marshallIn(SFc6_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId, real_T c6_b_y[6]);
static uint8_T c6_c_emlrt_marshallIn(SFc6_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c6_b_is_active_c6_PD_rbdl, const char_T *c6_identifier);
static uint8_T c6_d_emlrt_marshallIn(SFc6_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId);
static void init_dsm_address_info(SFc6_PD_rbdlInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc6_PD_rbdlInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c6_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c6_is_active_c6_PD_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct
  *chartInstance)
{
}

static void enable_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c6_do_animation_call_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct
  *chartInstance)
{
  const mxArray *c6_b_y = NULL;
  const mxArray *c6_c_y = NULL;
  const mxArray *c6_d_y = NULL;
  const mxArray *c6_st;
  c6_st = NULL;
  c6_b_y = NULL;
  sf_mex_assign(&c6_b_y, sf_mex_createcellmatrix(2, 1), false);
  c6_c_y = NULL;
  sf_mex_assign(&c6_c_y, sf_mex_create("y", *chartInstance->c6_y, 0, 0U, 1U, 0U,
    1, 6), false);
  sf_mex_setcell(c6_b_y, 0, c6_c_y);
  c6_d_y = NULL;
  sf_mex_assign(&c6_d_y, sf_mex_create("y",
    &chartInstance->c6_is_active_c6_PD_rbdl, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c6_b_y, 1, c6_d_y);
  sf_mex_assign(&c6_st, c6_b_y, false);
  return c6_st;
}

static void set_sim_state_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c6_st)
{
  const mxArray *c6_b_u;
  c6_b_u = sf_mex_dup(c6_st);
  c6_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c6_b_u, 0)), "y",
                      *chartInstance->c6_y);
  chartInstance->c6_is_active_c6_PD_rbdl = c6_c_emlrt_marshallIn(chartInstance,
    sf_mex_dup(sf_mex_getcell(c6_b_u, 1)), "is_active_c6_PD_rbdl");
  sf_mex_destroy(&c6_b_u);
  sf_mex_destroy(&c6_st);
}

static void sf_gateway_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c6_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c6_eML_blk_kernel(chartInstance, *chartInstance->c6_u, *chartInstance->c6_y);
  c6_do_animation_call_c6_PD_rbdl(chartInstance);
}

static void mdl_start_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
  hipError_t c6_errCode;
  c6_errCode = hipGetLastError();
  if (c6_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c6_errCode, (char_T *)hipGetErrorName
                       (c6_errCode), (char_T *)hipGetErrorString(c6_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c6_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct
  *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *
  chartInstance)
{
}

static void initSimStructsc6_PD_rbdl(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c6_PD_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c6_nameCaptureInfo = NULL;
  c6_nameCaptureInfo = NULL;
  sf_mex_assign(&c6_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c6_nameCaptureInfo;
}

static void c6_eML_blk_kernel(SFc6_PD_rbdlInstanceStruct *chartInstance, real_T
  c6_b_u[6], real_T c6_b_y[6])
{
  c6_b_y[0] = c6_b_u[1];
  c6_b_y[1] = c6_b_u[2];
  c6_b_y[2] = c6_b_u[3];
  c6_b_y[3] = c6_b_u[0];
  c6_b_y[4] = c6_b_u[4];
  c6_b_y[5] = c6_b_u[5];
}

static void c6_emlrt_marshallIn(SFc6_PD_rbdlInstanceStruct *chartInstance, const
  mxArray *c6_b_y, const char_T *c6_identifier, real_T c6_c_y[6])
{
  emlrtMsgIdentifier c6_thisId;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c6_b_y), &c6_thisId, c6_c_y);
  sf_mex_destroy(&c6_b_y);
}

static void c6_b_emlrt_marshallIn(SFc6_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId, real_T c6_b_y[6])
{
  real_T c6_dv[6];
  int32_T c6_i;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_b_u), c6_dv, 1, 0, 0U, 1, 0U, 1, 6);
  for (c6_i = 0; c6_i < 6; c6_i++) {
    c6_b_y[c6_i] = c6_dv[c6_i];
  }

  sf_mex_destroy(&c6_b_u);
}

static uint8_T c6_c_emlrt_marshallIn(SFc6_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c6_b_is_active_c6_PD_rbdl, const char_T *c6_identifier)
{
  emlrtMsgIdentifier c6_thisId;
  uint8_T c6_b_y;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_b_y = c6_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c6_b_is_active_c6_PD_rbdl), &c6_thisId);
  sf_mex_destroy(&c6_b_is_active_c6_PD_rbdl);
  return c6_b_y;
}

static uint8_T c6_d_emlrt_marshallIn(SFc6_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId)
{
  uint8_T c6_b_y;
  uint8_T c6_c_u;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_b_u), &c6_c_u, 1, 3, 0U, 0, 0U, 0);
  c6_b_y = c6_c_u;
  sf_mex_destroy(&c6_b_u);
  return c6_b_y;
}

static void init_dsm_address_info(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc6_PD_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c6_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c6_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c6_y = (real_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c6_PD_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1797144223U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(73479396U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1318086020U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3398444244U);
}

mxArray *sf_c6_PD_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c6_PD_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c6_PD_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c6_PD_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiD+wMjAwAakOYCYiQECWKF8RqgYI1ycBS6uAMQllQWpIPHiomTPFCCdl5gL5ie"
    "WVnjmpeWDzbdgQJjPhsV8RiTzOaHiEPDBnjL9Ig4g/QZI+lkI6BcAsiqh4QILH/LtV3CgTD/Efg"
    "8C7hdBcT+En1kcn5hcklmWGp9sFh/gEl+UlJLDgOQvAEYEF0U="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c6_PD_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sz4RAlAineuVo8CCV5Hr2sG";
}

static void sf_opaque_initialize_c6_PD_rbdl(void *chartInstanceVar)
{
  initialize_params_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c6_PD_rbdl(void *chartInstanceVar)
{
  enable_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c6_PD_rbdl(void *chartInstanceVar)
{
  disable_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c6_PD_rbdl(void *chartInstanceVar)
{
  sf_gateway_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c6_PD_rbdl(SimStruct* S)
{
  return get_sim_state_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c6_PD_rbdl(SimStruct* S, const mxArray *st)
{
  set_sim_state_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c6_PD_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_PD_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*)
      chartInstanceVar);
    ((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc6_PD_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c6_PD_rbdl(void *chartInstanceVar)
{
  mdl_start_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c6_PD_rbdl(void *chartInstanceVar)
{
  mdl_terminate_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c6_PD_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc6_PD_rbdl((SFc6_PD_rbdlInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c6_PD_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV01v00AQdaKCAEFVoUpwQKI3OCIEqFygrZNApJZWdVqO1daexKusd939SBskuHHkP3Dhyt+",
    "BOxdOqBIHrsw6bhoc22mJWgHCkuus/ebtzLzZ2a1Taa45eM3ife+641zE5yW8q87gupCOKyP34P",
    "2Mcycdv0MjbqINIkmknNKLkwg2QQlmNBW8ydsiF0Z5GyRwH7GxkLqITdHIMMq7DcN9y6dehtQPv",
    "VAYFqygLQnWOesjW2z0BvLUqARfNwACHUphOmGDkc7QY6n33RD8rjJRWQgKtGdi65ZaM0zTmEH9",
    "APwmV5qgx+rYN08TDa4+KAzTRqq8I6CIYkYJz402JMqDGBOsYSsO8O+60RhUFuaHROoVCEkP1Cr",
    "tJpyCQ5aTKvywSznRQlLC6hFzreG4bxsM/VkTAbCShKBvKxJINxaU62L9vQZGWudkl0ENdk2nmM",
    "2DPWPF36awD7Iwb21X9ECSDqzzwkmThNQPErWGVTIO0zSCbSKXfdRPQVBYvVg5yiOoE7TQoggGS",
    "ZBN1ZK0h+ktZDNR01bmpCVjooHYahIsYav3oEyFIVvD5y5hTBXCWiJehR6whLVGNCmHDVjzcUrR",
    "oCUwwba8i1eD4RSFT2Gu4AHNlauXASR95wU2ll+RvlFaRC4Wb211dfzzOKzJNcg28SGvC0hCFWD",
    "OkvQWswVUWe0RiF7pxL088KBCJqEc1Ta8ti9kF3NS0kSOQ7CKFgIj1UEtcSVsKVw0ZTCr5SScT/",
    "wQAttgKIM1XDaIzcmJsq1tGdddj+p+DZQvaZyjqsFVh22obguqH8MW73KxzxtSRF7a4wfpBcBqI",
    "5JT3lnB/iX7DZw9T4pkP3OO97OrJ9jPjuyyz7sjPJUcHmfkmZ33crV83ir+qqR2SyN21zLzzGTs",
    "LG4O7x8f9j5+uXnl67e3rz99f/ymNc3876un2/9n0/Gto0Y7XDi9sXq12Ocjfs3k8N8Y4Z9Lx+r",
    "Vg81ltkw5mG2x6LrbD5/L++pZwndYKfe3mvH36P2C7fhYZUl9Sr8ZpAcTOyZmsF1b/sURfy9OyM",
    "fl9P3gOnw6nf38UlbHmQn2c/irn6nb359/YWk6+5PpPZ/Rez7Z13eI7Raw4z/a2ajtyN2AZdfj7",
    "9b3ae2cc7b7W/z8b3f2up9kvzpvO+ec7aaN77T79p+GL+v/TgY/9wfHMe156qzxn53TnXtup+Mn",
    "w39t3JCyIOeUm37GI2s77+s/UKc/ASU/r2w=",
    ""
  };

  static char newstr [1237] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c6_PD_rbdl(SimStruct *S)
{
  const char* newstr = sf_c6_PD_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2926683639U));
  ssSetChecksum1(S,(3892975843U));
  ssSetChecksum2(S,(3615327726U));
  ssSetChecksum3(S,(1417558515U));
}

static void mdlRTW_c6_PD_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c6_PD_rbdl(SimStruct *S)
{
  SFc6_PD_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc6_PD_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc6_PD_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc6_PD_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc6_PD_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c6_PD_rbdl;
  chartInstance->chartInfo.initializeChart = sf_opaque_initialize_c6_PD_rbdl;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c6_PD_rbdl;
  chartInstance->chartInfo.mdlTerminate = sf_opaque_mdl_terminate_c6_PD_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c6_PD_rbdl;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c6_PD_rbdl;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c6_PD_rbdl;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c6_PD_rbdl;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c6_PD_rbdl;
  chartInstance->chartInfo.getSimStateInfo = sf_get_sim_state_info_c6_PD_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c6_PD_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c6_PD_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c6_JITStateAnimation,
    chartInstance->c6_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c6_PD_rbdl(chartInstance);
}

void c6_PD_rbdl_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c6_PD_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c6_PD_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c6_PD_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c6_PD_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
