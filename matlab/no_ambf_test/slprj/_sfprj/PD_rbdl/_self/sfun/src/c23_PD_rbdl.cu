#include "hip/hip_runtime.h"
/* Include files */

#include "PD_rbdl_sfun.h"
#include "c23_PD_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance);
static void initialize_params_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct
  *chartInstance);
static void enable_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance);
static void disable_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance);
static void c23_do_animation_call_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct
  *chartInstance);
static void ext_mode_exec_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct
  *chartInstance);
static void set_sim_state_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_st);
static void sf_gateway_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance);
static void mdl_start_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *
  chartInstance);
static void mdl_cleanup_runtime_resources_c23_PD_rbdl
  (SFc23_PD_rbdlInstanceStruct *chartInstance);
static void initSimStructsc23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance);
static void c23_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_b_msg, const char_T *c23_identifier,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_y);
static void c23_b_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_y);
static void c23_c_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId, uint8_T c23_y[8]);
static c23_SL_Bus_ROSVariableLengthArrayInfo c23_d_emlrt_marshallIn
  (SFc23_PD_rbdlInstanceStruct *chartInstance, const mxArray *c23_u, const
   emlrtMsgIdentifier *c23_parentId);
static uint32_T c23_e_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId);
static void c23_f_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId, real_T c23_y[6]);
static uint8_T c23_g_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_b_is_active_c23_PD_rbdl, const char_T *c23_identifier);
static uint8_T c23_h_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId);
static void c23_eML_blk_kernel(SFc23_PD_rbdlInstanceStruct *chartInstance,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg);
static __global__ void c23_sf_gateway_c23_PD_rbdl_kernel1
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg,
   c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg);
static __global__ void c23_sf_gateway_c23_PD_rbdl_kernel2
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg,
   c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg);
static __global__ void c23_sf_gateway_c23_PD_rbdl_kernel3
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg,
   c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_r);
static __global__ void c23_sf_gateway_c23_PD_rbdl_kernel4
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg,
   c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_r);
static __global__ void c23_sf_gateway_c23_PD_rbdl_kernel5(const
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_r,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg);
static __global__ void c23_sf_gateway_c23_PD_rbdl_kernel6(const
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_r,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg);
static __global__ void c23_eML_blk_kernel_kernel7
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg);
static __global__ void c23_eML_blk_kernel_kernel8(const uint8_T c23_uv[5],
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg);
static __global__ void c23_eML_blk_kernel_kernel9
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg);
static void init_dsm_address_info(SFc23_PD_rbdlInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc23_PD_rbdlInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c23_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c23_is_active_c23_PD_rbdl = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c23_gpu_uv, 5UL);
  hipMalloc(&chartInstance->c23_gpu_blankMsg, 184UL);
}

static void initialize_params_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct
  *chartInstance)
{
}

static void enable_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c23_do_animation_call_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct
  *chartInstance)
{
  static const char_T *c23_sv[8] = { "ModelName", "ModelName_SL_Info", "Q",
    "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  static const char_T *c23_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c23_b_y = NULL;
  const mxArray *c23_c_y = NULL;
  const mxArray *c23_d_y = NULL;
  const mxArray *c23_e_y = NULL;
  const mxArray *c23_f_y = NULL;
  const mxArray *c23_g_y = NULL;
  const mxArray *c23_h_y = NULL;
  const mxArray *c23_i_y = NULL;
  const mxArray *c23_j_y = NULL;
  const mxArray *c23_k_y = NULL;
  const mxArray *c23_l_y = NULL;
  const mxArray *c23_m_y = NULL;
  const mxArray *c23_n_y = NULL;
  const mxArray *c23_o_y = NULL;
  const mxArray *c23_p_y = NULL;
  const mxArray *c23_q_y = NULL;
  const mxArray *c23_r_y = NULL;
  const mxArray *c23_s_y = NULL;
  const mxArray *c23_st;
  const mxArray *c23_y = NULL;
  real_T c23_d_u[6];
  int32_T c23_i;
  int32_T c23_i1;
  int32_T c23_i2;
  int32_T c23_i3;
  uint32_T c23_b_u;
  uint32_T c23_c_u;
  uint32_T c23_e_u;
  uint32_T c23_f_u;
  uint32_T c23_g_u;
  uint32_T c23_h_u;
  uint32_T c23_i_u;
  uint32_T c23_j_u;
  uint32_T c23_u_CurrentLength;
  uint32_T c23_u_ReceivedLength;
  uint8_T c23_u[8];
  c23_st = NULL;
  c23_y = NULL;
  sf_mex_assign(&c23_y, sf_mex_createcellmatrix(2, 1), false);
  c23_b_y = NULL;
  sf_mex_assign(&c23_b_y, sf_mex_createstruct("structure", 8, c23_sv, 2, 1, 1),
                false);
  for (c23_i = 0; c23_i < 8; c23_i++) {
    c23_u[c23_i] = ((uint8_T *)&((char_T *)chartInstance->c23_msg)[0])[c23_i];
  }

  c23_c_y = NULL;
  sf_mex_assign(&c23_c_y, sf_mex_create("y", c23_u, 3, 0U, 1U, 0U, 1, 8), false);
  sf_mex_setfieldbynum(c23_b_y, 0, "ModelName", c23_c_y, 0);
  c23_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [8])[0];
  c23_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [8])[4];
  c23_d_y = NULL;
  sf_mex_assign(&c23_d_y, sf_mex_createstruct("structure", 2, c23_sv1, 2, 1, 1),
                false);
  c23_b_u = c23_u_CurrentLength;
  c23_e_y = NULL;
  sf_mex_assign(&c23_e_y, sf_mex_create("y", &c23_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_d_y, 0, "CurrentLength", c23_e_y, 0);
  c23_c_u = c23_u_ReceivedLength;
  c23_f_y = NULL;
  sf_mex_assign(&c23_f_y, sf_mex_create("y", &c23_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_d_y, 0, "ReceivedLength", c23_f_y, 1);
  sf_mex_setfieldbynum(c23_b_y, 0, "ModelName_SL_Info", c23_d_y, 1);
  for (c23_i1 = 0; c23_i1 < 6; c23_i1++) {
    c23_d_u[c23_i1] = ((real_T *)&((char_T *)chartInstance->c23_msg)[16])[c23_i1];
  }

  c23_g_y = NULL;
  sf_mex_assign(&c23_g_y, sf_mex_create("y", c23_d_u, 0, 0U, 1U, 0U, 1, 6),
                false);
  sf_mex_setfieldbynum(c23_b_y, 0, "Q", c23_g_y, 2);
  c23_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [64])[0];
  c23_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [64])[4];
  c23_h_y = NULL;
  sf_mex_assign(&c23_h_y, sf_mex_createstruct("structure", 2, c23_sv1, 2, 1, 1),
                false);
  c23_e_u = c23_u_CurrentLength;
  c23_i_y = NULL;
  sf_mex_assign(&c23_i_y, sf_mex_create("y", &c23_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_h_y, 0, "CurrentLength", c23_i_y, 0);
  c23_f_u = c23_u_ReceivedLength;
  c23_j_y = NULL;
  sf_mex_assign(&c23_j_y, sf_mex_create("y", &c23_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_h_y, 0, "ReceivedLength", c23_j_y, 1);
  sf_mex_setfieldbynum(c23_b_y, 0, "Q_SL_Info", c23_h_y, 3);
  for (c23_i2 = 0; c23_i2 < 6; c23_i2++) {
    c23_d_u[c23_i2] = ((real_T *)&((char_T *)chartInstance->c23_msg)[72])[c23_i2];
  }

  c23_k_y = NULL;
  sf_mex_assign(&c23_k_y, sf_mex_create("y", c23_d_u, 0, 0U, 1U, 0U, 1, 6),
                false);
  sf_mex_setfieldbynum(c23_b_y, 0, "Qd", c23_k_y, 4);
  c23_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [120])[0];
  c23_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [120])[4];
  c23_l_y = NULL;
  sf_mex_assign(&c23_l_y, sf_mex_createstruct("structure", 2, c23_sv1, 2, 1, 1),
                false);
  c23_g_u = c23_u_CurrentLength;
  c23_m_y = NULL;
  sf_mex_assign(&c23_m_y, sf_mex_create("y", &c23_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_l_y, 0, "CurrentLength", c23_m_y, 0);
  c23_h_u = c23_u_ReceivedLength;
  c23_n_y = NULL;
  sf_mex_assign(&c23_n_y, sf_mex_create("y", &c23_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_l_y, 0, "ReceivedLength", c23_n_y, 1);
  sf_mex_setfieldbynum(c23_b_y, 0, "Qd_SL_Info", c23_l_y, 5);
  for (c23_i3 = 0; c23_i3 < 6; c23_i3++) {
    c23_d_u[c23_i3] = ((real_T *)&((char_T *)chartInstance->c23_msg)[128])
      [c23_i3];
  }

  c23_o_y = NULL;
  sf_mex_assign(&c23_o_y, sf_mex_create("y", c23_d_u, 0, 0U, 1U, 0U, 1, 6),
                false);
  sf_mex_setfieldbynum(c23_b_y, 0, "Tau", c23_o_y, 6);
  c23_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [176])[0];
  c23_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [176])[4];
  c23_p_y = NULL;
  sf_mex_assign(&c23_p_y, sf_mex_createstruct("structure", 2, c23_sv1, 2, 1, 1),
                false);
  c23_i_u = c23_u_CurrentLength;
  c23_q_y = NULL;
  sf_mex_assign(&c23_q_y, sf_mex_create("y", &c23_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_p_y, 0, "CurrentLength", c23_q_y, 0);
  c23_j_u = c23_u_ReceivedLength;
  c23_r_y = NULL;
  sf_mex_assign(&c23_r_y, sf_mex_create("y", &c23_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_p_y, 0, "ReceivedLength", c23_r_y, 1);
  sf_mex_setfieldbynum(c23_b_y, 0, "Tau_SL_Info", c23_p_y, 7);
  sf_mex_setcell(c23_y, 0, c23_b_y);
  c23_s_y = NULL;
  sf_mex_assign(&c23_s_y, sf_mex_create("y",
    &chartInstance->c23_is_active_c23_PD_rbdl, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c23_y, 1, c23_s_y);
  sf_mex_assign(&c23_st, c23_y, false);
  return c23_st;
}

static void set_sim_state_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_st)
{
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest c23_r;
  const mxArray *c23_u;
  int32_T c23_i;
  int32_T c23_i1;
  c23_u = sf_mex_dup(c23_st);
  c23_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c23_u, 0)),
                       "msg", &c23_r);
  for (c23_i = 0; c23_i < 8; c23_i++) {
    ((uint8_T *)&((char_T *)chartInstance->c23_msg)[0])[c23_i] =
      c23_r.ModelName[c23_i];
  }

  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[0] = c23_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[4] = c23_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[64])[0] = c23_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[64])[4] = c23_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[120])[0] = c23_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[120])[4] = c23_r.Qd_SL_Info.ReceivedLength;
  for (c23_i1 = 0; c23_i1 < 6; c23_i1++) {
    ((real_T *)&((char_T *)chartInstance->c23_msg)[16])[c23_i1] = c23_r.Q[c23_i1];
    ((real_T *)&((char_T *)chartInstance->c23_msg)[72])[c23_i1] =
      c23_r.Qd[c23_i1];
    ((real_T *)&((char_T *)chartInstance->c23_msg)[128])[c23_i1] =
      c23_r.Tau[c23_i1];
  }

  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[176])[0] = c23_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[176])[4] = c23_r.Tau_SL_Info.ReceivedLength;
  chartInstance->c23_is_active_c23_PD_rbdl = c23_g_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c23_u, 1)),
     "is_active_c23_PD_rbdl");
  sf_mex_destroy(&c23_u);
  sf_mex_destroy(&c23_st);
}

static void sf_gateway_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest c23_r;
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_gpu_blankMsg;
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_gpu_msg;
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_gpu_r;
  hipMalloc(&c23_gpu_r, 184UL);
  hipMalloc(&c23_gpu_msg, 184UL);
  hipMalloc(&c23_b_gpu_blankMsg, 184UL);
  chartInstance->c23_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(c23_b_gpu_blankMsg, chartInstance->c23_blankMsg, 184UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 184UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_PD_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (c23_b_gpu_blankMsg, c23_gpu_msg);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 184UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c23_blankMsg, c23_b_gpu_blankMsg, 184UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[0] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[8])[0];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[4] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[8])[4];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[64])[0] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[64])[0];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[64])[4] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[64])[4];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[120])[0] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[120])[0];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[120])[4] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[120])[4];
  hipMemcpy(c23_b_gpu_blankMsg, chartInstance->c23_blankMsg, 184UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 184UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_PD_rbdl_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (c23_b_gpu_blankMsg, c23_gpu_msg);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 184UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c23_blankMsg, c23_b_gpu_blankMsg, 184UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[176])[0] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[176])[0];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[176])[4] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[176])[4];
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 184UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_PD_rbdl_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (c23_gpu_msg, c23_gpu_r);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 184UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c23_r, c23_gpu_r, 184UL, hipMemcpyDeviceToHost);
  c23_r.ModelName_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [8])[0];
  c23_r.ModelName_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [8])[4];
  c23_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [64])[0];
  c23_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [64])[4];
  c23_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [120])[0];
  c23_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [120])[4];
  hipMemcpy(c23_gpu_r, &c23_r, 184UL, hipMemcpyHostToDevice);
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 184UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_PD_rbdl_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (c23_gpu_msg, c23_gpu_r);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 184UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c23_r, c23_gpu_r, 184UL, hipMemcpyDeviceToHost);
  c23_r.Tau_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [176])[0];
  c23_r.Tau_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [176])[4];
  c23_eML_blk_kernel(chartInstance, &c23_r);
  hipMemcpy(c23_gpu_r, &c23_r, 184UL, hipMemcpyHostToDevice);
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 184UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_PD_rbdl_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (c23_gpu_r, c23_gpu_msg);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 184UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[0] = c23_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[4] = c23_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[64])[0] = c23_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[64])[4] = c23_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[120])[0] = c23_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[120])[4] = c23_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 184UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_PD_rbdl_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (c23_gpu_r, c23_gpu_msg);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 184UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[176])[0] = c23_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[176])[4] = c23_r.Tau_SL_Info.ReceivedLength;
  c23_do_animation_call_c23_PD_rbdl(chartInstance);
  hipFree(c23_b_gpu_blankMsg);
  hipFree(c23_gpu_msg);
  hipFree(c23_gpu_r);
}

static void mdl_start_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
  hipError_t c23_errCode;
  hipFree(*chartInstance->c23_gpu_uv);
  hipFree(chartInstance->c23_gpu_blankMsg);
  c23_errCode = hipGetLastError();
  if (c23_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c23_errCode, (char_T *)hipGetErrorName
                       (c23_errCode), (char_T *)hipGetErrorString(c23_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c23_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *
  chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c23_PD_rbdl
  (SFc23_PD_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc23_PD_rbdl(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c23_PD_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c23_nameCaptureInfo = NULL;
  c23_nameCaptureInfo = NULL;
  sf_mex_assign(&c23_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c23_nameCaptureInfo;
}

static void c23_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_b_msg, const char_T *c23_identifier,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_y)
{
  emlrtMsgIdentifier c23_thisId;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c23_b_msg), &c23_thisId,
    c23_y);
  sf_mex_destroy(&c23_b_msg);
}

static void c23_b_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_y)
{
  static const char_T *c23_fieldNames[8] = { "ModelName", "ModelName_SL_Info",
    "Q", "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  emlrtMsgIdentifier c23_thisId;
  c23_thisId.fParent = c23_parentId;
  c23_thisId.bParentIsCell = false;
  sf_mex_check_struct(c23_parentId, c23_u, 8, c23_fieldNames, 0U, NULL);
  c23_thisId.fIdentifier = "ModelName";
  c23_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c23_u,
    "ModelName", "ModelName", 0)), &c23_thisId, c23_y->ModelName);
  c23_thisId.fIdentifier = "ModelName_SL_Info";
  c23_y->ModelName_SL_Info = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "ModelName_SL_Info", "ModelName_SL_Info", 0)),
    &c23_thisId);
  c23_thisId.fIdentifier = "Q";
  c23_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c23_u, "Q",
    "Q", 0)), &c23_thisId, c23_y->Q);
  c23_thisId.fIdentifier = "Q_SL_Info";
  c23_y->Q_SL_Info = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "Q_SL_Info", "Q_SL_Info", 0)), &c23_thisId);
  c23_thisId.fIdentifier = "Qd";
  c23_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c23_u, "Qd",
    "Qd", 0)), &c23_thisId, c23_y->Qd);
  c23_thisId.fIdentifier = "Qd_SL_Info";
  c23_y->Qd_SL_Info = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c23_thisId);
  c23_thisId.fIdentifier = "Tau";
  c23_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c23_u, "Tau",
    "Tau", 0)), &c23_thisId, c23_y->Tau);
  c23_thisId.fIdentifier = "Tau_SL_Info";
  c23_y->Tau_SL_Info = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "Tau_SL_Info", "Tau_SL_Info", 0)), &c23_thisId);
  sf_mex_destroy(&c23_u);
}

static void c23_c_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId, uint8_T c23_y[8])
{
  int32_T c23_i;
  uint8_T c23_uv[8];
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), c23_uv, 1, 3, 0U, 1, 0U, 1, 8);
  for (c23_i = 0; c23_i < 8; c23_i++) {
    c23_y[c23_i] = c23_uv[c23_i];
  }

  sf_mex_destroy(&c23_u);
}

static c23_SL_Bus_ROSVariableLengthArrayInfo c23_d_emlrt_marshallIn
  (SFc23_PD_rbdlInstanceStruct *chartInstance, const mxArray *c23_u, const
   emlrtMsgIdentifier *c23_parentId)
{
  static const char_T *c23_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c23_SL_Bus_ROSVariableLengthArrayInfo c23_y;
  emlrtMsgIdentifier c23_thisId;
  c23_thisId.fParent = c23_parentId;
  c23_thisId.bParentIsCell = false;
  sf_mex_check_struct(c23_parentId, c23_u, 2, c23_fieldNames, 0U, NULL);
  c23_thisId.fIdentifier = "CurrentLength";
  c23_y.CurrentLength = c23_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "CurrentLength", "CurrentLength", 0)), &c23_thisId);
  c23_thisId.fIdentifier = "ReceivedLength";
  c23_y.ReceivedLength = c23_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "ReceivedLength", "ReceivedLength", 0)), &c23_thisId);
  sf_mex_destroy(&c23_u);
  return c23_y;
}

static uint32_T c23_e_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId)
{
  uint32_T c23_b_u;
  uint32_T c23_y;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), &c23_b_u, 1, 7, 0U, 0, 0U, 0);
  c23_y = c23_b_u;
  sf_mex_destroy(&c23_u);
  return c23_y;
}

static void c23_f_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId, real_T c23_y[6])
{
  real_T c23_dv[6];
  int32_T c23_i;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), c23_dv, 1, 0, 0U, 1, 0U, 1, 6);
  for (c23_i = 0; c23_i < 6; c23_i++) {
    c23_y[c23_i] = c23_dv[c23_i];
  }

  sf_mex_destroy(&c23_u);
}

static uint8_T c23_g_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_b_is_active_c23_PD_rbdl, const char_T *c23_identifier)
{
  emlrtMsgIdentifier c23_thisId;
  uint8_T c23_y;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_y = c23_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c23_b_is_active_c23_PD_rbdl), &c23_thisId);
  sf_mex_destroy(&c23_b_is_active_c23_PD_rbdl);
  return c23_y;
}

static uint8_T c23_h_emlrt_marshallIn(SFc23_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId)
{
  uint8_T c23_b_u;
  uint8_T c23_y;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), &c23_b_u, 1, 3, 0U, 0, 0U, 0);
  c23_y = c23_b_u;
  sf_mex_destroy(&c23_u);
  return c23_y;
}

static void c23_eML_blk_kernel(SFc23_PD_rbdlInstanceStruct *chartInstance,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg)
{
  static uint8_T c23_uv[5] = { 104U, 117U, 109U, 97U, 110U };

  hipMemcpy(chartInstance->c23_gpu_blankMsg, c23_b_blankMsg, 184UL,
             hipMemcpyHostToDevice);
  c23_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c23_gpu_blankMsg);
  hipMemcpy(*chartInstance->c23_gpu_uv, c23_uv, 5UL, hipMemcpyHostToDevice);
  c23_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c23_gpu_uv, chartInstance->c23_gpu_blankMsg);
  c23_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c23_gpu_blankMsg);
  hipMemcpy(c23_b_blankMsg, chartInstance->c23_gpu_blankMsg, 184UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_PD_rbdl_kernel1
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg,
   c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg)
{
  uint64_T c23_threadId;
  int32_T c23_i;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i = (int32_T)c23_threadId;
  if (c23_i < 8) {
    c23_b_msg->ModelName[c23_i] = c23_b_blankMsg->ModelName[c23_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_PD_rbdl_kernel2
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg,
   c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg)
{
  uint64_T c23_threadId;
  int32_T c23_i1;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i1 = (int32_T)c23_threadId;
  if (c23_i1 < 6) {
    c23_b_msg->Q[c23_i1] = c23_b_blankMsg->Q[c23_i1];
    c23_b_msg->Qd[c23_i1] = c23_b_blankMsg->Qd[c23_i1];
    c23_b_msg->Tau[c23_i1] = c23_b_blankMsg->Tau[c23_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_PD_rbdl_kernel3
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg,
   c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_r)
{
  uint64_T c23_threadId;
  int32_T c23_i2;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i2 = (int32_T)c23_threadId;
  if (c23_i2 < 8) {
    c23_r->ModelName[c23_i2] = c23_b_msg->ModelName[c23_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_PD_rbdl_kernel4
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg,
   c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_r)
{
  uint64_T c23_threadId;
  int32_T c23_i3;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i3 = (int32_T)c23_threadId;
  if (c23_i3 < 6) {
    c23_r->Q[c23_i3] = c23_b_msg->Q[c23_i3];
    c23_r->Qd[c23_i3] = c23_b_msg->Qd[c23_i3];
    c23_r->Tau[c23_i3] = c23_b_msg->Tau[c23_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_PD_rbdl_kernel5(const
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_r,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg)
{
  uint64_T c23_threadId;
  int32_T c23_i4;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i4 = (int32_T)c23_threadId;
  if (c23_i4 < 8) {
    c23_b_msg->ModelName[c23_i4] = c23_r->ModelName[c23_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_PD_rbdl_kernel6(const
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_r,
  c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_msg)
{
  uint64_T c23_threadId;
  int32_T c23_i5;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i5 = (int32_T)c23_threadId;
  if (c23_i5 < 6) {
    c23_b_msg->Q[c23_i5] = c23_r->Q[c23_i5];
    c23_b_msg->Qd[c23_i5] = c23_r->Qd[c23_i5];
    c23_b_msg->Tau[c23_i5] = c23_r->Tau[c23_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c23_eML_blk_kernel_kernel7
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg)
{
  uint64_T c23_threadId;
  int32_T c23_tmpIdx;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_tmpIdx = (int32_T)c23_threadId;
  if (c23_tmpIdx < 1) {
    c23_b_blankMsg->ModelName_SL_Info.CurrentLength = 5U;
  }
}

static __global__ __launch_bounds__(32, 1) void c23_eML_blk_kernel_kernel8(const
  uint8_T c23_uv[5], c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest
  *c23_b_blankMsg)
{
  uint64_T c23_threadId;
  int32_T c23_i;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i = (int32_T)c23_threadId;
  if (c23_i < 5) {
    c23_b_blankMsg->ModelName[c23_i] = c23_uv[c23_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c23_eML_blk_kernel_kernel9
  (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *c23_b_blankMsg)
{
  uint64_T c23_threadId;
  int32_T c23_tmpIdx;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_tmpIdx = (int32_T)c23_threadId;
  if (c23_tmpIdx < 1) {
    c23_b_blankMsg->Q_SL_Info.CurrentLength = 6U;
    c23_b_blankMsg->Qd_SL_Info.CurrentLength = 6U;
    c23_b_blankMsg->Tau_SL_Info.CurrentLength = 6U;
  }
}

static void init_dsm_address_info(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc23_PD_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c23_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c23_blankMsg =
    (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c23_msg =
    (c23_SL_Bus_PD_rbdl_rbdl_server_RBDLForwardDynamicsRequest *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c23_PD_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1908773061U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4252016980U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(525113536U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2993307740U);
}

mxArray *sf_c23_PD_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c23_PD_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c23_PD_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c23_PD_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiD+wMjAwAakOYCYiQECWKF8RqgYI1ycBS6uAMQllQWpIPHiomTPFCCdl5gL5ie"
    "WVnjmpeWDzbdgQJjPhsV8RiTzOaHiEPDBnjL9Ig4g/QZI+lmw6GdG0i8A5OUWp4PdDQsf8u1XcK"
    "BMP8R+DwLuF0VxP4SfWRyfmFySWZYan2xkHB/gEl+UlJIDNQ8EAAEIGEg="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c23_PD_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "scYvH40EpYswOoaWKJIjtgC";
}

static void sf_opaque_initialize_c23_PD_rbdl(void *chartInstanceVar)
{
  initialize_params_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c23_PD_rbdl(void *chartInstanceVar)
{
  enable_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c23_PD_rbdl(void *chartInstanceVar)
{
  disable_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c23_PD_rbdl(void *chartInstanceVar)
{
  sf_gateway_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c23_PD_rbdl(SimStruct* S)
{
  return get_sim_state_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c23_PD_rbdl(SimStruct* S, const mxArray *st)
{
  set_sim_state_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c23_PD_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_PD_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*)
      chartInstanceVar);
    ((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc23_PD_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c23_PD_rbdl(void *chartInstanceVar)
{
  mdl_start_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c23_PD_rbdl(void *chartInstanceVar)
{
  mdl_terminate_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c23_PD_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc23_PD_rbdl((SFc23_PD_rbdlInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c23_PD_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQphQnaIDEMIoaObRFckuOQeNDgAKNbUpC1NqxEdpJczLWy5G40XKX3R/KPvScU96",
    "hb9Br732JvEV8yyWHzFK0LFMkZVuw4RQlQNNLfvPt7Hwzsyuv0d308FrE+/G3nncLn9/g3fRG18",
    "183Ji4R+8XvIf5+D0aCRtvE0Vi7dVegsTwErTk1jApuqInS2FM9ECBoIhNpDJVbJrFljMx6FhBH",
    "Z9+HTEaBZG0PFxHWxJuCX6IbIk128jTYgqo6QCEJlLS9qMOJ/2xx8oM/QjoQNu4bgkaTGAT55be",
    "tNywhEP7AGhXaEPQY33iW2CIAd8cVC7TrVQHx0AZJ5wRUbraiOgAEgywgd0kxL9b1uCiijAaEWX",
    "WISIp6A02yDilgCIn0/hhnwlipGKEt2PuO8Np37Y5+rMpQ+A1AUHf1hWQQSKZMNX6Bx1caVuQfQ",
    "4t2Lf9arYA/rBO/FcMhqAq49bzZQqK9GFLVE6aBaR9kKk1zpJpmGExvCJqjaJ+GsLK7MXM0QFBn",
    "WAHLapgkC2yq3cUSzG8lWw27rrMnFUyNh6JrWfBMrZ2CnUqjNk6VPiEc10J25HJBqTAM9YWMaQe",
    "NmItx2nNwh2JAXbpXV0NVjAUPof5UoSsVK60AMj6zgtsLKeR1GojYx+Tt7WxMf15GtYVBlSPUCj",
    "rAoowDRizLLzVbCHTTnsEolcmc68MPMqQWShP96xoDaUaYExqmsjJEpyilcBY91FLrIRdjUVTB3",
    "NazsJRQiMIXYNhHDaxbBBbEhPtWtsa1l3KzGELNFUsKVHVYtVhG2q7hDpMYFcMhByKjpJxkPf4U",
    "XgBMNuIEkz017F/qcMOzl4mRbafeSf72Z0z7GfHdsXnowmeRgmPN/Esznu7WT9vE/9r5HarE3Z3",
    "C/MsFOwcbgnv1u+f//3447ufv09X/vwE//w9z/x/Nc+3/y/m4x+OG+24cNKpfHXY5xN+LZTw35v",
    "gX8rHmr5Jn688bidv9HBLkte//dp9a/p+xnfUqPe3WfD3+P0D1/Exy7L8VLQb5gcTNyZ2tF07/q",
    "cT/t6aEY/b+fvRdfRsPvvvVos6lsXrxql43XCVeypvLz7/g9X57M+m93JB7+VsX98jrlvAHv3py",
    "d52a0/th7xQjxfN7/PaeVds97X4+b/d5et+lv3qqu28K7abd33n3bevG76u/3sF/NI1Xse856nL",
    "xn/wznfuuZ+Pfxn/tPEjxsOSU27+GY+svbKv/4E8/QL/sq+P",
    ""
  };

  static char newstr [1249] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c23_PD_rbdl(SimStruct *S)
{
  const char* newstr = sf_c23_PD_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3288094788U));
  ssSetChecksum1(S,(998710767U));
  ssSetChecksum2(S,(2100590107U));
  ssSetChecksum3(S,(2948097527U));
}

static void mdlRTW_c23_PD_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c23_PD_rbdl(SimStruct *S)
{
  SFc23_PD_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc23_PD_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc23_PD_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc23_PD_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc23_PD_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c23_PD_rbdl;
  chartInstance->chartInfo.initializeChart = sf_opaque_initialize_c23_PD_rbdl;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c23_PD_rbdl;
  chartInstance->chartInfo.mdlTerminate = sf_opaque_mdl_terminate_c23_PD_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c23_PD_rbdl;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c23_PD_rbdl;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c23_PD_rbdl;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c23_PD_rbdl;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c23_PD_rbdl;
  chartInstance->chartInfo.getSimStateInfo = sf_get_sim_state_info_c23_PD_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c23_PD_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c23_PD_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c23_JITStateAnimation,
    chartInstance->c23_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c23_PD_rbdl(chartInstance);
}

void c23_PD_rbdl_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c23_PD_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c23_PD_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c23_PD_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c23_PD_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
