/* Include files */

#include "PD_rbdl_sfun.h"
#include "c25_PD_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance);
static void initialize_params_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct
  *chartInstance);
static void enable_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance);
static void disable_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance);
static void c25_do_animation_call_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct
  *chartInstance);
static void ext_mode_exec_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct
  *chartInstance);
static void set_sim_state_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_st);
static void sf_gateway_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance);
static void mdl_start_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *
  chartInstance);
static void mdl_cleanup_runtime_resources_c25_PD_rbdl
  (SFc25_PD_rbdlInstanceStruct *chartInstance);
static void initSimStructsc25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance);
static void c25_eML_blk_kernel(SFc25_PD_rbdlInstanceStruct *chartInstance,
  real_T c25_b_u[6], real_T c25_b_y[6]);
static void c25_emlrt_marshallIn(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_b_y, const char_T *c25_identifier, real_T c25_c_y[6]);
static void c25_b_emlrt_marshallIn(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_b_u, const emlrtMsgIdentifier *c25_parentId, real_T
  c25_b_y[6]);
static uint8_T c25_c_emlrt_marshallIn(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_b_is_active_c25_PD_rbdl, const char_T *c25_identifier);
static uint8_T c25_d_emlrt_marshallIn(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_b_u, const emlrtMsgIdentifier *c25_parentId);
static void init_dsm_address_info(SFc25_PD_rbdlInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc25_PD_rbdlInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c25_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c25_is_active_c25_PD_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct
  *chartInstance)
{
}

static void enable_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c25_do_animation_call_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct
  *chartInstance)
{
  const mxArray *c25_b_y = NULL;
  const mxArray *c25_c_y = NULL;
  const mxArray *c25_d_y = NULL;
  const mxArray *c25_st;
  c25_st = NULL;
  c25_b_y = NULL;
  sf_mex_assign(&c25_b_y, sf_mex_createcellmatrix(2, 1), false);
  c25_c_y = NULL;
  sf_mex_assign(&c25_c_y, sf_mex_create("y", *chartInstance->c25_y, 0, 0U, 1U,
    0U, 2, 1, 6), false);
  sf_mex_setcell(c25_b_y, 0, c25_c_y);
  c25_d_y = NULL;
  sf_mex_assign(&c25_d_y, sf_mex_create("y",
    &chartInstance->c25_is_active_c25_PD_rbdl, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c25_b_y, 1, c25_d_y);
  sf_mex_assign(&c25_st, c25_b_y, false);
  return c25_st;
}

static void set_sim_state_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_st)
{
  const mxArray *c25_b_u;
  c25_b_u = sf_mex_dup(c25_st);
  c25_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c25_b_u, 0)),
                       "y", *chartInstance->c25_y);
  chartInstance->c25_is_active_c25_PD_rbdl = c25_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c25_b_u, 1)),
     "is_active_c25_PD_rbdl");
  sf_mex_destroy(&c25_b_u);
  sf_mex_destroy(&c25_st);
}

static void sf_gateway_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c25_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c25_eML_blk_kernel(chartInstance, *chartInstance->c25_u, *chartInstance->c25_y);
  c25_do_animation_call_c25_PD_rbdl(chartInstance);
}

static void mdl_start_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
  hipError_t c25_errCode;
  c25_errCode = hipGetLastError();
  if (c25_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c25_errCode, (char_T *)hipGetErrorName
                       (c25_errCode), (char_T *)hipGetErrorString(c25_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c25_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *
  chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c25_PD_rbdl
  (SFc25_PD_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc25_PD_rbdl(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c25_PD_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c25_nameCaptureInfo = NULL;
  c25_nameCaptureInfo = NULL;
  sf_mex_assign(&c25_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c25_nameCaptureInfo;
}

static void c25_eML_blk_kernel(SFc25_PD_rbdlInstanceStruct *chartInstance,
  real_T c25_b_u[6], real_T c25_b_y[6])
{
  c25_b_y[0] = c25_b_u[3];
  c25_b_y[1] = c25_b_u[0];
  c25_b_y[2] = c25_b_u[1];
  c25_b_y[3] = c25_b_u[2];
  c25_b_y[4] = c25_b_u[4];
  c25_b_y[5] = c25_b_u[5];
}

static void c25_emlrt_marshallIn(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_b_y, const char_T *c25_identifier, real_T c25_c_y[6])
{
  emlrtMsgIdentifier c25_thisId;
  c25_thisId.fIdentifier = const_cast<const char_T *>(c25_identifier);
  c25_thisId.fParent = NULL;
  c25_thisId.bParentIsCell = false;
  c25_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c25_b_y), &c25_thisId,
    c25_c_y);
  sf_mex_destroy(&c25_b_y);
}

static void c25_b_emlrt_marshallIn(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_b_u, const emlrtMsgIdentifier *c25_parentId, real_T
  c25_b_y[6])
{
  real_T c25_dv[6];
  int32_T c25_i;
  sf_mex_import(c25_parentId, sf_mex_dup(c25_b_u), c25_dv, 1, 0, 0U, 1, 0U, 2, 1,
                6);
  for (c25_i = 0; c25_i < 6; c25_i++) {
    c25_b_y[c25_i] = c25_dv[c25_i];
  }

  sf_mex_destroy(&c25_b_u);
}

static uint8_T c25_c_emlrt_marshallIn(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_b_is_active_c25_PD_rbdl, const char_T *c25_identifier)
{
  emlrtMsgIdentifier c25_thisId;
  uint8_T c25_b_y;
  c25_thisId.fIdentifier = const_cast<const char_T *>(c25_identifier);
  c25_thisId.fParent = NULL;
  c25_thisId.bParentIsCell = false;
  c25_b_y = c25_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c25_b_is_active_c25_PD_rbdl), &c25_thisId);
  sf_mex_destroy(&c25_b_is_active_c25_PD_rbdl);
  return c25_b_y;
}

static uint8_T c25_d_emlrt_marshallIn(SFc25_PD_rbdlInstanceStruct *chartInstance,
  const mxArray *c25_b_u, const emlrtMsgIdentifier *c25_parentId)
{
  uint8_T c25_b_y;
  uint8_T c25_c_u;
  sf_mex_import(c25_parentId, sf_mex_dup(c25_b_u), &c25_c_u, 1, 3, 0U, 0, 0U, 0);
  c25_b_y = c25_c_u;
  sf_mex_destroy(&c25_b_u);
  return c25_b_y;
}

static void init_dsm_address_info(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc25_PD_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c25_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c25_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c25_y = (real_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c25_PD_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(355923390U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3254592024U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(515668378U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(305817613U);
}

mxArray *sf_c25_PD_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c25_PD_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c25_PD_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c25_PD_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiD+wMjAwAakOYCYiQECWKF8RqgYI1ycBS6uAMQllQWpIPHiomTPFCCdl5gL5ie"
    "WVnjmpeWDzbdgQJjPhsV8RiTzOaHiEPDBnjL9Ig4g/QZI+lkI6BcAsiqh4QILH/LtV3CgTD/Efg"
    "8C7hdFcT+En1kcn5hcklmWGp9sZBof4BJflJSSg+QvAEc2F3g="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c25_PD_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sDt8ga6s3SmK65qEWTBZYi";
}

static void sf_opaque_initialize_c25_PD_rbdl(void *chartInstanceVar)
{
  initialize_params_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c25_PD_rbdl(void *chartInstanceVar)
{
  enable_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c25_PD_rbdl(void *chartInstanceVar)
{
  disable_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c25_PD_rbdl(void *chartInstanceVar)
{
  sf_gateway_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c25_PD_rbdl(SimStruct* S)
{
  return get_sim_state_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c25_PD_rbdl(SimStruct* S, const mxArray *st)
{
  set_sim_state_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c25_PD_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_PD_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*)
      chartInstanceVar);
    ((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc25_PD_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c25_PD_rbdl(void *chartInstanceVar)
{
  mdl_start_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c25_PD_rbdl(void *chartInstanceVar)
{
  mdl_terminate_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c25_PD_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc25_PD_rbdl((SFc25_PD_rbdlInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c25_PD_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV11u00AQdqKCqKClQrRPSJQn4A2BQDxBmz8R0dAIpyB4qbb2JF5lvevuT9ocgjtwAx65BAf",
    "gGD0Cs7abBsd2WiIqQFhynbW/+XZmvtnZrVNpdxy8VvF+dMtxruLzGt5VJ7mupOPK1J28X3Lup+",
    "NPaMRN2CWShMopvTgJ4S0owYymgrd5X+TCKO+DBO4hNhJSF7EpGhpG+bBluGf51PuAeoEbCMP8G",
    "toSf5ezMbJFRneRp0EleLoF4OtACjMIWowMJh5LfVQPwBsqE5aFoEC7JrJuqY5hmkYMmsfgtbnS",
    "BD1WZ765mmio6+PCMG2kyj0FijBilPDcaAOiXIgwwRr2Ih//7hqNQWVhXkCkrkFARqB26DDmFBy",
    "ynFThhwPKiRaSEtYMWd0azvrWZehPR/jAShKCvtUkkGEkKNfF+rstjLTJyQGDBhyYQTGbC4fGiv",
    "+OwhHIwrz162IEkgxglxdOGiekeRyrNamSWZimIbwjcttD/RT4hdWLlaNcgjpBDy2KYBAH2VY9S",
    "UeY3kI2E7ZtZc5bMiZMxFbzYDFbcwRlKkzYWh6vE8ZUIawnoh0YAYtZG0STcljCmo9Tivo9gQm2",
    "5V28GgynKHwKqwvu01y5RhlA3HfeYGP5GekZpUVYx+Jt7OzMfp6FtbkG2Sce5HUBSagCzFmc3mI",
    "2nyqrPQLRKx27lwdOKmQeylF9wxtHQg4xJyVN5CwEq2ghMFQD1BJXwp7CRVMGs1rOw3nEC8C3DY",
    "Yy6OCyQWxOTpRtbdu47kZUjxugPEmjHFUNrjpsQ01bUOMI9viQiyPekiJ00x6fpBcAq41ITvmgh",
    "v1Ljls4e54U8X7mnO1nN86xn53aZZ8PpngqOTzO1DM773K1fN4q/qqkdltTdiuZeZYydha3hve9",
    "r50v3ZvXNx6yleXOxq1vi8z/uXqx/X81Hd85bbSThTOaqVeLfTXl11IO/8YU/1o6Vg39fECeqSd",
    "u+PrZ08Pm+17t4wea8J1Uyv2tZvw9fb9pOz5WWVyf0mv76cHEjolJtmvL/3zK36tz8rGcvk+uk5",
    "eL2d/eyuq4NMd+DX+NM3X76/Nvbi1mfz691zN6r8f7+j6x3QL2vcdP97uNfXngs0xcv1rfF7VzL",
    "tnub/Hzv93v1/08+9Vl2zmXbLdofBfdt/80fFn/dzL4tT84jkXPU78b/9252Lnnbjp+MfnXph5Q",
    "5uecctPPeGTt5339B+r0BzTBqeY=",
    ""
  };

  static char newstr [1229] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c25_PD_rbdl(SimStruct *S)
{
  const char* newstr = sf_c25_PD_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2941105953U));
  ssSetChecksum1(S,(369823568U));
  ssSetChecksum2(S,(151874601U));
  ssSetChecksum3(S,(3423737421U));
}

static void mdlRTW_c25_PD_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c25_PD_rbdl(SimStruct *S)
{
  SFc25_PD_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc25_PD_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc25_PD_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc25_PD_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc25_PD_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c25_PD_rbdl;
  chartInstance->chartInfo.initializeChart = sf_opaque_initialize_c25_PD_rbdl;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c25_PD_rbdl;
  chartInstance->chartInfo.mdlTerminate = sf_opaque_mdl_terminate_c25_PD_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c25_PD_rbdl;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c25_PD_rbdl;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c25_PD_rbdl;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c25_PD_rbdl;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c25_PD_rbdl;
  chartInstance->chartInfo.getSimStateInfo = sf_get_sim_state_info_c25_PD_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c25_PD_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c25_PD_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c25_JITStateAnimation,
    chartInstance->c25_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c25_PD_rbdl(chartInstance);
}

void c25_PD_rbdl_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c25_PD_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c25_PD_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c25_PD_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c25_PD_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
