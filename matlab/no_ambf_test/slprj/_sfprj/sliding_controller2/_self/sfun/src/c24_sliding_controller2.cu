#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c24_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void enable_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void disable_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void c24_do_animation_call_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance, const mxArray *c24_st);
static void sf_gateway_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void c24_eML_blk_kernel(SFc24_sliding_controller2InstanceStruct
  *chartInstance, real_T c24_b_u[6], real_T c24_b_y[7]);
static void c24_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_y, const char_T *c24_identifier, real_T
  c24_c_y[7]);
static void c24_b_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId,
  real_T c24_b_y[7]);
static uint8_T c24_c_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_is_active_c24_sliding_controller2, const
  char_T *c24_identifier);
static uint8_T c24_d_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId);
static __global__ void c24_eML_blk_kernel_kernel1(real_T c24_b_y[7]);
static void init_dsm_address_info(SFc24_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc24_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c24_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c24_is_active_c24_sliding_controller2 = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c24_gpu_y, 56UL);
}

static void initialize_params_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c24_do_animation_call_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c24_b_y = NULL;
  const mxArray *c24_c_y = NULL;
  const mxArray *c24_d_y = NULL;
  const mxArray *c24_st;
  c24_st = NULL;
  c24_b_y = NULL;
  sf_mex_assign(&c24_b_y, sf_mex_createcellmatrix(2, 1), false);
  c24_c_y = NULL;
  sf_mex_assign(&c24_c_y, sf_mex_create("y", *chartInstance->c24_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c24_b_y, 0, c24_c_y);
  c24_d_y = NULL;
  sf_mex_assign(&c24_d_y, sf_mex_create("y",
    &chartInstance->c24_is_active_c24_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c24_b_y, 1, c24_d_y);
  sf_mex_assign(&c24_st, c24_b_y, false);
  return c24_st;
}

static void set_sim_state_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance, const mxArray *c24_st)
{
  const mxArray *c24_b_u;
  c24_b_u = sf_mex_dup(c24_st);
  c24_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c24_b_u, 0)),
                       "y", *chartInstance->c24_y);
  chartInstance->c24_is_active_c24_sliding_controller2 = c24_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c24_b_u, 1)),
     "is_active_c24_sliding_controller2");
  sf_mex_destroy(&c24_b_u);
  sf_mex_destroy(&c24_st);
}

static void sf_gateway_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  real_T c24_dv1[7];
  real_T c24_dv[6];
  int32_T c24_i;
  int32_T c24_i1;
  chartInstance->c24_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c24_i = 0; c24_i < 6; c24_i++) {
    c24_dv[c24_i] = (*chartInstance->c24_u)[c24_i];
  }

  c24_eML_blk_kernel(chartInstance, c24_dv, c24_dv1);
  for (c24_i1 = 0; c24_i1 < 7; c24_i1++) {
    (*chartInstance->c24_y)[c24_i1] = c24_dv1[c24_i1];
  }

  c24_do_animation_call_c24_sliding_controller2(chartInstance);
}

static void mdl_start_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c24_errCode;
  hipFree(*chartInstance->c24_gpu_y);
  c24_errCode = hipGetLastError();
  if (c24_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c24_errCode, (char_T *)hipGetErrorName
                       (c24_errCode), (char_T *)hipGetErrorString(c24_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c24_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c24_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c24_nameCaptureInfo = NULL;
  c24_nameCaptureInfo = NULL;
  sf_mex_assign(&c24_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c24_nameCaptureInfo;
}

static void c24_eML_blk_kernel(SFc24_sliding_controller2InstanceStruct
  *chartInstance, real_T c24_b_u[6], real_T c24_b_y[7])
{
  const mxArray *c24_c_y = NULL;
  c24_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c24_gpu_y);
  hipMemcpy(c24_b_y, *chartInstance->c24_gpu_y, 56UL, hipMemcpyDeviceToHost);
  c24_b_y[0] = c24_b_u[3];
  c24_b_y[1] = 0.0;
  sf_mex_printf("%s =\\n", "reordered");
  c24_c_y = NULL;
  sf_mex_assign(&c24_c_y, sf_mex_create("y", c24_b_y, 0, 0U, 1U, 0U, 2, 1, 7),
                false);
  sf_mex_call(chartInstance->c24_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c24_c_y);
  c24_b_y[2] = c24_b_u[0];
  c24_b_y[3] = c24_b_u[1];
  c24_b_y[4] = c24_b_u[4];
  c24_b_y[5] = c24_b_u[5];
  c24_b_y[6] = c24_b_u[2];
}

static void c24_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_y, const char_T *c24_identifier, real_T
  c24_c_y[7])
{
  emlrtMsgIdentifier c24_thisId;
  c24_thisId.fIdentifier = const_cast<const char_T *>(c24_identifier);
  c24_thisId.fParent = NULL;
  c24_thisId.bParentIsCell = false;
  c24_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c24_b_y), &c24_thisId,
    c24_c_y);
  sf_mex_destroy(&c24_b_y);
}

static void c24_b_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId,
  real_T c24_b_y[7])
{
  real_T c24_dv[7];
  int32_T c24_i;
  sf_mex_import(c24_parentId, sf_mex_dup(c24_b_u), c24_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c24_i = 0; c24_i < 7; c24_i++) {
    c24_b_y[c24_i] = c24_dv[c24_i];
  }

  sf_mex_destroy(&c24_b_u);
}

static uint8_T c24_c_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_is_active_c24_sliding_controller2, const
  char_T *c24_identifier)
{
  emlrtMsgIdentifier c24_thisId;
  uint8_T c24_b_y;
  c24_thisId.fIdentifier = const_cast<const char_T *>(c24_identifier);
  c24_thisId.fParent = NULL;
  c24_thisId.bParentIsCell = false;
  c24_b_y = c24_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c24_b_is_active_c24_sliding_controller2), &c24_thisId);
  sf_mex_destroy(&c24_b_is_active_c24_sliding_controller2);
  return c24_b_y;
}

static uint8_T c24_d_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId)
{
  uint8_T c24_b_y;
  uint8_T c24_c_u;
  sf_mex_import(c24_parentId, sf_mex_dup(c24_b_u), &c24_c_u, 1, 3, 0U, 0, 0U, 0);
  c24_b_y = c24_c_u;
  sf_mex_destroy(&c24_b_u);
  return c24_b_y;
}

static __global__ __launch_bounds__(32, 1) void c24_eML_blk_kernel_kernel1
  (real_T c24_b_y[7])
{
  uint64_T c24_threadId;
  int32_T c24_i;
  c24_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c24_i = (int32_T)c24_threadId;
  if (c24_i < 7) {
    c24_b_y[c24_i] = 0.0;
  }
}

static void init_dsm_address_info(SFc24_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc24_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c24_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c24_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c24_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c24_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1104270009U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4002144469U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1451542329U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1457894201U);
}

mxArray *sf_c24_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c24_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c24_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c24_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8clGJvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABbZhvY"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c24_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "scU9zZ5H1kMhWDkDI28BpgD";
}

static void sf_opaque_initialize_c24_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c24_sliding_controller2
    ((SFc24_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c24_sliding_controller2(void *chartInstanceVar)
{
  enable_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c24_sliding_controller2(void *chartInstanceVar)
{
  disable_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c24_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c24_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c24_sliding_controller2
    ((SFc24_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c24_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c24_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc24_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c24_sliding_controller2
      ((SFc24_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc24_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc24_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c24_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c24_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c24_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c24_sliding_controller2
      ((SFc24_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc24_sliding_controller2
      ((SFc24_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c24_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV09v00gUd6OCQAJUrRZxQQJO7BEqWMEFSuJERGpotW4Bcamm45d4lPGMmT9pw1dAXFZc9sB",
    "ngM+AxHfgst9hbyvtiTe2kwbHdigRFYuw5Dpj/96b997v/Zl6K92eh9cFvDd+8bzT+DyDd8PLrl",
    "P5emXmzt6vetfz9SsUEjbeJorE2qu9BInhD9CSW8Ok6Iq+LIUx0QcFgiI2kcpUadMstpyJYccK6",
    "vTpJxGjURBJy8MmypJwS/Axakus2UY9PlNATQcgNJGSdhB1OBlMLVbmoBUBHWob17mgwQQ2cWbp",
    "nuWGJRzah0C7QhuCFusj2wJDDLTMYaWbzlMdTIAyTjgjotTbiOgAEgywgd0kxL9b1qBTRRiNiDJ",
    "NiMgI9CYbpjqlgKJOpvHDPhPESMUIb8e85QTnbdvmaE9PhsBrAoK2NRWQYSKZMNX8Bx30tC3IPg",
    "cf9u2gWlsAz60j/zGDA1CVceu35AgUGcCWqNw0DUj7MGVrmiXzMMNieEzUA4r8aQgrsxczRwcEe",
    "YIdlKiCQepkV+8oNsLwVmqzcddl5qKSsXFGtl4ES7W1R1DHwlRbh4oW4VxXwnZksgkj4KlWnxhS",
    "D8u0luO0ZuGOxAC79K6uBisYEp/DWlKErJSuUQGQ9p1H2Fg+R1KrjYxbmLz+5ub853lYVxhQfUK",
    "hrAsowjRgzNLwVmsLmXbcIxCtMql5ZeAsQxahPN23wj+QaogxqWkiRy44RiuBsR4gl1gJuxqLpg",
    "7muFyEo4RGELoGwzj0sGwQWxIT7VrbA6y7ETNjHzRVLClh1WLVYRtqu4QaJ7ArhkIeiI6ScZD3+",
    "Cy8AJhtRAkmBk3sX2rcwd3LqHDz7IZ3NM/OfcE8m8gVn7/N6Fkp0ePNPIv7nm3U79vAXyuT+Tsj",
    "d76wz2pBzuHW8L7438vX/75+8/bDn7/f/vuvD++X2f9d43jz/0K+vjxptNPCGc3lq8M+nLFrtUT",
    "/pRn9a/la0927L57dfnhz2Iue+EO/u36nmQz8jJ8F9jYK9k7eX3UdH7MszU9Fu2F+MHFrYrNx7f",
    "TfmbH39IJ4nM3fZ9c/95eT/3WjyOPqAvk1/DUu5O3X7391Yzn5bP+nC+y/VuD7WjrX94jrFrBH1",
    "2/tac5CLPU9KoVRknNQ6yX1+bX5flw574Tl/i92/pT79rx/yfw6aTnvhOWW9e+4c/x7w9fNA6+A",
    "X/uO/Vj2fPWt8R+9452DruTre9N/dVoR42HJqTf/jEfYftnXHyBPPwFtPrQQ",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c24_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c24_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2492201493U));
  ssSetChecksum1(S,(2946340087U));
  ssSetChecksum2(S,(892768965U));
  ssSetChecksum3(S,(3267729885U));
}

static void mdlRTW_c24_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c24_sliding_controller2(SimStruct *S)
{
  SFc24_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc24_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc24_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc24_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc24_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c24_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c24_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c24_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c24_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c24_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c24_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c24_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c24_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c24_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c24_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c24_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c24_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c24_JITStateAnimation,
    chartInstance->c24_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c24_sliding_controller2(chartInstance);
}

void c24_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c24_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c24_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c24_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c24_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
