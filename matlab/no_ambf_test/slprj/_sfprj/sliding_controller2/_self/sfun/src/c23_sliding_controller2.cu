#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c23_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void enable_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void disable_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void c23_do_animation_call_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance, const mxArray *c23_st);
static void sf_gateway_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void c23_eML_blk_kernel(SFc23_sliding_controller2InstanceStruct
  *chartInstance, real_T c23_b_u[6], real_T c23_b_y[7]);
static void c23_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_y, const char_T *c23_identifier, real_T
  c23_c_y[7]);
static void c23_b_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_u, const emlrtMsgIdentifier *c23_parentId,
  real_T c23_b_y[7]);
static uint8_T c23_c_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_is_active_c23_sliding_controller2, const
  char_T *c23_identifier);
static uint8_T c23_d_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_u, const emlrtMsgIdentifier *c23_parentId);
static __global__ void c23_eML_blk_kernel_kernel1(real_T c23_b_y[7]);
static void init_dsm_address_info(SFc23_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc23_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c23_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c23_is_active_c23_sliding_controller2 = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c23_gpu_y, 56UL);
}

static void initialize_params_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c23_do_animation_call_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c23_b_y = NULL;
  const mxArray *c23_c_y = NULL;
  const mxArray *c23_d_y = NULL;
  const mxArray *c23_st;
  c23_st = NULL;
  c23_b_y = NULL;
  sf_mex_assign(&c23_b_y, sf_mex_createcellmatrix(2, 1), false);
  c23_c_y = NULL;
  sf_mex_assign(&c23_c_y, sf_mex_create("y", *chartInstance->c23_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c23_b_y, 0, c23_c_y);
  c23_d_y = NULL;
  sf_mex_assign(&c23_d_y, sf_mex_create("y",
    &chartInstance->c23_is_active_c23_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c23_b_y, 1, c23_d_y);
  sf_mex_assign(&c23_st, c23_b_y, false);
  return c23_st;
}

static void set_sim_state_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance, const mxArray *c23_st)
{
  const mxArray *c23_b_u;
  c23_b_u = sf_mex_dup(c23_st);
  c23_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c23_b_u, 0)),
                       "y", *chartInstance->c23_y);
  chartInstance->c23_is_active_c23_sliding_controller2 = c23_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c23_b_u, 1)),
     "is_active_c23_sliding_controller2");
  sf_mex_destroy(&c23_b_u);
  sf_mex_destroy(&c23_st);
}

static void sf_gateway_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  real_T c23_dv1[7];
  real_T c23_dv[6];
  int32_T c23_i;
  int32_T c23_i1;
  chartInstance->c23_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c23_i = 0; c23_i < 6; c23_i++) {
    c23_dv[c23_i] = (*chartInstance->c23_u)[c23_i];
  }

  c23_eML_blk_kernel(chartInstance, c23_dv, c23_dv1);
  for (c23_i1 = 0; c23_i1 < 7; c23_i1++) {
    (*chartInstance->c23_y)[c23_i1] = c23_dv1[c23_i1];
  }

  c23_do_animation_call_c23_sliding_controller2(chartInstance);
}

static void mdl_start_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c23_errCode;
  hipFree(*chartInstance->c23_gpu_y);
  c23_errCode = hipGetLastError();
  if (c23_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c23_errCode, (char_T *)hipGetErrorName
                       (c23_errCode), (char_T *)hipGetErrorString(c23_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c23_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c23_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c23_nameCaptureInfo = NULL;
  c23_nameCaptureInfo = NULL;
  sf_mex_assign(&c23_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c23_nameCaptureInfo;
}

static void c23_eML_blk_kernel(SFc23_sliding_controller2InstanceStruct
  *chartInstance, real_T c23_b_u[6], real_T c23_b_y[7])
{
  const mxArray *c23_c_y = NULL;
  c23_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c23_gpu_y);
  hipMemcpy(c23_b_y, *chartInstance->c23_gpu_y, 56UL, hipMemcpyDeviceToHost);
  c23_b_y[0] = c23_b_u[3];
  c23_b_y[1] = 0.0;
  sf_mex_printf("%s =\\n", "reordered");
  c23_c_y = NULL;
  sf_mex_assign(&c23_c_y, sf_mex_create("y", c23_b_y, 0, 0U, 1U, 0U, 2, 1, 7),
                false);
  sf_mex_call(chartInstance->c23_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c23_c_y);
  c23_b_y[2] = c23_b_u[0];
  c23_b_y[3] = c23_b_u[1];
  c23_b_y[4] = c23_b_u[4];
  c23_b_y[5] = c23_b_u[5];
  c23_b_y[6] = c23_b_u[2];
}

static void c23_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_y, const char_T *c23_identifier, real_T
  c23_c_y[7])
{
  emlrtMsgIdentifier c23_thisId;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c23_b_y), &c23_thisId,
    c23_c_y);
  sf_mex_destroy(&c23_b_y);
}

static void c23_b_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_u, const emlrtMsgIdentifier *c23_parentId,
  real_T c23_b_y[7])
{
  real_T c23_dv[7];
  int32_T c23_i;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_b_u), c23_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c23_i = 0; c23_i < 7; c23_i++) {
    c23_b_y[c23_i] = c23_dv[c23_i];
  }

  sf_mex_destroy(&c23_b_u);
}

static uint8_T c23_c_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_is_active_c23_sliding_controller2, const
  char_T *c23_identifier)
{
  emlrtMsgIdentifier c23_thisId;
  uint8_T c23_b_y;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_b_y = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c23_b_is_active_c23_sliding_controller2), &c23_thisId);
  sf_mex_destroy(&c23_b_is_active_c23_sliding_controller2);
  return c23_b_y;
}

static uint8_T c23_d_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_u, const emlrtMsgIdentifier *c23_parentId)
{
  uint8_T c23_b_y;
  uint8_T c23_c_u;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_b_u), &c23_c_u, 1, 3, 0U, 0, 0U, 0);
  c23_b_y = c23_c_u;
  sf_mex_destroy(&c23_b_u);
  return c23_b_y;
}

static __global__ __launch_bounds__(32, 1) void c23_eML_blk_kernel_kernel1
  (real_T c23_b_y[7])
{
  uint64_T c23_threadId;
  int32_T c23_i;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i = (int32_T)c23_threadId;
  if (c23_i < 7) {
    c23_b_y[c23_i] = 0.0;
  }
}

static void init_dsm_address_info(SFc23_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc23_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c23_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c23_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c23_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c23_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1104270009U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4002144469U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1451542329U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1457894201U);
}

mxArray *sf_c23_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c23_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c23_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c23_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8clGxvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABbQhvX"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c23_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "scU9zZ5H1kMhWDkDI28BpgD";
}

static void sf_opaque_initialize_c23_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c23_sliding_controller2
    ((SFc23_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c23_sliding_controller2(void *chartInstanceVar)
{
  enable_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c23_sliding_controller2(void *chartInstanceVar)
{
  disable_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c23_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c23_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c23_sliding_controller2
    ((SFc23_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c23_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c23_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc23_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c23_sliding_controller2
      ((SFc23_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc23_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc23_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c23_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c23_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c23_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c23_sliding_controller2
      ((SFc23_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc23_sliding_controller2
      ((SFc23_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c23_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV09v00gUd6OCQAJUrRZxQQJO7BGKWMEFSuJERGpotW4Bcamm45d4lPGMmT9pw1dAXFZc9sB",
    "ngM+AxHfgst9hbyvtiTe2kwbHdigRFYuw5Dpj/96b997v/Zl6K92eh9cFvDd+8bzT+DyDd8PLrl",
    "P5emXmzt6vetfz9SsUEjbeJorE2qu9BInhD9CSW8Ok6Iq+LIUx0QcFgiI2kcpUadMstpyJYccK6",
    "vTpJxGjURBJy8MmypJwS/Axakus2UY9PlNATQcgNJGSdhB1OBlMLVbmoBUBHWob17mgwQQ2cWbp",
    "nuWGJRzah0C7QhuCFusj2wJDDLTMYaWbzlMdTIAyTjgjotTbiOgAEgywgd0kxL9b1qBTRRiNiDJ",
    "NiMgI9CYbpjqlgKJOpvHDPhPESMUIb8e85QTnbdvmaE9PhsBrAoK2NRWQYSKZMNX8Bx30tC3IPg",
    "cf9u2gWlsAz60j/zGDA1CVceu35AgUGcCWqNw0DUj7MGVrmiXzMMNieEzUA4r8aQgrsxczRwcEe",
    "YIdlKiCQepkV+8oNsLwVmqzcddl5qKSsXFGtl4ES7W1R1DHwlRbh4oW4VxXwnZksgkj4KlWnxhS",
    "D8u0luO0ZuGOxAC79K6uBisYEp/DWlKErJSuUQGQ9p1H2Fg+R1KrjYxbmLz+5ub853lYVxhQfUK",
    "hrAsowjRgzNLwVmsLmXbcIxCtMql5ZeAsQxahPN23wj+QaogxqWkiRy44RiuBsR4gl1gJuxqLpg",
    "7muFyEo4RGELoGwzj0sGwQWxIT7VrbA6y7ETNjHzRVLClh1WLVYRtqu4QaJ7ArhkIeiI6ScZD3+",
    "Cy8AJhtRAkmBk3sX2rcwd3LqHDz7IZ3NM/OfcE8m8gVn7/N6Fkp0ePNPIv7nm3U79vAXyuT+Tsj",
    "d76wz2pBzuHW8L7438vX/75+8/bDn7/f/vuvD++X2f9d43jz/0K+vjxptNPCGc3lq8M+nLFrtUT",
    "/pRn9a/la0927L57dfnhz2Iue+EO/u36nmQz8jJ8F9jYK9k7eX3UdH7MszU9Fu2F+MHFrYrNx7f",
    "TfmbH39IJ4nM3fZ9c/95eT/3WjyOPqAvk1/DUu5O3X7391Yzn5bP+nC+y/VuD7WjrX94jrFrBH1",
    "2/tac5CLPU9KoVRknNQ6yX1+bX5flw574Tl/i92/pT79rx/yfw6aTnvhOWW9e+4c/x7w9fNA6+A",
    "X/uO/Vj2fPWt8R+9452DruTre9N/dVoR42HJqTf/jEfYftnXHyBPPwFlirQP",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c23_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c23_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2492201493U));
  ssSetChecksum1(S,(2946340087U));
  ssSetChecksum2(S,(892768965U));
  ssSetChecksum3(S,(3267729885U));
}

static void mdlRTW_c23_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c23_sliding_controller2(SimStruct *S)
{
  SFc23_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc23_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc23_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc23_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc23_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c23_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c23_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c23_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c23_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c23_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c23_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c23_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c23_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c23_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c23_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c23_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c23_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c23_JITStateAnimation,
    chartInstance->c23_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c23_sliding_controller2(chartInstance);
}

void c23_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c23_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c23_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c23_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c23_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
