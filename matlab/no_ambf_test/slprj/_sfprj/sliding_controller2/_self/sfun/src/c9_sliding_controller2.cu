/* Include files */

#include "sliding_controller2_sfun.h"
#include "c9_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void enable_c9_sliding_controller2(SFc9_sliding_controller2InstanceStruct
  *chartInstance);
static void disable_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void c9_do_animation_call_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance, const mxArray *c9_st);
static void sf_gateway_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance);
static void c9_eML_blk_kernel(SFc9_sliding_controller2InstanceStruct
  *chartInstance, real_T c9_b_q[2], real_T c9_b_y[4]);
static void c9_emlrt_marshallIn(SFc9_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c9_b_y, const char_T *c9_identifier, real_T
  c9_c_y[4]);
static void c9_b_emlrt_marshallIn(SFc9_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId,
  real_T c9_b_y[4]);
static uint8_T c9_c_emlrt_marshallIn(SFc9_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c9_b_is_active_c9_sliding_controller2, const
  char_T *c9_identifier);
static uint8_T c9_d_emlrt_marshallIn(SFc9_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId);
static void init_dsm_address_info(SFc9_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc9_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c9_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c9_is_active_c9_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c9_sliding_controller2(SFc9_sliding_controller2InstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c9_do_animation_call_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c9_b_y = NULL;
  const mxArray *c9_c_y = NULL;
  const mxArray *c9_d_y = NULL;
  const mxArray *c9_st;
  c9_st = NULL;
  c9_b_y = NULL;
  sf_mex_assign(&c9_b_y, sf_mex_createcellmatrix(2, 1), false);
  c9_c_y = NULL;
  sf_mex_assign(&c9_c_y, sf_mex_create("y", *chartInstance->c9_y, 0, 0U, 1U, 0U,
    2, 2, 2), false);
  sf_mex_setcell(c9_b_y, 0, c9_c_y);
  c9_d_y = NULL;
  sf_mex_assign(&c9_d_y, sf_mex_create("y",
    &chartInstance->c9_is_active_c9_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c9_b_y, 1, c9_d_y);
  sf_mex_assign(&c9_st, c9_b_y, false);
  return c9_st;
}

static void set_sim_state_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance, const mxArray *c9_st)
{
  const mxArray *c9_u;
  c9_u = sf_mex_dup(c9_st);
  c9_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c9_u, 0)), "y",
                      *chartInstance->c9_y);
  chartInstance->c9_is_active_c9_sliding_controller2 = c9_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c9_u, 1)),
     "is_active_c9_sliding_controller2");
  sf_mex_destroy(&c9_u);
  sf_mex_destroy(&c9_st);
}

static void sf_gateway_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c9_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c9_eML_blk_kernel(chartInstance, *chartInstance->c9_q, *chartInstance->c9_y);
  c9_do_animation_call_c9_sliding_controller2(chartInstance);
}

static void mdl_start_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c9_errCode;
  c9_errCode = hipGetLastError();
  if (c9_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c9_errCode, (char_T *)hipGetErrorName
                       (c9_errCode), (char_T *)hipGetErrorString(c9_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c9_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc9_sliding_controller2
  (SFc9_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c9_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c9_nameCaptureInfo = NULL;
  c9_nameCaptureInfo = NULL;
  sf_mex_assign(&c9_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c9_nameCaptureInfo;
}

static void c9_eML_blk_kernel(SFc9_sliding_controller2InstanceStruct
  *chartInstance, real_T c9_b_q[2], real_T c9_b_y[4])
{
  c9_b_y[0] = -0.324 * muDoubleScalarSin(c9_b_q[0]) - 0.39 * muDoubleScalarSin
    (c9_b_q[0] + c9_b_q[1]);
  c9_b_y[2] = 0.324 * muDoubleScalarCos(c9_b_q[0]) + 0.39 * muDoubleScalarSin
    (c9_b_q[0] + c9_b_q[1]);
  c9_b_y[1] = -0.39 * muDoubleScalarSin(c9_b_q[0] + c9_b_q[1]);
  c9_b_y[3] = 0.39 * muDoubleScalarCos(c9_b_q[0] + c9_b_q[1]);
}

static void c9_emlrt_marshallIn(SFc9_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c9_b_y, const char_T *c9_identifier, real_T
  c9_c_y[4])
{
  emlrtMsgIdentifier c9_thisId;
  c9_thisId.fIdentifier = const_cast<const char_T *>(c9_identifier);
  c9_thisId.fParent = NULL;
  c9_thisId.bParentIsCell = false;
  c9_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c9_b_y), &c9_thisId, c9_c_y);
  sf_mex_destroy(&c9_b_y);
}

static void c9_b_emlrt_marshallIn(SFc9_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId,
  real_T c9_b_y[4])
{
  real_T c9_dv[4];
  int32_T c9_i;
  sf_mex_import(c9_parentId, sf_mex_dup(c9_u), c9_dv, 1, 0, 0U, 1, 0U, 2, 2, 2);
  for (c9_i = 0; c9_i < 4; c9_i++) {
    c9_b_y[c9_i] = c9_dv[c9_i];
  }

  sf_mex_destroy(&c9_u);
}

static uint8_T c9_c_emlrt_marshallIn(SFc9_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c9_b_is_active_c9_sliding_controller2, const
  char_T *c9_identifier)
{
  emlrtMsgIdentifier c9_thisId;
  uint8_T c9_b_y;
  c9_thisId.fIdentifier = const_cast<const char_T *>(c9_identifier);
  c9_thisId.fParent = NULL;
  c9_thisId.bParentIsCell = false;
  c9_b_y = c9_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c9_b_is_active_c9_sliding_controller2), &c9_thisId);
  sf_mex_destroy(&c9_b_is_active_c9_sliding_controller2);
  return c9_b_y;
}

static uint8_T c9_d_emlrt_marshallIn(SFc9_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId)
{
  uint8_T c9_b_u;
  uint8_T c9_b_y;
  sf_mex_import(c9_parentId, sf_mex_dup(c9_u), &c9_b_u, 1, 3, 0U, 0, 0U, 0);
  c9_b_y = c9_b_u;
  sf_mex_destroy(&c9_u);
  return c9_b_y;
}

static void init_dsm_address_info(SFc9_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc9_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c9_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c9_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c9_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c9_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(552332206U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2608078302U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1441289006U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4066884445U);
}

mxArray *sf_c9_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c9_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c9_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c9_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiD+wcjAwAakOYCYiQECWKF8RqgYI1ycBS6uAMQllQWpIPHiomTPFCCdl5gL5ie"
    "WVnjmpeWDzbdgQJjPhsV8RiTzOaHiEPDBnjL9Ig4g/QZI+lkI6BcAsiqh4QILH/LtV3CgTD/E/g"
    "AC7ldAcT+En1kcn5hcklmWGp9sGV+ck5mSmZcen5yfV1KUn5OTWmQE8x8AZzscmA=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c9_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sjS5Om55RaK9v6MEldWgzUF";
}

static void sf_opaque_initialize_c9_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c9_sliding_controller2
    ((SFc9_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c9_sliding_controller2((SFc9_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c9_sliding_controller2(void *chartInstanceVar)
{
  enable_c9_sliding_controller2((SFc9_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c9_sliding_controller2(void *chartInstanceVar)
{
  disable_c9_sliding_controller2((SFc9_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c9_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c9_sliding_controller2((SFc9_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c9_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c9_sliding_controller2
    ((SFc9_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c9_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c9_sliding_controller2((SFc9_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c9_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc9_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c9_sliding_controller2
      ((SFc9_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc9_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc9_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c9_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c9_sliding_controller2((SFc9_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c9_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c9_sliding_controller2((SFc9_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc9_sliding_controller2((SFc9_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c9_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c9_sliding_controller2
      ((SFc9_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc9_sliding_controller2((SFc9_sliding_controller2InstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c9_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV8tu20YUpQQnrYvEMIoA3QSIdskyCOCiXrR1rAcsxKqN0EqWxnh4JY41nGHmQVv5h6yK/kE",
    "WXfRbuu6+P1Cgmy5zh6RlhSKpuKoNtygBmhry3DP3fcdeoz/w8NrA+5svPe8uPj/Hu+ll15183Z",
    "i7s/dr3uN8/Q6FhI0OiSKR9movQSJ4CVpya5gUfTGSpTAmRqBAUMTGUpkqNs0iy5mY9Kygjk+/D",
    "hkN/VBaHuyiLAkOBJ8iW2zNIfJ0mAJqegCBCZW047DHyXimsTJn7RDoRNuozgQNxrexU0sPLDcs",
    "5tA9B9oX2hDUWF/q5htioG3OK810lmr/AiijmDMiSq0NifYhRgcbGMYB/j2wBo0qwmhIlNmFkCS",
    "g99kk5ZQCipxM44cTJoiRihHejXjbCS7qdshRn4EMgNc4BHXbVUAmsWTCVMff76GlXUFOOHTgxI",
    "6r2Xx4Y13wXzE4A1Xpt1FbJqDIGA5E5aapQ7rnabRmWbIIMyyCV0Q9pxg/DUFl9mLmaJ9gnOAIJ",
    "apgkBrZ10eKJejeSjYb9V1mLisZG2XB1stgKVs3gboozNh6VLQJ57oSdiTjfUiAp6wdYkg9LGMt",
    "x2nNgiOJDnbpXV0NVjAMfA5rSxGw0nAlBUDad37AxvIxklptZNTG5O3s7y9+XoT1hQE1IhTKuoA",
    "iTAP6LHVvNVvAtIs9AlErk6pXBs4yZBnK0yMrOmdSTdAnNU3k0gQX0UpgpMcYS6yEocaiqYO5WC",
    "7DUUJDCFyDYRwGWDaILfGJdq3tOdZdwsy0A5oqFpdE1WLVYRvquoSaxjAUEyHPRE/JyM97fOZeA",
    "Mw2ogQT413sX2raw93LQuHm2VPvcp7d+4R5diFXfD6Z42mU8Hhzz+K+6836fZv4q5HL7czJ3S/s",
    "s1aQc7hNvL84bf30Z+uX6a8/fvb73uP3366y/8/Nq83/jXz98KLRzgonWchXh92b02uthP+rOf7",
    "NfK1P/a2DaGvrJXmxnXw96PLg9fjtsJfy/dWo17dZ0Pfifct1fMyyND8V7Qf5wcStic3GdXoemt",
    "P37hJ/rOfvs+uP71eTf7BTjOPaEvlN/DUt5O3f37+1s5p8tv/hEv1bhXi30rl+TFy3gGO6faw5C",
    "7DSj6kURknOQT3b+Afq7Kpy3g3L/Vv0/F/u+uP+KXPrpuW8G5Zb1b6rzu/bhq+bA14Bv3mL7Vj1",
    "XHXd+N+8q51/HuXr72b/4rRDxoOS027+GY+uo7Kv/4E8/QD9erI7",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c9_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c9_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2501929482U));
  ssSetChecksum1(S,(2041323763U));
  ssSetChecksum2(S,(3808924622U));
  ssSetChecksum3(S,(1034168136U));
}

static void mdlRTW_c9_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c9_sliding_controller2(SimStruct *S)
{
  SFc9_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc9_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc9_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc9_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc9_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c9_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c9_sliding_controller2;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c9_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c9_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c9_sliding_controller2;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c9_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c9_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c9_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c9_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c9_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c9_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c9_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c9_JITStateAnimation,
    chartInstance->c9_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c9_sliding_controller2(chartInstance);
}

void c9_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c9_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c9_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c9_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c9_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
