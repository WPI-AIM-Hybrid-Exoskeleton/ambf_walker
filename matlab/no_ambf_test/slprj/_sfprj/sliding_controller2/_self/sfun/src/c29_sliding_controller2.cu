/* Include files */

#include "sliding_controller2_sfun.h"
#include "c29_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void enable_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void disable_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void c29_do_animation_call_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance, const mxArray *c29_st);
static void sf_gateway_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance);
static void c29_eML_blk_kernel(SFc29_sliding_controller2InstanceStruct
  *chartInstance, real_T c29_b_q[2], real_T c29_b_y[4]);
static void c29_emlrt_marshallIn(SFc29_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c29_b_y, const char_T *c29_identifier, real_T
  c29_c_y[4]);
static void c29_b_emlrt_marshallIn(SFc29_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c29_u, const emlrtMsgIdentifier *c29_parentId,
  real_T c29_b_y[4]);
static uint8_T c29_c_emlrt_marshallIn(SFc29_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c29_b_is_active_c29_sliding_controller2, const
  char_T *c29_identifier);
static uint8_T c29_d_emlrt_marshallIn(SFc29_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c29_u, const emlrtMsgIdentifier *c29_parentId);
static void init_dsm_address_info(SFc29_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc29_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c29_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c29_is_active_c29_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c29_do_animation_call_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c29_b_y = NULL;
  const mxArray *c29_c_y = NULL;
  const mxArray *c29_d_y = NULL;
  const mxArray *c29_st;
  c29_st = NULL;
  c29_b_y = NULL;
  sf_mex_assign(&c29_b_y, sf_mex_createcellmatrix(2, 1), false);
  c29_c_y = NULL;
  sf_mex_assign(&c29_c_y, sf_mex_create("y", *chartInstance->c29_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c29_b_y, 0, c29_c_y);
  c29_d_y = NULL;
  sf_mex_assign(&c29_d_y, sf_mex_create("y",
    &chartInstance->c29_is_active_c29_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c29_b_y, 1, c29_d_y);
  sf_mex_assign(&c29_st, c29_b_y, false);
  return c29_st;
}

static void set_sim_state_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance, const mxArray *c29_st)
{
  const mxArray *c29_u;
  c29_u = sf_mex_dup(c29_st);
  c29_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c29_u, 0)), "y",
                       *chartInstance->c29_y);
  chartInstance->c29_is_active_c29_sliding_controller2 = c29_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c29_u, 1)),
     "is_active_c29_sliding_controller2");
  sf_mex_destroy(&c29_u);
  sf_mex_destroy(&c29_st);
}

static void sf_gateway_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c29_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c29_eML_blk_kernel(chartInstance, *chartInstance->c29_q, *chartInstance->c29_y);
  c29_do_animation_call_c29_sliding_controller2(chartInstance);
}

static void mdl_start_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c29_errCode;
  c29_errCode = hipGetLastError();
  if (c29_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c29_errCode, (char_T *)hipGetErrorName
                       (c29_errCode), (char_T *)hipGetErrorString(c29_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c29_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc29_sliding_controller2
  (SFc29_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c29_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c29_nameCaptureInfo = NULL;
  c29_nameCaptureInfo = NULL;
  sf_mex_assign(&c29_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c29_nameCaptureInfo;
}

static void c29_eML_blk_kernel(SFc29_sliding_controller2InstanceStruct
  *chartInstance, real_T c29_b_q[2], real_T c29_b_y[4])
{
  c29_b_y[0] = -0.324 * muDoubleScalarSin(c29_b_q[0]) - 0.39 * muDoubleScalarSin
    (c29_b_q[0] + c29_b_q[1]);
  c29_b_y[2] = 0.324 * muDoubleScalarCos(c29_b_q[0]) + 0.39 * muDoubleScalarSin
    (c29_b_q[0] + c29_b_q[1]);
  c29_b_y[1] = -0.39 * muDoubleScalarSin(c29_b_q[0] + c29_b_q[1]);
  c29_b_y[3] = 0.39 * muDoubleScalarCos(c29_b_q[0] + c29_b_q[1]);
}

static void c29_emlrt_marshallIn(SFc29_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c29_b_y, const char_T *c29_identifier, real_T
  c29_c_y[4])
{
  emlrtMsgIdentifier c29_thisId;
  c29_thisId.fIdentifier = const_cast<const char_T *>(c29_identifier);
  c29_thisId.fParent = NULL;
  c29_thisId.bParentIsCell = false;
  c29_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c29_b_y), &c29_thisId,
    c29_c_y);
  sf_mex_destroy(&c29_b_y);
}

static void c29_b_emlrt_marshallIn(SFc29_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c29_u, const emlrtMsgIdentifier *c29_parentId,
  real_T c29_b_y[4])
{
  real_T c29_dv[4];
  int32_T c29_i;
  sf_mex_import(c29_parentId, sf_mex_dup(c29_u), c29_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c29_i = 0; c29_i < 4; c29_i++) {
    c29_b_y[c29_i] = c29_dv[c29_i];
  }

  sf_mex_destroy(&c29_u);
}

static uint8_T c29_c_emlrt_marshallIn(SFc29_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c29_b_is_active_c29_sliding_controller2, const
  char_T *c29_identifier)
{
  emlrtMsgIdentifier c29_thisId;
  uint8_T c29_b_y;
  c29_thisId.fIdentifier = const_cast<const char_T *>(c29_identifier);
  c29_thisId.fParent = NULL;
  c29_thisId.bParentIsCell = false;
  c29_b_y = c29_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c29_b_is_active_c29_sliding_controller2), &c29_thisId);
  sf_mex_destroy(&c29_b_is_active_c29_sliding_controller2);
  return c29_b_y;
}

static uint8_T c29_d_emlrt_marshallIn(SFc29_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c29_u, const emlrtMsgIdentifier *c29_parentId)
{
  uint8_T c29_b_u;
  uint8_T c29_b_y;
  sf_mex_import(c29_parentId, sf_mex_dup(c29_u), &c29_b_u, 1, 3, 0U, 0, 0U, 0);
  c29_b_y = c29_b_u;
  sf_mex_destroy(&c29_u);
  return c29_b_y;
}

static void init_dsm_address_info(SFc29_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc29_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c29_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c29_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c29_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c29_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(552332206U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2608078302U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1441289006U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4066884445U);
}

mxArray *sf_c29_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c29_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c29_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c29_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8clGlvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABcGhvd"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c29_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sjS5Om55RaK9v6MEldWgzUF";
}

static void sf_opaque_initialize_c29_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c29_sliding_controller2
    ((SFc29_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c29_sliding_controller2((SFc29_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c29_sliding_controller2(void *chartInstanceVar)
{
  enable_c29_sliding_controller2((SFc29_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c29_sliding_controller2(void *chartInstanceVar)
{
  disable_c29_sliding_controller2((SFc29_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c29_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c29_sliding_controller2((SFc29_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c29_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c29_sliding_controller2
    ((SFc29_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c29_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c29_sliding_controller2((SFc29_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c29_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc29_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c29_sliding_controller2
      ((SFc29_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc29_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc29_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c29_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c29_sliding_controller2((SFc29_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c29_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c29_sliding_controller2((SFc29_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc29_sliding_controller2((SFc29_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c29_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c29_sliding_controller2
      ((SFc29_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc29_sliding_controller2
      ((SFc29_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c29_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgUnjYs2MIoAuQSIc0qPgQEHyaGNbf3AQq3aMK0kN2O9HIlrLXeZ/aGtvENOQd6",
    "gxx76JD3nnhco0BfoLEnJKkVScYQYSVECNLXkN7Mz883P2lvp9jy8buO9/YPn3cTnLbwbXnbdyN",
    "crM3f2ftV7mK/foJCw0SFRJNJe7SVIBEegJbeGSdEVA1kKY2IACgRFbCyVqdKmWWQ5E6OOFdTp0",
    "y9CRkM/lJYHuyhLggPBx6gttuYQ9bSYAmo6AIEJlbTDsMPJcGqxMufNEOhI26jOBQ3Gt7EzS/cs",
    "Nyzm0L4A2hXaELRYX9rmG2KgaS4q3XSean8ClFHMGRGl3oZE+xBjgA304wD/HliDThVhNCTK7EJ",
    "IEtD7bJTqlAKKOpnGD6dMECMVI7wd8aYTnLftkKM9PRkArwkI2rargIxiyYSp5t/voKdtQU45tO",
    "DUDqu1+fDKOvKfMzgHVRm3QVMmoMgQDkTlpmlA2hcpW9MsmYcZFsFzonYo8qchqMxezBztE+QJj",
    "lGiCgapk119rFiC4a3UZqOuy8xFJWOjjGy9CJZqaydQx8JUW4eKJuFcV8KOZbwPCfBUa4sYUg/L",
    "tJbjtGbBscQAu/SurgYrGBKfw5pSBKyUrqQASPvOr9hY/o2kVhsZNTF5W/v785/nYV1hQA0IhbI",
    "uoAjTgDFLw1utLWDacY9AtMqk5pWBswxZhPL0wIrWuVQjjElNE7l0wTFaCYz0ELnESuhrLJo6mO",
    "NyEY4SGkLgGgzj0MOyQWxJTLRrbTtYdwkz4xZoqlhcwqrFqsM21HYJNY6hL0ZCnouOkpGf9/gsv",
    "ACYbUQJJoa72L/UuIO7l1Hh5tkj73KeffcR82wiV3z+OKNnpUSPN/Ms7rvWqN+3gb9WJvN3Ru77",
    "wj6rBTmHW8f727ONd39v/D7+8+03H/Ye/vbTMvv/0bja/L+dr+9NGu20cJK5fHXYvRm7Vkv0353",
    "Rv56v9Zm/dRBtbR2RX54mj3ttHrwYvu53Mn4W2Nso2Dt5v+E6PmZZmp+KdoP8YOLWxGbj2ul/Mm",
    "PvzQXxWMvfZ9dfz5aTv7Nd5HF1gfw6/hoX8vbT99/YXk4+2//lAvsfFPh+kM71E+K6BZzQzacnm",
    "rMAS/2ESmGU5BzUZkl9fmq+X1XOu2a5r8XO/+U+P+8fM7+uW867Zrll/bvqHP/S8HXzwCvg179g",
    "P5Y9X31u/Hvvaueg+/n65+m/Os2Q8aDk1Jt/xiPsoOzrfyBP/wHJMrGQ",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c29_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c29_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2501929482U));
  ssSetChecksum1(S,(2041323763U));
  ssSetChecksum2(S,(3808924622U));
  ssSetChecksum3(S,(1034168136U));
}

static void mdlRTW_c29_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c29_sliding_controller2(SimStruct *S)
{
  SFc29_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc29_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc29_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc29_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc29_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c29_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c29_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c29_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c29_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c29_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c29_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c29_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c29_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c29_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c29_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c29_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c29_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c29_JITStateAnimation,
    chartInstance->c29_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c29_sliding_controller2(chartInstance);
}

void c29_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c29_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c29_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c29_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c29_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
