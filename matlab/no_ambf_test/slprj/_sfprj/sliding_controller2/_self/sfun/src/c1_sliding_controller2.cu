/* Include files */

#include "sliding_controller2_sfun.h"
#include "c1_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void enable_c1_sliding_controller2(SFc1_sliding_controller2InstanceStruct
  *chartInstance);
static void disable_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void c1_do_animation_call_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance, const mxArray *c1_st);
static void sf_gateway_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void c1_eML_blk_kernel(SFc1_sliding_controller2InstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2]);
static void c1_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2]);
static void c1_b_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2]);
static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller2, const
  char_T *c1_identifier);
static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static void init_dsm_address_info(SFc1_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc1_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c1_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c1_is_active_c1_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c1_sliding_controller2(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c1_do_animation_call_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c1_b_y = NULL;
  const mxArray *c1_c_y = NULL;
  const mxArray *c1_d_y = NULL;
  const mxArray *c1_e_y = NULL;
  const mxArray *c1_f_y = NULL;
  const mxArray *c1_st;
  const mxArray *c1_y = NULL;
  c1_st = NULL;
  c1_y = NULL;
  sf_mex_assign(&c1_y, sf_mex_createcellmatrix(5, 1), false);
  c1_b_y = NULL;
  sf_mex_assign(&c1_b_y, sf_mex_create("y", *chartInstance->c1_ankle, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 0, c1_b_y);
  c1_c_y = NULL;
  sf_mex_assign(&c1_c_y, sf_mex_create("y", *chartInstance->c1_danke, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 1, c1_c_y);
  c1_d_y = NULL;
  sf_mex_assign(&c1_d_y, sf_mex_create("y", *chartInstance->c1_dknee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 2, c1_d_y);
  c1_e_y = NULL;
  sf_mex_assign(&c1_e_y, sf_mex_create("y", *chartInstance->c1_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 3, c1_e_y);
  c1_f_y = NULL;
  sf_mex_assign(&c1_f_y, sf_mex_create("y",
    &chartInstance->c1_is_active_c1_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c1_y, 4, c1_f_y);
  sf_mex_assign(&c1_st, c1_y, false);
  return c1_st;
}

static void set_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance, const mxArray *c1_st)
{
  const mxArray *c1_u;
  c1_u = sf_mex_dup(c1_st);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 0)),
                      "ankle", *chartInstance->c1_ankle);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 1)),
                      "danke", *chartInstance->c1_danke);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 2)),
                      "dknee", *chartInstance->c1_dknee);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 3)), "knee",
                      *chartInstance->c1_knee);
  chartInstance->c1_is_active_c1_sliding_controller2 = c1_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 4)),
     "is_active_c1_sliding_controller2");
  sf_mex_destroy(&c1_u);
  sf_mex_destroy(&c1_st);
}

static void sf_gateway_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c1_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c1_eML_blk_kernel(chartInstance, *chartInstance->c1_q, *chartInstance->c1_qd, *
                    chartInstance->c1_knee, *chartInstance->c1_dknee,
                    *chartInstance->c1_ankle, *chartInstance->c1_danke);
  c1_do_animation_call_c1_sliding_controller2(chartInstance);
}

static void mdl_start_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c1_errCode;
  c1_errCode = hipGetLastError();
  if (c1_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c1_errCode, (char_T *)hipGetErrorName
                       (c1_errCode), (char_T *)hipGetErrorString(c1_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c1_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c1_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c1_nameCaptureInfo = NULL;
  const char_T *c1_data[4] = {
    "789cc593cd4ac34014856fa516375537fa188e2d08dd5745118bd0e246244e929b66ccfcc8646a5b372ebbf4b1f475dcb8706592364d1b0c118bf542b8393999"
    "f9ee1c18a89c5f5600603b7a3e9e01ce6a90547dda6077d63760b9f27ea5a0a7b509d5a575b1ff1ef15e66da51d2e0c84c85a402e72b5d2598a4d2f4c60f081a",
    "43c51fd14d1c8f71ec3181dd45d18995385db0e622b6e2f7b68f4ed01d08d07e984dc8174592475c7705e7ad96e491af7c1ef9ff52dee897bc74fffd125eea7b"
    "4a0fa976ad804926a8614e6871ecaf7eee945b2b9c63eab86a6073cc786f2bf268216fd9bf39b925be124824353e950c795f71d7a3da260e35511ad63024a176",
    "0815b6670d290f509328204e6d2295957c35181af27d800762c629cb71e787e72aba4f75d84a7a6bef88ae9337f97c6dae9397d67ff1d6751f2f0ec7acf1c4ee"
    "83a0d96a771ab6ed5cbbe6389be3aa8453360714e8bfdeff0b89616e21", "" };

  c1_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c1_data[0], 1632U, &c1_nameCaptureInfo);
  return c1_nameCaptureInfo;
}

static void c1_eML_blk_kernel(SFc1_sliding_controller2InstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2])
{
  real_T c1_a;
  real_T c1_b_a;
  c1_b_knee[0] = -0.324 * muDoubleScalarSin(c1_b_q[0]);
  c1_b_knee[1] = -0.324 * muDoubleScalarCos(c1_b_q[0]);
  c1_a = -0.324 * c1_b_qd[0];
  c1_b_dknee[0] = c1_a * muDoubleScalarCos(c1_b_q[0]);
  c1_b_dknee[1] = c1_a * -muDoubleScalarSin(c1_b_q[0]);
  c1_b_a = -0.39 * (c1_b_qd[0] + c1_b_qd[1]);
  c1_b_ankle[0] = c1_b_knee[0] + -0.39 * muDoubleScalarSin(c1_b_q[0] + c1_b_q[1]);
  c1_b_ankle[1] = c1_b_knee[1] + -0.39 * muDoubleScalarCos(c1_b_q[0] + c1_b_q[1]);
  c1_b_danke[0] = c1_b_dknee[0] + c1_b_a * muDoubleScalarCos(c1_b_q[0] + c1_b_q
    [1]);
  c1_b_danke[1] = c1_b_dknee[1] + c1_b_a * -muDoubleScalarSin(c1_b_q[0] +
    c1_b_q[1]);
}

static void c1_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2])
{
  emlrtMsgIdentifier c1_thisId;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_b_ankle), &c1_thisId, c1_y);
  sf_mex_destroy(&c1_b_ankle);
}

static void c1_b_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2])
{
  real_T c1_dv[2];
  int32_T c1_i;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), c1_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c1_i = 0; c1_i < 2; c1_i++) {
    c1_y[c1_i] = c1_dv[c1_i];
  }

  sf_mex_destroy(&c1_u);
}

static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller2, const
  char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  uint8_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c1_b_is_active_c1_sliding_controller2), &c1_thisId);
  sf_mex_destroy(&c1_b_is_active_c1_sliding_controller2);
  return c1_y;
}

static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  uint8_T c1_b_u;
  uint8_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_b_u, 1, 3, 0U, 0, 0U, 0);
  c1_y = c1_b_u;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static void init_dsm_address_info(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c1_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c1_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c1_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c1_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c1_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c1_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c1_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c1_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c1_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c1_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6DFA4ifWRyfmFySWZYan"
    "2wYX5yTmZKZlx6fnJ9XUpSfk5NaZATzHwD+USxt"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c1_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c1_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c1_sliding_controller2
    ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c1_sliding_controller2(void *chartInstanceVar)
{
  enable_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c1_sliding_controller2(void *chartInstanceVar)
{
  disable_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c1_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c1_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c1_sliding_controller2
    ((SFc1_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c1_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c1_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c1_sliding_controller2
      ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc1_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c1_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c1_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c1_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c1_sliding_controller2
      ((SFc1_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c1_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQlaIDG8CFALwFq9JIeWsBtEaBFgcY2JSFC7Vop5RTIxVgvR+JCy112f2g771A",
    "k7xL00hdJznmCvkE7S9KyTJGUZSepW5QGJS/5zbfzzczOrrzWYM/Daw3vl/c87zZ+f4R328uvW8",
    "W4NXfnzzveg2L8GxoJGw+JIrH2Gi9BYvgZtOTWMCkGYiwrYUyMQYGgiE2kMnVsmsWWMzHtW0Edn",
    "/4lYjQKIml5uIO2JNwX/BTZEmuGyNNlCqjpA4QmUtJOoj4nk5nHyhz7EdCptnGTBA0msIlzS+9Z",
    "bljCoXcCdCC0IeixPvctMMSAb05qZTqlOjgDyjjhjIhKtRHRASQYYAMHSYif+9agqDKMRkSZHYh",
    "ICnqXTTNOKaDMyTS+OGKCGKkY4b2Y+85w0bchR3/2ZAi8ISDo244CMk0kE6Y+/0EflfYEOeLQhS",
    "M7qWcL4Ffrkv+UwTGo2riNfZmCIhPYF7WTZgHpnWTZmlXJIsywGJ4StU0xfxrC2urFytEBwTzBC",
    "C3qYJCJHOiRYimGt5bNxgNXmcuWjI3zZOtlsIytl0JTFmZsfSp8wrmuhY1ksgsp8Iy1SwxphuWs",
    "1TitWTiSGGBX3vWrwQqGiS9gvhQhq0xXWgJkfecnbCwXkdRqI2Mfi7e7u7v4ehE2EAbUmFCo6gK",
    "KMA0Ysyy89Wwh0y73CESvTOZeFTivkGUoT4+t6B5LNcWYNDSRcwkuo7XAWE8wl7gSDjQumiaYy+",
    "UyHCU0gtA1GMZhD5cNYitiol1r28Z1lzJz2gVNFUsqsmpx1WEb6rmCOk3gQEyFPBZ9JeOg6PF5e",
    "AGw2ogSTEx2sH+p0z7OXpUKt59teuf72ceX2M/O7Mrfn8/xtCp4vLnv8rx32hfx7dK8bfxrtXK7",
    "rTm7u6V5OiU7h1vH+/mrZ7//+PqPF18+ePvFi9H3fy2bv7Uwfyv7f60ArbL/rxXj+2eNdrZw0oV",
    "6ddjHc351Kvg/meNfL8b6SfRQpM8eR9tPyCgebH7z/LvN9MjP+IadZn9vlfw9e77hOj5WWVafig",
    "7C4mDixsTm27Xj/3bO39tL4nGneJ5ffz66nv39rbJ9Z4m+9WJMxJRDqX7/OR2fXVlHiEJujo6Nq",
    "+uY4qHhxui4t1XuD1U6Ohd0dDwn4d34vxjH1ezz+YdL/N8o5WEjOy8eErcLwSH96lBzFuIOckil",
    "MEpyDurrtUv276b+uaqd94Ht/i1+/m/3/vN+mfNQ+4p2nWuevz6U3XX1rXouvGn4pn3AK+HXb7C",
    "O657X3zf+jbfaufrTYvzD7KezHzEeVvyKKl7jT6Jx1dv/QJ3+DahHwwQ=",
    ""
  };

  static char newstr [1333] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c1_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c1_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042620282U));
  ssSetChecksum1(S,(2411386699U));
  ssSetChecksum2(S,(753018669U));
  ssSetChecksum3(S,(4282078351U));
}

static void mdlRTW_c1_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c1_sliding_controller2(SimStruct *S)
{
  SFc1_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc1_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc1_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc1_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc1_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c1_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c1_sliding_controller2;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c1_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c1_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c1_sliding_controller2;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c1_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c1_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c1_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c1_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c1_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c1_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c1_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c1_JITStateAnimation,
    chartInstance->c1_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c1_sliding_controller2(chartInstance);
}

void c1_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c1_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c1_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c1_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c1_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
