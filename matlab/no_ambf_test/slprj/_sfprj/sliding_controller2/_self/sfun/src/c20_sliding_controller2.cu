/* Include files */

#include "sliding_controller2_sfun.h"
#include "c20_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void enable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void disable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void c20_do_animation_call_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance, const mxArray *c20_st);
static void sf_gateway_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void c20_eML_blk_kernel(SFc20_sliding_controller2InstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2]);
static void c20_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2]);
static void c20_b_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2]);
static uint8_T c20_c_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_is_active_c20_sliding_controller2, const
  char_T *c20_identifier);
static uint8_T c20_d_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId);
static void init_dsm_address_info(SFc20_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc20_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c20_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c20_is_active_c20_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c20_do_animation_call_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c20_b_y = NULL;
  const mxArray *c20_c_y = NULL;
  const mxArray *c20_d_y = NULL;
  const mxArray *c20_e_y = NULL;
  const mxArray *c20_f_y = NULL;
  const mxArray *c20_st;
  const mxArray *c20_y = NULL;
  c20_st = NULL;
  c20_y = NULL;
  sf_mex_assign(&c20_y, sf_mex_createcellmatrix(5, 1), false);
  c20_b_y = NULL;
  sf_mex_assign(&c20_b_y, sf_mex_create("y", *chartInstance->c20_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 0, c20_b_y);
  c20_c_y = NULL;
  sf_mex_assign(&c20_c_y, sf_mex_create("y", *chartInstance->c20_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 1, c20_c_y);
  c20_d_y = NULL;
  sf_mex_assign(&c20_d_y, sf_mex_create("y", *chartInstance->c20_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 2, c20_d_y);
  c20_e_y = NULL;
  sf_mex_assign(&c20_e_y, sf_mex_create("y", *chartInstance->c20_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c20_y, 3, c20_e_y);
  c20_f_y = NULL;
  sf_mex_assign(&c20_f_y, sf_mex_create("y",
    &chartInstance->c20_is_active_c20_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c20_y, 4, c20_f_y);
  sf_mex_assign(&c20_st, c20_y, false);
  return c20_st;
}

static void set_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance, const mxArray *c20_st)
{
  const mxArray *c20_u;
  c20_u = sf_mex_dup(c20_st);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 0)),
                       "ankle", *chartInstance->c20_ankle);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 1)),
                       "danke", *chartInstance->c20_danke);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 2)),
                       "dknee", *chartInstance->c20_dknee);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 3)),
                       "knee", *chartInstance->c20_knee);
  chartInstance->c20_is_active_c20_sliding_controller2 = c20_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 4)),
     "is_active_c20_sliding_controller2");
  sf_mex_destroy(&c20_u);
  sf_mex_destroy(&c20_st);
}

static void sf_gateway_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c20_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c20_eML_blk_kernel(chartInstance, *chartInstance->c20_q,
                     *chartInstance->c20_qd, *chartInstance->c20_knee,
                     *chartInstance->c20_dknee, *chartInstance->c20_ankle,
                     *chartInstance->c20_danke);
  c20_do_animation_call_c20_sliding_controller2(chartInstance);
}

static void mdl_start_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c20_errCode;
  c20_errCode = hipGetLastError();
  if (c20_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c20_errCode, (char_T *)hipGetErrorName
                       (c20_errCode), (char_T *)hipGetErrorString(c20_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c20_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c20_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c20_nameCaptureInfo = NULL;
  const char_T *c20_data[4] = {
    "789cc593cd4ac34014856fa516375537fa188e2d08dd5745118bd0e246244e929b66ccfcc8646a5b372ebbf4b1f475dcb8706592364d1b0c118bf542b8393999"
    "f9ee1c18a89c5f5600603b7a3e9e01ce6a90547dda6077d63760b9f27ea5a0a7b509d5a575b1ff1ef15e66da51d2e0c84c85a402e72b5d2598a4d2f4c60f081a",
    "43c51fd14d1c8f71ec3181dd45d18995385db0e622b6e2f7b68f4ed01d08d07e984dc8174592475c7705e7ad96e491af7c1ef9ff52dee897bc74fffd125eea7b"
    "4a0fa976ad804926a8614e6871ecaf7eee945b2b9c63eab86a6073cc786f2bf268216fd9bf39b925be124824353e950c795f71d7a3da260e35511ad63024a176",
    "0815b6670d290f509328204e6d2295957c35181af27d800762c629cb71e787e72aba4f75d84a7a6bef88ae9337f97c6dae9397d67ff1d6751f2f0ec7acf1c4ee"
    "83a0d96a771ab6ed5cbbe6389be3aa8453360714e8bfdeff0b89616e21", "" };

  c20_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c20_data[0], 1632U, &c20_nameCaptureInfo);
  return c20_nameCaptureInfo;
}

static void c20_eML_blk_kernel(SFc20_sliding_controller2InstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2])
{
  real_T c20_a;
  real_T c20_b_a;
  c20_b_knee[0] = -0.324 * muDoubleScalarSin(c20_b_q[0]);
  c20_b_knee[1] = -0.324 * muDoubleScalarCos(c20_b_q[0]);
  c20_a = -0.324 * c20_b_qd[0];
  c20_b_dknee[0] = c20_a * muDoubleScalarCos(c20_b_q[0]);
  c20_b_dknee[1] = c20_a * -muDoubleScalarSin(c20_b_q[0]);
  c20_b_a = -0.39 * (c20_b_qd[0] + c20_b_qd[1]);
  c20_b_ankle[0] = c20_b_knee[0] + -0.39 * muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_ankle[1] = c20_b_knee[1] + -0.39 * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[0] = c20_b_dknee[0] + c20_b_a * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[1] = c20_b_dknee[1] + c20_b_a * -muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
}

static void c20_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2])
{
  emlrtMsgIdentifier c20_thisId;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c20_b_ankle), &c20_thisId,
    c20_y);
  sf_mex_destroy(&c20_b_ankle);
}

static void c20_b_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2])
{
  real_T c20_dv[2];
  int32_T c20_i;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), c20_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c20_i = 0; c20_i < 2; c20_i++) {
    c20_y[c20_i] = c20_dv[c20_i];
  }

  sf_mex_destroy(&c20_u);
}

static uint8_T c20_c_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_is_active_c20_sliding_controller2, const
  char_T *c20_identifier)
{
  emlrtMsgIdentifier c20_thisId;
  uint8_T c20_y;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_y = c20_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c20_b_is_active_c20_sliding_controller2), &c20_thisId);
  sf_mex_destroy(&c20_b_is_active_c20_sliding_controller2);
  return c20_y;
}

static uint8_T c20_d_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId)
{
  uint8_T c20_b_u;
  uint8_T c20_y;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), &c20_b_u, 1, 3, 0U, 0, 0U, 0);
  c20_y = c20_b_u;
  sf_mex_destroy(&c20_u);
  return c20_y;
}

static void init_dsm_address_info(SFc20_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc20_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c20_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c20_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c20_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c20_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c20_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c20_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c20_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c20_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c20_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c20_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6LFA4ifWRyfmFySWZYan"
    "2xkEF+ck5mSmZcen5yfV1KUn5OTWmSEZD4IAACGpiyw"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c20_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c20_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c20_sliding_controller2
    ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c20_sliding_controller2(void *chartInstanceVar)
{
  enable_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c20_sliding_controller2(void *chartInstanceVar)
{
  disable_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c20_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c20_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c20_sliding_controller2
    ((SFc20_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c20_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c20_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c20_sliding_controller2
      ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc20_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c20_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c20_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c20_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c20_sliding_controller2
      ((SFc20_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc20_sliding_controller2
      ((SFc20_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c20_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFALgGa5JIeGsBIEaBFgdY2JSFC7dop5STIxVgvR+JCy11mf2g771D",
    "k2vcoesmLpOc+Qd4gmSVpWaZIKrKb1ClKg5KX/Obb+WZmZ1dea7Dt4XUd799veN4V/L6Kd9vLr5",
    "Vi3Jq58+cd714x/g2NhI13iSKx9hovQWL4FbTk1jApBmIkK2FMjECBoIhNpDJ1bJrFljMx6VtBH",
    "Z9+GjEaBZG0PNxEWxLuCH6MbIk1u8jTZQqo6QOEJlLSjqM+J+Opx8oc+hHQibZxkwQNJrCJc0tv",
    "W25YwqF3BHQgtCHosT71LTDEgG+OamU6pTo4Aco44YyISrUR0QEkGGADe0mInzvWoKgyjEZEmU2",
    "ISAp6i00yTimgzMk0vjhgghipGOG9mPvOcN63XY7+bMsQeENA0LdNBWSSSCZMff6DPirtCXLAoQ",
    "sHdlzPFsAL65L/hMEhqNq4jXyZgiJj2BG1k2YB6R1l2ZpWyTzMsBieELVBMX8awtrqxcrRAcE8w",
    "RAt6mCQiRzooWIphreWzcYDV5mLloyN82TrRbCMrZdCUxambH0qfMK5roUNZbIFKfCMtUsMaYbl",
    "rNU4rVk4lBhgV971q8EKhokvYL4UIatMV1oCZH3nF2wsZ5HUaiNjH4u3u7U1/3oeNhAG1IhQqOo",
    "CijANGLMsvPVsIdMu9whEr0zmXhU4r5BFKE+PrOgeSjXBmDQ0kVMJLqO1wFiPMZe4EvY0LpommM",
    "vlIhwlNILQNRjGYRuXDWIrYqJda9vAdZcyc9wFTRVLKrJqcdVhG+q5gjpOYE9MhDwUfSXjoOjxe",
    "XgBsNqIEkyMN7F/qeM+zl6VCrefrXmn+9kXH7CfndiVv7+e4WlV8Hgz3+V5r7XP4tuledv412rl",
    "duszdl+W5umU7BxuFe+Xfzz/8+c3r1/dv/f3N6+GP7xbNH9rbv5W9r+zu7qy3P5/vRjfOmm004W",
    "TztWrwz6a8atTwX9zhn+1GOvH0UORPn8UbTwmw3iw9u3L79fSAz/je9Zp9nel5O/J89uu42OVZf",
    "Wp6CAsDiZuTGy+XTv+72b8vbIgHteK5/n19qeL2d9aL9t3FuhbLcZETDiU6vff03H33DpCFHJ5d",
    "Nw+v44JHhoujY4b6+X+UKWjc0ZHx3MS/hn/5+O4nH0+/7MF/t8p5eFOdl7cJ24Xgn36YG1fcxbi",
    "FrJPpTBKcg7qQUXfP28fXdbO+8R2n4uf/9t9/Lx/yLmofU67zgXPYZ/K7qL6lj0fXjZ8037glfC",
    "rl1jHRc/tHxv/l7fc+fqrYvzj9Ce0HzEeVvyaKl7jT6NR1dv/QJ2+B7Wew1c=",
    ""
  };

  static char newstr [1337] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c20_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c20_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042620282U));
  ssSetChecksum1(S,(2411386699U));
  ssSetChecksum2(S,(753018669U));
  ssSetChecksum3(S,(4282078351U));
}

static void mdlRTW_c20_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c20_sliding_controller2(SimStruct *S)
{
  SFc20_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc20_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc20_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc20_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc20_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c20_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c20_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c20_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c20_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c20_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c20_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c20_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c20_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c20_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c20_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c20_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c20_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c20_JITStateAnimation,
    chartInstance->c20_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c20_sliding_controller2(chartInstance);
}

void c20_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c20_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c20_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c20_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c20_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
