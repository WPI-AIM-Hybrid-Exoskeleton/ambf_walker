#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c18_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void enable_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void disable_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void c18_do_animation_call_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance, const mxArray *c18_st);
static void sf_gateway_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance);
static void c18_eML_blk_kernel(SFc18_sliding_controller2InstanceStruct
  *chartInstance, real_T c18_b_u[6], real_T c18_b_y[7]);
static void c18_emlrt_marshallIn(SFc18_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c18_b_y, const char_T *c18_identifier, real_T
  c18_c_y[7]);
static void c18_b_emlrt_marshallIn(SFc18_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c18_b_u, const emlrtMsgIdentifier *c18_parentId,
  real_T c18_b_y[7]);
static uint8_T c18_c_emlrt_marshallIn(SFc18_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c18_b_is_active_c18_sliding_controller2, const
  char_T *c18_identifier);
static uint8_T c18_d_emlrt_marshallIn(SFc18_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c18_b_u, const emlrtMsgIdentifier *c18_parentId);
static __global__ void c18_eML_blk_kernel_kernel1(real_T c18_b_y[7]);
static void init_dsm_address_info(SFc18_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc18_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c18_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c18_is_active_c18_sliding_controller2 = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c18_gpu_y, 56UL);
}

static void initialize_params_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c18_do_animation_call_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c18_b_y = NULL;
  const mxArray *c18_c_y = NULL;
  const mxArray *c18_d_y = NULL;
  const mxArray *c18_st;
  c18_st = NULL;
  c18_b_y = NULL;
  sf_mex_assign(&c18_b_y, sf_mex_createcellmatrix(2, 1), false);
  c18_c_y = NULL;
  sf_mex_assign(&c18_c_y, sf_mex_create("y", *chartInstance->c18_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c18_b_y, 0, c18_c_y);
  c18_d_y = NULL;
  sf_mex_assign(&c18_d_y, sf_mex_create("y",
    &chartInstance->c18_is_active_c18_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c18_b_y, 1, c18_d_y);
  sf_mex_assign(&c18_st, c18_b_y, false);
  return c18_st;
}

static void set_sim_state_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance, const mxArray *c18_st)
{
  const mxArray *c18_b_u;
  c18_b_u = sf_mex_dup(c18_st);
  c18_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c18_b_u, 0)),
                       "y", *chartInstance->c18_y);
  chartInstance->c18_is_active_c18_sliding_controller2 = c18_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c18_b_u, 1)),
     "is_active_c18_sliding_controller2");
  sf_mex_destroy(&c18_b_u);
  sf_mex_destroy(&c18_st);
}

static void sf_gateway_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
  real_T c18_dv1[7];
  real_T c18_dv[6];
  int32_T c18_i;
  int32_T c18_i1;
  chartInstance->c18_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c18_i = 0; c18_i < 6; c18_i++) {
    c18_dv[c18_i] = (*chartInstance->c18_u)[c18_i];
  }

  c18_eML_blk_kernel(chartInstance, c18_dv, c18_dv1);
  for (c18_i1 = 0; c18_i1 < 7; c18_i1++) {
    (*chartInstance->c18_y)[c18_i1] = c18_dv1[c18_i1];
  }

  c18_do_animation_call_c18_sliding_controller2(chartInstance);
}

static void mdl_start_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c18_errCode;
  hipFree(*chartInstance->c18_gpu_y);
  c18_errCode = hipGetLastError();
  if (c18_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c18_errCode, (char_T *)hipGetErrorName
                       (c18_errCode), (char_T *)hipGetErrorString(c18_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c18_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc18_sliding_controller2
  (SFc18_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c18_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c18_nameCaptureInfo = NULL;
  c18_nameCaptureInfo = NULL;
  sf_mex_assign(&c18_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c18_nameCaptureInfo;
}

static void c18_eML_blk_kernel(SFc18_sliding_controller2InstanceStruct
  *chartInstance, real_T c18_b_u[6], real_T c18_b_y[7])
{
  const mxArray *c18_c_y = NULL;
  c18_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c18_gpu_y);
  hipMemcpy(c18_b_y, *chartInstance->c18_gpu_y, 56UL, hipMemcpyDeviceToHost);
  c18_b_y[0] = c18_b_u[3];
  c18_b_y[1] = 0.0;
  sf_mex_printf("%s =\\n", "reordered");
  c18_c_y = NULL;
  sf_mex_assign(&c18_c_y, sf_mex_create("y", c18_b_y, 0, 0U, 1U, 0U, 2, 1, 7),
                false);
  sf_mex_call(chartInstance->c18_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c18_c_y);
  c18_b_y[2] = c18_b_u[0];
  c18_b_y[3] = c18_b_u[1];
  c18_b_y[4] = c18_b_u[4];
  c18_b_y[5] = c18_b_u[5];
  c18_b_y[6] = c18_b_u[2];
}

static void c18_emlrt_marshallIn(SFc18_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c18_b_y, const char_T *c18_identifier, real_T
  c18_c_y[7])
{
  emlrtMsgIdentifier c18_thisId;
  c18_thisId.fIdentifier = const_cast<const char_T *>(c18_identifier);
  c18_thisId.fParent = NULL;
  c18_thisId.bParentIsCell = false;
  c18_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c18_b_y), &c18_thisId,
    c18_c_y);
  sf_mex_destroy(&c18_b_y);
}

static void c18_b_emlrt_marshallIn(SFc18_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c18_b_u, const emlrtMsgIdentifier *c18_parentId,
  real_T c18_b_y[7])
{
  real_T c18_dv[7];
  int32_T c18_i;
  sf_mex_import(c18_parentId, sf_mex_dup(c18_b_u), c18_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c18_i = 0; c18_i < 7; c18_i++) {
    c18_b_y[c18_i] = c18_dv[c18_i];
  }

  sf_mex_destroy(&c18_b_u);
}

static uint8_T c18_c_emlrt_marshallIn(SFc18_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c18_b_is_active_c18_sliding_controller2, const
  char_T *c18_identifier)
{
  emlrtMsgIdentifier c18_thisId;
  uint8_T c18_b_y;
  c18_thisId.fIdentifier = const_cast<const char_T *>(c18_identifier);
  c18_thisId.fParent = NULL;
  c18_thisId.bParentIsCell = false;
  c18_b_y = c18_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c18_b_is_active_c18_sliding_controller2), &c18_thisId);
  sf_mex_destroy(&c18_b_is_active_c18_sliding_controller2);
  return c18_b_y;
}

static uint8_T c18_d_emlrt_marshallIn(SFc18_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c18_b_u, const emlrtMsgIdentifier *c18_parentId)
{
  uint8_T c18_b_y;
  uint8_T c18_c_u;
  sf_mex_import(c18_parentId, sf_mex_dup(c18_b_u), &c18_c_u, 1, 3, 0U, 0, 0U, 0);
  c18_b_y = c18_c_u;
  sf_mex_destroy(&c18_b_u);
  return c18_b_y;
}

static __global__ __launch_bounds__(32, 1) void c18_eML_blk_kernel_kernel1
  (real_T c18_b_y[7])
{
  uint64_T c18_threadId;
  int32_T c18_i;
  c18_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c18_i = (int32_T)c18_threadId;
  if (c18_i < 7) {
    c18_b_y[c18_i] = 0.0;
  }
}

static void init_dsm_address_info(SFc18_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc18_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c18_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c18_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c18_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c18_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1104270009U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4002144469U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1451542329U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1457894201U);
}

mxArray *sf_c18_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c18_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c18_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c18_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8cmGFvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABb0Rvb"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c18_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "scU9zZ5H1kMhWDkDI28BpgD";
}

static void sf_opaque_initialize_c18_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c18_sliding_controller2
    ((SFc18_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c18_sliding_controller2((SFc18_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c18_sliding_controller2(void *chartInstanceVar)
{
  enable_c18_sliding_controller2((SFc18_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c18_sliding_controller2(void *chartInstanceVar)
{
  disable_c18_sliding_controller2((SFc18_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c18_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c18_sliding_controller2((SFc18_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c18_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c18_sliding_controller2
    ((SFc18_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c18_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c18_sliding_controller2((SFc18_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c18_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc18_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c18_sliding_controller2
      ((SFc18_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc18_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc18_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c18_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c18_sliding_controller2((SFc18_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c18_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c18_sliding_controller2((SFc18_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc18_sliding_controller2((SFc18_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c18_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c18_sliding_controller2
      ((SFc18_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc18_sliding_controller2
      ((SFc18_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c18_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV09v00gUd6MuAglQhRZxWYly2j0uSKDuBdrEiYjUbKt1Cysu1XT8Eo8ynvHOn7ThKyAuqBc",
    "OfAb4DEh8By58h72ttKd9Yztp1rEdSkQFCEuuM/bvvXnv/d6fqbfS7Xl4XcV785rnXcDnRbwbXn",
    "b9kK9XZu7s/ar3c75+jkLCxrtEkVh7tZcgMfwBWnJrmBRd0ZelMCb6oEBQxCZSmSptmsWWMzHsW",
    "EGdPv04YjQKIml52ERZEu4IPkZtiTW7qMdnCqjpAIQmUtIOog4ng6nFyhy1IqBDbeM6FzSYwCbO",
    "LN2z3LCEQ/sYaFdoQ9BifWpbYIiBljmudNN5qoMJUMYJZ0SUehsRHUCCATawn4T4d8cadKoIoxF",
    "RpgkRGYHeZsNUpxRQ1Mk0fjhkghipGOHtmLec4Lxtuxzt6ckQeE1A0LamAjJMJBOmmv+gg562BT",
    "nk4MOhHVRrC+Av68h/xOAIVGXc+i05AkUGsCMqN00D0j5O2ZpmyTzMsBgeEbVFkT8NYWX2Yubog",
    "CBPsIcSVTBInezqPcVGGN5KbTbuusxcVDI2zsjWi2CptvYI6liYautQ0SKc60rYnky2YQQ81eoT",
    "Q+phmdZynNYs3JMYYJfe1dVgBUPic1hLipCV0jUqANK+8zs2lv8jqdVGxi1MXn97e/7zPKwrDKg",
    "+oVDWBRRhGjBmaXirtYVMO+4RiFaZ1LwycJYhi1Ce7lvhH0k1xJjUNJFTFxyjlcBYD5BLrIR9jU",
    "VTB3NcLsJRQiMIXYNhHHpYNogtiYl2rW0L627EzNgHTRVLSli1WHXYhtouocYJ7IuhkEeio2Qc5",
    "D0+Cy8AZhtRgolBE/uXGndw9zIq3Dz71TudZ5c/Yp5N5IrPX2b0rJTo8WaexX0vNer3beCvlcn8",
    "nZG7UthntSDncGt4X//32ck/J69ev3tx7+6Hl+/eLrP/m8bZ5v/VfP3TpNFOC2c0l68O+3DGrtU",
    "S/Tdm9K/la033f3v65O7D28Ne9Ngf+t07G81k4Gf8LLC3UbB38n7ddXzMsjQ/Fe2G+cHErYnNxr",
    "XTvzFj74UF8biUv8+uvx8sJ//jZpHH1QXya/hrXMjbT99/fXM5+Wz/PxfYf6vA9610rh8Q1y3gg",
    "N7eONCchVjqB1QKoyTnoO6U1Oen5vtZ5bxzlvta7Pwu9/l5/5j5dd5y3jnLLevfWef4l4avmwde",
    "Ab/2Bfux7Pnqc+Pfe2c7B93M1/en/+q0IsbDklNv/hmPsP2yr99Anv4HhFm0Ew==",
    ""
  };

  static char newstr [1265] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c18_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c18_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2492201493U));
  ssSetChecksum1(S,(2946340087U));
  ssSetChecksum2(S,(892768965U));
  ssSetChecksum3(S,(3267729885U));
}

static void mdlRTW_c18_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c18_sliding_controller2(SimStruct *S)
{
  SFc18_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc18_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc18_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc18_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc18_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c18_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c18_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c18_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c18_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c18_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c18_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c18_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c18_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c18_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c18_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c18_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c18_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c18_JITStateAnimation,
    chartInstance->c18_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c18_sliding_controller2(chartInstance);
}

void c18_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c18_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c18_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c18_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c18_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
