/* Include files */

#include "sliding_controller2_sfun.h"
#include "c12_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void enable_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void disable_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void c12_do_animation_call_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance, const mxArray *c12_st);
static void sf_gateway_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance);
static void c12_eML_blk_kernel(SFc12_sliding_controller2InstanceStruct
  *chartInstance, real_T c12_b_q[2], real_T c12_b_y[4]);
static void c12_emlrt_marshallIn(SFc12_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c12_b_y, const char_T *c12_identifier, real_T
  c12_c_y[4]);
static void c12_b_emlrt_marshallIn(SFc12_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId,
  real_T c12_b_y[4]);
static uint8_T c12_c_emlrt_marshallIn(SFc12_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c12_b_is_active_c12_sliding_controller2, const
  char_T *c12_identifier);
static uint8_T c12_d_emlrt_marshallIn(SFc12_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId);
static void init_dsm_address_info(SFc12_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc12_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c12_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c12_is_active_c12_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c12_do_animation_call_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c12_b_y = NULL;
  const mxArray *c12_c_y = NULL;
  const mxArray *c12_d_y = NULL;
  const mxArray *c12_st;
  c12_st = NULL;
  c12_b_y = NULL;
  sf_mex_assign(&c12_b_y, sf_mex_createcellmatrix(2, 1), false);
  c12_c_y = NULL;
  sf_mex_assign(&c12_c_y, sf_mex_create("y", *chartInstance->c12_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c12_b_y, 0, c12_c_y);
  c12_d_y = NULL;
  sf_mex_assign(&c12_d_y, sf_mex_create("y",
    &chartInstance->c12_is_active_c12_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c12_b_y, 1, c12_d_y);
  sf_mex_assign(&c12_st, c12_b_y, false);
  return c12_st;
}

static void set_sim_state_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance, const mxArray *c12_st)
{
  const mxArray *c12_u;
  c12_u = sf_mex_dup(c12_st);
  c12_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c12_u, 0)), "y",
                       *chartInstance->c12_y);
  chartInstance->c12_is_active_c12_sliding_controller2 = c12_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c12_u, 1)),
     "is_active_c12_sliding_controller2");
  sf_mex_destroy(&c12_u);
  sf_mex_destroy(&c12_st);
}

static void sf_gateway_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
  real_T c12_dv1[4];
  real_T c12_dv[2];
  int32_T c12_i;
  int32_T c12_i1;
  chartInstance->c12_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c12_i = 0; c12_i < 2; c12_i++) {
    c12_dv[c12_i] = (*chartInstance->c12_q)[c12_i];
  }

  c12_eML_blk_kernel(chartInstance, c12_dv, c12_dv1);
  for (c12_i1 = 0; c12_i1 < 4; c12_i1++) {
    (*chartInstance->c12_y)[c12_i1] = c12_dv1[c12_i1];
  }

  c12_do_animation_call_c12_sliding_controller2(chartInstance);
}

static void mdl_start_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c12_errCode;
  c12_errCode = hipGetLastError();
  if (c12_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c12_errCode, (char_T *)hipGetErrorName
                       (c12_errCode), (char_T *)hipGetErrorString(c12_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c12_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc12_sliding_controller2
  (SFc12_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c12_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c12_nameCaptureInfo = NULL;
  c12_nameCaptureInfo = NULL;
  sf_mex_assign(&c12_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c12_nameCaptureInfo;
}

static void c12_eML_blk_kernel(SFc12_sliding_controller2InstanceStruct
  *chartInstance, real_T c12_b_q[2], real_T c12_b_y[4])
{
  const mxArray *c12_c_y = NULL;
  c12_b_y[0] = -0.324 * muDoubleScalarCos(c12_b_q[0]) + -0.39 *
    muDoubleScalarCos(c12_b_q[0] + c12_b_q[1]);
  c12_b_y[2] = -0.39 * muDoubleScalarCos(c12_b_q[0] + c12_b_q[1]);
  c12_b_y[1] = 0.324 * muDoubleScalarSin(c12_b_q[0]) - -0.39 * muDoubleScalarSin
    (c12_b_q[0] + c12_b_q[1]);
  c12_b_y[3] = 0.39 * muDoubleScalarSin(c12_b_q[0] + c12_b_q[1]);
  sf_mex_printf("%s =\\n", "y");
  c12_c_y = NULL;
  sf_mex_assign(&c12_c_y, sf_mex_create("y", c12_b_y, 0, 0U, 1U, 0U, 2, 2, 2),
                false);
  sf_mex_call(chartInstance->c12_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c12_c_y);
}

static void c12_emlrt_marshallIn(SFc12_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c12_b_y, const char_T *c12_identifier, real_T
  c12_c_y[4])
{
  emlrtMsgIdentifier c12_thisId;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c12_b_y), &c12_thisId,
    c12_c_y);
  sf_mex_destroy(&c12_b_y);
}

static void c12_b_emlrt_marshallIn(SFc12_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId,
  real_T c12_b_y[4])
{
  real_T c12_dv[4];
  int32_T c12_i;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), c12_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c12_i = 0; c12_i < 4; c12_i++) {
    c12_b_y[c12_i] = c12_dv[c12_i];
  }

  sf_mex_destroy(&c12_u);
}

static uint8_T c12_c_emlrt_marshallIn(SFc12_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c12_b_is_active_c12_sliding_controller2, const
  char_T *c12_identifier)
{
  emlrtMsgIdentifier c12_thisId;
  uint8_T c12_b_y;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_b_y = c12_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c12_b_is_active_c12_sliding_controller2), &c12_thisId);
  sf_mex_destroy(&c12_b_is_active_c12_sliding_controller2);
  return c12_b_y;
}

static uint8_T c12_d_emlrt_marshallIn(SFc12_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId)
{
  uint8_T c12_b_u;
  uint8_T c12_b_y;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), &c12_b_u, 1, 3, 0U, 0, 0U, 0);
  c12_b_y = c12_b_u;
  sf_mex_destroy(&c12_u);
  return c12_b_y;
}

static void init_dsm_address_info(SFc12_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc12_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c12_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c12_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c12_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c12_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3193823814U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(267299597U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(504614851U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(981118328U);
}

mxArray *sf_c12_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c12_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c12_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c12_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8cmGRvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABa+RvV"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c12_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sITpOBJPN6OXIDOpjmctU7";
}

static void sf_opaque_initialize_c12_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c12_sliding_controller2
    ((SFc12_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c12_sliding_controller2((SFc12_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c12_sliding_controller2(void *chartInstanceVar)
{
  enable_c12_sliding_controller2((SFc12_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c12_sliding_controller2(void *chartInstanceVar)
{
  disable_c12_sliding_controller2((SFc12_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c12_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c12_sliding_controller2((SFc12_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c12_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c12_sliding_controller2
    ((SFc12_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c12_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c12_sliding_controller2((SFc12_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c12_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc12_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c12_sliding_controller2
      ((SFc12_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc12_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc12_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c12_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c12_sliding_controller2((SFc12_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c12_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c12_sliding_controller2((SFc12_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc12_sliding_controller2((SFc12_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c12_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c12_sliding_controller2
      ((SFc12_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc12_sliding_controller2
      ((SFc12_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c12_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV82O00gQ9kSzCCRAIxZxQmI4oOW4ywE4wZA/YTRDRjjDclg09LQrcZN2t+mfZCIh8QS8xp7",
    "3BfYh9saJwz7BPsJW204m69gOQ8QI0FryOG1/VV1VX/30eBv+nofXZbx3rnjeOXyex7vhZdcP+X",
    "pj4c7eb3o/5ev3KCRsvE8UibVXewkSwzPQklvDpPDFQJbCmBiAAkERm0hlqrRpFlvOxKhrBXX69",
    "K8Ro1EQScvDJsqSsCf4FLUl1uyjnjZTQE0XIDSRknYYdTkZzi1WZtKKgI60jetc0GACmziz9J7l",
    "hiUcOsdAfaENQYv1iW2BIQZa5rjSTeepDmZAGSecEVHqbUR0AAkG2MBBEuLfnjXoVBFGI6JMEyI",
    "yBr3LRqlOKaCok2n8cMQEMVIxwjsxbznBZdv2OdqzJ0PgNQFB25oKyCiRTJhq/oMuetoR5IhDG4",
    "7ssFpbAG+sI/85gwmoyrgNWnIMigyhJyo3TQPSOU7ZmmfJMsywGJ4T9YgifxrCyuzFzNEBQZ6gj",
    "xJVMEid9HVfsTGGt1KbjX2XmatKxsYZ2XoVLNXWGUMdC3NtXSpahHNdCevLZBfGwFOtbWJIPSzT",
    "Wo7TmoV9iQF26V1dDVYwJD6HtaQIWSld4wIg7TtPsbH8F0mtNjJuYfK2d3eXPy/DfGFADQiFsi6",
    "gCNOAMUvDW60tZNpxj0C0yqTmlYGzDFmF8vTAivZEqhHGpKaJnLjgGK0ExnqIXGIlHGgsmjqY43",
    "IVjhIaQegaDOOwh2WD2JKYaNfaHmHdjZmZtkFTxZISVi1WHbahjkuoaQIHYiTkRHSVjIO8x2fhB",
    "cBsI0owMWxi/1LTLu5eRoWbZz97J/Ps4ifMs5lc8Xl7Qc9GiR5v4Vnc90Kjft8G/tqYzd8FuUuF",
    "fTYLcg63hfeff7/97Xjy48uLf/3+7uP2rVfr7P9H43Tz/3K+vj5rtPPCGS/lq8M+XrBrs0T/tQX",
    "9W/la+/2k13yy//Ru74Xf7iWvY2oO7uX8rLC3UbB39n7bdXzMsjQ/FfXD/GDi1sRm49rpv79g77",
    "kV8biQv8+ufx6uJ391p8jj5gr5Lfw1LeTt5++/vbOefLb/ixX23yzwfTOd64fEdQs4pL/cOdSch",
    "Vjqh1QKoyTnoO6U1Ofn5vtp5bwzlvtW7Pxf7svz/inz66zlvDOWW9e/087xrw1fNw+8An7rK/Zj",
    "3fPVl8Z/8E53DrqRrx/M/9VpRYyHJafe/DMeYQdlX7+DPP0XvxGxng==",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c12_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c12_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1551688381U));
  ssSetChecksum1(S,(1561556856U));
  ssSetChecksum2(S,(2141638924U));
  ssSetChecksum3(S,(1613045982U));
}

static void mdlRTW_c12_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c12_sliding_controller2(SimStruct *S)
{
  SFc12_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc12_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc12_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc12_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc12_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c12_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c12_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c12_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c12_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c12_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c12_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c12_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c12_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c12_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c12_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c12_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c12_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c12_JITStateAnimation,
    chartInstance->c12_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c12_sliding_controller2(chartInstance);
}

void c12_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c12_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c12_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c12_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c12_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
