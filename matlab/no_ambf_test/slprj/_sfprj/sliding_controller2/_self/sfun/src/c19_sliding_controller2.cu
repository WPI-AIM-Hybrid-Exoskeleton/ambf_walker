/* Include files */

#include "sliding_controller2_sfun.h"
#include "c19_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void enable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void disable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void c19_do_animation_call_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance, const mxArray *c19_st);
static void sf_gateway_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void c19_eML_blk_kernel(SFc19_sliding_controller2InstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2]);
static void c19_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2]);
static void c19_b_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2]);
static uint8_T c19_c_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_is_active_c19_sliding_controller2, const
  char_T *c19_identifier);
static uint8_T c19_d_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId);
static void init_dsm_address_info(SFc19_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc19_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c19_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c19_is_active_c19_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c19_do_animation_call_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c19_b_y = NULL;
  const mxArray *c19_c_y = NULL;
  const mxArray *c19_d_y = NULL;
  const mxArray *c19_e_y = NULL;
  const mxArray *c19_f_y = NULL;
  const mxArray *c19_st;
  const mxArray *c19_y = NULL;
  c19_st = NULL;
  c19_y = NULL;
  sf_mex_assign(&c19_y, sf_mex_createcellmatrix(5, 1), false);
  c19_b_y = NULL;
  sf_mex_assign(&c19_b_y, sf_mex_create("y", *chartInstance->c19_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 0, c19_b_y);
  c19_c_y = NULL;
  sf_mex_assign(&c19_c_y, sf_mex_create("y", *chartInstance->c19_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 1, c19_c_y);
  c19_d_y = NULL;
  sf_mex_assign(&c19_d_y, sf_mex_create("y", *chartInstance->c19_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 2, c19_d_y);
  c19_e_y = NULL;
  sf_mex_assign(&c19_e_y, sf_mex_create("y", *chartInstance->c19_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c19_y, 3, c19_e_y);
  c19_f_y = NULL;
  sf_mex_assign(&c19_f_y, sf_mex_create("y",
    &chartInstance->c19_is_active_c19_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c19_y, 4, c19_f_y);
  sf_mex_assign(&c19_st, c19_y, false);
  return c19_st;
}

static void set_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance, const mxArray *c19_st)
{
  const mxArray *c19_u;
  c19_u = sf_mex_dup(c19_st);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 0)),
                       "ankle", *chartInstance->c19_ankle);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 1)),
                       "danke", *chartInstance->c19_danke);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 2)),
                       "dknee", *chartInstance->c19_dknee);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 3)),
                       "knee", *chartInstance->c19_knee);
  chartInstance->c19_is_active_c19_sliding_controller2 = c19_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 4)),
     "is_active_c19_sliding_controller2");
  sf_mex_destroy(&c19_u);
  sf_mex_destroy(&c19_st);
}

static void sf_gateway_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c19_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c19_eML_blk_kernel(chartInstance, *chartInstance->c19_q,
                     *chartInstance->c19_qd, *chartInstance->c19_knee,
                     *chartInstance->c19_dknee, *chartInstance->c19_ankle,
                     *chartInstance->c19_danke);
  c19_do_animation_call_c19_sliding_controller2(chartInstance);
}

static void mdl_start_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c19_errCode;
  c19_errCode = hipGetLastError();
  if (c19_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c19_errCode, (char_T *)hipGetErrorName
                       (c19_errCode), (char_T *)hipGetErrorString(c19_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c19_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c19_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c19_nameCaptureInfo = NULL;
  const char_T *c19_data[4] = {
    "789cc593cd4ac34014856fa516375537fa188e2d08dd5745118bd0e246244e929b66ccfcc8646a5b372ebbf4b1f475dcb8706592364d1b0c118bf542b8393999"
    "f9ee1c18a89c5f5600603b7a3e9e01ce6a90547dda6077d63760b9f27ea5a0a7b509d5a575b1ff1ef15e66da51d2e0c84c85a402e72b5d2598a4d2f4c60f081a",
    "43c51fd14d1c8f71ec3181dd45d18995385db0e622b6e2f7b68f4ed01d08d07e984dc8174592475c7705e7ad96e491af7c1ef9ff52dee897bc74fffd125eea7b"
    "4a0fa976ad804926a8614e6871ecaf7eee945b2b9c63eab86a6073cc786f2bf268216fd9bf39b925be124824353e950c795f71d7a3da260e35511ad63024a176",
    "0815b6670d290f509328204e6d2295957c35181af27d800762c629cb71e787e72aba4f75d84a7a6bef88ae9337f97c6dae9397d67ff1d6751f2f0ec7acf1c4ee"
    "83a0d96a771ab6ed5cbbe6389be3aa8453360714e8bfdeff0b89616e21", "" };

  c19_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c19_data[0], 1632U, &c19_nameCaptureInfo);
  return c19_nameCaptureInfo;
}

static void c19_eML_blk_kernel(SFc19_sliding_controller2InstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2])
{
  real_T c19_a;
  real_T c19_b_a;
  c19_b_knee[0] = -0.324 * muDoubleScalarSin(c19_b_q[0]);
  c19_b_knee[1] = -0.324 * muDoubleScalarCos(c19_b_q[0]);
  c19_a = -0.324 * c19_b_qd[0];
  c19_b_dknee[0] = c19_a * muDoubleScalarCos(c19_b_q[0]);
  c19_b_dknee[1] = c19_a * -muDoubleScalarSin(c19_b_q[0]);
  c19_b_a = -0.39 * (c19_b_qd[0] + c19_b_qd[1]);
  c19_b_ankle[0] = c19_b_knee[0] + -0.39 * muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_ankle[1] = c19_b_knee[1] + -0.39 * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[0] = c19_b_dknee[0] + c19_b_a * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[1] = c19_b_dknee[1] + c19_b_a * -muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
}

static void c19_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2])
{
  emlrtMsgIdentifier c19_thisId;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c19_b_ankle), &c19_thisId,
    c19_y);
  sf_mex_destroy(&c19_b_ankle);
}

static void c19_b_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2])
{
  real_T c19_dv[2];
  int32_T c19_i;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), c19_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c19_i = 0; c19_i < 2; c19_i++) {
    c19_y[c19_i] = c19_dv[c19_i];
  }

  sf_mex_destroy(&c19_u);
}

static uint8_T c19_c_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_is_active_c19_sliding_controller2, const
  char_T *c19_identifier)
{
  emlrtMsgIdentifier c19_thisId;
  uint8_T c19_y;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_y = c19_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c19_b_is_active_c19_sliding_controller2), &c19_thisId);
  sf_mex_destroy(&c19_b_is_active_c19_sliding_controller2);
  return c19_y;
}

static uint8_T c19_d_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId)
{
  uint8_T c19_b_u;
  uint8_T c19_y;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), &c19_b_u, 1, 3, 0U, 0, 0U, 0);
  c19_y = c19_b_u;
  sf_mex_destroy(&c19_u);
  return c19_y;
}

static void init_dsm_address_info(SFc19_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc19_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c19_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c19_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c19_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c19_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c19_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c19_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c19_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c19_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c19_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c19_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6LFA4ifWRyfmFySWZYan"
    "2xoGV+ck5mSmZcen5yfV1KUn5OTWmSEZD4IAACHxSy4"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c19_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c19_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c19_sliding_controller2
    ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c19_sliding_controller2(void *chartInstanceVar)
{
  enable_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c19_sliding_controller2(void *chartInstanceVar)
{
  disable_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c19_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c19_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c19_sliding_controller2
    ((SFc19_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c19_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c19_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c19_sliding_controller2
      ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc19_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c19_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c19_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c19_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c19_sliding_controller2
      ((SFc19_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc19_sliding_controller2
      ((SFc19_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c19_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA08CFALwGa9JIeUsBtUaBBgNQ2JSFC7dgp5TTIxVgvR+JCy112f2g77xD",
    "k2vcoeumLNOc8Qd4gnSVpWaZIKrKb1C1Cg5KX/Obb+WZmZ1dea7Dt4XUD799uet4V/L6Kd9vLr5",
    "Vi3Jq58+cd724xfolGwsa7RJFYe42XIDH8DFpya5gUAzGSlTAmRqBAUMQmUpk6Ns1iy5mY9K2gj",
    "k//EjEaBZG0PNxEWxLuCH6MbIk1u8jTZQqo6QOEJlLSjqM+J+Opx8oc+hHQibZxkwQNJrCJc0tv",
    "W25YwqF3BHQgtCHosT71LTDEgG+OamU6pTo4Aco44YyISrUR0QEkGGADe0mInzvWoKgyjEZEmU2",
    "ISAp6i00yTimgzMk0vjhgghipGOG9mPvOcN63XY7+bMsQeENA0LdNBWSSSCZMff6DPirtCXLAoQ",
    "sHdlzPFsCv1iX/KYNDULVxG/kyBUXGsCNqJ80C0jvKsjWtknmYYTE8JWqDYv40hLXVi5WjA4J5g",
    "iFa1MEgEznQQ8VSDG8tm40HrjIXLRkb58nWi2AZWy+FpixM2fpU+IRzXQsbymQLUuAZa5cY0gzL",
    "WatxWrNwKDHArrzrV4MVDBNfwHwpQlaZrrQEyPrOY2wsZ5HUaiNjH4u3u7U1/3oeNhAG1IhQqOo",
    "CijANGLMsvPVsIdMu9whEr0zmXhU4r5BFKE+PrOgeSjXBmDQ0kVMJLqO1wFiPMZe4EvY0LpommM",
    "vlIhwlNILQNRjGYRuXDWIrYqJda9vAdZcyc9wFTRVLKrJqcdVhG+q5gjpOYE9MhDwUfSXjoOjxe",
    "XgBsNqIEkyMN7F/qeM+zl6VCrefrXmn+9n199jPTuzK31/N8LQqeLyZ7/K819pn8e3SvG38a7Vy",
    "u/UZu89K83RKdg63iveL35//8dNff776+u6be6+GD94tmr81N38r+9/ZXV1Zbv+/UYxvnTTa6cJ",
    "J5+rVYR/N+NWp4P98hn+1GOsn0fciff4o2nhChvFg7bsX99fSAz/je9Zp9nel5O/J89uu42OVZf",
    "Wp6CAsDiZuTGy+XTv+H2b8vbIgHteK5/n19seL2d9aL9t3FuhbLcZETDiU6vff0/HluXWEKOTy6",
    "Lh9fh0TPDRcGh0318v9oUpH54yOjuck/DP+z8dxOft8/mcL/L9TysOd7Ly4T9wuBPv0m/v7mrMQ",
    "t5B9KoVRknNQ31b0/fP20WXtvI9s91/x85Pdh8/7+5yL2ue061zwHPax7C6qb9nz4WXDN+0HXgm",
    "/eol1XPTc/qHxr73lztdfFOOH05/QfsR4WPFrqniNP41GVW//B3X6N/M9w18=",
    ""
  };

  static char newstr [1337] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c19_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c19_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042620282U));
  ssSetChecksum1(S,(2411386699U));
  ssSetChecksum2(S,(753018669U));
  ssSetChecksum3(S,(4282078351U));
}

static void mdlRTW_c19_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c19_sliding_controller2(SimStruct *S)
{
  SFc19_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc19_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc19_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc19_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc19_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c19_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c19_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c19_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c19_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c19_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c19_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c19_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c19_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c19_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c19_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c19_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c19_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c19_JITStateAnimation,
    chartInstance->c19_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c19_sliding_controller2(chartInstance);
}

void c19_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c19_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c19_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c19_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c19_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
