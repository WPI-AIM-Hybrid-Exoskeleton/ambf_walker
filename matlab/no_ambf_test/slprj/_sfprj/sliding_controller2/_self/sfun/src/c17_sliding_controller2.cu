#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c17_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#include "MWGpuFunctionImplUtility.hpp"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include "MWCUSOLVERUtils.hpp"
#include "hip/hip_math_constants.h"

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void enable_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void disable_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void c17_do_animation_call_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance, const mxArray *c17_st);
static void sf_gateway_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void c17_eML_blk_kernel(SFc17_sliding_controller2InstanceStruct
  *chartInstance, real_T c17_b_Kp0[6], real_T c17_b_Kn0[6], real_T c17_b_Bp0[6],
  real_T c17_b_Bn0[6], real_T c17_b_beta[2], real_T c17_b_alpha[2], real_T
  c17_b_Tid[6], real_T c17_b_Th[6], real_T c17_b_qd[6], real_T c17_b_K[36],
  real_T c17_b_B[36]);
static void c17_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_b_B, const char_T *c17_identifier, real_T
  c17_y[36]);
static void c17_b_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId,
  real_T c17_y[36]);
static uint8_T c17_c_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_b_is_active_c17_sliding_controller2, const
  char_T *c17_identifier);
static uint8_T c17_d_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId);
static __global__ void c17_eML_blk_kernel_kernel1(real_T c17_b_Kp0[36]);
static __global__ void c17_eML_blk_kernel_kernel2(const real_T c17_b_Kp0[6],
  real_T c17_c_Kp0[36]);
static __global__ void c17_eML_blk_kernel_kernel3(real_T c17_b_Kn0[36]);
static __global__ void c17_eML_blk_kernel_kernel4(const real_T c17_b_Kn0[6],
  real_T c17_c_Kn0[36]);
static __global__ void c17_eML_blk_kernel_kernel5(real_T c17_b_Bp0[36]);
static __global__ void c17_eML_blk_kernel_kernel6(const real_T c17_b_Bp0[6],
  real_T c17_c_Bp0[36]);
static __global__ void c17_eML_blk_kernel_kernel7(real_T c17_b_Bn0[36]);
static __global__ void c17_eML_blk_kernel_kernel8(const real_T c17_b_Tid[6],
  const real_T c17_b_qd[6], const real_T c17_b_Th[6], const real_T c17_b_Bn0[6],
  ptrdiff_t c17_jpvt_t[6], real_T c17_A[6], real_T c17_x[6], real_T c17_b_x[6],
  real_T c17_c_Bn0[36]);
static __global__ void c17_eML_blk_kernel_kernel9(const ptrdiff_t c17_jpvt_t[6],
  int32_T c17_jpvt[6]);
static __global__ void c17_eML_blk_kernel_kernel10(int32_T c17_jpvt[6], real_T
  c17_A[6]);
static __global__ void c17_eML_blk_kernel_kernel11(real_T c17_change[36]);
static __global__ void c17_eML_blk_kernel_kernel12(const real_T c17_b_Th[6],
  real_T c17_b_B[6]);
static __global__ void c17_eML_blk_kernel_kernel13(real_T c17_b_B[6]);
static __global__ void c17_eML_blk_kernel_kernel14(const int32_T c17_jpvt[6],
  const int32_T c17_rankA, real_T c17_A[6], real_T c17_b_B[6], real_T
  c17_change[36]);
static __global__ void c17_eML_blk_kernel_kernel15(const real_T c17_change[36],
  real_T c17_Y[36]);
static __global__ void c17_eML_blk_kernel_kernel16(const real_T c17_Y[36],
  real_T c17_b_K[36], real_T c17_change[36]);
static __global__ void c17_eML_blk_kernel_kernel17(real_T c17_b_K[36]);
static __global__ void c17_eML_blk_kernel_kernel18(real_T c17_b_B[36]);
static __global__ void c17_eML_blk_kernel_kernel19(const real_T c17_b_Bn0[36],
  const real_T c17_b_alpha[2], const real_T c17_b_Bp0[36], const real_T
  c17_b_Kp0[36], const real_T c17_change[36], const real_T c17_b_beta[2], const
  real_T c17_b_Kn0[36], const real_T c17_x[6], const real_T c17_b_x[6], real_T
  c17_b_K[36], real_T c17_b_B[36]);
static void init_dsm_address_info(SFc17_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc17_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c17_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c17_is_active_c17_sliding_controller2 = 0U;
  cusolverEnsureInitialization();
  hipGetLastError();
  hipMalloc(&chartInstance->c17_gpu_beta, 16UL);
  hipMalloc(&chartInstance->c17_gpu_alpha, 16UL);
  hipMalloc(&chartInstance->c17_gpu_B, 288UL);
  hipMalloc(&chartInstance->c17_gpu_K, 288UL);
  hipMalloc(&chartInstance->c17_gpu_Y, 288UL);
  hipMalloc(&chartInstance->c17_gpu_info_t, 4UL);
  hipMalloc(&chartInstance->c17_gpu_tau, 8UL);
  hipMalloc(&chartInstance->c17_b_gpu_B, 48UL);
  hipMalloc(&chartInstance->c17_gpu_change, 288UL);
  hipMalloc(&chartInstance->c17_gpu_jpvt, 24UL);
  hipMalloc(&chartInstance->c17_gpu_x, 48UL);
  hipMalloc(&chartInstance->c17_b_gpu_x, 48UL);
  hipMalloc(&chartInstance->c17_gpu_A, 48UL);
  hipMalloc(&chartInstance->c17_gpu_jpvt_t, 48UL);
  hipMalloc(&chartInstance->c17_gpu_Bn0, 48UL);
  hipMalloc(&chartInstance->c17_gpu_Th, 48UL);
  hipMalloc(&chartInstance->c17_gpu_qd, 48UL);
  hipMalloc(&chartInstance->c17_gpu_Tid, 48UL);
  hipMalloc(&chartInstance->c17_b_gpu_Bn0, 288UL);
  hipMalloc(&chartInstance->c17_gpu_Bp0, 48UL);
  hipMalloc(&chartInstance->c17_b_gpu_Bp0, 288UL);
  hipMalloc(&chartInstance->c17_gpu_Kn0, 48UL);
  hipMalloc(&chartInstance->c17_b_gpu_Kn0, 288UL);
  hipMalloc(&chartInstance->c17_gpu_Kp0, 48UL);
  hipMalloc(&chartInstance->c17_b_gpu_Kp0, 288UL);
}

static void initialize_params_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c17_do_animation_call_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c17_b_y = NULL;
  const mxArray *c17_c_y = NULL;
  const mxArray *c17_d_y = NULL;
  const mxArray *c17_st;
  const mxArray *c17_y = NULL;
  c17_st = NULL;
  c17_y = NULL;
  sf_mex_assign(&c17_y, sf_mex_createcellmatrix(3, 1), false);
  c17_b_y = NULL;
  sf_mex_assign(&c17_b_y, sf_mex_create("y", *chartInstance->c17_B, 0, 0U, 1U,
    0U, 2, 6, 6), false);
  sf_mex_setcell(c17_y, 0, c17_b_y);
  c17_c_y = NULL;
  sf_mex_assign(&c17_c_y, sf_mex_create("y", *chartInstance->c17_K, 0, 0U, 1U,
    0U, 2, 6, 6), false);
  sf_mex_setcell(c17_y, 1, c17_c_y);
  c17_d_y = NULL;
  sf_mex_assign(&c17_d_y, sf_mex_create("y",
    &chartInstance->c17_is_active_c17_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c17_y, 2, c17_d_y);
  sf_mex_assign(&c17_st, c17_y, false);
  return c17_st;
}

static void set_sim_state_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance, const mxArray *c17_st)
{
  const mxArray *c17_u;
  c17_u = sf_mex_dup(c17_st);
  c17_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 0)), "B",
                       *chartInstance->c17_B);
  c17_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 1)), "K",
                       *chartInstance->c17_K);
  chartInstance->c17_is_active_c17_sliding_controller2 = c17_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 2)),
     "is_active_c17_sliding_controller2");
  sf_mex_destroy(&c17_u);
  sf_mex_destroy(&c17_st);
}

static void sf_gateway_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  real_T c17_dv10[36];
  real_T c17_dv9[36];
  real_T c17_dv[6];
  real_T c17_dv1[6];
  real_T c17_dv2[6];
  real_T c17_dv3[6];
  real_T c17_dv6[6];
  real_T c17_dv7[6];
  real_T c17_dv8[6];
  real_T c17_dv4[2];
  real_T c17_dv5[2];
  int32_T c17_i;
  int32_T c17_i1;
  int32_T c17_i10;
  int32_T c17_i2;
  int32_T c17_i3;
  int32_T c17_i4;
  int32_T c17_i5;
  int32_T c17_i6;
  int32_T c17_i7;
  int32_T c17_i8;
  int32_T c17_i9;
  chartInstance->c17_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c17_i = 0; c17_i < 6; c17_i++) {
    c17_dv[c17_i] = (*chartInstance->c17_Kp0)[c17_i];
  }

  for (c17_i1 = 0; c17_i1 < 6; c17_i1++) {
    c17_dv1[c17_i1] = (*chartInstance->c17_Kn0)[c17_i1];
  }

  for (c17_i2 = 0; c17_i2 < 6; c17_i2++) {
    c17_dv2[c17_i2] = (*chartInstance->c17_Bp0)[c17_i2];
  }

  for (c17_i3 = 0; c17_i3 < 6; c17_i3++) {
    c17_dv3[c17_i3] = (*chartInstance->c17_Bn0)[c17_i3];
  }

  for (c17_i4 = 0; c17_i4 < 2; c17_i4++) {
    c17_dv4[c17_i4] = (*chartInstance->c17_beta)[c17_i4];
  }

  for (c17_i5 = 0; c17_i5 < 2; c17_i5++) {
    c17_dv5[c17_i5] = (*chartInstance->c17_alpha)[c17_i5];
  }

  for (c17_i6 = 0; c17_i6 < 6; c17_i6++) {
    c17_dv6[c17_i6] = (*chartInstance->c17_Tid)[c17_i6];
  }

  for (c17_i7 = 0; c17_i7 < 6; c17_i7++) {
    c17_dv7[c17_i7] = (*chartInstance->c17_Th)[c17_i7];
  }

  for (c17_i8 = 0; c17_i8 < 6; c17_i8++) {
    c17_dv8[c17_i8] = (*chartInstance->c17_qd)[c17_i8];
  }

  c17_eML_blk_kernel(chartInstance, c17_dv, c17_dv1, c17_dv2, c17_dv3, c17_dv4,
                     c17_dv5, c17_dv6, c17_dv7, c17_dv8, c17_dv9, c17_dv10);
  for (c17_i9 = 0; c17_i9 < 36; c17_i9++) {
    (*chartInstance->c17_B)[c17_i9] = c17_dv10[c17_i9];
  }

  for (c17_i10 = 0; c17_i10 < 36; c17_i10++) {
    (*chartInstance->c17_K)[c17_i10] = c17_dv9[c17_i10];
  }

  c17_do_animation_call_c17_sliding_controller2(chartInstance);
}

static void mdl_start_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c17_errCode;
  hipFree(*chartInstance->c17_gpu_beta);
  hipFree(*chartInstance->c17_gpu_alpha);
  hipFree(*chartInstance->c17_gpu_B);
  hipFree(*chartInstance->c17_gpu_K);
  hipFree(*chartInstance->c17_gpu_Y);
  hipFree(chartInstance->c17_gpu_info_t);
  hipFree(chartInstance->c17_gpu_tau);
  hipFree(*chartInstance->c17_b_gpu_B);
  hipFree(*chartInstance->c17_gpu_change);
  hipFree(*chartInstance->c17_gpu_jpvt);
  hipFree(*chartInstance->c17_gpu_x);
  hipFree(*chartInstance->c17_b_gpu_x);
  hipFree(*chartInstance->c17_gpu_A);
  hipFree(*chartInstance->c17_gpu_jpvt_t);
  hipFree(*chartInstance->c17_gpu_Bn0);
  hipFree(*chartInstance->c17_gpu_Th);
  hipFree(*chartInstance->c17_gpu_qd);
  hipFree(*chartInstance->c17_gpu_Tid);
  hipFree(*chartInstance->c17_b_gpu_Bn0);
  hipFree(*chartInstance->c17_gpu_Bp0);
  hipFree(*chartInstance->c17_b_gpu_Bp0);
  hipFree(*chartInstance->c17_gpu_Kn0);
  hipFree(*chartInstance->c17_b_gpu_Kn0);
  hipFree(*chartInstance->c17_gpu_Kp0);
  hipFree(*chartInstance->c17_b_gpu_Kp0);
  c17_errCode = hipGetLastError();
  if (c17_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c17_errCode, (char_T *)hipGetErrorName
                       (c17_errCode), (char_T *)hipGetErrorString(c17_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c17_fEmlrtCtx);
  }

  cusolverDestroyWorkspace();
  cusolverEnsureDestruction();
}

static void mdl_setup_runtime_resources_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c17_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c17_nameCaptureInfo = NULL;
  c17_nameCaptureInfo = NULL;
  sf_mex_assign(&c17_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c17_nameCaptureInfo;
}

static void c17_eML_blk_kernel(SFc17_sliding_controller2InstanceStruct
  *chartInstance, real_T c17_b_Kp0[6], real_T c17_b_Kn0[6], real_T c17_b_Bp0[6],
  real_T c17_b_Bn0[6], real_T c17_b_beta[2], real_T c17_b_alpha[2], real_T
  c17_b_Tid[6], real_T c17_b_Th[6], real_T c17_b_qd[6], real_T c17_b_K[36],
  real_T c17_b_B[36])
{
  ptrdiff_t c17_jpvt_t[6];
  ptrdiff_t c17_info_t;
  real_T c17_A[6];
  real_T c17_tau;
  real_T c17_tol;
  int32_T c17_b_info_t;
  int32_T c17_rankA;
  boolean_T c17_A_dirtyOnCpu;
  boolean_T c17_A_dirtyOnGpu;
  c17_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Kp0);
  hipMemcpy(*chartInstance->c17_gpu_Kp0, c17_b_Kp0, 48UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Kp0, *chartInstance->c17_b_gpu_Kp0);
  c17_eML_blk_kernel_kernel3<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Kn0);
  hipMemcpy(*chartInstance->c17_gpu_Kn0, c17_b_Kn0, 48UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Kn0, *chartInstance->c17_b_gpu_Kn0);
  c17_eML_blk_kernel_kernel5<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Bp0);
  hipMemcpy(*chartInstance->c17_gpu_Bp0, c17_b_Bp0, 48UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Bp0, *chartInstance->c17_b_gpu_Bp0);
  c17_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Bn0);
  hipMemcpy(*chartInstance->c17_gpu_Tid, c17_b_Tid, 48UL,
             hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_qd, c17_b_qd, 48UL, hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_Th, c17_b_Th, 48UL, hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_Bn0, c17_b_Bn0, 48UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Tid, *chartInstance->c17_gpu_qd,
     *chartInstance->c17_gpu_Th, *chartInstance->c17_gpu_Bn0,
     *chartInstance->c17_gpu_jpvt_t, *chartInstance->c17_gpu_A,
     *chartInstance->c17_b_gpu_x, *chartInstance->c17_gpu_x,
     *chartInstance->c17_b_gpu_Bn0);
  hipMemcpy(c17_A, *chartInstance->c17_gpu_A, 48UL, hipMemcpyDeviceToHost);
  hipMemcpy(c17_jpvt_t, *chartInstance->c17_gpu_jpvt_t, 48UL,
             hipMemcpyDeviceToHost);
  c17_info_t = LAPACKE_dgeqp3(102, (ptrdiff_t)1, (ptrdiff_t)6, &c17_A[0],
    (ptrdiff_t)1, &c17_jpvt_t[0], &c17_tau);
  c17_A_dirtyOnGpu = false;
  c17_A_dirtyOnCpu = true;
  if ((int32_T)c17_info_t != 0) {
    c17_tau = rtNaN;
    c17_eML_blk_kernel_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_gpu_jpvt, *chartInstance->c17_gpu_A);
    c17_A_dirtyOnCpu = false;
    c17_A_dirtyOnGpu = true;
  } else {
    hipMemcpy(*chartInstance->c17_gpu_jpvt_t, c17_jpvt_t, 48UL,
               hipMemcpyHostToDevice);
    c17_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_gpu_jpvt_t, *chartInstance->c17_gpu_jpvt);
  }

  c17_rankA = 0;
  if (c17_A_dirtyOnGpu) {
    hipMemcpy(c17_A, *chartInstance->c17_gpu_A, 48UL, hipMemcpyDeviceToHost);
  }

  c17_tol = 1.3322676295501878E-14 * muDoubleScalarAbs(c17_A[0]);
  if (!(muDoubleScalarAbs(c17_A[0]) <= c17_tol)) {
    c17_rankA = 1;
  }

  c17_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Th, *chartInstance->c17_b_gpu_B);
  c17_b_info_t = 0;
  if (c17_A_dirtyOnCpu) {
    hipMemcpy(*chartInstance->c17_gpu_A, c17_A, 48UL, hipMemcpyHostToDevice);
  }

  hipMemcpy(chartInstance->c17_gpu_tau, &c17_tau, 8UL, hipMemcpyHostToDevice);
  hipsolverDnDormqr_bufferSize(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT,
    HIPBLAS_OP_T, 1, 6, 1, (double *)&(*chartInstance->c17_gpu_A)[0], 1, (double *)
    chartInstance->c17_gpu_tau, (double *)&(*chartInstance->c17_b_gpu_B)[0], 1,
    getCuSolverWorkspaceReq());
  setCuSolverWorkspaceTypeSize(8);
  cusolverInitWorkspace();
  hipMemcpy(chartInstance->c17_gpu_info_t, &c17_b_info_t, 4UL,
             hipMemcpyHostToDevice);
  hipsolverDnDormqr(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, 1,
                   6, 1, (double *)&(*chartInstance->c17_gpu_A)[0], 1, (double *)
                   chartInstance->c17_gpu_tau, (double *)
                   &(*chartInstance->c17_b_gpu_B)[0], 1, (real_T *)
                   getCuSolverWorkspaceBuff(), *getCuSolverWorkspaceReq(),
                   chartInstance->c17_gpu_info_t);
  hipMemcpy(&c17_b_info_t, chartInstance->c17_gpu_info_t, 4UL,
             hipMemcpyDeviceToHost);
  if (c17_b_info_t < 0) {
    c17_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_b_gpu_B);
  }

  c17_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_jpvt, c17_rankA, *chartInstance->c17_gpu_A,
     *chartInstance->c17_b_gpu_B, *chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel15<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_change, *chartInstance->c17_gpu_Y);
  c17_eML_blk_kernel_kernel16<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Y, *chartInstance->c17_gpu_K,
     *chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_K);
  c17_eML_blk_kernel_kernel18<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_B);
  hipMemcpy(*chartInstance->c17_gpu_alpha, c17_b_alpha, 16UL,
             hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_beta, c17_b_beta, 16UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Bn0, *chartInstance->c17_gpu_alpha,
     *chartInstance->c17_b_gpu_Bp0, *chartInstance->c17_b_gpu_Kp0,
     *chartInstance->c17_gpu_change, *chartInstance->c17_gpu_beta,
     *chartInstance->c17_b_gpu_Kn0, *chartInstance->c17_b_gpu_x,
     *chartInstance->c17_gpu_x, *chartInstance->c17_gpu_K,
     *chartInstance->c17_gpu_B);
  hipMemcpy(c17_b_K, *chartInstance->c17_gpu_K, 288UL, hipMemcpyDeviceToHost);
  hipMemcpy(c17_b_B, *chartInstance->c17_gpu_B, 288UL, hipMemcpyDeviceToHost);
}

static void c17_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_b_B, const char_T *c17_identifier, real_T
  c17_y[36])
{
  emlrtMsgIdentifier c17_thisId;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c17_b_B), &c17_thisId, c17_y);
  sf_mex_destroy(&c17_b_B);
}

static void c17_b_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId,
  real_T c17_y[36])
{
  real_T c17_dv[36];
  int32_T c17_i;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), c17_dv, 1, 0, 0U, 1, 0U, 2, 6,
                6);
  for (c17_i = 0; c17_i < 36; c17_i++) {
    c17_y[c17_i] = c17_dv[c17_i];
  }

  sf_mex_destroy(&c17_u);
}

static uint8_T c17_c_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_b_is_active_c17_sliding_controller2, const
  char_T *c17_identifier)
{
  emlrtMsgIdentifier c17_thisId;
  uint8_T c17_y;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_y = c17_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c17_b_is_active_c17_sliding_controller2), &c17_thisId);
  sf_mex_destroy(&c17_b_is_active_c17_sliding_controller2);
  return c17_y;
}

static uint8_T c17_d_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId)
{
  uint8_T c17_b_u;
  uint8_T c17_y;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), &c17_b_u, 1, 3, 0U, 0, 0U, 0);
  c17_y = c17_b_u;
  sf_mex_destroy(&c17_u);
  return c17_y;
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel1
  (real_T c17_b_Kp0[36])
{
  uint64_T c17_threadId;
  int32_T c17_i;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i = (int32_T)c17_threadId;
  if (c17_i < 36) {
    c17_b_Kp0[c17_i] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel2(const
  real_T c17_b_Kp0[6], real_T c17_c_Kp0[36])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 6) {
    c17_c_Kp0[c17_j + 6 * c17_j] = c17_b_Kp0[c17_j];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel3
  (real_T c17_b_Kn0[36])
{
  uint64_T c17_threadId;
  int32_T c17_i1;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i1 = (int32_T)c17_threadId;
  if (c17_i1 < 36) {
    c17_b_Kn0[c17_i1] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel4(const
  real_T c17_b_Kn0[6], real_T c17_c_Kn0[36])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 6) {
    c17_c_Kn0[c17_j + 6 * c17_j] = c17_b_Kn0[c17_j];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel5
  (real_T c17_b_Bp0[36])
{
  uint64_T c17_threadId;
  int32_T c17_i2;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i2 = (int32_T)c17_threadId;
  if (c17_i2 < 36) {
    c17_b_Bp0[c17_i2] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel6(const
  real_T c17_b_Bp0[6], real_T c17_c_Bp0[36])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 6) {
    c17_c_Bp0[c17_j + 6 * c17_j] = c17_b_Bp0[c17_j];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel7
  (real_T c17_b_Bn0[36])
{
  uint64_T c17_threadId;
  int32_T c17_i3;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i3 = (int32_T)c17_threadId;
  if (c17_i3 < 36) {
    c17_b_Bn0[c17_i3] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel8(const
  real_T c17_b_Tid[6], const real_T c17_b_qd[6], const real_T c17_b_Th[6], const
  real_T c17_b_Bn0[6], ptrdiff_t c17_jpvt_t[6], real_T c17_A[6], real_T c17_x[6],
  real_T c17_b_x[6], real_T c17_c_Bn0[36])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 6) {
    c17_c_Bn0[c17_j + 6 * c17_j] = c17_b_Bn0[c17_j];
    c17_b_x[c17_j] = c17_b_Th[c17_j];
    c17_b_x[c17_j] = mwGpuSign(c17_b_x[c17_j]);
    c17_x[c17_j] = c17_b_qd[c17_j];
    c17_x[c17_j] = mwGpuSign(c17_x[c17_j]);
    c17_A[c17_j] = c17_b_Tid[c17_j];
    c17_jpvt_t[c17_j] = (ptrdiff_t)0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel9(const
  ptrdiff_t c17_jpvt_t[6], int32_T c17_jpvt[6])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 6) {
    c17_jpvt[c17_k] = (int32_T)c17_jpvt_t[c17_k];
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel10
  (int32_T c17_jpvt[6], real_T c17_A[6])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 6) {
    c17_A[c17_j] = HIP_NAN;
    c17_jpvt[c17_j] = c17_j + 1;
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel11
  (real_T c17_change[36])
{
  uint64_T c17_threadId;
  int32_T c17_i4;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i4 = (int32_T)c17_threadId;
  if (c17_i4 < 36) {
    c17_change[c17_i4] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel12(
  const real_T c17_b_Th[6], real_T c17_b_B[6])
{
  uint64_T c17_threadId;
  int32_T c17_i5;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i5 = (int32_T)c17_threadId;
  if (c17_i5 < 6) {
    c17_b_B[c17_i5] = c17_b_Th[c17_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel13
  (real_T c17_b_B[6])
{
  uint64_T c17_threadId;
  int32_T c17_i6;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i6 = (int32_T)c17_threadId;
  if (c17_i6 < 6) {
    c17_b_B[c17_i6] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel14(
  const int32_T c17_jpvt[6], const int32_T c17_rankA, real_T c17_A[6], real_T
  c17_b_B[6], real_T c17_change[36])
{
  uint64_T c17_threadId;
  int32_T c17_i;
  int32_T c17_j;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 6) {
    for (c17_i = 0; c17_i < c17_rankA; c17_i++) {
      c17_change[(c17_jpvt[0] + 6 * c17_k) - 1] = c17_b_B[c17_k];
    }

    for (c17_j = 0; c17_j < c17_rankA; c17_j++) {
      c17_change[(c17_jpvt[c17_rankA - 1] + 6 * c17_k) - 1] /= c17_A[(c17_rankA
        + c17_rankA) - 2];
    }
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel15(
  const real_T c17_change[36], real_T c17_Y[36])
{
  uint64_T c17_threadId;
  int32_T c17_i7;
  int32_T c17_i8;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i8 = (int32_T)(c17_threadId % 6UL);
  c17_i7 = (int32_T)((c17_threadId - (uint64_T)c17_i8) / 6UL);
  if ((c17_i7 < 6) && (c17_i8 < 6)) {
    c17_Y[c17_i8 + 6 * c17_i7] = c17_change[c17_i7 + 6 * c17_i8];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel16(
  const real_T c17_Y[36], real_T c17_b_K[36], real_T c17_change[36])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 36) {
    c17_change[c17_k] = fabs(c17_Y[c17_k]);
    c17_b_K[c17_k] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel17
  (real_T c17_b_K[36])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 6) {
    c17_b_K[c17_k + 6 * c17_k] = 1.0;
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel18
  (real_T c17_b_B[36])
{
  uint64_T c17_threadId;
  int32_T c17_i9;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i9 = (int32_T)c17_threadId;
  if (c17_i9 < 36) {
    c17_b_B[c17_i9] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel19(
  const real_T c17_b_Bn0[36], const real_T c17_b_alpha[2], const real_T
  c17_b_Bp0[36], const real_T c17_b_Kp0[36], const real_T c17_change[36], const
  real_T c17_b_beta[2], const real_T c17_b_Kn0[36], const real_T c17_x[6], const
  real_T c17_b_x[6], real_T c17_b_K[36], real_T c17_b_B[36])
{
  uint64_T c17_threadId;
  int32_T c17_ii;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_ii = (int32_T)c17_threadId;
  if (c17_ii < 6) {
    c17_b_B[c17_ii + 6 * c17_ii] = 1.0;
    if (c17_b_x[c17_ii] == c17_x[c17_ii]) {
      c17_b_K[c17_ii + 6 * c17_ii] = c17_b_Kp0[c17_ii + 6 * c17_ii] +
        c17_b_beta[0] * c17_change[c17_ii];
      c17_b_B[c17_ii + 6 * c17_ii] = c17_b_Bp0[c17_ii + 6 * c17_ii] +
        c17_b_alpha[0] * c17_change[c17_ii];
    } else {
      c17_b_K[c17_ii + 6 * c17_ii] = c17_b_Kn0[c17_ii + 6 * c17_ii] -
        c17_b_beta[1] * c17_change[c17_ii];
      c17_b_B[c17_ii + 6 * c17_ii] = c17_b_Bn0[c17_ii + 6 * c17_ii] -
        c17_b_alpha[1] * c17_change[c17_ii];
    }
  }
}

static void init_dsm_address_info(SFc17_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc17_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c17_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c17_Kp0 = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c17_Kn0 = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_Bp0 = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c17_Bn0 = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c17_beta = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
  chartInstance->c17_alpha = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
  chartInstance->c17_Tid = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 6);
  chartInstance->c17_K = (real_T (*)[36])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_Th = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 7);
  chartInstance->c17_qd = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 8);
  chartInstance->c17_B = (real_T (*)[36])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c17_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(216709722U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(633274381U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(4277968906U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3250424110U);
}

mxArray *sf_c17_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,1);
  mxSetCell(mxcell3p, 0, mxCreateString("coder.internal.lapack.LAPACKApi"));
  return(mxcell3p);
}

mxArray *sf_c17_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("late");
  mxArray *hiddenFallbackReason = mxCreateString("ir_function_calls");
  mxArray *incompatibleSymbol = mxCreateString("__gpu_kernel");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c17_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c17_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiA+wMTAwAakOYCYiQECWKF8RiBmhtIQcRa4uAIQl1QWpILEi4uSPVOAdF5iLpi"
    "fWFrhmZeWDzbfggFhPhsW8xmRzOeEikPAB3vK9Cs5gPQbIOlnIaBfAMhygoYLLHwGzv0iZLnfm2"
    "ruV3CgTD/E/ggC7ldEcT+En1kcn5hcklmWGp9saB5fnJOZkpmXHp+cn1dSlJ+Tk1pkhGQ+CAAA7"
    "zcftQ=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c17_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "siB67nyLXRLMbo1kYaCR9pH";
}

static void sf_opaque_initialize_c17_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c17_sliding_controller2
    ((SFc17_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c17_sliding_controller2(void *chartInstanceVar)
{
  enable_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c17_sliding_controller2(void *chartInstanceVar)
{
  disable_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c17_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c17_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c17_sliding_controller2
    ((SFc17_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c17_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c17_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc17_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c17_sliding_controller2
      ((SFc17_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc17_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc17_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c17_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c17_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c17_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c17_sliding_controller2
      ((SFc17_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc17_sliding_controller2
      ((SFc17_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c17_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV01u20YUphQnqIvUENAA3RRo0k2ybIKiaRdNHVFSIlSKDdNO0pUxHj6JUw1n2PmRrRN0lQP",
    "0NjlB1930Ds0N8oakZIUiqdhq0rQoAZoa8nvffO9n3oy9Rn/o4bWDd+tTz7uGz4/wbnrZdTUfN5",
    "bu7P2Wdzsfv0AjYeN9okisvdpLkBgOQEtuDZOiL0ayFMbECBQIithEKlPFpllsOROTnhXU8elnE",
    "aNREEnLwzbaknBP8BmyJdbsI0+HKaCmBxCaSEk7jnqcjBeKlTn1I6ATbeM6FzSYwCZOlh5abljC",
    "oXsGtC+0IahYn2sLDDHgm7NKN52nOpgDZZxwRkSptxHRASQYYANHSYh/96xBp4owGhFl2hCRKeg",
    "Bm6ScUkCRk2n8cMIEMVIxwrsx953hqrZ9jnqGMgReExDU1lZAJolkwlTnP+ihp11BTjh04MSOq9",
    "kC+MW65D9lcAqqMm4jX05BkTHsicpJ04B0z9JsLapkFWZYDE+JekgxfxrCyurFytEBwTzBIVpUw",
    "SB1sq8PFZtieCvZbNx3lbluydg4S7ZeB0vZulOoy8KCrUeFTzjXlbBDmQxgCjxl7RBD6mEZazlO",
    "axYeSgywK+/q1WAFw8TnMF+KkJWma1oApH3nCTaWN5HUaiNjH4u3Mxisfl6F9YUBNSIUyrqAIkw",
    "DxiwNbzVbyLTLPQJRlUnllYGzClmH8vTIis6pVBOMSU0TOXfBZbQSGOsx5hJXwpHGRVMHc7lch6",
    "OERhC6BsM4DHHZILYkJtq1toe47qbMzDqgqWJJSVYtrjpsQ11XULMEjsREyFPRUzIO8h6fhRcAq",
    "40owcS4jf1LzXo4e1kq3H72lXe+n11/i/1sbld83lniaZTweEtPh99dwm8338RvF+Ztzt/Nrwqe",
    "TwrzbhV4HK7l0vLy5/DXV/e+/n3n+9/+3P34UTEORT2NFT2N9LezS65c7Dywk48/nzfexUKartS",
    "vwz5e0rVVwv/ZEn8rH2vW/ua+mA2eHwyGJ/Lu5CfiH3yXPE75Xjbr9V4p6J2/v+l2AKy6tF4V7Y",
    "f5QcWNic22b8f/7ZLea2visZ2/z66/ftjM/svdYh631ti38Fe7UMf/nP4bl9L/49+m/+buZvbZ/",
    "M/X6L9VqNdb6TnlmLjuB8f07v1jzVmIreuYSmGU5BzUvZJ+c9n1elE77z3b/Vt0/m/37vP+Nvvx",
    "9iXtmhvu/+/LblP/Lnou+dDwdfuBV8C3PmA/Nj0vvmv8H97FznFf5OMHi3/d/IjxsOQUn3/GI/m",
    "o7Ot/oE5fAwg+tTI=",
    ""
  };

  static char newstr [1293] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c17_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c17_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1684717667U));
  ssSetChecksum1(S,(875753863U));
  ssSetChecksum2(S,(2570915534U));
  ssSetChecksum3(S,(1191854301U));
}

static void mdlRTW_c17_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c17_sliding_controller2(SimStruct *S)
{
  SFc17_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc17_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc17_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc17_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc17_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c17_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c17_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c17_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c17_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c17_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c17_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c17_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c17_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c17_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c17_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c17_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c17_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c17_JITStateAnimation,
    chartInstance->c17_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c17_sliding_controller2(chartInstance);
}

void c17_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c17_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c17_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c17_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c17_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
