/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c1_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c1_do_animation_call_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c1_st);
static void sf_gateway_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c1_eML_blk_kernel(SFc1_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2]);
static void c1_emlrt_marshallIn(SFc1_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2]);
static void c1_b_emlrt_marshallIn(SFc1_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2]);
static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller_rbdl, const
  char_T *c1_identifier);
static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static void init_dsm_address_info(SFc1_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc1_sliding_controller_rbdlInstanceStruct *
  chartInstance);

/* Function Definitions */
static void initialize_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c1_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c1_is_active_c1_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c1_do_animation_call_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c1_b_y = NULL;
  const mxArray *c1_c_y = NULL;
  const mxArray *c1_d_y = NULL;
  const mxArray *c1_e_y = NULL;
  const mxArray *c1_f_y = NULL;
  const mxArray *c1_st;
  const mxArray *c1_y = NULL;
  c1_st = NULL;
  c1_y = NULL;
  sf_mex_assign(&c1_y, sf_mex_createcellmatrix(5, 1), false);
  c1_b_y = NULL;
  sf_mex_assign(&c1_b_y, sf_mex_create("y", *chartInstance->c1_ankle, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 0, c1_b_y);
  c1_c_y = NULL;
  sf_mex_assign(&c1_c_y, sf_mex_create("y", *chartInstance->c1_danke, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 1, c1_c_y);
  c1_d_y = NULL;
  sf_mex_assign(&c1_d_y, sf_mex_create("y", *chartInstance->c1_dknee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 2, c1_d_y);
  c1_e_y = NULL;
  sf_mex_assign(&c1_e_y, sf_mex_create("y", *chartInstance->c1_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 3, c1_e_y);
  c1_f_y = NULL;
  sf_mex_assign(&c1_f_y, sf_mex_create("y",
    &chartInstance->c1_is_active_c1_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c1_y, 4, c1_f_y);
  sf_mex_assign(&c1_st, c1_y, false);
  return c1_st;
}

static void set_sim_state_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c1_st)
{
  const mxArray *c1_u;
  c1_u = sf_mex_dup(c1_st);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 0)),
                      "ankle", *chartInstance->c1_ankle);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 1)),
                      "danke", *chartInstance->c1_danke);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 2)),
                      "dknee", *chartInstance->c1_dknee);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 3)), "knee",
                      *chartInstance->c1_knee);
  chartInstance->c1_is_active_c1_sliding_controller_rbdl = c1_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 4)),
     "is_active_c1_sliding_controller_rbdl");
  sf_mex_destroy(&c1_u);
  sf_mex_destroy(&c1_st);
}

static void sf_gateway_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c1_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c1_eML_blk_kernel(chartInstance, *chartInstance->c1_q, *chartInstance->c1_qd, *
                    chartInstance->c1_knee, *chartInstance->c1_dknee,
                    *chartInstance->c1_ankle, *chartInstance->c1_danke);
  c1_do_animation_call_c1_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c1_errCode;
  c1_errCode = hipGetLastError();
  if (c1_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c1_errCode, (char_T *)hipGetErrorName
                       (c1_errCode), (char_T *)hipGetErrorString(c1_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c1_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc1_sliding_controller_rbdl
  (SFc1_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c1_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c1_nameCaptureInfo = NULL;
  const char_T *c1_data[4] = {
    "789ce593cb4ac34014864fa51637f5b2e94b088e75e5be2a8a5884163722f1243969c6cc452653dbbaf10df4297c171fcb246d7a09868862371e08277ffecc7c"
    "677e18a85d5cd500603b795a2f00e70dc8aa396db03beb1bb05a45bf56d2f3da84facabad4df4b786f33ed6965696ca742a1a4f94a5f4bae50d9fee491c050ac",
    "c513f9991370417d2ea9b72cbaa992674bd65ca456fade09c98b7a4309268c17138a6591e591d67dc979eb157914ab9847f1bf9c37fe212fdfbf55c1cbfd409b"
    "111adf89b8e2122df76247d0e0f7e7ceb98dd239a68eaf87aea005efe3973c2ce5adfab7a7772cd49298421ba2e224065af8011a97796893349c51cc62e33194",
    "6ee08c504464581290409729ed645f2dc5967d1de0819c71aa72dcf9e6b9caee5313b6b2fefeba8febe4e5f55f78ebba8f978713de7ee60f517474dce9b65dd7"
    "bbf1edc9628eeb0a4ed51c50a2ff7aff4fb0e86ac3", "" };

  c1_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c1_data[0], 1632U, &c1_nameCaptureInfo);
  return c1_nameCaptureInfo;
}

static void c1_eML_blk_kernel(SFc1_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2])
{
  real_T c1_a;
  real_T c1_b_a;
  c1_b_knee[0] = -0.324 * muDoubleScalarSin(c1_b_q[0]);
  c1_b_knee[1] = -0.324 * muDoubleScalarCos(c1_b_q[0]);
  c1_a = -0.324 * c1_b_qd[0];
  c1_b_dknee[0] = c1_a * muDoubleScalarCos(c1_b_q[0]);
  c1_b_dknee[1] = c1_a * -muDoubleScalarSin(c1_b_q[0]);
  c1_b_a = -0.39 * (c1_b_qd[0] + c1_b_qd[1]);
  c1_b_ankle[0] = c1_b_knee[0] + -0.39 * muDoubleScalarSin(c1_b_q[0] + c1_b_q[1]);
  c1_b_ankle[1] = c1_b_knee[1] + -0.39 * muDoubleScalarCos(c1_b_q[0] + c1_b_q[1]);
  c1_b_danke[0] = c1_b_dknee[0] + c1_b_a * muDoubleScalarCos(c1_b_q[0] + c1_b_q
    [1]);
  c1_b_danke[1] = c1_b_dknee[1] + c1_b_a * -muDoubleScalarSin(c1_b_q[0] +
    c1_b_q[1]);
}

static void c1_emlrt_marshallIn(SFc1_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2])
{
  emlrtMsgIdentifier c1_thisId;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_b_ankle), &c1_thisId, c1_y);
  sf_mex_destroy(&c1_b_ankle);
}

static void c1_b_emlrt_marshallIn(SFc1_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2])
{
  real_T c1_dv[2];
  int32_T c1_i;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), c1_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c1_i = 0; c1_i < 2; c1_i++) {
    c1_y[c1_i] = c1_dv[c1_i];
  }

  sf_mex_destroy(&c1_u);
}

static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller_rbdl, const
  char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  uint8_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c1_b_is_active_c1_sliding_controller_rbdl), &c1_thisId);
  sf_mex_destroy(&c1_b_is_active_c1_sliding_controller_rbdl);
  return c1_y;
}

static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  uint8_T c1_b_u;
  uint8_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_b_u, 1, 3, 0U, 0, 0U, 0);
  c1_y = c1_b_u;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static void init_dsm_address_info(SFc1_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc1_sliding_controller_rbdlInstanceStruct *
  chartInstance)
{
  chartInstance->c1_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c1_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c1_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c1_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c1_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c1_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c1_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c1_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c1_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c1_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6DFA4ifWRyfmFySWZYan"
    "2wYX5yTmZKZlx6fnJ9XUpSfk5NaFF+UlJLDgBRPAKQWLlY="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c1_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c1_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c1_sliding_controller_rbdl
    ((SFc1_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c1_sliding_controller_rbdl
    ((SFc1_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c1_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c1_sliding_controller_rbdl((SFc1_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c1_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c1_sliding_controller_rbdl((SFc1_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c1_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c1_sliding_controller_rbdl
    ((SFc1_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c1_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c1_sliding_controller_rbdl
    ((SFc1_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c1_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c1_sliding_controller_rbdl
    ((SFc1_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c1_sliding_controller_rbdl(void *
  chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc1_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c1_sliding_controller_rbdl
      ((SFc1_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc1_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc1_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c1_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c1_sliding_controller_rbdl
    ((SFc1_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c1_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c1_sliding_controller_rbdl
    ((SFc1_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc1_sliding_controller_rbdl
    ((SFc1_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c1_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c1_sliding_controller_rbdl
      ((SFc1_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc1_sliding_controller_rbdl
      ((SFc1_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c1_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA2cAEDvQRoUARoi15cFAXaS2ObkhABduyUchr0IqyWI3Gh5S67P/TPO+S",
    "eSx+kr9An6BPkmjxCZ0lalimSiuwmdYvSoOQlv/l2vpnZ2ZXXGhx4eN3H+9WW593B77t4t7382i",
    "jGrYU7f97xvijGL9FI2PiIKBJrr/ESJIafQEtuDZNiICayEsbEBBQIithEKlPHpllsOROzvhXU8",
    "emfI0ajIJKWh3toS8JDwc+QLbHmCHm6TAE1fYDQREraadTnZDr3WJkTPwI60zZukqDBBDZxbukD",
    "yw1LOPROgQ6ENgQ91pe+BYYY8M1prUynVAcXQBknnBFRqTYiOoAEA2zgOAnx89AaFFWG0Ygoswc",
    "RSUHvs1nGKQWUOZnGF2MmiJGKEd6Lue8Ml3074ujPgQyBNwQEfdtTQGaJZMLU5z/oo9KeIGMOXR",
    "jbaT1bAL9al/znDE5A1cZt4ssUFJnCoaidNAtI7zTL1rxKlmGGxfCcqF2K+dMQ1lYvVo4OCOYJh",
    "mhRB4NM5EAPFUsxvLVsNh64yly1ZGycJ1uvgmVsvRSasjBn61PhE851LWwok31IgWesXWJIMyxn",
    "rcZpzcKhxAC78q5fDVYwTHwB86UIWWW60hIg6ztPsbFcRVKrjYx9LN7u/v7y62XYQBhQE0Khqgs",
    "owjRgzLLw1rOFTLvcIxC9Mpl7VeC8QlahPD2xonsi1Qxj0tBELiW4jNYCYz3FXOJKONa4aJpgLp",
    "ercJTQCELXYBiHA1w2iK2IiXatbRfXXcrMWRc0VSypyKrFVYdtqOcK6iyBYzET8kT0lYyDosfn4",
    "QXAaiNKMDHdw/6lzvo4e1Uq3H627V3uZx+9w352YVf+/nKBp1XB4y18l+e9176Kb5fmbeNfq5Xb",
    "7SzYfVyap1Oyc7hNvM9//xrePP4teh199cnWOfyxav7W0vyt7H9nd3djvf3/fjF+cNFo5wsnXap",
    "Xh32y4Fengv/TBf7NYqyfRd+J9Jcn0e4zMowH29+e/7Cdjv2M70Wn2d+Nkr8Xzx+6jo9VltWnoo",
    "OwOJi4MbH5du34v1/w986KeNwrnufX28c3s3+wU7bvrNC3WYyJmHEo1e8/p+Pza+sIUcjt0fHw+",
    "jpmeGi4NTq2dsr9oUpH54qOjuck/D3+L8dxPft8/hcr/H9UysOj7Lw4Im4XghH9ZqQ5C3EHGVEp",
    "jJKcgxqpccjLff+6fXRdO+8D2/1b/Pzf7v3n/V3ORe1r2nVueA77UHY31bfu+fC24Zv2A6+E37z",
    "FOm56bn/f+D+99c7XnxXjH+c/of2I8bDi11TxGn8aTare/gfq9C9568QS",
    ""
  };

  static char newstr [1333] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c1_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c1_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1697364346U));
  ssSetChecksum1(S,(1755135983U));
  ssSetChecksum2(S,(287926502U));
  ssSetChecksum3(S,(3395648020U));
}

static void mdlRTW_c1_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c1_sliding_controller_rbdl(SimStruct *S)
{
  SFc1_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc1_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc1_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc1_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc1_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c1_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c1_JITStateAnimation,
    chartInstance->c1_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c1_sliding_controller_rbdl(chartInstance);
}

void c1_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c1_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c1_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c1_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c1_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
