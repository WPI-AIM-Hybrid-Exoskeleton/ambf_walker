/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c33_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c33_do_animation_call_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c33_st);
static void sf_gateway_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c33_eML_blk_kernel(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c33_b_u[7], real_T c33_b_y[6]);
static void c33_emlrt_marshallIn(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c33_b_y, const char_T *c33_identifier, real_T
  c33_c_y[6]);
static void c33_b_emlrt_marshallIn(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c33_b_u, const emlrtMsgIdentifier *c33_parentId,
  real_T c33_b_y[6]);
static uint8_T c33_c_emlrt_marshallIn
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c33_b_is_active_c33_sliding_controller_rbdl, const char_T *c33_identifier);
static uint8_T c33_d_emlrt_marshallIn
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c33_b_u, const emlrtMsgIdentifier *c33_parentId);
static void init_dsm_address_info(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c33_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c33_is_active_c33_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c33_do_animation_call_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c33_b_y = NULL;
  const mxArray *c33_c_y = NULL;
  const mxArray *c33_d_y = NULL;
  const mxArray *c33_st;
  c33_st = NULL;
  c33_b_y = NULL;
  sf_mex_assign(&c33_b_y, sf_mex_createcellmatrix(2, 1), false);
  c33_c_y = NULL;
  sf_mex_assign(&c33_c_y, sf_mex_create("y", *chartInstance->c33_y, 0, 0U, 1U,
    0U, 1, 6), false);
  sf_mex_setcell(c33_b_y, 0, c33_c_y);
  c33_d_y = NULL;
  sf_mex_assign(&c33_d_y, sf_mex_create("y",
    &chartInstance->c33_is_active_c33_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c33_b_y, 1, c33_d_y);
  sf_mex_assign(&c33_st, c33_b_y, false);
  return c33_st;
}

static void set_sim_state_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c33_st)
{
  const mxArray *c33_b_u;
  c33_b_u = sf_mex_dup(c33_st);
  c33_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c33_b_u, 0)),
                       "y", *chartInstance->c33_y);
  chartInstance->c33_is_active_c33_sliding_controller_rbdl =
    c33_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c33_b_u, 1)),
    "is_active_c33_sliding_controller_rbdl");
  sf_mex_destroy(&c33_b_u);
  sf_mex_destroy(&c33_st);
}

static void sf_gateway_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c33_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c33_eML_blk_kernel(chartInstance, *chartInstance->c33_u, *chartInstance->c33_y);
  c33_do_animation_call_c33_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c33_errCode;
  c33_errCode = hipGetLastError();
  if (c33_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c33_errCode, (char_T *)hipGetErrorName
                       (c33_errCode), (char_T *)hipGetErrorString(c33_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c33_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc33_sliding_controller_rbdl
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c33_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c33_nameCaptureInfo = NULL;
  c33_nameCaptureInfo = NULL;
  sf_mex_assign(&c33_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c33_nameCaptureInfo;
}

static void c33_eML_blk_kernel(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c33_b_u[7], real_T c33_b_y[6])
{
  c33_b_y[0] = c33_b_u[2];
  c33_b_y[1] = c33_b_u[3];
  c33_b_y[2] = c33_b_u[6];
  c33_b_y[3] = c33_b_u[0];
  c33_b_y[4] = c33_b_u[4];
  c33_b_y[5] = c33_b_u[5];
}

static void c33_emlrt_marshallIn(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c33_b_y, const char_T *c33_identifier, real_T
  c33_c_y[6])
{
  emlrtMsgIdentifier c33_thisId;
  c33_thisId.fIdentifier = const_cast<const char_T *>(c33_identifier);
  c33_thisId.fParent = NULL;
  c33_thisId.bParentIsCell = false;
  c33_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c33_b_y), &c33_thisId,
    c33_c_y);
  sf_mex_destroy(&c33_b_y);
}

static void c33_b_emlrt_marshallIn(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c33_b_u, const emlrtMsgIdentifier *c33_parentId,
  real_T c33_b_y[6])
{
  real_T c33_dv[6];
  int32_T c33_i;
  sf_mex_import(c33_parentId, sf_mex_dup(c33_b_u), c33_dv, 1, 0, 0U, 1, 0U, 1, 6);
  for (c33_i = 0; c33_i < 6; c33_i++) {
    c33_b_y[c33_i] = c33_dv[c33_i];
  }

  sf_mex_destroy(&c33_b_u);
}

static uint8_T c33_c_emlrt_marshallIn
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c33_b_is_active_c33_sliding_controller_rbdl, const char_T *c33_identifier)
{
  emlrtMsgIdentifier c33_thisId;
  uint8_T c33_b_y;
  c33_thisId.fIdentifier = const_cast<const char_T *>(c33_identifier);
  c33_thisId.fParent = NULL;
  c33_thisId.bParentIsCell = false;
  c33_b_y = c33_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c33_b_is_active_c33_sliding_controller_rbdl), &c33_thisId);
  sf_mex_destroy(&c33_b_is_active_c33_sliding_controller_rbdl);
  return c33_b_y;
}

static uint8_T c33_d_emlrt_marshallIn
  (SFc33_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c33_b_u, const emlrtMsgIdentifier *c33_parentId)
{
  uint8_T c33_b_y;
  uint8_T c33_c_u;
  sf_mex_import(c33_parentId, sf_mex_dup(c33_b_u), &c33_c_u, 1, 3, 0U, 0, 0U, 0);
  c33_b_y = c33_c_u;
  sf_mex_destroy(&c33_b_u);
  return c33_b_y;
}

static void init_dsm_address_info(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc33_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c33_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c33_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c33_y = (real_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c33_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2339256118U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2737773724U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(583777407U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1172739197U);
}

mxArray *sf_c33_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c33_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c33_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c33_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cnGxvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB2RR2x"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c33_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "s0lpZ3UsJHBzJBVszsMYZmF";
}

static void sf_opaque_initialize_c33_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c33_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c33_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c33_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c33_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c33_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c33_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc33_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c33_sliding_controller_rbdl
      ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc33_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c33_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c33_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc33_sliding_controller_rbdl
    ((SFc33_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c33_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c33_sliding_controller_rbdl
      ((SFc33_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc33_sliding_controller_rbdl
      ((SFc33_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c33_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV91u00gUdqOCQFpQhZDQSiuBkBBcInrBXi0lTqKGbWiF2/JzE03HJ/Eo4xkzP2nDO+zFPgW",
    "3PA7ihjuueQTO2E4aHNuhRFSwWkuuM/Z3zpxzvvMz9da6PQ+vq3hvXfO8i/i8hHfDy64L+Xpt7s",
    "7er3t38/U/KCRsvEcUibVXewkSwzPQklvDpOiKgSyFMTEABYIiNpHKVGnTLLaciVHHCur06ecRo",
    "1EQScvDJsqScFfwCWpLrNlDPS2mgJoOQGgiJe0w6nAynFmszLEfAR1pG9e5oMEENnFm6Z7lhiUc",
    "2idAu0IbghbrU9sCQwz45qTSTeepDqZAGSecEVHqbUR0AAkG2MBBEuLfXWvQqSKMRkSZJkRkDHq",
    "HjVKdUkBRJ9P44YgJYqRihLdj7jvBRdv2ONrTkyHwmoCgbU0FZJRIJkw1/0EHPW0LcsShBUd2WK",
    "0tgNfWkX/I4BhUZdwGvhyDIkPYFZWbpgFpn6RszbJkEWZYDIdEPabIn4awMnsxc3RAkCfYR4kqG",
    "KROdvW+YmMMb6U2G3ddZi4rGRtnZOtlsFRbewx1LMy0dajwCee6ErYvkx0YA0+1togh9bBMazlO",
    "axbuSwywS+/qarCCIfE5zJciZKV0jQuAtO88xcbyNZJabWTsY/K2dnYWPy/CusKAGhAKZV1AEaY",
    "BY5aGt1pbyLTjHoFolUnNKwNnGbIM5emBFa1jqUYYk5omcuqCY7QSGOshcomVcKCxaOpgjstlOE",
    "poBKFrMIxDD8sGsSUx0a61Pca6GzMzaYGmiiUlrFqsOmxDbZdQkwQOxEjIY9FRMg7yHp+FFwCzj",
    "SjBxLCJ/UtNOrh7GRVunt33TufZb98wz6Zyxee9OT1rJXq8uWdx38uN+n0b+GttOn/n5K4U9lkv",
    "yDncBt63P7x94H/aZh//vUu6f//+cJX93zXONv+v5us/po12VjjjhXx12O05u9ZL9N+Y07+Rr/V",
    "9nrzaPNBPtptvnjQP9Rvde/kq7mT8LLG3UbB3+v6W6/iYZWl+KtoN84OJWxObjWun/885ey8uic",
    "fl/H12fX60mvz1rSKP60vkN/DXpJC337//ra3V5LP9Xyyx/06B7zvpXO8T1y2gTzc3+5qzEEu9T",
    "6UwSnIOqq+OQl7w83vz/axy3jnL/Sp2/i/343n/lvl13nLeOcut6t9Z5/jPhq+bB14Bv/ET+7Hq",
    "+epH4997ZzsH3czXf83+1fEjxsOSU2/+GY+wg7Kv/4E8/QJ4HrNn",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c33_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c33_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(849861154U));
  ssSetChecksum1(S,(1766385219U));
  ssSetChecksum2(S,(1629983199U));
  ssSetChecksum3(S,(924404553U));
}

static void mdlRTW_c33_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c33_sliding_controller_rbdl(SimStruct *S)
{
  SFc33_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc33_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc33_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc33_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc33_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c33_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c33_JITStateAnimation,
    chartInstance->c33_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c33_sliding_controller_rbdl(chartInstance);
}

void c33_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c33_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c33_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c33_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c33_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
