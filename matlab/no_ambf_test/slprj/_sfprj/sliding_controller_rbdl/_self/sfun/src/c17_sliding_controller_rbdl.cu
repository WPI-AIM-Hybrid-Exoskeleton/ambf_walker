#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c17_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#include "MWGpuFunctionImplUtility.hpp"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include "MWCUSOLVERUtils.hpp"
#include "hip/hip_math_constants.h"

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c17_do_animation_call_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c17_st);
static void sf_gateway_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c17_eML_blk_kernel(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c17_b_Kp0[7], real_T c17_b_Kn0[7], real_T c17_b_Bp0[7],
  real_T c17_b_Bn0[7], real_T c17_b_beta[2], real_T c17_b_alpha[2], real_T
  c17_b_Tid[7], real_T c17_b_Th[7], real_T c17_b_qd[7], real_T c17_b_K[49],
  real_T c17_b_B[49]);
static void c17_emlrt_marshallIn(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c17_b_B, const char_T *c17_identifier, real_T
  c17_y[49]);
static void c17_b_emlrt_marshallIn(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId,
  real_T c17_y[49]);
static uint8_T c17_c_emlrt_marshallIn
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c17_b_is_active_c17_sliding_controller_rbdl, const char_T *c17_identifier);
static uint8_T c17_d_emlrt_marshallIn
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c17_u, const emlrtMsgIdentifier *c17_parentId);
static __global__ void c17_eML_blk_kernel_kernel1(real_T c17_b_Kp0[49]);
static __global__ void c17_eML_blk_kernel_kernel2(const real_T c17_b_Kp0[7],
  real_T c17_c_Kp0[49]);
static __global__ void c17_eML_blk_kernel_kernel3(real_T c17_b_Kn0[49]);
static __global__ void c17_eML_blk_kernel_kernel4(const real_T c17_b_Kn0[7],
  real_T c17_c_Kn0[49]);
static __global__ void c17_eML_blk_kernel_kernel5(real_T c17_b_Bp0[49]);
static __global__ void c17_eML_blk_kernel_kernel6(const real_T c17_b_Bp0[7],
  real_T c17_c_Bp0[49]);
static __global__ void c17_eML_blk_kernel_kernel7(real_T c17_b_Bn0[49]);
static __global__ void c17_eML_blk_kernel_kernel8(const real_T c17_b_Tid[7],
  const real_T c17_b_qd[7], const real_T c17_b_Th[7], const real_T c17_b_Bn0[7],
  ptrdiff_t c17_jpvt_t[7], real_T c17_A[7], real_T c17_x[7], real_T c17_b_x[7],
  real_T c17_c_Bn0[49]);
static __global__ void c17_eML_blk_kernel_kernel9(const ptrdiff_t c17_jpvt_t[7],
  int32_T c17_jpvt[7]);
static __global__ void c17_eML_blk_kernel_kernel10(int32_T c17_jpvt[7], real_T
  c17_A[7]);
static __global__ void c17_eML_blk_kernel_kernel11(real_T c17_change[49]);
static __global__ void c17_eML_blk_kernel_kernel12(const real_T c17_b_Th[7],
  real_T c17_b_B[7]);
static __global__ void c17_eML_blk_kernel_kernel13(real_T c17_b_B[7]);
static __global__ void c17_eML_blk_kernel_kernel14(const int32_T c17_jpvt[7],
  const int32_T c17_rankA, real_T c17_A[7], real_T c17_b_B[7], real_T
  c17_change[49]);
static __global__ void c17_eML_blk_kernel_kernel15(const real_T c17_change[49],
  real_T c17_Y[49]);
static __global__ void c17_eML_blk_kernel_kernel16(const real_T c17_Y[49],
  real_T c17_b_K[49], real_T c17_change[49]);
static __global__ void c17_eML_blk_kernel_kernel17(real_T c17_b_K[49]);
static __global__ void c17_eML_blk_kernel_kernel18(real_T c17_b_B[49]);
static __global__ void c17_eML_blk_kernel_kernel19(const real_T c17_b_Bn0[49],
  const real_T c17_b_alpha[2], const real_T c17_b_Bp0[49], const real_T
  c17_b_Kp0[49], const real_T c17_change[49], const real_T c17_b_beta[2], const
  real_T c17_b_Kn0[49], const real_T c17_x[7], const real_T c17_b_x[7], real_T
  c17_b_K[49], real_T c17_b_B[49]);
static void init_dsm_address_info(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c17_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c17_is_active_c17_sliding_controller_rbdl = 0U;
  cusolverEnsureInitialization();
  hipGetLastError();
  hipMalloc(&chartInstance->c17_gpu_beta, 16UL);
  hipMalloc(&chartInstance->c17_gpu_alpha, 16UL);
  hipMalloc(&chartInstance->c17_gpu_B, 392UL);
  hipMalloc(&chartInstance->c17_gpu_K, 392UL);
  hipMalloc(&chartInstance->c17_gpu_Y, 392UL);
  hipMalloc(&chartInstance->c17_gpu_info_t, 4UL);
  hipMalloc(&chartInstance->c17_gpu_tau, 8UL);
  hipMalloc(&chartInstance->c17_b_gpu_B, 56UL);
  hipMalloc(&chartInstance->c17_gpu_change, 392UL);
  hipMalloc(&chartInstance->c17_gpu_jpvt, 28UL);
  hipMalloc(&chartInstance->c17_gpu_x, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_x, 56UL);
  hipMalloc(&chartInstance->c17_gpu_A, 56UL);
  hipMalloc(&chartInstance->c17_gpu_jpvt_t, 56UL);
  hipMalloc(&chartInstance->c17_gpu_Bn0, 56UL);
  hipMalloc(&chartInstance->c17_gpu_Th, 56UL);
  hipMalloc(&chartInstance->c17_gpu_qd, 56UL);
  hipMalloc(&chartInstance->c17_gpu_Tid, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_Bn0, 392UL);
  hipMalloc(&chartInstance->c17_gpu_Bp0, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_Bp0, 392UL);
  hipMalloc(&chartInstance->c17_gpu_Kn0, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_Kn0, 392UL);
  hipMalloc(&chartInstance->c17_gpu_Kp0, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_Kp0, 392UL);
}

static void initialize_params_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c17_do_animation_call_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c17_b_y = NULL;
  const mxArray *c17_c_y = NULL;
  const mxArray *c17_d_y = NULL;
  const mxArray *c17_st;
  const mxArray *c17_y = NULL;
  c17_st = NULL;
  c17_y = NULL;
  sf_mex_assign(&c17_y, sf_mex_createcellmatrix(3, 1), false);
  c17_b_y = NULL;
  sf_mex_assign(&c17_b_y, sf_mex_create("y", *chartInstance->c17_B, 0, 0U, 1U,
    0U, 2, 7, 7), false);
  sf_mex_setcell(c17_y, 0, c17_b_y);
  c17_c_y = NULL;
  sf_mex_assign(&c17_c_y, sf_mex_create("y", *chartInstance->c17_K, 0, 0U, 1U,
    0U, 2, 7, 7), false);
  sf_mex_setcell(c17_y, 1, c17_c_y);
  c17_d_y = NULL;
  sf_mex_assign(&c17_d_y, sf_mex_create("y",
    &chartInstance->c17_is_active_c17_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c17_y, 2, c17_d_y);
  sf_mex_assign(&c17_st, c17_y, false);
  return c17_st;
}

static void set_sim_state_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c17_st)
{
  const mxArray *c17_u;
  c17_u = sf_mex_dup(c17_st);
  c17_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 0)), "B",
                       *chartInstance->c17_B);
  c17_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 1)), "K",
                       *chartInstance->c17_K);
  chartInstance->c17_is_active_c17_sliding_controller_rbdl =
    c17_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 2)),
    "is_active_c17_sliding_controller_rbdl");
  sf_mex_destroy(&c17_u);
  sf_mex_destroy(&c17_st);
}

static void sf_gateway_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  real_T c17_dv10[49];
  real_T c17_dv9[49];
  real_T c17_dv[7];
  real_T c17_dv1[7];
  real_T c17_dv2[7];
  real_T c17_dv3[7];
  real_T c17_dv6[7];
  real_T c17_dv7[7];
  real_T c17_dv8[7];
  real_T c17_dv4[2];
  real_T c17_dv5[2];
  int32_T c17_i;
  int32_T c17_i1;
  int32_T c17_i10;
  int32_T c17_i2;
  int32_T c17_i3;
  int32_T c17_i4;
  int32_T c17_i5;
  int32_T c17_i6;
  int32_T c17_i7;
  int32_T c17_i8;
  int32_T c17_i9;
  chartInstance->c17_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c17_i = 0; c17_i < 7; c17_i++) {
    c17_dv[c17_i] = (*chartInstance->c17_Kp0)[c17_i];
  }

  for (c17_i1 = 0; c17_i1 < 7; c17_i1++) {
    c17_dv1[c17_i1] = (*chartInstance->c17_Kn0)[c17_i1];
  }

  for (c17_i2 = 0; c17_i2 < 7; c17_i2++) {
    c17_dv2[c17_i2] = (*chartInstance->c17_Bp0)[c17_i2];
  }

  for (c17_i3 = 0; c17_i3 < 7; c17_i3++) {
    c17_dv3[c17_i3] = (*chartInstance->c17_Bn0)[c17_i3];
  }

  for (c17_i4 = 0; c17_i4 < 2; c17_i4++) {
    c17_dv4[c17_i4] = (*chartInstance->c17_beta)[c17_i4];
  }

  for (c17_i5 = 0; c17_i5 < 2; c17_i5++) {
    c17_dv5[c17_i5] = (*chartInstance->c17_alpha)[c17_i5];
  }

  for (c17_i6 = 0; c17_i6 < 7; c17_i6++) {
    c17_dv6[c17_i6] = (*chartInstance->c17_Tid)[c17_i6];
  }

  for (c17_i7 = 0; c17_i7 < 7; c17_i7++) {
    c17_dv7[c17_i7] = (*chartInstance->c17_Th)[c17_i7];
  }

  for (c17_i8 = 0; c17_i8 < 7; c17_i8++) {
    c17_dv8[c17_i8] = (*chartInstance->c17_qd)[c17_i8];
  }

  c17_eML_blk_kernel(chartInstance, c17_dv, c17_dv1, c17_dv2, c17_dv3, c17_dv4,
                     c17_dv5, c17_dv6, c17_dv7, c17_dv8, c17_dv9, c17_dv10);
  for (c17_i9 = 0; c17_i9 < 49; c17_i9++) {
    (*chartInstance->c17_B)[c17_i9] = c17_dv10[c17_i9];
  }

  for (c17_i10 = 0; c17_i10 < 49; c17_i10++) {
    (*chartInstance->c17_K)[c17_i10] = c17_dv9[c17_i10];
  }

  c17_do_animation_call_c17_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c17_errCode;
  hipFree(*chartInstance->c17_gpu_beta);
  hipFree(*chartInstance->c17_gpu_alpha);
  hipFree(*chartInstance->c17_gpu_B);
  hipFree(*chartInstance->c17_gpu_K);
  hipFree(*chartInstance->c17_gpu_Y);
  hipFree(chartInstance->c17_gpu_info_t);
  hipFree(chartInstance->c17_gpu_tau);
  hipFree(*chartInstance->c17_b_gpu_B);
  hipFree(*chartInstance->c17_gpu_change);
  hipFree(*chartInstance->c17_gpu_jpvt);
  hipFree(*chartInstance->c17_gpu_x);
  hipFree(*chartInstance->c17_b_gpu_x);
  hipFree(*chartInstance->c17_gpu_A);
  hipFree(*chartInstance->c17_gpu_jpvt_t);
  hipFree(*chartInstance->c17_gpu_Bn0);
  hipFree(*chartInstance->c17_gpu_Th);
  hipFree(*chartInstance->c17_gpu_qd);
  hipFree(*chartInstance->c17_gpu_Tid);
  hipFree(*chartInstance->c17_b_gpu_Bn0);
  hipFree(*chartInstance->c17_gpu_Bp0);
  hipFree(*chartInstance->c17_b_gpu_Bp0);
  hipFree(*chartInstance->c17_gpu_Kn0);
  hipFree(*chartInstance->c17_b_gpu_Kn0);
  hipFree(*chartInstance->c17_gpu_Kp0);
  hipFree(*chartInstance->c17_b_gpu_Kp0);
  c17_errCode = hipGetLastError();
  if (c17_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c17_errCode, (char_T *)hipGetErrorName
                       (c17_errCode), (char_T *)hipGetErrorString(c17_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c17_fEmlrtCtx);
  }

  cusolverDestroyWorkspace();
  cusolverEnsureDestruction();
}

static void mdl_setup_runtime_resources_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc17_sliding_controller_rbdl
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c17_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c17_nameCaptureInfo = NULL;
  c17_nameCaptureInfo = NULL;
  sf_mex_assign(&c17_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c17_nameCaptureInfo;
}

static void c17_eML_blk_kernel(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c17_b_Kp0[7], real_T c17_b_Kn0[7], real_T c17_b_Bp0[7],
  real_T c17_b_Bn0[7], real_T c17_b_beta[2], real_T c17_b_alpha[2], real_T
  c17_b_Tid[7], real_T c17_b_Th[7], real_T c17_b_qd[7], real_T c17_b_K[49],
  real_T c17_b_B[49])
{
  ptrdiff_t c17_jpvt_t[7];
  ptrdiff_t c17_info_t;
  real_T c17_A[7];
  real_T c17_tau;
  real_T c17_tol;
  int32_T c17_b_info_t;
  int32_T c17_rankA;
  boolean_T c17_A_dirtyOnCpu;
  boolean_T c17_A_dirtyOnGpu;
  c17_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Kp0);
  hipMemcpy(*chartInstance->c17_gpu_Kp0, c17_b_Kp0, 56UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Kp0, *chartInstance->c17_b_gpu_Kp0);
  c17_eML_blk_kernel_kernel3<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Kn0);
  hipMemcpy(*chartInstance->c17_gpu_Kn0, c17_b_Kn0, 56UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Kn0, *chartInstance->c17_b_gpu_Kn0);
  c17_eML_blk_kernel_kernel5<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Bp0);
  hipMemcpy(*chartInstance->c17_gpu_Bp0, c17_b_Bp0, 56UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Bp0, *chartInstance->c17_b_gpu_Bp0);
  c17_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Bn0);
  hipMemcpy(*chartInstance->c17_gpu_Tid, c17_b_Tid, 56UL,
             hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_qd, c17_b_qd, 56UL, hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_Th, c17_b_Th, 56UL, hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_Bn0, c17_b_Bn0, 56UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Tid, *chartInstance->c17_gpu_qd,
     *chartInstance->c17_gpu_Th, *chartInstance->c17_gpu_Bn0,
     *chartInstance->c17_gpu_jpvt_t, *chartInstance->c17_gpu_A,
     *chartInstance->c17_b_gpu_x, *chartInstance->c17_gpu_x,
     *chartInstance->c17_b_gpu_Bn0);
  hipMemcpy(c17_A, *chartInstance->c17_gpu_A, 56UL, hipMemcpyDeviceToHost);
  hipMemcpy(c17_jpvt_t, *chartInstance->c17_gpu_jpvt_t, 56UL,
             hipMemcpyDeviceToHost);
  c17_info_t = LAPACKE_dgeqp3(102, (ptrdiff_t)1, (ptrdiff_t)7, &c17_A[0],
    (ptrdiff_t)1, &c17_jpvt_t[0], &c17_tau);
  c17_A_dirtyOnGpu = false;
  c17_A_dirtyOnCpu = true;
  if ((int32_T)c17_info_t != 0) {
    c17_tau = rtNaN;
    c17_eML_blk_kernel_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_gpu_jpvt, *chartInstance->c17_gpu_A);
    c17_A_dirtyOnCpu = false;
    c17_A_dirtyOnGpu = true;
  } else {
    hipMemcpy(*chartInstance->c17_gpu_jpvt_t, c17_jpvt_t, 56UL,
               hipMemcpyHostToDevice);
    c17_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_gpu_jpvt_t, *chartInstance->c17_gpu_jpvt);
  }

  c17_rankA = 0;
  if (c17_A_dirtyOnGpu) {
    hipMemcpy(c17_A, *chartInstance->c17_gpu_A, 56UL, hipMemcpyDeviceToHost);
  }

  c17_tol = 1.5543122344752192E-14 * muDoubleScalarAbs(c17_A[0]);
  if (!(muDoubleScalarAbs(c17_A[0]) <= c17_tol)) {
    c17_rankA = 1;
  }

  c17_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Th, *chartInstance->c17_b_gpu_B);
  c17_b_info_t = 0;
  if (c17_A_dirtyOnCpu) {
    hipMemcpy(*chartInstance->c17_gpu_A, c17_A, 56UL, hipMemcpyHostToDevice);
  }

  hipMemcpy(chartInstance->c17_gpu_tau, &c17_tau, 8UL, hipMemcpyHostToDevice);
  hipsolverDnDormqr_bufferSize(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT,
    HIPBLAS_OP_T, 1, 7, 1, (double *)&(*chartInstance->c17_gpu_A)[0], 1, (double *)
    chartInstance->c17_gpu_tau, (double *)&(*chartInstance->c17_b_gpu_B)[0], 1,
    getCuSolverWorkspaceReq());
  setCuSolverWorkspaceTypeSize(8);
  cusolverInitWorkspace();
  hipMemcpy(chartInstance->c17_gpu_info_t, &c17_b_info_t, 4UL,
             hipMemcpyHostToDevice);
  hipsolverDnDormqr(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, 1,
                   7, 1, (double *)&(*chartInstance->c17_gpu_A)[0], 1, (double *)
                   chartInstance->c17_gpu_tau, (double *)
                   &(*chartInstance->c17_b_gpu_B)[0], 1, (real_T *)
                   getCuSolverWorkspaceBuff(), *getCuSolverWorkspaceReq(),
                   chartInstance->c17_gpu_info_t);
  hipMemcpy(&c17_b_info_t, chartInstance->c17_gpu_info_t, 4UL,
             hipMemcpyDeviceToHost);
  if (c17_b_info_t < 0) {
    c17_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_b_gpu_B);
  }

  c17_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_jpvt, c17_rankA, *chartInstance->c17_gpu_A,
     *chartInstance->c17_b_gpu_B, *chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel15<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_change, *chartInstance->c17_gpu_Y);
  c17_eML_blk_kernel_kernel16<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Y, *chartInstance->c17_gpu_K,
     *chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_K);
  c17_eML_blk_kernel_kernel18<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_B);
  hipMemcpy(*chartInstance->c17_gpu_alpha, c17_b_alpha, 16UL,
             hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_beta, c17_b_beta, 16UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Bn0, *chartInstance->c17_gpu_alpha,
     *chartInstance->c17_b_gpu_Bp0, *chartInstance->c17_b_gpu_Kp0,
     *chartInstance->c17_gpu_change, *chartInstance->c17_gpu_beta,
     *chartInstance->c17_b_gpu_Kn0, *chartInstance->c17_b_gpu_x,
     *chartInstance->c17_gpu_x, *chartInstance->c17_gpu_K,
     *chartInstance->c17_gpu_B);
  hipMemcpy(c17_b_K, *chartInstance->c17_gpu_K, 392UL, hipMemcpyDeviceToHost);
  hipMemcpy(c17_b_B, *chartInstance->c17_gpu_B, 392UL, hipMemcpyDeviceToHost);
}

static void c17_emlrt_marshallIn(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c17_b_B, const char_T *c17_identifier, real_T
  c17_y[49])
{
  emlrtMsgIdentifier c17_thisId;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c17_b_B), &c17_thisId, c17_y);
  sf_mex_destroy(&c17_b_B);
}

static void c17_b_emlrt_marshallIn(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId,
  real_T c17_y[49])
{
  real_T c17_dv[49];
  int32_T c17_i;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), c17_dv, 1, 0, 0U, 1, 0U, 2, 7,
                7);
  for (c17_i = 0; c17_i < 49; c17_i++) {
    c17_y[c17_i] = c17_dv[c17_i];
  }

  sf_mex_destroy(&c17_u);
}

static uint8_T c17_c_emlrt_marshallIn
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c17_b_is_active_c17_sliding_controller_rbdl, const char_T *c17_identifier)
{
  emlrtMsgIdentifier c17_thisId;
  uint8_T c17_y;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_y = c17_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c17_b_is_active_c17_sliding_controller_rbdl), &c17_thisId);
  sf_mex_destroy(&c17_b_is_active_c17_sliding_controller_rbdl);
  return c17_y;
}

static uint8_T c17_d_emlrt_marshallIn
  (SFc17_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c17_u, const emlrtMsgIdentifier *c17_parentId)
{
  uint8_T c17_b_u;
  uint8_T c17_y;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), &c17_b_u, 1, 3, 0U, 0, 0U, 0);
  c17_y = c17_b_u;
  sf_mex_destroy(&c17_u);
  return c17_y;
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel1
  (real_T c17_b_Kp0[49])
{
  uint64_T c17_threadId;
  int32_T c17_i;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i = (int32_T)c17_threadId;
  if (c17_i < 49) {
    c17_b_Kp0[c17_i] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel2(const
  real_T c17_b_Kp0[7], real_T c17_c_Kp0[49])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_c_Kp0[c17_j + 7 * c17_j] = c17_b_Kp0[c17_j];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel3
  (real_T c17_b_Kn0[49])
{
  uint64_T c17_threadId;
  int32_T c17_i1;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i1 = (int32_T)c17_threadId;
  if (c17_i1 < 49) {
    c17_b_Kn0[c17_i1] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel4(const
  real_T c17_b_Kn0[7], real_T c17_c_Kn0[49])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_c_Kn0[c17_j + 7 * c17_j] = c17_b_Kn0[c17_j];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel5
  (real_T c17_b_Bp0[49])
{
  uint64_T c17_threadId;
  int32_T c17_i2;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i2 = (int32_T)c17_threadId;
  if (c17_i2 < 49) {
    c17_b_Bp0[c17_i2] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel6(const
  real_T c17_b_Bp0[7], real_T c17_c_Bp0[49])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_c_Bp0[c17_j + 7 * c17_j] = c17_b_Bp0[c17_j];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel7
  (real_T c17_b_Bn0[49])
{
  uint64_T c17_threadId;
  int32_T c17_i3;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i3 = (int32_T)c17_threadId;
  if (c17_i3 < 49) {
    c17_b_Bn0[c17_i3] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel8(const
  real_T c17_b_Tid[7], const real_T c17_b_qd[7], const real_T c17_b_Th[7], const
  real_T c17_b_Bn0[7], ptrdiff_t c17_jpvt_t[7], real_T c17_A[7], real_T c17_x[7],
  real_T c17_b_x[7], real_T c17_c_Bn0[49])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_c_Bn0[c17_j + 7 * c17_j] = c17_b_Bn0[c17_j];
    c17_b_x[c17_j] = c17_b_Th[c17_j];
    c17_b_x[c17_j] = mwGpuSign(c17_b_x[c17_j]);
    c17_x[c17_j] = c17_b_qd[c17_j];
    c17_x[c17_j] = mwGpuSign(c17_x[c17_j]);
    c17_A[c17_j] = c17_b_Tid[c17_j];
    c17_jpvt_t[c17_j] = (ptrdiff_t)0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel9(const
  ptrdiff_t c17_jpvt_t[7], int32_T c17_jpvt[7])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    c17_jpvt[c17_k] = (int32_T)c17_jpvt_t[c17_k];
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel10
  (int32_T c17_jpvt[7], real_T c17_A[7])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_A[c17_j] = HIP_NAN;
    c17_jpvt[c17_j] = c17_j + 1;
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel11
  (real_T c17_change[49])
{
  uint64_T c17_threadId;
  int32_T c17_i4;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i4 = (int32_T)c17_threadId;
  if (c17_i4 < 49) {
    c17_change[c17_i4] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel12(
  const real_T c17_b_Th[7], real_T c17_b_B[7])
{
  uint64_T c17_threadId;
  int32_T c17_i5;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i5 = (int32_T)c17_threadId;
  if (c17_i5 < 7) {
    c17_b_B[c17_i5] = c17_b_Th[c17_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel13
  (real_T c17_b_B[7])
{
  uint64_T c17_threadId;
  int32_T c17_i6;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i6 = (int32_T)c17_threadId;
  if (c17_i6 < 7) {
    c17_b_B[c17_i6] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel14(
  const int32_T c17_jpvt[7], const int32_T c17_rankA, real_T c17_A[7], real_T
  c17_b_B[7], real_T c17_change[49])
{
  uint64_T c17_threadId;
  int32_T c17_i;
  int32_T c17_j;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    for (c17_i = 0; c17_i < c17_rankA; c17_i++) {
      c17_change[(c17_jpvt[0] + 7 * c17_k) - 1] = c17_b_B[c17_k];
    }

    for (c17_j = 0; c17_j < c17_rankA; c17_j++) {
      c17_change[(c17_jpvt[c17_rankA - 1] + 7 * c17_k) - 1] /= c17_A[(c17_rankA
        + c17_rankA) - 2];
    }
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel15(
  const real_T c17_change[49], real_T c17_Y[49])
{
  uint64_T c17_threadId;
  int32_T c17_i7;
  int32_T c17_i8;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i8 = (int32_T)(c17_threadId % 7UL);
  c17_i7 = (int32_T)((c17_threadId - (uint64_T)c17_i8) / 7UL);
  if ((c17_i7 < 7) && (c17_i8 < 7)) {
    c17_Y[c17_i8 + 7 * c17_i7] = c17_change[c17_i7 + 7 * c17_i8];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel16(
  const real_T c17_Y[49], real_T c17_b_K[49], real_T c17_change[49])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 49) {
    c17_change[c17_k] = fabs(c17_Y[c17_k]);
    c17_b_K[c17_k] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel17
  (real_T c17_b_K[49])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    c17_b_K[c17_k + 7 * c17_k] = 1.0;
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel18
  (real_T c17_b_B[49])
{
  uint64_T c17_threadId;
  int32_T c17_i9;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i9 = (int32_T)c17_threadId;
  if (c17_i9 < 49) {
    c17_b_B[c17_i9] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel19(
  const real_T c17_b_Bn0[49], const real_T c17_b_alpha[2], const real_T
  c17_b_Bp0[49], const real_T c17_b_Kp0[49], const real_T c17_change[49], const
  real_T c17_b_beta[2], const real_T c17_b_Kn0[49], const real_T c17_x[7], const
  real_T c17_b_x[7], real_T c17_b_K[49], real_T c17_b_B[49])
{
  uint64_T c17_threadId;
  int32_T c17_ii;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_ii = (int32_T)c17_threadId;
  if (c17_ii < 7) {
    c17_b_B[c17_ii + 7 * c17_ii] = 1.0;
    if (c17_b_x[c17_ii] == c17_x[c17_ii]) {
      c17_b_K[c17_ii + 7 * c17_ii] = c17_b_Kp0[c17_ii + 7 * c17_ii] +
        c17_b_beta[0] * c17_change[c17_ii];
      c17_b_B[c17_ii + 7 * c17_ii] = c17_b_Bp0[c17_ii + 7 * c17_ii] +
        c17_b_alpha[0] * c17_change[c17_ii];
    } else {
      c17_b_K[c17_ii + 7 * c17_ii] = c17_b_Kn0[c17_ii + 7 * c17_ii] -
        c17_b_beta[1] * c17_change[c17_ii];
      c17_b_B[c17_ii + 7 * c17_ii] = c17_b_Bn0[c17_ii + 7 * c17_ii] -
        c17_b_alpha[1] * c17_change[c17_ii];
    }
  }
}

static void init_dsm_address_info(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc17_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c17_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c17_Kp0 = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c17_Kn0 = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_Bp0 = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c17_Bn0 = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c17_beta = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
  chartInstance->c17_alpha = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
  chartInstance->c17_Tid = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 6);
  chartInstance->c17_K = (real_T (*)[49])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_Th = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 7);
  chartInstance->c17_qd = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 8);
  chartInstance->c17_B = (real_T (*)[49])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c17_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(184866213U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3943974275U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1621223579U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3836141733U);
}

mxArray *sf_c17_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,1);
  mxSetCell(mxcell3p, 0, mxCreateString("coder.internal.lapack.LAPACKApi"));
  return(mxcell3p);
}

mxArray *sf_c17_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("late");
  mxArray *hiddenFallbackReason = mxCreateString("ir_function_calls");
  mxArray *incompatibleSymbol = mxCreateString("__gpu_kernel");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c17_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c17_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiA+wMTAwAakOYCYiQECWKF8RiBmhtIQcRa4uAIQl1QWpILEi4uSPVOAdF5iLpi"
    "fWFrhmZeWDzbfggFhPhsW8xmRzOeEikPAB3vK9Cs5gPQbIOlnIaBfAMhygoYLLHwGzv0iZLnfm2"
    "ruV3CgTD/E/ggC7ldFcT+En1kcn5hcklmWGp9saB5fnJOZkpmXHp+cn1dSlJ+Tk1oUX5SUkoPkT"
    "wAKJCGO"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c17_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sZ5qVxZToNvS2ZkiuiWRnEB";
}

static void sf_opaque_initialize_c17_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c17_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c17_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c17_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c17_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c17_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c17_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc17_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c17_sliding_controller_rbdl
      ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc17_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c17_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c17_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc17_sliding_controller_rbdl
    ((SFc17_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c17_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c17_sliding_controller_rbdl
      ((SFc17_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc17_sliding_controller_rbdl
      ((SFc17_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c17_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV8tu20YUpRQ3iIEmENoA3RRoUCBIlm3QoF0ljl4oUTs2QtkpvBHGwytxoOEMMw/a/od200U",
    "/oP2JLvsZ/Yds8gm5Q1KyQpFUbDVpWpQATQ157plzH3Nn7LX8PQ+vW3h3PvW86/i8gXfby6+Pin",
    "Fr6c7fb3n3ivHPaCRsfEAUibXXeAkSwzPQklvDpPDFRFbCmJiAAkERm0hl6tg0iy1nYja0gjo+/",
    "TxiNAoiaXnYRVsS7gt+jmyJNQfI02cKqBkChCZS0k6jISfThWJlTnsR0Jm2cZMLGkxgEydL71lu",
    "WMJhcAbUF9oQVKwvtAWGGOiZs1o3nac6mANlnHBGRKW3EdEBJBhgA4dJiH/3rUGnyjAaEWW6EJE",
    "U9C6bZZxSQJmTafxwwgQxUjHCBzHvOcNVbQcc9ezJEHhDQFBbVwGZJZIJU5//YIieDgQ54dCHEz",
    "utZwvghXXJP2JwCqo2bpOeTEGRKeyL2kmzgAzOsmwtqmQVZlgMR0Q9oZg/DWFt9WLl6IBgnmCEF",
    "nUwyJz09UixFMNby2Zj31XmuiVj4zzZeh0sYxuk0JSFBduQih7hXNfCRjLZhRR4xtonhjTDctZq",
    "nNYsHEkMsCvv+tVgBcPEF7CeFCGrTFdaAmR95yk2ljeR1Goj4x4Wb393d/XzKswXBtSEUKjqAoo",
    "wDRizLLz1bCHTLvcIRFUmk1cFzitkHcrTEyv6p1LNMCYNTeTCBZfRWmCsp5hLXAmHGhdNE8zlch",
    "2OEhpB6BoM47CHywaxFTHRrrU9wXWXMnPeB00VSyqyanHVYRsauII6T+BQzIQ8FUMl46Do8Xl4A",
    "bDaiBJMTLvYv9T5EGevSoXbz77yLvazj99iP5vblZ/3l3haFTze0tPhd5bw2+038duledvzd/Or",
    "hudmad6tEo/DdfD+45OHv91oP/B//f0n/5tfXtJyHMp6Wit6WtlvZ5dcu9x54FYx/nzeeBcLKV2",
    "pX4f9fknXVgX/Z0v8nWKsjx++ODo7HsmnafDgeMYse/5MDLoZ35/tZr3XSnrn7++4HQCrLqtXRf",
    "2wOKi4MbH59u34v1vSe31NPLaL9/n16vFm9l/ulPO4tca+g7+6pTr+5/TfvpL+H/42/Xd2NrPP5",
    "/9xjf67pXq9m51TxsR1PxjTr78da85CbF1jKoVRknNQY3US8pKfV12vl7Xz3rPdv0Xn/3bvPu9v",
    "sx9vX9GuveH+/77sNvXvsueSDw3ftB94JXznA/Zj0/Piu8b/5V3uHPdFMX60+NetFzEeVpzii89",
    "4JJ9Uff0P1OlrU3q3Bw==",
    ""
  };

  static char newstr [1297] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c17_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c17_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2721387192U));
  ssSetChecksum1(S,(1228014088U));
  ssSetChecksum2(S,(1233953686U));
  ssSetChecksum3(S,(1676382772U));
}

static void mdlRTW_c17_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c17_sliding_controller_rbdl(SimStruct *S)
{
  SFc17_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc17_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc17_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc17_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc17_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c17_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c17_JITStateAnimation,
    chartInstance->c17_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c17_sliding_controller_rbdl(chartInstance);
}

void c17_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c17_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c17_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c17_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c17_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
