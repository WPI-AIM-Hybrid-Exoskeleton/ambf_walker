/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c7_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c7_do_animation_call_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c7_st);
static void sf_gateway_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c7_eML_blk_kernel(SFc7_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c7_b_q[2], real_T c7_b_y[4]);
static void c7_emlrt_marshallIn(SFc7_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c7_b_y, const char_T *c7_identifier, real_T
  c7_c_y[4]);
static void c7_b_emlrt_marshallIn(SFc7_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId,
  real_T c7_b_y[4]);
static uint8_T c7_c_emlrt_marshallIn(SFc7_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c7_b_is_active_c7_sliding_controller_rbdl, const
  char_T *c7_identifier);
static uint8_T c7_d_emlrt_marshallIn(SFc7_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId);
static void init_dsm_address_info(SFc7_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc7_sliding_controller_rbdlInstanceStruct *
  chartInstance);

/* Function Definitions */
static void initialize_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c7_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c7_is_active_c7_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c7_do_animation_call_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c7_b_y = NULL;
  const mxArray *c7_c_y = NULL;
  const mxArray *c7_d_y = NULL;
  const mxArray *c7_st;
  c7_st = NULL;
  c7_b_y = NULL;
  sf_mex_assign(&c7_b_y, sf_mex_createcellmatrix(2, 1), false);
  c7_c_y = NULL;
  sf_mex_assign(&c7_c_y, sf_mex_create("y", *chartInstance->c7_y, 0, 0U, 1U, 0U,
    2, 2, 2), false);
  sf_mex_setcell(c7_b_y, 0, c7_c_y);
  c7_d_y = NULL;
  sf_mex_assign(&c7_d_y, sf_mex_create("y",
    &chartInstance->c7_is_active_c7_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c7_b_y, 1, c7_d_y);
  sf_mex_assign(&c7_st, c7_b_y, false);
  return c7_st;
}

static void set_sim_state_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c7_st)
{
  const mxArray *c7_u;
  c7_u = sf_mex_dup(c7_st);
  c7_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c7_u, 0)), "y",
                      *chartInstance->c7_y);
  chartInstance->c7_is_active_c7_sliding_controller_rbdl = c7_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c7_u, 1)),
     "is_active_c7_sliding_controller_rbdl");
  sf_mex_destroy(&c7_u);
  sf_mex_destroy(&c7_st);
}

static void sf_gateway_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c7_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c7_eML_blk_kernel(chartInstance, *chartInstance->c7_q, *chartInstance->c7_y);
  c7_do_animation_call_c7_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c7_errCode;
  c7_errCode = hipGetLastError();
  if (c7_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c7_errCode, (char_T *)hipGetErrorName
                       (c7_errCode), (char_T *)hipGetErrorString(c7_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c7_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc7_sliding_controller_rbdl
  (SFc7_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c7_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c7_nameCaptureInfo = NULL;
  c7_nameCaptureInfo = NULL;
  sf_mex_assign(&c7_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c7_nameCaptureInfo;
}

static void c7_eML_blk_kernel(SFc7_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c7_b_q[2], real_T c7_b_y[4])
{
  c7_b_y[0] = -0.324 * muDoubleScalarSin(c7_b_q[0]);
  c7_b_y[2] = 0.0;
  c7_b_y[1] = 0.324 * muDoubleScalarCos(c7_b_q[0]);
  c7_b_y[3] = 0.0;
}

static void c7_emlrt_marshallIn(SFc7_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c7_b_y, const char_T *c7_identifier, real_T
  c7_c_y[4])
{
  emlrtMsgIdentifier c7_thisId;
  c7_thisId.fIdentifier = const_cast<const char_T *>(c7_identifier);
  c7_thisId.fParent = NULL;
  c7_thisId.bParentIsCell = false;
  c7_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c7_b_y), &c7_thisId, c7_c_y);
  sf_mex_destroy(&c7_b_y);
}

static void c7_b_emlrt_marshallIn(SFc7_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId,
  real_T c7_b_y[4])
{
  real_T c7_dv[4];
  int32_T c7_i;
  sf_mex_import(c7_parentId, sf_mex_dup(c7_u), c7_dv, 1, 0, 0U, 1, 0U, 2, 2, 2);
  for (c7_i = 0; c7_i < 4; c7_i++) {
    c7_b_y[c7_i] = c7_dv[c7_i];
  }

  sf_mex_destroy(&c7_u);
}

static uint8_T c7_c_emlrt_marshallIn(SFc7_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c7_b_is_active_c7_sliding_controller_rbdl, const
  char_T *c7_identifier)
{
  emlrtMsgIdentifier c7_thisId;
  uint8_T c7_b_y;
  c7_thisId.fIdentifier = const_cast<const char_T *>(c7_identifier);
  c7_thisId.fParent = NULL;
  c7_thisId.bParentIsCell = false;
  c7_b_y = c7_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c7_b_is_active_c7_sliding_controller_rbdl), &c7_thisId);
  sf_mex_destroy(&c7_b_is_active_c7_sliding_controller_rbdl);
  return c7_b_y;
}

static uint8_T c7_d_emlrt_marshallIn(SFc7_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId)
{
  uint8_T c7_b_u;
  uint8_T c7_b_y;
  sf_mex_import(c7_parentId, sf_mex_dup(c7_u), &c7_b_u, 1, 3, 0U, 0, 0U, 0);
  c7_b_y = c7_b_u;
  sf_mex_destroy(&c7_u);
  return c7_b_y;
}

static void init_dsm_address_info(SFc7_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc7_sliding_controller_rbdlInstanceStruct *
  chartInstance)
{
  chartInstance->c7_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c7_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c7_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c7_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(536815311U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1945133011U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2900394615U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(32238838U);
}

mxArray *sf_c7_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c7_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c7_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c7_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfh"
    "UU90P4mcXxicklmWWp8cnm8cU5mSmZeenxyfl5JUX5OTmpRfFFSSk5DEj+BAB5JB2A"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c7_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sFqlSox9r5ekjZXT2w93GAD";
}

static void sf_opaque_initialize_c7_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c7_sliding_controller_rbdl
    ((SFc7_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c7_sliding_controller_rbdl
    ((SFc7_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c7_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c7_sliding_controller_rbdl((SFc7_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c7_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c7_sliding_controller_rbdl((SFc7_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c7_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c7_sliding_controller_rbdl
    ((SFc7_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c7_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c7_sliding_controller_rbdl
    ((SFc7_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c7_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c7_sliding_controller_rbdl
    ((SFc7_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c7_sliding_controller_rbdl(void *
  chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc7_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c7_sliding_controller_rbdl
      ((SFc7_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc7_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc7_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c7_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c7_sliding_controller_rbdl
    ((SFc7_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c7_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c7_sliding_controller_rbdl
    ((SFc7_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc7_sliding_controller_rbdl
    ((SFc7_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c7_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c7_sliding_controller_rbdl
      ((SFc7_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc7_sliding_controller_rbdl
      ((SFc7_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c7_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u00oUdqMWgeCiCiEhISRYIHGXCISADbTND0Q3va1wCohNNB2fxEPGM2Z+kuYF2PEmd3H",
    "390V4AjZseIR7xnbS4NhOS0QFCEuuM/Z3zpxzvvMz9dbaux5el/HeuuJ55/B5Hu+al14b2Xpt7k",
    "7fr3t3svUHFBI22ieKRNqrvASJ4AVoya1hUrRFXxbCmOiDAkERG0tlyrRpFlnOxLBlBXX69KuQ0",
    "dAPpeXBDsqSYE/wCWqLrdlHPQ2mgJoWQGBCJe0gbHEymFmszLgeAh1qG1W5oMH4NnZm6V3LDYs5",
    "NI+AtoU2BC3Wx7b5hhiom6NSN52n2p8CZRRzRkShtyHRPsQYYAMHcYB/96xBp/IwGhJldiAkI9A",
    "dNkx0SgF5nUzjh0MmiJGKEd6MeN0JLtq2z9GeXRkArwgI2rajgAxjyYQp599voadNQQ45NODQDs",
    "q1+fDOOvJfMhiDKo1bvy5HoMgA9kTppklAmkcJW7MsWYQZFsFLorYp8qchKM1ezBztE+QJuihRB",
    "oPEybbuKjbC8JZqs1HbZeaykrFRSrZeBku0NUdQxcJMW4uKOuFcl8K6Mu7ACHiitUEMqYalWotx",
    "WrOgKzHALr3Lq8EKhsRnsLoUASuka5QDJH3nb2wsXyOp1UZGdUzeRqez+HkR1hYGVJ9QKOoCijA",
    "NGLMkvOXaAqYd9whEq0xiXhE4zZBlKE/3rWiMpRpiTCqayLELjtFSYKQHyCVWwoHGoqmCOS6X4S",
    "ihIQSuwTAOu1g2iC2IiXatbRvrbsTMpAGaKhYXsGqx6rANNV1CTWI4EEMhx6KlZORnPT4NLwBmG",
    "1GCicEO9i81aeHuRVS4eXbXO55nl04wz6Zy+eefc3rWCvR4c8/8vhdq1fvW8NfadP7Oyf2R22c9",
    "J+dwm3hf/+fBp//u6w1+8a/O+8+vOqvs/2/tdPP/cra+MW20s8IZLeSrwz6fs2u9QP+1Of2b2Vq",
    "33nFfHj1WD2D49s3r7r3x4/vPthspP0vsreXsnb6/5To+ZlmSn4q2g+xg4tbEpuPa6X80Z++5Jf",
    "G4kL1Pry9PV5O/upXncX2J/Cb+muTy9tv3v7W1mny6/+sl9t/O8X07mes94roF9OjDnuYswErvU",
    "SmMkpyD6qnDgOfr81vz/bRy3hnL/Sx2/pb7/ryfZH6dtZx3xnKr+nfaOf6j4avmgZfDb/7Afqx6",
    "vvre+I/e6c5BN7P1k9m/OvWQ8aDg1Jt9xiNsv+jrL5Cn/wMAbLLO",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c7_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c7_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3862277146U));
  ssSetChecksum1(S,(91435963U));
  ssSetChecksum2(S,(1279986540U));
  ssSetChecksum3(S,(1280830342U));
}

static void mdlRTW_c7_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c7_sliding_controller_rbdl(SimStruct *S)
{
  SFc7_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc7_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc7_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc7_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc7_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c7_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c7_JITStateAnimation,
    chartInstance->c7_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c7_sliding_controller_rbdl(chartInstance);
}

void c7_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c7_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c7_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c7_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c7_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
