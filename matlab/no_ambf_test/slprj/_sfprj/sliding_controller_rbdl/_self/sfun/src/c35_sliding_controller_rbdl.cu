#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c35_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c35_do_animation_call_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c35_st);
static void sf_gateway_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c35_eML_blk_kernel(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c35_b_u[7], real_T c35_b_y[7]);
static void c35_emlrt_marshallIn(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c35_b_y, const char_T *c35_identifier, real_T
  c35_c_y[7]);
static void c35_b_emlrt_marshallIn(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c35_b_u, const emlrtMsgIdentifier *c35_parentId,
  real_T c35_b_y[7]);
static uint8_T c35_c_emlrt_marshallIn
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c35_b_is_active_c35_sliding_controller_rbdl, const char_T *c35_identifier);
static uint8_T c35_d_emlrt_marshallIn
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c35_b_u, const emlrtMsgIdentifier *c35_parentId);
static __global__ void c35_eML_blk_kernel_kernel1(real_T c35_b_y[7]);
static void init_dsm_address_info(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c35_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c35_is_active_c35_sliding_controller_rbdl = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c35_gpu_y, 56UL);
}

static void initialize_params_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c35_do_animation_call_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c35_b_y = NULL;
  const mxArray *c35_c_y = NULL;
  const mxArray *c35_d_y = NULL;
  const mxArray *c35_st;
  c35_st = NULL;
  c35_b_y = NULL;
  sf_mex_assign(&c35_b_y, sf_mex_createcellmatrix(2, 1), false);
  c35_c_y = NULL;
  sf_mex_assign(&c35_c_y, sf_mex_create("y", *chartInstance->c35_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c35_b_y, 0, c35_c_y);
  c35_d_y = NULL;
  sf_mex_assign(&c35_d_y, sf_mex_create("y",
    &chartInstance->c35_is_active_c35_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c35_b_y, 1, c35_d_y);
  sf_mex_assign(&c35_st, c35_b_y, false);
  return c35_st;
}

static void set_sim_state_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c35_st)
{
  const mxArray *c35_b_u;
  c35_b_u = sf_mex_dup(c35_st);
  c35_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c35_b_u, 0)),
                       "y", *chartInstance->c35_y);
  chartInstance->c35_is_active_c35_sliding_controller_rbdl =
    c35_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c35_b_u, 1)),
    "is_active_c35_sliding_controller_rbdl");
  sf_mex_destroy(&c35_b_u);
  sf_mex_destroy(&c35_st);
}

static void sf_gateway_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  real_T c35_dv[7];
  real_T c35_dv1[7];
  int32_T c35_i;
  int32_T c35_i1;
  chartInstance->c35_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c35_i = 0; c35_i < 7; c35_i++) {
    c35_dv[c35_i] = (*chartInstance->c35_u)[c35_i];
  }

  c35_eML_blk_kernel(chartInstance, c35_dv, c35_dv1);
  for (c35_i1 = 0; c35_i1 < 7; c35_i1++) {
    (*chartInstance->c35_y)[c35_i1] = c35_dv1[c35_i1];
  }

  c35_do_animation_call_c35_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c35_errCode;
  hipFree(*chartInstance->c35_gpu_y);
  c35_errCode = hipGetLastError();
  if (c35_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c35_errCode, (char_T *)hipGetErrorName
                       (c35_errCode), (char_T *)hipGetErrorString(c35_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c35_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc35_sliding_controller_rbdl
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c35_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c35_nameCaptureInfo = NULL;
  c35_nameCaptureInfo = NULL;
  sf_mex_assign(&c35_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c35_nameCaptureInfo;
}

static void c35_eML_blk_kernel(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c35_b_u[7], real_T c35_b_y[7])
{
  c35_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c35_gpu_y);
  hipMemcpy(c35_b_y, *chartInstance->c35_gpu_y, 56UL, hipMemcpyDeviceToHost);
  c35_b_y[0] = c35_b_u[3];
  c35_b_y[1] = c35_b_u[0];
  c35_b_y[2] = c35_b_u[1];
  c35_b_y[3] = c35_b_u[2];
  c35_b_y[4] = c35_b_u[4];
  c35_b_y[5] = c35_b_u[5];
}

static void c35_emlrt_marshallIn(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c35_b_y, const char_T *c35_identifier, real_T
  c35_c_y[7])
{
  emlrtMsgIdentifier c35_thisId;
  c35_thisId.fIdentifier = const_cast<const char_T *>(c35_identifier);
  c35_thisId.fParent = NULL;
  c35_thisId.bParentIsCell = false;
  c35_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c35_b_y), &c35_thisId,
    c35_c_y);
  sf_mex_destroy(&c35_b_y);
}

static void c35_b_emlrt_marshallIn(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c35_b_u, const emlrtMsgIdentifier *c35_parentId,
  real_T c35_b_y[7])
{
  real_T c35_dv[7];
  int32_T c35_i;
  sf_mex_import(c35_parentId, sf_mex_dup(c35_b_u), c35_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c35_i = 0; c35_i < 7; c35_i++) {
    c35_b_y[c35_i] = c35_dv[c35_i];
  }

  sf_mex_destroy(&c35_b_u);
}

static uint8_T c35_c_emlrt_marshallIn
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c35_b_is_active_c35_sliding_controller_rbdl, const char_T *c35_identifier)
{
  emlrtMsgIdentifier c35_thisId;
  uint8_T c35_b_y;
  c35_thisId.fIdentifier = const_cast<const char_T *>(c35_identifier);
  c35_thisId.fParent = NULL;
  c35_thisId.bParentIsCell = false;
  c35_b_y = c35_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c35_b_is_active_c35_sliding_controller_rbdl), &c35_thisId);
  sf_mex_destroy(&c35_b_is_active_c35_sliding_controller_rbdl);
  return c35_b_y;
}

static uint8_T c35_d_emlrt_marshallIn
  (SFc35_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c35_b_u, const emlrtMsgIdentifier *c35_parentId)
{
  uint8_T c35_b_y;
  uint8_T c35_c_u;
  sf_mex_import(c35_parentId, sf_mex_dup(c35_b_u), &c35_c_u, 1, 3, 0U, 0, 0U, 0);
  c35_b_y = c35_c_u;
  sf_mex_destroy(&c35_b_u);
  return c35_b_y;
}

static __global__ __launch_bounds__(32, 1) void c35_eML_blk_kernel_kernel1
  (real_T c35_b_y[7])
{
  uint64_T c35_threadId;
  int32_T c35_i;
  c35_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c35_i = (int32_T)c35_threadId;
  if (c35_i < 7) {
    c35_b_y[c35_i] = 0.0;
  }
}

static void init_dsm_address_info(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc35_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c35_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c35_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c35_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c35_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3695039179U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(714296334U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2541404152U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3670390759U);
}

mxArray *sf_c35_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c35_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c35_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c35_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cnGpvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB2jR2z"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c35_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "spfioONSEyxKl3rszkFZWnD";
}

static void sf_opaque_initialize_c35_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c35_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c35_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c35_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c35_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c35_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c35_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc35_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c35_sliding_controller_rbdl
      ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc35_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c35_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c35_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc35_sliding_controller_rbdl
    ((SFc35_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c35_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c35_sliding_controller_rbdl
      ((SFc35_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc35_sliding_controller_rbdl
      ((SFc35_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c35_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV91u00gUdqNuBVoWVQgJISEBF2hXXK1UIXEFbZNYRDS0wi2sVlpF0/FJPMp4xsxPmuztXu8",
    "DcMcj8BTcc7e3+wb7CHvGdtLg2E7biAoQllxn7O+cOed852fqrXW6Hl7X8d6+4Xkb+LyCd8PLrh",
    "/y9drcnb1f937O13+jkLDxAVEk1l7tJUgML0FLbg2ToiP6shTGRB8UCIrYRCpTpU2z2HImhr4V1",
    "OnTryNGoyCSloe7KEvCfcEnqC2x5gD1tJgCanyA0ERK2kHkczKYWazMSTMCOtQ2rnNBgwls4szS",
    "XcsNSzi0x0A7QhuCFutT2wJDDDTNuNJN56kOpkAZJ5wRUeptRHQACQbYwFES4t99a9CpIoxGRJl",
    "diMgI9B4bpjqlgKJOpvHDMRPESMUIb8e86QQXbTvgaE9XhsBrAoK27Sogw0QyYar5D3z0tC3IMY",
    "cWHNtBtbYA3lhH/isGJ6Aq49ZvyhEoMoB9UblpGpD2OGVrliWLMMNieEXUDkX+NISV2YuZowOCP",
    "MEhSlTBIHWyow8VG2F4K7XZuOMyc1nJ2DgjWy+DpdraI6hjYabNp6JJONeVsEOZ7MEIeKq1RQyp",
    "h2Vay3Fas/BQYoBdeldXgxUMic9hTSlCVkrXqABI+84LbCyfIqnVRsZNTN7W3t7i50VYRxhQfUK",
    "hrAsowjRgzNLwVmsLmXbcIxCtMql5ZeAsQ5ahPN23onUi1RBjUtNETl1wjFYCYz1ALrESjjQWTR",
    "3McbkMRwmNIHQNhnHoYtkgtiQm2rW2Hay7ETOTFmiqWFLCqsWqwzbUdgk1SeBIDIU8Eb6ScZD3+",
    "Cy8AJhtRAkmBrvYv9TEx93LqHDz7FfvdJ5dO8M8m8oVn7/M6Vkr0ePNPYv7Xm3U79vAX2vT+Tsn",
    "91Nhn/WCnMNt4v3u37/++PH2248PX94PP4z97ir7v2+cb/5fz9d3po12VjijhXx12Gdzdq2X6L8",
    "1p38zX+ukz+T+i6A9GT/nW0r/OfR/fy1aGT9L7G0U7J2+v+c6PmZZmp+KdsL8YOLWxGbj2ul/PG",
    "fvxpJ4XM3fZ9d/T1eTv7ld5HF9ifwm/poU8vbi+9/bXk0+2/+3JfY/KPD9IJ3rPeK6BfTo1qOe5",
    "izEUu9RKYySnIPqqeOQF/y8aL6fV867ZLmvxc7vcp+f97PMr8uW8y5ZblX/zjvHvzR83TzwCvjN",
    "L9iPVc9Xnxv/j3e+c9DdfP1k9q9OM2I8LDn15p/xCNsv+/oN5On/EZK0LA==",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c35_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c35_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1568924064U));
  ssSetChecksum1(S,(3516471563U));
  ssSetChecksum2(S,(1679905322U));
  ssSetChecksum3(S,(1296464070U));
}

static void mdlRTW_c35_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c35_sliding_controller_rbdl(SimStruct *S)
{
  SFc35_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc35_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc35_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc35_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc35_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c35_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c35_JITStateAnimation,
    chartInstance->c35_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c35_sliding_controller_rbdl(chartInstance);
}

void c35_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c35_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c35_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c35_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c35_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
