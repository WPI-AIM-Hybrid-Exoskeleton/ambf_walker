#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c26_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c26_do_animation_call_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_st);
static void sf_gateway_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c26_emlrt_marshallIn(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c26_b_msg, const char_T *c26_identifier,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_y);
static void c26_b_emlrt_marshallIn(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c26_u, const emlrtMsgIdentifier *c26_parentId,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_y);
static void c26_c_emlrt_marshallIn(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c26_u, const emlrtMsgIdentifier *c26_parentId,
  uint8_T c26_y[8]);
static c26_SL_Bus_ROSVariableLengthArrayInfo c26_d_emlrt_marshallIn
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_u, const emlrtMsgIdentifier *c26_parentId);
static uint32_T c26_e_emlrt_marshallIn
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_u, const emlrtMsgIdentifier *c26_parentId);
static void c26_f_emlrt_marshallIn(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c26_u, const emlrtMsgIdentifier *c26_parentId,
  real_T c26_y[7]);
static uint8_T c26_g_emlrt_marshallIn
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_b_is_active_c26_sliding_controller_rbdl, const char_T *c26_identifier);
static uint8_T c26_h_emlrt_marshallIn
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_u, const emlrtMsgIdentifier *c26_parentId);
static void c26_eML_blk_kernel(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c26_b_blankMsg);
static __global__ void c26_sf_gateway_c26_sliding_controller_rbdl_kernel1
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c26_b_blankMsg,
   c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg);
static __global__ void c26_sf_gateway_c26_sliding_controller_rbdl_kernel2
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c26_b_blankMsg,
   c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg);
static __global__ void c26_sf_gateway_c26_sliding_controller_rbdl_kernel3
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg,
   c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_r);
static __global__ void c26_sf_gateway_c26_sliding_controller_rbdl_kernel4
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg,
   c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_r);
static __global__ void c26_sf_gateway_c26_sliding_controller_rbdl_kernel5(const
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_r,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg);
static __global__ void c26_sf_gateway_c26_sliding_controller_rbdl_kernel6(const
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_r,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg);
static __global__ void c26_eML_blk_kernel_kernel7
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c26_b_blankMsg);
static __global__ void c26_eML_blk_kernel_kernel8(const uint8_T c26_uv[3],
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c26_b_blankMsg);
static __global__ void c26_eML_blk_kernel_kernel9
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c26_b_blankMsg);
static void init_dsm_address_info(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c26_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c26_is_active_c26_sliding_controller_rbdl = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c26_gpu_uv, 3UL);
  hipMalloc(&chartInstance->c26_gpu_blankMsg, 208UL);
}

static void initialize_params_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c26_do_animation_call_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  static const char_T *c26_sv[8] = { "ModelName", "ModelName_SL_Info", "Q",
    "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  static const char_T *c26_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c26_b_y = NULL;
  const mxArray *c26_c_y = NULL;
  const mxArray *c26_d_y = NULL;
  const mxArray *c26_e_y = NULL;
  const mxArray *c26_f_y = NULL;
  const mxArray *c26_g_y = NULL;
  const mxArray *c26_h_y = NULL;
  const mxArray *c26_i_y = NULL;
  const mxArray *c26_j_y = NULL;
  const mxArray *c26_k_y = NULL;
  const mxArray *c26_l_y = NULL;
  const mxArray *c26_m_y = NULL;
  const mxArray *c26_n_y = NULL;
  const mxArray *c26_o_y = NULL;
  const mxArray *c26_p_y = NULL;
  const mxArray *c26_q_y = NULL;
  const mxArray *c26_r_y = NULL;
  const mxArray *c26_s_y = NULL;
  const mxArray *c26_st;
  const mxArray *c26_y = NULL;
  real_T c26_d_u[7];
  int32_T c26_i;
  int32_T c26_i1;
  int32_T c26_i2;
  int32_T c26_i3;
  uint32_T c26_b_u;
  uint32_T c26_c_u;
  uint32_T c26_e_u;
  uint32_T c26_f_u;
  uint32_T c26_g_u;
  uint32_T c26_h_u;
  uint32_T c26_i_u;
  uint32_T c26_j_u;
  uint32_T c26_u_CurrentLength;
  uint32_T c26_u_ReceivedLength;
  uint8_T c26_u[8];
  c26_st = NULL;
  c26_y = NULL;
  sf_mex_assign(&c26_y, sf_mex_createcellmatrix(2, 1), false);
  c26_b_y = NULL;
  sf_mex_assign(&c26_b_y, sf_mex_createstruct("structure", 8, c26_sv, 2, 1, 1),
                false);
  for (c26_i = 0; c26_i < 8; c26_i++) {
    c26_u[c26_i] = ((uint8_T *)&((char_T *)chartInstance->c26_msg)[0])[c26_i];
  }

  c26_c_y = NULL;
  sf_mex_assign(&c26_c_y, sf_mex_create("y", c26_u, 3, 0U, 1U, 0U, 1, 8), false);
  sf_mex_setfieldbynum(c26_b_y, 0, "ModelName", c26_c_y, 0);
  c26_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [8])[0];
  c26_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [8])[4];
  c26_d_y = NULL;
  sf_mex_assign(&c26_d_y, sf_mex_createstruct("structure", 2, c26_sv1, 2, 1, 1),
                false);
  c26_b_u = c26_u_CurrentLength;
  c26_e_y = NULL;
  sf_mex_assign(&c26_e_y, sf_mex_create("y", &c26_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c26_d_y, 0, "CurrentLength", c26_e_y, 0);
  c26_c_u = c26_u_ReceivedLength;
  c26_f_y = NULL;
  sf_mex_assign(&c26_f_y, sf_mex_create("y", &c26_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c26_d_y, 0, "ReceivedLength", c26_f_y, 1);
  sf_mex_setfieldbynum(c26_b_y, 0, "ModelName_SL_Info", c26_d_y, 1);
  for (c26_i1 = 0; c26_i1 < 7; c26_i1++) {
    c26_d_u[c26_i1] = ((real_T *)&((char_T *)chartInstance->c26_msg)[16])[c26_i1];
  }

  c26_g_y = NULL;
  sf_mex_assign(&c26_g_y, sf_mex_create("y", c26_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c26_b_y, 0, "Q", c26_g_y, 2);
  c26_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [72])[0];
  c26_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [72])[4];
  c26_h_y = NULL;
  sf_mex_assign(&c26_h_y, sf_mex_createstruct("structure", 2, c26_sv1, 2, 1, 1),
                false);
  c26_e_u = c26_u_CurrentLength;
  c26_i_y = NULL;
  sf_mex_assign(&c26_i_y, sf_mex_create("y", &c26_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c26_h_y, 0, "CurrentLength", c26_i_y, 0);
  c26_f_u = c26_u_ReceivedLength;
  c26_j_y = NULL;
  sf_mex_assign(&c26_j_y, sf_mex_create("y", &c26_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c26_h_y, 0, "ReceivedLength", c26_j_y, 1);
  sf_mex_setfieldbynum(c26_b_y, 0, "Q_SL_Info", c26_h_y, 3);
  for (c26_i2 = 0; c26_i2 < 7; c26_i2++) {
    c26_d_u[c26_i2] = ((real_T *)&((char_T *)chartInstance->c26_msg)[80])[c26_i2];
  }

  c26_k_y = NULL;
  sf_mex_assign(&c26_k_y, sf_mex_create("y", c26_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c26_b_y, 0, "Qd", c26_k_y, 4);
  c26_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [136])[0];
  c26_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [136])[4];
  c26_l_y = NULL;
  sf_mex_assign(&c26_l_y, sf_mex_createstruct("structure", 2, c26_sv1, 2, 1, 1),
                false);
  c26_g_u = c26_u_CurrentLength;
  c26_m_y = NULL;
  sf_mex_assign(&c26_m_y, sf_mex_create("y", &c26_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c26_l_y, 0, "CurrentLength", c26_m_y, 0);
  c26_h_u = c26_u_ReceivedLength;
  c26_n_y = NULL;
  sf_mex_assign(&c26_n_y, sf_mex_create("y", &c26_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c26_l_y, 0, "ReceivedLength", c26_n_y, 1);
  sf_mex_setfieldbynum(c26_b_y, 0, "Qd_SL_Info", c26_l_y, 5);
  for (c26_i3 = 0; c26_i3 < 7; c26_i3++) {
    c26_d_u[c26_i3] = ((real_T *)&((char_T *)chartInstance->c26_msg)[144])
      [c26_i3];
  }

  c26_o_y = NULL;
  sf_mex_assign(&c26_o_y, sf_mex_create("y", c26_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c26_b_y, 0, "Tau", c26_o_y, 6);
  c26_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [200])[0];
  c26_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [200])[4];
  c26_p_y = NULL;
  sf_mex_assign(&c26_p_y, sf_mex_createstruct("structure", 2, c26_sv1, 2, 1, 1),
                false);
  c26_i_u = c26_u_CurrentLength;
  c26_q_y = NULL;
  sf_mex_assign(&c26_q_y, sf_mex_create("y", &c26_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c26_p_y, 0, "CurrentLength", c26_q_y, 0);
  c26_j_u = c26_u_ReceivedLength;
  c26_r_y = NULL;
  sf_mex_assign(&c26_r_y, sf_mex_create("y", &c26_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c26_p_y, 0, "ReceivedLength", c26_r_y, 1);
  sf_mex_setfieldbynum(c26_b_y, 0, "Tau_SL_Info", c26_p_y, 7);
  sf_mex_setcell(c26_y, 0, c26_b_y);
  c26_s_y = NULL;
  sf_mex_assign(&c26_s_y, sf_mex_create("y",
    &chartInstance->c26_is_active_c26_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c26_y, 1, c26_s_y);
  sf_mex_assign(&c26_st, c26_y, false);
  return c26_st;
}

static void set_sim_state_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_st)
{
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 c26_r;
  const mxArray *c26_u;
  int32_T c26_i;
  int32_T c26_i1;
  c26_u = sf_mex_dup(c26_st);
  c26_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c26_u, 0)),
                       "msg", &c26_r);
  for (c26_i = 0; c26_i < 8; c26_i++) {
    ((uint8_T *)&((char_T *)chartInstance->c26_msg)[0])[c26_i] =
      c26_r.ModelName[c26_i];
  }

  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[8])[0] = c26_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[8])[4] = c26_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[72])[0] = c26_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[72])[4] = c26_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[136])[0] = c26_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[136])[4] = c26_r.Qd_SL_Info.ReceivedLength;
  for (c26_i1 = 0; c26_i1 < 7; c26_i1++) {
    ((real_T *)&((char_T *)chartInstance->c26_msg)[16])[c26_i1] = c26_r.Q[c26_i1];
    ((real_T *)&((char_T *)chartInstance->c26_msg)[80])[c26_i1] =
      c26_r.Qd[c26_i1];
    ((real_T *)&((char_T *)chartInstance->c26_msg)[144])[c26_i1] =
      c26_r.Tau[c26_i1];
  }

  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[200])[0] = c26_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[200])[4] = c26_r.Tau_SL_Info.ReceivedLength;
  chartInstance->c26_is_active_c26_sliding_controller_rbdl =
    c26_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c26_u, 1)),
    "is_active_c26_sliding_controller_rbdl");
  sf_mex_destroy(&c26_u);
  sf_mex_destroy(&c26_st);
}

static void sf_gateway_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 c26_r;
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
    *c26_b_gpu_blankMsg;
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_gpu_msg;
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_gpu_r;
  hipMalloc(&c26_gpu_r, 208UL);
  hipMalloc(&c26_gpu_msg, 208UL);
  hipMalloc(&c26_b_gpu_blankMsg, 208UL);
  chartInstance->c26_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(c26_b_gpu_blankMsg, chartInstance->c26_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c26_gpu_msg, chartInstance->c26_msg, 208UL, hipMemcpyHostToDevice);
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c26_b_gpu_blankMsg, c26_gpu_msg);
  hipMemcpy(chartInstance->c26_msg, c26_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c26_blankMsg, c26_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[8])[0] = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_blankMsg)[8])[0];
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[8])[4] = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_blankMsg)[8])[4];
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[72])[0] = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_blankMsg)[72])[0];
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[72])[4] = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_blankMsg)[72])[4];
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[136])[0] = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_blankMsg)[136])[0];
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[136])[4] = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_blankMsg)[136])[4];
  hipMemcpy(c26_b_gpu_blankMsg, chartInstance->c26_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c26_gpu_msg, chartInstance->c26_msg, 208UL, hipMemcpyHostToDevice);
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel2<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c26_b_gpu_blankMsg, c26_gpu_msg);
  hipMemcpy(chartInstance->c26_msg, c26_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c26_blankMsg, c26_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_blankMsg)[200])[4];
  hipMemcpy(c26_gpu_msg, chartInstance->c26_msg, 208UL, hipMemcpyHostToDevice);
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel3<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c26_gpu_msg, c26_gpu_r);
  hipMemcpy(chartInstance->c26_msg, c26_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c26_r, c26_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c26_r.ModelName_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [8])[0];
  c26_r.ModelName_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [8])[4];
  c26_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [72])[0];
  c26_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [72])[4];
  c26_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [136])[0];
  c26_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [136])[4];
  hipMemcpy(c26_gpu_r, &c26_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c26_gpu_msg, chartInstance->c26_msg, 208UL, hipMemcpyHostToDevice);
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel4<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c26_gpu_msg, c26_gpu_r);
  hipMemcpy(chartInstance->c26_msg, c26_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c26_r, c26_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c26_r.Tau_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [200])[0];
  c26_r.Tau_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c26_msg)
    [200])[4];
  c26_eML_blk_kernel(chartInstance, &c26_r);
  hipMemcpy(c26_gpu_r, &c26_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c26_gpu_msg, chartInstance->c26_msg, 208UL, hipMemcpyHostToDevice);
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel5<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c26_gpu_r, c26_gpu_msg);
  hipMemcpy(chartInstance->c26_msg, c26_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[8])[0] = c26_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[8])[4] = c26_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[72])[0] = c26_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[72])[4] = c26_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[136])[0] = c26_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[136])[4] = c26_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c26_gpu_msg, chartInstance->c26_msg, 208UL, hipMemcpyHostToDevice);
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel6<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c26_gpu_r, c26_gpu_msg);
  hipMemcpy(chartInstance->c26_msg, c26_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[200])[0] = c26_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c26_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c26_msg)[200])[4] = c26_r.Tau_SL_Info.ReceivedLength;
  c26_do_animation_call_c26_sliding_controller_rbdl(chartInstance);
  hipFree(c26_b_gpu_blankMsg);
  hipFree(c26_gpu_msg);
  hipFree(c26_gpu_r);
}

static void mdl_start_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c26_errCode;
  hipFree(*chartInstance->c26_gpu_uv);
  hipFree(chartInstance->c26_gpu_blankMsg);
  c26_errCode = hipGetLastError();
  if (c26_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c26_errCode, (char_T *)hipGetErrorName
                       (c26_errCode), (char_T *)hipGetErrorString(c26_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c26_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc26_sliding_controller_rbdl
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c26_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c26_nameCaptureInfo = NULL;
  c26_nameCaptureInfo = NULL;
  sf_mex_assign(&c26_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c26_nameCaptureInfo;
}

static void c26_emlrt_marshallIn(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c26_b_msg, const char_T *c26_identifier,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_y)
{
  emlrtMsgIdentifier c26_thisId;
  c26_thisId.fIdentifier = const_cast<const char_T *>(c26_identifier);
  c26_thisId.fParent = NULL;
  c26_thisId.bParentIsCell = false;
  c26_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c26_b_msg), &c26_thisId,
    c26_y);
  sf_mex_destroy(&c26_b_msg);
}

static void c26_b_emlrt_marshallIn(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c26_u, const emlrtMsgIdentifier *c26_parentId,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_y)
{
  static const char_T *c26_fieldNames[8] = { "ModelName", "ModelName_SL_Info",
    "Q", "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  emlrtMsgIdentifier c26_thisId;
  c26_thisId.fParent = c26_parentId;
  c26_thisId.bParentIsCell = false;
  sf_mex_check_struct(c26_parentId, c26_u, 8, c26_fieldNames, 0U, NULL);
  c26_thisId.fIdentifier = "ModelName";
  c26_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c26_u,
    "ModelName", "ModelName", 0)), &c26_thisId, c26_y->ModelName);
  c26_thisId.fIdentifier = "ModelName_SL_Info";
  c26_y->ModelName_SL_Info = c26_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c26_u, "ModelName_SL_Info", "ModelName_SL_Info", 0)),
    &c26_thisId);
  c26_thisId.fIdentifier = "Q";
  c26_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c26_u, "Q",
    "Q", 0)), &c26_thisId, c26_y->Q);
  c26_thisId.fIdentifier = "Q_SL_Info";
  c26_y->Q_SL_Info = c26_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c26_u, "Q_SL_Info", "Q_SL_Info", 0)), &c26_thisId);
  c26_thisId.fIdentifier = "Qd";
  c26_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c26_u, "Qd",
    "Qd", 0)), &c26_thisId, c26_y->Qd);
  c26_thisId.fIdentifier = "Qd_SL_Info";
  c26_y->Qd_SL_Info = c26_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c26_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c26_thisId);
  c26_thisId.fIdentifier = "Tau";
  c26_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c26_u, "Tau",
    "Tau", 0)), &c26_thisId, c26_y->Tau);
  c26_thisId.fIdentifier = "Tau_SL_Info";
  c26_y->Tau_SL_Info = c26_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c26_u, "Tau_SL_Info", "Tau_SL_Info", 0)), &c26_thisId);
  sf_mex_destroy(&c26_u);
}

static void c26_c_emlrt_marshallIn(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c26_u, const emlrtMsgIdentifier *c26_parentId,
  uint8_T c26_y[8])
{
  int32_T c26_i;
  uint8_T c26_uv[8];
  sf_mex_import(c26_parentId, sf_mex_dup(c26_u), c26_uv, 1, 3, 0U, 1, 0U, 1, 8);
  for (c26_i = 0; c26_i < 8; c26_i++) {
    c26_y[c26_i] = c26_uv[c26_i];
  }

  sf_mex_destroy(&c26_u);
}

static c26_SL_Bus_ROSVariableLengthArrayInfo c26_d_emlrt_marshallIn
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_u, const emlrtMsgIdentifier *c26_parentId)
{
  static const char_T *c26_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c26_SL_Bus_ROSVariableLengthArrayInfo c26_y;
  emlrtMsgIdentifier c26_thisId;
  c26_thisId.fParent = c26_parentId;
  c26_thisId.bParentIsCell = false;
  sf_mex_check_struct(c26_parentId, c26_u, 2, c26_fieldNames, 0U, NULL);
  c26_thisId.fIdentifier = "CurrentLength";
  c26_y.CurrentLength = c26_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c26_u, "CurrentLength", "CurrentLength", 0)), &c26_thisId);
  c26_thisId.fIdentifier = "ReceivedLength";
  c26_y.ReceivedLength = c26_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c26_u, "ReceivedLength", "ReceivedLength", 0)), &c26_thisId);
  sf_mex_destroy(&c26_u);
  return c26_y;
}

static uint32_T c26_e_emlrt_marshallIn
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_u, const emlrtMsgIdentifier *c26_parentId)
{
  uint32_T c26_b_u;
  uint32_T c26_y;
  sf_mex_import(c26_parentId, sf_mex_dup(c26_u), &c26_b_u, 1, 7, 0U, 0, 0U, 0);
  c26_y = c26_b_u;
  sf_mex_destroy(&c26_u);
  return c26_y;
}

static void c26_f_emlrt_marshallIn(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c26_u, const emlrtMsgIdentifier *c26_parentId,
  real_T c26_y[7])
{
  real_T c26_dv[7];
  int32_T c26_i;
  sf_mex_import(c26_parentId, sf_mex_dup(c26_u), c26_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c26_i = 0; c26_i < 7; c26_i++) {
    c26_y[c26_i] = c26_dv[c26_i];
  }

  sf_mex_destroy(&c26_u);
}

static uint8_T c26_g_emlrt_marshallIn
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_b_is_active_c26_sliding_controller_rbdl, const char_T *c26_identifier)
{
  emlrtMsgIdentifier c26_thisId;
  uint8_T c26_y;
  c26_thisId.fIdentifier = const_cast<const char_T *>(c26_identifier);
  c26_thisId.fParent = NULL;
  c26_thisId.bParentIsCell = false;
  c26_y = c26_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c26_b_is_active_c26_sliding_controller_rbdl), &c26_thisId);
  sf_mex_destroy(&c26_b_is_active_c26_sliding_controller_rbdl);
  return c26_y;
}

static uint8_T c26_h_emlrt_marshallIn
  (SFc26_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c26_u, const emlrtMsgIdentifier *c26_parentId)
{
  uint8_T c26_b_u;
  uint8_T c26_y;
  sf_mex_import(c26_parentId, sf_mex_dup(c26_u), &c26_b_u, 1, 3, 0U, 0, 0U, 0);
  c26_y = c26_b_u;
  sf_mex_destroy(&c26_u);
  return c26_y;
}

static void c26_eML_blk_kernel(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c26_b_blankMsg)
{
  static uint8_T c26_uv[3] = { 101U, 120U, 111U };

  hipMemcpy(chartInstance->c26_gpu_blankMsg, c26_b_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  c26_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c26_gpu_blankMsg);
  hipMemcpy(*chartInstance->c26_gpu_uv, c26_uv, 3UL, hipMemcpyHostToDevice);
  c26_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c26_gpu_uv, chartInstance->c26_gpu_blankMsg);
  c26_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c26_gpu_blankMsg);
  hipMemcpy(c26_b_blankMsg, chartInstance->c26_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(32, 1) void
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel1
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c26_b_blankMsg,
   c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg)
{
  uint64_T c26_threadId;
  int32_T c26_i;
  c26_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c26_i = (int32_T)c26_threadId;
  if (c26_i < 8) {
    c26_b_msg->ModelName[c26_i] = c26_b_blankMsg->ModelName[c26_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel2
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c26_b_blankMsg,
   c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg)
{
  uint64_T c26_threadId;
  int32_T c26_i1;
  c26_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c26_i1 = (int32_T)c26_threadId;
  if (c26_i1 < 7) {
    c26_b_msg->Q[c26_i1] = c26_b_blankMsg->Q[c26_i1];
    c26_b_msg->Qd[c26_i1] = c26_b_blankMsg->Qd[c26_i1];
    c26_b_msg->Tau[c26_i1] = c26_b_blankMsg->Tau[c26_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel3
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg,
   c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_r)
{
  uint64_T c26_threadId;
  int32_T c26_i2;
  c26_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c26_i2 = (int32_T)c26_threadId;
  if (c26_i2 < 8) {
    c26_r->ModelName[c26_i2] = c26_b_msg->ModelName[c26_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel4
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg,
   c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_r)
{
  uint64_T c26_threadId;
  int32_T c26_i3;
  c26_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c26_i3 = (int32_T)c26_threadId;
  if (c26_i3 < 7) {
    c26_r->Q[c26_i3] = c26_b_msg->Q[c26_i3];
    c26_r->Qd[c26_i3] = c26_b_msg->Qd[c26_i3];
    c26_r->Tau[c26_i3] = c26_b_msg->Tau[c26_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel5(const
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_r,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg)
{
  uint64_T c26_threadId;
  int32_T c26_i4;
  c26_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c26_i4 = (int32_T)c26_threadId;
  if (c26_i4 < 8) {
    c26_b_msg->ModelName[c26_i4] = c26_r->ModelName[c26_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c26_sf_gateway_c26_sliding_controller_rbdl_kernel6(const
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_r,
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c26_b_msg)
{
  uint64_T c26_threadId;
  int32_T c26_i5;
  c26_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c26_i5 = (int32_T)c26_threadId;
  if (c26_i5 < 7) {
    c26_b_msg->Q[c26_i5] = c26_r->Q[c26_i5];
    c26_b_msg->Qd[c26_i5] = c26_r->Qd[c26_i5];
    c26_b_msg->Tau[c26_i5] = c26_r->Tau[c26_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c26_eML_blk_kernel_kernel7
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c26_b_blankMsg)
{
  uint64_T c26_threadId;
  int32_T c26_tmpIdx;
  c26_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c26_tmpIdx = (int32_T)c26_threadId;
  if (c26_tmpIdx < 1) {
    c26_b_blankMsg->ModelName_SL_Info.CurrentLength = 3U;
  }
}

static __global__ __launch_bounds__(32, 1) void c26_eML_blk_kernel_kernel8(const
  uint8_T c26_uv[3],
  c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c26_b_blankMsg)
{
  uint64_T c26_threadId;
  int32_T c26_i;
  c26_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c26_i = (int32_T)c26_threadId;
  if (c26_i < 3) {
    c26_b_blankMsg->ModelName[c26_i] = c26_uv[c26_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c26_eML_blk_kernel_kernel9
  (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c26_b_blankMsg)
{
  uint64_T c26_threadId;
  int32_T c26_tmpIdx;
  c26_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c26_tmpIdx = (int32_T)c26_threadId;
  if (c26_tmpIdx < 1) {
    c26_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
    c26_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
    c26_b_blankMsg->Tau_SL_Info.CurrentLength = 7U;
  }
}

static void init_dsm_address_info(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc26_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c26_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c26_blankMsg =
    (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c26_msg =
    (c26_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c26_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(590713265U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2431965276U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(4137784681U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(63624825U);
}

mxArray *sf_c26_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c26_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c26_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c26_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvyqK+yH8zOL4xOSSzLLU+GQjs/jinMyUzLz0+OT8vJKi/Jyc1KL4oqSUHFj8AwEAPaYeh"
    "Q=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c26_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sd6D4Ot9C8kffXwxtaQFSbB";
}

static void sf_opaque_initialize_c26_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c26_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c26_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c26_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c26_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c26_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c26_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc26_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c26_sliding_controller_rbdl
      ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc26_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c26_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c26_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc26_sliding_controller_rbdl
    ((SFc26_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c26_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c26_sliding_controller_rbdl
      ((SFc26_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc26_sliding_controller_rbdl
      ((SFc26_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c26_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV09v00gUd9MuAolFFUKCAxJcqt3jaoUQXKCNk4hILe2u28Jpo8n4JR5lPGPmT9J+AW4c4Hs",
    "g8Uk48xH2sIeV9gvsG9tJU8d2KBEVu8KS6479e2/ee7/3Z+Ktdfc8vG7gvX3T867g8yreDS+7fs",
    "jXa3N39n7D+ylfv0EhYeMDokisvdpLkBh+By25NUyKrhjIUhgTA1AgKGITqUyVNs1iy5kYdaygT",
    "p9+ETEaBZG0PGyiLAn3BT9FbYk1B6inxRRQ0wEITaSkHUYdToYzi5WZ+BHQkbZxnQsaTGATZ5be",
    "s9ywhEP7BGhXaEPQYn1mW2CIAd+cVLrpPNXBFCjjhDMiSr2NiA4gwQAbOEpC/LtvDTpVhNGIKNO",
    "EiIxB77JRqlMKKOpkGj/0mSBGKkZ4O+a+E1y07YCjPXsyBF4TELStqYCMEsmEqeY/6KCnbUH6HF",
    "rQt8NqbQG8so78YwYTUJVxG/hyDIoMYV9UbpoGpH2SsjXLkkWYYTEcE7VDkT8NYWX2YubogCBPc",
    "IgSVTBInezqQ8XGGN5KbTbuusxcVjI2zsjWy2CptvYY6liYaetQ4RPOdSXsUCa7MAaeam0RQ+ph",
    "mdZynNYsPJQYYJfe1dVgBUPic5gvRchK6RoXAGnfeY6N5TySWm1k7GPytnZ3Fz8vwrrCgBoQCmV",
    "dQBGmAWOWhrdaW8i04x6BaJVJzSsDZxmyDOXpgRWtiVQjjElNEzlzwTFaCYz1ELnESjjSWDR1MM",
    "flMhwlNILQNRjGYQ/LBrElMdGute1g3Y2ZOW2BpoolJaxarDpsQ22XUKcJHImRkBPRUTIO8h6fh",
    "RcAs40owcSwif1LnXZw9zIq3Dz7xTubZ9c/Y55N5YrPn+f0rJXo8eaexX2vNer3beB/a9P5Oyf3",
    "Y2GfjYKcw23ivbMe/9V9/f7PO62PjX/+ePd2lf0/NC42/2/k67vTRjsrnPFCvjrsszm7Nkr0357",
    "Tv5mvdfiw9WDfPPYfjQaDl5MTQ37rBP1mxs8SexsFe6fv77uOj1mW5qei3TA/mLg1sdm4dvofzd",
    "l7ZUk8ruXvs+vvp6vJ39ou8lgWr/Vz8Vp3lXsub798//vbq8ln+79cYv9Wge+tdK73iOsW0KO/P",
    "uxpzkIs9R6VwijJOaie6oe8UJ9fmu8XlfMuWe6/Yud3ua/P++fMr8uW8y5ZblX/LjrHvzV83Tzw",
    "CvjNb9iPVc9XXxv/ybvYOehevn4y+6njR4yHJafe/DMeYQdlX/8HefovuHy1Fw==",
    ""
  };

  static char newstr [1265] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c26_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c26_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3966567233U));
  ssSetChecksum1(S,(3903686217U));
  ssSetChecksum2(S,(47072280U));
  ssSetChecksum3(S,(2442288883U));
}

static void mdlRTW_c26_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c26_sliding_controller_rbdl(SimStruct *S)
{
  SFc26_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc26_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc26_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc26_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc26_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c26_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c26_JITStateAnimation,
    chartInstance->c26_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c26_sliding_controller_rbdl(chartInstance);
}

void c26_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c26_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c26_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c26_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c26_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
