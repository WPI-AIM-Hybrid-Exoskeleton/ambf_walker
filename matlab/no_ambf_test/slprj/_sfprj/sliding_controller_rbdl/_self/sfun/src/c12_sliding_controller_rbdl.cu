/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c12_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c12_do_animation_call_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_st);
static void sf_gateway_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c12_eML_blk_kernel(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c12_b_q[2], real_T c12_b_y[4]);
static void c12_emlrt_marshallIn(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c12_b_y, const char_T *c12_identifier, real_T
  c12_c_y[4]);
static void c12_b_emlrt_marshallIn(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId,
  real_T c12_b_y[4]);
static uint8_T c12_c_emlrt_marshallIn
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_b_is_active_c12_sliding_controller_rbdl, const char_T *c12_identifier);
static uint8_T c12_d_emlrt_marshallIn
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_u, const emlrtMsgIdentifier *c12_parentId);
static void init_dsm_address_info(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c12_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c12_is_active_c12_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c12_do_animation_call_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c12_b_y = NULL;
  const mxArray *c12_c_y = NULL;
  const mxArray *c12_d_y = NULL;
  const mxArray *c12_st;
  c12_st = NULL;
  c12_b_y = NULL;
  sf_mex_assign(&c12_b_y, sf_mex_createcellmatrix(2, 1), false);
  c12_c_y = NULL;
  sf_mex_assign(&c12_c_y, sf_mex_create("y", *chartInstance->c12_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c12_b_y, 0, c12_c_y);
  c12_d_y = NULL;
  sf_mex_assign(&c12_d_y, sf_mex_create("y",
    &chartInstance->c12_is_active_c12_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c12_b_y, 1, c12_d_y);
  sf_mex_assign(&c12_st, c12_b_y, false);
  return c12_st;
}

static void set_sim_state_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_st)
{
  const mxArray *c12_u;
  c12_u = sf_mex_dup(c12_st);
  c12_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c12_u, 0)), "y",
                       *chartInstance->c12_y);
  chartInstance->c12_is_active_c12_sliding_controller_rbdl =
    c12_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c12_u, 1)),
    "is_active_c12_sliding_controller_rbdl");
  sf_mex_destroy(&c12_u);
  sf_mex_destroy(&c12_st);
}

static void sf_gateway_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  real_T c12_dv1[4];
  real_T c12_dv[2];
  int32_T c12_i;
  int32_T c12_i1;
  chartInstance->c12_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c12_i = 0; c12_i < 2; c12_i++) {
    c12_dv[c12_i] = (*chartInstance->c12_q)[c12_i];
  }

  c12_eML_blk_kernel(chartInstance, c12_dv, c12_dv1);
  for (c12_i1 = 0; c12_i1 < 4; c12_i1++) {
    (*chartInstance->c12_y)[c12_i1] = c12_dv1[c12_i1];
  }

  c12_do_animation_call_c12_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c12_errCode;
  c12_errCode = hipGetLastError();
  if (c12_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c12_errCode, (char_T *)hipGetErrorName
                       (c12_errCode), (char_T *)hipGetErrorString(c12_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c12_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c12_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c12_nameCaptureInfo = NULL;
  c12_nameCaptureInfo = NULL;
  sf_mex_assign(&c12_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c12_nameCaptureInfo;
}

static void c12_eML_blk_kernel(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c12_b_q[2], real_T c12_b_y[4])
{
  const mxArray *c12_c_y = NULL;
  c12_b_y[0] = -0.324 * muDoubleScalarCos(c12_b_q[0]) + -0.39 *
    muDoubleScalarCos(c12_b_q[0] + c12_b_q[1]);
  c12_b_y[2] = -0.39 * muDoubleScalarCos(c12_b_q[0] + c12_b_q[1]);
  c12_b_y[1] = 0.324 * muDoubleScalarSin(c12_b_q[0]) - -0.39 * muDoubleScalarSin
    (c12_b_q[0] + c12_b_q[1]);
  c12_b_y[3] = 0.39 * muDoubleScalarSin(c12_b_q[0] + c12_b_q[1]);
  sf_mex_printf("%s =\\n", "y");
  c12_c_y = NULL;
  sf_mex_assign(&c12_c_y, sf_mex_create("y", c12_b_y, 0, 0U, 1U, 0U, 2, 2, 2),
                false);
  sf_mex_call(chartInstance->c12_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c12_c_y);
}

static void c12_emlrt_marshallIn(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c12_b_y, const char_T *c12_identifier, real_T
  c12_c_y[4])
{
  emlrtMsgIdentifier c12_thisId;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c12_b_y), &c12_thisId,
    c12_c_y);
  sf_mex_destroy(&c12_b_y);
}

static void c12_b_emlrt_marshallIn(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId,
  real_T c12_b_y[4])
{
  real_T c12_dv[4];
  int32_T c12_i;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), c12_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c12_i = 0; c12_i < 4; c12_i++) {
    c12_b_y[c12_i] = c12_dv[c12_i];
  }

  sf_mex_destroy(&c12_u);
}

static uint8_T c12_c_emlrt_marshallIn
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_b_is_active_c12_sliding_controller_rbdl, const char_T *c12_identifier)
{
  emlrtMsgIdentifier c12_thisId;
  uint8_T c12_b_y;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_b_y = c12_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c12_b_is_active_c12_sliding_controller_rbdl), &c12_thisId);
  sf_mex_destroy(&c12_b_is_active_c12_sliding_controller_rbdl);
  return c12_b_y;
}

static uint8_T c12_d_emlrt_marshallIn
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_u, const emlrtMsgIdentifier *c12_parentId)
{
  uint8_T c12_b_u;
  uint8_T c12_b_y;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), &c12_b_u, 1, 3, 0U, 0, 0U, 0);
  c12_b_y = c12_b_u;
  sf_mex_destroy(&c12_u);
  return c12_b_y;
}

static void init_dsm_address_info(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c12_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c12_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c12_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c12_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3193823814U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(267299597U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(504614851U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(981118328U);
}

mxArray *sf_c12_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c12_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c12_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c12_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cmGRvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB11x2u"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c12_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sITpOBJPN6OXIDOpjmctU7";
}

static void sf_opaque_initialize_c12_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c12_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c12_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c12_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c12_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c12_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c12_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc12_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c12_sliding_controller_rbdl
      ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc12_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c12_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c12_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c12_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c12_sliding_controller_rbdl
      ((SFc12_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc12_sliding_controller_rbdl
      ((SFc12_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c12_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgUnaIDEMJqipwD1xUiPSQ9tT42jP5SFHRmh7OTQQl0vR+JWy11mfyQLKNAn6Gv",
    "knBfIQ+SWUw59gjxCZklKViiSiiPESIsSoKklv5mdmW9+1t6Wf+ThtYP3wZeedx2fX+Dd8LLrWr",
    "7eWrqz99ve3Xz9NwoJGx8TRWLt1V6CxPAYtOTWMCl8MZSlMCaGoEBQxCZSmSptmsWWMzHuWkGdP",
    "v0kYjQKIml52ERZEvYEn6G2xJpj1NNmCqjpAoQmUtKOoi4no4XFykxbEdCxtnGdCxpMYBNnlj6y",
    "3LCEQ+ccqC+0IWixvrAtMMRAy5xXuuk81cEcKOOEMyJKvY2IDiDBABs4SUL827MGnSrCaESUaUJ",
    "EJqAP2TjVKQUUdTKNH86YIEYqRngn5i0nuGrbMUd7jmQIvCYgaFtTARknkglTzX/QRU87gpxxaM",
    "OZHVVrC+CZdeSfMpiCqozbsCUnoMgIeqJy0zQgnfOUrUWWrMIMi+GUqIcU+dMQVmYvZo4OCPIEf",
    "ZSogkHqpK/7ik0wvJXabOy7zFxXMjbOyNbrYKm2zgTqWFho61LRIpzrSlhfJocwAZ5qbRND6mGZ",
    "1nKc1izsSwywS+/qarCCIfE5rCVFyErpmhQAad95hI3lfSS12si4hcnbPjxc/bwK84UBNSQUyrq",
    "AIkwDxiwNb7W2kGnHPQLRKpOaVwbOMmQdytNDK9pTqcYYk5omcuGCY7QSGOsRcomVcKKxaOpgjs",
    "t1OEpoBKFrMIzDEZYNYktiol1re4h1N2Fm1gZNFUtKWLVYddiGOi6hZgmciLGQU9FVMg7yHp+FF",
    "wCzjSjBxKiJ/UvNurh7GRVunt3zLubZzQ+YZ3O54vPbJT1bJXq8pWdx3xuN+n0b+GtrPn+X5G4V",
    "9tkuyDncLt4v//nz1/Pp7d9uvnr+15u9/d832f9F43Lzfydf35k32kXhTFby1WF/XrJru0T/10v",
    "6d/O19vtJr/nL8aPve0/9di/5I6bm5IecnzX2Ngr2zt/vuY6PWZbmp6J+mB9M3JrYbFw7/T8u2X",
    "t9TTxu5O+z6+2DzeS/OijyuL1Gfhd/zQp5+/H77x1sJp/t/3SN/fsFvvfTuT4grlvAgN7/bqA5C",
    "7HUB1QKoyTnoAbqLOQFPz823y8r512x3L/Fzv/lPj3vHzK/rlrOu2K5Tf277Bz/3PB188Ar4Hc/",
    "Yz82PV99avxr73LnoG/y9U+Lf3VaEeNhyak3/4xH2GHZ1/9Anr4D09Gzdw==",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c12_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c12_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1551688381U));
  ssSetChecksum1(S,(1561556856U));
  ssSetChecksum2(S,(2141638924U));
  ssSetChecksum3(S,(1613045982U));
}

static void mdlRTW_c12_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c12_sliding_controller_rbdl(SimStruct *S)
{
  SFc12_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc12_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc12_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc12_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc12_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c12_JITStateAnimation,
    chartInstance->c12_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c12_sliding_controller_rbdl(chartInstance);
}

void c12_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c12_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c12_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c12_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c12_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
