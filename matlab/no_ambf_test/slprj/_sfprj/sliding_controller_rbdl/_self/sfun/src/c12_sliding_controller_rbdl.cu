/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c12_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c12_do_animation_call_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_st);
static void sf_gateway_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c12_eML_blk_kernel(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c12_b_q[2], real_T c12_b_y[4]);
static void c12_emlrt_marshallIn(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c12_b_y, const char_T *c12_identifier, real_T
  c12_c_y[4]);
static void c12_b_emlrt_marshallIn(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId,
  real_T c12_b_y[4]);
static uint8_T c12_c_emlrt_marshallIn
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_b_is_active_c12_sliding_controller_rbdl, const char_T *c12_identifier);
static uint8_T c12_d_emlrt_marshallIn
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_u, const emlrtMsgIdentifier *c12_parentId);
static void init_dsm_address_info(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c12_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c12_is_active_c12_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c12_do_animation_call_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c12_b_y = NULL;
  const mxArray *c12_c_y = NULL;
  const mxArray *c12_d_y = NULL;
  const mxArray *c12_st;
  c12_st = NULL;
  c12_b_y = NULL;
  sf_mex_assign(&c12_b_y, sf_mex_createcellmatrix(2, 1), false);
  c12_c_y = NULL;
  sf_mex_assign(&c12_c_y, sf_mex_create("y", *chartInstance->c12_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c12_b_y, 0, c12_c_y);
  c12_d_y = NULL;
  sf_mex_assign(&c12_d_y, sf_mex_create("y",
    &chartInstance->c12_is_active_c12_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c12_b_y, 1, c12_d_y);
  sf_mex_assign(&c12_st, c12_b_y, false);
  return c12_st;
}

static void set_sim_state_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_st)
{
  const mxArray *c12_u;
  c12_u = sf_mex_dup(c12_st);
  c12_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c12_u, 0)), "y",
                       *chartInstance->c12_y);
  chartInstance->c12_is_active_c12_sliding_controller_rbdl =
    c12_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c12_u, 1)),
    "is_active_c12_sliding_controller_rbdl");
  sf_mex_destroy(&c12_u);
  sf_mex_destroy(&c12_st);
}

static void sf_gateway_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c12_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c12_eML_blk_kernel(chartInstance, *chartInstance->c12_q, *chartInstance->c12_y);
  c12_do_animation_call_c12_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c12_errCode;
  c12_errCode = hipGetLastError();
  if (c12_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c12_errCode, (char_T *)hipGetErrorName
                       (c12_errCode), (char_T *)hipGetErrorString(c12_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c12_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc12_sliding_controller_rbdl
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c12_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c12_nameCaptureInfo = NULL;
  c12_nameCaptureInfo = NULL;
  sf_mex_assign(&c12_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c12_nameCaptureInfo;
}

static void c12_eML_blk_kernel(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c12_b_q[2], real_T c12_b_y[4])
{
  c12_b_y[0] = -0.324 * muDoubleScalarCos(c12_b_q[0]) + -0.39 *
    muDoubleScalarCos(c12_b_q[0] + c12_b_q[1]);
  c12_b_y[2] = -0.39 * muDoubleScalarCos(c12_b_q[0] + c12_b_q[1]);
  c12_b_y[1] = 0.324 * muDoubleScalarSin(c12_b_q[0]) - -0.39 * muDoubleScalarSin
    (c12_b_q[0] + c12_b_q[1]);
  c12_b_y[3] = 0.39 * muDoubleScalarSin(c12_b_q[0] + c12_b_q[1]);
}

static void c12_emlrt_marshallIn(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c12_b_y, const char_T *c12_identifier, real_T
  c12_c_y[4])
{
  emlrtMsgIdentifier c12_thisId;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c12_b_y), &c12_thisId,
    c12_c_y);
  sf_mex_destroy(&c12_b_y);
}

static void c12_b_emlrt_marshallIn(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId,
  real_T c12_b_y[4])
{
  real_T c12_dv[4];
  int32_T c12_i;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), c12_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c12_i = 0; c12_i < 4; c12_i++) {
    c12_b_y[c12_i] = c12_dv[c12_i];
  }

  sf_mex_destroy(&c12_u);
}

static uint8_T c12_c_emlrt_marshallIn
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_b_is_active_c12_sliding_controller_rbdl, const char_T *c12_identifier)
{
  emlrtMsgIdentifier c12_thisId;
  uint8_T c12_b_y;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_b_y = c12_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c12_b_is_active_c12_sliding_controller_rbdl), &c12_thisId);
  sf_mex_destroy(&c12_b_is_active_c12_sliding_controller_rbdl);
  return c12_b_y;
}

static uint8_T c12_d_emlrt_marshallIn
  (SFc12_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c12_u, const emlrtMsgIdentifier *c12_parentId)
{
  uint8_T c12_b_u;
  uint8_T c12_b_y;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), &c12_b_u, 1, 3, 0U, 0, 0U, 0);
  c12_b_y = c12_b_u;
  sf_mex_destroy(&c12_u);
  return c12_b_y;
}

static void init_dsm_address_info(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc12_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c12_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c12_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c12_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c12_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(439330706U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4089007251U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2590084767U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3145384635U);
}

mxArray *sf_c12_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c12_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c12_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c12_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cmGRvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB11x2u"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c12_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sebrKerZXvpAoYCEy2tsgbF";
}

static void sf_opaque_initialize_c12_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c12_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c12_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c12_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c12_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c12_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c12_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc12_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c12_sliding_controller_rbdl
      ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc12_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c12_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c12_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc12_sliding_controller_rbdl
    ((SFc12_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c12_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c12_sliding_controller_rbdl
      ((SFc12_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc12_sliding_controller_rbdl
      ((SFc12_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c12_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQplU3iIE2EIogOaRAfQnaY5tTT41tSkSF2LVR2mnai7BajsSNlrvM/tDWtff0mHs",
    "QIOf2bfoYfYTOkpSsUiQVR4iRBiFAU0t+Mzsz3/ysva3BkYfXLbz3vvC8G/i8iXfHK65Py/XW0l",
    "283/a+LtfPUUjY5IQokmiv9RIkgZ9BS24Nk2IgxrIWxsQYFAiK2FQq06RNs8RyJqaBFdTp07/Ej",
    "MZhLC2PDlCWRMeCz1Bbas0J6ukxBdQEAJGJlbSTOOBksrBYmXM/BjrVNmlzQYMJberM0keWG5Zy",
    "6F8AHQhtCFqsL20LDTHgm4tGN52nOpwDZZJyRkSttzHRIaQYYANnaYR/j61Bp6owGhNlDiAmGeh",
    "DNs11SgFVnUzjhxETxEjFCO8n3HeCq7adcLTnSEbAWwKCth0oINNUMmGa+Q8D9LQvyIhDD0Z20q",
    "wthGfWkf+YwTmoxriNfZmBIhM4Fo2b5gHpX+RsLbJkFWZYAo+J2qfIn4aoMXsxc3RIkCc4RYkmG",
    "ORODvSpYhmGt1GbTQYuM9eVjE0KsvU6WK6tn0EbCwttARU+4Vw3wk5leggZ8FxrjxjSDiu01uO0",
    "ZtGpxAC79G6uBisYEl/CfCkiVktXVgHkfecnbCz/RVKrjUx8TN7e4eHq51XYQBhQY0KhrgsowjR",
    "gzPLwNmuLmHbcIxCtMrl5deAiQ9ahPD22oncu1RRj0tJELl1wjDYCEz1BLrESzjQWTRvMcbkORw",
    "mNIXINhnE4wrJBbE1MtGtt+1h3GTOzHmiqWFrDqsWqwzbUdwk1S+FMTIU8F4GSSVj2+CK8AJhtR",
    "AkmJgfYv9QswN3rqHDz7Fvvcp599gbzbC5XfX6zpGerRo+39Kzuu9Np37eDv7bm83dJ7vPKPtsV",
    "OYfr4v3Xnafd31+9tn8c7wYvPrn3cpP9/+xcbf7fKtdfzhvtonCylXx12B+X7Nqu0X93SX+3XGs",
    "YqUegfnuSpfvyV78/e2D0ZBQU/Kyxt1Oxd/5+13V8zLI8PxUdROXBxK2JLca10//9kr031sRjp3",
    "xfXP883Ez+9l6Vx+018l38Navk7dvvv7u3mXyx/5M19t+v8H0/n+tD4roFDOl3D4aaswhLfUilM",
    "EpyDmqoRhGv+Pm2+X5VOe+a5f4vdn6Ue/e8v8n8um4575rlNvXvqnP8fcO3zQOvgu++x35ser56",
    "1/i/vaudg74q1z8s/tXxY8ajmlNv+RmPsOO6rx9Anv4LjCSzcw==",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c12_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c12_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(275388081U));
  ssSetChecksum1(S,(1973855106U));
  ssSetChecksum2(S,(1176522638U));
  ssSetChecksum3(S,(2702902166U));
}

static void mdlRTW_c12_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c12_sliding_controller_rbdl(SimStruct *S)
{
  SFc12_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc12_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc12_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc12_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc12_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c12_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c12_JITStateAnimation,
    chartInstance->c12_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c12_sliding_controller_rbdl(chartInstance);
}

void c12_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c12_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c12_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c12_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c12_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
