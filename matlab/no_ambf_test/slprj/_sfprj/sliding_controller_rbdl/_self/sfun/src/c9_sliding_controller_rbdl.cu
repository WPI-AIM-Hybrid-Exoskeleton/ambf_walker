/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c9_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c9_do_animation_call_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c9_st);
static void sf_gateway_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c9_eML_blk_kernel(SFc9_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c9_b_q[2], real_T c9_b_y[4]);
static void c9_emlrt_marshallIn(SFc9_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c9_b_y, const char_T *c9_identifier, real_T
  c9_c_y[4]);
static void c9_b_emlrt_marshallIn(SFc9_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId,
  real_T c9_b_y[4]);
static uint8_T c9_c_emlrt_marshallIn(SFc9_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c9_b_is_active_c9_sliding_controller_rbdl, const
  char_T *c9_identifier);
static uint8_T c9_d_emlrt_marshallIn(SFc9_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId);
static void init_dsm_address_info(SFc9_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc9_sliding_controller_rbdlInstanceStruct *
  chartInstance);

/* Function Definitions */
static void initialize_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c9_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c9_is_active_c9_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c9_do_animation_call_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c9_b_y = NULL;
  const mxArray *c9_c_y = NULL;
  const mxArray *c9_d_y = NULL;
  const mxArray *c9_st;
  c9_st = NULL;
  c9_b_y = NULL;
  sf_mex_assign(&c9_b_y, sf_mex_createcellmatrix(2, 1), false);
  c9_c_y = NULL;
  sf_mex_assign(&c9_c_y, sf_mex_create("y", *chartInstance->c9_y, 0, 0U, 1U, 0U,
    2, 2, 2), false);
  sf_mex_setcell(c9_b_y, 0, c9_c_y);
  c9_d_y = NULL;
  sf_mex_assign(&c9_d_y, sf_mex_create("y",
    &chartInstance->c9_is_active_c9_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c9_b_y, 1, c9_d_y);
  sf_mex_assign(&c9_st, c9_b_y, false);
  return c9_st;
}

static void set_sim_state_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c9_st)
{
  const mxArray *c9_u;
  c9_u = sf_mex_dup(c9_st);
  c9_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c9_u, 0)), "y",
                      *chartInstance->c9_y);
  chartInstance->c9_is_active_c9_sliding_controller_rbdl = c9_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c9_u, 1)),
     "is_active_c9_sliding_controller_rbdl");
  sf_mex_destroy(&c9_u);
  sf_mex_destroy(&c9_st);
}

static void sf_gateway_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c9_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c9_eML_blk_kernel(chartInstance, *chartInstance->c9_q, *chartInstance->c9_y);
  c9_do_animation_call_c9_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c9_errCode;
  c9_errCode = hipGetLastError();
  if (c9_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c9_errCode, (char_T *)hipGetErrorName
                       (c9_errCode), (char_T *)hipGetErrorString(c9_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c9_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc9_sliding_controller_rbdl
  (SFc9_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c9_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c9_nameCaptureInfo = NULL;
  c9_nameCaptureInfo = NULL;
  sf_mex_assign(&c9_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c9_nameCaptureInfo;
}

static void c9_eML_blk_kernel(SFc9_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c9_b_q[2], real_T c9_b_y[4])
{
  c9_b_y[0] = -0.324 * muDoubleScalarSin(c9_b_q[0]) - 0.39 * muDoubleScalarSin
    (c9_b_q[0] + c9_b_q[1]);
  c9_b_y[2] = 0.324 * muDoubleScalarCos(c9_b_q[0]) + 0.39 * muDoubleScalarSin
    (c9_b_q[0] + c9_b_q[1]);
  c9_b_y[1] = -0.39 * muDoubleScalarSin(c9_b_q[0] + c9_b_q[1]);
  c9_b_y[3] = 0.39 * muDoubleScalarCos(c9_b_q[0] + c9_b_q[1]);
}

static void c9_emlrt_marshallIn(SFc9_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c9_b_y, const char_T *c9_identifier, real_T
  c9_c_y[4])
{
  emlrtMsgIdentifier c9_thisId;
  c9_thisId.fIdentifier = const_cast<const char_T *>(c9_identifier);
  c9_thisId.fParent = NULL;
  c9_thisId.bParentIsCell = false;
  c9_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c9_b_y), &c9_thisId, c9_c_y);
  sf_mex_destroy(&c9_b_y);
}

static void c9_b_emlrt_marshallIn(SFc9_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId,
  real_T c9_b_y[4])
{
  real_T c9_dv[4];
  int32_T c9_i;
  sf_mex_import(c9_parentId, sf_mex_dup(c9_u), c9_dv, 1, 0, 0U, 1, 0U, 2, 2, 2);
  for (c9_i = 0; c9_i < 4; c9_i++) {
    c9_b_y[c9_i] = c9_dv[c9_i];
  }

  sf_mex_destroy(&c9_u);
}

static uint8_T c9_c_emlrt_marshallIn(SFc9_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c9_b_is_active_c9_sliding_controller_rbdl, const
  char_T *c9_identifier)
{
  emlrtMsgIdentifier c9_thisId;
  uint8_T c9_b_y;
  c9_thisId.fIdentifier = const_cast<const char_T *>(c9_identifier);
  c9_thisId.fParent = NULL;
  c9_thisId.bParentIsCell = false;
  c9_b_y = c9_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c9_b_is_active_c9_sliding_controller_rbdl), &c9_thisId);
  sf_mex_destroy(&c9_b_is_active_c9_sliding_controller_rbdl);
  return c9_b_y;
}

static uint8_T c9_d_emlrt_marshallIn(SFc9_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId)
{
  uint8_T c9_b_u;
  uint8_T c9_b_y;
  sf_mex_import(c9_parentId, sf_mex_dup(c9_u), &c9_b_u, 1, 3, 0U, 0, 0U, 0);
  c9_b_y = c9_b_u;
  sf_mex_destroy(&c9_u);
  return c9_b_y;
}

static void init_dsm_address_info(SFc9_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc9_sliding_controller_rbdlInstanceStruct *
  chartInstance)
{
  chartInstance->c9_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c9_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c9_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c9_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(552332206U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2608078302U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1441289006U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4066884445U);
}

mxArray *sf_c9_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c9_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c9_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c9_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfh"
    "UU90P4mcXxicklmWWp8cmW8cU5mSmZeenxyfl5JUX5OTmpRfFFSSk5DEj+BAB5bh2C"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c9_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sjS5Om55RaK9v6MEldWgzUF";
}

static void sf_opaque_initialize_c9_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c9_sliding_controller_rbdl
    ((SFc9_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c9_sliding_controller_rbdl
    ((SFc9_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c9_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c9_sliding_controller_rbdl((SFc9_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c9_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c9_sliding_controller_rbdl((SFc9_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c9_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c9_sliding_controller_rbdl
    ((SFc9_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c9_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c9_sliding_controller_rbdl
    ((SFc9_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c9_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c9_sliding_controller_rbdl
    ((SFc9_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c9_sliding_controller_rbdl(void *
  chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc9_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c9_sliding_controller_rbdl
      ((SFc9_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc9_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc9_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c9_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c9_sliding_controller_rbdl
    ((SFc9_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c9_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c9_sliding_controller_rbdl
    ((SFc9_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc9_sliding_controller_rbdl
    ((SFc9_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c9_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c9_sliding_controller_rbdl
      ((SFc9_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc9_sliding_controller_rbdl
      ((SFc9_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c9_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u1EgQdkZZ/rSgCCFxQSIHJPbIJQgOQMj8iBGZTYQT4DbqtGvGnWl3m/5xMrwDJ8QbcOS",
    "wT7Jn7vsCSLzAVtueyeCxPYQRESAsOZ62v6quqq9+Ot5Kt+fhdQXvzauedw6fF/BueNn1R75emb",
    "mz96ve7Xz9BoWEjXaJIpH2ai9BIngGWnJrmBRdMZClMCYGoEBQxMZSmSptmkWWMzHqWEGdPv0iZ",
    "DT0Q2l5sIWyJNgRfIzaYmt2UU+LKaCmAxCYUEk7DDucDKcWK3PUDIGOtI3qXNBgfBs7s3TPcsNi",
    "Du1joF2hDUGL9YltviEGmua40k3nqfYnQBnFnBFR6m1ItA8xBtjAfhzg3x1r0KkijIZEmS0ISQJ",
    "6m41SnVJAUSfT+OGACWKkYoS3I950gvO27XK0pycD4DUBQdu2FJBRLJkw1fz7HfS0LcgBhxYc2G",
    "G1Nh9eWUf+cwZHoCrjNmjKBBQZwo6o3DQNSPs4ZWuaJfMwwyJ4TtRjivxpCCqzFzNH+wR5gj2Uq",
    "IJB6mRX7ymWYHgrtdmo6zJzUcnYKCNbL4Kl2toJ1LEw1dahokk415WwPRlvQwI81doihtTDMq3l",
    "OK1ZsCcxwC69q6vBCobE57CmFAErpSspANK+8zc2li+R1GojoyYmb2t7e/7zPKwrDKgBoVDWBRR",
    "hGjBmaXirtQVMO+4RiFaZ1LwycJYhi1CeHljROpJqhDGpaSInLjhGK4GRHiKXWAn7GoumDua4XI",
    "SjhIYQuAbDOPSwbBBbEhPtWttjrLuEmXELNFUsLmHVYtVhG2q7hBrHsC9GQh6JjpKRn/f4LLwAm",
    "G1ECSaGW9i/1LiDu5dR4ebZHe9knv35FfNsIld8/jWjZ6VEjzfzLO57sVG/bwN/rUzm74zc5cI+",
    "qwU5h1vD+9Lh+rvP6x/G/749/9+T2+8fLLP/P43Tzf8r+frGpNFOCyeZy1eHfTJj12qJ/usz+tf",
    "ytT70N3aijY1n5On95G6vzYMXw9f7nYyfBfY2CvZO3q+7jo9Zluanot0gP5i4NbHZuHb6783Ye2",
    "5BPC7m77Pr06Pl5K9tFnlcXSC/hr/Ghbz99v3XN5eTz/Z/ucD+WwW+b6VzvU9ct4A+vd/XnAVY6",
    "X0qhVGSc1B9dRDwYn1+a76fVs47Y7mfxc7fct+f96+ZX2ct552x3LL+nXaO/2j4unngFfBrP7Af",
    "y56vvjf+o3e6c9DNfP1w+q9OM2Q8KDn15p/xCDso+/oL5On/VxmzNQ==",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c9_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c9_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2501929482U));
  ssSetChecksum1(S,(2041323763U));
  ssSetChecksum2(S,(3808924622U));
  ssSetChecksum3(S,(1034168136U));
}

static void mdlRTW_c9_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c9_sliding_controller_rbdl(SimStruct *S)
{
  SFc9_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc9_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc9_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc9_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc9_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c9_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c9_JITStateAnimation,
    chartInstance->c9_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c9_sliding_controller_rbdl(chartInstance);
}

void c9_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c9_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c9_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c9_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c9_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
