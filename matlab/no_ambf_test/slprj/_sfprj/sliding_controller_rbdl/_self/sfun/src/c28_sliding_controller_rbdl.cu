/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c28_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c28_do_animation_call_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c28_st);
static void sf_gateway_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c28_eML_blk_kernel(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c28_b_q[2], real_T c28_b_y[4]);
static void c28_emlrt_marshallIn(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c28_b_y, const char_T *c28_identifier, real_T
  c28_c_y[4]);
static void c28_b_emlrt_marshallIn(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c28_u, const emlrtMsgIdentifier *c28_parentId,
  real_T c28_b_y[4]);
static uint8_T c28_c_emlrt_marshallIn
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c28_b_is_active_c28_sliding_controller_rbdl, const char_T *c28_identifier);
static uint8_T c28_d_emlrt_marshallIn
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c28_u, const emlrtMsgIdentifier *c28_parentId);
static void init_dsm_address_info(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c28_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c28_is_active_c28_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c28_do_animation_call_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c28_b_y = NULL;
  const mxArray *c28_c_y = NULL;
  const mxArray *c28_d_y = NULL;
  const mxArray *c28_st;
  c28_st = NULL;
  c28_b_y = NULL;
  sf_mex_assign(&c28_b_y, sf_mex_createcellmatrix(2, 1), false);
  c28_c_y = NULL;
  sf_mex_assign(&c28_c_y, sf_mex_create("y", *chartInstance->c28_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c28_b_y, 0, c28_c_y);
  c28_d_y = NULL;
  sf_mex_assign(&c28_d_y, sf_mex_create("y",
    &chartInstance->c28_is_active_c28_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c28_b_y, 1, c28_d_y);
  sf_mex_assign(&c28_st, c28_b_y, false);
  return c28_st;
}

static void set_sim_state_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c28_st)
{
  const mxArray *c28_u;
  c28_u = sf_mex_dup(c28_st);
  c28_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c28_u, 0)), "y",
                       *chartInstance->c28_y);
  chartInstance->c28_is_active_c28_sliding_controller_rbdl =
    c28_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c28_u, 1)),
    "is_active_c28_sliding_controller_rbdl");
  sf_mex_destroy(&c28_u);
  sf_mex_destroy(&c28_st);
}

static void sf_gateway_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c28_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c28_eML_blk_kernel(chartInstance, *chartInstance->c28_q, *chartInstance->c28_y);
  c28_do_animation_call_c28_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c28_errCode;
  c28_errCode = hipGetLastError();
  if (c28_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c28_errCode, (char_T *)hipGetErrorName
                       (c28_errCode), (char_T *)hipGetErrorString(c28_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c28_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc28_sliding_controller_rbdl
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c28_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c28_nameCaptureInfo = NULL;
  c28_nameCaptureInfo = NULL;
  sf_mex_assign(&c28_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c28_nameCaptureInfo;
}

static void c28_eML_blk_kernel(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c28_b_q[2], real_T c28_b_y[4])
{
  c28_b_y[0] = -0.324 * muDoubleScalarSin(c28_b_q[0]);
  c28_b_y[2] = 0.0;
  c28_b_y[1] = 0.324 * muDoubleScalarCos(c28_b_q[0]);
  c28_b_y[3] = 0.0;
}

static void c28_emlrt_marshallIn(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c28_b_y, const char_T *c28_identifier, real_T
  c28_c_y[4])
{
  emlrtMsgIdentifier c28_thisId;
  c28_thisId.fIdentifier = const_cast<const char_T *>(c28_identifier);
  c28_thisId.fParent = NULL;
  c28_thisId.bParentIsCell = false;
  c28_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c28_b_y), &c28_thisId,
    c28_c_y);
  sf_mex_destroy(&c28_b_y);
}

static void c28_b_emlrt_marshallIn(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c28_u, const emlrtMsgIdentifier *c28_parentId,
  real_T c28_b_y[4])
{
  real_T c28_dv[4];
  int32_T c28_i;
  sf_mex_import(c28_parentId, sf_mex_dup(c28_u), c28_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c28_i = 0; c28_i < 4; c28_i++) {
    c28_b_y[c28_i] = c28_dv[c28_i];
  }

  sf_mex_destroy(&c28_u);
}

static uint8_T c28_c_emlrt_marshallIn
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c28_b_is_active_c28_sliding_controller_rbdl, const char_T *c28_identifier)
{
  emlrtMsgIdentifier c28_thisId;
  uint8_T c28_b_y;
  c28_thisId.fIdentifier = const_cast<const char_T *>(c28_identifier);
  c28_thisId.fParent = NULL;
  c28_thisId.bParentIsCell = false;
  c28_b_y = c28_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c28_b_is_active_c28_sliding_controller_rbdl), &c28_thisId);
  sf_mex_destroy(&c28_b_is_active_c28_sliding_controller_rbdl);
  return c28_b_y;
}

static uint8_T c28_d_emlrt_marshallIn
  (SFc28_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c28_u, const emlrtMsgIdentifier *c28_parentId)
{
  uint8_T c28_b_u;
  uint8_T c28_b_y;
  sf_mex_import(c28_parentId, sf_mex_dup(c28_u), &c28_b_u, 1, 3, 0U, 0, 0U, 0);
  c28_b_y = c28_b_u;
  sf_mex_destroy(&c28_u);
  return c28_b_y;
}

static void init_dsm_address_info(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc28_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c28_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c28_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c28_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c28_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(536815311U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1945133011U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2900394615U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(32238838U);
}

mxArray *sf_c28_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c28_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c28_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c28_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8clGFvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB21B21"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c28_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sFqlSox9r5ekjZXT2w93GAD";
}

static void sf_opaque_initialize_c28_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c28_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c28_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c28_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c28_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c28_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c28_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc28_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c28_sliding_controller_rbdl
      ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc28_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c28_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c28_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc28_sliding_controller_rbdl
    ((SFc28_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c28_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c28_sliding_controller_rbdl
      ((SFc28_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc28_sliding_controller_rbdl
      ((SFc28_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c28_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgXbaNA2MIoCBYIA8SVoj4UNA8mlta2fVIhcG6UcG70I6+VI3Gq5S++PZL1Ab32",
    "THHLPi/QJcsmlj9BZkpJViqRiCzbcogRoaslvZmfmm5+1t9Y+8vB6jPf+V563ic/P8K556bWRrd",
    "fm7vT9uvdttv4DhYSNTogikfYqL0Ei+AW05NYwKdqiLwthTPRBgaCIjaUyZdo0iyxnYtiygjp9+",
    "ixkNPRDaXlwiLIkOBZ8gtpia05QT4MpoKYFEJhQSTsIW5wMZhYrM66HQIfaRlUuaDC+jZ1Z+shy",
    "w2IOzSugbaENQYv1tW2+IQbq5qrUTeep9qdAGcWcEVHobUi0DzEG2MBpHODfY2vQqTyMhkSZQwj",
    "JCHSHDROdUkBeJ9P44YIJYqRihDcjXneCi7adcLTnSAbAKwKCth0qIMNYMmHK+fdb6GlTkAsODb",
    "iwg3JtPlxaR/4bBmNQpXHr1+UIFBnAsSjdNAlI8ypha5YlizDDInhD1AFF/jQEpdmLmaN9gjxBF",
    "yXKYJA42dZdxUYY3lJtNmq7zFxWMjZKydbLYIm25giqWJhpa1FRJ5zrUlhXxh0YAU+0Nogh1bBU",
    "azFOaxZ0JQbYpXd5NVjBkPgMVpciYIV0jXKApO/8jI3ln0hqtZFRHZO30eksfl6EtYUB1ScUirq",
    "AIkwDxiwJb7m2gGnHPQLRKpOYVwROM2QZytN9KxpjqYYYk4omcu2CY7QUGOkBcomVcKqxaKpgjs",
    "tlOEpoCIFrMIzDEZYNYgtiol1rO8C6GzEzaYCmisUFrFqsOmxDTZdQkxhOxVDIsWgpGflZj0/DC",
    "4DZRpRgYnCI/UtNWrh7ERVunn3vXc+zLz5hnk3l8s/v5vSsFejx5p75fR/Vqvet4a+16fydk/sy",
    "t896Ts7htvB+8nbvw/tdvcE/f935/eNZZ5X939VuNv8fZ+un00Y7K5zRQr467E9zdq0X6P9mTv9",
    "WttatS+7Lq5dqD4a//Xre3Rm/3H110Ej5WWJvLWfv9P226/iYZUl+KtoOsoOJWxObjmun/8WcvZ",
    "tL4vEoe59ef/24mvzX+3ke15fIb+GvSS5vb7//9v5q8un+50vsf57j+3ky13vEdQvo0Z0XPc1Zg",
    "KXeo1IYJTkH1VMXAc/5edt8v6mcd89y/xY7/5e7e94/ZX7dt5x3z3Kr+nfTOf7Q8FXzwMvhtx6w",
    "H6uer+4a/6d3s3PQs2z9w+xfnXrIeFBw6s0+4xG2X/T1P5CnfwOO+7MD",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c28_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c28_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3862277146U));
  ssSetChecksum1(S,(91435963U));
  ssSetChecksum2(S,(1279986540U));
  ssSetChecksum3(S,(1280830342U));
}

static void mdlRTW_c28_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c28_sliding_controller_rbdl(SimStruct *S)
{
  SFc28_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc28_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc28_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc28_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc28_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c28_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c28_JITStateAnimation,
    chartInstance->c28_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c28_sliding_controller_rbdl(chartInstance);
}

void c28_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c28_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c28_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c28_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c28_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
