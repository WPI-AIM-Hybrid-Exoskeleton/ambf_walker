#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c40_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c40_do_animation_call_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c40_st);
static void sf_gateway_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c40_eML_blk_kernel(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c40_b_u[7], real_T c40_b_y[7]);
static void c40_emlrt_marshallIn(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c40_b_y, const char_T *c40_identifier, real_T
  c40_c_y[7]);
static void c40_b_emlrt_marshallIn(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c40_b_u, const emlrtMsgIdentifier *c40_parentId,
  real_T c40_b_y[7]);
static uint8_T c40_c_emlrt_marshallIn
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c40_b_is_active_c40_sliding_controller_rbdl, const char_T *c40_identifier);
static uint8_T c40_d_emlrt_marshallIn
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c40_b_u, const emlrtMsgIdentifier *c40_parentId);
static __global__ void c40_eML_blk_kernel_kernel1(real_T c40_b_y[7]);
static void init_dsm_address_info(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c40_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c40_is_active_c40_sliding_controller_rbdl = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c40_gpu_y, 56UL);
}

static void initialize_params_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c40_do_animation_call_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c40_b_y = NULL;
  const mxArray *c40_c_y = NULL;
  const mxArray *c40_d_y = NULL;
  const mxArray *c40_st;
  c40_st = NULL;
  c40_b_y = NULL;
  sf_mex_assign(&c40_b_y, sf_mex_createcellmatrix(2, 1), false);
  c40_c_y = NULL;
  sf_mex_assign(&c40_c_y, sf_mex_create("y", *chartInstance->c40_y, 0, 0U, 1U,
    0U, 1, 7), false);
  sf_mex_setcell(c40_b_y, 0, c40_c_y);
  c40_d_y = NULL;
  sf_mex_assign(&c40_d_y, sf_mex_create("y",
    &chartInstance->c40_is_active_c40_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c40_b_y, 1, c40_d_y);
  sf_mex_assign(&c40_st, c40_b_y, false);
  return c40_st;
}

static void set_sim_state_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c40_st)
{
  const mxArray *c40_b_u;
  c40_b_u = sf_mex_dup(c40_st);
  c40_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c40_b_u, 0)),
                       "y", *chartInstance->c40_y);
  chartInstance->c40_is_active_c40_sliding_controller_rbdl =
    c40_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c40_b_u, 1)),
    "is_active_c40_sliding_controller_rbdl");
  sf_mex_destroy(&c40_b_u);
  sf_mex_destroy(&c40_st);
}

static void sf_gateway_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  real_T c40_dv[7];
  real_T c40_dv1[7];
  int32_T c40_i;
  int32_T c40_i1;
  chartInstance->c40_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c40_i = 0; c40_i < 7; c40_i++) {
    c40_dv[c40_i] = (*chartInstance->c40_u)[c40_i];
  }

  c40_eML_blk_kernel(chartInstance, c40_dv, c40_dv1);
  for (c40_i1 = 0; c40_i1 < 7; c40_i1++) {
    (*chartInstance->c40_y)[c40_i1] = c40_dv1[c40_i1];
  }

  c40_do_animation_call_c40_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c40_errCode;
  hipFree(*chartInstance->c40_gpu_y);
  c40_errCode = hipGetLastError();
  if (c40_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c40_errCode, (char_T *)hipGetErrorName
                       (c40_errCode), (char_T *)hipGetErrorString(c40_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c40_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc40_sliding_controller_rbdl
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c40_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c40_nameCaptureInfo = NULL;
  c40_nameCaptureInfo = NULL;
  sf_mex_assign(&c40_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c40_nameCaptureInfo;
}

static void c40_eML_blk_kernel(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c40_b_u[7], real_T c40_b_y[7])
{
  c40_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c40_gpu_y);
  hipMemcpy(c40_b_y, *chartInstance->c40_gpu_y, 56UL, hipMemcpyDeviceToHost);
  c40_b_y[0] = c40_b_u[1];
  c40_b_y[1] = c40_b_u[2];
  c40_b_y[2] = c40_b_u[3];
  c40_b_y[3] = c40_b_u[0];
  c40_b_y[4] = c40_b_u[4];
  c40_b_y[5] = c40_b_u[5];
}

static void c40_emlrt_marshallIn(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c40_b_y, const char_T *c40_identifier, real_T
  c40_c_y[7])
{
  emlrtMsgIdentifier c40_thisId;
  c40_thisId.fIdentifier = const_cast<const char_T *>(c40_identifier);
  c40_thisId.fParent = NULL;
  c40_thisId.bParentIsCell = false;
  c40_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c40_b_y), &c40_thisId,
    c40_c_y);
  sf_mex_destroy(&c40_b_y);
}

static void c40_b_emlrt_marshallIn(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c40_b_u, const emlrtMsgIdentifier *c40_parentId,
  real_T c40_b_y[7])
{
  real_T c40_dv[7];
  int32_T c40_i;
  sf_mex_import(c40_parentId, sf_mex_dup(c40_b_u), c40_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c40_i = 0; c40_i < 7; c40_i++) {
    c40_b_y[c40_i] = c40_dv[c40_i];
  }

  sf_mex_destroy(&c40_b_u);
}

static uint8_T c40_c_emlrt_marshallIn
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c40_b_is_active_c40_sliding_controller_rbdl, const char_T *c40_identifier)
{
  emlrtMsgIdentifier c40_thisId;
  uint8_T c40_b_y;
  c40_thisId.fIdentifier = const_cast<const char_T *>(c40_identifier);
  c40_thisId.fParent = NULL;
  c40_thisId.bParentIsCell = false;
  c40_b_y = c40_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c40_b_is_active_c40_sliding_controller_rbdl), &c40_thisId);
  sf_mex_destroy(&c40_b_is_active_c40_sliding_controller_rbdl);
  return c40_b_y;
}

static uint8_T c40_d_emlrt_marshallIn
  (SFc40_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c40_b_u, const emlrtMsgIdentifier *c40_parentId)
{
  uint8_T c40_b_y;
  uint8_T c40_c_u;
  sf_mex_import(c40_parentId, sf_mex_dup(c40_b_u), &c40_c_u, 1, 3, 0U, 0, 0U, 0);
  c40_b_y = c40_c_u;
  sf_mex_destroy(&c40_b_u);
  return c40_b_y;
}

static __global__ __launch_bounds__(32, 1) void c40_eML_blk_kernel_kernel1
  (real_T c40_b_y[7])
{
  uint64_T c40_threadId;
  int32_T c40_i;
  c40_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c40_i = (int32_T)c40_threadId;
  if (c40_i < 7) {
    c40_b_y[c40_i] = 0.0;
  }
}

static void init_dsm_address_info(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc40_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c40_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c40_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c40_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c40_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3761594018U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1451773495U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2587411843U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4121430803U);
}

mxArray *sf_c40_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c40_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c40_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c40_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8ckmBvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB1/h2v"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c40_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sEsyT33uilQMeLbc2q1KEGD";
}

static void sf_opaque_initialize_c40_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c40_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c40_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c40_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c40_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c40_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c40_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc40_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c40_sliding_controller_rbdl
      ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc40_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c40_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c40_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc40_sliding_controller_rbdl
    ((SFc40_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c40_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c40_sliding_controller_rbdl
      ((SFc40_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc40_sliding_controller_rbdl
      ((SFc40_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c40_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81uE0kQnlgJAmkXRQiJCxLhgNhjIBw4QYh/wFp7k91xAjer3VP2tNzTPfSPE1/3tJd9BV5",
    "gL4gn4DH2Cfa8l71v9czYMeOZMcEiAsRIk3HPfFVdVV/9dLyNdtfD6zre+zc87wo+r+Jd89JrK1",
    "tvLNzp+03vfrb+E4WEjY6IIpH2Ki9BIvgNtOTWMCnaYigLYUwMQYGgiI2lMmXaNIssZ2LcsoI6f",
    "fplyGjoh9Ly4ABlSXAo+BS1xdYcoZ4GU0BNCyAwoZJ2FLY4Gc0tVua0HgIdaxtVuaDB+DZ2Zumu",
    "5YbFHJpnQNtCG4IW63PbfEMM1M1ZqZvOU+3PgDKKOSOi0NuQaB9iDLCB4zjAv4fWoFN5GA2JMgc",
    "QkgnoDhsnOqWAvE6m8cOACWKkYoQ3I153gsu2HXG0pysD4BUBQdsOFJBxLJkw5fz7LfS0KciAQw",
    "MGdlSuzYfX1pF/wuAUVGnchnU5AUVGcChKN00C0jxL2JpnyTLMsAhOiHpGkT8NQWn2YuZonyBP0",
    "EOJMhgkTrZ1T7EJhrdUm43aLjNXlYyNUrL1KliirTmBKhbm2lpU1AnnuhTWk3EHJsATrQ1iSDUs",
    "1VqM05oFPYkBduldXg1WMCQ+g9WlCFghXZMcIOk7v2Bj+RBJrTYyqmPyNjqd5c/LsLYwoIaEQlE",
    "XUIRpwJgl4S3XFjDtuEcgWmUS84rAaYasQnl6aEXjVKoxxqSiiZy74BgtBUZ6hFxiJRxrLJoqmO",
    "NyFY4SGkLgGgzj0MWyQWxBTLRrbc+w7ibMTBugqWJxAasWqw7bUNMl1DSGYzEW8lS0lIz8rMen4",
    "QXAbCNKMDE6wP6lpi3cvYgKN892vfN59sNHzLOZXP7504KejQI93sIzv++1WvW+Nfy1MZu/C3I/",
    "5vbZzMk53Dbe73f+eCp+f7f16r9/9t/c/evNOvu/rV1s/l/P1rdnjXZeOJOlfHXYFwt2bRbov7W",
    "gfztb66ae9vb2LOO/dqEzoA9fP/i5+byR8rPC3lrO3tn7HdfxMcuS/FS0HWQHE7cmNh3XTv/jBX",
    "uvrIjHtex9ev37dD35m/t5HjdXyG/jr2kubz99/5399eTT/V+tsP9eju97yVzvE9ctoE8f7fY1Z",
    "wGWep9KYZTkHFRfDQKe8/NT8/2ict4ly30tdn6X+/y8f8z8umw575Ll1vXvonP8S8NXzQMvh9/+",
    "gv1Y93z1ufF/exc7B93J1k/m/+rUQ8aDglNv9hmPsMOir99Anv4Pacq0Nw==",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c40_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c40_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1065885888U));
  ssSetChecksum1(S,(95584878U));
  ssSetChecksum2(S,(1088877912U));
  ssSetChecksum3(S,(2662015390U));
}

static void mdlRTW_c40_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c40_sliding_controller_rbdl(SimStruct *S)
{
  SFc40_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc40_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc40_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc40_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc40_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c40_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c40_JITStateAnimation,
    chartInstance->c40_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c40_sliding_controller_rbdl(chartInstance);
}

void c40_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c40_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c40_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c40_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c40_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
