#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c8_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c8_do_animation_call_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c8_st);
static void sf_gateway_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c8_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c8_b_y, const char_T *c8_identifier, real_T
  c8_c_y[7]);
static void c8_b_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c8_b_u, const emlrtMsgIdentifier *c8_parentId,
  real_T c8_b_y[7]);
static uint8_T c8_c_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c8_b_is_active_c8_sliding_controller_rbdl, const
  char_T *c8_identifier);
static uint8_T c8_d_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c8_b_u, const emlrtMsgIdentifier *c8_parentId);
static void c8_eML_blk_kernel(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c8_b_u[7]);
static __global__ void c8_sf_gateway_c8_sliding_controller_rbdl_kernel1(real_T
  c8_b_u[7], real_T c8_b_y[7]);
static void init_dsm_address_info(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance);

/* Function Definitions */
static void initialize_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c8_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c8_is_active_c8_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c8_do_animation_call_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c8_b_y = NULL;
  const mxArray *c8_c_y = NULL;
  const mxArray *c8_d_y = NULL;
  const mxArray *c8_st;
  c8_st = NULL;
  c8_b_y = NULL;
  sf_mex_assign(&c8_b_y, sf_mex_createcellmatrix(2, 1), false);
  c8_c_y = NULL;
  sf_mex_assign(&c8_c_y, sf_mex_create("y", *chartInstance->c8_y, 0, 0U, 1U, 0U,
    1, 7), false);
  sf_mex_setcell(c8_b_y, 0, c8_c_y);
  c8_d_y = NULL;
  sf_mex_assign(&c8_d_y, sf_mex_create("y",
    &chartInstance->c8_is_active_c8_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c8_b_y, 1, c8_d_y);
  sf_mex_assign(&c8_st, c8_b_y, false);
  return c8_st;
}

static void set_sim_state_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c8_st)
{
  const mxArray *c8_b_u;
  c8_b_u = sf_mex_dup(c8_st);
  c8_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c8_b_u, 0)), "y",
                      *chartInstance->c8_y);
  chartInstance->c8_is_active_c8_sliding_controller_rbdl = c8_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c8_b_u, 1)),
     "is_active_c8_sliding_controller_rbdl");
  sf_mex_destroy(&c8_b_u);
  sf_mex_destroy(&c8_st);
}

static void sf_gateway_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  real_T (*c8_gpu_u)[7];
  real_T (*c8_gpu_y)[7];
  hipMalloc(&c8_gpu_y, 56UL);
  hipMalloc(&c8_gpu_u, 56UL);
  chartInstance->c8_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(*c8_gpu_u, *chartInstance->c8_u, 56UL, hipMemcpyHostToDevice);
  hipMemcpy(*c8_gpu_y, *chartInstance->c8_y, 56UL, hipMemcpyHostToDevice);
  c8_sf_gateway_c8_sliding_controller_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(*c8_gpu_u, *c8_gpu_y);
  hipMemcpy(*chartInstance->c8_y, *c8_gpu_y, 56UL, hipMemcpyDeviceToHost);
  hipMemcpy(*chartInstance->c8_u, *c8_gpu_u, 56UL, hipMemcpyDeviceToHost);
  c8_eML_blk_kernel(chartInstance, *chartInstance->c8_y);
  c8_do_animation_call_c8_sliding_controller_rbdl(chartInstance);
  hipFree(*c8_gpu_u);
  hipFree(*c8_gpu_y);
}

static void mdl_start_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c8_errCode;
  c8_errCode = hipGetLastError();
  if (c8_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c8_errCode, (char_T *)hipGetErrorName
                       (c8_errCode), (char_T *)hipGetErrorString(c8_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c8_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c8_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c8_nameCaptureInfo = NULL;
  c8_nameCaptureInfo = NULL;
  sf_mex_assign(&c8_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c8_nameCaptureInfo;
}

static void c8_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c8_b_y, const char_T *c8_identifier, real_T
  c8_c_y[7])
{
  emlrtMsgIdentifier c8_thisId;
  c8_thisId.fIdentifier = const_cast<const char_T *>(c8_identifier);
  c8_thisId.fParent = NULL;
  c8_thisId.bParentIsCell = false;
  c8_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c8_b_y), &c8_thisId, c8_c_y);
  sf_mex_destroy(&c8_b_y);
}

static void c8_b_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c8_b_u, const emlrtMsgIdentifier *c8_parentId,
  real_T c8_b_y[7])
{
  real_T c8_dv[7];
  int32_T c8_i;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_b_u), c8_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c8_i = 0; c8_i < 7; c8_i++) {
    c8_b_y[c8_i] = c8_dv[c8_i];
  }

  sf_mex_destroy(&c8_b_u);
}

static uint8_T c8_c_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c8_b_is_active_c8_sliding_controller_rbdl, const
  char_T *c8_identifier)
{
  emlrtMsgIdentifier c8_thisId;
  uint8_T c8_b_y;
  c8_thisId.fIdentifier = const_cast<const char_T *>(c8_identifier);
  c8_thisId.fParent = NULL;
  c8_thisId.bParentIsCell = false;
  c8_b_y = c8_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c8_b_is_active_c8_sliding_controller_rbdl), &c8_thisId);
  sf_mex_destroy(&c8_b_is_active_c8_sliding_controller_rbdl);
  return c8_b_y;
}

static uint8_T c8_d_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c8_b_u, const emlrtMsgIdentifier *c8_parentId)
{
  uint8_T c8_b_y;
  uint8_T c8_c_u;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_b_u), &c8_c_u, 1, 3, 0U, 0, 0U, 0);
  c8_b_y = c8_c_u;
  sf_mex_destroy(&c8_b_u);
  return c8_b_y;
}

static void c8_eML_blk_kernel(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c8_b_u[7])
{
}

static __global__ __launch_bounds__(32, 1) void
  c8_sf_gateway_c8_sliding_controller_rbdl_kernel1(real_T c8_b_u[7], real_T
  c8_b_y[7])
{
  uint64_T c8_threadId;
  int32_T c8_i;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i = (int32_T)c8_threadId;
  if (c8_i < 7) {
    c8_b_y[c8_i] = c8_b_u[c8_i];
  }
}

static void init_dsm_address_info(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance)
{
  chartInstance->c8_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c8_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c8_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c8_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2438127827U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3025077116U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(880705944U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3519700120U);
}

mxArray *sf_c8_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c8_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c8_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c8_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfh"
    "UU90P4mcXxicklmWWp8ckW8cU5mSmZeenxyfl5JUX5OTmpRfFFSSk5DEj+BAB5SR2B"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c8_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "svi1IOD2u267XmhzOouF1CC";
}

static void sf_opaque_initialize_c8_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c8_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c8_sliding_controller_rbdl((SFc8_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c8_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c8_sliding_controller_rbdl((SFc8_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c8_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c8_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c8_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c8_sliding_controller_rbdl(void *
  chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc8_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c8_sliding_controller_rbdl
      ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc8_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c8_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c8_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c8_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c8_sliding_controller_rbdl
      ((SFc8_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc8_sliding_controller_rbdl
      ((SFc8_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c8_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV91u00gUdqMuAgmqCq3gBgmkrbRcLki7cLNQ6iTaSC2p1i1wgRRNxyfxKOMZMz9usy/AxYp",
    "79imWSx6DSx5jH4EztpNmHduhRFSAsOQ6Y3/nzDnnOz9Tb6235+G1gff2Vc+7gM+LeLe8/PqhWK",
    "/N3fn7de/nYv0KhYSN94kisfYaL0Fi+BO05NYwKXpiKCthTAxBgaCITaQyddo0iy1nYty1gjp9+",
    "mnEaBRE0vJwB2VJ2Bd8gtoSa/ZRT5spoKYLEJpISTuKupyMZhYrc+xHQMfaxk0uaDCBTZxZes9y",
    "wxIOnROgPaENQYv1qW2BIQZ8c1LrpvNUB1OgjBPOiKj0NiI6gAQDbOAwCfFv3xp0qgyjEVFmByK",
    "Sgt5l40ynFFDWyTR+OGKCGKkY4Z2Y+05w0bZ9jvbsyRB4Q0DQth0FZJxIJkw9/0EXPe0IcsShDU",
    "d2VK8tgBfWkf+EwTGo2rgNfZmCIiPoi9pNs4B0TjK2ZlmyCDMshidEPaLIn4awNnsxc3RAkCc4Q",
    "Ik6GGRO9vSBYimGt1abjXsuM5eVjI1zsvUyWKatk0ITCzNtXSp8wrmuhR3IZBdS4JnWNjGkGZZr",
    "rcZpzcIDiQF26V1fDVYwJL6A+VKErJKutATI+s5jbCz/R1KrjYx9TN727u7i50VYTxhQQ0Khqgs",
    "owjRgzLLw1msLmXbcIxCtMpl5VeA8Q5ahPD20on0s1Rhj0tBETl1wjNYCYz1CLrESDjUWTRPMcb",
    "kMRwmNIHQNhnHYw7JBbEVMtGttj7DuUmYmbdBUsaSCVYtVh22o4xJqksChGAt5LLpKxkHR4/PwA",
    "mC2ESWYGO1g/1KTLu5eRYWbZ794p/Ps8kfMs6lc+Xl7Ts9ahR5v7lne91Kred8W/lqbzt85uSul",
    "fdZLcg63iffVl9eC4a9///T787f/vHv5+t9V9n/TOtv83yjWN6aNdlY46UK+Ouwfc3atV+i/Pqd",
    "/s1jrlN3p9dt37d3f7j2Lo7/60nbv+H7OzxJ7WyV7p+9vuY6PWZblp6K9sDiYuDWx+bh2+u/P2X",
    "thSTwuFe/z67+Hq8n/uF3mcX2J/Cb+mpTy9tP3v7W9mny+/7Ml9m+V+N7K5vqAuG4BA3p/oDkLs",
    "dIHVAqjJOegBuoo5OX6/NR8P6ucd85yX4ud3+U+P+8fM7/OW847Z7lV/TvrHP/S8E3zwCvhN79g",
    "P1Y9X31u/HvvbOegm8X6wexfHT9iPKw49Raf8Qg7rPr6DeTpByAMsuQ=",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c8_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c8_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1393985298U));
  ssSetChecksum1(S,(596325734U));
  ssSetChecksum2(S,(2578996285U));
  ssSetChecksum3(S,(2946009040U));
}

static void mdlRTW_c8_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c8_sliding_controller_rbdl(SimStruct *S)
{
  SFc8_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc8_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc8_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc8_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc8_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c8_JITStateAnimation,
    chartInstance->c8_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c8_sliding_controller_rbdl(chartInstance);
}

void c8_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c8_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c8_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c8_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c8_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
