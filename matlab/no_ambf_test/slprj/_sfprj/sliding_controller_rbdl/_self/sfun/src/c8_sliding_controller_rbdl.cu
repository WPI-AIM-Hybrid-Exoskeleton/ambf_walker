/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c8_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c8_do_animation_call_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c8_st);
static void sf_gateway_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c8_eML_blk_kernel(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c8_b_u[7], real_T c8_b_y[6]);
static void c8_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c8_b_y, const char_T *c8_identifier, real_T
  c8_c_y[6]);
static void c8_b_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c8_b_u, const emlrtMsgIdentifier *c8_parentId,
  real_T c8_b_y[6]);
static uint8_T c8_c_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c8_b_is_active_c8_sliding_controller_rbdl, const
  char_T *c8_identifier);
static uint8_T c8_d_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c8_b_u, const emlrtMsgIdentifier *c8_parentId);
static void init_dsm_address_info(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance);

/* Function Definitions */
static void initialize_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c8_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c8_is_active_c8_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c8_do_animation_call_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c8_b_y = NULL;
  const mxArray *c8_c_y = NULL;
  const mxArray *c8_d_y = NULL;
  const mxArray *c8_st;
  c8_st = NULL;
  c8_b_y = NULL;
  sf_mex_assign(&c8_b_y, sf_mex_createcellmatrix(2, 1), false);
  c8_c_y = NULL;
  sf_mex_assign(&c8_c_y, sf_mex_create("y", *chartInstance->c8_y, 0, 0U, 1U, 0U,
    1, 6), false);
  sf_mex_setcell(c8_b_y, 0, c8_c_y);
  c8_d_y = NULL;
  sf_mex_assign(&c8_d_y, sf_mex_create("y",
    &chartInstance->c8_is_active_c8_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c8_b_y, 1, c8_d_y);
  sf_mex_assign(&c8_st, c8_b_y, false);
  return c8_st;
}

static void set_sim_state_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c8_st)
{
  const mxArray *c8_b_u;
  c8_b_u = sf_mex_dup(c8_st);
  c8_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c8_b_u, 0)), "y",
                      *chartInstance->c8_y);
  chartInstance->c8_is_active_c8_sliding_controller_rbdl = c8_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c8_b_u, 1)),
     "is_active_c8_sliding_controller_rbdl");
  sf_mex_destroy(&c8_b_u);
  sf_mex_destroy(&c8_st);
}

static void sf_gateway_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c8_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c8_eML_blk_kernel(chartInstance, *chartInstance->c8_u, *chartInstance->c8_y);
  c8_do_animation_call_c8_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c8_errCode;
  c8_errCode = hipGetLastError();
  if (c8_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c8_errCode, (char_T *)hipGetErrorName
                       (c8_errCode), (char_T *)hipGetErrorString(c8_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c8_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc8_sliding_controller_rbdl
  (SFc8_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c8_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c8_nameCaptureInfo = NULL;
  c8_nameCaptureInfo = NULL;
  sf_mex_assign(&c8_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c8_nameCaptureInfo;
}

static void c8_eML_blk_kernel(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c8_b_u[7], real_T c8_b_y[6])
{
  c8_b_y[0] = c8_b_u[2];
  c8_b_y[1] = c8_b_u[3];
  c8_b_y[2] = c8_b_u[6];
  c8_b_y[3] = c8_b_u[0];
  c8_b_y[4] = c8_b_u[4];
  c8_b_y[5] = c8_b_u[5];
}

static void c8_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c8_b_y, const char_T *c8_identifier, real_T
  c8_c_y[6])
{
  emlrtMsgIdentifier c8_thisId;
  c8_thisId.fIdentifier = const_cast<const char_T *>(c8_identifier);
  c8_thisId.fParent = NULL;
  c8_thisId.bParentIsCell = false;
  c8_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c8_b_y), &c8_thisId, c8_c_y);
  sf_mex_destroy(&c8_b_y);
}

static void c8_b_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c8_b_u, const emlrtMsgIdentifier *c8_parentId,
  real_T c8_b_y[6])
{
  real_T c8_dv[6];
  int32_T c8_i;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_b_u), c8_dv, 1, 0, 0U, 1, 0U, 1, 6);
  for (c8_i = 0; c8_i < 6; c8_i++) {
    c8_b_y[c8_i] = c8_dv[c8_i];
  }

  sf_mex_destroy(&c8_b_u);
}

static uint8_T c8_c_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c8_b_is_active_c8_sliding_controller_rbdl, const
  char_T *c8_identifier)
{
  emlrtMsgIdentifier c8_thisId;
  uint8_T c8_b_y;
  c8_thisId.fIdentifier = const_cast<const char_T *>(c8_identifier);
  c8_thisId.fParent = NULL;
  c8_thisId.bParentIsCell = false;
  c8_b_y = c8_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c8_b_is_active_c8_sliding_controller_rbdl), &c8_thisId);
  sf_mex_destroy(&c8_b_is_active_c8_sliding_controller_rbdl);
  return c8_b_y;
}

static uint8_T c8_d_emlrt_marshallIn(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c8_b_u, const emlrtMsgIdentifier *c8_parentId)
{
  uint8_T c8_b_y;
  uint8_T c8_c_u;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_b_u), &c8_c_u, 1, 3, 0U, 0, 0U, 0);
  c8_b_y = c8_c_u;
  sf_mex_destroy(&c8_b_u);
  return c8_b_y;
}

static void init_dsm_address_info(SFc8_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc8_sliding_controller_rbdlInstanceStruct *
  chartInstance)
{
  chartInstance->c8_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c8_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c8_y = (real_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c8_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2339256118U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2737773724U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(583777407U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1172739197U);
}

mxArray *sf_c8_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c8_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c8_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c8_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfh"
    "UU90P4mcXxicklmWWp8ckW8cU5mSmZeenxyfl5JUX5OTmpRfFFSSk5DEj+BAB5SR2B"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c8_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "s0lpZ3UsJHBzJBVszsMYZmF";
}

static void sf_opaque_initialize_c8_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c8_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c8_sliding_controller_rbdl((SFc8_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c8_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c8_sliding_controller_rbdl((SFc8_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c8_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c8_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c8_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c8_sliding_controller_rbdl(void *
  chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc8_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c8_sliding_controller_rbdl
      ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc8_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c8_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c8_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc8_sliding_controller_rbdl
    ((SFc8_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c8_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c8_sliding_controller_rbdl
      ((SFc8_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc8_sliding_controller_rbdl
      ((SFc8_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c8_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV09v00gUd6MuAgmqCiEhJCTQCql7rHYPcIISJ1EDzbZat91dLtF0/BKPMp4x8ydt+A4c+BR",
    "75eMgLnvb836EfWM7adaxnZaIChCWXGfs33vz3vu9P1Nvrdvz8NrAe+e2513D53W8G152/ZCv1+",
    "bu7P26t5Wv36KQsPEBUSTWXu0lSAy/gZbcGiZFVwxkKYyJASgQFLGJVKZKm2ax5UyMOlZQp0//H",
    "jEaBZG0PGyiLAn3BZ+gtsSaA9TTYgqo6QCEJlLSDqMOJ8OZxcqc+hHQkbZxnQsaTGATZ5buWW5Y",
    "wqF9BrQrtCFosT63LTDEgG/OKt10nupgCpRxwhkRpd5GRAeQYIANHCUh/t23Bp0qwmhElGlCRMa",
    "g99go1SkFFHUyjR9OmCBGKkZ4O+a+E1y07YCjPT0ZAq8JCNrWVEBGiWTCVPMfdNDTtiAnHFpwYo",
    "fV2gJ4bR35xwxOQVXGbeDLMSgyhH1RuWkakPZZytYsSxZhhsVwTNRzivxpCCuzFzNHBwR5gkOUq",
    "IJB6mRXHyo2xvBWarNx12XmspKxcUa2XgZLtbXHUMfCTFuHCp9writhhzLZgzHwVGuLGFIPy7SW",
    "47Rm4aHEALv0rq4GKxgSn8N8KUJWSte4AEj7zq/YWP6PpFYbGfuYvK29vcXPi7CuMKAGhEJZF1C",
    "EacCYpeGt1hYy7bhHIFplUvPKwFmGLEN5emBF61SqEcakpomcu+AYrQTGeohcYiUcaSyaOpjjch",
    "mOEhpB6BoM49DDskFsSUy0a23Pse7GzExaoKliSQmrFqsO21DbJdQkgSMxEvJUdJSMg7zHZ+EFw",
    "GwjSjAxbGL/UpMO7l5GhZtn2975PLt5gXk2lSs+f5rTs1aix5t7Fve90ajft4G/1qbzd07uVmGf",
    "9YKcw23i/ePHv372/9llf7/bIt2X9x6vsv/7xuXm/0a+vj9ttLPCGS/kq8Puztm1XqL/7pz+zXy",
    "tt3ny6pcj/WK3+eZF81i/0b0/X8WdjJ8l9jYK9k7fP3QdH7MszU9Fu2F+MHFrYrNx7fQ/mbP32p",
    "J43MjfZ9e/z1aTv7NT5HF9ifwm/poU8vbT93+4s5p8tv8fS+x/VOD7UTrX+8R1C+jTJ33NWYiV3",
    "qdSGCU5B9VXJyEv1uen5vtl5bwrlvta7Pwu9/l5v8j8umo574rlVvXvsnP8S8PXzQOvgN/8gv1Y",
    "9Xz1ufEfvMudgx7k66ezf3X8iPGw5NSbf8Yj7KDs6zeQp/8BEBOzNw==",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c8_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c8_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(849861154U));
  ssSetChecksum1(S,(1766385219U));
  ssSetChecksum2(S,(1629983199U));
  ssSetChecksum3(S,(924404553U));
}

static void mdlRTW_c8_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c8_sliding_controller_rbdl(SimStruct *S)
{
  SFc8_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc8_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc8_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc8_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc8_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c8_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c8_JITStateAnimation,
    chartInstance->c8_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c8_sliding_controller_rbdl(chartInstance);
}

void c8_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c8_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c8_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c8_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c8_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
