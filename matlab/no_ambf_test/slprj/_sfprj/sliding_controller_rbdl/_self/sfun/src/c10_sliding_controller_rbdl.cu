/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c10_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c10_do_animation_call_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c10_st);
static void sf_gateway_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c10_eML_blk_kernel(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c10_b_q[2], real_T c10_b_qd[2], real_T c10_b_knee[2],
  real_T c10_b_dknee[2], real_T c10_b_ankle[2], real_T c10_b_danke[2]);
static void c10_emlrt_marshallIn(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c10_b_ankle, const char_T *c10_identifier,
  real_T c10_y[2]);
static void c10_b_emlrt_marshallIn(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId,
  real_T c10_y[2]);
static uint8_T c10_c_emlrt_marshallIn
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c10_b_is_active_c10_sliding_controller_rbdl, const char_T *c10_identifier);
static uint8_T c10_d_emlrt_marshallIn
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c10_u, const emlrtMsgIdentifier *c10_parentId);
static void init_dsm_address_info(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c10_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c10_is_active_c10_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c10_do_animation_call_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c10_b_y = NULL;
  const mxArray *c10_c_y = NULL;
  const mxArray *c10_d_y = NULL;
  const mxArray *c10_e_y = NULL;
  const mxArray *c10_f_y = NULL;
  const mxArray *c10_st;
  const mxArray *c10_y = NULL;
  c10_st = NULL;
  c10_y = NULL;
  sf_mex_assign(&c10_y, sf_mex_createcellmatrix(5, 1), false);
  c10_b_y = NULL;
  sf_mex_assign(&c10_b_y, sf_mex_create("y", *chartInstance->c10_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 0, c10_b_y);
  c10_c_y = NULL;
  sf_mex_assign(&c10_c_y, sf_mex_create("y", *chartInstance->c10_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 1, c10_c_y);
  c10_d_y = NULL;
  sf_mex_assign(&c10_d_y, sf_mex_create("y", *chartInstance->c10_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 2, c10_d_y);
  c10_e_y = NULL;
  sf_mex_assign(&c10_e_y, sf_mex_create("y", *chartInstance->c10_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c10_y, 3, c10_e_y);
  c10_f_y = NULL;
  sf_mex_assign(&c10_f_y, sf_mex_create("y",
    &chartInstance->c10_is_active_c10_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c10_y, 4, c10_f_y);
  sf_mex_assign(&c10_st, c10_y, false);
  return c10_st;
}

static void set_sim_state_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c10_st)
{
  const mxArray *c10_u;
  c10_u = sf_mex_dup(c10_st);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 0)),
                       "ankle", *chartInstance->c10_ankle);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 1)),
                       "danke", *chartInstance->c10_danke);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 2)),
                       "dknee", *chartInstance->c10_dknee);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 3)),
                       "knee", *chartInstance->c10_knee);
  chartInstance->c10_is_active_c10_sliding_controller_rbdl =
    c10_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 4)),
    "is_active_c10_sliding_controller_rbdl");
  sf_mex_destroy(&c10_u);
  sf_mex_destroy(&c10_st);
}

static void sf_gateway_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c10_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c10_eML_blk_kernel(chartInstance, *chartInstance->c10_q,
                     *chartInstance->c10_qd, *chartInstance->c10_knee,
                     *chartInstance->c10_dknee, *chartInstance->c10_ankle,
                     *chartInstance->c10_danke);
  c10_do_animation_call_c10_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c10_errCode;
  c10_errCode = hipGetLastError();
  if (c10_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c10_errCode, (char_T *)hipGetErrorName
                       (c10_errCode), (char_T *)hipGetErrorString(c10_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c10_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc10_sliding_controller_rbdl
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c10_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c10_nameCaptureInfo = NULL;
  const char_T *c10_data[4] = {
    "789cc593cd4ac34014856fa516375537fa188e2d08dd57c5222d428b1b9138496e9a31f32393a96dddb8ecd2c7d26772659226fd098688c57a21dc9c9ccc7c77"
    "0e0c543add0a00ec474ff715e0aa0649d5e70d0ed3be03eb95f72b053dab5da8aead8bfd4ec47b4bb5a3a4c189990b49052e56ba4a3049a5194c9f1034868a3f",
    "a39b381ee3386002fbaba2172b71b9622d446cc5ef6d1f9da03f12a0fd7039215f15491e713d149cb75a9247bef279e4ffcb78935ff2b2fd8f4b7899ef293da6"
    "dab5022699a08639a1c571b8f9b9336ead708eb9e3aa91cd71c9fbd890470b79ebfeddc53df1954022a9f1a964c8878abb1ed53671a889d2b0c62109b543a8b0",
    "3d6b4c79809a4401716a13a9ace4abc1d090ef033c1129a72cc7831f9eabe83ed5612fe9ada333ba4ddeecf3bdb94d5e56ffc5dbd67dbc3e9db2c60b7b0c8266"
    "abdd6bd8b673eb9af3e51c37259cb239a040fff5fe5f7dd56cd0", "" };

  c10_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c10_data[0], 1632U, &c10_nameCaptureInfo);
  return c10_nameCaptureInfo;
}

static void c10_eML_blk_kernel(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c10_b_q[2], real_T c10_b_qd[2], real_T c10_b_knee[2],
  real_T c10_b_dknee[2], real_T c10_b_ankle[2], real_T c10_b_danke[2])
{
  real_T c10_a;
  real_T c10_b_a;
  c10_b_knee[0] = -0.324 * muDoubleScalarSin(c10_b_q[0]);
  c10_b_knee[1] = -0.324 * muDoubleScalarCos(c10_b_q[0]);
  c10_a = -0.324 * c10_b_qd[0];
  c10_b_dknee[0] = c10_a * muDoubleScalarCos(c10_b_q[0]);
  c10_b_dknee[1] = c10_a * -muDoubleScalarSin(c10_b_q[0]);
  c10_b_a = -0.39 * (c10_b_qd[0] + c10_b_qd[1]);
  c10_b_ankle[0] = c10_b_knee[0] + -0.39 * muDoubleScalarSin(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_ankle[1] = c10_b_knee[1] + -0.39 * muDoubleScalarCos(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_danke[0] = c10_b_dknee[0] + c10_b_a * muDoubleScalarCos(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_danke[1] = c10_b_dknee[1] + c10_b_a * -muDoubleScalarSin(c10_b_q[0] +
    c10_b_q[1]);
}

static void c10_emlrt_marshallIn(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c10_b_ankle, const char_T *c10_identifier,
  real_T c10_y[2])
{
  emlrtMsgIdentifier c10_thisId;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c10_b_ankle), &c10_thisId,
    c10_y);
  sf_mex_destroy(&c10_b_ankle);
}

static void c10_b_emlrt_marshallIn(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId,
  real_T c10_y[2])
{
  real_T c10_dv[2];
  int32_T c10_i;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), c10_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c10_i = 0; c10_i < 2; c10_i++) {
    c10_y[c10_i] = c10_dv[c10_i];
  }

  sf_mex_destroy(&c10_u);
}

static uint8_T c10_c_emlrt_marshallIn
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c10_b_is_active_c10_sliding_controller_rbdl, const char_T *c10_identifier)
{
  emlrtMsgIdentifier c10_thisId;
  uint8_T c10_y;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_y = c10_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c10_b_is_active_c10_sliding_controller_rbdl), &c10_thisId);
  sf_mex_destroy(&c10_b_is_active_c10_sliding_controller_rbdl);
  return c10_y;
}

static uint8_T c10_d_emlrt_marshallIn
  (SFc10_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c10_u, const emlrtMsgIdentifier *c10_parentId)
{
  uint8_T c10_b_u;
  uint8_T c10_y;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), &c10_b_u, 1, 3, 0U, 0, 0U, 0);
  c10_y = c10_b_u;
  sf_mex_destroy(&c10_u);
  return c10_y;
}

static void init_dsm_address_info(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc10_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c10_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c10_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c10_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c10_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c10_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c10_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c10_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c10_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c10_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c10_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c10_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c10_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6LFA4ifWRyfmFySWZYan"
    "2xoEF+ck5mSmZcen5yfV1KUn5OTWhRflJSSgxRPAKFfLog="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c10_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c10_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c10_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c10_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c10_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c10_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c10_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c10_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc10_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c10_sliding_controller_rbdl
      ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc10_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c10_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c10_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc10_sliding_controller_rbdl
    ((SFc10_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c10_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c10_sliding_controller_rbdl
      ((SFc10_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc10_sliding_controller_rbdl
      ((SFc10_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c10_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFALwESFAiSQwu4KAq0KNDYpiREqF07pZwGuQir5UhcaLnL7A9t5x2",
    "KXPseRS59kfbcJ+gbtLMkLcsUSUV2k7pFaVDykt98O9/M7OzKaw32Pbxu4/3THc+7gd838W57+b",
    "VRjFsLd/684z0sxj+ikbDxIVEk1l7jJUgM34OW3BomxUBMZCWMiQkoEBSxiVSmjk2z2HImZn0rq",
    "OPTP0SMRkEkLQ930ZaEB4KfIltizSHydJkCavoAoYmUtNOoz8l07rEyx34EdKZt3CRBgwls4tzS",
    "+5YblnDonQAdCG0IeqzPfQsMMeCbk1qZTqkOzoAyTjgjolJtRHQACQbYwFES4ueBNSiqDKMRUWY",
    "XIpKC3mOzjFMKKHMyjS/GTBAjFSO8F3PfGS77dsjRn30ZAm8ICPq2q4DMEsmEqc9/0EelPUHGHL",
    "owttN6tgBeWpf8ZwyOQdXGbeLLFBSZwoGonTQLSO8ky9a8SpZhhsXwjKgdivnTENZWL1aODgjmC",
    "YZoUQeDTORADxVLMby1bDYeuMpctWRsnCdbr4JlbL0UmrIwZ+tT4RPOdS1sKJM9SIFnrF1iSDMs",
    "Z63Gac3CocQAu/KuXw1WMEx8AfOlCFllutISIOs732FjuYikVhsZ+1i83b295dfLsIEwoCaEQlU",
    "XUIRpwJhl4a1nC5l2uUcgemUy96rAeYWsQnl6YkX3WKoZxqShiZxLcBmtBcZ6irnElXCkcdE0wV",
    "wuV+EooRGErsEwDvu4bBBbERPtWtsOrruUmdMuaKpYUpFVi6sO21DPFdRpAkdiJuSx6CsZB0WPz",
    "8MLgNVGlGBiuov9S532cfaqVLj9bMs7388+eIv97Myu/P1ogadVweMtfJfnvdW+iG+X5m3jX6uV",
    "220v2H1YmqdTsnO4Tbxf/fzizbe//vL604e/f/J6+PWfq+ZvLc3fyv53djc31tv/bxfju2eNdr5",
    "w0qV6ddgnC351Kvg/WuDfLMb6afSFSF88iXaekmE82Pr81Vdb6djP+J53mv3dKPl79vy+6/hYZV",
    "l9KjoIi4OJGxObb9eO/8sFf2+siMet4nl+/fH4avZ3t8v2nRX6NosxETMOpfr953R8fGkdIQq5P",
    "jruX17HDA8N10bHne1yf6jS0bmgo+M5CX+P/8txXM8+n//5Cv8flPLwIDsvjojbhWBEP9saac5C",
    "3EJGVAqjJOegRmoc8lKeLttH17Xz3rPdv8XP/+3efd7f5lzUvqRd54rnsPdld1V9654Prxu+aT/",
    "wSvjNa6zjquf2d43/zVvvfH2vGH8z/wntR4yHFb+mitf402hS9fY/UKd/AcKpxS8=",
    ""
  };

  static char newstr [1341] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c10_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c10_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042620282U));
  ssSetChecksum1(S,(2411386699U));
  ssSetChecksum2(S,(753018669U));
  ssSetChecksum3(S,(4282078351U));
}

static void mdlRTW_c10_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c10_sliding_controller_rbdl(SimStruct *S)
{
  SFc10_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc10_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc10_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc10_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc10_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c10_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c10_JITStateAnimation,
    chartInstance->c10_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c10_sliding_controller_rbdl(chartInstance);
}

void c10_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c10_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c10_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c10_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c10_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
