/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c34_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c34_do_animation_call_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c34_st);
static void sf_gateway_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c34_eML_blk_kernel(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c34_b_u[7], real_T c34_b_y[6]);
static void c34_emlrt_marshallIn(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c34_b_y, const char_T *c34_identifier, real_T
  c34_c_y[6]);
static void c34_b_emlrt_marshallIn(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c34_b_u, const emlrtMsgIdentifier *c34_parentId,
  real_T c34_b_y[6]);
static uint8_T c34_c_emlrt_marshallIn
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c34_b_is_active_c34_sliding_controller_rbdl, const char_T *c34_identifier);
static uint8_T c34_d_emlrt_marshallIn
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c34_b_u, const emlrtMsgIdentifier *c34_parentId);
static void init_dsm_address_info(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c34_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c34_is_active_c34_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c34_do_animation_call_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c34_b_y = NULL;
  const mxArray *c34_c_y = NULL;
  const mxArray *c34_d_y = NULL;
  const mxArray *c34_st;
  c34_st = NULL;
  c34_b_y = NULL;
  sf_mex_assign(&c34_b_y, sf_mex_createcellmatrix(2, 1), false);
  c34_c_y = NULL;
  sf_mex_assign(&c34_c_y, sf_mex_create("y", *chartInstance->c34_y, 0, 0U, 1U,
    0U, 1, 6), false);
  sf_mex_setcell(c34_b_y, 0, c34_c_y);
  c34_d_y = NULL;
  sf_mex_assign(&c34_d_y, sf_mex_create("y",
    &chartInstance->c34_is_active_c34_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c34_b_y, 1, c34_d_y);
  sf_mex_assign(&c34_st, c34_b_y, false);
  return c34_st;
}

static void set_sim_state_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c34_st)
{
  const mxArray *c34_b_u;
  c34_b_u = sf_mex_dup(c34_st);
  c34_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c34_b_u, 0)),
                       "y", *chartInstance->c34_y);
  chartInstance->c34_is_active_c34_sliding_controller_rbdl =
    c34_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c34_b_u, 1)),
    "is_active_c34_sliding_controller_rbdl");
  sf_mex_destroy(&c34_b_u);
  sf_mex_destroy(&c34_st);
}

static void sf_gateway_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c34_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c34_eML_blk_kernel(chartInstance, *chartInstance->c34_u, *chartInstance->c34_y);
  c34_do_animation_call_c34_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c34_errCode;
  c34_errCode = hipGetLastError();
  if (c34_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c34_errCode, (char_T *)hipGetErrorName
                       (c34_errCode), (char_T *)hipGetErrorString(c34_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c34_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc34_sliding_controller_rbdl
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c34_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c34_nameCaptureInfo = NULL;
  c34_nameCaptureInfo = NULL;
  sf_mex_assign(&c34_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c34_nameCaptureInfo;
}

static void c34_eML_blk_kernel(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c34_b_u[7], real_T c34_b_y[6])
{
  c34_b_y[0] = c34_b_u[1];
  c34_b_y[1] = c34_b_u[2];
  c34_b_y[2] = c34_b_u[3];
  c34_b_y[3] = c34_b_u[0];
  c34_b_y[4] = c34_b_u[4];
  c34_b_y[5] = c34_b_u[5];
}

static void c34_emlrt_marshallIn(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c34_b_y, const char_T *c34_identifier, real_T
  c34_c_y[6])
{
  emlrtMsgIdentifier c34_thisId;
  c34_thisId.fIdentifier = const_cast<const char_T *>(c34_identifier);
  c34_thisId.fParent = NULL;
  c34_thisId.bParentIsCell = false;
  c34_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c34_b_y), &c34_thisId,
    c34_c_y);
  sf_mex_destroy(&c34_b_y);
}

static void c34_b_emlrt_marshallIn(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c34_b_u, const emlrtMsgIdentifier *c34_parentId,
  real_T c34_b_y[6])
{
  real_T c34_dv[6];
  int32_T c34_i;
  sf_mex_import(c34_parentId, sf_mex_dup(c34_b_u), c34_dv, 1, 0, 0U, 1, 0U, 1, 6);
  for (c34_i = 0; c34_i < 6; c34_i++) {
    c34_b_y[c34_i] = c34_dv[c34_i];
  }

  sf_mex_destroy(&c34_b_u);
}

static uint8_T c34_c_emlrt_marshallIn
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c34_b_is_active_c34_sliding_controller_rbdl, const char_T *c34_identifier)
{
  emlrtMsgIdentifier c34_thisId;
  uint8_T c34_b_y;
  c34_thisId.fIdentifier = const_cast<const char_T *>(c34_identifier);
  c34_thisId.fParent = NULL;
  c34_thisId.bParentIsCell = false;
  c34_b_y = c34_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c34_b_is_active_c34_sliding_controller_rbdl), &c34_thisId);
  sf_mex_destroy(&c34_b_is_active_c34_sliding_controller_rbdl);
  return c34_b_y;
}

static uint8_T c34_d_emlrt_marshallIn
  (SFc34_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c34_b_u, const emlrtMsgIdentifier *c34_parentId)
{
  uint8_T c34_b_y;
  uint8_T c34_c_u;
  sf_mex_import(c34_parentId, sf_mex_dup(c34_b_u), &c34_c_u, 1, 3, 0U, 0, 0U, 0);
  c34_b_y = c34_c_u;
  sf_mex_destroy(&c34_b_u);
  return c34_b_y;
}

static void init_dsm_address_info(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc34_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c34_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c34_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c34_y = (real_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c34_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3910899723U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1623685603U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(38977074U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3513601893U);
}

mxArray *sf_c34_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c34_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c34_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c34_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cnGJvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB2aR2y"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c34_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sbIZyOsp5zUyg8V8S5w2l4G";
}

static void sf_opaque_initialize_c34_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c34_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c34_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c34_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c34_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c34_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c34_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc34_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c34_sliding_controller_rbdl
      ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc34_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c34_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c34_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc34_sliding_controller_rbdl
    ((SFc34_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c34_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c34_sliding_controller_rbdl
      ((SFc34_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc34_sliding_controller_rbdl
      ((SFc34_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c34_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV91qFEkU7gxRFNYwiCCIsN7IeiMsUSE3apL52R1ITLCTuCzCUFN9ZrqY6qre+pnJ7DPssm+",
    "x1/soXvoSguC14KnunslsT3ePcTCo2NDpqe7vnDrnfOen4q119j28NvDevu55l/F5Be+al16Xsv",
    "Xa3J2+X/d+ytZ/o5Cw0SFRJNJe5SVIBM9BS24Nk6Ij+rIQxkQfFAiK2FgqU6ZNs8hyJoZtK6jTp",
    "1+EjIZ+KC0PdlGWBAeCT1BbbM0h6mkyBdS0AQITKmkHYZuTwcxiZcaNEOhQ26jKBQ3Gt7EzS+9b",
    "bljMoXUKtCO0IWixPrPNN8RAw5yWuuk81f4UKKOYMyIKvQ2J9iHGABs4jgP8e2ANOpWH0ZAosws",
    "hGYHeY8NEpxSQ18k0fugxQYxUjPBWxBtOcNG2Q4727MsAeEVA0LZdBWQYSyZMOf9+Gz1tCdLj0I",
    "SeHZRr8+EP68g/YTAGVRq3fkOOQJEBHIjSTZOAtE4TtmZZsggzLIITonYo8qchKM1ezBztE+QJj",
    "lCiDAaJkx19pNgIw1uqzUYdl5nLSsZGKdl6GSzR1hpBFQszbW0qGoRzXQo7kvEejIAnWpvEkGpY",
    "qrUYpzULjiQG2KV3eTVYwZD4DNaQImCFdI1ygKTvPMPG8n8ktdrIqIHJ29zbW/y8COsIA6pPKBR",
    "1AUWYBoxZEt5ybQHTjnsEolUmMa8InGbIMpSn+1Y0x1INMSYVTeTMBcdoKTDSA+QSK+FYY9FUwR",
    "yXy3CU0BAC12AYh30sG8QWxES71raDdTdiZtIETRWLC1i1WHXYhlouoSYxHIuhkGPRVjLysx6fh",
    "hcAs40owcRgF/uXmrRx9yIq3Dz72TubZz98xDybyuWf9+b0rBXo8eae+X2v1qr3reGvten8nZO7",
    "lttnPSfncHW8//r3Vf31u/vy1j+bbx6P379cZf//aueb/xvZ+va00c4KZ7SQrw7765xd6wX6b87",
    "pr2dr3ev8PjnQ8aM/jyeDrZMt/9F4kz/8JeVnib21nL3T93dcx8csS/JT0U6QHUzcmth0XDv9W3",
    "P2Xl4Sj6vZ+/R6+3Q1+RvbeR7Xl8jX8dckl7efvv+d7dXk0/1/W2L/3Rzfd5O53iWuW0CXPnjY1",
    "ZwFWOpdKoVRknNQXdULeM7PT83388p5Fyz3tdj5Xe7z8/4x8+ui5bwLllvVv/PO8S8NXzUPvBy+",
    "/gX7ser56nPjX3vnOwf9mK2fzP7VaYSMBwWn3uwzHmH7RV+/gTz9AC7wtPk=",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c34_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c34_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(282109580U));
  ssSetChecksum1(S,(1865282520U));
  ssSetChecksum2(S,(3979513626U));
  ssSetChecksum3(S,(1560115005U));
}

static void mdlRTW_c34_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c34_sliding_controller_rbdl(SimStruct *S)
{
  SFc34_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc34_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc34_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc34_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc34_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c34_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c34_JITStateAnimation,
    chartInstance->c34_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c34_sliding_controller_rbdl(chartInstance);
}

void c34_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c34_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c34_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c34_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c34_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
