#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c41_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c41_do_animation_call_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c41_st);
static void sf_gateway_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c41_eML_blk_kernel(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c41_b_u[7], real_T c41_b_y[7]);
static void c41_emlrt_marshallIn(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c41_b_y, const char_T *c41_identifier, real_T
  c41_c_y[7]);
static void c41_b_emlrt_marshallIn(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c41_b_u, const emlrtMsgIdentifier *c41_parentId,
  real_T c41_b_y[7]);
static uint8_T c41_c_emlrt_marshallIn
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c41_b_is_active_c41_sliding_controller_rbdl, const char_T *c41_identifier);
static uint8_T c41_d_emlrt_marshallIn
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c41_b_u, const emlrtMsgIdentifier *c41_parentId);
static __global__ void c41_eML_blk_kernel_kernel1(real_T c41_b_y[7]);
static void init_dsm_address_info(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c41_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c41_is_active_c41_sliding_controller_rbdl = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c41_gpu_y, 56UL);
}

static void initialize_params_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c41_do_animation_call_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c41_b_y = NULL;
  const mxArray *c41_c_y = NULL;
  const mxArray *c41_d_y = NULL;
  const mxArray *c41_st;
  c41_st = NULL;
  c41_b_y = NULL;
  sf_mex_assign(&c41_b_y, sf_mex_createcellmatrix(2, 1), false);
  c41_c_y = NULL;
  sf_mex_assign(&c41_c_y, sf_mex_create("y", *chartInstance->c41_y, 0, 0U, 1U,
    0U, 1, 7), false);
  sf_mex_setcell(c41_b_y, 0, c41_c_y);
  c41_d_y = NULL;
  sf_mex_assign(&c41_d_y, sf_mex_create("y",
    &chartInstance->c41_is_active_c41_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c41_b_y, 1, c41_d_y);
  sf_mex_assign(&c41_st, c41_b_y, false);
  return c41_st;
}

static void set_sim_state_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c41_st)
{
  const mxArray *c41_b_u;
  c41_b_u = sf_mex_dup(c41_st);
  c41_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c41_b_u, 0)),
                       "y", *chartInstance->c41_y);
  chartInstance->c41_is_active_c41_sliding_controller_rbdl =
    c41_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c41_b_u, 1)),
    "is_active_c41_sliding_controller_rbdl");
  sf_mex_destroy(&c41_b_u);
  sf_mex_destroy(&c41_st);
}

static void sf_gateway_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  real_T c41_dv[7];
  real_T c41_dv1[7];
  int32_T c41_i;
  int32_T c41_i1;
  chartInstance->c41_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c41_i = 0; c41_i < 7; c41_i++) {
    c41_dv[c41_i] = (*chartInstance->c41_u)[c41_i];
  }

  c41_eML_blk_kernel(chartInstance, c41_dv, c41_dv1);
  for (c41_i1 = 0; c41_i1 < 7; c41_i1++) {
    (*chartInstance->c41_y)[c41_i1] = c41_dv1[c41_i1];
  }

  c41_do_animation_call_c41_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c41_errCode;
  hipFree(*chartInstance->c41_gpu_y);
  c41_errCode = hipGetLastError();
  if (c41_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c41_errCode, (char_T *)hipGetErrorName
                       (c41_errCode), (char_T *)hipGetErrorString(c41_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c41_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc41_sliding_controller_rbdl
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c41_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c41_nameCaptureInfo = NULL;
  c41_nameCaptureInfo = NULL;
  sf_mex_assign(&c41_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c41_nameCaptureInfo;
}

static void c41_eML_blk_kernel(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c41_b_u[7], real_T c41_b_y[7])
{
  c41_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c41_gpu_y);
  hipMemcpy(c41_b_y, *chartInstance->c41_gpu_y, 56UL, hipMemcpyDeviceToHost);
  c41_b_y[0] = c41_b_u[1];
  c41_b_y[1] = c41_b_u[2];
  c41_b_y[2] = c41_b_u[3];
  c41_b_y[3] = c41_b_u[0];
  c41_b_y[4] = c41_b_u[4];
  c41_b_y[5] = c41_b_u[5];
}

static void c41_emlrt_marshallIn(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c41_b_y, const char_T *c41_identifier, real_T
  c41_c_y[7])
{
  emlrtMsgIdentifier c41_thisId;
  c41_thisId.fIdentifier = const_cast<const char_T *>(c41_identifier);
  c41_thisId.fParent = NULL;
  c41_thisId.bParentIsCell = false;
  c41_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c41_b_y), &c41_thisId,
    c41_c_y);
  sf_mex_destroy(&c41_b_y);
}

static void c41_b_emlrt_marshallIn(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c41_b_u, const emlrtMsgIdentifier *c41_parentId,
  real_T c41_b_y[7])
{
  real_T c41_dv[7];
  int32_T c41_i;
  sf_mex_import(c41_parentId, sf_mex_dup(c41_b_u), c41_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c41_i = 0; c41_i < 7; c41_i++) {
    c41_b_y[c41_i] = c41_dv[c41_i];
  }

  sf_mex_destroy(&c41_b_u);
}

static uint8_T c41_c_emlrt_marshallIn
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c41_b_is_active_c41_sliding_controller_rbdl, const char_T *c41_identifier)
{
  emlrtMsgIdentifier c41_thisId;
  uint8_T c41_b_y;
  c41_thisId.fIdentifier = const_cast<const char_T *>(c41_identifier);
  c41_thisId.fParent = NULL;
  c41_thisId.bParentIsCell = false;
  c41_b_y = c41_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c41_b_is_active_c41_sliding_controller_rbdl), &c41_thisId);
  sf_mex_destroy(&c41_b_is_active_c41_sliding_controller_rbdl);
  return c41_b_y;
}

static uint8_T c41_d_emlrt_marshallIn
  (SFc41_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c41_b_u, const emlrtMsgIdentifier *c41_parentId)
{
  uint8_T c41_b_y;
  uint8_T c41_c_u;
  sf_mex_import(c41_parentId, sf_mex_dup(c41_b_u), &c41_c_u, 1, 3, 0U, 0, 0U, 0);
  c41_b_y = c41_c_u;
  sf_mex_destroy(&c41_b_u);
  return c41_b_y;
}

static __global__ __launch_bounds__(32, 1) void c41_eML_blk_kernel_kernel1
  (real_T c41_b_y[7])
{
  uint64_T c41_threadId;
  int32_T c41_i;
  c41_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c41_i = (int32_T)c41_threadId;
  if (c41_i < 7) {
    c41_b_y[c41_i] = 0.0;
  }
}

static void init_dsm_address_info(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc41_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c41_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c41_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c41_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c41_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3761594018U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1451773495U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2587411843U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4121430803U);
}

mxArray *sf_c41_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c41_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c41_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c41_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8ckmhvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB2Ih2w"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c41_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sEsyT33uilQMeLbc2q1KEGD";
}

static void sf_opaque_initialize_c41_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c41_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c41_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c41_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c41_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c41_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c41_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc41_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c41_sliding_controller_rbdl
      ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc41_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c41_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c41_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc41_sliding_controller_rbdl
    ((SFc41_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c41_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c41_sliding_controller_rbdl
      ((SFc41_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc41_sliding_controller_rbdl
      ((SFc41_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c41_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV82O00gQ9kQDWiRAI7TSXpAYDgiOy8+B0zJMfiAiYWZxBrhFnXYlbqXdbfonM7ly4sIr8AJ",
    "cEE/AY/AEnLlwp9p2MsGxHYaIEazWksdp+6vqqvrqp8fbaHc9vC7ivXPJ887i8w+8a156ncnWGw",
    "t3+n7Tu56tX6OQsNE+USTSXuUlSARPQEtuDZOiLYayEMbEEBQIithYKlOmTbPIcibGLSuo06efh",
    "YyGfigtD3ZRlgR7gk9RW2zNPuppMAXUtAACEyppR2GLk9HcYmUO6yHQsbZRlQsajG9jZ5buWm5Y",
    "zKF5BLQttCFosT62zTfEQN0clbrpPNX+DCijmDMiCr0NifYhxgAbOIgD/LtnDTqVh9GQKLMLIZm",
    "A7rBxolMKyOtkGj8MmCBGKkZ4M+J1J7hs2z5He7oyAF4RELRtVwEZx5IJU86/30JPm4IMODRgYE",
    "fl2nx4YR35TxkcgiqN27AuJ6DICPZE6aZJQJpHCVvzLFmGGRbBU6LuU+RPQ1CavZg52ifIE/RQo",
    "gwGiZNt3VNsguEt1WajtsvMVSVjo5RsvQqWaGtOoIqFubYWFXXCuS6F9WTcgQnwRGuDGFINS7UW",
    "47RmQU9igF16l1eDFQyJz2B1KQJWSNckB0j6zmNsLN8iqdVGRnVM3kans/x5GdYWBtSQUCjqAoo",
    "wDRizJLzl2gKmHfcIRKtMYl4ROM2QVShPD61oHEo1xphUNJFjFxyjpcBIj5BLrIQDjUVTBXNcrs",
    "JRQkMIXINhHLpYNogtiIl2re0+1t2EmWkDNFUsLmDVYtVhG2q6hJrGcCDGQh6KlpKRn/X4NLwAm",
    "G1ECSZGu9i/1LSFuxdR4ebZ397xPDv/HfNsJpd/3ljQs1Ggx1t45vc9V6vet4a/Nmbzd0HuQm6f",
    "zZycw23h/WH71T3x8v2Z518+7by5+vbNOvu/q51s/l/M1pdnjXZeOJOlfHXYhwt2bRbo/2tB/1a",
    "21k097d2+bRn/twudAb314uaj5oNGys8Ke2s5e2fvt13HxyxL8lPRdpAdTNya2HRcO/13F+w9uy",
    "Ie57L36fX53nryf+7kedxcIb+Fv6a5vP3x/bd31pNP93++wv5rOb6vJXO9T1y3gD69c7OvOQuw1",
    "PtUCqMk56D6ahDwnJ8/mu8nlfNOWe53sfN/uZ/P+/fMr9OW805Zbl3/TjrHfzV81TzwcvitX9iP",
    "dc9XPxv/0TvZOehKtv5n/q9OPWQ8KDj1Zp/xCDss+vofyNOvcX60OA==",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c41_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c41_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1065885888U));
  ssSetChecksum1(S,(95584878U));
  ssSetChecksum2(S,(1088877912U));
  ssSetChecksum3(S,(2662015390U));
}

static void mdlRTW_c41_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c41_sliding_controller_rbdl(SimStruct *S)
{
  SFc41_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc41_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc41_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc41_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc41_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c41_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c41_JITStateAnimation,
    chartInstance->c41_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c41_sliding_controller_rbdl(chartInstance);
}

void c41_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c41_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c41_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c41_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c41_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
