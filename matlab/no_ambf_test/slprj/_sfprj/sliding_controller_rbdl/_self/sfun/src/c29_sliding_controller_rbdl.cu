/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c29_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c29_do_animation_call_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c29_st);
static void sf_gateway_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c29_eML_blk_kernel(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c29_b_q[2], real_T c29_b_y[4]);
static void c29_emlrt_marshallIn(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c29_b_y, const char_T *c29_identifier, real_T
  c29_c_y[4]);
static void c29_b_emlrt_marshallIn(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c29_u, const emlrtMsgIdentifier *c29_parentId,
  real_T c29_b_y[4]);
static uint8_T c29_c_emlrt_marshallIn
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c29_b_is_active_c29_sliding_controller_rbdl, const char_T *c29_identifier);
static uint8_T c29_d_emlrt_marshallIn
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c29_u, const emlrtMsgIdentifier *c29_parentId);
static void init_dsm_address_info(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c29_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c29_is_active_c29_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c29_do_animation_call_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c29_b_y = NULL;
  const mxArray *c29_c_y = NULL;
  const mxArray *c29_d_y = NULL;
  const mxArray *c29_st;
  c29_st = NULL;
  c29_b_y = NULL;
  sf_mex_assign(&c29_b_y, sf_mex_createcellmatrix(2, 1), false);
  c29_c_y = NULL;
  sf_mex_assign(&c29_c_y, sf_mex_create("y", *chartInstance->c29_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c29_b_y, 0, c29_c_y);
  c29_d_y = NULL;
  sf_mex_assign(&c29_d_y, sf_mex_create("y",
    &chartInstance->c29_is_active_c29_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c29_b_y, 1, c29_d_y);
  sf_mex_assign(&c29_st, c29_b_y, false);
  return c29_st;
}

static void set_sim_state_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c29_st)
{
  const mxArray *c29_u;
  c29_u = sf_mex_dup(c29_st);
  c29_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c29_u, 0)), "y",
                       *chartInstance->c29_y);
  chartInstance->c29_is_active_c29_sliding_controller_rbdl =
    c29_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c29_u, 1)),
    "is_active_c29_sliding_controller_rbdl");
  sf_mex_destroy(&c29_u);
  sf_mex_destroy(&c29_st);
}

static void sf_gateway_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c29_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c29_eML_blk_kernel(chartInstance, *chartInstance->c29_q, *chartInstance->c29_y);
  c29_do_animation_call_c29_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c29_errCode;
  c29_errCode = hipGetLastError();
  if (c29_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c29_errCode, (char_T *)hipGetErrorName
                       (c29_errCode), (char_T *)hipGetErrorString(c29_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c29_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc29_sliding_controller_rbdl
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c29_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c29_nameCaptureInfo = NULL;
  c29_nameCaptureInfo = NULL;
  sf_mex_assign(&c29_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c29_nameCaptureInfo;
}

static void c29_eML_blk_kernel(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c29_b_q[2], real_T c29_b_y[4])
{
  c29_b_y[0] = -0.324 * muDoubleScalarSin(c29_b_q[0]) - 0.39 * muDoubleScalarSin
    (c29_b_q[0] + c29_b_q[1]);
  c29_b_y[2] = 0.324 * muDoubleScalarCos(c29_b_q[0]) + 0.39 * muDoubleScalarSin
    (c29_b_q[0] + c29_b_q[1]);
  c29_b_y[1] = -0.39 * muDoubleScalarSin(c29_b_q[0] + c29_b_q[1]);
  c29_b_y[3] = 0.39 * muDoubleScalarCos(c29_b_q[0] + c29_b_q[1]);
}

static void c29_emlrt_marshallIn(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c29_b_y, const char_T *c29_identifier, real_T
  c29_c_y[4])
{
  emlrtMsgIdentifier c29_thisId;
  c29_thisId.fIdentifier = const_cast<const char_T *>(c29_identifier);
  c29_thisId.fParent = NULL;
  c29_thisId.bParentIsCell = false;
  c29_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c29_b_y), &c29_thisId,
    c29_c_y);
  sf_mex_destroy(&c29_b_y);
}

static void c29_b_emlrt_marshallIn(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c29_u, const emlrtMsgIdentifier *c29_parentId,
  real_T c29_b_y[4])
{
  real_T c29_dv[4];
  int32_T c29_i;
  sf_mex_import(c29_parentId, sf_mex_dup(c29_u), c29_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c29_i = 0; c29_i < 4; c29_i++) {
    c29_b_y[c29_i] = c29_dv[c29_i];
  }

  sf_mex_destroy(&c29_u);
}

static uint8_T c29_c_emlrt_marshallIn
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c29_b_is_active_c29_sliding_controller_rbdl, const char_T *c29_identifier)
{
  emlrtMsgIdentifier c29_thisId;
  uint8_T c29_b_y;
  c29_thisId.fIdentifier = const_cast<const char_T *>(c29_identifier);
  c29_thisId.fParent = NULL;
  c29_thisId.bParentIsCell = false;
  c29_b_y = c29_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c29_b_is_active_c29_sliding_controller_rbdl), &c29_thisId);
  sf_mex_destroy(&c29_b_is_active_c29_sliding_controller_rbdl);
  return c29_b_y;
}

static uint8_T c29_d_emlrt_marshallIn
  (SFc29_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c29_u, const emlrtMsgIdentifier *c29_parentId)
{
  uint8_T c29_b_u;
  uint8_T c29_b_y;
  sf_mex_import(c29_parentId, sf_mex_dup(c29_u), &c29_b_u, 1, 3, 0U, 0, 0U, 0);
  c29_b_y = c29_b_u;
  sf_mex_destroy(&c29_u);
  return c29_b_y;
}

static void init_dsm_address_info(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc29_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c29_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c29_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c29_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c29_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(552332206U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2608078302U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1441289006U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4066884445U);
}

mxArray *sf_c29_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c29_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c29_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c29_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8clGlvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB2+B22"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c29_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sjS5Om55RaK9v6MEldWgzUF";
}

static void sf_opaque_initialize_c29_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c29_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c29_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c29_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c29_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c29_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c29_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc29_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c29_sliding_controller_rbdl
      ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc29_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c29_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c29_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc29_sliding_controller_rbdl
    ((SFc29_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c29_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c29_sliding_controller_rbdl
      ((SFc29_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc29_sliding_controller_rbdl
      ((SFc29_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c29_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u5EQQdkZh2SBYRQhpL0jkEoUjQgoiByDJ/CijzZAoTrLcRp12zbgz7W7TP06Gd9jTijf",
    "guAeeZM975wWQeAGqbc9k8NieTUaJAsKS42n7q+qq+uqn4610ex5ez/De/dTznuDzKd4NL7s+yN",
    "crM3f2ftXbytevUEjY6JgoEmmv9hIkghPQklvDpOiKgSyFMTEABYIiNpbKVGnTLLKciVHHCur06",
    "Zcho6EfSsuDfZQlwZHgY9QWW3OMelpMATUdgMCEStph2OFkOLVYmatmCHSkbVTnggbj29iZpXuW",
    "GxZzaF8D7QptCFqsb2zzDTHQNNeVbjpPtT8ByijmjIhSb0OifYgxwAbO4gD/HlmDThVhNCTK7EN",
    "IEtCHbJTqlAKKOpnGDxdMECMVI7wd8aYTnLftmKM9PRkArwkI2ravgIxiyYSp5t/voKdtQS44tO",
    "DCDqu1+fCzdeSfM7gCVRm3QVMmoMgQjkTlpmlA2tcpW9MsmYcZFsE5UXsU+dMQVGYvZo72CfIEp",
    "yhRBYPUya4+VSzB8FZqs1HXZeaikrFRRrZeBEu1tROoY2GqrUNFk3CuK2GnMj6EBHiqtUUMqYdl",
    "WstxWrPgVGKAXXpXV4MVDInPYU0pAlZKV1IApH3nR2ws/0RSq42Mmpi8rcPD+c/zsK4woAaEQlk",
    "XUIRpwJil4a3WFjDtuEcgWmVS88rAWYYsQnl6YEXrSqoRxqSmidy44BitBEZ6iFxiJZxpLJo6mO",
    "NyEY4SGkLgGgzj0MOyQWxJTLRrbXtYdwkz4xZoqlhcwqrFqsM21HYJNY7hTIyEvBIdJSM/7/FZe",
    "AEw24gSTAz3sX+pcQd3L6PCzbOvvJt59vF7zLOJXPH55YyelRI93syzuO9ao37fBv5amczfGblP",
    "CvusFuQcbh3vjy43fv1r48347esP/zjY+u27Zfb/vXG7+f8sX38+abTTwknm8tVhD2bsWi3R/3x",
    "G/3q+1pf+9lG0vX1CXuwk3/TaPHg5/OWsk/GzwN5Gwd7J+w3X8THL0vxUtBvkBxO3JjYb107/tz",
    "P2PlkQj7X8fXb9+cNy8p/tFnlcXSC/jr/Ghby9+/4bu8vJZ/v/tMD+zQLfm+lc7xPXLaBPv97pa",
    "84CLPU+lcIoyTmovroIeMHPu+b7beW8B5b7t9j5v9z98/4+8+uh5bwHllvWv9vO8ceGr5sHXgG/",
    "/oj9WPZ8dd/4d97tzkFf5Ovvp//qNEPGg5JTb/4Zj7CDsq//gTz9G93ys2k=",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c29_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c29_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2501929482U));
  ssSetChecksum1(S,(2041323763U));
  ssSetChecksum2(S,(3808924622U));
  ssSetChecksum3(S,(1034168136U));
}

static void mdlRTW_c29_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c29_sliding_controller_rbdl(SimStruct *S)
{
  SFc29_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc29_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc29_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc29_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc29_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c29_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c29_JITStateAnimation,
    chartInstance->c29_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c29_sliding_controller_rbdl(chartInstance);
}

void c29_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c29_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c29_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c29_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c29_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
