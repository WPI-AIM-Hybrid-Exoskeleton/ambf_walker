#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c39_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c39_do_animation_call_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_st);
static void sf_gateway_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c39_emlrt_marshallIn(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c39_b_msg, const char_T *c39_identifier,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_y);
static void c39_b_emlrt_marshallIn(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c39_u, const emlrtMsgIdentifier *c39_parentId,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_y);
static void c39_c_emlrt_marshallIn(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c39_u, const emlrtMsgIdentifier *c39_parentId,
  uint8_T c39_y[8]);
static c39_SL_Bus_ROSVariableLengthArrayInfo c39_d_emlrt_marshallIn
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_u, const emlrtMsgIdentifier *c39_parentId);
static uint32_T c39_e_emlrt_marshallIn
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_u, const emlrtMsgIdentifier *c39_parentId);
static void c39_f_emlrt_marshallIn(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c39_u, const emlrtMsgIdentifier *c39_parentId,
  real_T c39_y[7]);
static uint8_T c39_g_emlrt_marshallIn
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_b_is_active_c39_sliding_controller_rbdl, const char_T *c39_identifier);
static uint8_T c39_h_emlrt_marshallIn
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_u, const emlrtMsgIdentifier *c39_parentId);
static void c39_eML_blk_kernel(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c39_b_blankMsg);
static __global__ void c39_sf_gateway_c39_sliding_controller_rbdl_kernel1
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c39_b_blankMsg,
   c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg);
static __global__ void c39_sf_gateway_c39_sliding_controller_rbdl_kernel2
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c39_b_blankMsg,
   c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg);
static __global__ void c39_sf_gateway_c39_sliding_controller_rbdl_kernel3
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg,
   c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_r);
static __global__ void c39_sf_gateway_c39_sliding_controller_rbdl_kernel4
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg,
   c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_r);
static __global__ void c39_sf_gateway_c39_sliding_controller_rbdl_kernel5(const
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_r,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg);
static __global__ void c39_sf_gateway_c39_sliding_controller_rbdl_kernel6(const
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_r,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg);
static __global__ void c39_eML_blk_kernel_kernel7
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c39_b_blankMsg);
static __global__ void c39_eML_blk_kernel_kernel8(const uint8_T c39_uv[5],
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c39_b_blankMsg);
static __global__ void c39_eML_blk_kernel_kernel9
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c39_b_blankMsg);
static void init_dsm_address_info(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c39_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c39_is_active_c39_sliding_controller_rbdl = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c39_gpu_uv, 5UL);
  hipMalloc(&chartInstance->c39_gpu_blankMsg, 208UL);
}

static void initialize_params_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c39_do_animation_call_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  static const char_T *c39_sv[8] = { "ModelName", "ModelName_SL_Info", "Q",
    "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  static const char_T *c39_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c39_b_y = NULL;
  const mxArray *c39_c_y = NULL;
  const mxArray *c39_d_y = NULL;
  const mxArray *c39_e_y = NULL;
  const mxArray *c39_f_y = NULL;
  const mxArray *c39_g_y = NULL;
  const mxArray *c39_h_y = NULL;
  const mxArray *c39_i_y = NULL;
  const mxArray *c39_j_y = NULL;
  const mxArray *c39_k_y = NULL;
  const mxArray *c39_l_y = NULL;
  const mxArray *c39_m_y = NULL;
  const mxArray *c39_n_y = NULL;
  const mxArray *c39_o_y = NULL;
  const mxArray *c39_p_y = NULL;
  const mxArray *c39_q_y = NULL;
  const mxArray *c39_r_y = NULL;
  const mxArray *c39_s_y = NULL;
  const mxArray *c39_st;
  const mxArray *c39_y = NULL;
  real_T c39_d_u[7];
  int32_T c39_i;
  int32_T c39_i1;
  int32_T c39_i2;
  int32_T c39_i3;
  uint32_T c39_b_u;
  uint32_T c39_c_u;
  uint32_T c39_e_u;
  uint32_T c39_f_u;
  uint32_T c39_g_u;
  uint32_T c39_h_u;
  uint32_T c39_i_u;
  uint32_T c39_j_u;
  uint32_T c39_u_CurrentLength;
  uint32_T c39_u_ReceivedLength;
  uint8_T c39_u[8];
  c39_st = NULL;
  c39_y = NULL;
  sf_mex_assign(&c39_y, sf_mex_createcellmatrix(2, 1), false);
  c39_b_y = NULL;
  sf_mex_assign(&c39_b_y, sf_mex_createstruct("structure", 8, c39_sv, 2, 1, 1),
                false);
  for (c39_i = 0; c39_i < 8; c39_i++) {
    c39_u[c39_i] = ((uint8_T *)&((char_T *)chartInstance->c39_msg)[0])[c39_i];
  }

  c39_c_y = NULL;
  sf_mex_assign(&c39_c_y, sf_mex_create("y", c39_u, 3, 0U, 1U, 0U, 1, 8), false);
  sf_mex_setfieldbynum(c39_b_y, 0, "ModelName", c39_c_y, 0);
  c39_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [8])[0];
  c39_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [8])[4];
  c39_d_y = NULL;
  sf_mex_assign(&c39_d_y, sf_mex_createstruct("structure", 2, c39_sv1, 2, 1, 1),
                false);
  c39_b_u = c39_u_CurrentLength;
  c39_e_y = NULL;
  sf_mex_assign(&c39_e_y, sf_mex_create("y", &c39_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c39_d_y, 0, "CurrentLength", c39_e_y, 0);
  c39_c_u = c39_u_ReceivedLength;
  c39_f_y = NULL;
  sf_mex_assign(&c39_f_y, sf_mex_create("y", &c39_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c39_d_y, 0, "ReceivedLength", c39_f_y, 1);
  sf_mex_setfieldbynum(c39_b_y, 0, "ModelName_SL_Info", c39_d_y, 1);
  for (c39_i1 = 0; c39_i1 < 7; c39_i1++) {
    c39_d_u[c39_i1] = ((real_T *)&((char_T *)chartInstance->c39_msg)[16])[c39_i1];
  }

  c39_g_y = NULL;
  sf_mex_assign(&c39_g_y, sf_mex_create("y", c39_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c39_b_y, 0, "Q", c39_g_y, 2);
  c39_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [72])[0];
  c39_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [72])[4];
  c39_h_y = NULL;
  sf_mex_assign(&c39_h_y, sf_mex_createstruct("structure", 2, c39_sv1, 2, 1, 1),
                false);
  c39_e_u = c39_u_CurrentLength;
  c39_i_y = NULL;
  sf_mex_assign(&c39_i_y, sf_mex_create("y", &c39_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c39_h_y, 0, "CurrentLength", c39_i_y, 0);
  c39_f_u = c39_u_ReceivedLength;
  c39_j_y = NULL;
  sf_mex_assign(&c39_j_y, sf_mex_create("y", &c39_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c39_h_y, 0, "ReceivedLength", c39_j_y, 1);
  sf_mex_setfieldbynum(c39_b_y, 0, "Q_SL_Info", c39_h_y, 3);
  for (c39_i2 = 0; c39_i2 < 7; c39_i2++) {
    c39_d_u[c39_i2] = ((real_T *)&((char_T *)chartInstance->c39_msg)[80])[c39_i2];
  }

  c39_k_y = NULL;
  sf_mex_assign(&c39_k_y, sf_mex_create("y", c39_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c39_b_y, 0, "Qd", c39_k_y, 4);
  c39_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [136])[0];
  c39_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [136])[4];
  c39_l_y = NULL;
  sf_mex_assign(&c39_l_y, sf_mex_createstruct("structure", 2, c39_sv1, 2, 1, 1),
                false);
  c39_g_u = c39_u_CurrentLength;
  c39_m_y = NULL;
  sf_mex_assign(&c39_m_y, sf_mex_create("y", &c39_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c39_l_y, 0, "CurrentLength", c39_m_y, 0);
  c39_h_u = c39_u_ReceivedLength;
  c39_n_y = NULL;
  sf_mex_assign(&c39_n_y, sf_mex_create("y", &c39_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c39_l_y, 0, "ReceivedLength", c39_n_y, 1);
  sf_mex_setfieldbynum(c39_b_y, 0, "Qd_SL_Info", c39_l_y, 5);
  for (c39_i3 = 0; c39_i3 < 7; c39_i3++) {
    c39_d_u[c39_i3] = ((real_T *)&((char_T *)chartInstance->c39_msg)[144])
      [c39_i3];
  }

  c39_o_y = NULL;
  sf_mex_assign(&c39_o_y, sf_mex_create("y", c39_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c39_b_y, 0, "Tau", c39_o_y, 6);
  c39_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [200])[0];
  c39_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [200])[4];
  c39_p_y = NULL;
  sf_mex_assign(&c39_p_y, sf_mex_createstruct("structure", 2, c39_sv1, 2, 1, 1),
                false);
  c39_i_u = c39_u_CurrentLength;
  c39_q_y = NULL;
  sf_mex_assign(&c39_q_y, sf_mex_create("y", &c39_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c39_p_y, 0, "CurrentLength", c39_q_y, 0);
  c39_j_u = c39_u_ReceivedLength;
  c39_r_y = NULL;
  sf_mex_assign(&c39_r_y, sf_mex_create("y", &c39_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c39_p_y, 0, "ReceivedLength", c39_r_y, 1);
  sf_mex_setfieldbynum(c39_b_y, 0, "Tau_SL_Info", c39_p_y, 7);
  sf_mex_setcell(c39_y, 0, c39_b_y);
  c39_s_y = NULL;
  sf_mex_assign(&c39_s_y, sf_mex_create("y",
    &chartInstance->c39_is_active_c39_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c39_y, 1, c39_s_y);
  sf_mex_assign(&c39_st, c39_y, false);
  return c39_st;
}

static void set_sim_state_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_st)
{
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 c39_r;
  const mxArray *c39_u;
  int32_T c39_i;
  int32_T c39_i1;
  c39_u = sf_mex_dup(c39_st);
  c39_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c39_u, 0)),
                       "msg", &c39_r);
  for (c39_i = 0; c39_i < 8; c39_i++) {
    ((uint8_T *)&((char_T *)chartInstance->c39_msg)[0])[c39_i] =
      c39_r.ModelName[c39_i];
  }

  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[8])[0] = c39_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[8])[4] = c39_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[72])[0] = c39_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[72])[4] = c39_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[136])[0] = c39_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[136])[4] = c39_r.Qd_SL_Info.ReceivedLength;
  for (c39_i1 = 0; c39_i1 < 7; c39_i1++) {
    ((real_T *)&((char_T *)chartInstance->c39_msg)[16])[c39_i1] = c39_r.Q[c39_i1];
    ((real_T *)&((char_T *)chartInstance->c39_msg)[80])[c39_i1] =
      c39_r.Qd[c39_i1];
    ((real_T *)&((char_T *)chartInstance->c39_msg)[144])[c39_i1] =
      c39_r.Tau[c39_i1];
  }

  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[200])[0] = c39_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[200])[4] = c39_r.Tau_SL_Info.ReceivedLength;
  chartInstance->c39_is_active_c39_sliding_controller_rbdl =
    c39_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c39_u, 1)),
    "is_active_c39_sliding_controller_rbdl");
  sf_mex_destroy(&c39_u);
  sf_mex_destroy(&c39_st);
}

static void sf_gateway_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 c39_r;
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
    *c39_b_gpu_blankMsg;
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_gpu_msg;
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_gpu_r;
  hipMalloc(&c39_gpu_r, 208UL);
  hipMalloc(&c39_gpu_msg, 208UL);
  hipMalloc(&c39_b_gpu_blankMsg, 208UL);
  chartInstance->c39_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(c39_b_gpu_blankMsg, chartInstance->c39_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c39_gpu_msg, chartInstance->c39_msg, 208UL, hipMemcpyHostToDevice);
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c39_b_gpu_blankMsg, c39_gpu_msg);
  hipMemcpy(chartInstance->c39_msg, c39_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c39_blankMsg, c39_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[8])[0] = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_blankMsg)[8])[0];
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[8])[4] = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_blankMsg)[8])[4];
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[72])[0] = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_blankMsg)[72])[0];
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[72])[4] = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_blankMsg)[72])[4];
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[136])[0] = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_blankMsg)[136])[0];
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[136])[4] = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_blankMsg)[136])[4];
  hipMemcpy(c39_b_gpu_blankMsg, chartInstance->c39_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c39_gpu_msg, chartInstance->c39_msg, 208UL, hipMemcpyHostToDevice);
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel2<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c39_b_gpu_blankMsg, c39_gpu_msg);
  hipMemcpy(chartInstance->c39_msg, c39_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c39_blankMsg, c39_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_blankMsg)[200])[4];
  hipMemcpy(c39_gpu_msg, chartInstance->c39_msg, 208UL, hipMemcpyHostToDevice);
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel3<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c39_gpu_msg, c39_gpu_r);
  hipMemcpy(chartInstance->c39_msg, c39_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c39_r, c39_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c39_r.ModelName_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [8])[0];
  c39_r.ModelName_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [8])[4];
  c39_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [72])[0];
  c39_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [72])[4];
  c39_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [136])[0];
  c39_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [136])[4];
  hipMemcpy(c39_gpu_r, &c39_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c39_gpu_msg, chartInstance->c39_msg, 208UL, hipMemcpyHostToDevice);
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel4<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c39_gpu_msg, c39_gpu_r);
  hipMemcpy(chartInstance->c39_msg, c39_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c39_r, c39_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c39_r.Tau_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [200])[0];
  c39_r.Tau_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c39_msg)
    [200])[4];
  c39_eML_blk_kernel(chartInstance, &c39_r);
  hipMemcpy(c39_gpu_r, &c39_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c39_gpu_msg, chartInstance->c39_msg, 208UL, hipMemcpyHostToDevice);
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel5<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c39_gpu_r, c39_gpu_msg);
  hipMemcpy(chartInstance->c39_msg, c39_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[8])[0] = c39_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[8])[4] = c39_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[72])[0] = c39_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[72])[4] = c39_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[136])[0] = c39_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[136])[4] = c39_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c39_gpu_msg, chartInstance->c39_msg, 208UL, hipMemcpyHostToDevice);
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel6<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c39_gpu_r, c39_gpu_msg);
  hipMemcpy(chartInstance->c39_msg, c39_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[200])[0] = c39_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c39_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c39_msg)[200])[4] = c39_r.Tau_SL_Info.ReceivedLength;
  c39_do_animation_call_c39_sliding_controller_rbdl(chartInstance);
  hipFree(c39_b_gpu_blankMsg);
  hipFree(c39_gpu_msg);
  hipFree(c39_gpu_r);
}

static void mdl_start_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c39_errCode;
  hipFree(*chartInstance->c39_gpu_uv);
  hipFree(chartInstance->c39_gpu_blankMsg);
  c39_errCode = hipGetLastError();
  if (c39_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c39_errCode, (char_T *)hipGetErrorName
                       (c39_errCode), (char_T *)hipGetErrorString(c39_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c39_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc39_sliding_controller_rbdl
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c39_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c39_nameCaptureInfo = NULL;
  c39_nameCaptureInfo = NULL;
  sf_mex_assign(&c39_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c39_nameCaptureInfo;
}

static void c39_emlrt_marshallIn(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c39_b_msg, const char_T *c39_identifier,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_y)
{
  emlrtMsgIdentifier c39_thisId;
  c39_thisId.fIdentifier = const_cast<const char_T *>(c39_identifier);
  c39_thisId.fParent = NULL;
  c39_thisId.bParentIsCell = false;
  c39_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c39_b_msg), &c39_thisId,
    c39_y);
  sf_mex_destroy(&c39_b_msg);
}

static void c39_b_emlrt_marshallIn(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c39_u, const emlrtMsgIdentifier *c39_parentId,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_y)
{
  static const char_T *c39_fieldNames[8] = { "ModelName", "ModelName_SL_Info",
    "Q", "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  emlrtMsgIdentifier c39_thisId;
  c39_thisId.fParent = c39_parentId;
  c39_thisId.bParentIsCell = false;
  sf_mex_check_struct(c39_parentId, c39_u, 8, c39_fieldNames, 0U, NULL);
  c39_thisId.fIdentifier = "ModelName";
  c39_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c39_u,
    "ModelName", "ModelName", 0)), &c39_thisId, c39_y->ModelName);
  c39_thisId.fIdentifier = "ModelName_SL_Info";
  c39_y->ModelName_SL_Info = c39_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c39_u, "ModelName_SL_Info", "ModelName_SL_Info", 0)),
    &c39_thisId);
  c39_thisId.fIdentifier = "Q";
  c39_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c39_u, "Q",
    "Q", 0)), &c39_thisId, c39_y->Q);
  c39_thisId.fIdentifier = "Q_SL_Info";
  c39_y->Q_SL_Info = c39_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c39_u, "Q_SL_Info", "Q_SL_Info", 0)), &c39_thisId);
  c39_thisId.fIdentifier = "Qd";
  c39_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c39_u, "Qd",
    "Qd", 0)), &c39_thisId, c39_y->Qd);
  c39_thisId.fIdentifier = "Qd_SL_Info";
  c39_y->Qd_SL_Info = c39_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c39_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c39_thisId);
  c39_thisId.fIdentifier = "Tau";
  c39_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c39_u, "Tau",
    "Tau", 0)), &c39_thisId, c39_y->Tau);
  c39_thisId.fIdentifier = "Tau_SL_Info";
  c39_y->Tau_SL_Info = c39_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c39_u, "Tau_SL_Info", "Tau_SL_Info", 0)), &c39_thisId);
  sf_mex_destroy(&c39_u);
}

static void c39_c_emlrt_marshallIn(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c39_u, const emlrtMsgIdentifier *c39_parentId,
  uint8_T c39_y[8])
{
  int32_T c39_i;
  uint8_T c39_uv[8];
  sf_mex_import(c39_parentId, sf_mex_dup(c39_u), c39_uv, 1, 3, 0U, 1, 0U, 1, 8);
  for (c39_i = 0; c39_i < 8; c39_i++) {
    c39_y[c39_i] = c39_uv[c39_i];
  }

  sf_mex_destroy(&c39_u);
}

static c39_SL_Bus_ROSVariableLengthArrayInfo c39_d_emlrt_marshallIn
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_u, const emlrtMsgIdentifier *c39_parentId)
{
  static const char_T *c39_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c39_SL_Bus_ROSVariableLengthArrayInfo c39_y;
  emlrtMsgIdentifier c39_thisId;
  c39_thisId.fParent = c39_parentId;
  c39_thisId.bParentIsCell = false;
  sf_mex_check_struct(c39_parentId, c39_u, 2, c39_fieldNames, 0U, NULL);
  c39_thisId.fIdentifier = "CurrentLength";
  c39_y.CurrentLength = c39_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c39_u, "CurrentLength", "CurrentLength", 0)), &c39_thisId);
  c39_thisId.fIdentifier = "ReceivedLength";
  c39_y.ReceivedLength = c39_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c39_u, "ReceivedLength", "ReceivedLength", 0)), &c39_thisId);
  sf_mex_destroy(&c39_u);
  return c39_y;
}

static uint32_T c39_e_emlrt_marshallIn
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_u, const emlrtMsgIdentifier *c39_parentId)
{
  uint32_T c39_b_u;
  uint32_T c39_y;
  sf_mex_import(c39_parentId, sf_mex_dup(c39_u), &c39_b_u, 1, 7, 0U, 0, 0U, 0);
  c39_y = c39_b_u;
  sf_mex_destroy(&c39_u);
  return c39_y;
}

static void c39_f_emlrt_marshallIn(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c39_u, const emlrtMsgIdentifier *c39_parentId,
  real_T c39_y[7])
{
  real_T c39_dv[7];
  int32_T c39_i;
  sf_mex_import(c39_parentId, sf_mex_dup(c39_u), c39_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c39_i = 0; c39_i < 7; c39_i++) {
    c39_y[c39_i] = c39_dv[c39_i];
  }

  sf_mex_destroy(&c39_u);
}

static uint8_T c39_g_emlrt_marshallIn
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_b_is_active_c39_sliding_controller_rbdl, const char_T *c39_identifier)
{
  emlrtMsgIdentifier c39_thisId;
  uint8_T c39_y;
  c39_thisId.fIdentifier = const_cast<const char_T *>(c39_identifier);
  c39_thisId.fParent = NULL;
  c39_thisId.bParentIsCell = false;
  c39_y = c39_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c39_b_is_active_c39_sliding_controller_rbdl), &c39_thisId);
  sf_mex_destroy(&c39_b_is_active_c39_sliding_controller_rbdl);
  return c39_y;
}

static uint8_T c39_h_emlrt_marshallIn
  (SFc39_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c39_u, const emlrtMsgIdentifier *c39_parentId)
{
  uint8_T c39_b_u;
  uint8_T c39_y;
  sf_mex_import(c39_parentId, sf_mex_dup(c39_u), &c39_b_u, 1, 3, 0U, 0, 0U, 0);
  c39_y = c39_b_u;
  sf_mex_destroy(&c39_u);
  return c39_y;
}

static void c39_eML_blk_kernel(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c39_b_blankMsg)
{
  static uint8_T c39_uv[5] = { 104U, 117U, 109U, 97U, 110U };

  hipMemcpy(chartInstance->c39_gpu_blankMsg, c39_b_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  c39_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c39_gpu_blankMsg);
  hipMemcpy(*chartInstance->c39_gpu_uv, c39_uv, 5UL, hipMemcpyHostToDevice);
  c39_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c39_gpu_uv, chartInstance->c39_gpu_blankMsg);
  c39_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c39_gpu_blankMsg);
  hipMemcpy(c39_b_blankMsg, chartInstance->c39_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(32, 1) void
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel1
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c39_b_blankMsg,
   c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg)
{
  uint64_T c39_threadId;
  int32_T c39_i;
  c39_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c39_i = (int32_T)c39_threadId;
  if (c39_i < 8) {
    c39_b_msg->ModelName[c39_i] = c39_b_blankMsg->ModelName[c39_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel2
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c39_b_blankMsg,
   c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg)
{
  uint64_T c39_threadId;
  int32_T c39_i1;
  c39_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c39_i1 = (int32_T)c39_threadId;
  if (c39_i1 < 7) {
    c39_b_msg->Q[c39_i1] = c39_b_blankMsg->Q[c39_i1];
    c39_b_msg->Qd[c39_i1] = c39_b_blankMsg->Qd[c39_i1];
    c39_b_msg->Tau[c39_i1] = c39_b_blankMsg->Tau[c39_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel3
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg,
   c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_r)
{
  uint64_T c39_threadId;
  int32_T c39_i2;
  c39_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c39_i2 = (int32_T)c39_threadId;
  if (c39_i2 < 8) {
    c39_r->ModelName[c39_i2] = c39_b_msg->ModelName[c39_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel4
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg,
   c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_r)
{
  uint64_T c39_threadId;
  int32_T c39_i3;
  c39_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c39_i3 = (int32_T)c39_threadId;
  if (c39_i3 < 7) {
    c39_r->Q[c39_i3] = c39_b_msg->Q[c39_i3];
    c39_r->Qd[c39_i3] = c39_b_msg->Qd[c39_i3];
    c39_r->Tau[c39_i3] = c39_b_msg->Tau[c39_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel5(const
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_r,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg)
{
  uint64_T c39_threadId;
  int32_T c39_i4;
  c39_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c39_i4 = (int32_T)c39_threadId;
  if (c39_i4 < 8) {
    c39_b_msg->ModelName[c39_i4] = c39_r->ModelName[c39_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c39_sf_gateway_c39_sliding_controller_rbdl_kernel6(const
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_r,
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c39_b_msg)
{
  uint64_T c39_threadId;
  int32_T c39_i5;
  c39_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c39_i5 = (int32_T)c39_threadId;
  if (c39_i5 < 7) {
    c39_b_msg->Q[c39_i5] = c39_r->Q[c39_i5];
    c39_b_msg->Qd[c39_i5] = c39_r->Qd[c39_i5];
    c39_b_msg->Tau[c39_i5] = c39_r->Tau[c39_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c39_eML_blk_kernel_kernel7
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c39_b_blankMsg)
{
  uint64_T c39_threadId;
  int32_T c39_tmpIdx;
  c39_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c39_tmpIdx = (int32_T)c39_threadId;
  if (c39_tmpIdx < 1) {
    c39_b_blankMsg->ModelName_SL_Info.CurrentLength = 5U;
  }
}

static __global__ __launch_bounds__(32, 1) void c39_eML_blk_kernel_kernel8(const
  uint8_T c39_uv[5],
  c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c39_b_blankMsg)
{
  uint64_T c39_threadId;
  int32_T c39_i;
  c39_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c39_i = (int32_T)c39_threadId;
  if (c39_i < 5) {
    c39_b_blankMsg->ModelName[c39_i] = c39_uv[c39_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c39_eML_blk_kernel_kernel9
  (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c39_b_blankMsg)
{
  uint64_T c39_threadId;
  int32_T c39_tmpIdx;
  c39_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c39_tmpIdx = (int32_T)c39_threadId;
  if (c39_tmpIdx < 1) {
    c39_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
    c39_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
    c39_b_blankMsg->Tau_SL_Info.CurrentLength = 7U;
  }
}

static void init_dsm_address_info(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc39_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c39_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c39_blankMsg =
    (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c39_msg =
    (c39_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c39_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1446757649U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1328543342U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3523006734U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2043636117U);
}

mxArray *sf_c39_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c39_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c39_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c39_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvyqK+yH8zOL4xOSSzLLU+GRjy/jinMyUzLz0+OT8vJKi/Jyc1KL4oqSUHFj8AwEAPjcei"
    "Q=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c39_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sCgaqjKANzQMOTrp5zbdh0C";
}

static void sf_opaque_initialize_c39_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c39_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c39_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c39_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c39_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c39_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c39_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc39_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c39_sliding_controller_rbdl
      ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc39_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c39_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c39_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc39_sliding_controller_rbdl
    ((SFc39_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c39_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c39_sliding_controller_rbdl
      ((SFc39_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc39_sliding_controller_rbdl
      ((SFc39_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c39_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpmU3SIA0MIICyaFAcwnaY4CiQHppbVMSKtSK3VJOi1yE1XIkbrTcZfZHtvwOeYU",
    "8Q54hyEMEOQV5ggBF750lKVmmSCqOECMJSoCml/xmdma++Vl5G52uh9cNvHduet4VfF7Fu+Fl11",
    "f5emPhzt5ved/n66coJGx8SBSJtVd7CRLDn6Alt4ZJ0RFDWQpjYggKBEVsIpWp0qZZbDkT47YV1",
    "OnTf0WMRkEkLQ/3UJaEB4JPUVtizSHqaTIF1LQBQhMpaUdRm5PR3GJljv0I6FjbuM4FDSawiTNL",
    "dy03LOHQOgHaEdoQtFif2RYYYsA3J5VuOk91MAPKOOGMiFJvI6IDSDDABo6SEP8eWINOFWE0Isr",
    "sQUQmoPfZONUpBRR1Mo0fBkwQIxUjvBVz3wku23bI0Z6uDIHXBARt21NAxolkwlTzH7TR05YgAw",
    "5NGNhRtbYAnlhH/kMGx6Aq4zb05QQUGcGBqNw0DUjrJGVrniXLMMNieEjULkX+NISV2YuZowOCP",
    "EEPJapgkDrZ0T3FJhjeSm027rjMXFUyNs7I1qtgqbbWBOpYmGtrU+ETznUlrCeTfZgAT7U2iSH1",
    "sExrOU5rFvYkBtild3U1WMGQ+BzmSxGyUromBUDadx5gYzmPpFYbGfuYvM39/eXPy7COMKCGhEJ",
    "ZF1CEacCYpeGt1hYy7bhHIFplUvPKwFmGrEJ5emhF81iqMcakpomcueAYrQTGeoRcYiUcaSyaOp",
    "jjchWOEhpB6BoM49DFskFsSUy0a227WHcTZqZN0FSxpIRVi1WHbajlEmqawJEYC3ks2krGQd7js",
    "/ACYLYRJZgY7WH/UtM27l5GhZtn97yzeXb9PebZTK74/GFBz0aJHm/hWdz3WqN+3wb+tzGbvwty",
    "Xxf22SrIOdw23o/+ffPqJX92a+v265vTF2//WWf/542Lzf8b+frbWaOdF85kKV8d9rcFu7ZK9N9",
    "a0L+dr7U/Ik8e/7774PSP7kFPJT+dDsLonp/xs8LeRsHe2fs7ruNjlqX5qWgnzA8mbk1sNq6d/v",
    "sL9l5ZEY9r+fvsevfrevLf7BR5LIvX5rl4bbrKPZe3H77/nZ315LP9/15h/90C33fTud4nrltAn",
    "/74c19zFmKp96kURknOQfXVIOSF+vzQfL+onHfJcp+Lnf/LfXze32d+Xbacd8ly6/p30Tn+qeHr",
    "5oFXwG9/wn6se7762PhX3sXOQd/l61/mP3X8iPGw5NSbf8Yj7LDs6xeQp/8BAwm2wQ==",
    ""
  };

  static char newstr [1269] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c39_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c39_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3638621530U));
  ssSetChecksum1(S,(396258500U));
  ssSetChecksum2(S,(316282884U));
  ssSetChecksum3(S,(4091724409U));
}

static void mdlRTW_c39_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c39_sliding_controller_rbdl(SimStruct *S)
{
  SFc39_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc39_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc39_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc39_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc39_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c39_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c39_JITStateAnimation,
    chartInstance->c39_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c39_sliding_controller_rbdl(chartInstance);
}

void c39_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c39_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c39_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c39_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c39_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
