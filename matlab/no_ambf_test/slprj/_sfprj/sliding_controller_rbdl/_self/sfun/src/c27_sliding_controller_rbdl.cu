#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c27_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c27_do_animation_call_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_st);
static void sf_gateway_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c27_emlrt_marshallIn(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c27_b_msg, const char_T *c27_identifier,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_y);
static void c27_b_emlrt_marshallIn(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c27_u, const emlrtMsgIdentifier *c27_parentId,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_y);
static void c27_c_emlrt_marshallIn(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c27_u, const emlrtMsgIdentifier *c27_parentId,
  uint8_T c27_y[8]);
static c27_SL_Bus_ROSVariableLengthArrayInfo c27_d_emlrt_marshallIn
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_u, const emlrtMsgIdentifier *c27_parentId);
static uint32_T c27_e_emlrt_marshallIn
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_u, const emlrtMsgIdentifier *c27_parentId);
static void c27_f_emlrt_marshallIn(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c27_u, const emlrtMsgIdentifier *c27_parentId,
  real_T c27_y[7]);
static uint8_T c27_g_emlrt_marshallIn
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_b_is_active_c27_sliding_controller_rbdl, const char_T *c27_identifier);
static uint8_T c27_h_emlrt_marshallIn
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_u, const emlrtMsgIdentifier *c27_parentId);
static void c27_eML_blk_kernel(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c27_b_blankMsg);
static __global__ void c27_sf_gateway_c27_sliding_controller_rbdl_kernel1
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c27_b_blankMsg,
   c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg);
static __global__ void c27_sf_gateway_c27_sliding_controller_rbdl_kernel2
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c27_b_blankMsg,
   c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg);
static __global__ void c27_sf_gateway_c27_sliding_controller_rbdl_kernel3
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg,
   c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_r);
static __global__ void c27_sf_gateway_c27_sliding_controller_rbdl_kernel4
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg,
   c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_r);
static __global__ void c27_sf_gateway_c27_sliding_controller_rbdl_kernel5(const
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_r,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg);
static __global__ void c27_sf_gateway_c27_sliding_controller_rbdl_kernel6(const
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_r,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg);
static __global__ void c27_eML_blk_kernel_kernel7
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c27_b_blankMsg);
static __global__ void c27_eML_blk_kernel_kernel8(const uint8_T c27_uv[5],
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c27_b_blankMsg);
static __global__ void c27_eML_blk_kernel_kernel9
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c27_b_blankMsg);
static void init_dsm_address_info(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c27_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c27_is_active_c27_sliding_controller_rbdl = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c27_gpu_uv, 5UL);
  hipMalloc(&chartInstance->c27_gpu_blankMsg, 208UL);
}

static void initialize_params_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c27_do_animation_call_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  static const char_T *c27_sv[8] = { "ModelName", "ModelName_SL_Info", "Q",
    "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  static const char_T *c27_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c27_b_y = NULL;
  const mxArray *c27_c_y = NULL;
  const mxArray *c27_d_y = NULL;
  const mxArray *c27_e_y = NULL;
  const mxArray *c27_f_y = NULL;
  const mxArray *c27_g_y = NULL;
  const mxArray *c27_h_y = NULL;
  const mxArray *c27_i_y = NULL;
  const mxArray *c27_j_y = NULL;
  const mxArray *c27_k_y = NULL;
  const mxArray *c27_l_y = NULL;
  const mxArray *c27_m_y = NULL;
  const mxArray *c27_n_y = NULL;
  const mxArray *c27_o_y = NULL;
  const mxArray *c27_p_y = NULL;
  const mxArray *c27_q_y = NULL;
  const mxArray *c27_r_y = NULL;
  const mxArray *c27_s_y = NULL;
  const mxArray *c27_st;
  const mxArray *c27_y = NULL;
  real_T c27_d_u[7];
  int32_T c27_i;
  int32_T c27_i1;
  int32_T c27_i2;
  int32_T c27_i3;
  uint32_T c27_b_u;
  uint32_T c27_c_u;
  uint32_T c27_e_u;
  uint32_T c27_f_u;
  uint32_T c27_g_u;
  uint32_T c27_h_u;
  uint32_T c27_i_u;
  uint32_T c27_j_u;
  uint32_T c27_u_CurrentLength;
  uint32_T c27_u_ReceivedLength;
  uint8_T c27_u[8];
  c27_st = NULL;
  c27_y = NULL;
  sf_mex_assign(&c27_y, sf_mex_createcellmatrix(2, 1), false);
  c27_b_y = NULL;
  sf_mex_assign(&c27_b_y, sf_mex_createstruct("structure", 8, c27_sv, 2, 1, 1),
                false);
  for (c27_i = 0; c27_i < 8; c27_i++) {
    c27_u[c27_i] = ((uint8_T *)&((char_T *)chartInstance->c27_msg)[0])[c27_i];
  }

  c27_c_y = NULL;
  sf_mex_assign(&c27_c_y, sf_mex_create("y", c27_u, 3, 0U, 1U, 0U, 1, 8), false);
  sf_mex_setfieldbynum(c27_b_y, 0, "ModelName", c27_c_y, 0);
  c27_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [8])[0];
  c27_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [8])[4];
  c27_d_y = NULL;
  sf_mex_assign(&c27_d_y, sf_mex_createstruct("structure", 2, c27_sv1, 2, 1, 1),
                false);
  c27_b_u = c27_u_CurrentLength;
  c27_e_y = NULL;
  sf_mex_assign(&c27_e_y, sf_mex_create("y", &c27_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c27_d_y, 0, "CurrentLength", c27_e_y, 0);
  c27_c_u = c27_u_ReceivedLength;
  c27_f_y = NULL;
  sf_mex_assign(&c27_f_y, sf_mex_create("y", &c27_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c27_d_y, 0, "ReceivedLength", c27_f_y, 1);
  sf_mex_setfieldbynum(c27_b_y, 0, "ModelName_SL_Info", c27_d_y, 1);
  for (c27_i1 = 0; c27_i1 < 7; c27_i1++) {
    c27_d_u[c27_i1] = ((real_T *)&((char_T *)chartInstance->c27_msg)[16])[c27_i1];
  }

  c27_g_y = NULL;
  sf_mex_assign(&c27_g_y, sf_mex_create("y", c27_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c27_b_y, 0, "Q", c27_g_y, 2);
  c27_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [72])[0];
  c27_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [72])[4];
  c27_h_y = NULL;
  sf_mex_assign(&c27_h_y, sf_mex_createstruct("structure", 2, c27_sv1, 2, 1, 1),
                false);
  c27_e_u = c27_u_CurrentLength;
  c27_i_y = NULL;
  sf_mex_assign(&c27_i_y, sf_mex_create("y", &c27_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c27_h_y, 0, "CurrentLength", c27_i_y, 0);
  c27_f_u = c27_u_ReceivedLength;
  c27_j_y = NULL;
  sf_mex_assign(&c27_j_y, sf_mex_create("y", &c27_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c27_h_y, 0, "ReceivedLength", c27_j_y, 1);
  sf_mex_setfieldbynum(c27_b_y, 0, "Q_SL_Info", c27_h_y, 3);
  for (c27_i2 = 0; c27_i2 < 7; c27_i2++) {
    c27_d_u[c27_i2] = ((real_T *)&((char_T *)chartInstance->c27_msg)[80])[c27_i2];
  }

  c27_k_y = NULL;
  sf_mex_assign(&c27_k_y, sf_mex_create("y", c27_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c27_b_y, 0, "Qd", c27_k_y, 4);
  c27_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [136])[0];
  c27_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [136])[4];
  c27_l_y = NULL;
  sf_mex_assign(&c27_l_y, sf_mex_createstruct("structure", 2, c27_sv1, 2, 1, 1),
                false);
  c27_g_u = c27_u_CurrentLength;
  c27_m_y = NULL;
  sf_mex_assign(&c27_m_y, sf_mex_create("y", &c27_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c27_l_y, 0, "CurrentLength", c27_m_y, 0);
  c27_h_u = c27_u_ReceivedLength;
  c27_n_y = NULL;
  sf_mex_assign(&c27_n_y, sf_mex_create("y", &c27_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c27_l_y, 0, "ReceivedLength", c27_n_y, 1);
  sf_mex_setfieldbynum(c27_b_y, 0, "Qd_SL_Info", c27_l_y, 5);
  for (c27_i3 = 0; c27_i3 < 7; c27_i3++) {
    c27_d_u[c27_i3] = ((real_T *)&((char_T *)chartInstance->c27_msg)[144])
      [c27_i3];
  }

  c27_o_y = NULL;
  sf_mex_assign(&c27_o_y, sf_mex_create("y", c27_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c27_b_y, 0, "Tau", c27_o_y, 6);
  c27_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [200])[0];
  c27_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [200])[4];
  c27_p_y = NULL;
  sf_mex_assign(&c27_p_y, sf_mex_createstruct("structure", 2, c27_sv1, 2, 1, 1),
                false);
  c27_i_u = c27_u_CurrentLength;
  c27_q_y = NULL;
  sf_mex_assign(&c27_q_y, sf_mex_create("y", &c27_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c27_p_y, 0, "CurrentLength", c27_q_y, 0);
  c27_j_u = c27_u_ReceivedLength;
  c27_r_y = NULL;
  sf_mex_assign(&c27_r_y, sf_mex_create("y", &c27_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c27_p_y, 0, "ReceivedLength", c27_r_y, 1);
  sf_mex_setfieldbynum(c27_b_y, 0, "Tau_SL_Info", c27_p_y, 7);
  sf_mex_setcell(c27_y, 0, c27_b_y);
  c27_s_y = NULL;
  sf_mex_assign(&c27_s_y, sf_mex_create("y",
    &chartInstance->c27_is_active_c27_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c27_y, 1, c27_s_y);
  sf_mex_assign(&c27_st, c27_y, false);
  return c27_st;
}

static void set_sim_state_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_st)
{
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 c27_r;
  const mxArray *c27_u;
  int32_T c27_i;
  int32_T c27_i1;
  c27_u = sf_mex_dup(c27_st);
  c27_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c27_u, 0)),
                       "msg", &c27_r);
  for (c27_i = 0; c27_i < 8; c27_i++) {
    ((uint8_T *)&((char_T *)chartInstance->c27_msg)[0])[c27_i] =
      c27_r.ModelName[c27_i];
  }

  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[8])[0] = c27_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[8])[4] = c27_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[72])[0] = c27_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[72])[4] = c27_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[136])[0] = c27_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[136])[4] = c27_r.Qd_SL_Info.ReceivedLength;
  for (c27_i1 = 0; c27_i1 < 7; c27_i1++) {
    ((real_T *)&((char_T *)chartInstance->c27_msg)[16])[c27_i1] = c27_r.Q[c27_i1];
    ((real_T *)&((char_T *)chartInstance->c27_msg)[80])[c27_i1] =
      c27_r.Qd[c27_i1];
    ((real_T *)&((char_T *)chartInstance->c27_msg)[144])[c27_i1] =
      c27_r.Tau[c27_i1];
  }

  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[200])[0] = c27_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[200])[4] = c27_r.Tau_SL_Info.ReceivedLength;
  chartInstance->c27_is_active_c27_sliding_controller_rbdl =
    c27_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c27_u, 1)),
    "is_active_c27_sliding_controller_rbdl");
  sf_mex_destroy(&c27_u);
  sf_mex_destroy(&c27_st);
}

static void sf_gateway_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 c27_r;
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
    *c27_b_gpu_blankMsg;
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_gpu_msg;
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_gpu_r;
  hipMalloc(&c27_gpu_r, 208UL);
  hipMalloc(&c27_gpu_msg, 208UL);
  hipMalloc(&c27_b_gpu_blankMsg, 208UL);
  chartInstance->c27_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(c27_b_gpu_blankMsg, chartInstance->c27_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c27_gpu_msg, chartInstance->c27_msg, 208UL, hipMemcpyHostToDevice);
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c27_b_gpu_blankMsg, c27_gpu_msg);
  hipMemcpy(chartInstance->c27_msg, c27_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c27_blankMsg, c27_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[8])[0] = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_blankMsg)[8])[0];
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[8])[4] = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_blankMsg)[8])[4];
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[72])[0] = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_blankMsg)[72])[0];
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[72])[4] = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_blankMsg)[72])[4];
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[136])[0] = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_blankMsg)[136])[0];
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[136])[4] = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_blankMsg)[136])[4];
  hipMemcpy(c27_b_gpu_blankMsg, chartInstance->c27_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c27_gpu_msg, chartInstance->c27_msg, 208UL, hipMemcpyHostToDevice);
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel2<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c27_b_gpu_blankMsg, c27_gpu_msg);
  hipMemcpy(chartInstance->c27_msg, c27_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c27_blankMsg, c27_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_blankMsg)[200])[4];
  hipMemcpy(c27_gpu_msg, chartInstance->c27_msg, 208UL, hipMemcpyHostToDevice);
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel3<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c27_gpu_msg, c27_gpu_r);
  hipMemcpy(chartInstance->c27_msg, c27_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c27_r, c27_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c27_r.ModelName_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [8])[0];
  c27_r.ModelName_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [8])[4];
  c27_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [72])[0];
  c27_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [72])[4];
  c27_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [136])[0];
  c27_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [136])[4];
  hipMemcpy(c27_gpu_r, &c27_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c27_gpu_msg, chartInstance->c27_msg, 208UL, hipMemcpyHostToDevice);
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel4<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c27_gpu_msg, c27_gpu_r);
  hipMemcpy(chartInstance->c27_msg, c27_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c27_r, c27_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c27_r.Tau_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [200])[0];
  c27_r.Tau_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c27_msg)
    [200])[4];
  c27_eML_blk_kernel(chartInstance, &c27_r);
  hipMemcpy(c27_gpu_r, &c27_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c27_gpu_msg, chartInstance->c27_msg, 208UL, hipMemcpyHostToDevice);
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel5<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c27_gpu_r, c27_gpu_msg);
  hipMemcpy(chartInstance->c27_msg, c27_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[8])[0] = c27_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[8])[4] = c27_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[72])[0] = c27_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[72])[4] = c27_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[136])[0] = c27_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[136])[4] = c27_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c27_gpu_msg, chartInstance->c27_msg, 208UL, hipMemcpyHostToDevice);
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel6<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c27_gpu_r, c27_gpu_msg);
  hipMemcpy(chartInstance->c27_msg, c27_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[200])[0] = c27_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c27_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c27_msg)[200])[4] = c27_r.Tau_SL_Info.ReceivedLength;
  c27_do_animation_call_c27_sliding_controller_rbdl(chartInstance);
  hipFree(c27_b_gpu_blankMsg);
  hipFree(c27_gpu_msg);
  hipFree(c27_gpu_r);
}

static void mdl_start_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c27_errCode;
  hipFree(*chartInstance->c27_gpu_uv);
  hipFree(chartInstance->c27_gpu_blankMsg);
  c27_errCode = hipGetLastError();
  if (c27_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c27_errCode, (char_T *)hipGetErrorName
                       (c27_errCode), (char_T *)hipGetErrorString(c27_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c27_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc27_sliding_controller_rbdl
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c27_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c27_nameCaptureInfo = NULL;
  c27_nameCaptureInfo = NULL;
  sf_mex_assign(&c27_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c27_nameCaptureInfo;
}

static void c27_emlrt_marshallIn(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c27_b_msg, const char_T *c27_identifier,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_y)
{
  emlrtMsgIdentifier c27_thisId;
  c27_thisId.fIdentifier = const_cast<const char_T *>(c27_identifier);
  c27_thisId.fParent = NULL;
  c27_thisId.bParentIsCell = false;
  c27_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c27_b_msg), &c27_thisId,
    c27_y);
  sf_mex_destroy(&c27_b_msg);
}

static void c27_b_emlrt_marshallIn(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c27_u, const emlrtMsgIdentifier *c27_parentId,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_y)
{
  static const char_T *c27_fieldNames[8] = { "ModelName", "ModelName_SL_Info",
    "Q", "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  emlrtMsgIdentifier c27_thisId;
  c27_thisId.fParent = c27_parentId;
  c27_thisId.bParentIsCell = false;
  sf_mex_check_struct(c27_parentId, c27_u, 8, c27_fieldNames, 0U, NULL);
  c27_thisId.fIdentifier = "ModelName";
  c27_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c27_u,
    "ModelName", "ModelName", 0)), &c27_thisId, c27_y->ModelName);
  c27_thisId.fIdentifier = "ModelName_SL_Info";
  c27_y->ModelName_SL_Info = c27_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c27_u, "ModelName_SL_Info", "ModelName_SL_Info", 0)),
    &c27_thisId);
  c27_thisId.fIdentifier = "Q";
  c27_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c27_u, "Q",
    "Q", 0)), &c27_thisId, c27_y->Q);
  c27_thisId.fIdentifier = "Q_SL_Info";
  c27_y->Q_SL_Info = c27_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c27_u, "Q_SL_Info", "Q_SL_Info", 0)), &c27_thisId);
  c27_thisId.fIdentifier = "Qd";
  c27_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c27_u, "Qd",
    "Qd", 0)), &c27_thisId, c27_y->Qd);
  c27_thisId.fIdentifier = "Qd_SL_Info";
  c27_y->Qd_SL_Info = c27_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c27_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c27_thisId);
  c27_thisId.fIdentifier = "Tau";
  c27_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c27_u, "Tau",
    "Tau", 0)), &c27_thisId, c27_y->Tau);
  c27_thisId.fIdentifier = "Tau_SL_Info";
  c27_y->Tau_SL_Info = c27_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c27_u, "Tau_SL_Info", "Tau_SL_Info", 0)), &c27_thisId);
  sf_mex_destroy(&c27_u);
}

static void c27_c_emlrt_marshallIn(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c27_u, const emlrtMsgIdentifier *c27_parentId,
  uint8_T c27_y[8])
{
  int32_T c27_i;
  uint8_T c27_uv[8];
  sf_mex_import(c27_parentId, sf_mex_dup(c27_u), c27_uv, 1, 3, 0U, 1, 0U, 1, 8);
  for (c27_i = 0; c27_i < 8; c27_i++) {
    c27_y[c27_i] = c27_uv[c27_i];
  }

  sf_mex_destroy(&c27_u);
}

static c27_SL_Bus_ROSVariableLengthArrayInfo c27_d_emlrt_marshallIn
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_u, const emlrtMsgIdentifier *c27_parentId)
{
  static const char_T *c27_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c27_SL_Bus_ROSVariableLengthArrayInfo c27_y;
  emlrtMsgIdentifier c27_thisId;
  c27_thisId.fParent = c27_parentId;
  c27_thisId.bParentIsCell = false;
  sf_mex_check_struct(c27_parentId, c27_u, 2, c27_fieldNames, 0U, NULL);
  c27_thisId.fIdentifier = "CurrentLength";
  c27_y.CurrentLength = c27_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c27_u, "CurrentLength", "CurrentLength", 0)), &c27_thisId);
  c27_thisId.fIdentifier = "ReceivedLength";
  c27_y.ReceivedLength = c27_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c27_u, "ReceivedLength", "ReceivedLength", 0)), &c27_thisId);
  sf_mex_destroy(&c27_u);
  return c27_y;
}

static uint32_T c27_e_emlrt_marshallIn
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_u, const emlrtMsgIdentifier *c27_parentId)
{
  uint32_T c27_b_u;
  uint32_T c27_y;
  sf_mex_import(c27_parentId, sf_mex_dup(c27_u), &c27_b_u, 1, 7, 0U, 0, 0U, 0);
  c27_y = c27_b_u;
  sf_mex_destroy(&c27_u);
  return c27_y;
}

static void c27_f_emlrt_marshallIn(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c27_u, const emlrtMsgIdentifier *c27_parentId,
  real_T c27_y[7])
{
  real_T c27_dv[7];
  int32_T c27_i;
  sf_mex_import(c27_parentId, sf_mex_dup(c27_u), c27_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c27_i = 0; c27_i < 7; c27_i++) {
    c27_y[c27_i] = c27_dv[c27_i];
  }

  sf_mex_destroy(&c27_u);
}

static uint8_T c27_g_emlrt_marshallIn
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_b_is_active_c27_sliding_controller_rbdl, const char_T *c27_identifier)
{
  emlrtMsgIdentifier c27_thisId;
  uint8_T c27_y;
  c27_thisId.fIdentifier = const_cast<const char_T *>(c27_identifier);
  c27_thisId.fParent = NULL;
  c27_thisId.bParentIsCell = false;
  c27_y = c27_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c27_b_is_active_c27_sliding_controller_rbdl), &c27_thisId);
  sf_mex_destroy(&c27_b_is_active_c27_sliding_controller_rbdl);
  return c27_y;
}

static uint8_T c27_h_emlrt_marshallIn
  (SFc27_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c27_u, const emlrtMsgIdentifier *c27_parentId)
{
  uint8_T c27_b_u;
  uint8_T c27_y;
  sf_mex_import(c27_parentId, sf_mex_dup(c27_u), &c27_b_u, 1, 3, 0U, 0, 0U, 0);
  c27_y = c27_b_u;
  sf_mex_destroy(&c27_u);
  return c27_y;
}

static void c27_eML_blk_kernel(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c27_b_blankMsg)
{
  static uint8_T c27_uv[5] = { 104U, 117U, 109U, 97U, 110U };

  hipMemcpy(chartInstance->c27_gpu_blankMsg, c27_b_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  c27_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c27_gpu_blankMsg);
  hipMemcpy(*chartInstance->c27_gpu_uv, c27_uv, 5UL, hipMemcpyHostToDevice);
  c27_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c27_gpu_uv, chartInstance->c27_gpu_blankMsg);
  c27_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c27_gpu_blankMsg);
  hipMemcpy(c27_b_blankMsg, chartInstance->c27_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(32, 1) void
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel1
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c27_b_blankMsg,
   c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg)
{
  uint64_T c27_threadId;
  int32_T c27_i;
  c27_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c27_i = (int32_T)c27_threadId;
  if (c27_i < 8) {
    c27_b_msg->ModelName[c27_i] = c27_b_blankMsg->ModelName[c27_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel2
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c27_b_blankMsg,
   c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg)
{
  uint64_T c27_threadId;
  int32_T c27_i1;
  c27_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c27_i1 = (int32_T)c27_threadId;
  if (c27_i1 < 7) {
    c27_b_msg->Q[c27_i1] = c27_b_blankMsg->Q[c27_i1];
    c27_b_msg->Qd[c27_i1] = c27_b_blankMsg->Qd[c27_i1];
    c27_b_msg->Tau[c27_i1] = c27_b_blankMsg->Tau[c27_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel3
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg,
   c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_r)
{
  uint64_T c27_threadId;
  int32_T c27_i2;
  c27_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c27_i2 = (int32_T)c27_threadId;
  if (c27_i2 < 8) {
    c27_r->ModelName[c27_i2] = c27_b_msg->ModelName[c27_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel4
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg,
   c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_r)
{
  uint64_T c27_threadId;
  int32_T c27_i3;
  c27_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c27_i3 = (int32_T)c27_threadId;
  if (c27_i3 < 7) {
    c27_r->Q[c27_i3] = c27_b_msg->Q[c27_i3];
    c27_r->Qd[c27_i3] = c27_b_msg->Qd[c27_i3];
    c27_r->Tau[c27_i3] = c27_b_msg->Tau[c27_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel5(const
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_r,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg)
{
  uint64_T c27_threadId;
  int32_T c27_i4;
  c27_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c27_i4 = (int32_T)c27_threadId;
  if (c27_i4 < 8) {
    c27_b_msg->ModelName[c27_i4] = c27_r->ModelName[c27_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c27_sf_gateway_c27_sliding_controller_rbdl_kernel6(const
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_r,
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c27_b_msg)
{
  uint64_T c27_threadId;
  int32_T c27_i5;
  c27_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c27_i5 = (int32_T)c27_threadId;
  if (c27_i5 < 7) {
    c27_b_msg->Q[c27_i5] = c27_r->Q[c27_i5];
    c27_b_msg->Qd[c27_i5] = c27_r->Qd[c27_i5];
    c27_b_msg->Tau[c27_i5] = c27_r->Tau[c27_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c27_eML_blk_kernel_kernel7
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c27_b_blankMsg)
{
  uint64_T c27_threadId;
  int32_T c27_tmpIdx;
  c27_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c27_tmpIdx = (int32_T)c27_threadId;
  if (c27_tmpIdx < 1) {
    c27_b_blankMsg->ModelName_SL_Info.CurrentLength = 5U;
  }
}

static __global__ __launch_bounds__(32, 1) void c27_eML_blk_kernel_kernel8(const
  uint8_T c27_uv[5],
  c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c27_b_blankMsg)
{
  uint64_T c27_threadId;
  int32_T c27_i;
  c27_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c27_i = (int32_T)c27_threadId;
  if (c27_i < 5) {
    c27_b_blankMsg->ModelName[c27_i] = c27_uv[c27_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c27_eML_blk_kernel_kernel9
  (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c27_b_blankMsg)
{
  uint64_T c27_threadId;
  int32_T c27_tmpIdx;
  c27_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c27_tmpIdx = (int32_T)c27_threadId;
  if (c27_tmpIdx < 1) {
    c27_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
    c27_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
    c27_b_blankMsg->Tau_SL_Info.CurrentLength = 7U;
  }
}

static void init_dsm_address_info(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc27_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c27_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c27_blankMsg =
    (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c27_msg =
    (c27_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c27_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1446757649U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1328543342U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3523006734U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2043636117U);
}

mxArray *sf_c27_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c27_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c27_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c27_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvyqK+yH8zOL4xOSSzLLU+GQj8/jinMyUzLz0+OT8vJKi/Jyc1KL4oqSUHFj8AwEAPcoeh"
    "g=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c27_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sCgaqjKANzQMOTrp5zbdh0C";
}

static void sf_opaque_initialize_c27_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c27_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c27_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c27_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c27_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c27_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c27_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc27_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c27_sliding_controller_rbdl
      ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc27_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c27_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c27_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc27_sliding_controller_rbdl
    ((SFc27_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c27_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c27_sliding_controller_rbdl
      ((SFc27_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc27_sliding_controller_rbdl
      ((SFc27_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c27_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpmU3SIA0MIICyaFAcwnaY1CgSE+tbUpChVqxW8ppkYuwWo7EjZa7zP7Ilt8hr5B",
    "nyDMEeYggpyBPEKDovbMkJcsUScURYiRBCdD0kt/Mzsw3Pytvo9P18LqB985Nz7uCz6t4N7zs+i",
    "pfbyzc2fst7/t8/RSFhI0PiSKx9movQWL4E7Tk1jApOmIoS2FMDEGBoIhNpDJV2jSLLWdi3LaCO",
    "n36r4jRKIik5eEeypLwQPApakusOUQ9TaaAmjZAaCIl7ShqczKaW6zMsR8BHWsb17mgwQQ2cWbp",
    "ruWGJRxaJ0A7QhuCFusz2wJDDPjmpNJN56kOZkAZJ5wRUeptRHQACQbYwFES4t8Da9CpIoxGRJk",
    "9iMgE9D4bpzqlgKJOpvHDgAlipGKEt2LuO8Fl2w452tOVIfCagKBtewrIOJFMmGr+gzZ62hJkwK",
    "EJAzuq1hbAE+vIf8jgGFRl3Ia+nIAiIzgQlZumAWmdpGzNs2QZZlgMD4napcifhrAyezFzdECQJ",
    "+ihRBUMUic7uqfYBMNbqc3GHZeZq0rGxhnZehUs1daaQB0Lc21tKnzCua6E9WSyDxPgqdYmMaQe",
    "lmktx2nNwp7EALv0rq4GKxgSn8N8KUJWStekAEj7zgNsLOeR1GojYx+Tt7m/v/x5GdYRBtSQUCj",
    "rAoowDRizNLzV2kKmHfcIRKtMal4ZOMuQVShPD61oHks1xpjUNJEzFxyjlcBYj5BLrIQjjUVTB3",
    "NcrsJRQiMIXYNhHLpYNogtiYl2rW0X627CzLQJmiqWlLBqseqwDbVcQk0TOBJjIY9FW8k4yHt8F",
    "l4AzDaiBBOjPexfatrG3cuocPPsnnc2z66/xzybyRWfPyzo2SjR4y08i/tea9Tv28D/Nmbzd0Hu",
    "68I+WwU5h9vG+9G/b1695M9ubd1+fXP64u0/6+z/vHGx+X8jX387a7Tzwpks5avD/rZg11aJ/ls",
    "L+rfztfZH5Mnj33cfnP7RPeip5KfTQRjd8zN+VtjbKNg7e3/HdXzMsjQ/Fe2E+cHErYnNxrXT//",
    "OCvVdWxONa/j673v26nvw3O0Uey+K1eS5em65yz+Xth+9/Z2c9+Wz/v1fYf7fA9910rveJ6xbQp",
    "z/e72vOQiz1PpXCKMk5qL4ahLxQnx+a7xeV8y5Z7nOx83+5j8/7+8yvy5bzLlluXf8uOsc/NXzd",
    "PPAK+O1P2I91z1cfG//Ku9g56Lt8/cv8p44fMR6WnHrzz3iEHZZ9/QLy9D/r3ba+",
    ""
  };

  static char newstr [1265] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c27_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c27_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3638621530U));
  ssSetChecksum1(S,(396258500U));
  ssSetChecksum2(S,(316282884U));
  ssSetChecksum3(S,(4091724409U));
}

static void mdlRTW_c27_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c27_sliding_controller_rbdl(SimStruct *S)
{
  SFc27_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc27_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc27_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc27_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc27_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c27_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c27_JITStateAnimation,
    chartInstance->c27_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c27_sliding_controller_rbdl(chartInstance);
}

void c27_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c27_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c27_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c27_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c27_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
