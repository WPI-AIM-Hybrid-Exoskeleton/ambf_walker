/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c19_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c19_do_animation_call_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_st);
static void sf_gateway_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c19_eML_blk_kernel(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2]);
static void c19_emlrt_marshallIn(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2]);
static void c19_b_emlrt_marshallIn(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2]);
static uint8_T c19_c_emlrt_marshallIn
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_b_is_active_c19_sliding_controller_rbdl, const char_T *c19_identifier);
static uint8_T c19_d_emlrt_marshallIn
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_u, const emlrtMsgIdentifier *c19_parentId);
static void init_dsm_address_info(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c19_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c19_is_active_c19_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c19_do_animation_call_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c19_b_y = NULL;
  const mxArray *c19_c_y = NULL;
  const mxArray *c19_d_y = NULL;
  const mxArray *c19_e_y = NULL;
  const mxArray *c19_f_y = NULL;
  const mxArray *c19_st;
  const mxArray *c19_y = NULL;
  c19_st = NULL;
  c19_y = NULL;
  sf_mex_assign(&c19_y, sf_mex_createcellmatrix(5, 1), false);
  c19_b_y = NULL;
  sf_mex_assign(&c19_b_y, sf_mex_create("y", *chartInstance->c19_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 0, c19_b_y);
  c19_c_y = NULL;
  sf_mex_assign(&c19_c_y, sf_mex_create("y", *chartInstance->c19_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 1, c19_c_y);
  c19_d_y = NULL;
  sf_mex_assign(&c19_d_y, sf_mex_create("y", *chartInstance->c19_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 2, c19_d_y);
  c19_e_y = NULL;
  sf_mex_assign(&c19_e_y, sf_mex_create("y", *chartInstance->c19_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c19_y, 3, c19_e_y);
  c19_f_y = NULL;
  sf_mex_assign(&c19_f_y, sf_mex_create("y",
    &chartInstance->c19_is_active_c19_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c19_y, 4, c19_f_y);
  sf_mex_assign(&c19_st, c19_y, false);
  return c19_st;
}

static void set_sim_state_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_st)
{
  const mxArray *c19_u;
  c19_u = sf_mex_dup(c19_st);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 0)),
                       "ankle", *chartInstance->c19_ankle);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 1)),
                       "danke", *chartInstance->c19_danke);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 2)),
                       "dknee", *chartInstance->c19_dknee);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 3)),
                       "knee", *chartInstance->c19_knee);
  chartInstance->c19_is_active_c19_sliding_controller_rbdl =
    c19_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 4)),
    "is_active_c19_sliding_controller_rbdl");
  sf_mex_destroy(&c19_u);
  sf_mex_destroy(&c19_st);
}

static void sf_gateway_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c19_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c19_eML_blk_kernel(chartInstance, *chartInstance->c19_q,
                     *chartInstance->c19_qd, *chartInstance->c19_knee,
                     *chartInstance->c19_dknee, *chartInstance->c19_ankle,
                     *chartInstance->c19_danke);
  c19_do_animation_call_c19_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c19_errCode;
  c19_errCode = hipGetLastError();
  if (c19_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c19_errCode, (char_T *)hipGetErrorName
                       (c19_errCode), (char_T *)hipGetErrorString(c19_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c19_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c19_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c19_nameCaptureInfo = NULL;
  const char_T *c19_data[4] = {
    "789cc593cb4ac34014864fa516375537fa188e2d08dd57c50b2d428b1b9138494e9a317391c9d4b66eba74e963e933b93297a697608858ac07c2c99f3f33df99"
    "1f062a979d0a00ec464f670a705183a4ea6983fd59df82d5cafb95829ed5365457d6c5fe55c47b9b6947498363930a4905ce57ba4a3049a5e94f9e1034868a3f",
    "a39b381ee3d867027bcba21b2b71be64cd456cc5ef6d1f9da03714a0fd7031215f16491e713d149cb75a9247bef279e4ffcb78e35ff2b2fd0f4b7899ef293da2"
    "dab5022699a08639a1c571b0feb9336ead708ed471d5d0e6b8e07dacc9a385bc55ffeeec9ef84a2091d4f85432e403c55d8f6a9b38d4446958a39084da2154d8",
    "9e35a23c404da28038b5895456f2d56068c8f7011e8919a72cc7bd1f9eabe83ed56127e9ad8313ba49deebe77b7393bcacfe8bb7a9fb787d3c618d17f61804cd"
    "56bbdbb06de7d635a78b396e4a3865734081feebfdbf0083b96cd1", "" };

  c19_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c19_data[0], 1632U, &c19_nameCaptureInfo);
  return c19_nameCaptureInfo;
}

static void c19_eML_blk_kernel(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2])
{
  real_T c19_a;
  real_T c19_b_a;
  c19_b_knee[0] = -0.324 * muDoubleScalarSin(c19_b_q[0]);
  c19_b_knee[1] = -0.324 * muDoubleScalarCos(c19_b_q[0]);
  c19_a = -0.324 * c19_b_qd[0];
  c19_b_dknee[0] = c19_a * muDoubleScalarCos(c19_b_q[0]);
  c19_b_dknee[1] = c19_a * -muDoubleScalarSin(c19_b_q[0]);
  c19_b_a = -0.39 * (c19_b_qd[0] + c19_b_qd[1]);
  c19_b_ankle[0] = c19_b_knee[0] + -0.39 * muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_ankle[1] = c19_b_knee[1] + -0.39 * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[0] = c19_b_dknee[0] + c19_b_a * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[1] = c19_b_dknee[1] + c19_b_a * -muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
}

static void c19_emlrt_marshallIn(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2])
{
  emlrtMsgIdentifier c19_thisId;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c19_b_ankle), &c19_thisId,
    c19_y);
  sf_mex_destroy(&c19_b_ankle);
}

static void c19_b_emlrt_marshallIn(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2])
{
  real_T c19_dv[2];
  int32_T c19_i;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), c19_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c19_i = 0; c19_i < 2; c19_i++) {
    c19_y[c19_i] = c19_dv[c19_i];
  }

  sf_mex_destroy(&c19_u);
}

static uint8_T c19_c_emlrt_marshallIn
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_b_is_active_c19_sliding_controller_rbdl, const char_T *c19_identifier)
{
  emlrtMsgIdentifier c19_thisId;
  uint8_T c19_y;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_y = c19_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c19_b_is_active_c19_sliding_controller_rbdl), &c19_thisId);
  sf_mex_destroy(&c19_b_is_active_c19_sliding_controller_rbdl);
  return c19_y;
}

static uint8_T c19_d_emlrt_marshallIn
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_u, const emlrtMsgIdentifier *c19_parentId)
{
  uint8_T c19_b_u;
  uint8_T c19_y;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), &c19_b_u, 1, 3, 0U, 0, 0U, 0);
  c19_y = c19_b_u;
  sf_mex_destroy(&c19_u);
  return c19_y;
}

static void init_dsm_address_info(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c19_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c19_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c19_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c19_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c19_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c19_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c19_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c19_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c19_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c19_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6LFA4ifWRyfmFySWZYan"
    "2xoGV+ck5mSmZcen5yfV1KUn5OTWhRflJSSgxRPAKKjLpE="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c19_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c19_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c19_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c19_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c19_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c19_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c19_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c19_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc19_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c19_sliding_controller_rbdl
      ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc19_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c19_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c19_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c19_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c19_sliding_controller_rbdl
      ((SFc19_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc19_sliding_controller_rbdl
      ((SFc19_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c19_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFALwEaBAjSQwu4KAo0KNDapiREqF07pZwGuQir5UhcaLnL7A9t5x2",
    "KXPseRS95kfTcJ+gbtLMkLcsUSUV2k7pFaVDykt98O9/M7OzKaw32Pbxu4/3zHc+7gd838W57+b",
    "VRjFsLd/684z0sxj+hkbDxIVEk1l7jJUgMP4CW3BomxUBMZCWMiQkoEBSxiVSmjk2z2HImZn0rq",
    "OPTP0aMRkEkLQ930ZaEB4KfIltizSHydJkCavoAoYmUtNOoz8l07rEyx34EdKZt3CRBgwls4tzS",
    "+5YblnDonQAdCG0IeqzPfQsMMeCbk1qZTqkOzoAyTjgjolJtRHQACQbYwFES4ueBNSiqDKMRUWY",
    "XIpKC3mOzjFMKKHMyjS/GTBAjFSO8F3PfGS77dsjRn30ZAm8ICPq2q4DMEsmEqc9/0EelPUHGHL",
    "owttN6tgBeWJf8pwyOQdXGbeLLFBSZwoGonTQLSO8ky9a8SpZhhsXwlKgdivnTENZWL1aODgjmC",
    "YZoUQeDTORADxVLMby1bDYeuMpctWRsnCdbr4JlbL0UmrIwZ+tT4RPOdS1sKJM9SIFnrF1iSDMs",
    "Z63Gac3CocQAu/KuXw1WMEx8AfOlCFllutISIOs732NjuYikVhsZ+1i83b295dfLsIEwoCaEQlU",
    "XUIRpwJhl4a1nC5l2uUcgemUy96rAeYWsQnl6YkX3WKoZxqShiZxLcBmtBcZ6irnElXCkcdE0wV",
    "wuV+EooRGErsEwDvu4bBBbERPtWtsOrruUmdMuaKpYUpFVi6sO21DPFdRpAkdiJuSx6CsZB0WPz",
    "8MLgNVGlGBiuov9S532cfaqVLj9bMs7388+eIv97Myu/P3JAk+rgsdb+C7Pe6t9Ed8uzdvGv1Yr",
    "t9tesPuwNE+nZOdwm3i//OX5r9+9ef3qs4e/f/pq+PWfq+ZvLc3fyv53djc31tv/bxfju2eNdr5",
    "w0qV6ddjHC351Kvg/WuDfLMb6SfSlSJ8/jnaekGE82Pri5aOtdOxnfM86zf5ulPw9e37PdXyssq",
    "w+FR2ExcHEjYnNt2vH/9WCvzdWxONW8Ty//vj2avZ3t8v2nRX6NosxETMOpfr953Tcv7SOEIVcH",
    "x33Lq9jhoeGa6Pjzna5P1Tp6FzQ0fGchL/H/+U4rmefz/9shf8PSnl4kJ0XR8TtQjCinz8aac5C",
    "3EJGVAqjJOegRmoc8lKeLttH17Xz3rPdv8XP/+3efd7f5lzUvqRd54rnsPdld1V9654Prxu+aT/",
    "wSvjNa6zjquf2d43/zVvvfP1xMf5m/hPajxgPK35NFa/xp9Gk6u1/oE7/AggMxTg=",
    ""
  };

  static char newstr [1341] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c19_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c19_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042620282U));
  ssSetChecksum1(S,(2411386699U));
  ssSetChecksum2(S,(753018669U));
  ssSetChecksum3(S,(4282078351U));
}

static void mdlRTW_c19_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c19_sliding_controller_rbdl(SimStruct *S)
{
  SFc19_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc19_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc19_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc19_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc19_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c19_JITStateAnimation,
    chartInstance->c19_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c19_sliding_controller_rbdl(chartInstance);
}

void c19_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c19_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c19_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c19_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c19_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
