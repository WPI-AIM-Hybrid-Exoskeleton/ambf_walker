/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c19_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c19_do_animation_call_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_st);
static void sf_gateway_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c19_eML_blk_kernel(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2]);
static void c19_emlrt_marshallIn(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2]);
static void c19_b_emlrt_marshallIn(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2]);
static uint8_T c19_c_emlrt_marshallIn
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_b_is_active_c19_sliding_controller_rbdl, const char_T *c19_identifier);
static uint8_T c19_d_emlrt_marshallIn
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_u, const emlrtMsgIdentifier *c19_parentId);
static void init_dsm_address_info(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c19_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c19_is_active_c19_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c19_do_animation_call_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c19_b_y = NULL;
  const mxArray *c19_c_y = NULL;
  const mxArray *c19_d_y = NULL;
  const mxArray *c19_e_y = NULL;
  const mxArray *c19_f_y = NULL;
  const mxArray *c19_st;
  const mxArray *c19_y = NULL;
  c19_st = NULL;
  c19_y = NULL;
  sf_mex_assign(&c19_y, sf_mex_createcellmatrix(5, 1), false);
  c19_b_y = NULL;
  sf_mex_assign(&c19_b_y, sf_mex_create("y", *chartInstance->c19_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 0, c19_b_y);
  c19_c_y = NULL;
  sf_mex_assign(&c19_c_y, sf_mex_create("y", *chartInstance->c19_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 1, c19_c_y);
  c19_d_y = NULL;
  sf_mex_assign(&c19_d_y, sf_mex_create("y", *chartInstance->c19_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 2, c19_d_y);
  c19_e_y = NULL;
  sf_mex_assign(&c19_e_y, sf_mex_create("y", *chartInstance->c19_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c19_y, 3, c19_e_y);
  c19_f_y = NULL;
  sf_mex_assign(&c19_f_y, sf_mex_create("y",
    &chartInstance->c19_is_active_c19_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c19_y, 4, c19_f_y);
  sf_mex_assign(&c19_st, c19_y, false);
  return c19_st;
}

static void set_sim_state_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_st)
{
  const mxArray *c19_u;
  c19_u = sf_mex_dup(c19_st);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 0)),
                       "ankle", *chartInstance->c19_ankle);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 1)),
                       "danke", *chartInstance->c19_danke);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 2)),
                       "dknee", *chartInstance->c19_dknee);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 3)),
                       "knee", *chartInstance->c19_knee);
  chartInstance->c19_is_active_c19_sliding_controller_rbdl =
    c19_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 4)),
    "is_active_c19_sliding_controller_rbdl");
  sf_mex_destroy(&c19_u);
  sf_mex_destroy(&c19_st);
}

static void sf_gateway_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c19_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c19_eML_blk_kernel(chartInstance, *chartInstance->c19_q,
                     *chartInstance->c19_qd, *chartInstance->c19_knee,
                     *chartInstance->c19_dknee, *chartInstance->c19_ankle,
                     *chartInstance->c19_danke);
  c19_do_animation_call_c19_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c19_errCode;
  c19_errCode = hipGetLastError();
  if (c19_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c19_errCode, (char_T *)hipGetErrorName
                       (c19_errCode), (char_T *)hipGetErrorString(c19_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c19_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc19_sliding_controller_rbdl
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c19_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c19_nameCaptureInfo = NULL;
  const char_T *c19_data[4] = {
    "789ce593cb4ac34014864fa51637f5b2e94b088e75e5be2a8a5884163722f1243969c6cc452653dbbaf10df4297c171fcb246d7a09868862371e08277ffecc7c"
    "677e18a85d5cd500603b795a2f00e70dc8aa396db03beb1bb05a45bf56d2f3da84facabad4df4b786f33ed6965696ca742a1a4f94a5f4bae50d9fee491c050ac",
    "c513f9991370417d2ea9b72cbaa992674bd65ca456fade09c98b7a4309268c17138a6591e591d67dc979eb157914ab9847f1bf9c37fe212fdfbf55c1cbfd409b"
    "111adf89b8e2122df76247d0e0f7e7ceb98dd239a68eaf87aea005efe3973c2ce5adfab7a7772cd49298421ba2e224065af8011a97796893349c51cc62e33194",
    "6ee08c504464581290409729ed645f2dc5967d1de0819c71aa72dcf9e6b9caee5313b6b2fefeba8febe4e5f55f78ebba8f978713de7ee60f517474dce9b65dd7"
    "bbf1edc9628eeb0a4ed51c50a2ff7aff4fb0e86ac3", "" };

  c19_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c19_data[0], 1632U, &c19_nameCaptureInfo);
  return c19_nameCaptureInfo;
}

static void c19_eML_blk_kernel(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2])
{
  real_T c19_a;
  real_T c19_b_a;
  c19_b_knee[0] = -0.324 * muDoubleScalarSin(c19_b_q[0]);
  c19_b_knee[1] = -0.324 * muDoubleScalarCos(c19_b_q[0]);
  c19_a = -0.324 * c19_b_qd[0];
  c19_b_dknee[0] = c19_a * muDoubleScalarCos(c19_b_q[0]);
  c19_b_dknee[1] = c19_a * -muDoubleScalarSin(c19_b_q[0]);
  c19_b_a = -0.39 * (c19_b_qd[0] + c19_b_qd[1]);
  c19_b_ankle[0] = c19_b_knee[0] + -0.39 * muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_ankle[1] = c19_b_knee[1] + -0.39 * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[0] = c19_b_dknee[0] + c19_b_a * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[1] = c19_b_dknee[1] + c19_b_a * -muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
}

static void c19_emlrt_marshallIn(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2])
{
  emlrtMsgIdentifier c19_thisId;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c19_b_ankle), &c19_thisId,
    c19_y);
  sf_mex_destroy(&c19_b_ankle);
}

static void c19_b_emlrt_marshallIn(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2])
{
  real_T c19_dv[2];
  int32_T c19_i;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), c19_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c19_i = 0; c19_i < 2; c19_i++) {
    c19_y[c19_i] = c19_dv[c19_i];
  }

  sf_mex_destroy(&c19_u);
}

static uint8_T c19_c_emlrt_marshallIn
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_b_is_active_c19_sliding_controller_rbdl, const char_T *c19_identifier)
{
  emlrtMsgIdentifier c19_thisId;
  uint8_T c19_y;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_y = c19_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c19_b_is_active_c19_sliding_controller_rbdl), &c19_thisId);
  sf_mex_destroy(&c19_b_is_active_c19_sliding_controller_rbdl);
  return c19_y;
}

static uint8_T c19_d_emlrt_marshallIn
  (SFc19_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c19_u, const emlrtMsgIdentifier *c19_parentId)
{
  uint8_T c19_b_u;
  uint8_T c19_y;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), &c19_b_u, 1, 3, 0U, 0, 0U, 0);
  c19_y = c19_b_u;
  sf_mex_destroy(&c19_u);
  return c19_y;
}

static void init_dsm_address_info(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc19_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c19_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c19_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c19_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c19_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c19_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c19_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c19_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c19_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c19_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c19_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6LFA4ifWRyfmFySWZYan"
    "2xoGV+ck5mSmZcen5yfV1KUn5OTWhRflJSSgxRPAKKjLpE="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c19_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c19_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c19_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c19_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c19_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c19_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c19_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c19_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc19_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c19_sliding_controller_rbdl
      ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc19_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c19_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c19_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc19_sliding_controller_rbdl
    ((SFc19_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c19_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c19_sliding_controller_rbdl
      ((SFc19_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc19_sliding_controller_rbdl
      ((SFc19_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c19_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA2cAEDvQRIUCBIi15cFAWaS2ObkhABdu2Uchr0IqyWI3Gh5S67P/TPO/T",
    "eSx8kr5AnyBP02j5CZ0lalimSiuwmdYvSoOQlv/l2vpnZ2ZXXGhx4eN3H+9ctz7uD33fxbnv5tV",
    "GMWwt3/rzjPSnGv6CRsPERUSTWXuMlSAw/gJbcGibFQExkJYyJCSgQFLGJVKaOTbPYciZmfSuo4",
    "9M/RoxGQSQtD/fQloSHgp8hW2LNEfJ0mQJq+gChiZS006jPyXTusTInfgR0pm3cJEGDCWzi3NIH",
    "lhuWcOidAh0IbQh6rC99Cwwx4JvTWplOqQ4ugDJOOCOiUm1EdAAJBtjAcRLi56E1KKoMoxFRZg8",
    "ikoLeZ7OMUwooczKNL8ZMECMVI7wXc98ZLvt2xNGfAxkCbwgI+rangMwSyYSpz3/QR6U9QcYcuj",
    "C203q2AH62LvkvGZyAqo3bxJcpKDKFQ1E7aRaQ3mmWrXmVLMMMi+ElUbsU86chrK1erBwdEMwTD",
    "NGiDgaZyIEeKpZieGvZbDxwlblqydg4T7ZeBcvYeik0ZWHO1qfCJ5zrWthQJvuQAs9Yu8SQZljO",
    "Wo3TmoVDiQF25V2/GqxgmPgC5ksRssp0pSVA1ne+x8ZyFUmtNjL2sXi7+/vLr5dhA2FATQiFqi6",
    "gCNOAMcvCW88WMu1yj0D0ymTuVYHzClmF8vTEiu6JVDOMSUMTuZTgMloLjPUUc4kr4VjjommCuV",
    "yuwlFCIwhdg2EcDnDZILYiJtq1tl1cdykzZ13QVLGkIqsWVx22oZ4rqLMEjsVMyBPRVzIOih6fh",
    "xcAq40owcR0D/uXOuvj7FWpcPvZtne5n330DvvZhV35+/MFnlYFj7fwXZ73Xvsqvl2at41/rVZu",
    "t7Ng93Fpnk7JzuE28T5//SX88ey36Pfoi0+2zuHNqvlbS/O3sv+d3d2N9fb/+8X4wUWjnS+cdKl",
    "eHfb5gl+dCv5PF/g3i7F+EX0j0p+eR7svyDAebH99/nQ7HfsZ36tOs78bJX8vnj9yHR+rLKtPRQ",
    "dhcTBxY2Lz7drxf7vg750V8bhXPM+vP5/dzP7BTtm+s0LfZjEmYsahVL//nI7Prq0jRCG3R8ej6",
    "+uY4aHh1ujY2in3hyodnSs6Op6T8Pf4vxzH9ezz+V+t8P9xKQ+Ps/PiiLhdCEb0q6cjzVmIW8iI",
    "SmGU5BzUSI1DXsrTdfvounbeB7b7t/j5v937z/u7nIva17Tr3PAc9qHsbqpv3fPhbcM37QdeCb9",
    "5i3Xc9Nz+vvFvvfXO1w+L8Xfzn9B+xHhY8WuqeI0/jSZVb/8DdfoXNsbETQ==",
    ""
  };

  static char newstr [1337] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c19_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c19_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1697364346U));
  ssSetChecksum1(S,(1755135983U));
  ssSetChecksum2(S,(287926502U));
  ssSetChecksum3(S,(3395648020U));
}

static void mdlRTW_c19_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c19_sliding_controller_rbdl(SimStruct *S)
{
  SFc19_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc19_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc19_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc19_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc19_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c19_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c19_JITStateAnimation,
    chartInstance->c19_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c19_sliding_controller_rbdl(chartInstance);
}

void c19_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c19_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c19_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c19_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c19_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
