/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c22_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c22_do_animation_call_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c22_st);
static void sf_gateway_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c22_eML_blk_kernel(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c22_b_u[6], real_T c22_b_y[7]);
static void c22_emlrt_marshallIn(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c22_b_y, const char_T *c22_identifier, real_T
  c22_c_y[7]);
static void c22_b_emlrt_marshallIn(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId,
  real_T c22_b_y[7]);
static uint8_T c22_c_emlrt_marshallIn
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c22_b_is_active_c22_sliding_controller_rbdl, const char_T *c22_identifier);
static uint8_T c22_d_emlrt_marshallIn
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c22_b_u, const emlrtMsgIdentifier *c22_parentId);
static void init_dsm_address_info(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c22_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c22_is_active_c22_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c22_do_animation_call_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c22_b_y = NULL;
  const mxArray *c22_c_y = NULL;
  const mxArray *c22_d_y = NULL;
  const mxArray *c22_st;
  c22_st = NULL;
  c22_b_y = NULL;
  sf_mex_assign(&c22_b_y, sf_mex_createcellmatrix(2, 1), false);
  c22_c_y = NULL;
  sf_mex_assign(&c22_c_y, sf_mex_create("y", *chartInstance->c22_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c22_b_y, 0, c22_c_y);
  c22_d_y = NULL;
  sf_mex_assign(&c22_d_y, sf_mex_create("y",
    &chartInstance->c22_is_active_c22_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c22_b_y, 1, c22_d_y);
  sf_mex_assign(&c22_st, c22_b_y, false);
  return c22_st;
}

static void set_sim_state_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c22_st)
{
  const mxArray *c22_b_u;
  c22_b_u = sf_mex_dup(c22_st);
  c22_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c22_b_u, 0)),
                       "y", *chartInstance->c22_y);
  chartInstance->c22_is_active_c22_sliding_controller_rbdl =
    c22_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c22_b_u, 1)),
    "is_active_c22_sliding_controller_rbdl");
  sf_mex_destroy(&c22_b_u);
  sf_mex_destroy(&c22_st);
}

static void sf_gateway_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c22_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c22_eML_blk_kernel(chartInstance, *chartInstance->c22_u, *chartInstance->c22_y);
  c22_do_animation_call_c22_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c22_errCode;
  c22_errCode = hipGetLastError();
  if (c22_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c22_errCode, (char_T *)hipGetErrorName
                       (c22_errCode), (char_T *)hipGetErrorString(c22_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c22_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc22_sliding_controller_rbdl
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c22_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c22_nameCaptureInfo = NULL;
  c22_nameCaptureInfo = NULL;
  sf_mex_assign(&c22_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c22_nameCaptureInfo;
}

static void c22_eML_blk_kernel(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c22_b_u[6], real_T c22_b_y[7])
{
  c22_b_y[0] = c22_b_u[3];
  c22_b_y[1] = 0.0;
  c22_b_y[2] = c22_b_u[0];
  c22_b_y[3] = c22_b_u[1];
  c22_b_y[4] = c22_b_u[4];
  c22_b_y[5] = c22_b_u[5];
  c22_b_y[6] = c22_b_u[2];
}

static void c22_emlrt_marshallIn(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c22_b_y, const char_T *c22_identifier, real_T
  c22_c_y[7])
{
  emlrtMsgIdentifier c22_thisId;
  c22_thisId.fIdentifier = const_cast<const char_T *>(c22_identifier);
  c22_thisId.fParent = NULL;
  c22_thisId.bParentIsCell = false;
  c22_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c22_b_y), &c22_thisId,
    c22_c_y);
  sf_mex_destroy(&c22_b_y);
}

static void c22_b_emlrt_marshallIn(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId,
  real_T c22_b_y[7])
{
  real_T c22_dv[7];
  int32_T c22_i;
  sf_mex_import(c22_parentId, sf_mex_dup(c22_b_u), c22_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c22_i = 0; c22_i < 7; c22_i++) {
    c22_b_y[c22_i] = c22_dv[c22_i];
  }

  sf_mex_destroy(&c22_b_u);
}

static uint8_T c22_c_emlrt_marshallIn
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c22_b_is_active_c22_sliding_controller_rbdl, const char_T *c22_identifier)
{
  emlrtMsgIdentifier c22_thisId;
  uint8_T c22_b_y;
  c22_thisId.fIdentifier = const_cast<const char_T *>(c22_identifier);
  c22_thisId.fParent = NULL;
  c22_thisId.bParentIsCell = false;
  c22_b_y = c22_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c22_b_is_active_c22_sliding_controller_rbdl), &c22_thisId);
  sf_mex_destroy(&c22_b_is_active_c22_sliding_controller_rbdl);
  return c22_b_y;
}

static uint8_T c22_d_emlrt_marshallIn
  (SFc22_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c22_b_u, const emlrtMsgIdentifier *c22_parentId)
{
  uint8_T c22_b_y;
  uint8_T c22_c_u;
  sf_mex_import(c22_parentId, sf_mex_dup(c22_b_u), &c22_c_u, 1, 3, 0U, 0, 0U, 0);
  c22_b_y = c22_c_u;
  sf_mex_destroy(&c22_b_u);
  return c22_b_y;
}

static void init_dsm_address_info(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc22_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c22_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c22_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c22_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c22_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2517670966U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4150444637U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1981160258U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2051201109U);
}

mxArray *sf_c22_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c22_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c22_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c22_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8clGRvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB1/B2v"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c22_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "svnY9loGNdJEcvDOTuc9pcC";
}

static void sf_opaque_initialize_c22_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c22_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c22_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c22_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c22_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c22_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c22_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc22_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c22_sliding_controller_rbdl
      ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc22_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c22_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c22_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc22_sliding_controller_rbdl
    ((SFc22_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c22_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c22_sliding_controller_rbdl
      ((SFc22_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc22_sliding_controller_rbdl
      ((SFc22_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c22_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV0uP00gQ9kQDAgnQCCFxQYIL2j0iLgsXGMZJIGhewhkep6inXYlbaXebfngm4sqVn8Av2Mv",
    "+nL1w5bziwpVq28lkHdthiBgBwpLHafur6qr66tHjrfV2PLyu4L151fPO4/MC3i0vv84V67W5O3",
    "+/7v1RrN+hkLDxPlEk1l7jJUgMz0BLbg2ToieGshLGxBAUCIrYRCpTp02z2HImxl0rqNOnX0SMR",
    "kEkLQ+3UJaEe4JPUFtizT7qaTMF1HQBQhMpaUdRl5PRzGJljvwI6FjbuMkFDSawiTNL71huWMKh",
    "cwy0J7QhaLE+sS0wxIBvjmvddJ7qYAqUccIZEZXeRkQHkGCADRwkIf7dswadKsNoRJTZgoikoLf",
    "ZONMpBZR1Mo0fDpkgRipGeCfmvhNctG2foz07MgTeEBC0bUsBGSeSCVPPf9BFTzuCHHJow6Ed1W",
    "sL4LV15D9ncASqNm5DX6agyAj2RO2mWUA6xxlbsyxZhBkWw3OiHlHkT0NYm72YOTogyBP0UaIOB",
    "pmTPd1XLMXw1mqzcc9l5rKSsXFOtl4Gy7R1UmhiYaatS4VPONe1sL5MtiEFnmltE0OaYbnWapzW",
    "LOxLDLBL7/pqsIIh8QXMlyJklXSlJUDWd3axsfwfSa02MvYxedvb24ufF2E9YUANCYWqLqAI04A",
    "xy8Jbry1k2nGPQLTKZOZVgfMMWYby9NCK9pFUY4xJQxM5ccExWguM9Qi5xEo40Fg0TTDH5TIcJT",
    "SC0DUYxmEHywaxFTHRrrU9wrpLmZm0QVPFkgpWLVYdtqGOS6hJAgdiLOSR6CoZB0WPz8MLgNlGl",
    "GBitIX9S026uHsVFW6e3fFO5tmlr5hnU7ny8885PWsVery5Z3nfi63mfVv4a206f+fkLpf2WS/J",
    "OdwG3h/f3/nUf/Oh9dfbv+NXnye7q+z/T+t08/9Ksb4xbbSzwkkX8tVhn8zZtV6h//qc/o1irVP",
    "x6j6Xj3fDpx2atvf6lt5PqJ/zs8TeVsne6ftbruNjlmX5qWgvLA4mbk1sPq6d/ntz9p5fEo+Lxf",
    "v8+u/havLXNss8ri+R38Bfk1Lefvv+tzZXk8/3f7nE/tslvm9nc31AXLeAAb17d6A5C7HUB1QKo",
    "yTnoAbqMOQlP781308r552x3M9i52+578/718yvs5bzzlhuVf9OO8d/NHzTPPBK+I0f2I9Vz1ff",
    "G/+vd7pz0M1i/WD2r44fMR5WnHqLz3iEHVZ9/QXy9AtTwrUz",
    ""
  };

  static char newstr [1249] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c22_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c22_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(4063272422U));
  ssSetChecksum1(S,(48069460U));
  ssSetChecksum2(S,(1839957047U));
  ssSetChecksum3(S,(1316616025U));
}

static void mdlRTW_c22_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c22_sliding_controller_rbdl(SimStruct *S)
{
  SFc22_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc22_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc22_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc22_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc22_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c22_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c22_JITStateAnimation,
    chartInstance->c22_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c22_sliding_controller_rbdl(chartInstance);
}

void c22_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c22_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c22_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c22_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c22_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
