/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c18_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c18_do_animation_call_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_st);
static void sf_gateway_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c18_eML_blk_kernel(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c18_b_u[7], real_T c18_b_y[7]);
static void c18_emlrt_marshallIn(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c18_b_y, const char_T *c18_identifier, real_T
  c18_c_y[7]);
static void c18_b_emlrt_marshallIn(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c18_b_u, const emlrtMsgIdentifier *c18_parentId,
  real_T c18_b_y[7]);
static uint8_T c18_c_emlrt_marshallIn
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_b_is_active_c18_sliding_controller_rbdl, const char_T *c18_identifier);
static uint8_T c18_d_emlrt_marshallIn
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_b_u, const emlrtMsgIdentifier *c18_parentId);
static void init_dsm_address_info(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c18_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c18_is_active_c18_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c18_do_animation_call_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c18_b_y = NULL;
  const mxArray *c18_c_y = NULL;
  const mxArray *c18_d_y = NULL;
  const mxArray *c18_st;
  c18_st = NULL;
  c18_b_y = NULL;
  sf_mex_assign(&c18_b_y, sf_mex_createcellmatrix(2, 1), false);
  c18_c_y = NULL;
  sf_mex_assign(&c18_c_y, sf_mex_create("y", *chartInstance->c18_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c18_b_y, 0, c18_c_y);
  c18_d_y = NULL;
  sf_mex_assign(&c18_d_y, sf_mex_create("y",
    &chartInstance->c18_is_active_c18_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c18_b_y, 1, c18_d_y);
  sf_mex_assign(&c18_st, c18_b_y, false);
  return c18_st;
}

static void set_sim_state_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_st)
{
  const mxArray *c18_b_u;
  c18_b_u = sf_mex_dup(c18_st);
  c18_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c18_b_u, 0)),
                       "y", *chartInstance->c18_y);
  chartInstance->c18_is_active_c18_sliding_controller_rbdl =
    c18_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c18_b_u, 1)),
    "is_active_c18_sliding_controller_rbdl");
  sf_mex_destroy(&c18_b_u);
  sf_mex_destroy(&c18_st);
}

static void sf_gateway_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c18_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c18_eML_blk_kernel(chartInstance, *chartInstance->c18_u, *chartInstance->c18_y);
  c18_do_animation_call_c18_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c18_errCode;
  c18_errCode = hipGetLastError();
  if (c18_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c18_errCode, (char_T *)hipGetErrorName
                       (c18_errCode), (char_T *)hipGetErrorString(c18_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c18_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c18_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c18_nameCaptureInfo = NULL;
  c18_nameCaptureInfo = NULL;
  sf_mex_assign(&c18_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c18_nameCaptureInfo;
}

static void c18_eML_blk_kernel(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c18_b_u[7], real_T c18_b_y[7])
{
  c18_b_y[0] = c18_b_u[3];
  c18_b_y[1] = c18_b_u[6];
  c18_b_y[2] = c18_b_u[0];
  c18_b_y[3] = c18_b_u[1];
  c18_b_y[4] = c18_b_u[4];
  c18_b_y[5] = c18_b_u[5];
  c18_b_y[6] = c18_b_u[2];
}

static void c18_emlrt_marshallIn(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c18_b_y, const char_T *c18_identifier, real_T
  c18_c_y[7])
{
  emlrtMsgIdentifier c18_thisId;
  c18_thisId.fIdentifier = const_cast<const char_T *>(c18_identifier);
  c18_thisId.fParent = NULL;
  c18_thisId.bParentIsCell = false;
  c18_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c18_b_y), &c18_thisId,
    c18_c_y);
  sf_mex_destroy(&c18_b_y);
}

static void c18_b_emlrt_marshallIn(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c18_b_u, const emlrtMsgIdentifier *c18_parentId,
  real_T c18_b_y[7])
{
  real_T c18_dv[7];
  int32_T c18_i;
  sf_mex_import(c18_parentId, sf_mex_dup(c18_b_u), c18_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c18_i = 0; c18_i < 7; c18_i++) {
    c18_b_y[c18_i] = c18_dv[c18_i];
  }

  sf_mex_destroy(&c18_b_u);
}

static uint8_T c18_c_emlrt_marshallIn
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_b_is_active_c18_sliding_controller_rbdl, const char_T *c18_identifier)
{
  emlrtMsgIdentifier c18_thisId;
  uint8_T c18_b_y;
  c18_thisId.fIdentifier = const_cast<const char_T *>(c18_identifier);
  c18_thisId.fParent = NULL;
  c18_thisId.bParentIsCell = false;
  c18_b_y = c18_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c18_b_is_active_c18_sliding_controller_rbdl), &c18_thisId);
  sf_mex_destroy(&c18_b_is_active_c18_sliding_controller_rbdl);
  return c18_b_y;
}

static uint8_T c18_d_emlrt_marshallIn
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_b_u, const emlrtMsgIdentifier *c18_parentId)
{
  uint8_T c18_b_y;
  uint8_T c18_c_u;
  sf_mex_import(c18_parentId, sf_mex_dup(c18_b_u), &c18_c_u, 1, 3, 0U, 0, 0U, 0);
  c18_b_y = c18_c_u;
  sf_mex_destroy(&c18_b_u);
  return c18_b_y;
}

static void init_dsm_address_info(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c18_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c18_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c18_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c18_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3007509169U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1926089682U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2239166282U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3770191902U);
}

mxArray *sf_c18_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c18_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c18_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c18_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cmGFvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB2rx20"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c18_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sdegTANPF1GH9p3UVy5QBRH";
}

static void sf_opaque_initialize_c18_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c18_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c18_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c18_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c18_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c18_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c18_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc18_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c18_sliding_controller_rbdl
      ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc18_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c18_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c18_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c18_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c18_sliding_controller_rbdl
      ((SFc18_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc18_sliding_controller_rbdl
      ((SFc18_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c18_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u1EgQdkYBLRJE0QppLwi4RHADhFZiL7vJ/JGREhLwJHBBo067Ztyadrfpn0nmzBHxFLw",
    "AF96Fp0Bc4IZEte2ZDB7bQxgRsau15Hja/qq6qr766XgrnV0PrzW8N3/3vIv4/A3vmpdeF7L1ys",
    "ydvl/1bmXr1ygkbLRPFIm0V3kJEsET0JJbw6ToiL4shDHRBwWCIjaWypRp0yyynIlh2wrq9OmnI",
    "aOhH0rLgzrKkmBP8DFqi63ZRz1NpoCaNkBgQiXtIGxzMpharMxxIwQ61DaqckGD8W3szNK7lhsW",
    "c2idAO0IbQharE9t8w0x0DAnpW46T7U/Acoo5oyIQm9Don2IMcAGDuIA/+5Zg07lYTQkytQhJCP",
    "QO2yY6JQC8jqZxg9HTBAjFSO8FfGGE5y3bZ+jPbsyAF4RELStroAMY8mEKeffb6OnLUGOODThyA",
    "7KtfnwwjryDxkcgyqNW78hR6DIAPZE6aZJQFonCVvTLJmHGRbBIVFbFPnTEJRmL2aO9gnyBF2UK",
    "INB4mRHdxUbYXhLtdmo4zJzUcnYKCVbL4Il2lojqGJhqq1NRYNwrkthXRnvwAh4orVJDKmGpVqL",
    "cVqzoCsxwC69y6vBCobEZ7CGFAErpGuUAyR95xE2lm+R1GojowYmb3NnZ/7zPKwjDKg+oVDUBRR",
    "hGjBmSXjLtQVMO+4RiFaZxLwicJohi1Ce7lvRPJZqiDGpaCKnLjhGS4GRHiCXWAkHGoumCua4XI",
    "SjhIYQuAbDOOxi2SC2ICbatbYtrLsRM+MmaKpYXMCqxarDNtRyCTWO4UAMhTwWbSUjP+vxaXgBM",
    "NuIEkwM6ti/1LiNuxdR4ebZXe90nl3+jnk2kcs/b8/oWSnQ48088/teqlXvW8NfK5P5OyN3JbfP",
    "ak7O4dbx/nL16btXd15+WntDnn/4vHF9mf3f1s42/9ey9bVJo50WzmguXx12e8au1QL9f8zoX8/",
    "WOoBBd+vRfvvew+2/4vsHh+M/H9efbKf8LLC3lrN38v6m6/iYZUl+KtoJsoOJWxObjmun/8GMvR",
    "cXxONS9j69Pv6znPzVzTyPqwvk1/HXOJe3P77/zc3l5NP9ny2wfyPH90Yy13vEdQvo0XsPepqzA",
    "Eu9R6UwSnIOqqeOAp7z80fz/axy3jnL/Vvs/F/u5/P+PfPrvOW8c5Zb1r+zzvFfDV81D7wcfv0X",
    "9mPZ89XPxr/3znYOupGt/57+q9MIGQ8KTr3ZZzzC9ou+/gfy9CufRrPr",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c18_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c18_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042383101U));
  ssSetChecksum1(S,(4119015307U));
  ssSetChecksum2(S,(1566679822U));
  ssSetChecksum3(S,(505804526U));
}

static void mdlRTW_c18_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c18_sliding_controller_rbdl(SimStruct *S)
{
  SFc18_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc18_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc18_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc18_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc18_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c18_JITStateAnimation,
    chartInstance->c18_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c18_sliding_controller_rbdl(chartInstance);
}

void c18_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c18_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c18_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c18_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c18_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
