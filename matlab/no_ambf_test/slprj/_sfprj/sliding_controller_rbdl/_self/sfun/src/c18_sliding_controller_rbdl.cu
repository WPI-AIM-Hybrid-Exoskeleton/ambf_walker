/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c18_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c18_do_animation_call_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_st);
static void sf_gateway_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c18_eML_blk_kernel(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c18_b_u[6], real_T c18_b_y[7]);
static void c18_emlrt_marshallIn(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c18_b_y, const char_T *c18_identifier, real_T
  c18_c_y[7]);
static void c18_b_emlrt_marshallIn(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c18_b_u, const emlrtMsgIdentifier *c18_parentId,
  real_T c18_b_y[7]);
static uint8_T c18_c_emlrt_marshallIn
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_b_is_active_c18_sliding_controller_rbdl, const char_T *c18_identifier);
static uint8_T c18_d_emlrt_marshallIn
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_b_u, const emlrtMsgIdentifier *c18_parentId);
static void init_dsm_address_info(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c18_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c18_is_active_c18_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c18_do_animation_call_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c18_b_y = NULL;
  const mxArray *c18_c_y = NULL;
  const mxArray *c18_d_y = NULL;
  const mxArray *c18_st;
  c18_st = NULL;
  c18_b_y = NULL;
  sf_mex_assign(&c18_b_y, sf_mex_createcellmatrix(2, 1), false);
  c18_c_y = NULL;
  sf_mex_assign(&c18_c_y, sf_mex_create("y", *chartInstance->c18_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c18_b_y, 0, c18_c_y);
  c18_d_y = NULL;
  sf_mex_assign(&c18_d_y, sf_mex_create("y",
    &chartInstance->c18_is_active_c18_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c18_b_y, 1, c18_d_y);
  sf_mex_assign(&c18_st, c18_b_y, false);
  return c18_st;
}

static void set_sim_state_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_st)
{
  const mxArray *c18_b_u;
  c18_b_u = sf_mex_dup(c18_st);
  c18_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c18_b_u, 0)),
                       "y", *chartInstance->c18_y);
  chartInstance->c18_is_active_c18_sliding_controller_rbdl =
    c18_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c18_b_u, 1)),
    "is_active_c18_sliding_controller_rbdl");
  sf_mex_destroy(&c18_b_u);
  sf_mex_destroy(&c18_st);
}

static void sf_gateway_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c18_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c18_eML_blk_kernel(chartInstance, *chartInstance->c18_u, *chartInstance->c18_y);
  c18_do_animation_call_c18_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c18_errCode;
  c18_errCode = hipGetLastError();
  if (c18_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c18_errCode, (char_T *)hipGetErrorName
                       (c18_errCode), (char_T *)hipGetErrorString(c18_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c18_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc18_sliding_controller_rbdl
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c18_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c18_nameCaptureInfo = NULL;
  c18_nameCaptureInfo = NULL;
  sf_mex_assign(&c18_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c18_nameCaptureInfo;
}

static void c18_eML_blk_kernel(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c18_b_u[6], real_T c18_b_y[7])
{
  c18_b_y[0] = c18_b_u[3];
  c18_b_y[1] = 0.0;
  c18_b_y[2] = c18_b_u[0];
  c18_b_y[3] = c18_b_u[1];
  c18_b_y[4] = c18_b_u[4];
  c18_b_y[5] = c18_b_u[5];
  c18_b_y[6] = c18_b_u[2];
}

static void c18_emlrt_marshallIn(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c18_b_y, const char_T *c18_identifier, real_T
  c18_c_y[7])
{
  emlrtMsgIdentifier c18_thisId;
  c18_thisId.fIdentifier = const_cast<const char_T *>(c18_identifier);
  c18_thisId.fParent = NULL;
  c18_thisId.bParentIsCell = false;
  c18_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c18_b_y), &c18_thisId,
    c18_c_y);
  sf_mex_destroy(&c18_b_y);
}

static void c18_b_emlrt_marshallIn(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c18_b_u, const emlrtMsgIdentifier *c18_parentId,
  real_T c18_b_y[7])
{
  real_T c18_dv[7];
  int32_T c18_i;
  sf_mex_import(c18_parentId, sf_mex_dup(c18_b_u), c18_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c18_i = 0; c18_i < 7; c18_i++) {
    c18_b_y[c18_i] = c18_dv[c18_i];
  }

  sf_mex_destroy(&c18_b_u);
}

static uint8_T c18_c_emlrt_marshallIn
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_b_is_active_c18_sliding_controller_rbdl, const char_T *c18_identifier)
{
  emlrtMsgIdentifier c18_thisId;
  uint8_T c18_b_y;
  c18_thisId.fIdentifier = const_cast<const char_T *>(c18_identifier);
  c18_thisId.fParent = NULL;
  c18_thisId.bParentIsCell = false;
  c18_b_y = c18_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c18_b_is_active_c18_sliding_controller_rbdl), &c18_thisId);
  sf_mex_destroy(&c18_b_is_active_c18_sliding_controller_rbdl);
  return c18_b_y;
}

static uint8_T c18_d_emlrt_marshallIn
  (SFc18_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c18_b_u, const emlrtMsgIdentifier *c18_parentId)
{
  uint8_T c18_b_y;
  uint8_T c18_c_u;
  sf_mex_import(c18_parentId, sf_mex_dup(c18_b_u), &c18_c_u, 1, 3, 0U, 0, 0U, 0);
  c18_b_y = c18_c_u;
  sf_mex_destroy(&c18_b_u);
  return c18_b_y;
}

static void init_dsm_address_info(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc18_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c18_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c18_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c18_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c18_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3669229500U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(490348266U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2982423110U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1200777508U);
}

mxArray *sf_c18_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c18_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c18_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c18_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cmGFvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB2rx20"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c18_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sStKXtzIzHLETxtLQhJoVxG";
}

static void sf_opaque_initialize_c18_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c18_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c18_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c18_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c18_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c18_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c18_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc18_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c18_sliding_controller_rbdl
      ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc18_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c18_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c18_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc18_sliding_controller_rbdl
    ((SFc18_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c18_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c18_sliding_controller_rbdl
      ((SFc18_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc18_sliding_controller_rbdl
      ((SFc18_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c18_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u00oUdqOCqASoQkhsELBBsITFlVhxS/MDgZQWnPYiNtF0fBKPMp4x85MmvAEL3gUJ8R4",
    "s2bHhAXgEzthOGhzboURUcIUl1xn7O2fOOd/5mXpr7R0Pr4t4b13yvLP4PId3zUuvM9l6be5O36",
    "97t7L1WxQSNtojikTaq7wEieA5aMmtYVK0RV8WwpjogwJBERtLZcq0aRZZzsSwZQV1+vR/IaOhH",
    "0rLg22UJcGu4BPUFluzh3oaTAE1LYDAhEraQdjiZDCzWJmjegh0qG1U5YIG49vYmaV3LDcs5tAc",
    "A20LbQharI9t8w0xUDfjUjedp9qfAmUUc0ZEobch0T7EGGAD+3GAf3etQafyMBoSZbYhJCPQHTZ",
    "MdEoBeZ1M44dDJoiRihHejHjdCS7atsfRnh0ZAK8ICNq2rYAMY8mEKeffb6GnTUEOOTTg0A7Ktf",
    "nwyjryDxgcgSqNW78uR6DIAHZF6aZJQJrjhK1ZlizCDIvggKgHFPnTEJRmL2aO9gnyBF2UKINB4",
    "mRbdxUbYXhLtdmo7TJzWcnYKCVbL4Ml2pojqGJhpq1FRZ1wrkthXRl3YAQ80doghlTDUq3FOK1Z",
    "0JUYYJfe5dVgBUPiM1hdioAV0jXKAZK+8xQby/dIarWRUR2Tt9HpLH5ehLWFAdUnFIq6gCJMA8Y",
    "sCW+5toBpxz0C0SqTmFcETjNkGcrTfSsaR1INMSYVTeTYBcdoKTDSA+QSK2FfY9FUwRyXy3CU0B",
    "AC12AYhx0sG8QWxES71vYA627EzKQBmioWF7BqseqwDTVdQk1i2BdDIY9ES8nIz3p8Gl4AzDaiB",
    "BODbexfatLC3YuocPPsjnc8z87/wDybyuWft+f0rBXo8eae+X03atX71vDX2nT+zsldyO2znpNz",
    "uE2833z2Dz6//Pjl2j/vxeWvH66ssv+72snm/8VsfXXaaGeFM1rIV4d9NGfXeoH+K3P6N7O19s2",
    "TF+Z1+/WjTrM7Np1n4WN5MH6Y8rPE3lrO3un7G67jY5Yl+aloO8gOJm5NbDqunf57c/aeXRKPje",
    "x9en39dzX5y1t5HteXyG/ir0kub39+/xtbq8mn+79YYv/NHN83k7neI65bQI/evdfTnAVY6j0qh",
    "VGSc1A9dRjwnJ8/m+8nlfNOWe5PsfOv3K/n/Ufm12nLeacst6p/J53jvxu+ah54Ofzmb+zHquer",
    "X43/5J3sHHQ9W9+f/atTDxkPCk692Wc8wvaLvv4P8vQbW3O1hA==",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c18_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c18_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1448337289U));
  ssSetChecksum1(S,(3838859999U));
  ssSetChecksum2(S,(1857238302U));
  ssSetChecksum3(S,(397799444U));
}

static void mdlRTW_c18_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c18_sliding_controller_rbdl(SimStruct *S)
{
  SFc18_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc18_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc18_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc18_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc18_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c18_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c18_JITStateAnimation,
    chartInstance->c18_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c18_sliding_controller_rbdl(chartInstance);
}

void c18_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c18_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c18_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c18_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c18_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
