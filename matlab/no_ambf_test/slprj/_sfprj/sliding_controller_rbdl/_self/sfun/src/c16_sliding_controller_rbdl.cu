#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c16_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c16_do_animation_call_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_st);
static void sf_gateway_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c16_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_b_msg, const char_T *c16_identifier,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_y);
static void c16_b_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_y);
static void c16_c_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  real_T c16_y[7]);
static c16_SL_Bus_ROSVariableLengthArrayInfo c16_d_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId);
static uint32_T c16_e_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId);
static void c16_f_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  uint8_T c16_y[4]);
static uint8_T c16_g_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_b_is_active_c16_sliding_controller_rbdl, const char_T *c16_identifier);
static uint8_T c16_h_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId);
static void c16_eML_blk_kernel(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq
  *c16_b_blankMsg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel1
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel2
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel3
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel4
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel5
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel6(const
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel7(const
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel8(const
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg);
static void init_dsm_address_info(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c16_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c16_is_active_c16_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c16_do_animation_call_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  static const char_T *c16_sv[10] = { "Q", "Q_SL_Info", "Qd", "Qd_SL_Info",
    "Qdd", "Qdd_SL_Info", "Controller", "Controller_SL_Info", "Other",
    "Other_SL_Info" };

  static const char_T *c16_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c16_b_y = NULL;
  const mxArray *c16_c_y = NULL;
  const mxArray *c16_d_y = NULL;
  const mxArray *c16_e_y = NULL;
  const mxArray *c16_f_y = NULL;
  const mxArray *c16_g_y = NULL;
  const mxArray *c16_h_y = NULL;
  const mxArray *c16_i_y = NULL;
  const mxArray *c16_j_y = NULL;
  const mxArray *c16_k_y = NULL;
  const mxArray *c16_l_y = NULL;
  const mxArray *c16_m_y = NULL;
  const mxArray *c16_n_y = NULL;
  const mxArray *c16_o_y = NULL;
  const mxArray *c16_p_y = NULL;
  const mxArray *c16_q_y = NULL;
  const mxArray *c16_r_y = NULL;
  const mxArray *c16_s_y = NULL;
  const mxArray *c16_st;
  const mxArray *c16_t_y = NULL;
  const mxArray *c16_u_y = NULL;
  const mxArray *c16_v_y = NULL;
  const mxArray *c16_w_y = NULL;
  const mxArray *c16_y = NULL;
  real_T c16_u[7];
  int32_T c16_i;
  int32_T c16_i1;
  int32_T c16_i2;
  int32_T c16_i3;
  int32_T c16_i4;
  uint32_T c16_b_u;
  uint32_T c16_c_u;
  uint32_T c16_d_u;
  uint32_T c16_e_u;
  uint32_T c16_f_u;
  uint32_T c16_g_u;
  uint32_T c16_i_u;
  uint32_T c16_j_u;
  uint32_T c16_k_u;
  uint32_T c16_l_u;
  uint32_T c16_u_CurrentLength;
  uint32_T c16_u_ReceivedLength;
  uint8_T c16_h_u[4];
  c16_st = NULL;
  c16_y = NULL;
  sf_mex_assign(&c16_y, sf_mex_createcellmatrix(2, 1), false);
  c16_b_y = NULL;
  sf_mex_assign(&c16_b_y, sf_mex_createstruct("structure", 10, c16_sv, 2, 1, 1),
                false);
  for (c16_i = 0; c16_i < 7; c16_i++) {
    c16_u[c16_i] = ((real_T *)&((char_T *)chartInstance->c16_msg)[0])[c16_i];
  }

  c16_c_y = NULL;
  sf_mex_assign(&c16_c_y, sf_mex_create("y", c16_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Q", c16_c_y, 0);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [56])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [56])[4];
  c16_d_y = NULL;
  sf_mex_assign(&c16_d_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_b_u = c16_u_CurrentLength;
  c16_e_y = NULL;
  sf_mex_assign(&c16_e_y, sf_mex_create("y", &c16_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_d_y, 0, "CurrentLength", c16_e_y, 0);
  c16_c_u = c16_u_ReceivedLength;
  c16_f_y = NULL;
  sf_mex_assign(&c16_f_y, sf_mex_create("y", &c16_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_d_y, 0, "ReceivedLength", c16_f_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Q_SL_Info", c16_d_y, 1);
  for (c16_i1 = 0; c16_i1 < 7; c16_i1++) {
    c16_u[c16_i1] = ((real_T *)&((char_T *)chartInstance->c16_msg)[64])[c16_i1];
  }

  c16_g_y = NULL;
  sf_mex_assign(&c16_g_y, sf_mex_create("y", c16_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qd", c16_g_y, 2);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [120])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [120])[4];
  c16_h_y = NULL;
  sf_mex_assign(&c16_h_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_d_u = c16_u_CurrentLength;
  c16_i_y = NULL;
  sf_mex_assign(&c16_i_y, sf_mex_create("y", &c16_d_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_h_y, 0, "CurrentLength", c16_i_y, 0);
  c16_e_u = c16_u_ReceivedLength;
  c16_j_y = NULL;
  sf_mex_assign(&c16_j_y, sf_mex_create("y", &c16_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_h_y, 0, "ReceivedLength", c16_j_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qd_SL_Info", c16_h_y, 3);
  for (c16_i2 = 0; c16_i2 < 7; c16_i2++) {
    c16_u[c16_i2] = ((real_T *)&((char_T *)chartInstance->c16_msg)[128])[c16_i2];
  }

  c16_k_y = NULL;
  sf_mex_assign(&c16_k_y, sf_mex_create("y", c16_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qdd", c16_k_y, 4);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [184])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [184])[4];
  c16_l_y = NULL;
  sf_mex_assign(&c16_l_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_f_u = c16_u_CurrentLength;
  c16_m_y = NULL;
  sf_mex_assign(&c16_m_y, sf_mex_create("y", &c16_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_l_y, 0, "CurrentLength", c16_m_y, 0);
  c16_g_u = c16_u_ReceivedLength;
  c16_n_y = NULL;
  sf_mex_assign(&c16_n_y, sf_mex_create("y", &c16_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_l_y, 0, "ReceivedLength", c16_n_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qdd_SL_Info", c16_l_y, 5);
  for (c16_i3 = 0; c16_i3 < 4; c16_i3++) {
    c16_h_u[c16_i3] = ((uint8_T *)&((char_T *)chartInstance->c16_msg)[192])
      [c16_i3];
  }

  c16_o_y = NULL;
  sf_mex_assign(&c16_o_y, sf_mex_create("y", c16_h_u, 3, 0U, 1U, 0U, 1, 4),
                false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Controller", c16_o_y, 6);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[4];
  c16_p_y = NULL;
  sf_mex_assign(&c16_p_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_i_u = c16_u_CurrentLength;
  c16_q_y = NULL;
  sf_mex_assign(&c16_q_y, sf_mex_create("y", &c16_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_p_y, 0, "CurrentLength", c16_q_y, 0);
  c16_j_u = c16_u_ReceivedLength;
  c16_r_y = NULL;
  sf_mex_assign(&c16_r_y, sf_mex_create("y", &c16_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_p_y, 0, "ReceivedLength", c16_r_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Controller_SL_Info", c16_p_y, 7);
  for (c16_i4 = 0; c16_i4 < 7; c16_i4++) {
    c16_u[c16_i4] = ((real_T *)&((char_T *)chartInstance->c16_msg)[208])[c16_i4];
  }

  c16_s_y = NULL;
  sf_mex_assign(&c16_s_y, sf_mex_create("y", c16_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Other", c16_s_y, 8);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [264])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [264])[4];
  c16_t_y = NULL;
  sf_mex_assign(&c16_t_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_k_u = c16_u_CurrentLength;
  c16_u_y = NULL;
  sf_mex_assign(&c16_u_y, sf_mex_create("y", &c16_k_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_t_y, 0, "CurrentLength", c16_u_y, 0);
  c16_l_u = c16_u_ReceivedLength;
  c16_v_y = NULL;
  sf_mex_assign(&c16_v_y, sf_mex_create("y", &c16_l_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_t_y, 0, "ReceivedLength", c16_v_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Other_SL_Info", c16_t_y, 9);
  sf_mex_setcell(c16_y, 0, c16_b_y);
  c16_w_y = NULL;
  sf_mex_assign(&c16_w_y, sf_mex_create("y",
    &chartInstance->c16_is_active_c16_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c16_y, 1, c16_w_y);
  sf_mex_assign(&c16_st, c16_y, false);
  return c16_st;
}

static void set_sim_state_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_st)
{
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq c16_r;
  const mxArray *c16_u;
  int32_T c16_i;
  int32_T c16_i1;
  int32_T c16_i2;
  c16_u = sf_mex_dup(c16_st);
  c16_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c16_u, 0)),
                       "msg", &c16_r);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[0] = c16_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[4] = c16_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[0] = c16_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[4] = c16_r.Qd_SL_Info.ReceivedLength;
  for (c16_i = 0; c16_i < 7; c16_i++) {
    ((real_T *)&((char_T *)chartInstance->c16_msg)[0])[c16_i] = c16_r.Q[c16_i];
    ((real_T *)&((char_T *)chartInstance->c16_msg)[64])[c16_i] = c16_r.Qd[c16_i];
    ((real_T *)&((char_T *)chartInstance->c16_msg)[128])[c16_i] =
      c16_r.Qdd[c16_i];
  }

  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[0] = c16_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[4] = c16_r.Qdd_SL_Info.ReceivedLength;
  for (c16_i1 = 0; c16_i1 < 4; c16_i1++) {
    ((uint8_T *)&((char_T *)chartInstance->c16_msg)[192])[c16_i1] =
      c16_r.Controller[c16_i1];
  }

  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[0] = c16_r.Controller_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[4] = c16_r.Controller_SL_Info.ReceivedLength;
  for (c16_i2 = 0; c16_i2 < 7; c16_i2++) {
    ((real_T *)&((char_T *)chartInstance->c16_msg)[208])[c16_i2] =
      c16_r.Other[c16_i2];
  }

  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[0] = c16_r.Other_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[4] = c16_r.Other_SL_Info.ReceivedLength;
  chartInstance->c16_is_active_c16_sliding_controller_rbdl =
    c16_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c16_u, 1)),
    "is_active_c16_sliding_controller_rbdl");
  sf_mex_destroy(&c16_u);
  sf_mex_destroy(&c16_st);
}

static void sf_gateway_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq c16_r;
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq
    *c16_gpu_blankMsg;
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_gpu_msg;
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_gpu_r;
  hipMalloc(&c16_gpu_r, 272UL);
  hipMalloc(&c16_gpu_msg, 272UL);
  hipMalloc(&c16_gpu_blankMsg, 272UL);
  chartInstance->c16_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[56])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[56])[4];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[120])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[120])[4];
  hipMemcpy(c16_gpu_blankMsg, chartInstance->c16_blankMsg, 272UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_blankMsg, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c16_blankMsg, c16_gpu_blankMsg, 272UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[184])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[184])[4];
  hipMemcpy(c16_gpu_blankMsg, chartInstance->c16_blankMsg, 272UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel2<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_blankMsg, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c16_blankMsg, c16_gpu_blankMsg, 272UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[200])[4];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[264])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[264])[4];
  c16_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [56])[0];
  c16_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [56])[4];
  c16_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [120])[0];
  c16_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [120])[4];
  hipMemcpy(c16_gpu_r, &c16_r, 272UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_blankMsg, chartInstance->c16_blankMsg, 272UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel3<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_r, c16_gpu_blankMsg, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c16_blankMsg, c16_gpu_blankMsg, 272UL,
             hipMemcpyDeviceToHost);
  hipMemcpy(&c16_r, c16_gpu_r, 272UL, hipMemcpyDeviceToHost);
  c16_r.Qdd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [184])[0];
  c16_r.Qdd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [184])[4];
  hipMemcpy(c16_gpu_r, &c16_r, 272UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel4<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_msg, c16_gpu_r);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c16_r, c16_gpu_r, 272UL, hipMemcpyDeviceToHost);
  c16_r.Controller_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[0];
  c16_r.Controller_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[4];
  hipMemcpy(c16_gpu_r, &c16_r, 272UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel5<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_msg, c16_gpu_r);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c16_r, c16_gpu_r, 272UL, hipMemcpyDeviceToHost);
  c16_r.Other_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [264])[0];
  c16_r.Other_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [264])[4];
  c16_eML_blk_kernel(chartInstance, &c16_r);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[0] = c16_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[4] = c16_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[0] = c16_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[4] = c16_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c16_gpu_r, &c16_r, 272UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel6<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_r, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[0] = c16_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[4] = c16_r.Qdd_SL_Info.ReceivedLength;
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel7<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_r, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[0] = c16_r.Controller_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[4] = c16_r.Controller_SL_Info.ReceivedLength;
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel8<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_r, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[0] = c16_r.Other_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[4] = c16_r.Other_SL_Info.ReceivedLength;
  c16_do_animation_call_c16_sliding_controller_rbdl(chartInstance);
  hipFree(c16_gpu_blankMsg);
  hipFree(c16_gpu_msg);
  hipFree(c16_gpu_r);
}

static void mdl_start_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c16_errCode;
  c16_errCode = hipGetLastError();
  if (c16_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c16_errCode, (char_T *)hipGetErrorName
                       (c16_errCode), (char_T *)hipGetErrorString(c16_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c16_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c16_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c16_nameCaptureInfo = NULL;
  c16_nameCaptureInfo = NULL;
  sf_mex_assign(&c16_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c16_nameCaptureInfo;
}

static void c16_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_b_msg, const char_T *c16_identifier,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_y)
{
  emlrtMsgIdentifier c16_thisId;
  c16_thisId.fIdentifier = const_cast<const char_T *>(c16_identifier);
  c16_thisId.fParent = NULL;
  c16_thisId.bParentIsCell = false;
  c16_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c16_b_msg), &c16_thisId,
    c16_y);
  sf_mex_destroy(&c16_b_msg);
}

static void c16_b_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_y)
{
  static const char_T *c16_fieldNames[10] = { "Q", "Q_SL_Info", "Qd",
    "Qd_SL_Info", "Qdd", "Qdd_SL_Info", "Controller", "Controller_SL_Info",
    "Other", "Other_SL_Info" };

  emlrtMsgIdentifier c16_thisId;
  c16_thisId.fParent = c16_parentId;
  c16_thisId.bParentIsCell = false;
  sf_mex_check_struct(c16_parentId, c16_u, 10, c16_fieldNames, 0U, NULL);
  c16_thisId.fIdentifier = "Q";
  c16_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u, "Q",
    "Q", 0)), &c16_thisId, c16_y->Q);
  c16_thisId.fIdentifier = "Q_SL_Info";
  c16_y->Q_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Q_SL_Info", "Q_SL_Info", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "Qd";
  c16_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u, "Qd",
    "Qd", 0)), &c16_thisId, c16_y->Qd);
  c16_thisId.fIdentifier = "Qd_SL_Info";
  c16_y->Qd_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "Qdd";
  c16_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u, "Qdd",
    "Qdd", 0)), &c16_thisId, c16_y->Qdd);
  c16_thisId.fIdentifier = "Qdd_SL_Info";
  c16_y->Qdd_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Qdd_SL_Info", "Qdd_SL_Info", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "Controller";
  c16_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u,
    "Controller", "Controller", 0)), &c16_thisId, c16_y->Controller);
  c16_thisId.fIdentifier = "Controller_SL_Info";
  c16_y->Controller_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Controller_SL_Info", "Controller_SL_Info", 0)),
    &c16_thisId);
  c16_thisId.fIdentifier = "Other";
  c16_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u,
    "Other", "Other", 0)), &c16_thisId, c16_y->Other);
  c16_thisId.fIdentifier = "Other_SL_Info";
  c16_y->Other_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Other_SL_Info", "Other_SL_Info", 0)), &c16_thisId);
  sf_mex_destroy(&c16_u);
}

static void c16_c_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  real_T c16_y[7])
{
  real_T c16_dv[7];
  int32_T c16_i;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), c16_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c16_i = 0; c16_i < 7; c16_i++) {
    c16_y[c16_i] = c16_dv[c16_i];
  }

  sf_mex_destroy(&c16_u);
}

static c16_SL_Bus_ROSVariableLengthArrayInfo c16_d_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  static const char_T *c16_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c16_SL_Bus_ROSVariableLengthArrayInfo c16_y;
  emlrtMsgIdentifier c16_thisId;
  c16_thisId.fParent = c16_parentId;
  c16_thisId.bParentIsCell = false;
  sf_mex_check_struct(c16_parentId, c16_u, 2, c16_fieldNames, 0U, NULL);
  c16_thisId.fIdentifier = "CurrentLength";
  c16_y.CurrentLength = c16_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "CurrentLength", "CurrentLength", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "ReceivedLength";
  c16_y.ReceivedLength = c16_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "ReceivedLength", "ReceivedLength", 0)), &c16_thisId);
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static uint32_T c16_e_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  uint32_T c16_b_u;
  uint32_T c16_y;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), &c16_b_u, 1, 7, 0U, 0, 0U, 0);
  c16_y = c16_b_u;
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static void c16_f_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  uint8_T c16_y[4])
{
  int32_T c16_i;
  uint8_T c16_uv[4];
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), c16_uv, 1, 3, 0U, 1, 0U, 1, 4);
  for (c16_i = 0; c16_i < 4; c16_i++) {
    c16_y[c16_i] = c16_uv[c16_i];
  }

  sf_mex_destroy(&c16_u);
}

static uint8_T c16_g_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_b_is_active_c16_sliding_controller_rbdl, const char_T *c16_identifier)
{
  emlrtMsgIdentifier c16_thisId;
  uint8_T c16_y;
  c16_thisId.fIdentifier = const_cast<const char_T *>(c16_identifier);
  c16_thisId.fParent = NULL;
  c16_thisId.bParentIsCell = false;
  c16_y = c16_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c16_b_is_active_c16_sliding_controller_rbdl), &c16_thisId);
  sf_mex_destroy(&c16_b_is_active_c16_sliding_controller_rbdl);
  return c16_y;
}

static uint8_T c16_h_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  uint8_T c16_b_u;
  uint8_T c16_y;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), &c16_b_u, 1, 3, 0U, 0, 0U, 0);
  c16_y = c16_b_u;
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static void c16_eML_blk_kernel(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq
  *c16_b_blankMsg)
{
  c16_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
  c16_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
  c16_b_blankMsg->Qdd_SL_Info.CurrentLength = 7U;
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel1
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i = (int32_T)c16_threadId;
  if (c16_i < 7) {
    c16_b_msg->Q[c16_i] = c16_b_blankMsg->Q[c16_i];
    c16_b_msg->Qd[c16_i] = c16_b_blankMsg->Qd[c16_i];
    c16_b_msg->Qdd[c16_i] = c16_b_blankMsg->Qdd[c16_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel2
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i1;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i1 = (int32_T)c16_threadId;
  if (c16_i1 < 4) {
    c16_b_msg->Controller[c16_i1] = c16_b_blankMsg->Controller[c16_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel3
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i2;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i2 = (int32_T)c16_threadId;
  if (c16_i2 < 7) {
    c16_b_msg->Other[c16_i2] = c16_b_blankMsg->Other[c16_i2];
    c16_r->Q[c16_i2] = c16_b_msg->Q[c16_i2];
    c16_r->Qd[c16_i2] = c16_b_msg->Qd[c16_i2];
    c16_r->Qdd[c16_i2] = c16_b_msg->Qdd[c16_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel4
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r)
{
  uint64_T c16_threadId;
  int32_T c16_i3;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i3 = (int32_T)c16_threadId;
  if (c16_i3 < 4) {
    c16_r->Controller[c16_i3] = c16_b_msg->Controller[c16_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel5
  (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg,
   c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r)
{
  uint64_T c16_threadId;
  int32_T c16_i4;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i4 = (int32_T)c16_threadId;
  if (c16_i4 < 7) {
    c16_r->Other[c16_i4] = c16_b_msg->Other[c16_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel6(const
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i5;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i5 = (int32_T)c16_threadId;
  if (c16_i5 < 7) {
    c16_b_msg->Q[c16_i5] = c16_r->Q[c16_i5];
    c16_b_msg->Qd[c16_i5] = c16_r->Qd[c16_i5];
    c16_b_msg->Qdd[c16_i5] = c16_r->Qdd[c16_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel7(const
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i6;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i6 = (int32_T)c16_threadId;
  if (c16_i6 < 4) {
    c16_b_msg->Controller[c16_i6] = c16_r->Controller[c16_i6];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel8(const
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i7;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i7 = (int32_T)c16_threadId;
  if (c16_i7 < 7) {
    c16_b_msg->Other[c16_i7] = c16_r->Other[c16_i7];
  }
}

static void init_dsm_address_info(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c16_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c16_blankMsg =
    (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c16_msg =
    (c16_SL_Bus_sliding_controller_rbdl_DesiredJointsCmdRequest_rfbsq *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c16_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2412018572U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1870368763U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(254930633U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2940839482U);
}

mxArray *sf_c16_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c16_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c16_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c16_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvyqK+yH8zOL4xOSSzLLU+GRDs/jinMyUzLz0+OT8vJKi/Jyc1KL4oqSUHFj8AwEAPYEeh"
    "A=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c16_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sbp4IWq9s75xwXA1Mt6Z23";
}

static void sf_opaque_initialize_c16_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c16_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c16_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c16_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c16_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c16_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c16_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc16_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c16_sliding_controller_rbdl
      ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc16_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c16_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c16_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c16_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c16_sliding_controller_rbdl
      ((SFc16_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc16_sliding_controller_rbdl
      ((SFc16_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c16_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQphXXaIA0MIogJwPNIUZ7bP7bS2pbP4gAqzZKOwl6EVbLkbjQcpfZH8p+gp6Cvkp",
    "vOeU9ij5Cc0mAPEBnSUqWKZKKI8RIixKg6SW/mZ2Zb35W3lq35+F1He+drz1vA59f4t3wsuuLfL",
    "02d2fv171v8/VLFBI2OiSKRNqrvQSJ4BfQklvDpOiKoSyFMTEEBYIiNpbKVGnTLLKciXHHCur06",
    "Wcho6EfSsuDPZQlwYHgp6gttuYQ9bSYAmo6AIEJlbSjsMPJaGaxMpNmCHSsbVTnggbj29iZpXuW",
    "GxZzaJ8A7QptCFqsz2zzDTHQNCeVbjpPtT8FyijmjIhSb0OifYgxwAaO4wD/HliDThVhNCTK7EF",
    "IEtD7bJzqlAKKOpnGDwMmiJGKEd6OeNMJLtp2yNGengyA1wQEbdtTQMaxZMJU8+930NO2IAMOLR",
    "jYUbU2H15YR/5TBhNQlXEbNmUCiozgQFRumgakfZKyNcuSRZhhETwlapcifxqCyuzFzNE+QZ7gC",
    "CWqYJA62dVHiiUY3kptNuq6zFxWMjbKyNbLYKm2dgJ1LMy0dahoEs51JexIxvuQAE+1togh9bBM",
    "azlOaxYcSQywS+/qarCCIfE5rClFwErpSgqAtO/8jI3lPJJabWTUxORt7e8vfl6EdYUBNSQUyrq",
    "AIkwDxiwNb7W2gGnHPQLRKpOaVwbOMmQZytNDK1oTqcYYk5omcuaCY7QSGOkRcomVcKyxaOpgjs",
    "tlOEpoCIFrMIxDD8sGsSUx0a617WLdJcyctkBTxeISVi1WHbahtkuo0xiOxVjIiegoGfl5j8/CC",
    "4DZRpRgYrSH/UuddnD3MircPPveO5tn1z5gnk3lis/v5vSslejx5p7Ffa826vdt4H9r0/k7J/dV",
    "YZ/1gpzDbeL919bt7eT1+9823vz9+7tG8GqV/f9oXGz+X8/XW9NGOyucZCFfHfbJnF3rJfpvzun",
    "fzNd6EN/vPnvxo3704GTyfPdOzzz89e69nJ8l9jYK9k7f33IdH7MszU9Fu0F+MHFrYrNx7fT/MG",
    "fvxpJ4XM3fZ9fbn1aTv7FT5LEsXlfOxeuKq9xzefvx+9/aWU0+2//5Evu3C3xvp3O9T1y3gD698",
    "7CvOQuw1PtUCqMk56D6ahDwQn1+bL5fVM67ZLl/i53/y3163j9kfl22nHfJcqv6d9E5/rnh6+aB",
    "V8BvfsZ+rHq++tT4P72LnYO+ydePZz91miHjQcmpN/+MR9hh2df/QJ7+AwkwtQw=",
    ""
  };

  static char newstr [1265] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c16_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c16_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(623123674U));
  ssSetChecksum1(S,(2280963446U));
  ssSetChecksum2(S,(2414537478U));
  ssSetChecksum3(S,(3076784881U));
}

static void mdlRTW_c16_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c16_sliding_controller_rbdl(SimStruct *S)
{
  SFc16_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc16_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc16_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc16_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc16_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c16_JITStateAnimation,
    chartInstance->c16_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c16_sliding_controller_rbdl(chartInstance);
}

void c16_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c16_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c16_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c16_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c16_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
