#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c16_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c16_do_animation_call_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_st);
static void sf_gateway_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c16_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_b_msg, const char_T *c16_identifier,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_y);
static void c16_b_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_y);
static void c16_c_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  uint8_T c16_y[4]);
static c16_SL_Bus_ROSVariableLengthArrayInfo c16_d_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId);
static uint32_T c16_e_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId);
static void c16_f_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  real_T c16_y[7]);
static uint8_T c16_g_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_b_is_active_c16_sliding_controller_rbdl, const char_T *c16_identifier);
static uint8_T c16_h_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId);
static void c16_eML_blk_kernel(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm
  *c16_b_blankMsg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel1
  (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel2
  (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel3
  (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg,
   c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_r);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel4
  (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg,
   c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_r);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel5(const
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller_rbdl_kernel6(const
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg);
static void init_dsm_address_info(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c16_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c16_is_active_c16_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c16_do_animation_call_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  static const char_T *c16_sv[8] = { "ModelName", "ModelName_SL_Info", "Q",
    "Q_SL_Info", "Qd", "Qd_SL_Info", "Qdd", "Qdd_SL_Info" };

  static const char_T *c16_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c16_b_y = NULL;
  const mxArray *c16_c_y = NULL;
  const mxArray *c16_d_y = NULL;
  const mxArray *c16_e_y = NULL;
  const mxArray *c16_f_y = NULL;
  const mxArray *c16_g_y = NULL;
  const mxArray *c16_h_y = NULL;
  const mxArray *c16_i_y = NULL;
  const mxArray *c16_j_y = NULL;
  const mxArray *c16_k_y = NULL;
  const mxArray *c16_l_y = NULL;
  const mxArray *c16_m_y = NULL;
  const mxArray *c16_n_y = NULL;
  const mxArray *c16_o_y = NULL;
  const mxArray *c16_p_y = NULL;
  const mxArray *c16_q_y = NULL;
  const mxArray *c16_r_y = NULL;
  const mxArray *c16_s_y = NULL;
  const mxArray *c16_st;
  const mxArray *c16_y = NULL;
  real_T c16_d_u[7];
  int32_T c16_i;
  int32_T c16_i1;
  int32_T c16_i2;
  int32_T c16_i3;
  uint32_T c16_b_u;
  uint32_T c16_c_u;
  uint32_T c16_e_u;
  uint32_T c16_f_u;
  uint32_T c16_g_u;
  uint32_T c16_h_u;
  uint32_T c16_i_u;
  uint32_T c16_j_u;
  uint32_T c16_u_CurrentLength;
  uint32_T c16_u_ReceivedLength;
  uint8_T c16_u[4];
  c16_st = NULL;
  c16_y = NULL;
  sf_mex_assign(&c16_y, sf_mex_createcellmatrix(2, 1), false);
  c16_b_y = NULL;
  sf_mex_assign(&c16_b_y, sf_mex_createstruct("structure", 8, c16_sv, 2, 1, 1),
                false);
  for (c16_i = 0; c16_i < 4; c16_i++) {
    c16_u[c16_i] = ((uint8_T *)&((char_T *)chartInstance->c16_msg)[0])[c16_i];
  }

  c16_c_y = NULL;
  sf_mex_assign(&c16_c_y, sf_mex_create("y", c16_u, 3, 0U, 1U, 0U, 1, 4), false);
  sf_mex_setfieldbynum(c16_b_y, 0, "ModelName", c16_c_y, 0);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [8])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [8])[4];
  c16_d_y = NULL;
  sf_mex_assign(&c16_d_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_b_u = c16_u_CurrentLength;
  c16_e_y = NULL;
  sf_mex_assign(&c16_e_y, sf_mex_create("y", &c16_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_d_y, 0, "CurrentLength", c16_e_y, 0);
  c16_c_u = c16_u_ReceivedLength;
  c16_f_y = NULL;
  sf_mex_assign(&c16_f_y, sf_mex_create("y", &c16_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_d_y, 0, "ReceivedLength", c16_f_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "ModelName_SL_Info", c16_d_y, 1);
  for (c16_i1 = 0; c16_i1 < 7; c16_i1++) {
    c16_d_u[c16_i1] = ((real_T *)&((char_T *)chartInstance->c16_msg)[16])[c16_i1];
  }

  c16_g_y = NULL;
  sf_mex_assign(&c16_g_y, sf_mex_create("y", c16_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Q", c16_g_y, 2);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [72])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [72])[4];
  c16_h_y = NULL;
  sf_mex_assign(&c16_h_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_e_u = c16_u_CurrentLength;
  c16_i_y = NULL;
  sf_mex_assign(&c16_i_y, sf_mex_create("y", &c16_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_h_y, 0, "CurrentLength", c16_i_y, 0);
  c16_f_u = c16_u_ReceivedLength;
  c16_j_y = NULL;
  sf_mex_assign(&c16_j_y, sf_mex_create("y", &c16_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_h_y, 0, "ReceivedLength", c16_j_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Q_SL_Info", c16_h_y, 3);
  for (c16_i2 = 0; c16_i2 < 7; c16_i2++) {
    c16_d_u[c16_i2] = ((real_T *)&((char_T *)chartInstance->c16_msg)[80])[c16_i2];
  }

  c16_k_y = NULL;
  sf_mex_assign(&c16_k_y, sf_mex_create("y", c16_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qd", c16_k_y, 4);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [136])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [136])[4];
  c16_l_y = NULL;
  sf_mex_assign(&c16_l_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_g_u = c16_u_CurrentLength;
  c16_m_y = NULL;
  sf_mex_assign(&c16_m_y, sf_mex_create("y", &c16_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_l_y, 0, "CurrentLength", c16_m_y, 0);
  c16_h_u = c16_u_ReceivedLength;
  c16_n_y = NULL;
  sf_mex_assign(&c16_n_y, sf_mex_create("y", &c16_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_l_y, 0, "ReceivedLength", c16_n_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qd_SL_Info", c16_l_y, 5);
  for (c16_i3 = 0; c16_i3 < 7; c16_i3++) {
    c16_d_u[c16_i3] = ((real_T *)&((char_T *)chartInstance->c16_msg)[144])
      [c16_i3];
  }

  c16_o_y = NULL;
  sf_mex_assign(&c16_o_y, sf_mex_create("y", c16_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qdd", c16_o_y, 6);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[4];
  c16_p_y = NULL;
  sf_mex_assign(&c16_p_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_i_u = c16_u_CurrentLength;
  c16_q_y = NULL;
  sf_mex_assign(&c16_q_y, sf_mex_create("y", &c16_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_p_y, 0, "CurrentLength", c16_q_y, 0);
  c16_j_u = c16_u_ReceivedLength;
  c16_r_y = NULL;
  sf_mex_assign(&c16_r_y, sf_mex_create("y", &c16_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_p_y, 0, "ReceivedLength", c16_r_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qdd_SL_Info", c16_p_y, 7);
  sf_mex_setcell(c16_y, 0, c16_b_y);
  c16_s_y = NULL;
  sf_mex_assign(&c16_s_y, sf_mex_create("y",
    &chartInstance->c16_is_active_c16_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c16_y, 1, c16_s_y);
  sf_mex_assign(&c16_st, c16_y, false);
  return c16_st;
}

static void set_sim_state_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_st)
{
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm c16_r;
  const mxArray *c16_u;
  int32_T c16_i;
  int32_T c16_i1;
  c16_u = sf_mex_dup(c16_st);
  c16_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c16_u, 0)),
                       "msg", &c16_r);
  for (c16_i = 0; c16_i < 4; c16_i++) {
    ((uint8_T *)&((char_T *)chartInstance->c16_msg)[0])[c16_i] =
      c16_r.ModelName[c16_i];
  }

  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[8])[0] = c16_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[8])[4] = c16_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[72])[0] = c16_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[72])[4] = c16_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[136])[0] = c16_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[136])[4] = c16_r.Qd_SL_Info.ReceivedLength;
  for (c16_i1 = 0; c16_i1 < 7; c16_i1++) {
    ((real_T *)&((char_T *)chartInstance->c16_msg)[16])[c16_i1] = c16_r.Q[c16_i1];
    ((real_T *)&((char_T *)chartInstance->c16_msg)[80])[c16_i1] =
      c16_r.Qd[c16_i1];
    ((real_T *)&((char_T *)chartInstance->c16_msg)[144])[c16_i1] =
      c16_r.Qdd[c16_i1];
  }

  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[0] = c16_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[4] = c16_r.Qdd_SL_Info.ReceivedLength;
  chartInstance->c16_is_active_c16_sliding_controller_rbdl =
    c16_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c16_u, 1)),
    "is_active_c16_sliding_controller_rbdl");
  sf_mex_destroy(&c16_u);
  sf_mex_destroy(&c16_st);
}

static void sf_gateway_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm c16_r;
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm
    *c16_gpu_blankMsg;
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_gpu_msg;
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_gpu_r;
  hipMalloc(&c16_gpu_r, 208UL);
  hipMalloc(&c16_gpu_msg, 208UL);
  hipMalloc(&c16_gpu_blankMsg, 208UL);
  chartInstance->c16_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(c16_gpu_blankMsg, chartInstance->c16_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 208UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_blankMsg, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c16_blankMsg, c16_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[8])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[8])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[8])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[8])[4];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[72])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[72])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[72])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[72])[4];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[136])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[136])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[136])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[136])[4];
  hipMemcpy(c16_gpu_blankMsg, chartInstance->c16_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 208UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel2<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_blankMsg, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c16_blankMsg, c16_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[200])[4];
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 208UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel3<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_msg, c16_gpu_r);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c16_r, c16_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c16_r.ModelName_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [8])[0];
  c16_r.ModelName_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [8])[4];
  c16_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [72])[0];
  c16_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [72])[4];
  c16_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [136])[0];
  c16_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [136])[4];
  hipMemcpy(c16_gpu_r, &c16_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 208UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel4<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_msg, c16_gpu_r);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c16_r, c16_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c16_r.Qdd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[0];
  c16_r.Qdd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[4];
  c16_eML_blk_kernel(chartInstance, &c16_r);
  hipMemcpy(c16_gpu_r, &c16_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 208UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel5<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_r, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[8])[0] = c16_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[8])[4] = c16_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[72])[0] = c16_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[72])[4] = c16_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[136])[0] = c16_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[136])[4] = c16_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 208UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel6<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c16_gpu_r, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[0] = c16_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[4] = c16_r.Qdd_SL_Info.ReceivedLength;
  c16_do_animation_call_c16_sliding_controller_rbdl(chartInstance);
  hipFree(c16_gpu_blankMsg);
  hipFree(c16_gpu_msg);
  hipFree(c16_gpu_r);
}

static void mdl_start_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c16_errCode;
  c16_errCode = hipGetLastError();
  if (c16_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c16_errCode, (char_T *)hipGetErrorName
                       (c16_errCode), (char_T *)hipGetErrorString(c16_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c16_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc16_sliding_controller_rbdl
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c16_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c16_nameCaptureInfo = NULL;
  c16_nameCaptureInfo = NULL;
  sf_mex_assign(&c16_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c16_nameCaptureInfo;
}

static void c16_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_b_msg, const char_T *c16_identifier,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_y)
{
  emlrtMsgIdentifier c16_thisId;
  c16_thisId.fIdentifier = const_cast<const char_T *>(c16_identifier);
  c16_thisId.fParent = NULL;
  c16_thisId.bParentIsCell = false;
  c16_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c16_b_msg), &c16_thisId,
    c16_y);
  sf_mex_destroy(&c16_b_msg);
}

static void c16_b_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_y)
{
  static const char_T *c16_fieldNames[8] = { "ModelName", "ModelName_SL_Info",
    "Q", "Q_SL_Info", "Qd", "Qd_SL_Info", "Qdd", "Qdd_SL_Info" };

  emlrtMsgIdentifier c16_thisId;
  c16_thisId.fParent = c16_parentId;
  c16_thisId.bParentIsCell = false;
  sf_mex_check_struct(c16_parentId, c16_u, 8, c16_fieldNames, 0U, NULL);
  c16_thisId.fIdentifier = "ModelName";
  c16_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u,
    "ModelName", "ModelName", 0)), &c16_thisId, c16_y->ModelName);
  c16_thisId.fIdentifier = "ModelName_SL_Info";
  c16_y->ModelName_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "ModelName_SL_Info", "ModelName_SL_Info", 0)),
    &c16_thisId);
  c16_thisId.fIdentifier = "Q";
  c16_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u, "Q",
    "Q", 0)), &c16_thisId, c16_y->Q);
  c16_thisId.fIdentifier = "Q_SL_Info";
  c16_y->Q_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Q_SL_Info", "Q_SL_Info", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "Qd";
  c16_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u, "Qd",
    "Qd", 0)), &c16_thisId, c16_y->Qd);
  c16_thisId.fIdentifier = "Qd_SL_Info";
  c16_y->Qd_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "Qdd";
  c16_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u, "Qdd",
    "Qdd", 0)), &c16_thisId, c16_y->Qdd);
  c16_thisId.fIdentifier = "Qdd_SL_Info";
  c16_y->Qdd_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Qdd_SL_Info", "Qdd_SL_Info", 0)), &c16_thisId);
  sf_mex_destroy(&c16_u);
}

static void c16_c_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  uint8_T c16_y[4])
{
  int32_T c16_i;
  uint8_T c16_uv[4];
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), c16_uv, 1, 3, 0U, 1, 0U, 1, 4);
  for (c16_i = 0; c16_i < 4; c16_i++) {
    c16_y[c16_i] = c16_uv[c16_i];
  }

  sf_mex_destroy(&c16_u);
}

static c16_SL_Bus_ROSVariableLengthArrayInfo c16_d_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  static const char_T *c16_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c16_SL_Bus_ROSVariableLengthArrayInfo c16_y;
  emlrtMsgIdentifier c16_thisId;
  c16_thisId.fParent = c16_parentId;
  c16_thisId.bParentIsCell = false;
  sf_mex_check_struct(c16_parentId, c16_u, 2, c16_fieldNames, 0U, NULL);
  c16_thisId.fIdentifier = "CurrentLength";
  c16_y.CurrentLength = c16_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "CurrentLength", "CurrentLength", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "ReceivedLength";
  c16_y.ReceivedLength = c16_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "ReceivedLength", "ReceivedLength", 0)), &c16_thisId);
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static uint32_T c16_e_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  uint32_T c16_b_u;
  uint32_T c16_y;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), &c16_b_u, 1, 7, 0U, 0, 0U, 0);
  c16_y = c16_b_u;
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static void c16_f_emlrt_marshallIn(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  real_T c16_y[7])
{
  real_T c16_dv[7];
  int32_T c16_i;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), c16_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c16_i = 0; c16_i < 7; c16_i++) {
    c16_y[c16_i] = c16_dv[c16_i];
  }

  sf_mex_destroy(&c16_u);
}

static uint8_T c16_g_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_b_is_active_c16_sliding_controller_rbdl, const char_T *c16_identifier)
{
  emlrtMsgIdentifier c16_thisId;
  uint8_T c16_y;
  c16_thisId.fIdentifier = const_cast<const char_T *>(c16_identifier);
  c16_thisId.fParent = NULL;
  c16_thisId.bParentIsCell = false;
  c16_y = c16_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c16_b_is_active_c16_sliding_controller_rbdl), &c16_thisId);
  sf_mex_destroy(&c16_b_is_active_c16_sliding_controller_rbdl);
  return c16_y;
}

static uint8_T c16_h_emlrt_marshallIn
  (SFc16_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  uint8_T c16_b_u;
  uint8_T c16_y;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), &c16_b_u, 1, 3, 0U, 0, 0U, 0);
  c16_y = c16_b_u;
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static void c16_eML_blk_kernel(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm
  *c16_b_blankMsg)
{
  c16_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
  c16_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
  c16_b_blankMsg->Qdd_SL_Info.CurrentLength = 7U;
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel1
  (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i = (int32_T)c16_threadId;
  if (c16_i < 4) {
    c16_b_msg->ModelName[c16_i] = c16_b_blankMsg->ModelName[c16_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel2
  (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm
   *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i1;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i1 = (int32_T)c16_threadId;
  if (c16_i1 < 7) {
    c16_b_msg->Q[c16_i1] = c16_b_blankMsg->Q[c16_i1];
    c16_b_msg->Qd[c16_i1] = c16_b_blankMsg->Qd[c16_i1];
    c16_b_msg->Qdd[c16_i1] = c16_b_blankMsg->Qdd[c16_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel3
  (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg,
   c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_r)
{
  uint64_T c16_threadId;
  int32_T c16_i2;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i2 = (int32_T)c16_threadId;
  if (c16_i2 < 4) {
    c16_r->ModelName[c16_i2] = c16_b_msg->ModelName[c16_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel4
  (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg,
   c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_r)
{
  uint64_T c16_threadId;
  int32_T c16_i3;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i3 = (int32_T)c16_threadId;
  if (c16_i3 < 7) {
    c16_r->Q[c16_i3] = c16_b_msg->Q[c16_i3];
    c16_r->Qd[c16_i3] = c16_b_msg->Qd[c16_i3];
    c16_r->Qdd[c16_i3] = c16_b_msg->Qdd[c16_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel5(const
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i4;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i4 = (int32_T)c16_threadId;
  if (c16_i4 < 4) {
    c16_b_msg->ModelName[c16_i4] = c16_r->ModelName[c16_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller_rbdl_kernel6(const
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_r,
  c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i5;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i5 = (int32_T)c16_threadId;
  if (c16_i5 < 7) {
    c16_b_msg->Q[c16_i5] = c16_r->Q[c16_i5];
    c16_b_msg->Qd[c16_i5] = c16_r->Qd[c16_i5];
    c16_b_msg->Qdd[c16_i5] = c16_r->Qdd[c16_i5];
  }
}

static void init_dsm_address_info(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc16_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c16_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c16_blankMsg =
    (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c16_msg =
    (c16_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqm *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c16_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3270392620U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4166510679U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3229183098U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3467634171U);
}

mxArray *sf_c16_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c16_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c16_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c16_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvyqK+yH8zOL4xOSSzLLU+GRDs/jinMyUzLz0+OT8vJKi/Jyc1KL4oqSUHFj8AwEAPYEeh"
    "A=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c16_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sku70yXmmpvw9wiNyjwFDG";
}

static void sf_opaque_initialize_c16_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c16_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c16_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c16_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c16_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c16_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c16_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc16_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c16_sliding_controller_rbdl
      ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc16_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c16_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c16_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc16_sliding_controller_rbdl
    ((SFc16_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c16_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c16_sliding_controller_rbdl
      ((SFc16_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc16_sliding_controller_rbdl
      ((SFc16_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c16_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u1EgQdiYBgQRRdkGckOCC4MheYLnshswPjJSQCCcst1FPu2bcO+1u0z+ezJkj4jngyOP",
    "AlSfgEai2PZOJx/YQRkSwWkuO0/ZX1VX11U+Pt9bd8/DaxHv7d8+7iM9LeDe87LqQr9fm7uz9hn",
    "c3X79FIWGjA6JIpL3aS5AInoOW3BomRVcMZCmMiQEoEBSxsVSmSptmkeVMjDpWUKdP/xMyGvqht",
    "DzYQVkS7As+QW2xNQeop8UUUNMBCEyopB2GHU6GM4uVGTdDoCNtozoXNBjfxs4svWe5YTGH9jHQ",
    "rtCGoMX6xDbfEANNc1zppvNU+1OgjGLOiCj1NiTahxgDbOAoDvDvvjXoVBFGQ6LMDoQkAb3LRql",
    "OKaCok2n80GeCGKkY4e2IN53gom0HHO3ZkwHwmoCgbTsKyCiWTJhq/v0OetoWpM+hBX07rNbmwy",
    "vryH/BYAyqMm6DpkxAkSHsi8pN04C0j1O2ZlmyCDMsghdEPabIn4agMnsxc7RPkCc4RIkqGKROd",
    "vWhYgmGt1KbjbouM5eVjI0ysvUyWKqtnUAdCzNtHSqahHNdCTuU8S4kwFOtLWJIPSzTWo7TmgWH",
    "EgPs0ru6GqxgSHwOa0oRsFK6kgIg7TvPsLGcRlKrjYyamLyt3d3Fz4uwrjCgBoRCWRdQhGnAmKX",
    "hrdYWMO24RyBaZVLzysBZhixDeXpgRWss1QhjUtNETlxwjFYCIz1ELrESjjQWTR3McbkMRwkNIX",
    "ANhnHYw7JBbElMtGttj7HuEmYmLdBUsbiEVYtVh22o7RJqEsORGAk5Fh0lIz/v8Vl4ATDbiBJMD",
    "Hewf6lJB3cvo8LNs/veyTy78g3zbCpXfN6b07NWosebexb3vdyo37eB/61N5++c3NXCPhsFOYfb",
    "wvt9f/P11pt31z5/urf+KPhNr7L/h8bZ5v9mvr45bbSzwkkW8tVhn87ZtVGi/8ac/q18rUf24f3",
    "JyyiKk/GjMXs2+XfcaT3J+Vlib6Ng7/T9bdfxMcvS/FS0G+QHE7cmNhvXTv+fc/ZeXBKPy/n77P",
    "ry92ry17eLPJbFa/1UvNZd5Z7K2+/f//b2avLZ/i+X2H+nwPeddK73iOsW0KN/POhpzgIs9R6Vw",
    "ijJOaie6ge8UJ/fm+9nlfPOWe5XsfN/uR/P+7fMr/OW885ZblX/zjrHfzZ83TzwCvitn9iPVc9X",
    "Pxr/0TvbOehWvv5r9lOnGTIelJx68894hB2Uff0P5OlXoR6z4Q==",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c16_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c16_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2198758056U));
  ssSetChecksum1(S,(329747216U));
  ssSetChecksum2(S,(53008866U));
  ssSetChecksum3(S,(1930519609U));
}

static void mdlRTW_c16_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c16_sliding_controller_rbdl(SimStruct *S)
{
  SFc16_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc16_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc16_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc16_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc16_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c16_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c16_JITStateAnimation,
    chartInstance->c16_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c16_sliding_controller_rbdl(chartInstance);
}

void c16_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c16_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c16_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c16_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c16_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
