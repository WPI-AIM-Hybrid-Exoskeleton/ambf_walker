/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c20_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c20_do_animation_call_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_st);
static void sf_gateway_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c20_eML_blk_kernel(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2]);
static void c20_emlrt_marshallIn(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2]);
static void c20_b_emlrt_marshallIn(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2]);
static uint8_T c20_c_emlrt_marshallIn
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_b_is_active_c20_sliding_controller_rbdl, const char_T *c20_identifier);
static uint8_T c20_d_emlrt_marshallIn
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_u, const emlrtMsgIdentifier *c20_parentId);
static void init_dsm_address_info(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c20_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c20_is_active_c20_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c20_do_animation_call_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c20_b_y = NULL;
  const mxArray *c20_c_y = NULL;
  const mxArray *c20_d_y = NULL;
  const mxArray *c20_e_y = NULL;
  const mxArray *c20_f_y = NULL;
  const mxArray *c20_st;
  const mxArray *c20_y = NULL;
  c20_st = NULL;
  c20_y = NULL;
  sf_mex_assign(&c20_y, sf_mex_createcellmatrix(5, 1), false);
  c20_b_y = NULL;
  sf_mex_assign(&c20_b_y, sf_mex_create("y", *chartInstance->c20_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 0, c20_b_y);
  c20_c_y = NULL;
  sf_mex_assign(&c20_c_y, sf_mex_create("y", *chartInstance->c20_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 1, c20_c_y);
  c20_d_y = NULL;
  sf_mex_assign(&c20_d_y, sf_mex_create("y", *chartInstance->c20_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 2, c20_d_y);
  c20_e_y = NULL;
  sf_mex_assign(&c20_e_y, sf_mex_create("y", *chartInstance->c20_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c20_y, 3, c20_e_y);
  c20_f_y = NULL;
  sf_mex_assign(&c20_f_y, sf_mex_create("y",
    &chartInstance->c20_is_active_c20_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c20_y, 4, c20_f_y);
  sf_mex_assign(&c20_st, c20_y, false);
  return c20_st;
}

static void set_sim_state_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_st)
{
  const mxArray *c20_u;
  c20_u = sf_mex_dup(c20_st);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 0)),
                       "ankle", *chartInstance->c20_ankle);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 1)),
                       "danke", *chartInstance->c20_danke);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 2)),
                       "dknee", *chartInstance->c20_dknee);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 3)),
                       "knee", *chartInstance->c20_knee);
  chartInstance->c20_is_active_c20_sliding_controller_rbdl =
    c20_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 4)),
    "is_active_c20_sliding_controller_rbdl");
  sf_mex_destroy(&c20_u);
  sf_mex_destroy(&c20_st);
}

static void sf_gateway_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c20_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c20_eML_blk_kernel(chartInstance, *chartInstance->c20_q,
                     *chartInstance->c20_qd, *chartInstance->c20_knee,
                     *chartInstance->c20_dknee, *chartInstance->c20_ankle,
                     *chartInstance->c20_danke);
  c20_do_animation_call_c20_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c20_errCode;
  c20_errCode = hipGetLastError();
  if (c20_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c20_errCode, (char_T *)hipGetErrorName
                       (c20_errCode), (char_T *)hipGetErrorString(c20_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c20_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c20_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c20_nameCaptureInfo = NULL;
  const char_T *c20_data[4] = {
    "789ce593cb4ac34014864fa51637f5b2e94b088e75e5be2a8a5884163722f1243969c6cc452653dbbaf10df4297c171fcb246d7a09868862371e08277ffecc7c"
    "677e18a85d5cd500603b795a2f00e70dc8aa396db03beb1bb05a45bf56d2f3da84facabad4df4b786f33ed6965696ca742a1a4f94a5f4bae50d9fee491c050ac",
    "c513f9991370417d2ea9b72cbaa992674bd65ca456fade09c98b7a4309268c17138a6591e591d67dc979eb157914ab9847f1bf9c37fe212fdfbf55c1cbfd409b"
    "111adf89b8e2122df76247d0e0f7e7ceb98dd239a68eaf87aea005efe3973c2ce5adfab7a7772cd49298421ba2e224065af8011a97796893349c51cc62e33194",
    "6ee08c504464581290409729ed645f2dc5967d1de0819c71aa72dcf9e6b9caee5313b6b2fefeba8febe4e5f55f78ebba8f978713de7ee60f517474dce9b65dd7"
    "bbf1edc9628eeb0a4ed51c50a2ff7aff4fb0e86ac3", "" };

  c20_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c20_data[0], 1632U, &c20_nameCaptureInfo);
  return c20_nameCaptureInfo;
}

static void c20_eML_blk_kernel(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2])
{
  real_T c20_a;
  real_T c20_b_a;
  c20_b_knee[0] = -0.324 * muDoubleScalarSin(c20_b_q[0]);
  c20_b_knee[1] = -0.324 * muDoubleScalarCos(c20_b_q[0]);
  c20_a = -0.324 * c20_b_qd[0];
  c20_b_dknee[0] = c20_a * muDoubleScalarCos(c20_b_q[0]);
  c20_b_dknee[1] = c20_a * -muDoubleScalarSin(c20_b_q[0]);
  c20_b_a = -0.39 * (c20_b_qd[0] + c20_b_qd[1]);
  c20_b_ankle[0] = c20_b_knee[0] + -0.39 * muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_ankle[1] = c20_b_knee[1] + -0.39 * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[0] = c20_b_dknee[0] + c20_b_a * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[1] = c20_b_dknee[1] + c20_b_a * -muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
}

static void c20_emlrt_marshallIn(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2])
{
  emlrtMsgIdentifier c20_thisId;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c20_b_ankle), &c20_thisId,
    c20_y);
  sf_mex_destroy(&c20_b_ankle);
}

static void c20_b_emlrt_marshallIn(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2])
{
  real_T c20_dv[2];
  int32_T c20_i;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), c20_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c20_i = 0; c20_i < 2; c20_i++) {
    c20_y[c20_i] = c20_dv[c20_i];
  }

  sf_mex_destroy(&c20_u);
}

static uint8_T c20_c_emlrt_marshallIn
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_b_is_active_c20_sliding_controller_rbdl, const char_T *c20_identifier)
{
  emlrtMsgIdentifier c20_thisId;
  uint8_T c20_y;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_y = c20_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c20_b_is_active_c20_sliding_controller_rbdl), &c20_thisId);
  sf_mex_destroy(&c20_b_is_active_c20_sliding_controller_rbdl);
  return c20_y;
}

static uint8_T c20_d_emlrt_marshallIn
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_u, const emlrtMsgIdentifier *c20_parentId)
{
  uint8_T c20_b_u;
  uint8_T c20_y;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), &c20_b_u, 1, 3, 0U, 0, 0U, 0);
  c20_y = c20_b_u;
  sf_mex_destroy(&c20_u);
  return c20_y;
}

static void init_dsm_address_info(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c20_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c20_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c20_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c20_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c20_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c20_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c20_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c20_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c20_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c20_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6LFA4ifWRyfmFySWZYan"
    "2xkEF+ck5mSmZcen5yfV1KUn5OTWhRflJSSgxRPAKGELok="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c20_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c20_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c20_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c20_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c20_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c20_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c20_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c20_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc20_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c20_sliding_controller_rbdl
      ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc20_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c20_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c20_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c20_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c20_sliding_controller_rbdl
      ((SFc20_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc20_sliding_controller_rbdl
      ((SFc20_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c20_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA2cAEDvQRoUCBoi16MFgXaS2ObkhABduyUchr0IqyWI3Gh5S67P/TPO+S",
    "eSx+kr9An6BPkmjxCZ0lalimSiuwmdYvSoOQlv/l2vpnZ2ZXXGhx4eN3H+9WW593B77t4t7382i",
    "jGrYU7f97xvijGL9FI2PiIKBJrr/ESJIafQEtuDZNiICayEsbEBBQIithEKlPHpllsOROzvhXU8",
    "emfI0ajIJKWh3toS8JDwc+QLbHmCHm6TAE1fYDQREraadTnZDr3WJkTPwI60zZukqDBBDZxbukD",
    "yw1LOPROgQ6ENgQ91pe+BYYY8M1prUynVAcXQBknnBFRqTYiOoAEA2zgOAnx89AaFFWG0Ygoswc",
    "RSUHvs1nGKQWUOZnGF2MmiJGKEd6Lue8Ml3074ujPgQyBNwQEfdtTQGaJZMLU5z/oo9KeIGMOXR",
    "jbaT1bAL9al/znDE5A1cZt4ssUFJnCoaidNAtI7zTL1rxKlmGGxfCcqF2K+dMQ1lYvVo4OCOYJh",
    "mhRB4NM5EAPFUsxvLVsNh64yly1ZGycJ1uvgmVsvRSasjBn61PhE851LWwok31IgWesXWJIMyxn",
    "rcZpzcKhxAC78q5fDVYwTHwB86UIWWW60hIg6ztPsbFcRVKrjYx9LN7u/v7y62XYQBhQE0Khqgs",
    "owjRgzLLw1rOFTLvcIxC9Mpl7VeC8QlahPD2xonsi1Qxj0tBELiW4jNYCYz3FXOJKONa4aJpgLp",
    "ercJTQCELXYBiHA1w2iK2IiXatbRfXXcrMWRc0VSypyKrFVYdtqOcK6iyBYzET8kT0lYyDosfn4",
    "QXAaiNKMDHdw/6lzvo4e1Uq3H627V3uZx+9w352YVf+/nKBp1XB4y18l+e9176Kb5fmbeNfq5Xb",
    "7SzYfVyap1Oyc7hNvM9//xrePP4teh199cnWOfyxav7W0vyt7H9nd3djvf3/fjF+cNFo5wsnXap",
    "Xh32y4Fengv/TBf7NYqyfRd+J9Jcn0e4zMowH29+e/7Cdjv2M70Wn2d+Nkr8Xzx+6jo9VltWnoo",
    "OwOJi4MbH5du34v1/w986KeNwrnufX28c3s3+wU7bvrNC3WYyJmHEo1e8/p+Pza+sIUcjt0fHw+",
    "jpmeGi4NTq2dsr9oUpH54qOjuck/D3+L8dxPft8/hcr/H9UysOj7Lw4Im4XghH9ZnukOQtxCxlR",
    "KYySnIMaqXHIS3m6bh9d1877wHb/Fj//t3v/eX+Xc1H7mnadG57DPpTdTfWtez68bfim/cAr4Td",
    "vsY6bntvfN/5Pb73z9WfF+Mf5T2g/Yjys+DVVvMafRpOqt/+BOv0L+RjERQ==",
    ""
  };

  static char newstr [1337] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c20_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c20_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1697364346U));
  ssSetChecksum1(S,(1755135983U));
  ssSetChecksum2(S,(287926502U));
  ssSetChecksum3(S,(3395648020U));
}

static void mdlRTW_c20_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c20_sliding_controller_rbdl(SimStruct *S)
{
  SFc20_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc20_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc20_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc20_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc20_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c20_JITStateAnimation,
    chartInstance->c20_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c20_sliding_controller_rbdl(chartInstance);
}

void c20_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c20_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c20_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c20_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c20_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
