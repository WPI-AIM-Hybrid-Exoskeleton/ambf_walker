/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c20_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c20_do_animation_call_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_st);
static void sf_gateway_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c20_eML_blk_kernel(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2]);
static void c20_emlrt_marshallIn(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2]);
static void c20_b_emlrt_marshallIn(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2]);
static uint8_T c20_c_emlrt_marshallIn
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_b_is_active_c20_sliding_controller_rbdl, const char_T *c20_identifier);
static uint8_T c20_d_emlrt_marshallIn
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_u, const emlrtMsgIdentifier *c20_parentId);
static void init_dsm_address_info(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c20_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c20_is_active_c20_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c20_do_animation_call_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c20_b_y = NULL;
  const mxArray *c20_c_y = NULL;
  const mxArray *c20_d_y = NULL;
  const mxArray *c20_e_y = NULL;
  const mxArray *c20_f_y = NULL;
  const mxArray *c20_st;
  const mxArray *c20_y = NULL;
  c20_st = NULL;
  c20_y = NULL;
  sf_mex_assign(&c20_y, sf_mex_createcellmatrix(5, 1), false);
  c20_b_y = NULL;
  sf_mex_assign(&c20_b_y, sf_mex_create("y", *chartInstance->c20_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 0, c20_b_y);
  c20_c_y = NULL;
  sf_mex_assign(&c20_c_y, sf_mex_create("y", *chartInstance->c20_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 1, c20_c_y);
  c20_d_y = NULL;
  sf_mex_assign(&c20_d_y, sf_mex_create("y", *chartInstance->c20_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 2, c20_d_y);
  c20_e_y = NULL;
  sf_mex_assign(&c20_e_y, sf_mex_create("y", *chartInstance->c20_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c20_y, 3, c20_e_y);
  c20_f_y = NULL;
  sf_mex_assign(&c20_f_y, sf_mex_create("y",
    &chartInstance->c20_is_active_c20_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c20_y, 4, c20_f_y);
  sf_mex_assign(&c20_st, c20_y, false);
  return c20_st;
}

static void set_sim_state_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_st)
{
  const mxArray *c20_u;
  c20_u = sf_mex_dup(c20_st);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 0)),
                       "ankle", *chartInstance->c20_ankle);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 1)),
                       "danke", *chartInstance->c20_danke);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 2)),
                       "dknee", *chartInstance->c20_dknee);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 3)),
                       "knee", *chartInstance->c20_knee);
  chartInstance->c20_is_active_c20_sliding_controller_rbdl =
    c20_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 4)),
    "is_active_c20_sliding_controller_rbdl");
  sf_mex_destroy(&c20_u);
  sf_mex_destroy(&c20_st);
}

static void sf_gateway_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c20_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c20_eML_blk_kernel(chartInstance, *chartInstance->c20_q,
                     *chartInstance->c20_qd, *chartInstance->c20_knee,
                     *chartInstance->c20_dknee, *chartInstance->c20_ankle,
                     *chartInstance->c20_danke);
  c20_do_animation_call_c20_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c20_errCode;
  c20_errCode = hipGetLastError();
  if (c20_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c20_errCode, (char_T *)hipGetErrorName
                       (c20_errCode), (char_T *)hipGetErrorString(c20_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c20_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc20_sliding_controller_rbdl
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c20_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c20_nameCaptureInfo = NULL;
  const char_T *c20_data[4] = {
    "789cc593cb4ac34014864fa516375537fa188e2d08dd57c50b2d428b1b9138494e9a317391c9d4b66eba74e963e933b93297a697608858ac07c2c99f3f33df99"
    "1f062a979d0a00ec464f670a705183a4ea6983fd59df82d5cafb95829ed5365457d6c5fe55c47b9b6947498363930a4905ce57ba4a3049a5e94f9e1034868a3f",
    "a39b381ee3d867027bcba21b2b71be64cd456cc5ef6d1f9da03714a0fd7031215f16491e713d149cb75a9247bef279e4ffcb78e35ff2b2fd0f4b7899ef293da2"
    "dab5022699a08639a1c571b0feb9336ead708ed471d5d0e6b8e07dacc9a385bc55ffeeec9ef84a2091d4f85432e403c55d8f6a9b38d4446958a39084da2154d8",
    "9e35a23c404da28038b5895456f2d56068c8f7011e8919a72cc7bd1f9eabe83ed56127e9ad8313ba49deebe77b7393bcacfe8bb7a9fb787d3c618d17f61804cd"
    "56bbdbb06de7d635a78b396e4a3865734081feebfdbf0083b96cd1", "" };

  c20_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c20_data[0], 1632U, &c20_nameCaptureInfo);
  return c20_nameCaptureInfo;
}

static void c20_eML_blk_kernel(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2])
{
  real_T c20_a;
  real_T c20_b_a;
  c20_b_knee[0] = -0.324 * muDoubleScalarSin(c20_b_q[0]);
  c20_b_knee[1] = -0.324 * muDoubleScalarCos(c20_b_q[0]);
  c20_a = -0.324 * c20_b_qd[0];
  c20_b_dknee[0] = c20_a * muDoubleScalarCos(c20_b_q[0]);
  c20_b_dknee[1] = c20_a * -muDoubleScalarSin(c20_b_q[0]);
  c20_b_a = -0.39 * (c20_b_qd[0] + c20_b_qd[1]);
  c20_b_ankle[0] = c20_b_knee[0] + -0.39 * muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_ankle[1] = c20_b_knee[1] + -0.39 * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[0] = c20_b_dknee[0] + c20_b_a * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[1] = c20_b_dknee[1] + c20_b_a * -muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
}

static void c20_emlrt_marshallIn(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2])
{
  emlrtMsgIdentifier c20_thisId;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c20_b_ankle), &c20_thisId,
    c20_y);
  sf_mex_destroy(&c20_b_ankle);
}

static void c20_b_emlrt_marshallIn(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2])
{
  real_T c20_dv[2];
  int32_T c20_i;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), c20_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c20_i = 0; c20_i < 2; c20_i++) {
    c20_y[c20_i] = c20_dv[c20_i];
  }

  sf_mex_destroy(&c20_u);
}

static uint8_T c20_c_emlrt_marshallIn
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_b_is_active_c20_sliding_controller_rbdl, const char_T *c20_identifier)
{
  emlrtMsgIdentifier c20_thisId;
  uint8_T c20_y;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_y = c20_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c20_b_is_active_c20_sliding_controller_rbdl), &c20_thisId);
  sf_mex_destroy(&c20_b_is_active_c20_sliding_controller_rbdl);
  return c20_y;
}

static uint8_T c20_d_emlrt_marshallIn
  (SFc20_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c20_u, const emlrtMsgIdentifier *c20_parentId)
{
  uint8_T c20_b_u;
  uint8_T c20_y;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), &c20_b_u, 1, 3, 0U, 0, 0U, 0);
  c20_y = c20_b_u;
  sf_mex_destroy(&c20_u);
  return c20_y;
}

static void init_dsm_address_info(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc20_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c20_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c20_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c20_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c20_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c20_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c20_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c20_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c20_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c20_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c20_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6LFA4ifWRyfmFySWZYan"
    "2xkEF+ck5mSmZcen5yfV1KUn5OTWhRflJSSgxRPAKGELok="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c20_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c20_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c20_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c20_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c20_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c20_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c20_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c20_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc20_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c20_sliding_controller_rbdl
      ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc20_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c20_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c20_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc20_sliding_controller_rbdl
    ((SFc20_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c20_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c20_sliding_controller_rbdl
      ((SFc20_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc20_sliding_controller_rbdl
      ((SFc20_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c20_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFALwESFAiSQwsYLQq0KNDYpiREqF07pZwGuQir5UhcaLnL7A9t5x2",
    "KXPseRS59kfbcJ+gbtLMkLcsUSUV2k7pFaVDykt98O9/M7OzKaw32Pbxu4/3THc+7gd838W57+b",
    "VRjFsLd/684z0sxj+ikbDxIVEk1l7jJUgM34OW3BomxUBMZCWMiQkoEBSxiVSmjk2z2HImZn0rq",
    "OPTP0SMRkEkLQ930ZaEB4KfIltizSHydJkCavoAoYmUtNOoz8l07rEyx34EdKZt3CRBgwls4tzS",
    "+5YblnDonQAdCG0IeqzPfQsMMeCbk1qZTqkOzoAyTjgjolJtRHQACQbYwFES4ueBNSiqDKMRUWY",
    "XIpKC3mOzjFMKKHMyjS/GTBAjFSO8F3PfGS77dsjRn30ZAm8ICPq2q4DMEsmEqc9/0EelPUHGHL",
    "owttN6tgBeWpf8ZwyOQdXGbeLLFBSZwoGonTQLSO8ky9a8SpZhhsXwjKgdivnTENZWL1aODgjmC",
    "YZoUQeDTORADxVLMby1bDYeuMpctWRsnCdbr4JlbL0UmrIwZ+tT4RPOdS1sKJM9SIFnrF1iSDMs",
    "Z63Gac3CocQAu/KuXw1WMEx8AfOlCFllutISIOs732FjuYikVhsZ+1i83b295dfLsIEwoCaEQlU",
    "XUIRpwJhl4a1nC5l2uUcgemUy96rAeYWsQnl6YkX3WKoZxqShiZxLcBmtBcZ6irnElXCkcdE0wV",
    "wuV+EooRGErsEwDvu4bBBbERPtWtsOrruUmdMuaKpYUpFVi6sO21DPFdRpAkdiJuSx6CsZB0WPz",
    "8MLgNVGlGBiuov9S532cfaqVLj9bMs7388+eIv97Myu/P1ogadVweMtfJfnvdW+iG+X5m3jX6uV",
    "220v2H1YmqdTsnO4Tbxf/fzizbe//vL604e/f/J6+PWfq+ZvLc3fyv53djc31tv/bxfju2eNdr5",
    "w0qV6ddgnC351Kvg/WuDfLMb6afSFSF88iXaekmE82Pr81Vdb6djP+J53mv3dKPl79vy+6/hYZV",
    "l9KjoIi4OJGxObb9eO/8sFf2+siMet4nl+/fH4avZ3t8v2nRX6NosxETMOpfr953R8fGkdIQq5P",
    "jruX17HDA8N10bHne1yf6jS0bmgo+M5CX+P/8txXM8+n//5Cv8flPLwIDsvjojbhWBEP9saac5C",
    "3EJGVAqjJOegRmoc8lKeLttH17Xz3rPdv8XP/+3efd7f5lzUvqRd54rnsPdld1V9654Prxu+aT/",
    "wSvjNa6zjquf2d43/zVvvfH2vGH8z/wntR4yHFb+mitf402hS9fY/UKd/AcpexTA=",
    ""
  };

  static char newstr [1341] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c20_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c20_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042620282U));
  ssSetChecksum1(S,(2411386699U));
  ssSetChecksum2(S,(753018669U));
  ssSetChecksum3(S,(4282078351U));
}

static void mdlRTW_c20_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c20_sliding_controller_rbdl(SimStruct *S)
{
  SFc20_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc20_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc20_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc20_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc20_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c20_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c20_JITStateAnimation,
    chartInstance->c20_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c20_sliding_controller_rbdl(chartInstance);
}

void c20_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c20_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c20_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c20_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c20_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
