#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c23_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c23_do_animation_call_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_st);
static void sf_gateway_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c23_emlrt_marshallIn(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_b_msg, const char_T *c23_identifier,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_y);
static void c23_b_emlrt_marshallIn(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_y);
static void c23_c_emlrt_marshallIn(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId,
  uint8_T c23_y[8]);
static c23_SL_Bus_ROSVariableLengthArrayInfo c23_d_emlrt_marshallIn
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_u, const emlrtMsgIdentifier *c23_parentId);
static uint32_T c23_e_emlrt_marshallIn
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_u, const emlrtMsgIdentifier *c23_parentId);
static void c23_f_emlrt_marshallIn(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId,
  real_T c23_y[7]);
static uint8_T c23_g_emlrt_marshallIn
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_b_is_active_c23_sliding_controller_rbdl, const char_T *c23_identifier);
static uint8_T c23_h_emlrt_marshallIn
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_u, const emlrtMsgIdentifier *c23_parentId);
static void c23_eML_blk_kernel(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c23_b_blankMsg);
static __global__ void c23_sf_gateway_c23_sliding_controller_rbdl_kernel1
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c23_b_blankMsg,
   c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg);
static __global__ void c23_sf_gateway_c23_sliding_controller_rbdl_kernel2
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c23_b_blankMsg,
   c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg);
static __global__ void c23_sf_gateway_c23_sliding_controller_rbdl_kernel3
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg,
   c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_r);
static __global__ void c23_sf_gateway_c23_sliding_controller_rbdl_kernel4
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg,
   c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_r);
static __global__ void c23_sf_gateway_c23_sliding_controller_rbdl_kernel5(const
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_r,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg);
static __global__ void c23_sf_gateway_c23_sliding_controller_rbdl_kernel6(const
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_r,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg);
static __global__ void c23_eML_blk_kernel_kernel7
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c23_b_blankMsg);
static __global__ void c23_eML_blk_kernel_kernel8(const uint8_T c23_uv[5],
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c23_b_blankMsg);
static __global__ void c23_eML_blk_kernel_kernel9
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c23_b_blankMsg);
static void init_dsm_address_info(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c23_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c23_is_active_c23_sliding_controller_rbdl = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c23_gpu_uv, 5UL);
  hipMalloc(&chartInstance->c23_gpu_blankMsg, 208UL);
}

static void initialize_params_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c23_do_animation_call_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  static const char_T *c23_sv[8] = { "ModelName", "ModelName_SL_Info", "Q",
    "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  static const char_T *c23_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c23_b_y = NULL;
  const mxArray *c23_c_y = NULL;
  const mxArray *c23_d_y = NULL;
  const mxArray *c23_e_y = NULL;
  const mxArray *c23_f_y = NULL;
  const mxArray *c23_g_y = NULL;
  const mxArray *c23_h_y = NULL;
  const mxArray *c23_i_y = NULL;
  const mxArray *c23_j_y = NULL;
  const mxArray *c23_k_y = NULL;
  const mxArray *c23_l_y = NULL;
  const mxArray *c23_m_y = NULL;
  const mxArray *c23_n_y = NULL;
  const mxArray *c23_o_y = NULL;
  const mxArray *c23_p_y = NULL;
  const mxArray *c23_q_y = NULL;
  const mxArray *c23_r_y = NULL;
  const mxArray *c23_s_y = NULL;
  const mxArray *c23_st;
  const mxArray *c23_y = NULL;
  real_T c23_d_u[7];
  int32_T c23_i;
  int32_T c23_i1;
  int32_T c23_i2;
  int32_T c23_i3;
  uint32_T c23_b_u;
  uint32_T c23_c_u;
  uint32_T c23_e_u;
  uint32_T c23_f_u;
  uint32_T c23_g_u;
  uint32_T c23_h_u;
  uint32_T c23_i_u;
  uint32_T c23_j_u;
  uint32_T c23_u_CurrentLength;
  uint32_T c23_u_ReceivedLength;
  uint8_T c23_u[8];
  c23_st = NULL;
  c23_y = NULL;
  sf_mex_assign(&c23_y, sf_mex_createcellmatrix(2, 1), false);
  c23_b_y = NULL;
  sf_mex_assign(&c23_b_y, sf_mex_createstruct("structure", 8, c23_sv, 2, 1, 1),
                false);
  for (c23_i = 0; c23_i < 8; c23_i++) {
    c23_u[c23_i] = ((uint8_T *)&((char_T *)chartInstance->c23_msg)[0])[c23_i];
  }

  c23_c_y = NULL;
  sf_mex_assign(&c23_c_y, sf_mex_create("y", c23_u, 3, 0U, 1U, 0U, 1, 8), false);
  sf_mex_setfieldbynum(c23_b_y, 0, "ModelName", c23_c_y, 0);
  c23_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [8])[0];
  c23_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [8])[4];
  c23_d_y = NULL;
  sf_mex_assign(&c23_d_y, sf_mex_createstruct("structure", 2, c23_sv1, 2, 1, 1),
                false);
  c23_b_u = c23_u_CurrentLength;
  c23_e_y = NULL;
  sf_mex_assign(&c23_e_y, sf_mex_create("y", &c23_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_d_y, 0, "CurrentLength", c23_e_y, 0);
  c23_c_u = c23_u_ReceivedLength;
  c23_f_y = NULL;
  sf_mex_assign(&c23_f_y, sf_mex_create("y", &c23_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_d_y, 0, "ReceivedLength", c23_f_y, 1);
  sf_mex_setfieldbynum(c23_b_y, 0, "ModelName_SL_Info", c23_d_y, 1);
  for (c23_i1 = 0; c23_i1 < 7; c23_i1++) {
    c23_d_u[c23_i1] = ((real_T *)&((char_T *)chartInstance->c23_msg)[16])[c23_i1];
  }

  c23_g_y = NULL;
  sf_mex_assign(&c23_g_y, sf_mex_create("y", c23_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c23_b_y, 0, "Q", c23_g_y, 2);
  c23_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [72])[0];
  c23_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [72])[4];
  c23_h_y = NULL;
  sf_mex_assign(&c23_h_y, sf_mex_createstruct("structure", 2, c23_sv1, 2, 1, 1),
                false);
  c23_e_u = c23_u_CurrentLength;
  c23_i_y = NULL;
  sf_mex_assign(&c23_i_y, sf_mex_create("y", &c23_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_h_y, 0, "CurrentLength", c23_i_y, 0);
  c23_f_u = c23_u_ReceivedLength;
  c23_j_y = NULL;
  sf_mex_assign(&c23_j_y, sf_mex_create("y", &c23_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_h_y, 0, "ReceivedLength", c23_j_y, 1);
  sf_mex_setfieldbynum(c23_b_y, 0, "Q_SL_Info", c23_h_y, 3);
  for (c23_i2 = 0; c23_i2 < 7; c23_i2++) {
    c23_d_u[c23_i2] = ((real_T *)&((char_T *)chartInstance->c23_msg)[80])[c23_i2];
  }

  c23_k_y = NULL;
  sf_mex_assign(&c23_k_y, sf_mex_create("y", c23_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c23_b_y, 0, "Qd", c23_k_y, 4);
  c23_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [136])[0];
  c23_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [136])[4];
  c23_l_y = NULL;
  sf_mex_assign(&c23_l_y, sf_mex_createstruct("structure", 2, c23_sv1, 2, 1, 1),
                false);
  c23_g_u = c23_u_CurrentLength;
  c23_m_y = NULL;
  sf_mex_assign(&c23_m_y, sf_mex_create("y", &c23_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_l_y, 0, "CurrentLength", c23_m_y, 0);
  c23_h_u = c23_u_ReceivedLength;
  c23_n_y = NULL;
  sf_mex_assign(&c23_n_y, sf_mex_create("y", &c23_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_l_y, 0, "ReceivedLength", c23_n_y, 1);
  sf_mex_setfieldbynum(c23_b_y, 0, "Qd_SL_Info", c23_l_y, 5);
  for (c23_i3 = 0; c23_i3 < 7; c23_i3++) {
    c23_d_u[c23_i3] = ((real_T *)&((char_T *)chartInstance->c23_msg)[144])
      [c23_i3];
  }

  c23_o_y = NULL;
  sf_mex_assign(&c23_o_y, sf_mex_create("y", c23_d_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c23_b_y, 0, "Tau", c23_o_y, 6);
  c23_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [200])[0];
  c23_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [200])[4];
  c23_p_y = NULL;
  sf_mex_assign(&c23_p_y, sf_mex_createstruct("structure", 2, c23_sv1, 2, 1, 1),
                false);
  c23_i_u = c23_u_CurrentLength;
  c23_q_y = NULL;
  sf_mex_assign(&c23_q_y, sf_mex_create("y", &c23_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_p_y, 0, "CurrentLength", c23_q_y, 0);
  c23_j_u = c23_u_ReceivedLength;
  c23_r_y = NULL;
  sf_mex_assign(&c23_r_y, sf_mex_create("y", &c23_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c23_p_y, 0, "ReceivedLength", c23_r_y, 1);
  sf_mex_setfieldbynum(c23_b_y, 0, "Tau_SL_Info", c23_p_y, 7);
  sf_mex_setcell(c23_y, 0, c23_b_y);
  c23_s_y = NULL;
  sf_mex_assign(&c23_s_y, sf_mex_create("y",
    &chartInstance->c23_is_active_c23_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c23_y, 1, c23_s_y);
  sf_mex_assign(&c23_st, c23_y, false);
  return c23_st;
}

static void set_sim_state_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_st)
{
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 c23_r;
  const mxArray *c23_u;
  int32_T c23_i;
  int32_T c23_i1;
  c23_u = sf_mex_dup(c23_st);
  c23_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c23_u, 0)),
                       "msg", &c23_r);
  for (c23_i = 0; c23_i < 8; c23_i++) {
    ((uint8_T *)&((char_T *)chartInstance->c23_msg)[0])[c23_i] =
      c23_r.ModelName[c23_i];
  }

  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[0] = c23_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[4] = c23_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[72])[0] = c23_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[72])[4] = c23_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[136])[0] = c23_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[136])[4] = c23_r.Qd_SL_Info.ReceivedLength;
  for (c23_i1 = 0; c23_i1 < 7; c23_i1++) {
    ((real_T *)&((char_T *)chartInstance->c23_msg)[16])[c23_i1] = c23_r.Q[c23_i1];
    ((real_T *)&((char_T *)chartInstance->c23_msg)[80])[c23_i1] =
      c23_r.Qd[c23_i1];
    ((real_T *)&((char_T *)chartInstance->c23_msg)[144])[c23_i1] =
      c23_r.Tau[c23_i1];
  }

  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[200])[0] = c23_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[200])[4] = c23_r.Tau_SL_Info.ReceivedLength;
  chartInstance->c23_is_active_c23_sliding_controller_rbdl =
    c23_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c23_u, 1)),
    "is_active_c23_sliding_controller_rbdl");
  sf_mex_destroy(&c23_u);
  sf_mex_destroy(&c23_st);
}

static void sf_gateway_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 c23_r;
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
    *c23_b_gpu_blankMsg;
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_gpu_msg;
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_gpu_r;
  hipMalloc(&c23_gpu_r, 208UL);
  hipMalloc(&c23_gpu_msg, 208UL);
  hipMalloc(&c23_b_gpu_blankMsg, 208UL);
  chartInstance->c23_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(c23_b_gpu_blankMsg, chartInstance->c23_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 208UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c23_b_gpu_blankMsg, c23_gpu_msg);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c23_blankMsg, c23_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[0] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[8])[0];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[4] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[8])[4];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[72])[0] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[72])[0];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[72])[4] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[72])[4];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[136])[0] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[136])[0];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[136])[4] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[136])[4];
  hipMemcpy(c23_b_gpu_blankMsg, chartInstance->c23_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 208UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel2<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c23_b_gpu_blankMsg, c23_gpu_msg);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c23_blankMsg, c23_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_blankMsg)[200])[4];
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 208UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel3<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c23_gpu_msg, c23_gpu_r);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c23_r, c23_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c23_r.ModelName_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [8])[0];
  c23_r.ModelName_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [8])[4];
  c23_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [72])[0];
  c23_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [72])[4];
  c23_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [136])[0];
  c23_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [136])[4];
  hipMemcpy(c23_gpu_r, &c23_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 208UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel4<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c23_gpu_msg, c23_gpu_r);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c23_r, c23_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c23_r.Tau_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [200])[0];
  c23_r.Tau_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c23_msg)
    [200])[4];
  c23_eML_blk_kernel(chartInstance, &c23_r);
  hipMemcpy(c23_gpu_r, &c23_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 208UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel5<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c23_gpu_r, c23_gpu_msg);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[0] = c23_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[8])[4] = c23_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[72])[0] = c23_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[72])[4] = c23_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[136])[0] = c23_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[136])[4] = c23_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c23_gpu_msg, chartInstance->c23_msg, 208UL, hipMemcpyHostToDevice);
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel6<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c23_gpu_r, c23_gpu_msg);
  hipMemcpy(chartInstance->c23_msg, c23_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[200])[0] = c23_r.Tau_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c23_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c23_msg)[200])[4] = c23_r.Tau_SL_Info.ReceivedLength;
  c23_do_animation_call_c23_sliding_controller_rbdl(chartInstance);
  hipFree(c23_b_gpu_blankMsg);
  hipFree(c23_gpu_msg);
  hipFree(c23_gpu_r);
}

static void mdl_start_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c23_errCode;
  hipFree(*chartInstance->c23_gpu_uv);
  hipFree(chartInstance->c23_gpu_blankMsg);
  c23_errCode = hipGetLastError();
  if (c23_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c23_errCode, (char_T *)hipGetErrorName
                       (c23_errCode), (char_T *)hipGetErrorString(c23_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c23_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc23_sliding_controller_rbdl
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c23_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c23_nameCaptureInfo = NULL;
  c23_nameCaptureInfo = NULL;
  sf_mex_assign(&c23_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c23_nameCaptureInfo;
}

static void c23_emlrt_marshallIn(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_b_msg, const char_T *c23_identifier,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_y)
{
  emlrtMsgIdentifier c23_thisId;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c23_b_msg), &c23_thisId,
    c23_y);
  sf_mex_destroy(&c23_b_msg);
}

static void c23_b_emlrt_marshallIn(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_y)
{
  static const char_T *c23_fieldNames[8] = { "ModelName", "ModelName_SL_Info",
    "Q", "Q_SL_Info", "Qd", "Qd_SL_Info", "Tau", "Tau_SL_Info" };

  emlrtMsgIdentifier c23_thisId;
  c23_thisId.fParent = c23_parentId;
  c23_thisId.bParentIsCell = false;
  sf_mex_check_struct(c23_parentId, c23_u, 8, c23_fieldNames, 0U, NULL);
  c23_thisId.fIdentifier = "ModelName";
  c23_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c23_u,
    "ModelName", "ModelName", 0)), &c23_thisId, c23_y->ModelName);
  c23_thisId.fIdentifier = "ModelName_SL_Info";
  c23_y->ModelName_SL_Info = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "ModelName_SL_Info", "ModelName_SL_Info", 0)),
    &c23_thisId);
  c23_thisId.fIdentifier = "Q";
  c23_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c23_u, "Q",
    "Q", 0)), &c23_thisId, c23_y->Q);
  c23_thisId.fIdentifier = "Q_SL_Info";
  c23_y->Q_SL_Info = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "Q_SL_Info", "Q_SL_Info", 0)), &c23_thisId);
  c23_thisId.fIdentifier = "Qd";
  c23_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c23_u, "Qd",
    "Qd", 0)), &c23_thisId, c23_y->Qd);
  c23_thisId.fIdentifier = "Qd_SL_Info";
  c23_y->Qd_SL_Info = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c23_thisId);
  c23_thisId.fIdentifier = "Tau";
  c23_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c23_u, "Tau",
    "Tau", 0)), &c23_thisId, c23_y->Tau);
  c23_thisId.fIdentifier = "Tau_SL_Info";
  c23_y->Tau_SL_Info = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "Tau_SL_Info", "Tau_SL_Info", 0)), &c23_thisId);
  sf_mex_destroy(&c23_u);
}

static void c23_c_emlrt_marshallIn(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId,
  uint8_T c23_y[8])
{
  int32_T c23_i;
  uint8_T c23_uv[8];
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), c23_uv, 1, 3, 0U, 1, 0U, 1, 8);
  for (c23_i = 0; c23_i < 8; c23_i++) {
    c23_y[c23_i] = c23_uv[c23_i];
  }

  sf_mex_destroy(&c23_u);
}

static c23_SL_Bus_ROSVariableLengthArrayInfo c23_d_emlrt_marshallIn
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_u, const emlrtMsgIdentifier *c23_parentId)
{
  static const char_T *c23_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c23_SL_Bus_ROSVariableLengthArrayInfo c23_y;
  emlrtMsgIdentifier c23_thisId;
  c23_thisId.fParent = c23_parentId;
  c23_thisId.bParentIsCell = false;
  sf_mex_check_struct(c23_parentId, c23_u, 2, c23_fieldNames, 0U, NULL);
  c23_thisId.fIdentifier = "CurrentLength";
  c23_y.CurrentLength = c23_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "CurrentLength", "CurrentLength", 0)), &c23_thisId);
  c23_thisId.fIdentifier = "ReceivedLength";
  c23_y.ReceivedLength = c23_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c23_u, "ReceivedLength", "ReceivedLength", 0)), &c23_thisId);
  sf_mex_destroy(&c23_u);
  return c23_y;
}

static uint32_T c23_e_emlrt_marshallIn
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_u, const emlrtMsgIdentifier *c23_parentId)
{
  uint32_T c23_b_u;
  uint32_T c23_y;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), &c23_b_u, 1, 7, 0U, 0, 0U, 0);
  c23_y = c23_b_u;
  sf_mex_destroy(&c23_u);
  return c23_y;
}

static void c23_f_emlrt_marshallIn(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c23_u, const emlrtMsgIdentifier *c23_parentId,
  real_T c23_y[7])
{
  real_T c23_dv[7];
  int32_T c23_i;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), c23_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c23_i = 0; c23_i < 7; c23_i++) {
    c23_y[c23_i] = c23_dv[c23_i];
  }

  sf_mex_destroy(&c23_u);
}

static uint8_T c23_g_emlrt_marshallIn
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_b_is_active_c23_sliding_controller_rbdl, const char_T *c23_identifier)
{
  emlrtMsgIdentifier c23_thisId;
  uint8_T c23_y;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_y = c23_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c23_b_is_active_c23_sliding_controller_rbdl), &c23_thisId);
  sf_mex_destroy(&c23_b_is_active_c23_sliding_controller_rbdl);
  return c23_y;
}

static uint8_T c23_h_emlrt_marshallIn
  (SFc23_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c23_u, const emlrtMsgIdentifier *c23_parentId)
{
  uint8_T c23_b_u;
  uint8_T c23_y;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_u), &c23_b_u, 1, 3, 0U, 0, 0U, 0);
  c23_y = c23_b_u;
  sf_mex_destroy(&c23_u);
  return c23_y;
}

static void c23_eML_blk_kernel(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c23_b_blankMsg)
{
  static uint8_T c23_uv[5] = { 104U, 117U, 109U, 97U, 110U };

  hipMemcpy(chartInstance->c23_gpu_blankMsg, c23_b_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  c23_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c23_gpu_blankMsg);
  hipMemcpy(*chartInstance->c23_gpu_uv, c23_uv, 5UL, hipMemcpyHostToDevice);
  c23_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c23_gpu_uv, chartInstance->c23_gpu_blankMsg);
  c23_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c23_gpu_blankMsg);
  hipMemcpy(c23_b_blankMsg, chartInstance->c23_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel1
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c23_b_blankMsg,
   c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg)
{
  uint64_T c23_threadId;
  int32_T c23_i;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i = (int32_T)c23_threadId;
  if (c23_i < 8) {
    c23_b_msg->ModelName[c23_i] = c23_b_blankMsg->ModelName[c23_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel2
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c23_b_blankMsg,
   c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg)
{
  uint64_T c23_threadId;
  int32_T c23_i1;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i1 = (int32_T)c23_threadId;
  if (c23_i1 < 7) {
    c23_b_msg->Q[c23_i1] = c23_b_blankMsg->Q[c23_i1];
    c23_b_msg->Qd[c23_i1] = c23_b_blankMsg->Qd[c23_i1];
    c23_b_msg->Tau[c23_i1] = c23_b_blankMsg->Tau[c23_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel3
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg,
   c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_r)
{
  uint64_T c23_threadId;
  int32_T c23_i2;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i2 = (int32_T)c23_threadId;
  if (c23_i2 < 8) {
    c23_r->ModelName[c23_i2] = c23_b_msg->ModelName[c23_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel4
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg,
   c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_r)
{
  uint64_T c23_threadId;
  int32_T c23_i3;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i3 = (int32_T)c23_threadId;
  if (c23_i3 < 7) {
    c23_r->Q[c23_i3] = c23_b_msg->Q[c23_i3];
    c23_r->Qd[c23_i3] = c23_b_msg->Qd[c23_i3];
    c23_r->Tau[c23_i3] = c23_b_msg->Tau[c23_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel5(const
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_r,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg)
{
  uint64_T c23_threadId;
  int32_T c23_i4;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i4 = (int32_T)c23_threadId;
  if (c23_i4 < 8) {
    c23_b_msg->ModelName[c23_i4] = c23_r->ModelName[c23_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c23_sf_gateway_c23_sliding_controller_rbdl_kernel6(const
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_r,
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *c23_b_msg)
{
  uint64_T c23_threadId;
  int32_T c23_i5;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i5 = (int32_T)c23_threadId;
  if (c23_i5 < 7) {
    c23_b_msg->Q[c23_i5] = c23_r->Q[c23_i5];
    c23_b_msg->Qd[c23_i5] = c23_r->Qd[c23_i5];
    c23_b_msg->Tau[c23_i5] = c23_r->Tau[c23_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c23_eML_blk_kernel_kernel7
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c23_b_blankMsg)
{
  uint64_T c23_threadId;
  int32_T c23_tmpIdx;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_tmpIdx = (int32_T)c23_threadId;
  if (c23_tmpIdx < 1) {
    c23_b_blankMsg->ModelName_SL_Info.CurrentLength = 5U;
  }
}

static __global__ __launch_bounds__(32, 1) void c23_eML_blk_kernel_kernel8(const
  uint8_T c23_uv[5],
  c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
  *c23_b_blankMsg)
{
  uint64_T c23_threadId;
  int32_T c23_i;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_i = (int32_T)c23_threadId;
  if (c23_i < 5) {
    c23_b_blankMsg->ModelName[c23_i] = c23_uv[c23_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c23_eML_blk_kernel_kernel9
  (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57
   *c23_b_blankMsg)
{
  uint64_T c23_threadId;
  int32_T c23_tmpIdx;
  c23_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c23_tmpIdx = (int32_T)c23_threadId;
  if (c23_tmpIdx < 1) {
    c23_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
    c23_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
    c23_b_blankMsg->Tau_SL_Info.CurrentLength = 7U;
  }
}

static void init_dsm_address_info(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc23_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c23_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c23_blankMsg =
    (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c23_msg =
    (c23_SL_Bus_sliding_controller_rbdl_RBDLForwardDynamicsReques_u57 *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c23_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1446757649U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1328543342U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3523006734U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2043636117U);
}

mxArray *sf_c23_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c23_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c23_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c23_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvyqK+yH8zOL4xOSSzLLU+GQj4/jinMyUzLz0+OT8vJKi/Jyc1KL4oqSUHFj8AwEAPToeg"
    "g=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c23_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sCgaqjKANzQMOTrp5zbdh0C";
}

static void sf_opaque_initialize_c23_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c23_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c23_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c23_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c23_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c23_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c23_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc23_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c23_sliding_controller_rbdl
      ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc23_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c23_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c23_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc23_sliding_controller_rbdl
    ((SFc23_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c23_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c23_sliding_controller_rbdl
      ((SFc23_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc23_sliding_controller_rbdl
      ((SFc23_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c23_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpmU3SIA0MIICyaFAcwnaY9AiQE+tbUpChVqxW8ppkYuwWo7EjZa7zP7Ilt8hr5B",
    "nyDMEeYggpyBPEKDovbMkJcsUScURYiRBCdD0kt/Mzsw3Pytvo9P18LqB985Nz7uCz6t4N7zs+i",
    "pfbyzc2fst7/t8/RSFhI0PiSKx9movQWL4E7Tk1jApOmIoS2FMDEGBoIhNpDJV2jSLLWdi3LaCO",
    "n36r4jRKIik5eEeypLwQPApakusOUQ9TaaAmjZAaCIl7ShqczKaW6zMsR8BHWsb17mgwQQ2cWbp",
    "ruWGJRxaJ0A7QhuCFusz2wJDDPjmpNJN56kOZkAZJ5wRUeptRHQACQbYwFES4t8Da9CpIoxGRJk",
    "9iMgE9D4bpzqlgKJOpvHDgAlipGKEt2LuO8Fl2w452tOVIfCagKBtewrIOJFMmGr+gzZ62hJkwK",
    "EJAzuq1hbAE+vIf8jgGFRl3Ia+nIAiIzgQlZumAWmdpGzNs2QZZlgMD4napcifhrAyezFzdECQJ",
    "+ihRBUMUic7uqfYBMNbqc3GHZeZq0rGxhnZehUs1daaQB0Lc21tKnzCua6E9WSyDxPgqdYmMaQe",
    "lmktx2nNwp7EALv0rq4GKxgSn8N8KUJWStekAEj7zgNsLOeR1GojYx+Tt7m/v/x5GdYRBtSQUCj",
    "rAoowDRizNLzV2kKmHfcIRKtMal4ZOMuQVShPD61oHks1xpjUNJEzFxyjlcBYj5BLrIQjjUVTB3",
    "NcrsJRQiMIXYNhHLpYNogtiYl2rW0X627CzLQJmiqWlLBqseqwDbVcQk0TOBJjIY9FW8k4yHt8F",
    "l4AzDaiBBOjPexfatrG3cuocPPsnnc2z66/xzybyRWfPyzo2SjR4y08i/tea9Tv28D/Nmbzd0Hu",
    "68I+WwU5h9vG+9G/b1695M9ubd1+fXP64u0/6+z/vHGx+X8jX387a7Tzwpks5avD/rZg11aJ/ls",
    "L+rfztfZH5Mnj33cfnP7RPeip5P7pIIzu+Rk/K+xtFOydvb/jOj5mWZqfinbC/GDi1sRm49rp/3",
    "nB3isr4nEtf59d735dT/6bnSKPZfHaPBevTVe55/L2w/e/s7OefLb/3yvsv1vg+2461/vEdQvo0",
    "x9/6mvOQiz1PpXCKMk5qL4ahLxQnx+a7xeV8y5Z7nOx83+5j8/7+8yvy5bzLlluXf8uOsc/NXzd",
    "PPAK+O1P2I91z1cfG//Ku9g56Lt8/cv8p44fMR6WnHrzz3iEHZZ9/QLy9D/NDba6",
    ""
  };

  static char newstr [1265] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c23_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c23_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3638621530U));
  ssSetChecksum1(S,(396258500U));
  ssSetChecksum2(S,(316282884U));
  ssSetChecksum3(S,(4091724409U));
}

static void mdlRTW_c23_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c23_sliding_controller_rbdl(SimStruct *S)
{
  SFc23_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc23_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc23_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc23_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc23_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c23_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c23_JITStateAnimation,
    chartInstance->c23_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c23_sliding_controller_rbdl(chartInstance);
}

void c23_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c23_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c23_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c23_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c23_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
