/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c11_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c11_do_animation_call_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c11_st);
static void sf_gateway_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c11_eML_blk_kernel(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c11_b_q[2], real_T c11_b_y[4]);
static void c11_emlrt_marshallIn(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c11_b_y, const char_T *c11_identifier, real_T
  c11_c_y[4]);
static void c11_b_emlrt_marshallIn(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId,
  real_T c11_b_y[4]);
static uint8_T c11_c_emlrt_marshallIn
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c11_b_is_active_c11_sliding_controller_rbdl, const char_T *c11_identifier);
static uint8_T c11_d_emlrt_marshallIn
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c11_u, const emlrtMsgIdentifier *c11_parentId);
static void init_dsm_address_info(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c11_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c11_is_active_c11_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c11_do_animation_call_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c11_b_y = NULL;
  const mxArray *c11_c_y = NULL;
  const mxArray *c11_d_y = NULL;
  const mxArray *c11_st;
  c11_st = NULL;
  c11_b_y = NULL;
  sf_mex_assign(&c11_b_y, sf_mex_createcellmatrix(2, 1), false);
  c11_c_y = NULL;
  sf_mex_assign(&c11_c_y, sf_mex_create("y", *chartInstance->c11_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c11_b_y, 0, c11_c_y);
  c11_d_y = NULL;
  sf_mex_assign(&c11_d_y, sf_mex_create("y",
    &chartInstance->c11_is_active_c11_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c11_b_y, 1, c11_d_y);
  sf_mex_assign(&c11_st, c11_b_y, false);
  return c11_st;
}

static void set_sim_state_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c11_st)
{
  const mxArray *c11_u;
  c11_u = sf_mex_dup(c11_st);
  c11_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c11_u, 0)), "y",
                       *chartInstance->c11_y);
  chartInstance->c11_is_active_c11_sliding_controller_rbdl =
    c11_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c11_u, 1)),
    "is_active_c11_sliding_controller_rbdl");
  sf_mex_destroy(&c11_u);
  sf_mex_destroy(&c11_st);
}

static void sf_gateway_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c11_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c11_eML_blk_kernel(chartInstance, *chartInstance->c11_q, *chartInstance->c11_y);
  c11_do_animation_call_c11_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c11_errCode;
  c11_errCode = hipGetLastError();
  if (c11_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c11_errCode, (char_T *)hipGetErrorName
                       (c11_errCode), (char_T *)hipGetErrorString(c11_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c11_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc11_sliding_controller_rbdl
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c11_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c11_nameCaptureInfo = NULL;
  c11_nameCaptureInfo = NULL;
  sf_mex_assign(&c11_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c11_nameCaptureInfo;
}

static void c11_eML_blk_kernel(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c11_b_q[2], real_T c11_b_y[4])
{
  c11_b_y[0] = -0.324 * muDoubleScalarCos(c11_b_q[0]) + -0.39 *
    muDoubleScalarCos(c11_b_q[0] + c11_b_q[1]);
  c11_b_y[2] = 0.0;
  c11_b_y[1] = 0.324 * muDoubleScalarSin(c11_b_q[0]) - -0.39 * muDoubleScalarSin
    (c11_b_q[0] + c11_b_q[1]);
  c11_b_y[3] = 0.0;
}

static void c11_emlrt_marshallIn(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c11_b_y, const char_T *c11_identifier, real_T
  c11_c_y[4])
{
  emlrtMsgIdentifier c11_thisId;
  c11_thisId.fIdentifier = const_cast<const char_T *>(c11_identifier);
  c11_thisId.fParent = NULL;
  c11_thisId.bParentIsCell = false;
  c11_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c11_b_y), &c11_thisId,
    c11_c_y);
  sf_mex_destroy(&c11_b_y);
}

static void c11_b_emlrt_marshallIn(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId,
  real_T c11_b_y[4])
{
  real_T c11_dv[4];
  int32_T c11_i;
  sf_mex_import(c11_parentId, sf_mex_dup(c11_u), c11_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c11_i = 0; c11_i < 4; c11_i++) {
    c11_b_y[c11_i] = c11_dv[c11_i];
  }

  sf_mex_destroy(&c11_u);
}

static uint8_T c11_c_emlrt_marshallIn
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c11_b_is_active_c11_sliding_controller_rbdl, const char_T *c11_identifier)
{
  emlrtMsgIdentifier c11_thisId;
  uint8_T c11_b_y;
  c11_thisId.fIdentifier = const_cast<const char_T *>(c11_identifier);
  c11_thisId.fParent = NULL;
  c11_thisId.bParentIsCell = false;
  c11_b_y = c11_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c11_b_is_active_c11_sliding_controller_rbdl), &c11_thisId);
  sf_mex_destroy(&c11_b_is_active_c11_sliding_controller_rbdl);
  return c11_b_y;
}

static uint8_T c11_d_emlrt_marshallIn
  (SFc11_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c11_u, const emlrtMsgIdentifier *c11_parentId)
{
  uint8_T c11_b_u;
  uint8_T c11_b_y;
  sf_mex_import(c11_parentId, sf_mex_dup(c11_u), &c11_b_u, 1, 3, 0U, 0, 0U, 0);
  c11_b_y = c11_b_u;
  sf_mex_destroy(&c11_u);
  return c11_b_y;
}

static void init_dsm_address_info(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc11_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c11_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c11_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c11_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c11_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2645569171U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4271796742U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(4000730253U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3632441041U);
}

mxArray *sf_c11_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c11_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c11_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c11_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cmGhvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB1sx2t"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c11_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sgWZraubfEkzgYQvNqDcgxB";
}

static void sf_opaque_initialize_c11_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c11_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c11_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c11_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c11_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c11_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c11_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc11_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c11_sliding_controller_rbdl
      ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc11_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c11_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c11_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc11_sliding_controller_rbdl
    ((SFc11_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c11_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c11_sliding_controller_rbdl
      ((SFc11_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc11_sliding_controller_rbdl
      ((SFc11_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c11_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u1EgQdkYBLRKgaLXSXkDLBcERbnBhw/yxIyUkrJPA7mXU066xW9PuNv3jZLiv9rSHfRO",
    "OvAZIIB5jH4Fq2zMZPLaHMCICtJYcT9tfVVfVVz8db2Ow6+F1Fe/tHz3vIj5/wLvl5deFYr2xcO",
    "fvN71bxfofFBI23ieKxNprvASJ4XfQklvDpBiIsayEMTEGBYIiNpHK1GnTLLaciUnfCur06acRo",
    "5EfScuDNsqSYE/wKWpLrNlHPV2mgJo+QGAiJW0Y9TkJ5xYrc9yJgE60jZtc0GB8mziz9K7lhiUc",
    "eidAB0IbghbrU9t8Qwx0zEmtm85T7c+AMk44I6LS24hoHxIMsIHDJMC/e9agU2UYjYgybYhICnq",
    "HTTKdUkBZJ9P4YcQEMVIxwnsx7zjBZdv2OdqzKwPgDQFB29oKyCSRTJh6/v0+etoTZMShCyMb1m",
    "vz4bl15B8xOAZVG7dxR6agSAh7onbTLCC9k4yteZYswwyL4YiohxT50xDUZi9mjvYJ8gQHKFEHg",
    "8zJgT5QLMXw1mqz8cBl5qqSsXFOtl4Fy7T1UmhiYa6tT0WHcK5rYQcy2YEUeKa1SwxphuVaq3Fa",
    "s+BAYoBdetdXgxUMiS9gHSkCVklXWgJkfecxNpaPkdRqI+MOJm93Z2f58zJsIAyoMaFQ1QUUYRo",
    "wZll467UFTDvuEYhWmcy8KnCeIatQnh5b0T2WaoIxaWgipy44RmuBsQ6RS6yEQ41F0wRzXK7CUU",
    "IjCFyDYRx2sWwQWxET7VrbQ6y7lJlpFzRVLKlg1WLVYRvquYSaJnAoJkIei76SsV/0+Dy8AJhtR",
    "Akmwjb2LzXt4+5VVLh5dsc7nWeXP2GezeTKz9sLejYq9HgLz/K+l1rN+7bw18Zs/i7IXSnts1mS",
    "c7gtvP/+96/rr9+lcP9RfO/V0ds36+z/snW2+X+1WF+bNdp54aRL+eqwvy3YtVmh/+cF/VvFWod",
    "P/1TEjsa9yYvwjyfp4+ddGp60c35W2Nsq2Tt7f8N1fMyyLD8VHQTFwcStic3HtdN/b8Heiyvica",
    "l4n1///bqe/E/bZR43V8hv4a9pKW8/f/8b2+vJ5/s/W2H/zRLfN7O5PiSuW8CQ3r071JwFWOpDK",
    "oVRknNQQzUKeMnPz833s8p55yz3rdj5v9yX5/1T5td5y3nnLLeuf2ed418bvmkeeCX81lfsx7rn",
    "qy+Nf++d7Rz0S7F+MP9XpxMxHlSceovPeIQdV339DvL0A6q+tao=",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c11_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c11_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(495292807U));
  ssSetChecksum1(S,(1702287057U));
  ssSetChecksum2(S,(946685753U));
  ssSetChecksum3(S,(3537196728U));
}

static void mdlRTW_c11_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c11_sliding_controller_rbdl(SimStruct *S)
{
  SFc11_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc11_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc11_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc11_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc11_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c11_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c11_JITStateAnimation,
    chartInstance->c11_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c11_sliding_controller_rbdl(chartInstance);
}

void c11_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c11_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c11_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c11_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c11_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
