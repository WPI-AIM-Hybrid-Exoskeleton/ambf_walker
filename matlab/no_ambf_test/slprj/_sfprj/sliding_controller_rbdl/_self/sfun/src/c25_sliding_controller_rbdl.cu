/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c25_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c25_do_animation_call_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c25_st);
static void sf_gateway_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c25_eML_blk_kernel(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c25_b_u[6], real_T c25_b_y[6]);
static void c25_emlrt_marshallIn(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c25_b_y, const char_T *c25_identifier, real_T
  c25_c_y[6]);
static void c25_b_emlrt_marshallIn(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c25_b_u, const emlrtMsgIdentifier *c25_parentId,
  real_T c25_b_y[6]);
static uint8_T c25_c_emlrt_marshallIn
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c25_b_is_active_c25_sliding_controller_rbdl, const char_T *c25_identifier);
static uint8_T c25_d_emlrt_marshallIn
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c25_b_u, const emlrtMsgIdentifier *c25_parentId);
static void init_dsm_address_info(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c25_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c25_is_active_c25_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c25_do_animation_call_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c25_b_y = NULL;
  const mxArray *c25_c_y = NULL;
  const mxArray *c25_d_y = NULL;
  const mxArray *c25_st;
  c25_st = NULL;
  c25_b_y = NULL;
  sf_mex_assign(&c25_b_y, sf_mex_createcellmatrix(2, 1), false);
  c25_c_y = NULL;
  sf_mex_assign(&c25_c_y, sf_mex_create("y", *chartInstance->c25_y, 0, 0U, 1U,
    0U, 2, 1, 6), false);
  sf_mex_setcell(c25_b_y, 0, c25_c_y);
  c25_d_y = NULL;
  sf_mex_assign(&c25_d_y, sf_mex_create("y",
    &chartInstance->c25_is_active_c25_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c25_b_y, 1, c25_d_y);
  sf_mex_assign(&c25_st, c25_b_y, false);
  return c25_st;
}

static void set_sim_state_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c25_st)
{
  const mxArray *c25_b_u;
  c25_b_u = sf_mex_dup(c25_st);
  c25_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c25_b_u, 0)),
                       "y", *chartInstance->c25_y);
  chartInstance->c25_is_active_c25_sliding_controller_rbdl =
    c25_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c25_b_u, 1)),
    "is_active_c25_sliding_controller_rbdl");
  sf_mex_destroy(&c25_b_u);
  sf_mex_destroy(&c25_st);
}

static void sf_gateway_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c25_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c25_eML_blk_kernel(chartInstance, *chartInstance->c25_u, *chartInstance->c25_y);
  c25_do_animation_call_c25_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c25_errCode;
  c25_errCode = hipGetLastError();
  if (c25_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c25_errCode, (char_T *)hipGetErrorName
                       (c25_errCode), (char_T *)hipGetErrorString(c25_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c25_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc25_sliding_controller_rbdl
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c25_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c25_nameCaptureInfo = NULL;
  c25_nameCaptureInfo = NULL;
  sf_mex_assign(&c25_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c25_nameCaptureInfo;
}

static void c25_eML_blk_kernel(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c25_b_u[6], real_T c25_b_y[6])
{
  c25_b_y[0] = c25_b_u[3];
  c25_b_y[1] = c25_b_u[0];
  c25_b_y[2] = c25_b_u[1];
  c25_b_y[3] = c25_b_u[2];
  c25_b_y[4] = c25_b_u[4];
  c25_b_y[5] = c25_b_u[5];
}

static void c25_emlrt_marshallIn(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c25_b_y, const char_T *c25_identifier, real_T
  c25_c_y[6])
{
  emlrtMsgIdentifier c25_thisId;
  c25_thisId.fIdentifier = const_cast<const char_T *>(c25_identifier);
  c25_thisId.fParent = NULL;
  c25_thisId.bParentIsCell = false;
  c25_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c25_b_y), &c25_thisId,
    c25_c_y);
  sf_mex_destroy(&c25_b_y);
}

static void c25_b_emlrt_marshallIn(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c25_b_u, const emlrtMsgIdentifier *c25_parentId,
  real_T c25_b_y[6])
{
  real_T c25_dv[6];
  int32_T c25_i;
  sf_mex_import(c25_parentId, sf_mex_dup(c25_b_u), c25_dv, 1, 0, 0U, 1, 0U, 2, 1,
                6);
  for (c25_i = 0; c25_i < 6; c25_i++) {
    c25_b_y[c25_i] = c25_dv[c25_i];
  }

  sf_mex_destroy(&c25_b_u);
}

static uint8_T c25_c_emlrt_marshallIn
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c25_b_is_active_c25_sliding_controller_rbdl, const char_T *c25_identifier)
{
  emlrtMsgIdentifier c25_thisId;
  uint8_T c25_b_y;
  c25_thisId.fIdentifier = const_cast<const char_T *>(c25_identifier);
  c25_thisId.fParent = NULL;
  c25_thisId.bParentIsCell = false;
  c25_b_y = c25_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c25_b_is_active_c25_sliding_controller_rbdl), &c25_thisId);
  sf_mex_destroy(&c25_b_is_active_c25_sliding_controller_rbdl);
  return c25_b_y;
}

static uint8_T c25_d_emlrt_marshallIn
  (SFc25_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c25_b_u, const emlrtMsgIdentifier *c25_parentId)
{
  uint8_T c25_b_y;
  uint8_T c25_c_u;
  sf_mex_import(c25_parentId, sf_mex_dup(c25_b_u), &c25_c_u, 1, 3, 0U, 0, 0U, 0);
  c25_b_y = c25_c_u;
  sf_mex_destroy(&c25_b_u);
  return c25_b_y;
}

static void init_dsm_address_info(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc25_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c25_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c25_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c25_y = (real_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c25_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(355923390U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3254592024U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(515668378U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(305817613U);
}

mxArray *sf_c25_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c25_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c25_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c25_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8clGpvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB2aB2y"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c25_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sDt8ga6s3SmK65qEWTBZYi";
}

static void sf_opaque_initialize_c25_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c25_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c25_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c25_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c25_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c25_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c25_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc25_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c25_sliding_controller_rbdl
      ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc25_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c25_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c25_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc25_sliding_controller_rbdl
    ((SFc25_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c25_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c25_sliding_controller_rbdl
      ((SFc25_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc25_sliding_controller_rbdl
      ((SFc25_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c25_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgXXqJEmNYLApwBJDkHSW5HARk6toz9EiBUbpWynuQjr5UhcaLnL7I9sPUSepb3",
    "1JfoAfYw8QmZJSlYokoot2HCLEqCpJb+ZnZlvftbeWqfr4XUP7737nreBz+/xrnnp9V22Xpu70/",
    "fr3rNs/QmFhI0OiSKR9iovQSL4DbTk1jApOmIgC2FMDECBoIiNpTJl2jSLLGdi1LaCOn36JGQ09",
    "ENpeVBHWRIcCD5BbbE1h6inyRRQ0wYITKikHYZtToYzi5U5a4RAR9pGVS5oML6NnVm6a7lhMYfW",
    "OdCO0IagxfrCNt8QAw1zXuqm81T7U6CMYs6IKPQ2JNqHGANs4CgO8O+BNehUHkZDokwdQjIGvc9",
    "GiU4pIK+TafxwygQxUjHCWxFvOMFF2w452tOVAfCKgKBtdQVkFEsmTDn/fhs9bQlyyqEJp3ZYrs",
    "2Hj9aRf8zgDFRp3AYNOQZFhnAgSjdNAtI6T9iaZckizLAIjol6TZE/DUFp9mLmaJ8gT9BDiTIYJ",
    "E52dE+xMYa3VJuNOi4zl5WMjVKy9TJYoq01hioWZtraVDQI57oU1pPxPoyBJ1qbxJBqWKq1GKc1",
    "C3oSA+zSu7warGBIfAZrSBGwQrrGOUDSd95hY/kaSa02Mmpg8jb39xc/L8I6woAaEApFXUARpgF",
    "jloS3XFvAtOMegWiVScwrAqcZsgzl6YEVzTOpRhiTiiZy4YJjtBQY6SFyiZVwpLFoqmCOy2U4Sm",
    "gIgWswjEMXywaxBTHRrrW9xrobMzNpgqaKxQWsWqw6bEMtl1CTGI7ESMgz0VYy8rMen4YXALONK",
    "MHEsI79S03auHsRFW6e/exdzLMfvmGeTeXyz+dzetYK9Hhzz/y+m7XqfWv4a206f+fk7ub2Wc/J",
    "OdwW3k/+6v5x+OOd7Z/43c3u9v2/V9n/z9rl5v+9bP1w2mhnhTNeyFeHfTNn13qB/u05/VvZWjf",
    "NqyHZ1S/96O3uzsfWSa/+4XeW8bPE3lrO3un7x67jY5Yl+aloJ8gOJm5NbDqunf5Xc/ZuLInHZv",
    "Y+vT7/upr8g708j+tL5Lfw1ySXt1ff//HeavLp/u+X2P80x/fTZK73iesW0KcvdvqaswBLvU+lM",
    "EpyDqqvTgOe8/Oq+X5ZOe+G5f4tdv4vd/28f8v8umk574blVvXvsnP8tuGr5oGXw2/dYj9WPV9d",
    "N/4f73LnoEfZ+pfZvzqNkPGg4NSbfcYj7KDo638gT78ABLuwQA==",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c25_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c25_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2941105953U));
  ssSetChecksum1(S,(369823568U));
  ssSetChecksum2(S,(151874601U));
  ssSetChecksum3(S,(3423737421U));
}

static void mdlRTW_c25_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c25_sliding_controller_rbdl(SimStruct *S)
{
  SFc25_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc25_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc25_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc25_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc25_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c25_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c25_JITStateAnimation,
    chartInstance->c25_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c25_sliding_controller_rbdl(chartInstance);
}

void c25_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c25_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c25_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c25_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c25_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
