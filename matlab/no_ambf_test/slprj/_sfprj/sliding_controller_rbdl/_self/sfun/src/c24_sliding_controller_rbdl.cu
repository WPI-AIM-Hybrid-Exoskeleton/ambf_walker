/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c24_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c24_do_animation_call_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c24_st);
static void sf_gateway_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c24_eML_blk_kernel(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c24_b_u[7], real_T c24_b_y[7]);
static void c24_emlrt_marshallIn(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c24_b_y, const char_T *c24_identifier, real_T
  c24_c_y[7]);
static void c24_b_emlrt_marshallIn(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId,
  real_T c24_b_y[7]);
static uint8_T c24_c_emlrt_marshallIn
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c24_b_is_active_c24_sliding_controller_rbdl, const char_T *c24_identifier);
static uint8_T c24_d_emlrt_marshallIn
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c24_b_u, const emlrtMsgIdentifier *c24_parentId);
static void init_dsm_address_info(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c24_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c24_is_active_c24_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c24_do_animation_call_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c24_b_y = NULL;
  const mxArray *c24_c_y = NULL;
  const mxArray *c24_d_y = NULL;
  const mxArray *c24_st;
  c24_st = NULL;
  c24_b_y = NULL;
  sf_mex_assign(&c24_b_y, sf_mex_createcellmatrix(2, 1), false);
  c24_c_y = NULL;
  sf_mex_assign(&c24_c_y, sf_mex_create("y", *chartInstance->c24_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c24_b_y, 0, c24_c_y);
  c24_d_y = NULL;
  sf_mex_assign(&c24_d_y, sf_mex_create("y",
    &chartInstance->c24_is_active_c24_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c24_b_y, 1, c24_d_y);
  sf_mex_assign(&c24_st, c24_b_y, false);
  return c24_st;
}

static void set_sim_state_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c24_st)
{
  const mxArray *c24_b_u;
  c24_b_u = sf_mex_dup(c24_st);
  c24_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c24_b_u, 0)),
                       "y", *chartInstance->c24_y);
  chartInstance->c24_is_active_c24_sliding_controller_rbdl =
    c24_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c24_b_u, 1)),
    "is_active_c24_sliding_controller_rbdl");
  sf_mex_destroy(&c24_b_u);
  sf_mex_destroy(&c24_st);
}

static void sf_gateway_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c24_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c24_eML_blk_kernel(chartInstance, *chartInstance->c24_u, *chartInstance->c24_y);
  c24_do_animation_call_c24_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c24_errCode;
  c24_errCode = hipGetLastError();
  if (c24_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c24_errCode, (char_T *)hipGetErrorName
                       (c24_errCode), (char_T *)hipGetErrorString(c24_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c24_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc24_sliding_controller_rbdl
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c24_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c24_nameCaptureInfo = NULL;
  c24_nameCaptureInfo = NULL;
  sf_mex_assign(&c24_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c24_nameCaptureInfo;
}

static void c24_eML_blk_kernel(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c24_b_u[7], real_T c24_b_y[7])
{
  c24_b_y[0] = c24_b_u[3];
  c24_b_y[1] = c24_b_u[6];
  c24_b_y[2] = c24_b_u[0];
  c24_b_y[3] = c24_b_u[1];
  c24_b_y[4] = c24_b_u[4];
  c24_b_y[5] = c24_b_u[5];
  c24_b_y[6] = c24_b_u[2];
}

static void c24_emlrt_marshallIn(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c24_b_y, const char_T *c24_identifier, real_T
  c24_c_y[7])
{
  emlrtMsgIdentifier c24_thisId;
  c24_thisId.fIdentifier = const_cast<const char_T *>(c24_identifier);
  c24_thisId.fParent = NULL;
  c24_thisId.bParentIsCell = false;
  c24_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c24_b_y), &c24_thisId,
    c24_c_y);
  sf_mex_destroy(&c24_b_y);
}

static void c24_b_emlrt_marshallIn(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId,
  real_T c24_b_y[7])
{
  real_T c24_dv[7];
  int32_T c24_i;
  sf_mex_import(c24_parentId, sf_mex_dup(c24_b_u), c24_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c24_i = 0; c24_i < 7; c24_i++) {
    c24_b_y[c24_i] = c24_dv[c24_i];
  }

  sf_mex_destroy(&c24_b_u);
}

static uint8_T c24_c_emlrt_marshallIn
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c24_b_is_active_c24_sliding_controller_rbdl, const char_T *c24_identifier)
{
  emlrtMsgIdentifier c24_thisId;
  uint8_T c24_b_y;
  c24_thisId.fIdentifier = const_cast<const char_T *>(c24_identifier);
  c24_thisId.fParent = NULL;
  c24_thisId.bParentIsCell = false;
  c24_b_y = c24_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c24_b_is_active_c24_sliding_controller_rbdl), &c24_thisId);
  sf_mex_destroy(&c24_b_is_active_c24_sliding_controller_rbdl);
  return c24_b_y;
}

static uint8_T c24_d_emlrt_marshallIn
  (SFc24_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c24_b_u, const emlrtMsgIdentifier *c24_parentId)
{
  uint8_T c24_b_y;
  uint8_T c24_c_u;
  sf_mex_import(c24_parentId, sf_mex_dup(c24_b_u), &c24_c_u, 1, 3, 0U, 0, 0U, 0);
  c24_b_y = c24_c_u;
  sf_mex_destroy(&c24_b_u);
  return c24_b_y;
}

static void init_dsm_address_info(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc24_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c24_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c24_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c24_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c24_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3007509169U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1926089682U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2239166282U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3770191902U);
}

mxArray *sf_c24_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c24_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c24_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c24_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8clGJvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB2RB2x"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c24_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sdegTANPF1GH9p3UVy5QBRH";
}

static void sf_opaque_initialize_c24_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c24_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c24_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c24_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c24_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c24_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c24_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc24_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c24_sliding_controller_rbdl
      ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc24_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c24_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c24_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc24_sliding_controller_rbdl
    ((SFc24_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c24_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c24_sliding_controller_rbdl
      ((SFc24_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc24_sliding_controller_rbdl
      ((SFc24_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c24_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u1EgQdkYBLRJE0QppLwi4RHCDhV1p97KbzB8ZKSEBT8JeVqNOu2bcmna36Z9J5swR8RS",
    "8ABfehadAXOCGRLXtmcx6bA9hRMQiLDmetr+qrqqvfjreSmfXw2sN782fPe8iPn/Cu+al14VsvT",
    "Jzp+9XvVvZ+gUKCRvtE0Ui7VVegkTwGLTk1jApOqIvC2FM9EGBoIiNpTJl2jSLLGdi2LaCOn36S",
    "cho6IfS8qCOsiTYE3yM2mJr9lFPkymgpg0QmFBJOwjbnAymFitz3AiBDrWNqlzQYHwbO7P0ruWG",
    "xRxaJ0A7QhuCFutT23xDDDTMSambzlPtT4AyijkjotDbkGgfYgywgYM4wL971qBTeRgNiTJ1CMk",
    "I9A4bJjqlgLxOpvHDERPESMUIb0W84QTnbdvnaM+uDIBXBARtqysgw1gyYcr599voaUuQIw5NOL",
    "KDcm0+PLWO/EMGx6BK49ZvyBEoMoA9UbppEpDWScLWNEvmYYZFcEjUFkX+NASl2YuZo32CPEEXJ",
    "cpgkDjZ0V3FRhjeUm026rjMXFQyNkrJ1otgibbWCKpYmGprU9EgnOtSWFfGOzACnmhtEkOqYanW",
    "YpzWLOhKDLBL7/JqsIIh8RmsIUXACuka5QBJ33mIjeW/SGq1kVEDk7e5szP/eR7WEQZUn1Ao6gK",
    "KMA0YsyS85doCph33CESrTGJeETjNkEUoT/etaB5LNcSYVDSRUxcco6XASA+QS6yEA41FUwVzXC",
    "7CUUJDCFyDYRx2sWwQWxAT7VrbFtbdiJlxEzRVLC5g1WLVYRtquYQax3AghkIei7aSkZ/1+DS8A",
    "JhtRAkmBnXsX2rcxt2LqHDz7K53Os8uf8Y8m8jln7dn9KwU6PFmnvl9L9Wq963hr5XJ/J2Ru5Lb",
    "ZzUn53DreH+8+uT18zvP3q+9JP++/bBxfZn9X9XONv/XsvW1SaOdFs5oLl8ddnvGrtUC/b/M6F/",
    "P1jqAQXfr4X771wfbf8b3Dw7Hvz+qP95O+Vlgby1n7+T9TdfxMcuS/FS0E2QHE7cmNh3XTv8fM/",
    "ZeXBCPS9n79Hr393LyVzfzPK4ukF/HX+Nc3n75/jc3l5NP9/9ngf0bOb43krneI65bQI/e+62nO",
    "Quw1HtUCqMk56B66ijgOT+/NN/PKueds9z/xc4fcl+f98+ZX+ct552z3LL+nXWOf2v4qnng5fDr",
    "37Afy56vvjb+jXe2c9CNbP3X9F+dRsh4UHDqzT7jEbZf9PU7yNNPiCuz6A==",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c24_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c24_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042383101U));
  ssSetChecksum1(S,(4119015307U));
  ssSetChecksum2(S,(1566679822U));
  ssSetChecksum3(S,(505804526U));
}

static void mdlRTW_c24_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c24_sliding_controller_rbdl(SimStruct *S)
{
  SFc24_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc24_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc24_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc24_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc24_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c24_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c24_JITStateAnimation,
    chartInstance->c24_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c24_sliding_controller_rbdl(chartInstance);
}

void c24_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c24_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c24_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c24_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c24_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
