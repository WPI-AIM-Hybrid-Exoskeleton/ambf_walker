#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c6_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c6_do_animation_call_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c6_st);
static void sf_gateway_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c6_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c6_b_y, const char_T *c6_identifier, real_T
  c6_c_y[7]);
static void c6_b_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId,
  real_T c6_b_y[7]);
static uint8_T c6_c_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c6_b_is_active_c6_sliding_controller_rbdl, const
  char_T *c6_identifier);
static uint8_T c6_d_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId);
static void c6_eML_blk_kernel(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c6_b_u[7]);
static __global__ void c6_sf_gateway_c6_sliding_controller_rbdl_kernel1(real_T
  c6_b_u[7], real_T c6_b_y[7]);
static void init_dsm_address_info(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance);

/* Function Definitions */
static void initialize_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c6_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c6_is_active_c6_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c6_do_animation_call_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c6_b_y = NULL;
  const mxArray *c6_c_y = NULL;
  const mxArray *c6_d_y = NULL;
  const mxArray *c6_st;
  c6_st = NULL;
  c6_b_y = NULL;
  sf_mex_assign(&c6_b_y, sf_mex_createcellmatrix(2, 1), false);
  c6_c_y = NULL;
  sf_mex_assign(&c6_c_y, sf_mex_create("y", *chartInstance->c6_y, 0, 0U, 1U, 0U,
    1, 7), false);
  sf_mex_setcell(c6_b_y, 0, c6_c_y);
  c6_d_y = NULL;
  sf_mex_assign(&c6_d_y, sf_mex_create("y",
    &chartInstance->c6_is_active_c6_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c6_b_y, 1, c6_d_y);
  sf_mex_assign(&c6_st, c6_b_y, false);
  return c6_st;
}

static void set_sim_state_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c6_st)
{
  const mxArray *c6_b_u;
  c6_b_u = sf_mex_dup(c6_st);
  c6_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c6_b_u, 0)), "y",
                      *chartInstance->c6_y);
  chartInstance->c6_is_active_c6_sliding_controller_rbdl = c6_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c6_b_u, 1)),
     "is_active_c6_sliding_controller_rbdl");
  sf_mex_destroy(&c6_b_u);
  sf_mex_destroy(&c6_st);
}

static void sf_gateway_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  real_T (*c6_gpu_u)[7];
  real_T (*c6_gpu_y)[7];
  hipMalloc(&c6_gpu_y, 56UL);
  hipMalloc(&c6_gpu_u, 56UL);
  chartInstance->c6_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(*c6_gpu_u, *chartInstance->c6_u, 56UL, hipMemcpyHostToDevice);
  hipMemcpy(*c6_gpu_y, *chartInstance->c6_y, 56UL, hipMemcpyHostToDevice);
  c6_sf_gateway_c6_sliding_controller_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(*c6_gpu_u, *c6_gpu_y);
  hipMemcpy(*chartInstance->c6_y, *c6_gpu_y, 56UL, hipMemcpyDeviceToHost);
  hipMemcpy(*chartInstance->c6_u, *c6_gpu_u, 56UL, hipMemcpyDeviceToHost);
  c6_eML_blk_kernel(chartInstance, *chartInstance->c6_y);
  c6_do_animation_call_c6_sliding_controller_rbdl(chartInstance);
  hipFree(*c6_gpu_u);
  hipFree(*c6_gpu_y);
}

static void mdl_start_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c6_errCode;
  c6_errCode = hipGetLastError();
  if (c6_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c6_errCode, (char_T *)hipGetErrorName
                       (c6_errCode), (char_T *)hipGetErrorString(c6_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c6_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c6_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c6_nameCaptureInfo = NULL;
  c6_nameCaptureInfo = NULL;
  sf_mex_assign(&c6_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c6_nameCaptureInfo;
}

static void c6_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c6_b_y, const char_T *c6_identifier, real_T
  c6_c_y[7])
{
  emlrtMsgIdentifier c6_thisId;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c6_b_y), &c6_thisId, c6_c_y);
  sf_mex_destroy(&c6_b_y);
}

static void c6_b_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId,
  real_T c6_b_y[7])
{
  real_T c6_dv[7];
  int32_T c6_i;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_b_u), c6_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c6_i = 0; c6_i < 7; c6_i++) {
    c6_b_y[c6_i] = c6_dv[c6_i];
  }

  sf_mex_destroy(&c6_b_u);
}

static uint8_T c6_c_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c6_b_is_active_c6_sliding_controller_rbdl, const
  char_T *c6_identifier)
{
  emlrtMsgIdentifier c6_thisId;
  uint8_T c6_b_y;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_b_y = c6_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c6_b_is_active_c6_sliding_controller_rbdl), &c6_thisId);
  sf_mex_destroy(&c6_b_is_active_c6_sliding_controller_rbdl);
  return c6_b_y;
}

static uint8_T c6_d_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId)
{
  uint8_T c6_b_y;
  uint8_T c6_c_u;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_b_u), &c6_c_u, 1, 3, 0U, 0, 0U, 0);
  c6_b_y = c6_c_u;
  sf_mex_destroy(&c6_b_u);
  return c6_b_y;
}

static void c6_eML_blk_kernel(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c6_b_u[7])
{
}

static __global__ __launch_bounds__(32, 1) void
  c6_sf_gateway_c6_sliding_controller_rbdl_kernel1(real_T c6_b_u[7], real_T
  c6_b_y[7])
{
  uint64_T c6_threadId;
  int32_T c6_i;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i = (int32_T)c6_threadId;
  if (c6_i < 7) {
    c6_b_y[c6_i] = c6_b_u[c6_i];
  }
}

static void init_dsm_address_info(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance)
{
  chartInstance->c6_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c6_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c6_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c6_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2438127827U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3025077116U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(880705944U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3519700120U);
}

mxArray *sf_c6_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c6_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c6_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c6_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfh"
    "UU90P4mcXxicklmWWp8clm8cU5mSmZeenxyfl5JUX5OTmpRfFFSSk5DEj+BAB4/x1/"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c6_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "svi1IOD2u267XmhzOouF1CC";
}

static void sf_opaque_initialize_c6_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c6_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c6_sliding_controller_rbdl((SFc6_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c6_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c6_sliding_controller_rbdl((SFc6_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c6_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c6_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c6_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c6_sliding_controller_rbdl(void *
  chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc6_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c6_sliding_controller_rbdl
      ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc6_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c6_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c6_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c6_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c6_sliding_controller_rbdl
      ((SFc6_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc6_sliding_controller_rbdl
      ((SFc6_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c6_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV91u00gUdqMuAgmqCiG4QQJpKy2XC9LC3iyUOok2UkuqdQtcIEXT8Uk8ynjGzI/b7AtwseK",
    "efYrlksfgksfYR+CM7aRZx3YoERWLsOQ6Y3/nzDnnOz9Tb6235+G1gff2Vc+7gM+LeLe8/PqhWK",
    "/N3fn7de+nYv0ahYSN94kisfYaL0Fi+AO05NYwKXpiKCthTAxBgaCITaQyddo0iy1nYty1gjp9+",
    "lnEaBRE0vJwB2VJ2Bd8gtoSa/ZRT5spoKYLEJpISTuKupyMZhYrc+xHQMfaxk0uaDCBTZxZes9y",
    "wxIOnROgPaENQYv1qW2BIQZ8c1LrpvNUB1OgjBPOiKj0NiI6gAQDbOAwCfFv3xp0qgyjEVFmByK",
    "Sgt5l40ynFFDWyTR+OGKCGKkY4Z2Y+05w0bZ9jvbsyRB4Q0DQth0FZJxIJkw9/0EXPe0IcsShDU",
    "d2VK8tgJfWkf+UwTGo2rgNfZmCIiPoi9pNs4B0TjK2ZlmyCDMshqdEPabIn4awNnsxc3RAkCc4Q",
    "Ik6GGRO9vSBYimGt1abjXsuM5eVjI1zsvUyWKatk0ITCzNtXSp8wrmuhR3IZBdS4JnWNjGkGZZr",
    "rcZpzcIDiQF26V1fDVYwJL6A+VKErJKutATI+s4TbCz/RVKrjYx9TN727u7i50VYTxhQQ0Khqgs",
    "owjRgzLLw1msLmXbcIxCtMpl5VeA8Q5ahPD20on0s1Rhj0tBETl1wjNYCYz1CLrESDjUWTRPMcb",
    "kMRwmNIHQNhnHYw7JBbEVMtGttj7HuUmYmbdBUsaSCVYtVh22o4xJqksChGAt5LLpKxkHR4/PwA",
    "mC2ESWYGO1g/1KTLu5eRYWbZz97p/Ps8ifMs6lc+XlnTs9ahR5v7lne91Kred8W/lqbzt85uSul",
    "fdZLcg63iffVV9eD4S9//fjbi3d/v3/15p9V9n/bOtv83yjWN6eNdlY46UK+Ouzvc3atV+i/Mad",
    "/s1jrlN3t9dv37L37D57H0Z99abt3fT/nZ4m9rZK90/e3XcfHLMvyU9FeWBxM3JrYfFw7/b/O2X",
    "thSTwuFe/z699Hq8lf2y7zuL5EfhN/TUp5+/n7395eTT7f//kS+7dKfG9lc31AXLeAAb0/0JyFW",
    "OkDKoVRknNQA3UU8nJ9fm6+n1XOO2e5/4ud3+W+PO+fMr/OW847Z7lV/TvrHP/a8E3zwCvhN79i",
    "P1Y9X31p/AfvbOegW8X64exfHT9iPKw49Raf8Qg7rPr6DeTpRxCisuI=",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c6_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c6_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1393985298U));
  ssSetChecksum1(S,(596325734U));
  ssSetChecksum2(S,(2578996285U));
  ssSetChecksum3(S,(2946009040U));
}

static void mdlRTW_c6_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c6_sliding_controller_rbdl(SimStruct *S)
{
  SFc6_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc6_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc6_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc6_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc6_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c6_JITStateAnimation,
    chartInstance->c6_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c6_sliding_controller_rbdl(chartInstance);
}

void c6_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c6_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c6_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c6_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c6_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
