/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c6_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c6_do_animation_call_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c6_st);
static void sf_gateway_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c6_eML_blk_kernel(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c6_b_u[7], real_T c6_b_y[6]);
static void c6_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c6_b_y, const char_T *c6_identifier, real_T
  c6_c_y[6]);
static void c6_b_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId,
  real_T c6_b_y[6]);
static uint8_T c6_c_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c6_b_is_active_c6_sliding_controller_rbdl, const
  char_T *c6_identifier);
static uint8_T c6_d_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId);
static void init_dsm_address_info(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance);

/* Function Definitions */
static void initialize_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c6_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c6_is_active_c6_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c6_do_animation_call_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c6_b_y = NULL;
  const mxArray *c6_c_y = NULL;
  const mxArray *c6_d_y = NULL;
  const mxArray *c6_st;
  c6_st = NULL;
  c6_b_y = NULL;
  sf_mex_assign(&c6_b_y, sf_mex_createcellmatrix(2, 1), false);
  c6_c_y = NULL;
  sf_mex_assign(&c6_c_y, sf_mex_create("y", *chartInstance->c6_y, 0, 0U, 1U, 0U,
    1, 6), false);
  sf_mex_setcell(c6_b_y, 0, c6_c_y);
  c6_d_y = NULL;
  sf_mex_assign(&c6_d_y, sf_mex_create("y",
    &chartInstance->c6_is_active_c6_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c6_b_y, 1, c6_d_y);
  sf_mex_assign(&c6_st, c6_b_y, false);
  return c6_st;
}

static void set_sim_state_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c6_st)
{
  const mxArray *c6_b_u;
  c6_b_u = sf_mex_dup(c6_st);
  c6_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c6_b_u, 0)), "y",
                      *chartInstance->c6_y);
  chartInstance->c6_is_active_c6_sliding_controller_rbdl = c6_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c6_b_u, 1)),
     "is_active_c6_sliding_controller_rbdl");
  sf_mex_destroy(&c6_b_u);
  sf_mex_destroy(&c6_st);
}

static void sf_gateway_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c6_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c6_eML_blk_kernel(chartInstance, *chartInstance->c6_u, *chartInstance->c6_y);
  c6_do_animation_call_c6_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c6_errCode;
  c6_errCode = hipGetLastError();
  if (c6_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c6_errCode, (char_T *)hipGetErrorName
                       (c6_errCode), (char_T *)hipGetErrorString(c6_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c6_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc6_sliding_controller_rbdl
  (SFc6_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c6_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c6_nameCaptureInfo = NULL;
  c6_nameCaptureInfo = NULL;
  sf_mex_assign(&c6_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c6_nameCaptureInfo;
}

static void c6_eML_blk_kernel(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c6_b_u[7], real_T c6_b_y[6])
{
  c6_b_y[0] = c6_b_u[1];
  c6_b_y[1] = c6_b_u[2];
  c6_b_y[2] = c6_b_u[3];
  c6_b_y[3] = c6_b_u[0];
  c6_b_y[4] = c6_b_u[4];
  c6_b_y[5] = c6_b_u[5];
}

static void c6_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c6_b_y, const char_T *c6_identifier, real_T
  c6_c_y[6])
{
  emlrtMsgIdentifier c6_thisId;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c6_b_y), &c6_thisId, c6_c_y);
  sf_mex_destroy(&c6_b_y);
}

static void c6_b_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId,
  real_T c6_b_y[6])
{
  real_T c6_dv[6];
  int32_T c6_i;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_b_u), c6_dv, 1, 0, 0U, 1, 0U, 1, 6);
  for (c6_i = 0; c6_i < 6; c6_i++) {
    c6_b_y[c6_i] = c6_dv[c6_i];
  }

  sf_mex_destroy(&c6_b_u);
}

static uint8_T c6_c_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c6_b_is_active_c6_sliding_controller_rbdl, const
  char_T *c6_identifier)
{
  emlrtMsgIdentifier c6_thisId;
  uint8_T c6_b_y;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_b_y = c6_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c6_b_is_active_c6_sliding_controller_rbdl), &c6_thisId);
  sf_mex_destroy(&c6_b_is_active_c6_sliding_controller_rbdl);
  return c6_b_y;
}

static uint8_T c6_d_emlrt_marshallIn(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c6_b_u, const emlrtMsgIdentifier *c6_parentId)
{
  uint8_T c6_b_y;
  uint8_T c6_c_u;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_b_u), &c6_c_u, 1, 3, 0U, 0, 0U, 0);
  c6_b_y = c6_c_u;
  sf_mex_destroy(&c6_b_u);
  return c6_b_y;
}

static void init_dsm_address_info(SFc6_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc6_sliding_controller_rbdlInstanceStruct *
  chartInstance)
{
  chartInstance->c6_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c6_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c6_y = (real_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c6_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3910899723U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1623685603U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(38977074U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3513601893U);
}

mxArray *sf_c6_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c6_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c6_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c6_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfh"
    "UU90P4mcXxicklmWWp8clm8cU5mSmZeenxyfl5JUX5OTmpRfFFSSk5DEj+BAB4/x1/"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c6_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sbIZyOsp5zUyg8V8S5w2l4G";
}

static void sf_opaque_initialize_c6_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c6_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c6_sliding_controller_rbdl((SFc6_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c6_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c6_sliding_controller_rbdl((SFc6_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c6_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c6_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c6_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c6_sliding_controller_rbdl(void *
  chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc6_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c6_sliding_controller_rbdl
      ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc6_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c6_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c6_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc6_sliding_controller_rbdl
    ((SFc6_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c6_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c6_sliding_controller_rbdl
      ((SFc6_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc6_sliding_controller_rbdl
      ((SFc6_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c6_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV91qFEkU7gxZUVAZZEGQBb0I6M2CiEpu1CTzsw4km2AnUUQYaqrPTBdTXdVbPzOZfQbFt/D",
    "aR9nLvMTCgtcLe6q7ZzL2dPcYB4O7bEOnp7q/c+qc852firfW2fPwuo731g3Pu4TPy3jXvPT6IV",
    "uvzd3p+3XvbrZ+h0LCRgdEkUh7lZcgEbwALbk1TIqO6MtCGBN9UCAoYmOpTJk2zSLLmRi2raBOn",
    "34ZMhr6obQ82EFZEuwLPkFtsTUHqKfJFFDTBghMqKQdhG1OBjOLlRk3QqBDbaMqFzQY38bOLL1n",
    "uWExh9YJ0I7QhqDF+sw23xADDXNS6qbzVPtToIxizogo9DYk2ocYA2zgKA7w77416FQeRkOizA6",
    "EZAR6lw0TnVJAXifT+KHHBDFSMcJbEW84wUXbDjjasycD4BUBQdt2FJBhLJkw5fz7bfS0JUiPQx",
    "N6dlCuzYffrCP/mMEYVGnc+g05AkUGsC9KN00C0jpJ2JplySLMsAiOidqmyJ+GoDR7MXO0T5AnO",
    "ESJMhgkTnb0oWIjDG+pNht1XGYuKxkbpWTrZbBEW2sEVSzMtLWpaBDOdSnsUMa7MAKeaG0SQ6ph",
    "qdZinNYsOJQYYJfe5dVgBUPiM1hDioAV0jXKAZK+8ys2ls+R1GojowYmb3N3d/HzIqwjDKg+oVD",
    "UBRRhGjBmSXjLtQVMO+4RiFaZxLwicJohy1Ce7lvRHEs1xJhUNJEzFxyjpcBID5BLrIQjjUVTBX",
    "NcLsNRQkMIXINhHPawbBBbEBPtWts21t2ImUkTNFUsLmDVYtVhG2q5hJrEcCSGQo5FW8nIz3p8G",
    "l4AzDaiBBODHexfatLG3YuocPPsvnc2z65+wTybyuWf9+b0rBXo8eae+X2v1Kr3reGvten8nZO7",
    "lttnPSfncHW83374o3766Wd56/2DP5+M/36zyv4fa+eb/9ez9U/TRjsrnNFCvjrs8zm71gv035z",
    "TX8/Wutd5PdnX8aPfjyaDzeNN/9H4AX/4S8rPEntrOXun7++4jo9ZluSnop0gO5i4NbHpuHb6N+",
    "fsvbQkHley9+n117PV5H/cyvO4vkS+jr8mubz9+v3vbK0mn+7/aon9Gzm+N5K53iWuW0CXPu5qz",
    "gKs9C6VwijJOaiu6gU8X59fm+/nlfMuWO7fYuf/ct+e9y+ZXxct512w3Kr+nXeOf2/4qnng5fD1",
    "79iPVc9X3xp/6p3vHHQ7Wz+d/avTCBkPCk692Wc8wvaLvv4H8vQfr7i0xg==",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c6_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c6_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(282109580U));
  ssSetChecksum1(S,(1865282520U));
  ssSetChecksum2(S,(3979513626U));
  ssSetChecksum3(S,(1560115005U));
}

static void mdlRTW_c6_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c6_sliding_controller_rbdl(SimStruct *S)
{
  SFc6_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc6_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc6_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc6_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc6_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c6_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c6_JITStateAnimation,
    chartInstance->c6_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c6_sliding_controller_rbdl(chartInstance);
}

void c6_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c6_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c6_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c6_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c6_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
