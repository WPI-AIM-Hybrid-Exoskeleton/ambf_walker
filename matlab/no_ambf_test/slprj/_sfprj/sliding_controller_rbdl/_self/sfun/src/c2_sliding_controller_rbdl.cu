#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c2_sliding_controller_rbdl.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c2_do_animation_call_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c2_st);
static void sf_gateway_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c2_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_b_msg, const char_T *c2_identifier,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_y);
static void c2_b_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_y);
static void c2_c_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  uint8_T c2_y[4]);
static c2_SL_Bus_ROSVariableLengthArrayInfo c2_d_emlrt_marshallIn
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c2_u, const emlrtMsgIdentifier *c2_parentId);
static uint32_T c2_e_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static void c2_f_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real_T c2_y[7]);
static uint8_T c2_g_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c2_b_is_active_c2_sliding_controller_rbdl, const
  char_T *c2_identifier);
static uint8_T c2_h_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static void c2_eML_blk_kernel(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
  *c2_b_blankMsg);
static __global__ void c2_sf_gateway_c2_sliding_controller_rbdl_kernel1
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
   *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg);
static __global__ void c2_sf_gateway_c2_sliding_controller_rbdl_kernel2
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
   *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg);
static __global__ void c2_sf_gateway_c2_sliding_controller_rbdl_kernel3
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg,
   c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_r);
static __global__ void c2_sf_gateway_c2_sliding_controller_rbdl_kernel4
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg,
   c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_r);
static __global__ void c2_sf_gateway_c2_sliding_controller_rbdl_kernel5(const
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_r,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg);
static __global__ void c2_sf_gateway_c2_sliding_controller_rbdl_kernel6(const
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_r,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg);
static __global__ void c2_eML_blk_kernel_kernel7
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
   *c2_b_blankMsg);
static __global__ void c2_eML_blk_kernel_kernel8(const uint8_T c2_uv[3],
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
  *c2_b_blankMsg);
static __global__ void c2_eML_blk_kernel_kernel9
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
   *c2_b_blankMsg);
static void init_dsm_address_info(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc2_sliding_controller_rbdlInstanceStruct *
  chartInstance);

/* Function Definitions */
static void initialize_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c2_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c2_is_active_c2_sliding_controller_rbdl = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c2_gpu_uv, 3UL);
  hipMalloc(&chartInstance->c2_gpu_blankMsg, 208UL);
}

static void initialize_params_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c2_do_animation_call_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  static const char_T *c2_sv[8] = { "ModelName", "ModelName_SL_Info", "Q",
    "Q_SL_Info", "Qd", "Qd_SL_Info", "Qdd", "Qdd_SL_Info" };

  static const char_T *c2_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c2_b_y = NULL;
  const mxArray *c2_c_y = NULL;
  const mxArray *c2_d_y = NULL;
  const mxArray *c2_e_y = NULL;
  const mxArray *c2_f_y = NULL;
  const mxArray *c2_g_y = NULL;
  const mxArray *c2_h_y = NULL;
  const mxArray *c2_i_y = NULL;
  const mxArray *c2_j_y = NULL;
  const mxArray *c2_k_y = NULL;
  const mxArray *c2_l_y = NULL;
  const mxArray *c2_m_y = NULL;
  const mxArray *c2_n_y = NULL;
  const mxArray *c2_o_y = NULL;
  const mxArray *c2_p_y = NULL;
  const mxArray *c2_q_y = NULL;
  const mxArray *c2_r_y = NULL;
  const mxArray *c2_s_y = NULL;
  const mxArray *c2_st;
  const mxArray *c2_y = NULL;
  real_T c2_d_u[7];
  int32_T c2_i;
  int32_T c2_i1;
  int32_T c2_i2;
  int32_T c2_i3;
  uint32_T c2_b_u;
  uint32_T c2_c_u;
  uint32_T c2_e_u;
  uint32_T c2_f_u;
  uint32_T c2_g_u;
  uint32_T c2_h_u;
  uint32_T c2_i_u;
  uint32_T c2_j_u;
  uint32_T c2_u_CurrentLength;
  uint32_T c2_u_ReceivedLength;
  uint8_T c2_u[4];
  c2_st = NULL;
  c2_y = NULL;
  sf_mex_assign(&c2_y, sf_mex_createcellmatrix(2, 1), false);
  c2_b_y = NULL;
  sf_mex_assign(&c2_b_y, sf_mex_createstruct("structure", 8, c2_sv, 2, 1, 1),
                false);
  for (c2_i = 0; c2_i < 4; c2_i++) {
    c2_u[c2_i] = ((uint8_T *)&((char_T *)chartInstance->c2_msg)[0])[c2_i];
  }

  c2_c_y = NULL;
  sf_mex_assign(&c2_c_y, sf_mex_create("y", c2_u, 3, 0U, 1U, 0U, 1, 4), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "ModelName", c2_c_y, 0);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [4];
  c2_d_y = NULL;
  sf_mex_assign(&c2_d_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_b_u = c2_u_CurrentLength;
  c2_e_y = NULL;
  sf_mex_assign(&c2_e_y, sf_mex_create("y", &c2_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_d_y, 0, "CurrentLength", c2_e_y, 0);
  c2_c_u = c2_u_ReceivedLength;
  c2_f_y = NULL;
  sf_mex_assign(&c2_f_y, sf_mex_create("y", &c2_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_d_y, 0, "ReceivedLength", c2_f_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "ModelName_SL_Info", c2_d_y, 1);
  for (c2_i1 = 0; c2_i1 < 7; c2_i1++) {
    c2_d_u[c2_i1] = ((real_T *)&((char_T *)chartInstance->c2_msg)[16])[c2_i1];
  }

  c2_g_y = NULL;
  sf_mex_assign(&c2_g_y, sf_mex_create("y", c2_d_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "Q", c2_g_y, 2);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[4];
  c2_h_y = NULL;
  sf_mex_assign(&c2_h_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_e_u = c2_u_CurrentLength;
  c2_i_y = NULL;
  sf_mex_assign(&c2_i_y, sf_mex_create("y", &c2_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_h_y, 0, "CurrentLength", c2_i_y, 0);
  c2_f_u = c2_u_ReceivedLength;
  c2_j_y = NULL;
  sf_mex_assign(&c2_j_y, sf_mex_create("y", &c2_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_h_y, 0, "ReceivedLength", c2_j_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "Q_SL_Info", c2_h_y, 3);
  for (c2_i2 = 0; c2_i2 < 7; c2_i2++) {
    c2_d_u[c2_i2] = ((real_T *)&((char_T *)chartInstance->c2_msg)[80])[c2_i2];
  }

  c2_k_y = NULL;
  sf_mex_assign(&c2_k_y, sf_mex_create("y", c2_d_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qd", c2_k_y, 4);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[4];
  c2_l_y = NULL;
  sf_mex_assign(&c2_l_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_g_u = c2_u_CurrentLength;
  c2_m_y = NULL;
  sf_mex_assign(&c2_m_y, sf_mex_create("y", &c2_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_l_y, 0, "CurrentLength", c2_m_y, 0);
  c2_h_u = c2_u_ReceivedLength;
  c2_n_y = NULL;
  sf_mex_assign(&c2_n_y, sf_mex_create("y", &c2_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_l_y, 0, "ReceivedLength", c2_n_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qd_SL_Info", c2_l_y, 5);
  for (c2_i3 = 0; c2_i3 < 7; c2_i3++) {
    c2_d_u[c2_i3] = ((real_T *)&((char_T *)chartInstance->c2_msg)[144])[c2_i3];
  }

  c2_o_y = NULL;
  sf_mex_assign(&c2_o_y, sf_mex_create("y", c2_d_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qdd", c2_o_y, 6);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[4];
  c2_p_y = NULL;
  sf_mex_assign(&c2_p_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_i_u = c2_u_CurrentLength;
  c2_q_y = NULL;
  sf_mex_assign(&c2_q_y, sf_mex_create("y", &c2_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_p_y, 0, "CurrentLength", c2_q_y, 0);
  c2_j_u = c2_u_ReceivedLength;
  c2_r_y = NULL;
  sf_mex_assign(&c2_r_y, sf_mex_create("y", &c2_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_p_y, 0, "ReceivedLength", c2_r_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qdd_SL_Info", c2_p_y, 7);
  sf_mex_setcell(c2_y, 0, c2_b_y);
  c2_s_y = NULL;
  sf_mex_assign(&c2_s_y, sf_mex_create("y",
    &chartInstance->c2_is_active_c2_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c2_y, 1, c2_s_y);
  sf_mex_assign(&c2_st, c2_y, false);
  return c2_st;
}

static void set_sim_state_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c2_st)
{
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb c2_r;
  const mxArray *c2_u;
  int32_T c2_i;
  int32_T c2_i1;
  c2_u = sf_mex_dup(c2_st);
  c2_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 0)), "msg",
                      &c2_r);
  for (c2_i = 0; c2_i < 4; c2_i++) {
    ((uint8_T *)&((char_T *)chartInstance->c2_msg)[0])[c2_i] =
      c2_r.ModelName[c2_i];
  }

  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[0] = c2_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[4] = c2_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[0] = c2_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[4] = c2_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[0] = c2_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[4] = c2_r.Qd_SL_Info.ReceivedLength;
  for (c2_i1 = 0; c2_i1 < 7; c2_i1++) {
    ((real_T *)&((char_T *)chartInstance->c2_msg)[16])[c2_i1] = c2_r.Q[c2_i1];
    ((real_T *)&((char_T *)chartInstance->c2_msg)[80])[c2_i1] = c2_r.Qd[c2_i1];
    ((real_T *)&((char_T *)chartInstance->c2_msg)[144])[c2_i1] = c2_r.Qdd[c2_i1];
  }

  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[0] = c2_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[4] = c2_r.Qdd_SL_Info.ReceivedLength;
  chartInstance->c2_is_active_c2_sliding_controller_rbdl = c2_g_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 1)),
     "is_active_c2_sliding_controller_rbdl");
  sf_mex_destroy(&c2_u);
  sf_mex_destroy(&c2_st);
}

static void sf_gateway_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb c2_r;
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
    *c2_b_gpu_blankMsg;
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_gpu_msg;
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_gpu_r;
  hipMalloc(&c2_gpu_r, 208UL);
  hipMalloc(&c2_gpu_msg, 208UL);
  hipMalloc(&c2_b_gpu_blankMsg, 208UL);
  chartInstance->c2_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(c2_b_gpu_blankMsg, chartInstance->c2_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel1<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_b_gpu_blankMsg, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c2_blankMsg, c2_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[8])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[8])[4];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[72])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[72])[4];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[136])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[136])[4];
  hipMemcpy(c2_b_gpu_blankMsg, chartInstance->c2_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel2<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_b_gpu_blankMsg, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c2_blankMsg, c2_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[200])[4];
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel3<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_gpu_msg, c2_gpu_r);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c2_r, c2_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c2_r.ModelName_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [0];
  c2_r.ModelName_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [4];
  c2_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[0];
  c2_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[4];
  c2_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[0];
  c2_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[4];
  hipMemcpy(c2_gpu_r, &c2_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel4<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_gpu_msg, c2_gpu_r);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c2_r, c2_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c2_r.Qdd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[0];
  c2_r.Qdd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[4];
  c2_eML_blk_kernel(chartInstance, &c2_r);
  hipMemcpy(c2_gpu_r, &c2_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel5<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_gpu_r, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[0] = c2_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[4] = c2_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[0] = c2_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[4] = c2_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[0] = c2_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[4] = c2_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel6<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_gpu_r, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[0] = c2_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[4] = c2_r.Qdd_SL_Info.ReceivedLength;
  c2_do_animation_call_c2_sliding_controller_rbdl(chartInstance);
  hipFree(c2_b_gpu_blankMsg);
  hipFree(c2_gpu_msg);
  hipFree(c2_gpu_r);
}

static void mdl_start_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c2_errCode;
  hipFree(*chartInstance->c2_gpu_uv);
  hipFree(chartInstance->c2_gpu_blankMsg);
  c2_errCode = hipGetLastError();
  if (c2_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c2_errCode, (char_T *)hipGetErrorName
                       (c2_errCode), (char_T *)hipGetErrorString(c2_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c2_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc2_sliding_controller_rbdl
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c2_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c2_nameCaptureInfo = NULL;
  c2_nameCaptureInfo = NULL;
  sf_mex_assign(&c2_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c2_nameCaptureInfo;
}

static void c2_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_b_msg, const char_T *c2_identifier,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_y)
{
  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c2_b_msg), &c2_thisId, c2_y);
  sf_mex_destroy(&c2_b_msg);
}

static void c2_b_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_y)
{
  static const char_T *c2_fieldNames[8] = { "ModelName", "ModelName_SL_Info",
    "Q", "Q_SL_Info", "Qd", "Qd_SL_Info", "Qdd", "Qdd_SL_Info" };

  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fParent = c2_parentId;
  c2_thisId.bParentIsCell = false;
  sf_mex_check_struct(c2_parentId, c2_u, 8, c2_fieldNames, 0U, NULL);
  c2_thisId.fIdentifier = "ModelName";
  c2_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u,
    "ModelName", "ModelName", 0)), &c2_thisId, c2_y->ModelName);
  c2_thisId.fIdentifier = "ModelName_SL_Info";
  c2_y->ModelName_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "ModelName_SL_Info", "ModelName_SL_Info", 0)),
    &c2_thisId);
  c2_thisId.fIdentifier = "Q";
  c2_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u, "Q", "Q",
    0)), &c2_thisId, c2_y->Q);
  c2_thisId.fIdentifier = "Q_SL_Info";
  c2_y->Q_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "Q_SL_Info", "Q_SL_Info", 0)), &c2_thisId);
  c2_thisId.fIdentifier = "Qd";
  c2_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u, "Qd",
    "Qd", 0)), &c2_thisId, c2_y->Qd);
  c2_thisId.fIdentifier = "Qd_SL_Info";
  c2_y->Qd_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c2_thisId);
  c2_thisId.fIdentifier = "Qdd";
  c2_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u, "Qdd",
    "Qdd", 0)), &c2_thisId, c2_y->Qdd);
  c2_thisId.fIdentifier = "Qdd_SL_Info";
  c2_y->Qdd_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "Qdd_SL_Info", "Qdd_SL_Info", 0)), &c2_thisId);
  sf_mex_destroy(&c2_u);
}

static void c2_c_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  uint8_T c2_y[4])
{
  int32_T c2_i;
  uint8_T c2_uv[4];
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), c2_uv, 1, 3, 0U, 1, 0U, 1, 4);
  for (c2_i = 0; c2_i < 4; c2_i++) {
    c2_y[c2_i] = c2_uv[c2_i];
  }

  sf_mex_destroy(&c2_u);
}

static c2_SL_Bus_ROSVariableLengthArrayInfo c2_d_emlrt_marshallIn
  (SFc2_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  static const char_T *c2_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c2_SL_Bus_ROSVariableLengthArrayInfo c2_y;
  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fParent = c2_parentId;
  c2_thisId.bParentIsCell = false;
  sf_mex_check_struct(c2_parentId, c2_u, 2, c2_fieldNames, 0U, NULL);
  c2_thisId.fIdentifier = "CurrentLength";
  c2_y.CurrentLength = c2_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "CurrentLength", "CurrentLength", 0)), &c2_thisId);
  c2_thisId.fIdentifier = "ReceivedLength";
  c2_y.ReceivedLength = c2_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "ReceivedLength", "ReceivedLength", 0)), &c2_thisId);
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static uint32_T c2_e_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint32_T c2_b_u;
  uint32_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 7, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static void c2_f_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real_T c2_y[7])
{
  real_T c2_dv[7];
  int32_T c2_i;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), c2_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c2_i = 0; c2_i < 7; c2_i++) {
    c2_y[c2_i] = c2_dv[c2_i];
  }

  sf_mex_destroy(&c2_u);
}

static uint8_T c2_g_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c2_b_is_active_c2_sliding_controller_rbdl, const
  char_T *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  uint8_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c2_b_is_active_c2_sliding_controller_rbdl), &c2_thisId);
  sf_mex_destroy(&c2_b_is_active_c2_sliding_controller_rbdl);
  return c2_y;
}

static uint8_T c2_h_emlrt_marshallIn(SFc2_sliding_controller_rbdlInstanceStruct *
  chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint8_T c2_b_u;
  uint8_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 3, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static void c2_eML_blk_kernel(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
  *c2_b_blankMsg)
{
  static uint8_T c2_uv[3] = { 101U, 120U, 111U };

  hipMemcpy(chartInstance->c2_gpu_blankMsg, c2_b_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  c2_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c2_gpu_blankMsg);
  hipMemcpy(*chartInstance->c2_gpu_uv, c2_uv, 3UL, hipMemcpyHostToDevice);
  c2_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c2_gpu_uv, chartInstance->c2_gpu_blankMsg);
  c2_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c2_gpu_blankMsg);
  hipMemcpy(c2_b_blankMsg, chartInstance->c2_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel1
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
   *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i = (int32_T)c2_threadId;
  if (c2_i < 4) {
    c2_b_msg->ModelName[c2_i] = c2_b_blankMsg->ModelName[c2_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel2
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
   *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i1;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i1 = (int32_T)c2_threadId;
  if (c2_i1 < 7) {
    c2_b_msg->Q[c2_i1] = c2_b_blankMsg->Q[c2_i1];
    c2_b_msg->Qd[c2_i1] = c2_b_blankMsg->Qd[c2_i1];
    c2_b_msg->Qdd[c2_i1] = c2_b_blankMsg->Qdd[c2_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel3
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg,
   c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_r)
{
  uint64_T c2_threadId;
  int32_T c2_i2;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i2 = (int32_T)c2_threadId;
  if (c2_i2 < 4) {
    c2_r->ModelName[c2_i2] = c2_b_msg->ModelName[c2_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel4
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg,
   c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_r)
{
  uint64_T c2_threadId;
  int32_T c2_i3;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i3 = (int32_T)c2_threadId;
  if (c2_i3 < 7) {
    c2_r->Q[c2_i3] = c2_b_msg->Q[c2_i3];
    c2_r->Qd[c2_i3] = c2_b_msg->Qd[c2_i3];
    c2_r->Qdd[c2_i3] = c2_b_msg->Qdd[c2_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel5(const
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_r,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i4;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i4 = (int32_T)c2_threadId;
  if (c2_i4 < 4) {
    c2_b_msg->ModelName[c2_i4] = c2_r->ModelName[c2_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_rbdl_kernel6(const
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_r,
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i5;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i5 = (int32_T)c2_threadId;
  if (c2_i5 < 7) {
    c2_b_msg->Q[c2_i5] = c2_r->Q[c2_i5];
    c2_b_msg->Qd[c2_i5] = c2_r->Qd[c2_i5];
    c2_b_msg->Qdd[c2_i5] = c2_r->Qdd[c2_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel7
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
   *c2_b_blankMsg)
{
  uint64_T c2_threadId;
  int32_T c2_tmpIdx;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_tmpIdx = (int32_T)c2_threadId;
  if (c2_tmpIdx < 1) {
    c2_b_blankMsg->ModelName_SL_Info.CurrentLength = 3U;
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel8(const
  uint8_T c2_uv[3],
  c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
  *c2_b_blankMsg)
{
  uint64_T c2_threadId;
  int32_T c2_i;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i = (int32_T)c2_threadId;
  if (c2_i < 3) {
    c2_b_blankMsg->ModelName[c2_i] = c2_uv[c2_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel9
  (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb
   *c2_b_blankMsg)
{
  uint64_T c2_threadId;
  int32_T c2_tmpIdx;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_tmpIdx = (int32_T)c2_threadId;
  if (c2_tmpIdx < 1) {
    c2_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
    c2_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
    c2_b_blankMsg->Qdd_SL_Info.CurrentLength = 7U;
  }
}

static void init_dsm_address_info(SFc2_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc2_sliding_controller_rbdlInstanceStruct *
  chartInstance)
{
  chartInstance->c2_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c2_blankMsg =
    (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c2_msg =
    (c2_SL_Bus_sliding_controller_rbdl_RBDLInverseDynamicsReques_yqmb *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c2_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(895492264U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3401536428U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2965174071U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(321400159U);
}

mxArray *sf_c2_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c2_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c2_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c2_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvwqK+yH8zOL4xOSSzLLU+GSj+OKczJTMvPT45Py8kqL8nJzUoviipJQcePwDAQA/hR5N"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c2_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQcVcJOajzDCVpNPzYn1u6B";
}

static void sf_opaque_initialize_c2_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c2_sliding_controller_rbdl
    ((SFc2_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c2_sliding_controller_rbdl
    ((SFc2_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c2_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c2_sliding_controller_rbdl((SFc2_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c2_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c2_sliding_controller_rbdl((SFc2_sliding_controller_rbdlInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c2_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c2_sliding_controller_rbdl
    ((SFc2_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c2_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c2_sliding_controller_rbdl
    ((SFc2_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c2_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c2_sliding_controller_rbdl
    ((SFc2_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c2_sliding_controller_rbdl(void *
  chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc2_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c2_sliding_controller_rbdl
      ((SFc2_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc2_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc2_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c2_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c2_sliding_controller_rbdl
    ((SFc2_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c2_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c2_sliding_controller_rbdl
    ((SFc2_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc2_sliding_controller_rbdl
    ((SFc2_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c2_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c2_sliding_controller_rbdl
      ((SFc2_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc2_sliding_controller_rbdl
      ((SFc2_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c2_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u00oUdkNBIF1QhZDYIMECCVYIWCBWUOIkIqilvbgtICSi6fgkHjKeMfOTtix4A5YseIu",
    "7uNv7GEh3dZ/hPgJnbCdNHduhRFSAsOS6Y3/nzDnnOz8Tb6m77uF1Ae/Vi553Bp9n8W542XU6Xy",
    "9N3dn7Ze9Gvv6AQsLGm0SRWHu1lyAxPAMtuTVMiq7oy1IYE31QIChiE6lMlTbNYsuZGHasoE6ff",
    "h4xGgWRtDxsoiwJNwQ/QG2JNZuop8UUUNMBCE2kpB1EHU4GE4uV2fMjoENt4zoXNJjAJs4svW65",
    "YQmH9j7QrtCGoMX60LbAEAO+2a9003mqgzFQxglnRJR6GxEdQIIBNrCdhPh3wxp0qgijEVGmCRE",
    "ZgV5jw1SnFFDUyTR+2GWCGKkY4e2Y+05w1rZNjvasyxB4TUDQtqYCMkwkE6aa/6CDnrYF2eXQgl",
    "07qNYWwFvryN9hsAeqMm59X45AkQFsiMpN04C091O2JlkyCzMshh2iHlHkT0NYmb2YOTogyBNso",
    "UQVDFInu3pLsRGGt1KbjbsuM+eVjI0zsvU8WKqtPYI6FibaOlT4hHNdCduSyRqMgKdaW8SQelim",
    "tRynNQu3JAbYpXd1NVjBkPgc5ksRslK6RgVA2neeYmM5iqRWGxn7mLyttbXZz7OwrjCg+oRCWRd",
    "QhGnAmKXhrdYWMu24RyBaZVLzysBZhsxDebpvRWtPqiHGpKaJHLrgGK0ExnqAXGIlbGssmjqY43",
    "IejhIaQegaDOOwjmWD2JKYaNfaHmHdjZg5aIGmiiUlrFqsOmxDbZdQBwlsi6GQe6KjZBzkPT4LL",
    "wBmG1GCiUET+5c66ODuZVS4eXbbO5xnf3zFPBvLFZ83p/Qslejxpp7Ffc816vdt4H9L4/k7JXe+",
    "sM9yQc7hVvD+9Lr95u9/Hn++9f4Vjz7+11hk/78ax5v/F/L1lXGjnRTOaCZfHfbxlF3LJfovT+l",
    "fydf6T7pDn2yQN+9a/k7ydPPdS3HH3mtm/Myxt1Gwd/z+muv4mGVpfiraDfODiVsTm41rp//+lL",
    "1n5sTjXP4+u/5/uJj8pdUij2XxOnUkXqdc5R7J22/f/9rqYvLZ/i/m2H+9wPf1dK73iOsW0KN3e",
    "5qzECu9R6UwSnIOqqd2Q16sz2/N9+PKeScs97PY+Vvu+/P+NfPrpOW8E5Zb1L/jzvEfDV83D7wC",
    "fuUH9mPR89X3xv/rHe8cdDVfP5j81PEjxsOSU2/+GY+w/bKvv0CefgGKtbTt",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c2_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c2_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1782931097U));
  ssSetChecksum1(S,(3578314164U));
  ssSetChecksum2(S,(1817935406U));
  ssSetChecksum3(S,(48141672U));
}

static void mdlRTW_c2_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c2_sliding_controller_rbdl(SimStruct *S)
{
  SFc2_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc2_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc2_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc2_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance) SFc2_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c2_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c2_JITStateAnimation,
    chartInstance->c2_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c2_sliding_controller_rbdl(chartInstance);
}

void c2_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c2_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c2_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c2_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c2_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
