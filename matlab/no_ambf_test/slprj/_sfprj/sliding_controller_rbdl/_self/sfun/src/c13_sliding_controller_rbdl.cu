/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c13_sliding_controller_rbdl.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c13_do_animation_call_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c13_st);
static void sf_gateway_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c13_eML_blk_kernel(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c13_b_q[2], real_T c13_b_y[4]);
static void c13_emlrt_marshallIn(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c13_b_y, const char_T *c13_identifier, real_T
  c13_c_y[4]);
static void c13_b_emlrt_marshallIn(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId,
  real_T c13_b_y[4]);
static uint8_T c13_c_emlrt_marshallIn
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c13_b_is_active_c13_sliding_controller_rbdl, const char_T *c13_identifier);
static uint8_T c13_d_emlrt_marshallIn
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c13_u, const emlrtMsgIdentifier *c13_parentId);
static void init_dsm_address_info(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c13_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c13_is_active_c13_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c13_do_animation_call_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c13_b_y = NULL;
  const mxArray *c13_c_y = NULL;
  const mxArray *c13_d_y = NULL;
  const mxArray *c13_st;
  c13_st = NULL;
  c13_b_y = NULL;
  sf_mex_assign(&c13_b_y, sf_mex_createcellmatrix(2, 1), false);
  c13_c_y = NULL;
  sf_mex_assign(&c13_c_y, sf_mex_create("y", *chartInstance->c13_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c13_b_y, 0, c13_c_y);
  c13_d_y = NULL;
  sf_mex_assign(&c13_d_y, sf_mex_create("y",
    &chartInstance->c13_is_active_c13_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c13_b_y, 1, c13_d_y);
  sf_mex_assign(&c13_st, c13_b_y, false);
  return c13_st;
}

static void set_sim_state_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c13_st)
{
  const mxArray *c13_u;
  c13_u = sf_mex_dup(c13_st);
  c13_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c13_u, 0)), "y",
                       *chartInstance->c13_y);
  chartInstance->c13_is_active_c13_sliding_controller_rbdl =
    c13_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c13_u, 1)),
    "is_active_c13_sliding_controller_rbdl");
  sf_mex_destroy(&c13_u);
  sf_mex_destroy(&c13_st);
}

static void sf_gateway_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c13_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c13_eML_blk_kernel(chartInstance, *chartInstance->c13_q, *chartInstance->c13_y);
  c13_do_animation_call_c13_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c13_errCode;
  c13_errCode = hipGetLastError();
  if (c13_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c13_errCode, (char_T *)hipGetErrorName
                       (c13_errCode), (char_T *)hipGetErrorString(c13_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c13_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc13_sliding_controller_rbdl
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c13_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c13_nameCaptureInfo = NULL;
  c13_nameCaptureInfo = NULL;
  sf_mex_assign(&c13_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c13_nameCaptureInfo;
}

static void c13_eML_blk_kernel(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c13_b_q[2], real_T c13_b_y[4])
{
  c13_b_y[0] = -0.324 * muDoubleScalarCos(c13_b_q[0]) + -0.39 *
    muDoubleScalarCos(c13_b_q[0] + c13_b_q[1]);
  c13_b_y[2] = 0.0;
  c13_b_y[1] = 0.324 * muDoubleScalarSin(c13_b_q[0]) - -0.39 * muDoubleScalarSin
    (c13_b_q[0] + c13_b_q[1]);
  c13_b_y[3] = 0.0;
}

static void c13_emlrt_marshallIn(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c13_b_y, const char_T *c13_identifier, real_T
  c13_c_y[4])
{
  emlrtMsgIdentifier c13_thisId;
  c13_thisId.fIdentifier = const_cast<const char_T *>(c13_identifier);
  c13_thisId.fParent = NULL;
  c13_thisId.bParentIsCell = false;
  c13_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c13_b_y), &c13_thisId,
    c13_c_y);
  sf_mex_destroy(&c13_b_y);
}

static void c13_b_emlrt_marshallIn(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId,
  real_T c13_b_y[4])
{
  real_T c13_dv[4];
  int32_T c13_i;
  sf_mex_import(c13_parentId, sf_mex_dup(c13_u), c13_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c13_i = 0; c13_i < 4; c13_i++) {
    c13_b_y[c13_i] = c13_dv[c13_i];
  }

  sf_mex_destroy(&c13_u);
}

static uint8_T c13_c_emlrt_marshallIn
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c13_b_is_active_c13_sliding_controller_rbdl, const char_T *c13_identifier)
{
  emlrtMsgIdentifier c13_thisId;
  uint8_T c13_b_y;
  c13_thisId.fIdentifier = const_cast<const char_T *>(c13_identifier);
  c13_thisId.fParent = NULL;
  c13_thisId.bParentIsCell = false;
  c13_b_y = c13_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c13_b_is_active_c13_sliding_controller_rbdl), &c13_thisId);
  sf_mex_destroy(&c13_b_is_active_c13_sliding_controller_rbdl);
  return c13_b_y;
}

static uint8_T c13_d_emlrt_marshallIn
  (SFc13_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c13_u, const emlrtMsgIdentifier *c13_parentId)
{
  uint8_T c13_b_u;
  uint8_T c13_b_y;
  sf_mex_import(c13_parentId, sf_mex_dup(c13_u), &c13_b_u, 1, 3, 0U, 0, 0U, 0);
  c13_b_y = c13_b_u;
  sf_mex_destroy(&c13_u);
  return c13_b_y;
}

static void init_dsm_address_info(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc13_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c13_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c13_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c13_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c13_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3354674951U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1529774139U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2473256034U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2602264389U);
}

mxArray *sf_c13_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c13_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c13_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c13_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cmGxvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB1+x2v"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c13_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sKQAtHLtNrhPdUXOtv0bf0";
}

static void sf_opaque_initialize_c13_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c13_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c13_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c13_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c13_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c13_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c13_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc13_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c13_sliding_controller_rbdl
      ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc13_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c13_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c13_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc13_sliding_controller_rbdl
    ((SFc13_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c13_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c13_sliding_controller_rbdl
      ((SFc13_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc13_sliding_controller_rbdl
      ((SFc13_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c13_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u00oUdqOCqASoQlesroANgmURG1bctvkREekPuAV20WR8Eo8ynvGdH7d5CCQehT0SD8A",
    "T8ArsWLNAnLGdNDi2Q4mo4Opacp2xv3PmnPOdn6m31t3z8LqO9/YNz7uMzyt4N7zsupSv1+bu7P",
    "26dy9fv0YhYaNDokikvdpLkAieg5bcGiZFVwxlKYyJISgQFLGxVKZKm2aR5UyMO1ZQp0+/DBkN/",
    "VBaHuyiLAkOBJ+gttiaQ9TTYgqo6QAEJlTSjsIOJ6OZxcqcNEOgY22jOhc0GN/Gziy9Z7lhMYf2",
    "KdCu0IagxfrMNt8QA01zWumm81T7U6CMYs6IKPU2JNqHGANs4DgO8O+BNehUEUZDoswuhCQB3WP",
    "jVKcUUNTJNH4YMEGMVIzwdsSbTnDRtkOO9uzJAHhNQNC2XQVkHEsmTDX/fgc9bQsy4NCCgR1Va/",
    "PhX+vIf8HgBFRl3IZNmYAiIzgQlZumAWmfpmzNsmQRZlgEL4jaocifhqAyezFztE+QJzhCiSoYp",
    "E529ZFiCYa3UpuNui4zl5WMjTKy9TJYqq2dQB0LM20dKpqEc10JO5JxDxLgqdYWMaQelmktx2nN",
    "giOJAXbpXV0NVjAkPoc1pQhYKV1JAZD2nX1sLN8jqdVGRk1M3lavt/h5EdYVBtSQUCjrAoowDRi",
    "zNLzV2gKmHfcIRKtMal4ZOMuQZShPD61onUg1xpjUNJEzFxyjlcBIj5BLrIRjjUVTB3NcLsNRQk",
    "MIXINhHPawbBBbEhPtWtsO1l3CzKQFmioWl7BqseqwDbVdQk1iOBZjIU9ER8nIz3t8Fl4AzDaiB",
    "BOjXexfatLB3cuocPNsyzubZ1d/YJ5N5YrP+3N61kr0eHPP4r4bjfp9G/hrbTp/5+SuFfZZL8g5",
    "3KYrBvN40vj0bv/9m40vX599uLXK/m8b55v/1/P139NGOyucZCFfHfbJnF3rJfpvzunfzNf66bM",
    "d86Rn9lV4GBy/OjDJ1mC4lfOzxN5Gwd7p+zuu42OWpfmpaDfIDyZuTWw2rp3+R3P2Xl4Sj438fX",
    "Z9/mc1+b+2izyuL5HfxF+TQt7+/P53tleTz/Z/tcT+uwW+76ZzvU9ct4A+ffCwrzkLsNT7VAqjJ",
    "Oeg+moQ8IKfP5vv55XzLljuT7Hzf7lfz/uPzK+LlvMuWG5V/847x383fN088Ar4zd/Yj1XPV78a",
    "/9E73znodr5+PPtXpxkyHpScevPPeIQdln39D+TpNzSrtJQ=",
    ""
  };

  static char newstr [1249] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c13_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c13_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2034136182U));
  ssSetChecksum1(S,(1320937730U));
  ssSetChecksum2(S,(4194930878U));
  ssSetChecksum3(S,(516575742U));
}

static void mdlRTW_c13_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c13_sliding_controller_rbdl(SimStruct *S)
{
  SFc13_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc13_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc13_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc13_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc13_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c13_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c13_JITStateAnimation,
    chartInstance->c13_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c13_sliding_controller_rbdl(chartInstance);
}

void c13_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c13_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c13_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c13_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c13_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
