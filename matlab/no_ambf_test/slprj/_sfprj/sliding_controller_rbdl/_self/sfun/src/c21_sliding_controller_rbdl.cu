/* Include files */

#include "sliding_controller_rbdl_sfun.h"
#include "c21_sliding_controller_rbdl.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initialize_params_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void enable_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void disable_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c21_do_animation_call_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void ext_mode_exec_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void set_sim_state_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c21_st);
static void sf_gateway_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_start_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_terminate_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void initSimStructsc21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance);
static void c21_eML_blk_kernel(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c21_b_u[6], real_T c21_b_y[7]);
static void c21_emlrt_marshallIn(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c21_b_y, const char_T *c21_identifier, real_T
  c21_c_y[7]);
static void c21_b_emlrt_marshallIn(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId,
  real_T c21_b_y[7]);
static uint8_T c21_c_emlrt_marshallIn
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c21_b_is_active_c21_sliding_controller_rbdl, const char_T *c21_identifier);
static uint8_T c21_d_emlrt_marshallIn
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c21_b_u, const emlrtMsgIdentifier *c21_parentId);
static void init_dsm_address_info(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c21_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c21_is_active_c21_sliding_controller_rbdl = 0U;
  hipGetLastError();
}

static void initialize_params_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void enable_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c21_do_animation_call_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  const mxArray *c21_b_y = NULL;
  const mxArray *c21_c_y = NULL;
  const mxArray *c21_d_y = NULL;
  const mxArray *c21_st;
  c21_st = NULL;
  c21_b_y = NULL;
  sf_mex_assign(&c21_b_y, sf_mex_createcellmatrix(2, 1), false);
  c21_c_y = NULL;
  sf_mex_assign(&c21_c_y, sf_mex_create("y", *chartInstance->c21_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c21_b_y, 0, c21_c_y);
  c21_d_y = NULL;
  sf_mex_assign(&c21_d_y, sf_mex_create("y",
    &chartInstance->c21_is_active_c21_sliding_controller_rbdl, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c21_b_y, 1, c21_d_y);
  sf_mex_assign(&c21_st, c21_b_y, false);
  return c21_st;
}

static void set_sim_state_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c21_st)
{
  const mxArray *c21_b_u;
  c21_b_u = sf_mex_dup(c21_st);
  c21_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c21_b_u, 0)),
                       "y", *chartInstance->c21_y);
  chartInstance->c21_is_active_c21_sliding_controller_rbdl =
    c21_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c21_b_u, 1)),
    "is_active_c21_sliding_controller_rbdl");
  sf_mex_destroy(&c21_b_u);
  sf_mex_destroy(&c21_st);
}

static void sf_gateway_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  chartInstance->c21_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c21_eML_blk_kernel(chartInstance, *chartInstance->c21_u, *chartInstance->c21_y);
  c21_do_animation_call_c21_sliding_controller_rbdl(chartInstance);
}

static void mdl_start_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  hipError_t c21_errCode;
  c21_errCode = hipGetLastError();
  if (c21_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c21_errCode, (char_T *)hipGetErrorName
                       (c21_errCode), (char_T *)hipGetErrorString(c21_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c21_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

static void initSimStructsc21_sliding_controller_rbdl
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance)
{
}

const mxArray *sf_c21_sliding_controller_rbdl_get_eml_resolved_functions_info()
{
  const mxArray *c21_nameCaptureInfo = NULL;
  c21_nameCaptureInfo = NULL;
  sf_mex_assign(&c21_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c21_nameCaptureInfo;
}

static void c21_eML_blk_kernel(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance, real_T c21_b_u[6], real_T c21_b_y[7])
{
  c21_b_y[0] = c21_b_u[3];
  c21_b_y[1] = 0.0;
  c21_b_y[2] = c21_b_u[0];
  c21_b_y[3] = c21_b_u[1];
  c21_b_y[4] = c21_b_u[4];
  c21_b_y[5] = c21_b_u[5];
  c21_b_y[6] = c21_b_u[2];
}

static void c21_emlrt_marshallIn(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c21_b_y, const char_T *c21_identifier, real_T
  c21_c_y[7])
{
  emlrtMsgIdentifier c21_thisId;
  c21_thisId.fIdentifier = const_cast<const char_T *>(c21_identifier);
  c21_thisId.fParent = NULL;
  c21_thisId.bParentIsCell = false;
  c21_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c21_b_y), &c21_thisId,
    c21_c_y);
  sf_mex_destroy(&c21_b_y);
}

static void c21_b_emlrt_marshallIn(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId,
  real_T c21_b_y[7])
{
  real_T c21_dv[7];
  int32_T c21_i;
  sf_mex_import(c21_parentId, sf_mex_dup(c21_b_u), c21_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c21_i = 0; c21_i < 7; c21_i++) {
    c21_b_y[c21_i] = c21_dv[c21_i];
  }

  sf_mex_destroy(&c21_b_u);
}

static uint8_T c21_c_emlrt_marshallIn
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c21_b_is_active_c21_sliding_controller_rbdl, const char_T *c21_identifier)
{
  emlrtMsgIdentifier c21_thisId;
  uint8_T c21_b_y;
  c21_thisId.fIdentifier = const_cast<const char_T *>(c21_identifier);
  c21_thisId.fParent = NULL;
  c21_thisId.bParentIsCell = false;
  c21_b_y = c21_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c21_b_is_active_c21_sliding_controller_rbdl), &c21_thisId);
  sf_mex_destroy(&c21_b_is_active_c21_sliding_controller_rbdl);
  return c21_b_y;
}

static uint8_T c21_d_emlrt_marshallIn
  (SFc21_sliding_controller_rbdlInstanceStruct *chartInstance, const mxArray
   *c21_b_u, const emlrtMsgIdentifier *c21_parentId)
{
  uint8_T c21_b_y;
  uint8_T c21_c_u;
  sf_mex_import(c21_parentId, sf_mex_dup(c21_b_u), &c21_c_u, 1, 3, 0U, 0, 0U, 0);
  c21_b_y = c21_c_u;
  sf_mex_destroy(&c21_b_u);
  return c21_b_y;
}

static void init_dsm_address_info(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc21_sliding_controller_rbdlInstanceStruct
  *chartInstance)
{
  chartInstance->c21_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c21_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c21_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c21_sliding_controller_rbdl_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3669229500U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(490348266U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2982423110U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1200777508U);
}

mxArray *sf_c21_sliding_controller_rbdl_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c21_sliding_controller_rbdl_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c21_sliding_controller_rbdl_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c21_sliding_controller_rbdl(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8clGhvHFOZkpmXnp8cn5eSVF+Tk5qUXxRUkpOUj+BAB12B2u"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c21_sliding_controller_rbdl_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sStKXtzIzHLETxtLQhJoVxG";
}

static void sf_opaque_initialize_c21_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  initialize_params_c21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
  initialize_c21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c21_sliding_controller_rbdl(void *chartInstanceVar)
{
  enable_c21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c21_sliding_controller_rbdl(void *chartInstanceVar)
{
  disable_c21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c21_sliding_controller_rbdl(void *chartInstanceVar)
{
  sf_gateway_c21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c21_sliding_controller_rbdl
  (SimStruct* S)
{
  return get_sim_state_c21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c21_sliding_controller_rbdl(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c21_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc21_sliding_controller_rbdlInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_rbdl_optimization_info();
    }

    mdl_cleanup_runtime_resources_c21_sliding_controller_rbdl
      ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
    ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar)->
      ~SFc21_sliding_controller_rbdlInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c21_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_start_c21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c21_sliding_controller_rbdl(void
  *chartInstanceVar)
{
  mdl_terminate_c21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc21_sliding_controller_rbdl
    ((SFc21_sliding_controller_rbdlInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c21_sliding_controller_rbdl(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c21_sliding_controller_rbdl
      ((SFc21_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc21_sliding_controller_rbdl
      ((SFc21_sliding_controller_rbdlInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c21_sliding_controller_rbdl_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV91u00gUdqNSgbRbVQiJGwTcoN3LBWklrqA0P5Al3XZxWqq9iabjk3iU8YyZnzThDfZi3wU",
    "J8R57uXfc7AP0EfaM7aTBsZ22ERUgLLnO2N85c875zs/UW2vvenht4r190/M28Hkd75qXXtey9d",
    "rcnb5f937K1n+jkLDRPlEk0l7lJUgEr0BLbg2Toi36shDGRB8UCIrYWCpTpk2zyHImhi0rqNOnX",
    "4eMhn4oLQ92UJYEe4JPUFtszT7qaTAF1LQAAhMqaQdhi5PBzGJlTuoh0KG2UZULGoxvY2eW3rXc",
    "sJhDcwy0LbQhaLE+s803xEDdjEvddJ5qfwqUUcwZEYXehkT7EGOADRzEAf7dswadysNoSJTZgZC",
    "MQHfYMNEpBeR1Mo0fjpkgRipGeDPidSe4aNs+R3t2ZQC8IiBo244CMowlE6acf7+FnjYFOebQgG",
    "M7KNfmwxvryD9kcAKqNG79uhyBIgPYE6WbJgFpjhO2ZlmyCDMsgkOinlHkT0NQmr2YOdonyBN0U",
    "aIMBomTbd1VbIThLdVmo7bLzGUlY6OUbL0MlmhrjqCKhZm2FhV1wrkuhXVl3IER8ERrgxhSDUu1",
    "FuO0ZkFXYoBdepdXgxUMic9gdSkCVkjXKAdI+s7v2Fg+RVKrjYzqmLyNTmfx8yKsLQyoPqFQ1AU",
    "UYRowZkl4y7UFTDvuEYhWmcS8InCaIctQnu5b0TiRaogxqWgiZy44RkuBkR4gl1gJBxqLpgrmuF",
    "yGo4SGELgGwzjsYtkgtiAm2rW2Z1h3I2YmDdBUsbiAVYtVh22o6RJqEsOBGAp5IlpKRn7W49PwA",
    "mC2ESWYGOxg/1KTFu5eRIWbZ794Z/Psh3PMs6lc/vnznJ61Aj3e3DO/741a9b41/LU2nb9zcj/m",
    "9lnPyTncFt5/ffQPP/75z393f30vbp1+uL3K/u9qF5v/m9n6zrTRzgpntJCvDvtizq71Av235/R",
    "vZWvtm5dH5m377YtOszs2nT/C3+Th+HnKzxJ7azl7p+/vu46PWZbkp6LtIDuYuDWx6bh2+h/P2b",
    "uxJB43svfpdfp0Nflb23ke15fIb+GvSS5vL7///e3V5NP9j5bY/yDH94NkrveI6xbQo48e9jRnA",
    "ZZ6j0phlOQcVE8dBzzn52Xz/aJy3hXLfS12fpf7/LyfZ35dtZx3xXKr+nfROf6l4avmgZfDb33B",
    "fqx6vvrc+H+9i52D7mXrJ7N/deoh40HBqTf7jEfYftHXbyBP/wctPLV+",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c21_sliding_controller_rbdl(SimStruct *S)
{
  const char* newstr = sf_c21_sliding_controller_rbdl_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(1448337289U));
  ssSetChecksum1(S,(3838859999U));
  ssSetChecksum2(S,(1857238302U));
  ssSetChecksum3(S,(397799444U));
}

static void mdlRTW_c21_sliding_controller_rbdl(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c21_sliding_controller_rbdl(SimStruct *S)
{
  SFc21_sliding_controller_rbdlInstanceStruct *chartInstance;
  chartInstance = (SFc21_sliding_controller_rbdlInstanceStruct *)utMalloc(sizeof
    (SFc21_sliding_controller_rbdlInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc21_sliding_controller_rbdlInstanceStruct));
  chartInstance = new (chartInstance)
    SFc21_sliding_controller_rbdlInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c21_sliding_controller_rbdl;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c21_JITStateAnimation,
    chartInstance->c21_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c21_sliding_controller_rbdl(chartInstance);
}

void c21_sliding_controller_rbdl_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c21_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c21_sliding_controller_rbdl(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c21_sliding_controller_rbdl(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c21_sliding_controller_rbdl_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
