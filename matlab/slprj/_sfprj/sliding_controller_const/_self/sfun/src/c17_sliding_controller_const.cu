#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c17_sliding_controller_const.h"
#include "MWCudaDimUtility.hpp"
#include "MWGpuFunctionImplUtility.hpp"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include "MWCUSOLVERUtils.hpp"
#include "hip/hip_math_constants.h"

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void c17_do_animation_call_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c17_st);
static void sf_gateway_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);
static void c17_eML_blk_kernel(SFc17_sliding_controller_constInstanceStruct
  *chartInstance, real_T c17_b_Kp0[7], real_T c17_b_Kn0[7], real_T c17_b_Bp0[7],
  real_T c17_b_Bn0[7], real_T c17_b_beta[2], real_T c17_b_alpha[2], real_T
  c17_b_Tid[7], real_T c17_b_Th[7], real_T c17_b_qd[7], real_T c17_b_K[49],
  real_T c17_b_B[49]);
static void c17_emlrt_marshallIn(SFc17_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c17_b_B, const char_T *c17_identifier, real_T
  c17_y[49]);
static void c17_b_emlrt_marshallIn(SFc17_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId,
  real_T c17_y[49]);
static uint8_T c17_c_emlrt_marshallIn
  (SFc17_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c17_b_is_active_c17_sliding_controller_const, const char_T *c17_identifier);
static uint8_T c17_d_emlrt_marshallIn
  (SFc17_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c17_u, const emlrtMsgIdentifier *c17_parentId);
static __global__ void c17_eML_blk_kernel_kernel1(real_T c17_b_Kp0[49]);
static __global__ void c17_eML_blk_kernel_kernel2(const real_T c17_b_Kp0[7],
  real_T c17_c_Kp0[49]);
static __global__ void c17_eML_blk_kernel_kernel3(real_T c17_b_Kn0[49]);
static __global__ void c17_eML_blk_kernel_kernel4(const real_T c17_b_Kn0[7],
  real_T c17_c_Kn0[49]);
static __global__ void c17_eML_blk_kernel_kernel5(real_T c17_b_Bp0[49]);
static __global__ void c17_eML_blk_kernel_kernel6(const real_T c17_b_Bp0[7],
  real_T c17_c_Bp0[49]);
static __global__ void c17_eML_blk_kernel_kernel7(real_T c17_b_Bn0[49]);
static __global__ void c17_eML_blk_kernel_kernel8(const real_T c17_b_Tid[7],
  const real_T c17_b_qd[7], const real_T c17_b_Th[7], const real_T c17_b_Bn0[7],
  ptrdiff_t c17_jpvt_t[7], real_T c17_A[7], real_T c17_x[7], real_T c17_b_x[7],
  real_T c17_c_Bn0[49]);
static __global__ void c17_eML_blk_kernel_kernel9(const ptrdiff_t c17_jpvt_t[7],
  int32_T c17_jpvt[7]);
static __global__ void c17_eML_blk_kernel_kernel10(int32_T c17_jpvt[7], real_T
  c17_A[7]);
static __global__ void c17_eML_blk_kernel_kernel11(real_T c17_change[49]);
static __global__ void c17_eML_blk_kernel_kernel12(const real_T c17_b_Th[7],
  real_T c17_b_B[7]);
static __global__ void c17_eML_blk_kernel_kernel13(real_T c17_b_B[7]);
static __global__ void c17_eML_blk_kernel_kernel14(const int32_T c17_jpvt[7],
  const int32_T c17_rankA, real_T c17_A[7], real_T c17_b_B[7], real_T
  c17_change[49]);
static __global__ void c17_eML_blk_kernel_kernel15(const real_T c17_change[49],
  real_T c17_Y[49]);
static __global__ void c17_eML_blk_kernel_kernel16(const real_T c17_Y[49],
  real_T c17_b_K[49], real_T c17_change[49]);
static __global__ void c17_eML_blk_kernel_kernel17(real_T c17_b_K[49]);
static __global__ void c17_eML_blk_kernel_kernel18(real_T c17_b_B[49]);
static __global__ void c17_eML_blk_kernel_kernel19(const real_T c17_b_Bn0[49],
  const real_T c17_b_alpha[2], const real_T c17_b_Bp0[49], const real_T
  c17_b_Kp0[49], const real_T c17_change[49], const real_T c17_b_beta[2], const
  real_T c17_b_Kn0[49], const real_T c17_x[7], const real_T c17_b_x[7], real_T
  c17_b_K[49], real_T c17_b_B[49]);
static void init_dsm_address_info(SFc17_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address
  (SFc17_sliding_controller_constInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c17_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c17_is_active_c17_sliding_controller_const = 0U;
  cusolverEnsureInitialization();
  hipGetLastError();
  hipMalloc(&chartInstance->c17_gpu_beta, 16UL);
  hipMalloc(&chartInstance->c17_gpu_alpha, 16UL);
  hipMalloc(&chartInstance->c17_gpu_B, 392UL);
  hipMalloc(&chartInstance->c17_gpu_K, 392UL);
  hipMalloc(&chartInstance->c17_gpu_Y, 392UL);
  hipMalloc(&chartInstance->c17_gpu_info_t, 4UL);
  hipMalloc(&chartInstance->c17_gpu_tau, 8UL);
  hipMalloc(&chartInstance->c17_b_gpu_B, 56UL);
  hipMalloc(&chartInstance->c17_gpu_change, 392UL);
  hipMalloc(&chartInstance->c17_gpu_jpvt, 28UL);
  hipMalloc(&chartInstance->c17_gpu_x, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_x, 56UL);
  hipMalloc(&chartInstance->c17_gpu_A, 56UL);
  hipMalloc(&chartInstance->c17_gpu_jpvt_t, 56UL);
  hipMalloc(&chartInstance->c17_gpu_Bn0, 56UL);
  hipMalloc(&chartInstance->c17_gpu_Th, 56UL);
  hipMalloc(&chartInstance->c17_gpu_qd, 56UL);
  hipMalloc(&chartInstance->c17_gpu_Tid, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_Bn0, 392UL);
  hipMalloc(&chartInstance->c17_gpu_Bp0, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_Bp0, 392UL);
  hipMalloc(&chartInstance->c17_gpu_Kn0, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_Kn0, 392UL);
  hipMalloc(&chartInstance->c17_gpu_Kp0, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_Kp0, 392UL);
}

static void initialize_params_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c17_do_animation_call_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c17_b_y = NULL;
  const mxArray *c17_c_y = NULL;
  const mxArray *c17_d_y = NULL;
  const mxArray *c17_st;
  const mxArray *c17_y = NULL;
  c17_st = NULL;
  c17_y = NULL;
  sf_mex_assign(&c17_y, sf_mex_createcellmatrix(3, 1), false);
  c17_b_y = NULL;
  sf_mex_assign(&c17_b_y, sf_mex_create("y", *chartInstance->c17_B, 0, 0U, 1U,
    0U, 2, 7, 7), false);
  sf_mex_setcell(c17_y, 0, c17_b_y);
  c17_c_y = NULL;
  sf_mex_assign(&c17_c_y, sf_mex_create("y", *chartInstance->c17_K, 0, 0U, 1U,
    0U, 2, 7, 7), false);
  sf_mex_setcell(c17_y, 1, c17_c_y);
  c17_d_y = NULL;
  sf_mex_assign(&c17_d_y, sf_mex_create("y",
    &chartInstance->c17_is_active_c17_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c17_y, 2, c17_d_y);
  sf_mex_assign(&c17_st, c17_y, false);
  return c17_st;
}

static void set_sim_state_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c17_st)
{
  const mxArray *c17_u;
  c17_u = sf_mex_dup(c17_st);
  c17_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 0)), "B",
                       *chartInstance->c17_B);
  c17_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 1)), "K",
                       *chartInstance->c17_K);
  chartInstance->c17_is_active_c17_sliding_controller_const =
    c17_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 2)),
    "is_active_c17_sliding_controller_const");
  sf_mex_destroy(&c17_u);
  sf_mex_destroy(&c17_st);
}

static void sf_gateway_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
  real_T c17_dv10[49];
  real_T c17_dv9[49];
  real_T c17_dv[7];
  real_T c17_dv1[7];
  real_T c17_dv2[7];
  real_T c17_dv3[7];
  real_T c17_dv6[7];
  real_T c17_dv7[7];
  real_T c17_dv8[7];
  real_T c17_dv4[2];
  real_T c17_dv5[2];
  int32_T c17_i;
  int32_T c17_i1;
  int32_T c17_i10;
  int32_T c17_i2;
  int32_T c17_i3;
  int32_T c17_i4;
  int32_T c17_i5;
  int32_T c17_i6;
  int32_T c17_i7;
  int32_T c17_i8;
  int32_T c17_i9;
  chartInstance->c17_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c17_i = 0; c17_i < 7; c17_i++) {
    c17_dv[c17_i] = (*chartInstance->c17_Kp0)[c17_i];
  }

  for (c17_i1 = 0; c17_i1 < 7; c17_i1++) {
    c17_dv1[c17_i1] = (*chartInstance->c17_Kn0)[c17_i1];
  }

  for (c17_i2 = 0; c17_i2 < 7; c17_i2++) {
    c17_dv2[c17_i2] = (*chartInstance->c17_Bp0)[c17_i2];
  }

  for (c17_i3 = 0; c17_i3 < 7; c17_i3++) {
    c17_dv3[c17_i3] = (*chartInstance->c17_Bn0)[c17_i3];
  }

  for (c17_i4 = 0; c17_i4 < 2; c17_i4++) {
    c17_dv4[c17_i4] = (*chartInstance->c17_beta)[c17_i4];
  }

  for (c17_i5 = 0; c17_i5 < 2; c17_i5++) {
    c17_dv5[c17_i5] = (*chartInstance->c17_alpha)[c17_i5];
  }

  for (c17_i6 = 0; c17_i6 < 7; c17_i6++) {
    c17_dv6[c17_i6] = (*chartInstance->c17_Tid)[c17_i6];
  }

  for (c17_i7 = 0; c17_i7 < 7; c17_i7++) {
    c17_dv7[c17_i7] = (*chartInstance->c17_Th)[c17_i7];
  }

  for (c17_i8 = 0; c17_i8 < 7; c17_i8++) {
    c17_dv8[c17_i8] = (*chartInstance->c17_qd)[c17_i8];
  }

  c17_eML_blk_kernel(chartInstance, c17_dv, c17_dv1, c17_dv2, c17_dv3, c17_dv4,
                     c17_dv5, c17_dv6, c17_dv7, c17_dv8, c17_dv9, c17_dv10);
  for (c17_i9 = 0; c17_i9 < 49; c17_i9++) {
    (*chartInstance->c17_B)[c17_i9] = c17_dv10[c17_i9];
  }

  for (c17_i10 = 0; c17_i10 < 49; c17_i10++) {
    (*chartInstance->c17_K)[c17_i10] = c17_dv9[c17_i10];
  }

  c17_do_animation_call_c17_sliding_controller_const(chartInstance);
}

static void mdl_start_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c17_errCode;
  hipFree(*chartInstance->c17_gpu_beta);
  hipFree(*chartInstance->c17_gpu_alpha);
  hipFree(*chartInstance->c17_gpu_B);
  hipFree(*chartInstance->c17_gpu_K);
  hipFree(*chartInstance->c17_gpu_Y);
  hipFree(chartInstance->c17_gpu_info_t);
  hipFree(chartInstance->c17_gpu_tau);
  hipFree(*chartInstance->c17_b_gpu_B);
  hipFree(*chartInstance->c17_gpu_change);
  hipFree(*chartInstance->c17_gpu_jpvt);
  hipFree(*chartInstance->c17_gpu_x);
  hipFree(*chartInstance->c17_b_gpu_x);
  hipFree(*chartInstance->c17_gpu_A);
  hipFree(*chartInstance->c17_gpu_jpvt_t);
  hipFree(*chartInstance->c17_gpu_Bn0);
  hipFree(*chartInstance->c17_gpu_Th);
  hipFree(*chartInstance->c17_gpu_qd);
  hipFree(*chartInstance->c17_gpu_Tid);
  hipFree(*chartInstance->c17_b_gpu_Bn0);
  hipFree(*chartInstance->c17_gpu_Bp0);
  hipFree(*chartInstance->c17_b_gpu_Bp0);
  hipFree(*chartInstance->c17_gpu_Kn0);
  hipFree(*chartInstance->c17_b_gpu_Kn0);
  hipFree(*chartInstance->c17_gpu_Kp0);
  hipFree(*chartInstance->c17_b_gpu_Kp0);
  c17_errCode = hipGetLastError();
  if (c17_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c17_errCode, (char_T *)hipGetErrorName
                       (c17_errCode), (char_T *)hipGetErrorString(c17_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c17_fEmlrtCtx);
  }

  cusolverDestroyWorkspace();
  cusolverEnsureDestruction();
}

static void mdl_setup_runtime_resources_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc17_sliding_controller_const
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c17_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c17_nameCaptureInfo = NULL;
  c17_nameCaptureInfo = NULL;
  sf_mex_assign(&c17_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c17_nameCaptureInfo;
}

static void c17_eML_blk_kernel(SFc17_sliding_controller_constInstanceStruct
  *chartInstance, real_T c17_b_Kp0[7], real_T c17_b_Kn0[7], real_T c17_b_Bp0[7],
  real_T c17_b_Bn0[7], real_T c17_b_beta[2], real_T c17_b_alpha[2], real_T
  c17_b_Tid[7], real_T c17_b_Th[7], real_T c17_b_qd[7], real_T c17_b_K[49],
  real_T c17_b_B[49])
{
  ptrdiff_t c17_jpvt_t[7];
  ptrdiff_t c17_info_t;
  real_T c17_A[7];
  real_T c17_tau;
  real_T c17_tol;
  int32_T c17_b_info_t;
  int32_T c17_rankA;
  boolean_T c17_A_dirtyOnCpu;
  boolean_T c17_A_dirtyOnGpu;
  c17_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Kp0);
  hipMemcpy(*chartInstance->c17_gpu_Kp0, c17_b_Kp0, 56UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Kp0, *chartInstance->c17_b_gpu_Kp0);
  c17_eML_blk_kernel_kernel3<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Kn0);
  hipMemcpy(*chartInstance->c17_gpu_Kn0, c17_b_Kn0, 56UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Kn0, *chartInstance->c17_b_gpu_Kn0);
  c17_eML_blk_kernel_kernel5<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Bp0);
  hipMemcpy(*chartInstance->c17_gpu_Bp0, c17_b_Bp0, 56UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Bp0, *chartInstance->c17_b_gpu_Bp0);
  c17_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Bn0);
  hipMemcpy(*chartInstance->c17_gpu_Tid, c17_b_Tid, 56UL,
             hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_qd, c17_b_qd, 56UL, hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_Th, c17_b_Th, 56UL, hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_Bn0, c17_b_Bn0, 56UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Tid, *chartInstance->c17_gpu_qd,
     *chartInstance->c17_gpu_Th, *chartInstance->c17_gpu_Bn0,
     *chartInstance->c17_gpu_jpvt_t, *chartInstance->c17_gpu_A,
     *chartInstance->c17_b_gpu_x, *chartInstance->c17_gpu_x,
     *chartInstance->c17_b_gpu_Bn0);
  hipMemcpy(c17_A, *chartInstance->c17_gpu_A, 56UL, hipMemcpyDeviceToHost);
  hipMemcpy(c17_jpvt_t, *chartInstance->c17_gpu_jpvt_t, 56UL,
             hipMemcpyDeviceToHost);
  c17_info_t = LAPACKE_dgeqp3(102, (ptrdiff_t)1, (ptrdiff_t)7, &c17_A[0],
    (ptrdiff_t)1, &c17_jpvt_t[0], &c17_tau);
  c17_A_dirtyOnGpu = false;
  c17_A_dirtyOnCpu = true;
  if ((int32_T)c17_info_t != 0) {
    c17_tau = rtNaN;
    c17_eML_blk_kernel_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_gpu_jpvt, *chartInstance->c17_gpu_A);
    c17_A_dirtyOnCpu = false;
    c17_A_dirtyOnGpu = true;
  } else {
    hipMemcpy(*chartInstance->c17_gpu_jpvt_t, c17_jpvt_t, 56UL,
               hipMemcpyHostToDevice);
    c17_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_gpu_jpvt_t, *chartInstance->c17_gpu_jpvt);
  }

  c17_rankA = 0;
  if (c17_A_dirtyOnGpu) {
    hipMemcpy(c17_A, *chartInstance->c17_gpu_A, 56UL, hipMemcpyDeviceToHost);
  }

  c17_tol = 1.5543122344752192E-14 * muDoubleScalarAbs(c17_A[0]);
  if (!(muDoubleScalarAbs(c17_A[0]) <= c17_tol)) {
    c17_rankA = 1;
  }

  c17_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Th, *chartInstance->c17_b_gpu_B);
  c17_b_info_t = 0;
  if (c17_A_dirtyOnCpu) {
    hipMemcpy(*chartInstance->c17_gpu_A, c17_A, 56UL, hipMemcpyHostToDevice);
  }

  hipMemcpy(chartInstance->c17_gpu_tau, &c17_tau, 8UL, hipMemcpyHostToDevice);
  hipsolverDnDormqr_bufferSize(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT,
    HIPBLAS_OP_T, 1, 7, 1, (double *)&(*chartInstance->c17_gpu_A)[0], 1, (double *)
    chartInstance->c17_gpu_tau, (double *)&(*chartInstance->c17_b_gpu_B)[0], 1,
    getCuSolverWorkspaceReq());
  setCuSolverWorkspaceTypeSize(8);
  cusolverInitWorkspace();
  hipMemcpy(chartInstance->c17_gpu_info_t, &c17_b_info_t, 4UL,
             hipMemcpyHostToDevice);
  hipsolverDnDormqr(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, 1,
                   7, 1, (double *)&(*chartInstance->c17_gpu_A)[0], 1, (double *)
                   chartInstance->c17_gpu_tau, (double *)
                   &(*chartInstance->c17_b_gpu_B)[0], 1, (real_T *)
                   getCuSolverWorkspaceBuff(), *getCuSolverWorkspaceReq(),
                   chartInstance->c17_gpu_info_t);
  hipMemcpy(&c17_b_info_t, chartInstance->c17_gpu_info_t, 4UL,
             hipMemcpyDeviceToHost);
  if (c17_b_info_t < 0) {
    c17_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_b_gpu_B);
  }

  c17_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_jpvt, c17_rankA, *chartInstance->c17_gpu_A,
     *chartInstance->c17_b_gpu_B, *chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel15<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_change, *chartInstance->c17_gpu_Y);
  c17_eML_blk_kernel_kernel16<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Y, *chartInstance->c17_gpu_K,
     *chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_K);
  c17_eML_blk_kernel_kernel18<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_B);
  hipMemcpy(*chartInstance->c17_gpu_alpha, c17_b_alpha, 16UL,
             hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_beta, c17_b_beta, 16UL,
             hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_b_gpu_Bn0, *chartInstance->c17_gpu_alpha,
     *chartInstance->c17_b_gpu_Bp0, *chartInstance->c17_b_gpu_Kp0,
     *chartInstance->c17_gpu_change, *chartInstance->c17_gpu_beta,
     *chartInstance->c17_b_gpu_Kn0, *chartInstance->c17_b_gpu_x,
     *chartInstance->c17_gpu_x, *chartInstance->c17_gpu_K,
     *chartInstance->c17_gpu_B);
  hipMemcpy(c17_b_K, *chartInstance->c17_gpu_K, 392UL, hipMemcpyDeviceToHost);
  hipMemcpy(c17_b_B, *chartInstance->c17_gpu_B, 392UL, hipMemcpyDeviceToHost);
}

static void c17_emlrt_marshallIn(SFc17_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c17_b_B, const char_T *c17_identifier, real_T
  c17_y[49])
{
  emlrtMsgIdentifier c17_thisId;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c17_b_B), &c17_thisId, c17_y);
  sf_mex_destroy(&c17_b_B);
}

static void c17_b_emlrt_marshallIn(SFc17_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId,
  real_T c17_y[49])
{
  real_T c17_dv[49];
  int32_T c17_i;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), c17_dv, 1, 0, 0U, 1, 0U, 2, 7,
                7);
  for (c17_i = 0; c17_i < 49; c17_i++) {
    c17_y[c17_i] = c17_dv[c17_i];
  }

  sf_mex_destroy(&c17_u);
}

static uint8_T c17_c_emlrt_marshallIn
  (SFc17_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c17_b_is_active_c17_sliding_controller_const, const char_T *c17_identifier)
{
  emlrtMsgIdentifier c17_thisId;
  uint8_T c17_y;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_y = c17_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c17_b_is_active_c17_sliding_controller_const), &c17_thisId);
  sf_mex_destroy(&c17_b_is_active_c17_sliding_controller_const);
  return c17_y;
}

static uint8_T c17_d_emlrt_marshallIn
  (SFc17_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c17_u, const emlrtMsgIdentifier *c17_parentId)
{
  uint8_T c17_b_u;
  uint8_T c17_y;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), &c17_b_u, 1, 3, 0U, 0, 0U, 0);
  c17_y = c17_b_u;
  sf_mex_destroy(&c17_u);
  return c17_y;
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel1
  (real_T c17_b_Kp0[49])
{
  uint64_T c17_threadId;
  int32_T c17_i;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i = (int32_T)c17_threadId;
  if (c17_i < 49) {
    c17_b_Kp0[c17_i] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel2(const
  real_T c17_b_Kp0[7], real_T c17_c_Kp0[49])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_c_Kp0[c17_j + 7 * c17_j] = c17_b_Kp0[c17_j];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel3
  (real_T c17_b_Kn0[49])
{
  uint64_T c17_threadId;
  int32_T c17_i1;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i1 = (int32_T)c17_threadId;
  if (c17_i1 < 49) {
    c17_b_Kn0[c17_i1] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel4(const
  real_T c17_b_Kn0[7], real_T c17_c_Kn0[49])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_c_Kn0[c17_j + 7 * c17_j] = c17_b_Kn0[c17_j];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel5
  (real_T c17_b_Bp0[49])
{
  uint64_T c17_threadId;
  int32_T c17_i2;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i2 = (int32_T)c17_threadId;
  if (c17_i2 < 49) {
    c17_b_Bp0[c17_i2] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel6(const
  real_T c17_b_Bp0[7], real_T c17_c_Bp0[49])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_c_Bp0[c17_j + 7 * c17_j] = c17_b_Bp0[c17_j];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel7
  (real_T c17_b_Bn0[49])
{
  uint64_T c17_threadId;
  int32_T c17_i3;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i3 = (int32_T)c17_threadId;
  if (c17_i3 < 49) {
    c17_b_Bn0[c17_i3] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel8(const
  real_T c17_b_Tid[7], const real_T c17_b_qd[7], const real_T c17_b_Th[7], const
  real_T c17_b_Bn0[7], ptrdiff_t c17_jpvt_t[7], real_T c17_A[7], real_T c17_x[7],
  real_T c17_b_x[7], real_T c17_c_Bn0[49])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_c_Bn0[c17_j + 7 * c17_j] = c17_b_Bn0[c17_j];
    c17_b_x[c17_j] = c17_b_Th[c17_j];
    c17_b_x[c17_j] = mwGpuSign(c17_b_x[c17_j]);
    c17_x[c17_j] = c17_b_qd[c17_j];
    c17_x[c17_j] = mwGpuSign(c17_x[c17_j]);
    c17_A[c17_j] = c17_b_Tid[c17_j];
    c17_jpvt_t[c17_j] = (ptrdiff_t)0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel9(const
  ptrdiff_t c17_jpvt_t[7], int32_T c17_jpvt[7])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    c17_jpvt[c17_k] = (int32_T)c17_jpvt_t[c17_k];
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel10
  (int32_T c17_jpvt[7], real_T c17_A[7])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_A[c17_j] = HIP_NAN;
    c17_jpvt[c17_j] = c17_j + 1;
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel11
  (real_T c17_change[49])
{
  uint64_T c17_threadId;
  int32_T c17_i4;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i4 = (int32_T)c17_threadId;
  if (c17_i4 < 49) {
    c17_change[c17_i4] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel12(
  const real_T c17_b_Th[7], real_T c17_b_B[7])
{
  uint64_T c17_threadId;
  int32_T c17_i5;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i5 = (int32_T)c17_threadId;
  if (c17_i5 < 7) {
    c17_b_B[c17_i5] = c17_b_Th[c17_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel13
  (real_T c17_b_B[7])
{
  uint64_T c17_threadId;
  int32_T c17_i6;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i6 = (int32_T)c17_threadId;
  if (c17_i6 < 7) {
    c17_b_B[c17_i6] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel14(
  const int32_T c17_jpvt[7], const int32_T c17_rankA, real_T c17_A[7], real_T
  c17_b_B[7], real_T c17_change[49])
{
  uint64_T c17_threadId;
  int32_T c17_i;
  int32_T c17_j;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    for (c17_i = 0; c17_i < c17_rankA; c17_i++) {
      c17_change[(c17_jpvt[0] + 7 * c17_k) - 1] = c17_b_B[c17_k];
    }

    for (c17_j = 0; c17_j < c17_rankA; c17_j++) {
      c17_change[(c17_jpvt[c17_rankA - 1] + 7 * c17_k) - 1] /= c17_A[(c17_rankA
        + c17_rankA) - 2];
    }
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel15(
  const real_T c17_change[49], real_T c17_Y[49])
{
  uint64_T c17_threadId;
  int32_T c17_i7;
  int32_T c17_i8;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i8 = (int32_T)(c17_threadId % 7UL);
  c17_i7 = (int32_T)((c17_threadId - (uint64_T)c17_i8) / 7UL);
  if ((c17_i7 < 7) && (c17_i8 < 7)) {
    c17_Y[c17_i8 + 7 * c17_i7] = c17_change[c17_i7 + 7 * c17_i8];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel16(
  const real_T c17_Y[49], real_T c17_b_K[49], real_T c17_change[49])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 49) {
    c17_change[c17_k] = fabs(c17_Y[c17_k]);
    c17_b_K[c17_k] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel17
  (real_T c17_b_K[49])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    c17_b_K[c17_k + 7 * c17_k] = 1.0;
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel18
  (real_T c17_b_B[49])
{
  uint64_T c17_threadId;
  int32_T c17_i9;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i9 = (int32_T)c17_threadId;
  if (c17_i9 < 49) {
    c17_b_B[c17_i9] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel19(
  const real_T c17_b_Bn0[49], const real_T c17_b_alpha[2], const real_T
  c17_b_Bp0[49], const real_T c17_b_Kp0[49], const real_T c17_change[49], const
  real_T c17_b_beta[2], const real_T c17_b_Kn0[49], const real_T c17_x[7], const
  real_T c17_b_x[7], real_T c17_b_K[49], real_T c17_b_B[49])
{
  uint64_T c17_threadId;
  int32_T c17_ii;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_ii = (int32_T)c17_threadId;
  if (c17_ii < 7) {
    c17_b_B[c17_ii + 7 * c17_ii] = 1.0;
    if (c17_b_x[c17_ii] == c17_x[c17_ii]) {
      c17_b_K[c17_ii + 7 * c17_ii] = c17_b_Kp0[c17_ii + 7 * c17_ii] +
        c17_b_beta[0] * c17_change[c17_ii];
      c17_b_B[c17_ii + 7 * c17_ii] = c17_b_Bp0[c17_ii + 7 * c17_ii] +
        c17_b_alpha[0] * c17_change[c17_ii];
    } else {
      c17_b_K[c17_ii + 7 * c17_ii] = c17_b_Kn0[c17_ii + 7 * c17_ii] -
        c17_b_beta[1] * c17_change[c17_ii];
      c17_b_B[c17_ii + 7 * c17_ii] = c17_b_Bn0[c17_ii + 7 * c17_ii] -
        c17_b_alpha[1] * c17_change[c17_ii];
    }
  }
}

static void init_dsm_address_info(SFc17_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address
  (SFc17_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c17_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c17_Kp0 = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c17_Kn0 = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_Bp0 = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c17_Bn0 = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c17_beta = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 4);
  chartInstance->c17_alpha = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 5);
  chartInstance->c17_Tid = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 6);
  chartInstance->c17_K = (real_T (*)[49])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_Th = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 7);
  chartInstance->c17_qd = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 8);
  chartInstance->c17_B = (real_T (*)[49])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c17_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(184866213U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3943974275U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1621223579U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3836141733U);
}

mxArray *sf_c17_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,1);
  mxSetCell(mxcell3p, 0, mxCreateString("coder.internal.lapack.LAPACKApi"));
  return(mxcell3p);
}

mxArray *sf_c17_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("late");
  mxArray *hiddenFallbackReason = mxCreateString("ir_function_calls");
  mxArray *incompatibleSymbol = mxCreateString("__gpu_kernel");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c17_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c17_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiA+wMTAwAakOYCYiQECWKF8RiBmhtIQcRa4uAIQl1QWpILEi4uSPVOAdF5iLpi"
    "fWFrhmZeWDzbfggFhPhsW8xmRzOeEikPAB3vK9Cs5gPQbIOlnIaBfAMhygoYLLHwGzv0iZLnfm2"
    "ruV3CgTD/E/ggC7ldDcT+En1kcn5hcklmWGp9saB5fnJOZkpmXHp+cn1dSlJ+Tk1oEYhaXIPwJA"
    "BBDIhM="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c17_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sZ5qVxZToNvS2ZkiuiWRnEB";
}

static void sf_opaque_initialize_c17_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c17_sliding_controller_const(void *chartInstanceVar)
{
  enable_c17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c17_sliding_controller_const(void
  *chartInstanceVar)
{
  disable_c17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c17_sliding_controller_const(void
  *chartInstanceVar)
{
  sf_gateway_c17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c17_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c17_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c17_sliding_controller_const
  (void *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc17_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c17_sliding_controller_const
      ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc17_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c17_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c17_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc17_sliding_controller_const
    ((SFc17_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c17_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c17_sliding_controller_const
      ((SFc17_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
    initSimStructsc17_sliding_controller_const
      ((SFc17_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
  }
}

const char* sf_c17_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV8tu20YUpRQ3iIEkENoC3RRI0EWaZRs0aFepoxdK1I6NUHYKb4zx8EocaDjDzIO2/yHddNE",
    "PaH8iy3xG/iGbfELvkJQsUyQVW02aFiVAkUOee3juY+6MvJa/4+FxG8/OZ553Ha838Gx7+fFJMW",
    "4tnPnzDe/rYvwrGgkb7xFFYu01HoLE8BS05NYwKXwxlpUwJsagQFDEJlKZOjbNYsuZmA6toI5PP",
    "4sYjYJIWh520ZaEu4KfIVtizR7y9JkCaoYAoYmUtJNoyMlkrliZk14EdKpt3OSCBhPYxMnSO5Yb",
    "lnAYnAL1hTYEFetzbYEhBnrmtNZN56kOZkAZJ5wRUeltRHQACQbYwH4S4u+uNehUGUYjokwXIpK",
    "C3mbTjFMKKHMyjS+OmSBGKkb4IOY9Z7isbY+jnh0ZAm8ICGrrKiDTRDJh6vMfDNHTgSDHHPpwbC",
    "f1bAE8ty75BwxOQNXGbdyTKSgygV1R+9EsIIPTLFvzKlmGGRbDAVGPKeZPQ1hbvVg5OiCYJxihR",
    "R0MMid9PVIsxfDWstnYd5W5asrYOE+2XgXL2AYpNGVhzjakokc417WwkUy2IQWesfaJIc2wnLUa",
    "pzULRxID7Mq7fjZYwTDxBawnRcgq05WWAFnfeYKN5SKSWm1k3MPi7W9vL79ehvnCgBoTClVdQBG",
    "mAWOWhbeeLWTa5R6BqMpk8qrAeYWsQnl6bEX/RKopxqShiZy74DJaC4z1BHOJM2Ff46Rpgrlcrs",
    "JRQiMIXYNhHHZw2iC2IibatbbHOO9SZs76oKliSUVWLc46bEMDV1BnCeyLqZAnYqhkHBQ9Pg8vA",
    "FYbUYKJSRf7lzob4terUuHWs2+88/Xs5jusZzO78vX+Ak+rgsdbuDr81gJ+s30Rv1n6bnv2bHbU",
    "8NwqfXejxONwHTxffvrwjxvtB/7vf77wv/vtDS3HoayntaSnld07u+Ta5fYDt4vxl7PGO59I6VL",
    "9OuxPC7o2Kvi/WODvFGN9+PD5wenhSD5JgweHU2bZs6di0M34XrWb9V4r6Z09v+tWAKy6rF4V9c",
    "Nio+LGxObLt+P/YUHv9RXx2Cye58fbH9ez/2qrnMeNFfYdvOuW6vif0//5lfT//Lfpv7u1nn3+/",
    "V9W6L9Xqtd72T7liLjuB0f02++PNGchtq4jKoVRknNQ7labi35edb5e1s77wHb/Fp3/273/vL/L",
    "erx5Rbv2muv/h7Jb17/L7ks+NnzTeuCV8J2P2I9194vvG//au9w+7k4xfjT/69aLGA8rdvHFa9y",
    "Sj6ve/gfq9C9HpbeM",
    ""
  };

  static char newstr [1293] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c17_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c17_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2721387192U));
  ssSetChecksum1(S,(1228014088U));
  ssSetChecksum2(S,(1233953686U));
  ssSetChecksum3(S,(1676382772U));
}

static void mdlRTW_c17_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c17_sliding_controller_const(SimStruct *S)
{
  SFc17_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc17_sliding_controller_constInstanceStruct *)utMalloc
    (sizeof(SFc17_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc17_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc17_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c17_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c17_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c17_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c17_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c17_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c17_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c17_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c17_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c17_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c17_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c17_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c17_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c17_JITStateAnimation,
    chartInstance->c17_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c17_sliding_controller_const(chartInstance);
}

void c17_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c17_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c17_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c17_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c17_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
