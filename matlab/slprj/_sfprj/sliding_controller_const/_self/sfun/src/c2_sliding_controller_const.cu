#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c2_sliding_controller_const.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void c2_do_animation_call_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c2_st);
static void sf_gateway_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance);
static void c2_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_b_msg, const char_T *c2_identifier,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_y);
static void c2_b_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_y);
static void c2_c_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  uint8_T c2_y[4]);
static c2_SL_Bus_ROSVariableLengthArrayInfo c2_d_emlrt_marshallIn
  (SFc2_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c2_u, const emlrtMsgIdentifier *c2_parentId);
static uint32_T c2_e_emlrt_marshallIn
  (SFc2_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c2_u, const emlrtMsgIdentifier *c2_parentId);
static void c2_f_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real_T c2_y[7]);
static uint8_T c2_g_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_sliding_controller_const,
  const char_T *c2_identifier);
static uint8_T c2_h_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static void c2_eML_blk_kernel(SFc2_sliding_controller_constInstanceStruct
  *chartInstance,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
  *c2_b_blankMsg);
static __global__ void c2_sf_gateway_c2_sliding_controller_const_kernel1
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
   *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg);
static __global__ void c2_sf_gateway_c2_sliding_controller_const_kernel2
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
   *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg);
static __global__ void c2_sf_gateway_c2_sliding_controller_const_kernel3
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg,
   c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_r);
static __global__ void c2_sf_gateway_c2_sliding_controller_const_kernel4
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg,
   c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_r);
static __global__ void c2_sf_gateway_c2_sliding_controller_const_kernel5(const
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_r,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg);
static __global__ void c2_sf_gateway_c2_sliding_controller_const_kernel6(const
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_r,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg);
static __global__ void c2_eML_blk_kernel_kernel7
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
   *c2_b_blankMsg);
static __global__ void c2_eML_blk_kernel_kernel8(const uint8_T c2_uv[3],
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
  *c2_b_blankMsg);
static __global__ void c2_eML_blk_kernel_kernel9
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
   *c2_b_blankMsg);
static void init_dsm_address_info(SFc2_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc2_sliding_controller_constInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c2_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c2_is_active_c2_sliding_controller_const = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c2_gpu_uv, 3UL);
  hipMalloc(&chartInstance->c2_gpu_blankMsg, 208UL);
}

static void initialize_params_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c2_do_animation_call_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
  static const char_T *c2_sv[8] = { "ModelName", "ModelName_SL_Info", "Q",
    "Q_SL_Info", "Qd", "Qd_SL_Info", "Qdd", "Qdd_SL_Info" };

  static const char_T *c2_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c2_b_y = NULL;
  const mxArray *c2_c_y = NULL;
  const mxArray *c2_d_y = NULL;
  const mxArray *c2_e_y = NULL;
  const mxArray *c2_f_y = NULL;
  const mxArray *c2_g_y = NULL;
  const mxArray *c2_h_y = NULL;
  const mxArray *c2_i_y = NULL;
  const mxArray *c2_j_y = NULL;
  const mxArray *c2_k_y = NULL;
  const mxArray *c2_l_y = NULL;
  const mxArray *c2_m_y = NULL;
  const mxArray *c2_n_y = NULL;
  const mxArray *c2_o_y = NULL;
  const mxArray *c2_p_y = NULL;
  const mxArray *c2_q_y = NULL;
  const mxArray *c2_r_y = NULL;
  const mxArray *c2_s_y = NULL;
  const mxArray *c2_st;
  const mxArray *c2_y = NULL;
  real_T c2_d_u[7];
  int32_T c2_i;
  int32_T c2_i1;
  int32_T c2_i2;
  int32_T c2_i3;
  uint32_T c2_b_u;
  uint32_T c2_c_u;
  uint32_T c2_e_u;
  uint32_T c2_f_u;
  uint32_T c2_g_u;
  uint32_T c2_h_u;
  uint32_T c2_i_u;
  uint32_T c2_j_u;
  uint32_T c2_u_CurrentLength;
  uint32_T c2_u_ReceivedLength;
  uint8_T c2_u[4];
  c2_st = NULL;
  c2_y = NULL;
  sf_mex_assign(&c2_y, sf_mex_createcellmatrix(2, 1), false);
  c2_b_y = NULL;
  sf_mex_assign(&c2_b_y, sf_mex_createstruct("structure", 8, c2_sv, 2, 1, 1),
                false);
  for (c2_i = 0; c2_i < 4; c2_i++) {
    c2_u[c2_i] = ((uint8_T *)&((char_T *)chartInstance->c2_msg)[0])[c2_i];
  }

  c2_c_y = NULL;
  sf_mex_assign(&c2_c_y, sf_mex_create("y", c2_u, 3, 0U, 1U, 0U, 1, 4), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "ModelName", c2_c_y, 0);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [4];
  c2_d_y = NULL;
  sf_mex_assign(&c2_d_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_b_u = c2_u_CurrentLength;
  c2_e_y = NULL;
  sf_mex_assign(&c2_e_y, sf_mex_create("y", &c2_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_d_y, 0, "CurrentLength", c2_e_y, 0);
  c2_c_u = c2_u_ReceivedLength;
  c2_f_y = NULL;
  sf_mex_assign(&c2_f_y, sf_mex_create("y", &c2_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_d_y, 0, "ReceivedLength", c2_f_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "ModelName_SL_Info", c2_d_y, 1);
  for (c2_i1 = 0; c2_i1 < 7; c2_i1++) {
    c2_d_u[c2_i1] = ((real_T *)&((char_T *)chartInstance->c2_msg)[16])[c2_i1];
  }

  c2_g_y = NULL;
  sf_mex_assign(&c2_g_y, sf_mex_create("y", c2_d_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "Q", c2_g_y, 2);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[4];
  c2_h_y = NULL;
  sf_mex_assign(&c2_h_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_e_u = c2_u_CurrentLength;
  c2_i_y = NULL;
  sf_mex_assign(&c2_i_y, sf_mex_create("y", &c2_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_h_y, 0, "CurrentLength", c2_i_y, 0);
  c2_f_u = c2_u_ReceivedLength;
  c2_j_y = NULL;
  sf_mex_assign(&c2_j_y, sf_mex_create("y", &c2_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_h_y, 0, "ReceivedLength", c2_j_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "Q_SL_Info", c2_h_y, 3);
  for (c2_i2 = 0; c2_i2 < 7; c2_i2++) {
    c2_d_u[c2_i2] = ((real_T *)&((char_T *)chartInstance->c2_msg)[80])[c2_i2];
  }

  c2_k_y = NULL;
  sf_mex_assign(&c2_k_y, sf_mex_create("y", c2_d_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qd", c2_k_y, 4);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[4];
  c2_l_y = NULL;
  sf_mex_assign(&c2_l_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_g_u = c2_u_CurrentLength;
  c2_m_y = NULL;
  sf_mex_assign(&c2_m_y, sf_mex_create("y", &c2_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_l_y, 0, "CurrentLength", c2_m_y, 0);
  c2_h_u = c2_u_ReceivedLength;
  c2_n_y = NULL;
  sf_mex_assign(&c2_n_y, sf_mex_create("y", &c2_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_l_y, 0, "ReceivedLength", c2_n_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qd_SL_Info", c2_l_y, 5);
  for (c2_i3 = 0; c2_i3 < 7; c2_i3++) {
    c2_d_u[c2_i3] = ((real_T *)&((char_T *)chartInstance->c2_msg)[144])[c2_i3];
  }

  c2_o_y = NULL;
  sf_mex_assign(&c2_o_y, sf_mex_create("y", c2_d_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qdd", c2_o_y, 6);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[4];
  c2_p_y = NULL;
  sf_mex_assign(&c2_p_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_i_u = c2_u_CurrentLength;
  c2_q_y = NULL;
  sf_mex_assign(&c2_q_y, sf_mex_create("y", &c2_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_p_y, 0, "CurrentLength", c2_q_y, 0);
  c2_j_u = c2_u_ReceivedLength;
  c2_r_y = NULL;
  sf_mex_assign(&c2_r_y, sf_mex_create("y", &c2_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_p_y, 0, "ReceivedLength", c2_r_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qdd_SL_Info", c2_p_y, 7);
  sf_mex_setcell(c2_y, 0, c2_b_y);
  c2_s_y = NULL;
  sf_mex_assign(&c2_s_y, sf_mex_create("y",
    &chartInstance->c2_is_active_c2_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c2_y, 1, c2_s_y);
  sf_mex_assign(&c2_st, c2_y, false);
  return c2_st;
}

static void set_sim_state_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c2_st)
{
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm c2_r;
  const mxArray *c2_u;
  int32_T c2_i;
  int32_T c2_i1;
  c2_u = sf_mex_dup(c2_st);
  c2_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 0)), "msg",
                      &c2_r);
  for (c2_i = 0; c2_i < 4; c2_i++) {
    ((uint8_T *)&((char_T *)chartInstance->c2_msg)[0])[c2_i] =
      c2_r.ModelName[c2_i];
  }

  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[0] = c2_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[4] = c2_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[0] = c2_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[4] = c2_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[0] = c2_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[4] = c2_r.Qd_SL_Info.ReceivedLength;
  for (c2_i1 = 0; c2_i1 < 7; c2_i1++) {
    ((real_T *)&((char_T *)chartInstance->c2_msg)[16])[c2_i1] = c2_r.Q[c2_i1];
    ((real_T *)&((char_T *)chartInstance->c2_msg)[80])[c2_i1] = c2_r.Qd[c2_i1];
    ((real_T *)&((char_T *)chartInstance->c2_msg)[144])[c2_i1] = c2_r.Qdd[c2_i1];
  }

  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[0] = c2_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[4] = c2_r.Qdd_SL_Info.ReceivedLength;
  chartInstance->c2_is_active_c2_sliding_controller_const =
    c2_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 1)),
    "is_active_c2_sliding_controller_const");
  sf_mex_destroy(&c2_u);
  sf_mex_destroy(&c2_st);
}

static void sf_gateway_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm c2_r;
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
    *c2_b_gpu_blankMsg;
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_gpu_msg;
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_gpu_r;
  hipMalloc(&c2_gpu_r, 208UL);
  hipMalloc(&c2_gpu_msg, 208UL);
  hipMalloc(&c2_b_gpu_blankMsg, 208UL);
  chartInstance->c2_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(c2_b_gpu_blankMsg, chartInstance->c2_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_const_kernel1<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_b_gpu_blankMsg, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c2_blankMsg, c2_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[8])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[8])[4];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[72])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[72])[4];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[136])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[136])[4];
  hipMemcpy(c2_b_gpu_blankMsg, chartInstance->c2_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_const_kernel2<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_b_gpu_blankMsg, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c2_blankMsg, c2_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[200])[4];
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_const_kernel3<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_gpu_msg, c2_gpu_r);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c2_r, c2_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c2_r.ModelName_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [0];
  c2_r.ModelName_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [4];
  c2_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[0];
  c2_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[4];
  c2_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[0];
  c2_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[4];
  hipMemcpy(c2_gpu_r, &c2_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_const_kernel4<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_gpu_msg, c2_gpu_r);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c2_r, c2_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c2_r.Qdd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[0];
  c2_r.Qdd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[4];
  c2_eML_blk_kernel(chartInstance, &c2_r);
  hipMemcpy(c2_gpu_r, &c2_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_const_kernel5<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_gpu_r, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[0] = c2_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[4] = c2_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[0] = c2_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[4] = c2_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[0] = c2_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[4] = c2_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller_const_kernel6<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c2_gpu_r, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[0] = c2_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[4] = c2_r.Qdd_SL_Info.ReceivedLength;
  c2_do_animation_call_c2_sliding_controller_const(chartInstance);
  hipFree(c2_b_gpu_blankMsg);
  hipFree(c2_gpu_msg);
  hipFree(c2_gpu_r);
}

static void mdl_start_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c2_errCode;
  hipFree(*chartInstance->c2_gpu_uv);
  hipFree(chartInstance->c2_gpu_blankMsg);
  c2_errCode = hipGetLastError();
  if (c2_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c2_errCode, (char_T *)hipGetErrorName
                       (c2_errCode), (char_T *)hipGetErrorString(c2_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c2_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc2_sliding_controller_const
  (SFc2_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c2_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c2_nameCaptureInfo = NULL;
  c2_nameCaptureInfo = NULL;
  sf_mex_assign(&c2_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c2_nameCaptureInfo;
}

static void c2_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_b_msg, const char_T *c2_identifier,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_y)
{
  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c2_b_msg), &c2_thisId, c2_y);
  sf_mex_destroy(&c2_b_msg);
}

static void c2_b_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_y)
{
  static const char_T *c2_fieldNames[8] = { "ModelName", "ModelName_SL_Info",
    "Q", "Q_SL_Info", "Qd", "Qd_SL_Info", "Qdd", "Qdd_SL_Info" };

  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fParent = c2_parentId;
  c2_thisId.bParentIsCell = false;
  sf_mex_check_struct(c2_parentId, c2_u, 8, c2_fieldNames, 0U, NULL);
  c2_thisId.fIdentifier = "ModelName";
  c2_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u,
    "ModelName", "ModelName", 0)), &c2_thisId, c2_y->ModelName);
  c2_thisId.fIdentifier = "ModelName_SL_Info";
  c2_y->ModelName_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "ModelName_SL_Info", "ModelName_SL_Info", 0)),
    &c2_thisId);
  c2_thisId.fIdentifier = "Q";
  c2_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u, "Q", "Q",
    0)), &c2_thisId, c2_y->Q);
  c2_thisId.fIdentifier = "Q_SL_Info";
  c2_y->Q_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "Q_SL_Info", "Q_SL_Info", 0)), &c2_thisId);
  c2_thisId.fIdentifier = "Qd";
  c2_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u, "Qd",
    "Qd", 0)), &c2_thisId, c2_y->Qd);
  c2_thisId.fIdentifier = "Qd_SL_Info";
  c2_y->Qd_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c2_thisId);
  c2_thisId.fIdentifier = "Qdd";
  c2_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u, "Qdd",
    "Qdd", 0)), &c2_thisId, c2_y->Qdd);
  c2_thisId.fIdentifier = "Qdd_SL_Info";
  c2_y->Qdd_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "Qdd_SL_Info", "Qdd_SL_Info", 0)), &c2_thisId);
  sf_mex_destroy(&c2_u);
}

static void c2_c_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  uint8_T c2_y[4])
{
  int32_T c2_i;
  uint8_T c2_uv[4];
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), c2_uv, 1, 3, 0U, 1, 0U, 1, 4);
  for (c2_i = 0; c2_i < 4; c2_i++) {
    c2_y[c2_i] = c2_uv[c2_i];
  }

  sf_mex_destroy(&c2_u);
}

static c2_SL_Bus_ROSVariableLengthArrayInfo c2_d_emlrt_marshallIn
  (SFc2_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  static const char_T *c2_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c2_SL_Bus_ROSVariableLengthArrayInfo c2_y;
  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fParent = c2_parentId;
  c2_thisId.bParentIsCell = false;
  sf_mex_check_struct(c2_parentId, c2_u, 2, c2_fieldNames, 0U, NULL);
  c2_thisId.fIdentifier = "CurrentLength";
  c2_y.CurrentLength = c2_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "CurrentLength", "CurrentLength", 0)), &c2_thisId);
  c2_thisId.fIdentifier = "ReceivedLength";
  c2_y.ReceivedLength = c2_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "ReceivedLength", "ReceivedLength", 0)), &c2_thisId);
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static uint32_T c2_e_emlrt_marshallIn
  (SFc2_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint32_T c2_b_u;
  uint32_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 7, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static void c2_f_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real_T c2_y[7])
{
  real_T c2_dv[7];
  int32_T c2_i;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), c2_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c2_i = 0; c2_i < 7; c2_i++) {
    c2_y[c2_i] = c2_dv[c2_i];
  }

  sf_mex_destroy(&c2_u);
}

static uint8_T c2_g_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_sliding_controller_const,
  const char_T *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  uint8_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c2_b_is_active_c2_sliding_controller_const), &c2_thisId);
  sf_mex_destroy(&c2_b_is_active_c2_sliding_controller_const);
  return c2_y;
}

static uint8_T c2_h_emlrt_marshallIn(SFc2_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint8_T c2_b_u;
  uint8_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 3, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static void c2_eML_blk_kernel(SFc2_sliding_controller_constInstanceStruct
  *chartInstance,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
  *c2_b_blankMsg)
{
  static uint8_T c2_uv[3] = { 101U, 120U, 111U };

  hipMemcpy(chartInstance->c2_gpu_blankMsg, c2_b_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  c2_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c2_gpu_blankMsg);
  hipMemcpy(*chartInstance->c2_gpu_uv, c2_uv, 3UL, hipMemcpyHostToDevice);
  c2_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c2_gpu_uv, chartInstance->c2_gpu_blankMsg);
  c2_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c2_gpu_blankMsg);
  hipMemcpy(c2_b_blankMsg, chartInstance->c2_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_const_kernel1
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
   *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i = (int32_T)c2_threadId;
  if (c2_i < 4) {
    c2_b_msg->ModelName[c2_i] = c2_b_blankMsg->ModelName[c2_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_const_kernel2
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
   *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i1;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i1 = (int32_T)c2_threadId;
  if (c2_i1 < 7) {
    c2_b_msg->Q[c2_i1] = c2_b_blankMsg->Q[c2_i1];
    c2_b_msg->Qd[c2_i1] = c2_b_blankMsg->Qd[c2_i1];
    c2_b_msg->Qdd[c2_i1] = c2_b_blankMsg->Qdd[c2_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_const_kernel3
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg,
   c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_r)
{
  uint64_T c2_threadId;
  int32_T c2_i2;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i2 = (int32_T)c2_threadId;
  if (c2_i2 < 4) {
    c2_r->ModelName[c2_i2] = c2_b_msg->ModelName[c2_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_const_kernel4
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg,
   c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_r)
{
  uint64_T c2_threadId;
  int32_T c2_i3;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i3 = (int32_T)c2_threadId;
  if (c2_i3 < 7) {
    c2_r->Q[c2_i3] = c2_b_msg->Q[c2_i3];
    c2_r->Qd[c2_i3] = c2_b_msg->Qd[c2_i3];
    c2_r->Qdd[c2_i3] = c2_b_msg->Qdd[c2_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_const_kernel5(const
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_r,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i4;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i4 = (int32_T)c2_threadId;
  if (c2_i4 < 4) {
    c2_b_msg->ModelName[c2_i4] = c2_r->ModelName[c2_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller_const_kernel6(const
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_r,
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i5;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i5 = (int32_T)c2_threadId;
  if (c2_i5 < 7) {
    c2_b_msg->Q[c2_i5] = c2_r->Q[c2_i5];
    c2_b_msg->Qd[c2_i5] = c2_r->Qd[c2_i5];
    c2_b_msg->Qdd[c2_i5] = c2_r->Qdd[c2_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel7
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
   *c2_b_blankMsg)
{
  uint64_T c2_threadId;
  int32_T c2_tmpIdx;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_tmpIdx = (int32_T)c2_threadId;
  if (c2_tmpIdx < 1) {
    c2_b_blankMsg->ModelName_SL_Info.CurrentLength = 3U;
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel8(const
  uint8_T c2_uv[3],
  c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
  *c2_b_blankMsg)
{
  uint64_T c2_threadId;
  int32_T c2_i;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i = (int32_T)c2_threadId;
  if (c2_i < 3) {
    c2_b_blankMsg->ModelName[c2_i] = c2_uv[c2_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel9
  (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm
   *c2_b_blankMsg)
{
  uint64_T c2_threadId;
  int32_T c2_tmpIdx;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_tmpIdx = (int32_T)c2_threadId;
  if (c2_tmpIdx < 1) {
    c2_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
    c2_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
    c2_b_blankMsg->Qdd_SL_Info.CurrentLength = 7U;
  }
}

static void init_dsm_address_info(SFc2_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc2_sliding_controller_constInstanceStruct
  *chartInstance)
{
  chartInstance->c2_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c2_blankMsg =
    (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c2_msg =
    (c2_SL_Bus_sliding_controller_const_RBDLInverseDynamicsReques_9pm *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c2_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1643287079U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2194062691U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(495046957U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2677343194U);
}

mxArray *sf_c2_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c2_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c2_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c2_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvyqK+yH8zOL4xOSSzLLU+GSj+OKczJTMvPT45Py8kqL8nJzUIhCzuAQW/0AAAEYnHtI="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c2_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "ssuB11YlGjgEXxkbnZ3FZSD";
}

static void sf_opaque_initialize_c2_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c2_sliding_controller_const(void *chartInstanceVar)
{
  enable_c2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c2_sliding_controller_const(void *chartInstanceVar)
{
  disable_c2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c2_sliding_controller_const(void *chartInstanceVar)
{
  sf_gateway_c2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c2_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c2_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c2_sliding_controller_const(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc2_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c2_sliding_controller_const
      ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc2_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c2_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c2_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc2_sliding_controller_const
    ((SFc2_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c2_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c2_sliding_controller_const
      ((SFc2_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc2_sliding_controller_const
      ((SFc2_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c2_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV0tv00AQdkNBIAGqEBISQqKXCsSF14UTtHlBpJZWuOUloWq7nsRL1rtmH2n7Czhx4DfwB/g",
    "fnDgg8TO4cWXWdtLUsR1KRAUIS66z9jezM/PNY+vNddY8vM7jvXzB807h8zTeNS+9TmbrubE7fT",
    "/vXcvW71BI2GiDKBJpr/ISJIInoCW3hknREV1ZCGOiCwoERWwslSnTpllkORP9thXU6dPPQkZDP",
    "5SWB3WUJcG64PuoLbZmA/U0mQJq2gCBCZW0vbDNSW9ksTK7jRBoX9uoygUNxrexM0uvWW5YzKG1",
    "B7QjtCFosT6wzTfEQMPslbrpPNX+ECijmDMiCr0NifYhxgAb2IoD/LtuDTqVh9GQKFOHkAxAr7J",
    "+olMKyOtkGj/sMEGMVIzwVsQbTnDStg2O9qzJAHhFQNC2ugLSjyUTppx/v42etgTZ4dCEHdsr1+",
    "bDG+vIf8pgF1Rp3LoNOQBFerAuSjdNAtLaS9gaZckkzLAInhK1QpE/DUFp9mLmaJ8gT7CJEmUwS",
    "Jzs6E3FBhjeUm026rjMnFYyNkrJ1tNgibbWAKpYGGlrU9EgnOtS2KaMV2EAPNHaJIZUw1KtxTit",
    "WbApMcAuvcurwQqGxGewhhQBK6RrkAMkfecxNpbDSGq1kVEDk7e5ujr5eRLWEQZUl1Ao6gKKMA0",
    "YsyS85doCph33CESrTGJeETjNkGkoT3etaO5K1ceYVDSRAxcco6XASPeQS6yELY1FUwVzXE7DUU",
    "JDCFyDYRzWsGwQWxAT7VrbCtbdgJn9JmiqWFzAqsWqwzbUcgm1H8OW6Au5K9pKRn7W49PwAmC2E",
    "SWY6NWxf6n9Nu5eRIWbZ7e8g3l29ifm2VAu/7w+pmeuQI839szve6ZWvW8Nf80N5++Y3LncPvM5",
    "OYdbcLg++/6ef7755dWny/bD2xuz7P+xdrT5fz5bXxk22lHhDCby1WEfjdk1X6D/0pj+hWytta3",
    "fvv2CP3zdaz3f6++Il3fbL/1mys8Ue2s5e4fvF13HxyxL8lPRTpAdTNya2HRcO/33xuw9NSUeZ7",
    "L36fXtwWzyF5fzPBbF68SheJ1wlXsob399/8Xl2eTT/Z9PsX8px/dSMte3iesWsE3vbGvOAqz0b",
    "SqFUZJzUO6nNrn6/NV8P6qcd8xyf4ud/+V+P+8/M7+OW847ZrlZ/TvqHP/T8FXzwMvhF/5gP2Y9",
    "X/1u/FfvaOegq9n6/uhfnUbIeFBw6s0+4xG2W/T1H8jTH5rBtYs=",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c2_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c2_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(4150881037U));
  ssSetChecksum1(S,(802253969U));
  ssSetChecksum2(S,(449535445U));
  ssSetChecksum3(S,(713532277U));
}

static void mdlRTW_c2_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c2_sliding_controller_const(SimStruct *S)
{
  SFc2_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc2_sliding_controller_constInstanceStruct *)utMalloc(sizeof
    (SFc2_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc2_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc2_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c2_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c2_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c2_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c2_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c2_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c2_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c2_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c2_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c2_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c2_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c2_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c2_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c2_JITStateAnimation,
    chartInstance->c2_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c2_sliding_controller_const(chartInstance);
}

void c2_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c2_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c2_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c2_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c2_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
