/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c19_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void c19_do_animation_call_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c19_st);
static void sf_gateway_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);
static void c19_eML_blk_kernel(SFc19_sliding_controller_constInstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2]);
static void c19_emlrt_marshallIn(SFc19_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2]);
static void c19_b_emlrt_marshallIn(SFc19_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2]);
static uint8_T c19_c_emlrt_marshallIn
  (SFc19_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c19_b_is_active_c19_sliding_controller_const, const char_T *c19_identifier);
static uint8_T c19_d_emlrt_marshallIn
  (SFc19_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c19_u, const emlrtMsgIdentifier *c19_parentId);
static void init_dsm_address_info(SFc19_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address
  (SFc19_sliding_controller_constInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c19_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c19_is_active_c19_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c19_do_animation_call_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c19_b_y = NULL;
  const mxArray *c19_c_y = NULL;
  const mxArray *c19_d_y = NULL;
  const mxArray *c19_e_y = NULL;
  const mxArray *c19_f_y = NULL;
  const mxArray *c19_st;
  const mxArray *c19_y = NULL;
  c19_st = NULL;
  c19_y = NULL;
  sf_mex_assign(&c19_y, sf_mex_createcellmatrix(5, 1), false);
  c19_b_y = NULL;
  sf_mex_assign(&c19_b_y, sf_mex_create("y", *chartInstance->c19_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 0, c19_b_y);
  c19_c_y = NULL;
  sf_mex_assign(&c19_c_y, sf_mex_create("y", *chartInstance->c19_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 1, c19_c_y);
  c19_d_y = NULL;
  sf_mex_assign(&c19_d_y, sf_mex_create("y", *chartInstance->c19_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 2, c19_d_y);
  c19_e_y = NULL;
  sf_mex_assign(&c19_e_y, sf_mex_create("y", *chartInstance->c19_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c19_y, 3, c19_e_y);
  c19_f_y = NULL;
  sf_mex_assign(&c19_f_y, sf_mex_create("y",
    &chartInstance->c19_is_active_c19_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c19_y, 4, c19_f_y);
  sf_mex_assign(&c19_st, c19_y, false);
  return c19_st;
}

static void set_sim_state_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c19_st)
{
  const mxArray *c19_u;
  c19_u = sf_mex_dup(c19_st);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 0)),
                       "ankle", *chartInstance->c19_ankle);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 1)),
                       "danke", *chartInstance->c19_danke);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 2)),
                       "dknee", *chartInstance->c19_dknee);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 3)),
                       "knee", *chartInstance->c19_knee);
  chartInstance->c19_is_active_c19_sliding_controller_const =
    c19_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 4)),
    "is_active_c19_sliding_controller_const");
  sf_mex_destroy(&c19_u);
  sf_mex_destroy(&c19_st);
}

static void sf_gateway_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c19_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c19_eML_blk_kernel(chartInstance, *chartInstance->c19_q,
                     *chartInstance->c19_qd, *chartInstance->c19_knee,
                     *chartInstance->c19_dknee, *chartInstance->c19_ankle,
                     *chartInstance->c19_danke);
  c19_do_animation_call_c19_sliding_controller_const(chartInstance);
}

static void mdl_start_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c19_errCode;
  c19_errCode = hipGetLastError();
  if (c19_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c19_errCode, (char_T *)hipGetErrorName
                       (c19_errCode), (char_T *)hipGetErrorString(c19_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c19_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc19_sliding_controller_const
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c19_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c19_nameCaptureInfo = NULL;
  const char_T *c19_data[4] = {
    "789cc553cb4ac34014bd955adc545d09825b372e1cdd886b8b850a2db50f5c88c69b64d20c9d874c52dbbaf113fc1dbfc7853bffc1246d9a26182216eb8170e7"
    "7032f7dc7b60a0d4689600603bf83e5e00ce2b10a13a2bb03baf1b9046562fe5d4189b504edd0bf5f7c0ef75ce2d257d3af16744a2a08b9bb6124ca2f47bd347",
    "0a9a7a8a3f513b521cc6698f09da5d26ad9089fa92b420a1149e6b2eb586dd9100ed7ac9847c9944798478c8d9b75c904716d93cb2ffc57e935ffac5fdf70afc"
    "62dd517a8cda36864c32813eb33c83d3c1ea7bc7be95dc39668aad4626a789dfdb8a7ebd5cbfb47e7b79475c252891e8bb2819e503c56d07b5492cf483348cb1",
    "473c6d1114a6638c910fa92641401c4df27d66c722ec5b94dbce0ff7c87b3f55d88aea677b1fd7e9777f7470b84ebf18ffe5b7aef777d3bec093b3e72bd6e9e8"
    "7af3badf6df44f5bb5648e76814fd11c90c3ffbaff177b596734", "" };

  c19_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c19_data[0], 1616U, &c19_nameCaptureInfo);
  return c19_nameCaptureInfo;
}

static void c19_eML_blk_kernel(SFc19_sliding_controller_constInstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2])
{
  real_T c19_a;
  real_T c19_b_a;
  c19_b_knee[0] = -0.324 * muDoubleScalarCos(c19_b_q[0]);
  c19_b_knee[1] = -0.324 * muDoubleScalarSin(c19_b_q[0]);
  c19_a = -0.324 * c19_b_qd[0];
  c19_b_dknee[0] = c19_a * -muDoubleScalarSin(c19_b_q[0]);
  c19_b_dknee[1] = c19_a * muDoubleScalarCos(c19_b_q[0]);
  c19_b_a = -0.39 * (c19_b_qd[0] + c19_b_qd[1]);
  c19_b_ankle[0] = c19_b_knee[0] - -0.39 * muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_ankle[1] = c19_b_knee[1] - -0.39 * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[0] = c19_b_dknee[0] + c19_b_a * -muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[1] = c19_b_dknee[1] + c19_b_a * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
}

static void c19_emlrt_marshallIn(SFc19_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2])
{
  emlrtMsgIdentifier c19_thisId;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c19_b_ankle), &c19_thisId,
    c19_y);
  sf_mex_destroy(&c19_b_ankle);
}

static void c19_b_emlrt_marshallIn(SFc19_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2])
{
  real_T c19_dv[2];
  int32_T c19_i;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), c19_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c19_i = 0; c19_i < 2; c19_i++) {
    c19_y[c19_i] = c19_dv[c19_i];
  }

  sf_mex_destroy(&c19_u);
}

static uint8_T c19_c_emlrt_marshallIn
  (SFc19_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c19_b_is_active_c19_sliding_controller_const, const char_T *c19_identifier)
{
  emlrtMsgIdentifier c19_thisId;
  uint8_T c19_y;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_y = c19_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c19_b_is_active_c19_sliding_controller_const), &c19_thisId);
  sf_mex_destroy(&c19_b_is_active_c19_sliding_controller_const);
  return c19_y;
}

static uint8_T c19_d_emlrt_marshallIn
  (SFc19_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c19_u, const emlrtMsgIdentifier *c19_parentId)
{
  uint8_T c19_b_u;
  uint8_T c19_y;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), &c19_b_u, 1, 3, 0U, 0, 0U, 0);
  c19_y = c19_b_u;
  sf_mex_destroy(&c19_u);
  return c19_y;
}

static void init_dsm_address_info(SFc19_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address
  (SFc19_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c19_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c19_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c19_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c19_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c19_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c19_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c19_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c19_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c19_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c19_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6HFA4ifWRyfmFySWZYan"
    "2xoGV+ck5mSmZcen5yfV1KUn5OTWgRiFpcg4gkAqMIvFg=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c19_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c19_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c19_sliding_controller_const(void *chartInstanceVar)
{
  enable_c19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c19_sliding_controller_const(void
  *chartInstanceVar)
{
  disable_c19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c19_sliding_controller_const(void
  *chartInstanceVar)
{
  sf_gateway_c19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c19_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c19_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c19_sliding_controller_const
  (void *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc19_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c19_sliding_controller_const
      ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc19_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c19_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c19_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc19_sliding_controller_const
    ((SFc19_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c19_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c19_sliding_controller_const
      ((SFc19_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
    initSimStructsc19_sliding_controller_const
      ((SFc19_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
  }
}

const char* sf_c19_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFAL0EbFEXaSwEHQYHm0tqmJESFHTulnAQ51FgvR+JCy112f2ir75C",
    "ce+m9D9BLn6BAn6CnPkORJ8gsScsyRVKR3aRuURq0tOQ33843Mzu78lqDXQ+vm3j/eMvzruHndb",
    "zbXn6tFePW3J0/73ifFuMXaCRsvE8UibXXeAkSw7egJbeGSTEQI1kJY2IECgRFbCKVqWPTLLaci",
    "UnfCur49NOI0SiIpOXhNtqScE/wKbIl1uwjT5cpoKYPEJpISTuO+pyMZx4rc+xHQCfaxk0SNJjA",
    "Js4tvWu5YQmH3gnQgdCGoMf6zLfAEAO+OamV6ZTq4BQo44QzIirVRkQHkGCADRwkIf7fswZFlWE",
    "0IspsQ0RS0DtsknFKAWVOpvHFERPESMUI78Xcd4aLvu1z9GdXhsAbAoK+bSsgk0QyYerzH/RRaU",
    "+QIw5dOLLjerYAvrcu+U8YHIOqjdvIlykoMoY9UTtpFpDeSZatWZUswgyL4QlRWxTzpyGsrV6sH",
    "B0QzBMM0aIOBpnIgR4qlmJ4a9lsPHCVuWzJ2DhPtl4Gy9h6KTRlYcbWp8InnOta2FAmO5ACz1i7",
    "xJBmWM5ajdOahUOJAXblXb8arGCY+ALmSxGyynSlJUDWdx5hYzmPpFYbGftYvN2dncXXi7CBMKB",
    "GhEJVF1CEacCYZeGtZwuZdrlHIHplMveqwHmFLEN5emRF91iqCcakoYmcSXAZrQXGeoy5xJVwoH",
    "HRNMFcLpfhKKERhK7BMA67uGwQWxET7VrbFq67lJlpFzRVLKnIqsVVh22o5wpqmsCBmAh5LPpKx",
    "kHR4/PwAmC1ESWYGG9j/1LTPs5elQq3n214Z/vZe2+wn53alT8/m+NpVfB4c5/leW+0z+PbpXnb",
    "+Ndq5Xabc3bvl+bplOwcbh3vb/786ZNHH/7y6rffP385/fnX75bN31qYv5V9d3bX11bb/28W49u",
    "njXa2cNKFenXYh3N+dSr4P5jjXy/G+nH0hUifP4y2HpNhPNi4/8ODjfTIz/iedZr9XSv5e/r8ju",
    "v4WGVZfSo6CIuDiRsTm2/Xjv/LOX+vLYnHjeJ5fv319eXsb2+W7TtL9K0XYyImHEr1+8/p+PjCO",
    "kIUcnV03Lm4jgkeGq6Mjlub5f5QpaNzTkfHcxL+Hv8X47iafT7/syX+3y3l4W52XjwkbheCQ3rv",
    "waHmLMQt5JBKYZTkHJT7qs35PF20j65q571ju3+Ln//bvf28v8m5qH1Bu84lz2Hvyu6y+lY9H14",
    "1fNN+4JXw61dYx2XP7W8b/4e32vn6o2L81ewntB8xHlb8mipe40+jUdXb/0CdvgZyvcYW",
    ""
  };

  static char newstr [1345] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c19_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c19_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(614325834U));
  ssSetChecksum1(S,(4071890510U));
  ssSetChecksum2(S,(2385365703U));
  ssSetChecksum3(S,(1589225593U));
}

static void mdlRTW_c19_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c19_sliding_controller_const(SimStruct *S)
{
  SFc19_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc19_sliding_controller_constInstanceStruct *)utMalloc
    (sizeof(SFc19_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc19_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc19_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c19_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c19_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c19_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c19_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c19_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c19_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c19_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c19_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c19_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c19_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c19_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c19_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c19_JITStateAnimation,
    chartInstance->c19_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c19_sliding_controller_const(chartInstance);
}

void c19_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c19_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c19_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c19_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c19_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
