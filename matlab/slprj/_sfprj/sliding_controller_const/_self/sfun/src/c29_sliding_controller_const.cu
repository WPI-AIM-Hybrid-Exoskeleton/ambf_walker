/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c29_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void c29_do_animation_call_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c29_st);
static void sf_gateway_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);
static void c29_eML_blk_kernel(SFc29_sliding_controller_constInstanceStruct
  *chartInstance, real_T c29_b_q[2], real_T c29_b_y[4]);
static void c29_emlrt_marshallIn(SFc29_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c29_b_y, const char_T *c29_identifier, real_T
  c29_c_y[4]);
static void c29_b_emlrt_marshallIn(SFc29_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c29_u, const emlrtMsgIdentifier *c29_parentId,
  real_T c29_b_y[4]);
static uint8_T c29_c_emlrt_marshallIn
  (SFc29_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c29_b_is_active_c29_sliding_controller_const, const char_T *c29_identifier);
static uint8_T c29_d_emlrt_marshallIn
  (SFc29_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c29_u, const emlrtMsgIdentifier *c29_parentId);
static void init_dsm_address_info(SFc29_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address
  (SFc29_sliding_controller_constInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c29_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c29_is_active_c29_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c29_do_animation_call_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c29_b_y = NULL;
  const mxArray *c29_c_y = NULL;
  const mxArray *c29_d_y = NULL;
  const mxArray *c29_st;
  c29_st = NULL;
  c29_b_y = NULL;
  sf_mex_assign(&c29_b_y, sf_mex_createcellmatrix(2, 1), false);
  c29_c_y = NULL;
  sf_mex_assign(&c29_c_y, sf_mex_create("y", *chartInstance->c29_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c29_b_y, 0, c29_c_y);
  c29_d_y = NULL;
  sf_mex_assign(&c29_d_y, sf_mex_create("y",
    &chartInstance->c29_is_active_c29_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c29_b_y, 1, c29_d_y);
  sf_mex_assign(&c29_st, c29_b_y, false);
  return c29_st;
}

static void set_sim_state_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c29_st)
{
  const mxArray *c29_u;
  c29_u = sf_mex_dup(c29_st);
  c29_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c29_u, 0)), "y",
                       *chartInstance->c29_y);
  chartInstance->c29_is_active_c29_sliding_controller_const =
    c29_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c29_u, 1)),
    "is_active_c29_sliding_controller_const");
  sf_mex_destroy(&c29_u);
  sf_mex_destroy(&c29_st);
}

static void sf_gateway_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c29_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c29_eML_blk_kernel(chartInstance, *chartInstance->c29_q, *chartInstance->c29_y);
  c29_do_animation_call_c29_sliding_controller_const(chartInstance);
}

static void mdl_start_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c29_errCode;
  c29_errCode = hipGetLastError();
  if (c29_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c29_errCode, (char_T *)hipGetErrorName
                       (c29_errCode), (char_T *)hipGetErrorString(c29_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c29_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc29_sliding_controller_const
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c29_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c29_nameCaptureInfo = NULL;
  c29_nameCaptureInfo = NULL;
  sf_mex_assign(&c29_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c29_nameCaptureInfo;
}

static void c29_eML_blk_kernel(SFc29_sliding_controller_constInstanceStruct
  *chartInstance, real_T c29_b_q[2], real_T c29_b_y[4])
{
  c29_b_y[0] = -0.324 * muDoubleScalarSin(c29_b_q[0]) - 0.39 * muDoubleScalarSin
    (c29_b_q[0] + c29_b_q[1]);
  c29_b_y[2] = 0.324 * muDoubleScalarCos(c29_b_q[0]) + 0.39 * muDoubleScalarSin
    (c29_b_q[0] + c29_b_q[1]);
  c29_b_y[1] = -0.39 * muDoubleScalarSin(c29_b_q[0] + c29_b_q[1]);
  c29_b_y[3] = 0.39 * muDoubleScalarCos(c29_b_q[0] + c29_b_q[1]);
}

static void c29_emlrt_marshallIn(SFc29_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c29_b_y, const char_T *c29_identifier, real_T
  c29_c_y[4])
{
  emlrtMsgIdentifier c29_thisId;
  c29_thisId.fIdentifier = const_cast<const char_T *>(c29_identifier);
  c29_thisId.fParent = NULL;
  c29_thisId.bParentIsCell = false;
  c29_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c29_b_y), &c29_thisId,
    c29_c_y);
  sf_mex_destroy(&c29_b_y);
}

static void c29_b_emlrt_marshallIn(SFc29_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c29_u, const emlrtMsgIdentifier *c29_parentId,
  real_T c29_b_y[4])
{
  real_T c29_dv[4];
  int32_T c29_i;
  sf_mex_import(c29_parentId, sf_mex_dup(c29_u), c29_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c29_i = 0; c29_i < 4; c29_i++) {
    c29_b_y[c29_i] = c29_dv[c29_i];
  }

  sf_mex_destroy(&c29_u);
}

static uint8_T c29_c_emlrt_marshallIn
  (SFc29_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c29_b_is_active_c29_sliding_controller_const, const char_T *c29_identifier)
{
  emlrtMsgIdentifier c29_thisId;
  uint8_T c29_b_y;
  c29_thisId.fIdentifier = const_cast<const char_T *>(c29_identifier);
  c29_thisId.fParent = NULL;
  c29_thisId.bParentIsCell = false;
  c29_b_y = c29_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c29_b_is_active_c29_sliding_controller_const), &c29_thisId);
  sf_mex_destroy(&c29_b_is_active_c29_sliding_controller_const);
  return c29_b_y;
}

static uint8_T c29_d_emlrt_marshallIn
  (SFc29_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c29_u, const emlrtMsgIdentifier *c29_parentId)
{
  uint8_T c29_b_u;
  uint8_T c29_b_y;
  sf_mex_import(c29_parentId, sf_mex_dup(c29_u), &c29_b_u, 1, 3, 0U, 0, 0U, 0);
  c29_b_y = c29_b_u;
  sf_mex_destroy(&c29_u);
  return c29_b_y;
}

static void init_dsm_address_info(SFc29_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address
  (SFc29_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c29_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c29_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c29_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c29_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(552332206U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2608078302U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1441289006U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4066884445U);
}

mxArray *sf_c29_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c29_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c29_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c29_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfj"
    "UU90P4mcXxicklmWWp8clGlvHFOZkpmXnp8cn5eSVF+Tk5qUUgZnEJwp8AfRceOw=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c29_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sjS5Om55RaK9v6MEldWgzUF";
}

static void sf_opaque_initialize_c29_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c29_sliding_controller_const(void *chartInstanceVar)
{
  enable_c29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c29_sliding_controller_const(void
  *chartInstanceVar)
{
  disable_c29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c29_sliding_controller_const(void
  *chartInstanceVar)
{
  sf_gateway_c29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c29_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c29_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c29_sliding_controller_const
  (void *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc29_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c29_sliding_controller_const
      ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc29_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c29_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c29_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc29_sliding_controller_const
    ((SFc29_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c29_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c29_sliding_controller_const
      ((SFc29_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
    initSimStructsc29_sliding_controller_const
      ((SFc29_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
  }
}

const char* sf_c29_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgUnjYs2MIoAuQSIT0mPQQEHyaGNbf3AQq3aMK0kN2O9HIlrLXeZ/aGtvENOQd6",
    "gxx76JD3nnhco0BfoLEnJMkVScYQYSVECFLncb2Zn5pudWXkr3Z6H1228t37wvJv4vIV3w8uuG/",
    "l4ZebOvq96D/PxGxQSNjogikTaq70EieAQtOTWMCm6YiBLYUwMQIGgiI2lMlXaNIssZ2LUsYI6f",
    "fpFyGjoh9LyYAdlSbAv+Bi1xdYcoJ4WU0BNByAwoZJ2GHY4GU4tVuasGQIdaRvVuaDB+DZ2Zume",
    "5YbFHNrnQLtCG4IW6wvbfEMMNM15pZvOU+1PgDKKOSOi1NuQaB9iDLCBfhzg77416FQRRkOizA6",
    "EJAG9x0apTimgqJNpnDhhghipGOHtiDed4LxtBxzt6ckAeE1A0LYdBWQUSyZMNf9+Bz1tC3LCoQ",
    "UndlitzYdX1pH/nMEZqMq4DZoyAUWGsC8qF00D0j5P2ZpmyTzMsAieE7VNkT8NQWX2YuZonyBPc",
    "IQSVTBInezqI8USDG+lNht1XWYu2jI2ysjWi2CptnYCdSxMtXWoaBLOdSXsSMZ7kABPtbaIIfWw",
    "TGs5TmsWHEkMsEvv6t1gBUPic1hTioCV0pUUAGnd+Q0Ly2UktdrIqInJ29rbm5+eh3WFATUgFMq",
    "qgCJMA8YsDW+1toBpxz0C0SqTmlcGzjJkEcrTAytaZ1KNMCY1ReTCBcdoJTDSQ+QSd0Jf46apgz",
    "kuF+EooSEErsAwDj3cNogtiYl2pW0b913CzLgFmioWl7BqcddhGWq7hBrH0BcjIc9ER8nIz2t8F",
    "l4AzDaiBBPDHaxfatzB1cuocP3skXfRz777iH42kSs+f5zRs1Kix5t5Ftdda9Sv28C3lUn/nZH7",
    "vrDOakHO4dbx/vZ0490/G3+M/3r7zYfdh7//vMz6fzau1v9v5+N7k0I73TjJXL467O6MXasl+u/",
    "O6F/Px/rU39yPNjcPya9Pk8e9Ng9eDF/3Oxk/C+xtFOydfN9wFR+zLM1PRbtBfjBxY2Kzdu30P5",
    "mx9+aCeKzl37Pr72fLyd/ZKvK4ukB+Hd/Ghbz99PU3tpaTz9Z/ucD+BwW+H6R9/Zi4agHH9Kenx",
    "5qzALf6MZXCKMk5KPeqzWU/PzXfryrnXbPc12Ln/3Kfn/eP6V/XLedds9yy/l21j39p+Lp+4BXw",
    "61+wH8uerz43/r13tXPQ/Xz8y/SvTjNkPCg59ebTeIQdlM3+B/L0X9Ids+4=",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c29_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c29_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2501929482U));
  ssSetChecksum1(S,(2041323763U));
  ssSetChecksum2(S,(3808924622U));
  ssSetChecksum3(S,(1034168136U));
}

static void mdlRTW_c29_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c29_sliding_controller_const(SimStruct *S)
{
  SFc29_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc29_sliding_controller_constInstanceStruct *)utMalloc
    (sizeof(SFc29_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc29_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc29_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c29_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c29_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c29_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c29_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c29_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c29_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c29_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c29_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c29_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c29_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c29_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c29_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c29_JITStateAnimation,
    chartInstance->c29_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c29_sliding_controller_const(chartInstance);
}

void c29_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c29_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c29_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c29_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c29_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
