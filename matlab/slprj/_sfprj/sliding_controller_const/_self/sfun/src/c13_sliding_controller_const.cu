/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c13_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void c13_do_animation_call_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c13_st);
static void sf_gateway_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);
static void c13_eML_blk_kernel(SFc13_sliding_controller_constInstanceStruct
  *chartInstance, real_T c13_b_q[2], real_T c13_b_y[4]);
static void c13_emlrt_marshallIn(SFc13_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c13_b_y, const char_T *c13_identifier, real_T
  c13_c_y[4]);
static void c13_b_emlrt_marshallIn(SFc13_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId,
  real_T c13_b_y[4]);
static uint8_T c13_c_emlrt_marshallIn
  (SFc13_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c13_b_is_active_c13_sliding_controller_const, const char_T *c13_identifier);
static uint8_T c13_d_emlrt_marshallIn
  (SFc13_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c13_u, const emlrtMsgIdentifier *c13_parentId);
static void init_dsm_address_info(SFc13_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address
  (SFc13_sliding_controller_constInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c13_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c13_is_active_c13_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c13_do_animation_call_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c13_b_y = NULL;
  const mxArray *c13_c_y = NULL;
  const mxArray *c13_d_y = NULL;
  const mxArray *c13_st;
  c13_st = NULL;
  c13_b_y = NULL;
  sf_mex_assign(&c13_b_y, sf_mex_createcellmatrix(2, 1), false);
  c13_c_y = NULL;
  sf_mex_assign(&c13_c_y, sf_mex_create("y", *chartInstance->c13_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c13_b_y, 0, c13_c_y);
  c13_d_y = NULL;
  sf_mex_assign(&c13_d_y, sf_mex_create("y",
    &chartInstance->c13_is_active_c13_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c13_b_y, 1, c13_d_y);
  sf_mex_assign(&c13_st, c13_b_y, false);
  return c13_st;
}

static void set_sim_state_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c13_st)
{
  const mxArray *c13_u;
  c13_u = sf_mex_dup(c13_st);
  c13_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c13_u, 0)), "y",
                       *chartInstance->c13_y);
  chartInstance->c13_is_active_c13_sliding_controller_const =
    c13_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c13_u, 1)),
    "is_active_c13_sliding_controller_const");
  sf_mex_destroy(&c13_u);
  sf_mex_destroy(&c13_st);
}

static void sf_gateway_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c13_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c13_eML_blk_kernel(chartInstance, *chartInstance->c13_q, *chartInstance->c13_y);
  c13_do_animation_call_c13_sliding_controller_const(chartInstance);
}

static void mdl_start_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c13_errCode;
  c13_errCode = hipGetLastError();
  if (c13_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c13_errCode, (char_T *)hipGetErrorName
                       (c13_errCode), (char_T *)hipGetErrorString(c13_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c13_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc13_sliding_controller_const
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c13_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c13_nameCaptureInfo = NULL;
  c13_nameCaptureInfo = NULL;
  sf_mex_assign(&c13_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c13_nameCaptureInfo;
}

static void c13_eML_blk_kernel(SFc13_sliding_controller_constInstanceStruct
  *chartInstance, real_T c13_b_q[2], real_T c13_b_y[4])
{
  c13_b_y[0] = -0.324 * muDoubleScalarSin(c13_b_q[0]);
  c13_b_y[2] = 0.0;
  c13_b_y[1] = 0.324 * muDoubleScalarCos(c13_b_q[0]);
  c13_b_y[3] = 0.0;
}

static void c13_emlrt_marshallIn(SFc13_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c13_b_y, const char_T *c13_identifier, real_T
  c13_c_y[4])
{
  emlrtMsgIdentifier c13_thisId;
  c13_thisId.fIdentifier = const_cast<const char_T *>(c13_identifier);
  c13_thisId.fParent = NULL;
  c13_thisId.bParentIsCell = false;
  c13_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c13_b_y), &c13_thisId,
    c13_c_y);
  sf_mex_destroy(&c13_b_y);
}

static void c13_b_emlrt_marshallIn(SFc13_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId,
  real_T c13_b_y[4])
{
  real_T c13_dv[4];
  int32_T c13_i;
  sf_mex_import(c13_parentId, sf_mex_dup(c13_u), c13_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c13_i = 0; c13_i < 4; c13_i++) {
    c13_b_y[c13_i] = c13_dv[c13_i];
  }

  sf_mex_destroy(&c13_u);
}

static uint8_T c13_c_emlrt_marshallIn
  (SFc13_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c13_b_is_active_c13_sliding_controller_const, const char_T *c13_identifier)
{
  emlrtMsgIdentifier c13_thisId;
  uint8_T c13_b_y;
  c13_thisId.fIdentifier = const_cast<const char_T *>(c13_identifier);
  c13_thisId.fParent = NULL;
  c13_thisId.bParentIsCell = false;
  c13_b_y = c13_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c13_b_is_active_c13_sliding_controller_const), &c13_thisId);
  sf_mex_destroy(&c13_b_is_active_c13_sliding_controller_const);
  return c13_b_y;
}

static uint8_T c13_d_emlrt_marshallIn
  (SFc13_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c13_u, const emlrtMsgIdentifier *c13_parentId)
{
  uint8_T c13_b_u;
  uint8_T c13_b_y;
  sf_mex_import(c13_parentId, sf_mex_dup(c13_u), &c13_b_u, 1, 3, 0U, 0, 0U, 0);
  c13_b_y = c13_b_u;
  sf_mex_destroy(&c13_u);
  return c13_b_y;
}

static void init_dsm_address_info(SFc13_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address
  (SFc13_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c13_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c13_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c13_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c13_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(536815311U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1945133011U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2900394615U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(32238838U);
}

mxArray *sf_c13_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c13_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c13_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c13_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfj"
    "UU90P4mcXxicklmWWp8cmGxvHFOZkpmXnp8cn5eSVF+Tk5qUUgZnEJwp8AfBoeNA=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c13_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sFqlSox9r5ekjZXT2w93GAD";
}

static void sf_opaque_initialize_c13_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c13_sliding_controller_const(void *chartInstanceVar)
{
  enable_c13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c13_sliding_controller_const(void
  *chartInstanceVar)
{
  disable_c13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c13_sliding_controller_const(void
  *chartInstanceVar)
{
  sf_gateway_c13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c13_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c13_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c13_sliding_controller_const
  (void *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc13_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c13_sliding_controller_const
      ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc13_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c13_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c13_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc13_sliding_controller_const
    ((SFc13_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c13_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c13_sliding_controller_const
      ((SFc13_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
    initSimStructsc13_sliding_controller_const
      ((SFc13_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
  }
}

const char* sf_c13_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgXHqNHWMIICBYoC9SntMalhILmktvXTCpVro5R/0IuxXo7ErZa79P5I1gv01jf",
    "pIfe8SJ4gl176CJ0lKVmmSMq2YCMtQoAil/vN7Mx8szMrb6V94OG1gffuU89bw+cneNe89HqSjV",
    "dm7vT7qvdtNv4ThYSNjogikfYqL0Ei+BW05NYwKdqiJwthTPRAgaCIjaUyZdo0iyxnYtCygjp9+",
    "jRkNPRDaXmwj7IkOBR8jNpia45QT4MpoKYFEJhQSdsPW5z0pxYrM6qHQAfaRlUuaDC+jZ1Z+sBy",
    "w2IOzSugbaENQYv1tW2+IQbq5qrUTeep9idAGcWcEVHobUi0DzEG2MBxHODvoTXoVB5GQ6LMPoR",
    "kCLrDBolOKSCvk2mcuGCCGKkY4c2I153gvG1HHO05kAHwioCgbfsKyCCWTJhy/v0WetoU5IJDAy",
    "5sv1ybD5fWkX/CYASqNG69uhyCIn04FKWLJgFpXiVsTbNkHmZYBCdE7VHkT0NQmr2YOdonyBN0U",
    "aIMBomTbd1VbIjhLdVmo7bLzEVbxkYp2XoRLNHWHEIVC1NtLSrqhHNdCuvKuAND4InWBjGkGpZq",
    "LcZpzYKuxAC79C7fDVYwJD6D1aUIWCFdwxwgqTu/YGG5iaRWGxnVMXkbnc789DysLQyoHqFQVAU",
    "UYRowZkl4y7UFTDvuEYhWmcS8InCaIYtQnu5Z0RhJNcCYVBSRaxcco6XASPeRS9wJxxo3TRXMcb",
    "kIRwkNIXAFhnE4wG2D2IKYaFfa9nDfDZkZN0BTxeICVi3uOixDTZdQ4xiOxUDIkWgpGflZjU/DC",
    "4DZRpRgor+P9UuNW7h6ERWunz33rvvZZ7foZxO5/PO7GT0rBXq8mWd+3fVa9bo1fFuZ9N8Zuc9z",
    "66zm5BxuE++v/tp5/3ZbP+Gf/tz54+/TzjLrv6ndrf9vZOOvJ4V2unGGc/nqsD/N2LVaoP/LGf2",
    "b2Vi3Lrkvr16pHRj8/ttZ9/vRq+0f9xopPwvsreXsnXzfchUfsyzJT0XbQXYwcWNi03bt9L+csX",
    "dtQTzWs+/p9c8Py8l/sZvncXWB/Ca+jXN5e//1t3aXk0/XP1tg/7Mc38+Svn5OXLWAc/pi+1xzF",
    "uBWP6dSGCU5B+Vetbnp533z/a5y3iPL/Vfs/Cj38Lzfpn89tpz3yHLL+nfXPv6h4av6gZfDb37A",
    "fix7vnpo/Dvvbuegb7Lx6+lfnXrIeFBw6s2m8QjbK5r9H+Tpv1Tts4I=",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c13_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c13_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3862277146U));
  ssSetChecksum1(S,(91435963U));
  ssSetChecksum2(S,(1279986540U));
  ssSetChecksum3(S,(1280830342U));
}

static void mdlRTW_c13_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c13_sliding_controller_const(SimStruct *S)
{
  SFc13_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc13_sliding_controller_constInstanceStruct *)utMalloc
    (sizeof(SFc13_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc13_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc13_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c13_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c13_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c13_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c13_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c13_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c13_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c13_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c13_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c13_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c13_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c13_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c13_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c13_JITStateAnimation,
    chartInstance->c13_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c13_sliding_controller_const(chartInstance);
}

void c13_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c13_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c13_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c13_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c13_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
