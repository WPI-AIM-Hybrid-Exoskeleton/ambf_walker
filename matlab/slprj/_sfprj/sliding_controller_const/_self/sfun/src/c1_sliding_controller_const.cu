/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c1_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void c1_do_animation_call_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c1_st);
static void sf_gateway_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance);
static void c1_eML_blk_kernel(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2]);
static void c1_emlrt_marshallIn(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2]);
static void c1_b_emlrt_marshallIn(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2]);
static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller_const,
  const char_T *c1_identifier);
static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static void init_dsm_address_info(SFc1_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc1_sliding_controller_constInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c1_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c1_is_active_c1_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c1_do_animation_call_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c1_b_y = NULL;
  const mxArray *c1_c_y = NULL;
  const mxArray *c1_d_y = NULL;
  const mxArray *c1_e_y = NULL;
  const mxArray *c1_f_y = NULL;
  const mxArray *c1_st;
  const mxArray *c1_y = NULL;
  c1_st = NULL;
  c1_y = NULL;
  sf_mex_assign(&c1_y, sf_mex_createcellmatrix(5, 1), false);
  c1_b_y = NULL;
  sf_mex_assign(&c1_b_y, sf_mex_create("y", *chartInstance->c1_ankle, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 0, c1_b_y);
  c1_c_y = NULL;
  sf_mex_assign(&c1_c_y, sf_mex_create("y", *chartInstance->c1_danke, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 1, c1_c_y);
  c1_d_y = NULL;
  sf_mex_assign(&c1_d_y, sf_mex_create("y", *chartInstance->c1_dknee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 2, c1_d_y);
  c1_e_y = NULL;
  sf_mex_assign(&c1_e_y, sf_mex_create("y", *chartInstance->c1_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 3, c1_e_y);
  c1_f_y = NULL;
  sf_mex_assign(&c1_f_y, sf_mex_create("y",
    &chartInstance->c1_is_active_c1_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c1_y, 4, c1_f_y);
  sf_mex_assign(&c1_st, c1_y, false);
  return c1_st;
}

static void set_sim_state_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c1_st)
{
  const mxArray *c1_u;
  c1_u = sf_mex_dup(c1_st);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 0)),
                      "ankle", *chartInstance->c1_ankle);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 1)),
                      "danke", *chartInstance->c1_danke);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 2)),
                      "dknee", *chartInstance->c1_dknee);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 3)), "knee",
                      *chartInstance->c1_knee);
  chartInstance->c1_is_active_c1_sliding_controller_const =
    c1_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 4)),
    "is_active_c1_sliding_controller_const");
  sf_mex_destroy(&c1_u);
  sf_mex_destroy(&c1_st);
}

static void sf_gateway_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c1_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c1_eML_blk_kernel(chartInstance, *chartInstance->c1_q, *chartInstance->c1_qd, *
                    chartInstance->c1_knee, *chartInstance->c1_dknee,
                    *chartInstance->c1_ankle, *chartInstance->c1_danke);
  c1_do_animation_call_c1_sliding_controller_const(chartInstance);
}

static void mdl_start_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c1_errCode;
  c1_errCode = hipGetLastError();
  if (c1_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c1_errCode, (char_T *)hipGetErrorName
                       (c1_errCode), (char_T *)hipGetErrorString(c1_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c1_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc1_sliding_controller_const
  (SFc1_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c1_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c1_nameCaptureInfo = NULL;
  const char_T *c1_data[4] = {
    "789cc553cb4ac34014bd955adc545d09825b372e1cdd886b8b850a2db50f5c88c69b64d20c9d874c52dbbaf113fc1dbfc7853bffc1246d9a26182216eb8170e7"
    "7032f7dc7b60a0d4689600603bf83e5e00ce2b10a13a2bb03baf1b9046562fe5d4189b504edd0bf5f7c0ef75ce2d257d3af16744a2a08b9bb6124ca2f47bd347",
    "0a9a7a8a3f513b521cc6698f09da5d26ad9089fa92b420a1149e6b2eb586dd9100ed7ac9847c9944798478c8d9b75c904716d93cb2ffc57e935ffac5fdf70afc"
    "62dd517a8cda36864c32813eb33c83d3c1ea7bc7be95dc39668aad4626a789dfdb8a7ebd5cbfb47e7b79475c252891e8bb2819e503c56d07b5492cf483348cb1",
    "473c6d1114a6638c910fa92641401c4df27d66c722ec5b94dbce0ff7c87b3f55d88aea677b1fd7e9777f7470b84ebf18ffe5b7aef777d3bec093b3e72bd6e9e8"
    "7af3badf6df44f5bb5648e76814fd11c90c3ffbaff177b596734", "" };

  c1_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c1_data[0], 1616U, &c1_nameCaptureInfo);
  return c1_nameCaptureInfo;
}

static void c1_eML_blk_kernel(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2])
{
  real_T c1_a;
  real_T c1_b_a;
  c1_b_knee[0] = -0.324 * muDoubleScalarCos(c1_b_q[0]);
  c1_b_knee[1] = -0.324 * muDoubleScalarSin(c1_b_q[0]);
  c1_a = -0.324 * c1_b_qd[0];
  c1_b_dknee[0] = c1_a * -muDoubleScalarSin(c1_b_q[0]);
  c1_b_dknee[1] = c1_a * muDoubleScalarCos(c1_b_q[0]);
  c1_b_a = -0.39 * (c1_b_qd[0] + c1_b_qd[1]);
  c1_b_ankle[0] = c1_b_knee[0] - -0.39 * muDoubleScalarSin(c1_b_q[0] + c1_b_q[1]);
  c1_b_ankle[1] = c1_b_knee[1] - -0.39 * muDoubleScalarCos(c1_b_q[0] + c1_b_q[1]);
  c1_b_danke[0] = c1_b_dknee[0] + c1_b_a * -muDoubleScalarSin(c1_b_q[0] +
    c1_b_q[1]);
  c1_b_danke[1] = c1_b_dknee[1] + c1_b_a * muDoubleScalarCos(c1_b_q[0] + c1_b_q
    [1]);
}

static void c1_emlrt_marshallIn(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2])
{
  emlrtMsgIdentifier c1_thisId;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_b_ankle), &c1_thisId, c1_y);
  sf_mex_destroy(&c1_b_ankle);
}

static void c1_b_emlrt_marshallIn(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2])
{
  real_T c1_dv[2];
  int32_T c1_i;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), c1_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c1_i = 0; c1_i < 2; c1_i++) {
    c1_y[c1_i] = c1_dv[c1_i];
  }

  sf_mex_destroy(&c1_u);
}

static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller_const,
  const char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  uint8_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c1_b_is_active_c1_sliding_controller_const), &c1_thisId);
  sf_mex_destroy(&c1_b_is_active_c1_sliding_controller_const);
  return c1_y;
}

static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  uint8_T c1_b_u;
  uint8_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_b_u, 1, 3, 0U, 0, 0U, 0);
  c1_y = c1_b_u;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static void init_dsm_address_info(SFc1_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc1_sliding_controller_constInstanceStruct
  *chartInstance)
{
  chartInstance->c1_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c1_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c1_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c1_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c1_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c1_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c1_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c1_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c1_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c1_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6LFA4ifWRyfmFySWZYan"
    "2wYX5yTmZKZlx6fnJ9XUpSfk5NaBGIWlyDFEwCquC7b"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c1_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c1_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c1_sliding_controller_const(void *chartInstanceVar)
{
  enable_c1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c1_sliding_controller_const(void *chartInstanceVar)
{
  disable_c1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c1_sliding_controller_const(void *chartInstanceVar)
{
  sf_gateway_c1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c1_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c1_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c1_sliding_controller_const(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc1_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c1_sliding_controller_const
      ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc1_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c1_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c1_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc1_sliding_controller_const
    ((SFc1_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c1_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c1_sliding_controller_const
      ((SFc1_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc1_sliding_controller_const
      ((SFc1_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c1_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIAm8CFAL0EbFAnaSwEXRYH20timJESFHTulnAY91FgvR+JCy112f2ir75C",
    "ec8k9D5BLnqBAn6CnPkPRJ+gsScsyRVKR3aRuURq0tOQ33843Mzu78lqDXQ+vW3g/v+151/DzOt",
    "5tL7/WinFr7s6fd7yPi/EzNBI23ieKxNprvASJ4VvQklvDpBiIkayEMTECBYIiNpHK1LFpFlvOx",
    "KRvBXV8+ruI0SiIpOXhNtqScE/wKbIl1uwjT5cpoKYPEJpISTuO+pyMZx4rc+xHQCfaxk0SNJjA",
    "Js4tvWu5YQmH3gnQgdCGoMf6zLfAEAO+OamV6ZTq4BQo44QzIirVRkQHkGCADRwkIf7fswZFlWE",
    "0IspsQ0RS0DtsknFKAWVOpvHFERPESMUI78Xcd4aLvu1z9GdXhsAbAoK+bSsgk0QyYerzH/RRaU",
    "+QIw5dOLLjerYAfrQu+U8YHIOqjdvIlykoMoY9UTtpFpDeSZatWZUswgyL4QlRWxTzpyGsrV6sH",
    "B0QzBMM0aIOBpnIgR4qlmJ4a9lsPHCVuWzJ2DhPtl4Gy9h6KTRlYcbWp8InnOta2FAmO5ACz1i7",
    "xJBmWM5ajdOahUOJAXblXb8arGCY+ALmSxGyynSlJUDWdx5hYzmPpFYbGftYvN2dncXXi7CBMKB",
    "GhEJVF1CEacCYZeGtZwuZdrlHIHplMveqwHmFLEN5emRF91iqCcakoYmcSXAZrQXGeoy5xJVwoH",
    "HRNMFcLpfhKKERhK7BMA67uGwQWxET7VrbFq67lJlpFzRVLKnIqsVVh22o5wpqmsCBmAh5LPpKx",
    "kHR4/PwAmC1ESWYGG9j/1LTPs5elQq3n214Z/vZe2+wn53alT8/meNpVfB4c5/leW+0z+PbpXnb",
    "+Ndq5Xabc3Y3S/N0SnYOt473N7+/uPfog1d//vLrpz9PX77+Ydn8rYX5W9l3Z3d9bbX9/1YxvnP",
    "aaGcLJ12oV4d9OOdXp4L//Tn+9WKsH0dfiPT7h9HWYzKMBxuf//TVRnrkZ3xPO83+rpX8PX1+13",
    "V8rLKsPhUdhMXBxI2Jzbdrx//lnL/XlsTjRvE8v/54cDn7O5tl+84SfevFmIgJh1L9/nM6Prqwj",
    "hCFXB0ddy+uY4KHhiuj4/ZmuT9U6eic09HxnIS/x//FOK5mn8//dIn/90t5uJ+dFw+J24XgkH52",
    "qDkLcQc5pFIYJTkH5b5qU8rTRfvoqnbeO7b7t/j5v93bz/ubnIvaF7TrXPIc9q7sLqtv1fPhVcM",
    "37QdeCb9+hXVc9tz+tvG/eaudrz8sxl/PfkL7EeNhxa+p4jX+NBpVvf0P1OlftmXF2w==",
    ""
  };

  static char newstr [1345] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c1_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c1_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(614325834U));
  ssSetChecksum1(S,(4071890510U));
  ssSetChecksum2(S,(2385365703U));
  ssSetChecksum3(S,(1589225593U));
}

static void mdlRTW_c1_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c1_sliding_controller_const(SimStruct *S)
{
  SFc1_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc1_sliding_controller_constInstanceStruct *)utMalloc(sizeof
    (SFc1_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc1_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc1_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c1_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c1_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c1_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c1_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c1_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c1_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c1_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c1_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c1_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c1_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c1_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c1_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c1_JITStateAnimation,
    chartInstance->c1_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c1_sliding_controller_const(chartInstance);
}

void c1_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c1_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c1_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c1_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c1_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
