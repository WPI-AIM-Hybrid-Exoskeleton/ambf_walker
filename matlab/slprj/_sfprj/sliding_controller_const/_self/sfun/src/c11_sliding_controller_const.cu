/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c11_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void c11_do_animation_call_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c11_st);
static void sf_gateway_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);
static void c11_eML_blk_kernel(SFc11_sliding_controller_constInstanceStruct
  *chartInstance, real_T c11_b_q[2], real_T c11_b_y[4]);
static void c11_emlrt_marshallIn(SFc11_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c11_b_y, const char_T *c11_identifier, real_T
  c11_c_y[4]);
static void c11_b_emlrt_marshallIn(SFc11_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId,
  real_T c11_b_y[4]);
static uint8_T c11_c_emlrt_marshallIn
  (SFc11_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c11_b_is_active_c11_sliding_controller_const, const char_T *c11_identifier);
static uint8_T c11_d_emlrt_marshallIn
  (SFc11_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c11_u, const emlrtMsgIdentifier *c11_parentId);
static void init_dsm_address_info(SFc11_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address
  (SFc11_sliding_controller_constInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c11_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c11_is_active_c11_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c11_do_animation_call_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c11_b_y = NULL;
  const mxArray *c11_c_y = NULL;
  const mxArray *c11_d_y = NULL;
  const mxArray *c11_st;
  c11_st = NULL;
  c11_b_y = NULL;
  sf_mex_assign(&c11_b_y, sf_mex_createcellmatrix(2, 1), false);
  c11_c_y = NULL;
  sf_mex_assign(&c11_c_y, sf_mex_create("y", *chartInstance->c11_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c11_b_y, 0, c11_c_y);
  c11_d_y = NULL;
  sf_mex_assign(&c11_d_y, sf_mex_create("y",
    &chartInstance->c11_is_active_c11_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c11_b_y, 1, c11_d_y);
  sf_mex_assign(&c11_st, c11_b_y, false);
  return c11_st;
}

static void set_sim_state_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c11_st)
{
  const mxArray *c11_u;
  c11_u = sf_mex_dup(c11_st);
  c11_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c11_u, 0)), "y",
                       *chartInstance->c11_y);
  chartInstance->c11_is_active_c11_sliding_controller_const =
    c11_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c11_u, 1)),
    "is_active_c11_sliding_controller_const");
  sf_mex_destroy(&c11_u);
  sf_mex_destroy(&c11_st);
}

static void sf_gateway_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c11_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c11_eML_blk_kernel(chartInstance, *chartInstance->c11_q, *chartInstance->c11_y);
  c11_do_animation_call_c11_sliding_controller_const(chartInstance);
}

static void mdl_start_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c11_errCode;
  c11_errCode = hipGetLastError();
  if (c11_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c11_errCode, (char_T *)hipGetErrorName
                       (c11_errCode), (char_T *)hipGetErrorString(c11_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c11_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc11_sliding_controller_const
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c11_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c11_nameCaptureInfo = NULL;
  c11_nameCaptureInfo = NULL;
  sf_mex_assign(&c11_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c11_nameCaptureInfo;
}

static void c11_eML_blk_kernel(SFc11_sliding_controller_constInstanceStruct
  *chartInstance, real_T c11_b_q[2], real_T c11_b_y[4])
{
  c11_b_y[0] = -0.324 * muDoubleScalarSin(c11_b_q[0]);
  c11_b_y[2] = 0.0;
  c11_b_y[1] = 0.324 * muDoubleScalarCos(c11_b_q[0]);
  c11_b_y[3] = 0.0;
}

static void c11_emlrt_marshallIn(SFc11_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c11_b_y, const char_T *c11_identifier, real_T
  c11_c_y[4])
{
  emlrtMsgIdentifier c11_thisId;
  c11_thisId.fIdentifier = const_cast<const char_T *>(c11_identifier);
  c11_thisId.fParent = NULL;
  c11_thisId.bParentIsCell = false;
  c11_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c11_b_y), &c11_thisId,
    c11_c_y);
  sf_mex_destroy(&c11_b_y);
}

static void c11_b_emlrt_marshallIn(SFc11_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId,
  real_T c11_b_y[4])
{
  real_T c11_dv[4];
  int32_T c11_i;
  sf_mex_import(c11_parentId, sf_mex_dup(c11_u), c11_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c11_i = 0; c11_i < 4; c11_i++) {
    c11_b_y[c11_i] = c11_dv[c11_i];
  }

  sf_mex_destroy(&c11_u);
}

static uint8_T c11_c_emlrt_marshallIn
  (SFc11_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c11_b_is_active_c11_sliding_controller_const, const char_T *c11_identifier)
{
  emlrtMsgIdentifier c11_thisId;
  uint8_T c11_b_y;
  c11_thisId.fIdentifier = const_cast<const char_T *>(c11_identifier);
  c11_thisId.fParent = NULL;
  c11_thisId.bParentIsCell = false;
  c11_b_y = c11_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c11_b_is_active_c11_sliding_controller_const), &c11_thisId);
  sf_mex_destroy(&c11_b_is_active_c11_sliding_controller_const);
  return c11_b_y;
}

static uint8_T c11_d_emlrt_marshallIn
  (SFc11_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c11_u, const emlrtMsgIdentifier *c11_parentId)
{
  uint8_T c11_b_u;
  uint8_T c11_b_y;
  sf_mex_import(c11_parentId, sf_mex_dup(c11_u), &c11_b_u, 1, 3, 0U, 0, 0U, 0);
  c11_b_y = c11_b_u;
  sf_mex_destroy(&c11_u);
  return c11_b_y;
}

static void init_dsm_address_info(SFc11_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address
  (SFc11_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c11_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c11_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c11_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c11_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(536815311U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1945133011U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2900394615U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(32238838U);
}

mxArray *sf_c11_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c11_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c11_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c11_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfj"
    "UU90P4mcXxicklmWWp8cmGhvHFOZkpmXnp8cn5eSVF+Tk5qUUgZnEJwp8Ae9IeMg=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c11_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sFqlSox9r5ekjZXT2w93GAD";
}

static void sf_opaque_initialize_c11_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c11_sliding_controller_const(void *chartInstanceVar)
{
  enable_c11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c11_sliding_controller_const(void
  *chartInstanceVar)
{
  disable_c11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c11_sliding_controller_const(void
  *chartInstanceVar)
{
  sf_gateway_c11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c11_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c11_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c11_sliding_controller_const
  (void *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc11_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c11_sliding_controller_const
      ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc11_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c11_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c11_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc11_sliding_controller_const
    ((SFc11_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c11_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c11_sliding_controller_const
      ((SFc11_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
    initSimStructsc11_sliding_controller_const
      ((SFc11_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
  }
}

const char* sf_c11_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u00AQdqOCQPyoQkhICAlOwJEfIcEF2uYHIlJa4RQQl2q7nsRL1rtmf5LmBbjxJhy48yI",
    "8ARcuPAKztpMGx3YoERUgLDn2er+ZnZlvdmbjrbS3PLzO471+wfNO4vMU3jUvvU5k45WZO/2+6t",
    "3Ixu9RSNhohygSaa/yEiSC56Alt4ZJ0RY9WQhjogcKBEVsLJUp06ZZZDkTg5YV1OnTL0NGQz+Ul",
    "gebKEuCbcHHqC22Zgf1NJgCaloAgQmVtP2wxUl/arEyo3oIdKBtVOWCBuPb2Jmltyw3LObQPADa",
    "FtoQtFgf2uYbYqBuDkrddJ5qfwKUUcwZEYXehkT7EGOADezGAf5uW4NO5WE0JMpsQkiGoDtskOi",
    "UAvI6mcaJfSaIkYoR3ox43QnO27bD0Z4tGQCvCAjatqmADGLJhCnn32+hp01B9jk0YN/2y7X58N",
    "Y68l8wGIEqjVuvLoegSB+2RemiSUCaBwlb0yyZhxkWwQuiNijypyEozV7MHO0T5Am6KFEGg8TJt",
    "u4qNsTwlmqzUdtl5qItY6OUbL0IlmhrDqGKham2FhV1wrkuhXVl3IEh8ERrgxhSDUu1FuO0ZkFX",
    "YoBdepfvBisYEp/B6lIErJCuYQ6Q1J1nWFh+RFKrjYzqmLyNTmd+eh7WFgZUj1AoqgKKMA0YsyS",
    "85doCph33CESrTGJeETjNkEUoT/esaIykGmBMKorIoQuO0VJgpPvIJe6EXY2bpgrmuFyEo4SGEL",
    "gCwzhs4bZBbEFMtCttG7jvhsyMG6CpYnEBqxZ3HZahpkuocQy7YiDkSLSUjPysxqfhBcBsI0ow0",
    "d/E+qXGLVy9iArXz255h/3s7E/0s4lc/nlzRs9KgR5v5plf93Stet0avq1M+u+M3LncOqs5OYdb",
    "w/vyh3tfPt3VJ/iZp513X192lln/Y+1o/f98Nr4yKbTTjTOcy1eHfTJj12qB/ksz+teysW695b4",
    "8eKDuweDN61fdO6MHdx9vNFJ+Fthby9k7+X7NVXzMsiQ/FW0H2cHEjYlN27XTf3/G3pML4nE6+5",
    "5e3x4tJ39xPc/j6gL5NXwb5/L219e/tr6cfLr+qwX2X8/xfT3p63vEVQvYo7dv72nOAtzqe1QKo",
    "yTnoNyrNj/6+av5flQ575jl/hY7/8v9ft5/pn8dt5x3zHLL+nfUPv6n4av6gZfDr/3Bfix7vvrd",
    "+M/e0c5BV7Pxw+lfnXrIeFBw6s2m8QjbK5r9B/L0O0WFs4A=",
    ""
  };

  static char newstr [1249] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c11_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c11_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3862277146U));
  ssSetChecksum1(S,(91435963U));
  ssSetChecksum2(S,(1279986540U));
  ssSetChecksum3(S,(1280830342U));
}

static void mdlRTW_c11_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c11_sliding_controller_const(SimStruct *S)
{
  SFc11_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc11_sliding_controller_constInstanceStruct *)utMalloc
    (sizeof(SFc11_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc11_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc11_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c11_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c11_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c11_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c11_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c11_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c11_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c11_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c11_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c11_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c11_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c11_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c11_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c11_JITStateAnimation,
    chartInstance->c11_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c11_sliding_controller_const(chartInstance);
}

void c11_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c11_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c11_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c11_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c11_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
