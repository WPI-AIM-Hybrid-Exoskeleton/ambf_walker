#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c8_sliding_controller_const.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void c8_do_animation_call_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c8_st);
static void sf_gateway_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance);
static void c8_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_b_msg, const char_T *c8_identifier,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_y);
static void c8_b_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_y);
static void c8_c_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  c8_SL_Bus_sliding_controller_const_std_msgs_Header *c8_y);
static uint32_T c8_d_emlrt_marshallIn
  (SFc8_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c8_u, const emlrtMsgIdentifier *c8_parentId);
static c8_SL_Bus_sliding_controller_const_ros_time_Time c8_e_emlrt_marshallIn
  (SFc8_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c8_u, const emlrtMsgIdentifier *c8_parentId);
static real_T c8_f_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId);
static void c8_g_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  uint8_T c8_y[128]);
static c8_SL_Bus_ROSVariableLengthArrayInfo c8_h_emlrt_marshallIn
  (SFc8_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c8_u, const emlrtMsgIdentifier *c8_parentId);
static void c8_i_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  c8_SL_Bus_sliding_controller_const_std_msgs_String c8_y[16]);
static void c8_j_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  real_T c8_y[7]);
static uint8_T c8_k_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_b_is_active_c8_sliding_controller_const,
  const char_T *c8_identifier);
static uint8_T c8_l_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId);
static void c8_eML_blk_kernel(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c8_b_blankMsg);
static __global__ void c8_sf_gateway_c8_sliding_controller_const_kernel1
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg,
   c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg);
static __global__ void c8_sf_gateway_c8_sliding_controller_const_kernel2(const
  int32_T c8_i1, c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c8_b_blankMsg, c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c8_b_msg);
static __global__ void c8_sf_gateway_c8_sliding_controller_const_kernel3
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg,
   c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg);
static __global__ void c8_sf_gateway_c8_sliding_controller_const_kernel4
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg,
   c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r);
static __global__ void c8_sf_gateway_c8_sliding_controller_const_kernel5(const
  int32_T c8_i5, c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c8_b_msg, c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r);
static __global__ void c8_sf_gateway_c8_sliding_controller_const_kernel6
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg,
   c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r);
static __global__ void c8_sf_gateway_c8_sliding_controller_const_kernel7(const
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg);
static __global__ void c8_sf_gateway_c8_sliding_controller_const_kernel8(const
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r, const int32_T
  c8_i9, c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg);
static __global__ void c8_sf_gateway_c8_sliding_controller_const_kernel9(const
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg);
static __global__ void c8_eML_blk_kernel_kernel10
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel11(const uint8_T c8_uv[12],
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel12
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel13(const uint8_T c8_uv1[13],
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel14
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel15(const uint8_T c8_uv2[14],
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel16
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel17(const uint8_T c8_uv3[13],
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel18
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel19(const uint8_T c8_uv4[14],
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel20
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel21(const uint8_T c8_uv5[15],
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static __global__ void c8_eML_blk_kernel_kernel22
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg);
static void init_dsm_address_info(SFc8_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc8_sliding_controller_constInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c8_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c8_is_active_c8_sliding_controller_const = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c8_gpu_uv5, 15UL);
  hipMalloc(&chartInstance->c8_gpu_uv4, 14UL);
  hipMalloc(&chartInstance->c8_gpu_uv3, 13UL);
  hipMalloc(&chartInstance->c8_gpu_uv2, 14UL);
  hipMalloc(&chartInstance->c8_gpu_uv1, 13UL);
  hipMalloc(&chartInstance->c8_gpu_uv, 12UL);
  hipMalloc(&chartInstance->c8_gpu_blankMsg, 2536UL);
}

static void initialize_params_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c8_do_animation_call_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
  static const char_T *c8_sv[9] = { "Header", "Name", "Name_SL_Info", "Position",
    "Position_SL_Info", "Velocity", "Velocity_SL_Info", "Effort",
    "Effort_SL_Info" };

  static const char_T *c8_sv1[4] = { "Seq", "Stamp", "FrameId",
    "FrameId_SL_Info" };

  static const char_T *c8_sv2[2] = { "Sec", "Nsec" };

  static const char_T *c8_sv3[2] = { "CurrentLength", "ReceivedLength" };

  static const char_T *c8_sv4[2] = { "Data", "Data_SL_Info" };

  c8_SL_Bus_sliding_controller_const_std_msgs_String c8_g_u[16];
  const c8_SL_Bus_sliding_controller_const_std_msgs_String *c8_r;
  const mxArray *c8_ab_y = NULL;
  const mxArray *c8_b_y = NULL;
  const mxArray *c8_bb_y = NULL;
  const mxArray *c8_c_y = NULL;
  const mxArray *c8_cb_y = NULL;
  const mxArray *c8_d_y = NULL;
  const mxArray *c8_db_y = NULL;
  const mxArray *c8_e_y = NULL;
  const mxArray *c8_eb_y = NULL;
  const mxArray *c8_f_y = NULL;
  const mxArray *c8_fb_y = NULL;
  const mxArray *c8_g_y = NULL;
  const mxArray *c8_gb_y = NULL;
  const mxArray *c8_h_y = NULL;
  const mxArray *c8_i_y = NULL;
  const mxArray *c8_j_y = NULL;
  const mxArray *c8_k_y = NULL;
  const mxArray *c8_l_y = NULL;
  const mxArray *c8_m_y = NULL;
  const mxArray *c8_n_y = NULL;
  const mxArray *c8_o_y = NULL;
  const mxArray *c8_p_y = NULL;
  const mxArray *c8_q_y = NULL;
  const mxArray *c8_r_y = NULL;
  const mxArray *c8_s_y = NULL;
  const mxArray *c8_st;
  const mxArray *c8_t_y = NULL;
  const mxArray *c8_u_y = NULL;
  const mxArray *c8_v_y = NULL;
  const mxArray *c8_w_y = NULL;
  const mxArray *c8_x_y = NULL;
  const mxArray *c8_y = NULL;
  const mxArray *c8_y_y = NULL;
  real_T c8_l_u[7];
  real_T c8_b_u;
  real_T c8_c_u;
  real_T c8_u_Stamp_Nsec;
  real_T c8_u_Stamp_Sec;
  int32_T c8_iv[1];
  int32_T c8_b_i;
  int32_T c8_i;
  int32_T c8_i1;
  int32_T c8_i2;
  int32_T c8_i3;
  int32_T c8_i4;
  int32_T c8_i5;
  int32_T c8_i6;
  int32_T c8_i7;
  uint32_T c8_e_u;
  uint32_T c8_f_u;
  uint32_T c8_h_u;
  uint32_T c8_i_u;
  uint32_T c8_j_u;
  uint32_T c8_k_u;
  uint32_T c8_m_u;
  uint32_T c8_n_u;
  uint32_T c8_o_u;
  uint32_T c8_p_u;
  uint32_T c8_q_u;
  uint32_T c8_r_u;
  uint32_T c8_u;
  uint32_T c8_u_CurrentLength;
  uint32_T c8_u_FrameId_SL_Info_CurrentLength;
  uint32_T c8_u_FrameId_SL_Info_ReceivedLength;
  uint32_T c8_u_ReceivedLength;
  uint32_T c8_u_Seq;
  uint8_T c8_d_u[128];
  uint8_T c8_u_FrameId[128];
  c8_st = NULL;
  c8_y = NULL;
  sf_mex_assign(&c8_y, sf_mex_createcellmatrix(2, 1), false);
  c8_b_y = NULL;
  sf_mex_assign(&c8_b_y, sf_mex_createstruct("structure", 9, c8_sv, 2, 1, 1),
                false);
  c8_u_Seq = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[0];
  c8_u_Stamp_Sec = *(real_T *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[8])[0];
  c8_u_Stamp_Nsec = *(real_T *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[8])[8];
  for (c8_i = 0; c8_i < 128; c8_i++) {
    c8_u_FrameId[c8_i] = ((uint8_T *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
      chartInstance->c8_msg)[0])[24])[c8_i];
  }

  c8_u_FrameId_SL_Info_CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[0];
  c8_u_FrameId_SL_Info_ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[4];
  c8_c_y = NULL;
  sf_mex_assign(&c8_c_y, sf_mex_createstruct("structure", 4, c8_sv1, 2, 1, 1),
                false);
  c8_u = c8_u_Seq;
  c8_d_y = NULL;
  sf_mex_assign(&c8_d_y, sf_mex_create("y", &c8_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_c_y, 0, "Seq", c8_d_y, 0);
  c8_e_y = NULL;
  sf_mex_assign(&c8_e_y, sf_mex_createstruct("structure", 2, c8_sv2, 2, 1, 1),
                false);
  c8_b_u = c8_u_Stamp_Sec;
  c8_f_y = NULL;
  sf_mex_assign(&c8_f_y, sf_mex_create("y", &c8_b_u, 0, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_e_y, 0, "Sec", c8_f_y, 0);
  c8_c_u = c8_u_Stamp_Nsec;
  c8_g_y = NULL;
  sf_mex_assign(&c8_g_y, sf_mex_create("y", &c8_c_u, 0, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_e_y, 0, "Nsec", c8_g_y, 1);
  sf_mex_setfieldbynum(c8_c_y, 0, "Stamp", c8_e_y, 1);
  for (c8_i1 = 0; c8_i1 < 128; c8_i1++) {
    c8_d_u[c8_i1] = c8_u_FrameId[c8_i1];
  }

  c8_h_y = NULL;
  sf_mex_assign(&c8_h_y, sf_mex_create("y", c8_d_u, 3, 0U, 1U, 0U, 1, 128),
                false);
  sf_mex_setfieldbynum(c8_c_y, 0, "FrameId", c8_h_y, 2);
  c8_i_y = NULL;
  sf_mex_assign(&c8_i_y, sf_mex_createstruct("structure", 2, c8_sv3, 2, 1, 1),
                false);
  c8_e_u = c8_u_FrameId_SL_Info_CurrentLength;
  c8_j_y = NULL;
  sf_mex_assign(&c8_j_y, sf_mex_create("y", &c8_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_i_y, 0, "CurrentLength", c8_j_y, 0);
  c8_f_u = c8_u_FrameId_SL_Info_ReceivedLength;
  c8_k_y = NULL;
  sf_mex_assign(&c8_k_y, sf_mex_create("y", &c8_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_i_y, 0, "ReceivedLength", c8_k_y, 1);
  sf_mex_setfieldbynum(c8_c_y, 0, "FrameId_SL_Info", c8_i_y, 3);
  sf_mex_setfieldbynum(c8_b_y, 0, "Header", c8_c_y, 0);
  for (c8_i2 = 0; c8_i2 < 16; c8_i2++) {
    for (c8_i3 = 0; c8_i3 < 128; c8_i3++) {
      c8_g_u[c8_i2].Data[c8_i3] = ((uint8_T *)&((char_T *)
        (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
        (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
        chartInstance->c8_msg)[160])[136 * (int32_T)c8_i2])[0])[c8_i3];
    }

    c8_g_u[c8_i2].Data_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
      (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i2])[128])[0];
    c8_g_u[c8_i2].Data_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
      (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i2])[128])[4];
  }

  c8_l_y = NULL;
  c8_iv[0] = 16;
  sf_mex_assign(&c8_l_y, sf_mex_createstructarray("structure", 1, c8_iv, 2,
    c8_sv4), false);
  for (c8_b_i = 0; c8_b_i < 16; c8_b_i++) {
    c8_r = &c8_g_u[c8_b_i];
    for (c8_i4 = 0; c8_i4 < 128; c8_i4++) {
      c8_d_u[c8_i4] = c8_r->Data[c8_i4];
    }

    c8_n_y = NULL;
    sf_mex_assign(&c8_n_y, sf_mex_create("y", c8_d_u, 3, 0U, 1U, 0U, 1, 128),
                  false);
    sf_mex_setfieldbynum(c8_l_y, c8_b_i, "Data", c8_n_y, 0);
    c8_u_CurrentLength = c8_r->Data_SL_Info.CurrentLength;
    c8_u_ReceivedLength = c8_r->Data_SL_Info.ReceivedLength;
    c8_p_y = NULL;
    sf_mex_assign(&c8_p_y, sf_mex_createstruct("structure", 2, c8_sv3, 2, 1, 1),
                  false);
    c8_j_u = c8_u_CurrentLength;
    c8_r_y = NULL;
    sf_mex_assign(&c8_r_y, sf_mex_create("y", &c8_j_u, 7, 0U, 0U, 0U, 0), false);
    sf_mex_setfieldbynum(c8_p_y, 0, "CurrentLength", c8_r_y, 0);
    c8_k_u = c8_u_ReceivedLength;
    c8_s_y = NULL;
    sf_mex_assign(&c8_s_y, sf_mex_create("y", &c8_k_u, 7, 0U, 0U, 0U, 0), false);
    sf_mex_setfieldbynum(c8_p_y, 0, "ReceivedLength", c8_s_y, 1);
    sf_mex_setfieldbynum(c8_l_y, c8_b_i, "Data_SL_Info", c8_p_y, 1);
  }

  sf_mex_setfieldbynum(c8_b_y, 0, "Name", c8_l_y, 1);
  c8_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2336])[0];
  c8_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2336])[4];
  c8_m_y = NULL;
  sf_mex_assign(&c8_m_y, sf_mex_createstruct("structure", 2, c8_sv3, 2, 1, 1),
                false);
  c8_h_u = c8_u_CurrentLength;
  c8_o_y = NULL;
  sf_mex_assign(&c8_o_y, sf_mex_create("y", &c8_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_m_y, 0, "CurrentLength", c8_o_y, 0);
  c8_i_u = c8_u_ReceivedLength;
  c8_q_y = NULL;
  sf_mex_assign(&c8_q_y, sf_mex_create("y", &c8_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_m_y, 0, "ReceivedLength", c8_q_y, 1);
  sf_mex_setfieldbynum(c8_b_y, 0, "Name_SL_Info", c8_m_y, 2);
  for (c8_i5 = 0; c8_i5 < 7; c8_i5++) {
    c8_l_u[c8_i5] = ((real_T *)&((char_T *)chartInstance->c8_msg)[2344])[c8_i5];
  }

  c8_t_y = NULL;
  sf_mex_assign(&c8_t_y, sf_mex_create("y", c8_l_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c8_b_y, 0, "Position", c8_t_y, 3);
  c8_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2400])[0];
  c8_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2400])[4];
  c8_u_y = NULL;
  sf_mex_assign(&c8_u_y, sf_mex_createstruct("structure", 2, c8_sv3, 2, 1, 1),
                false);
  c8_m_u = c8_u_CurrentLength;
  c8_v_y = NULL;
  sf_mex_assign(&c8_v_y, sf_mex_create("y", &c8_m_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_u_y, 0, "CurrentLength", c8_v_y, 0);
  c8_n_u = c8_u_ReceivedLength;
  c8_w_y = NULL;
  sf_mex_assign(&c8_w_y, sf_mex_create("y", &c8_n_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_u_y, 0, "ReceivedLength", c8_w_y, 1);
  sf_mex_setfieldbynum(c8_b_y, 0, "Position_SL_Info", c8_u_y, 4);
  for (c8_i6 = 0; c8_i6 < 7; c8_i6++) {
    c8_l_u[c8_i6] = ((real_T *)&((char_T *)chartInstance->c8_msg)[2408])[c8_i6];
  }

  c8_x_y = NULL;
  sf_mex_assign(&c8_x_y, sf_mex_create("y", c8_l_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c8_b_y, 0, "Velocity", c8_x_y, 5);
  c8_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2464])[0];
  c8_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2464])[4];
  c8_y_y = NULL;
  sf_mex_assign(&c8_y_y, sf_mex_createstruct("structure", 2, c8_sv3, 2, 1, 1),
                false);
  c8_o_u = c8_u_CurrentLength;
  c8_ab_y = NULL;
  sf_mex_assign(&c8_ab_y, sf_mex_create("y", &c8_o_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_y_y, 0, "CurrentLength", c8_ab_y, 0);
  c8_p_u = c8_u_ReceivedLength;
  c8_bb_y = NULL;
  sf_mex_assign(&c8_bb_y, sf_mex_create("y", &c8_p_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_y_y, 0, "ReceivedLength", c8_bb_y, 1);
  sf_mex_setfieldbynum(c8_b_y, 0, "Velocity_SL_Info", c8_y_y, 6);
  for (c8_i7 = 0; c8_i7 < 7; c8_i7++) {
    c8_l_u[c8_i7] = ((real_T *)&((char_T *)chartInstance->c8_msg)[2472])[c8_i7];
  }

  c8_cb_y = NULL;
  sf_mex_assign(&c8_cb_y, sf_mex_create("y", c8_l_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c8_b_y, 0, "Effort", c8_cb_y, 7);
  c8_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2528])[0];
  c8_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2528])[4];
  c8_db_y = NULL;
  sf_mex_assign(&c8_db_y, sf_mex_createstruct("structure", 2, c8_sv3, 2, 1, 1),
                false);
  c8_q_u = c8_u_CurrentLength;
  c8_eb_y = NULL;
  sf_mex_assign(&c8_eb_y, sf_mex_create("y", &c8_q_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_db_y, 0, "CurrentLength", c8_eb_y, 0);
  c8_r_u = c8_u_ReceivedLength;
  c8_fb_y = NULL;
  sf_mex_assign(&c8_fb_y, sf_mex_create("y", &c8_r_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c8_db_y, 0, "ReceivedLength", c8_fb_y, 1);
  sf_mex_setfieldbynum(c8_b_y, 0, "Effort_SL_Info", c8_db_y, 8);
  sf_mex_setcell(c8_y, 0, c8_b_y);
  c8_gb_y = NULL;
  sf_mex_assign(&c8_gb_y, sf_mex_create("y",
    &chartInstance->c8_is_active_c8_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c8_y, 1, c8_gb_y);
  sf_mex_assign(&c8_st, c8_y, false);
  return c8_st;
}

static void set_sim_state_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c8_st)
{
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState c8_r;
  const mxArray *c8_u;
  int32_T c8_i;
  int32_T c8_i1;
  int32_T c8_i2;
  int32_T c8_i3;
  c8_u = sf_mex_dup(c8_st);
  c8_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c8_u, 0)), "msg",
                      &c8_r);
  *(uint32_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c8_msg)[0])[0] = c8_r.Header.Seq;
  *(real_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c8_msg)[0])[8])[0] =
    c8_r.Header.Stamp.Sec;
  *(real_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c8_msg)[0])[8])[8] =
    c8_r.Header.Stamp.Nsec;
  for (c8_i = 0; c8_i < 128; c8_i++) {
    ((uint8_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                  &((char_T *)chartInstance->c8_msg)[0])[24])[c8_i] =
      c8_r.Header.FrameId[c8_i];
  }

  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[0] =
    c8_r.Header.FrameId_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[4] =
    c8_r.Header.FrameId_SL_Info.ReceivedLength;
  for (c8_i1 = 0; c8_i1 < 16; c8_i1++) {
    for (c8_i2 = 0; c8_i2 < 128; c8_i2++) {
      ((uint8_T *)&((char_T *)
                    (c8_SL_Bus_sliding_controller_const_std_msgs_String *)
                    &((char_T *)
                      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)
                      &((char_T *)chartInstance->c8_msg)[160])[136 * (int32_T)
                    c8_i1])[0])[c8_i2] = c8_r.Name[c8_i1].Data[c8_i2];
    }

    *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i1])[128])[0] =
      c8_r.Name[c8_i1].Data_SL_Info.CurrentLength;
    *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i1])[128])[4] =
      c8_r.Name[c8_i1].Data_SL_Info.ReceivedLength;
  }

  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2336])[0] = c8_r.Name_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2336])[4] = c8_r.Name_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2400])[0] = c8_r.Position_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2400])[4] = c8_r.Position_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2464])[0] = c8_r.Velocity_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2464])[4] = c8_r.Velocity_SL_Info.ReceivedLength;
  for (c8_i3 = 0; c8_i3 < 7; c8_i3++) {
    ((real_T *)&((char_T *)chartInstance->c8_msg)[2344])[c8_i3] =
      c8_r.Position[c8_i3];
    ((real_T *)&((char_T *)chartInstance->c8_msg)[2408])[c8_i3] =
      c8_r.Velocity[c8_i3];
    ((real_T *)&((char_T *)chartInstance->c8_msg)[2472])[c8_i3] =
      c8_r.Effort[c8_i3];
  }

  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2528])[0] = c8_r.Effort_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2528])[4] = c8_r.Effort_SL_Info.ReceivedLength;
  chartInstance->c8_is_active_c8_sliding_controller_const =
    c8_k_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c8_u, 1)),
    "is_active_c8_sliding_controller_const");
  sf_mex_destroy(&c8_u);
  sf_mex_destroy(&c8_st);
}

static void sf_gateway_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState c8_r;
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_gpu_blankMsg;
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_gpu_msg;
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_gpu_r;
  int32_T c8_i1;
  int32_T c8_i5;
  int32_T c8_i9;
  hipMalloc(&c8_gpu_r, 2536UL);
  hipMalloc(&c8_gpu_msg, 2536UL);
  hipMalloc(&c8_b_gpu_blankMsg, 2536UL);
  chartInstance->c8_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  *(uint32_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c8_msg)[0])[0] = *(uint32_T *)
    &((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
       chartInstance->c8_blankMsg)[0])[0];
  *(real_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c8_msg)[0])[8])[0] = *(real_T *)
    &((char_T *)(c8_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
       (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
        chartInstance->c8_blankMsg)[0])[8])[0];
  *(real_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c8_msg)[0])[8])[8] = *(real_T *)
    &((char_T *)(c8_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
       (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
        chartInstance->c8_blankMsg)[0])[8])[8];
  hipMemcpy(c8_b_gpu_blankMsg, chartInstance->c8_blankMsg, 2536UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c8_gpu_msg, chartInstance->c8_msg, 2536UL, hipMemcpyHostToDevice);
  c8_sf_gateway_c8_sliding_controller_const_kernel1<<<dim3(1U, 1U, 1U), dim3
    (128U, 1U, 1U)>>>(c8_b_gpu_blankMsg, c8_gpu_msg);
  hipMemcpy(chartInstance->c8_msg, c8_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c8_blankMsg, c8_b_gpu_blankMsg, 2536UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[0] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_blankMsg)[0])[152])[0];
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[4] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_blankMsg)[0])[152])[4];
  for (c8_i1 = 0; c8_i1 < 16; c8_i1++) {
    hipMemcpy(c8_b_gpu_blankMsg, chartInstance->c8_blankMsg, 2536UL,
               hipMemcpyHostToDevice);
    hipMemcpy(c8_gpu_msg, chartInstance->c8_msg, 2536UL, hipMemcpyHostToDevice);
    c8_sf_gateway_c8_sliding_controller_const_kernel2<<<dim3(1U, 1U, 1U), dim3
      (128U, 1U, 1U)>>>(c8_i1, c8_b_gpu_blankMsg, c8_gpu_msg);
    hipMemcpy(chartInstance->c8_msg, c8_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
    hipMemcpy(chartInstance->c8_blankMsg, c8_b_gpu_blankMsg, 2536UL,
               hipMemcpyDeviceToHost);
    *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i1])[128])[0] = *(uint32_T *)
      &((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
         (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
          (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
           chartInstance->c8_blankMsg)[160])[136 * (int32_T)c8_i1])[128])[0];
    *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i1])[128])[4] = *(uint32_T *)
      &((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
         (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
          (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
           chartInstance->c8_blankMsg)[160])[136 * (int32_T)c8_i1])[128])[4];
  }

  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2336])[0] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_blankMsg)[2336])[0];
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2336])[4] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_blankMsg)[2336])[4];
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2400])[0] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_blankMsg)[2400])[0];
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2400])[4] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_blankMsg)[2400])[4];
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2464])[0] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_blankMsg)[2464])[0];
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2464])[4] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_blankMsg)[2464])[4];
  hipMemcpy(c8_b_gpu_blankMsg, chartInstance->c8_blankMsg, 2536UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c8_gpu_msg, chartInstance->c8_msg, 2536UL, hipMemcpyHostToDevice);
  c8_sf_gateway_c8_sliding_controller_const_kernel3<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c8_b_gpu_blankMsg, c8_gpu_msg);
  hipMemcpy(chartInstance->c8_msg, c8_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c8_blankMsg, c8_b_gpu_blankMsg, 2536UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2528])[0] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_blankMsg)[2528])[0];
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2528])[4] = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_blankMsg)[2528])[4];
  c8_r.Header.Seq = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[0];
  c8_r.Header.Stamp.Sec = *(real_T *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[8])[0];
  c8_r.Header.Stamp.Nsec = *(real_T *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[8])[8];
  hipMemcpy(c8_gpu_r, &c8_r, 2536UL, hipMemcpyHostToDevice);
  hipMemcpy(c8_gpu_msg, chartInstance->c8_msg, 2536UL, hipMemcpyHostToDevice);
  c8_sf_gateway_c8_sliding_controller_const_kernel4<<<dim3(1U, 1U, 1U), dim3
    (128U, 1U, 1U)>>>(c8_gpu_msg, c8_gpu_r);
  hipMemcpy(chartInstance->c8_msg, c8_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c8_r, c8_gpu_r, 2536UL, hipMemcpyDeviceToHost);
  c8_r.Header.FrameId_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[0];
  c8_r.Header.FrameId_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[4];
  for (c8_i5 = 0; c8_i5 < 16; c8_i5++) {
    hipMemcpy(c8_gpu_r, &c8_r, 2536UL, hipMemcpyHostToDevice);
    hipMemcpy(c8_gpu_msg, chartInstance->c8_msg, 2536UL, hipMemcpyHostToDevice);
    c8_sf_gateway_c8_sliding_controller_const_kernel5<<<dim3(1U, 1U, 1U), dim3
      (128U, 1U, 1U)>>>(c8_i5, c8_gpu_msg, c8_gpu_r);
    hipMemcpy(chartInstance->c8_msg, c8_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
    hipMemcpy(&c8_r, c8_gpu_r, 2536UL, hipMemcpyDeviceToHost);
    c8_r.Name[c8_i5].Data_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
      (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i5])[128])[0];
    c8_r.Name[c8_i5].Data_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
      (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i5])[128])[4];
  }

  c8_r.Name_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2336])[0];
  c8_r.Name_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2336])[4];
  c8_r.Position_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2400])[0];
  c8_r.Position_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2400])[4];
  c8_r.Velocity_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2464])[0];
  c8_r.Velocity_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2464])[4];
  hipMemcpy(c8_gpu_r, &c8_r, 2536UL, hipMemcpyHostToDevice);
  hipMemcpy(c8_gpu_msg, chartInstance->c8_msg, 2536UL, hipMemcpyHostToDevice);
  c8_sf_gateway_c8_sliding_controller_const_kernel6<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c8_gpu_msg, c8_gpu_r);
  hipMemcpy(chartInstance->c8_msg, c8_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c8_r, c8_gpu_r, 2536UL, hipMemcpyDeviceToHost);
  c8_r.Effort_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2528])[0];
  c8_r.Effort_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c8_msg)
    [2528])[4];
  c8_eML_blk_kernel(chartInstance, &c8_r);
  *(uint32_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c8_msg)[0])[0] = c8_r.Header.Seq;
  *(real_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c8_msg)[0])[8])[0] =
    c8_r.Header.Stamp.Sec;
  *(real_T *)&((char_T *)(c8_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c8_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c8_msg)[0])[8])[8] =
    c8_r.Header.Stamp.Nsec;
  hipMemcpy(c8_gpu_r, &c8_r, 2536UL, hipMemcpyHostToDevice);
  hipMemcpy(c8_gpu_msg, chartInstance->c8_msg, 2536UL, hipMemcpyHostToDevice);
  c8_sf_gateway_c8_sliding_controller_const_kernel7<<<dim3(1U, 1U, 1U), dim3
    (128U, 1U, 1U)>>>(c8_gpu_r, c8_gpu_msg);
  hipMemcpy(chartInstance->c8_msg, c8_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[0] =
    c8_r.Header.FrameId_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c8_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c8_msg)[0])[152])[4] =
    c8_r.Header.FrameId_SL_Info.ReceivedLength;
  for (c8_i9 = 0; c8_i9 < 16; c8_i9++) {
    hipMemcpy(c8_gpu_msg, chartInstance->c8_msg, 2536UL, hipMemcpyHostToDevice);
    c8_sf_gateway_c8_sliding_controller_const_kernel8<<<dim3(1U, 1U, 1U), dim3
      (128U, 1U, 1U)>>>(c8_gpu_r, c8_i9, c8_gpu_msg);
    hipMemcpy(chartInstance->c8_msg, c8_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
    *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i9])[128])[0] =
      c8_r.Name[c8_i9].Data_SL_Info.CurrentLength;
    *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c8_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c8_msg)[160])[136 * (int32_T)c8_i9])[128])[4] =
      c8_r.Name[c8_i9].Data_SL_Info.ReceivedLength;
  }

  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2336])[0] = c8_r.Name_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2336])[4] = c8_r.Name_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2400])[0] = c8_r.Position_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2400])[4] = c8_r.Position_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2464])[0] = c8_r.Velocity_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2464])[4] = c8_r.Velocity_SL_Info.ReceivedLength;
  hipMemcpy(c8_gpu_msg, chartInstance->c8_msg, 2536UL, hipMemcpyHostToDevice);
  c8_sf_gateway_c8_sliding_controller_const_kernel9<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c8_gpu_r, c8_gpu_msg);
  hipMemcpy(chartInstance->c8_msg, c8_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2528])[0] = c8_r.Effort_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c8_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c8_msg)[2528])[4] = c8_r.Effort_SL_Info.ReceivedLength;
  c8_do_animation_call_c8_sliding_controller_const(chartInstance);
  hipFree(c8_b_gpu_blankMsg);
  hipFree(c8_gpu_msg);
  hipFree(c8_gpu_r);
}

static void mdl_start_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c8_errCode;
  hipFree(*chartInstance->c8_gpu_uv5);
  hipFree(*chartInstance->c8_gpu_uv4);
  hipFree(*chartInstance->c8_gpu_uv3);
  hipFree(*chartInstance->c8_gpu_uv2);
  hipFree(*chartInstance->c8_gpu_uv1);
  hipFree(*chartInstance->c8_gpu_uv);
  hipFree(chartInstance->c8_gpu_blankMsg);
  c8_errCode = hipGetLastError();
  if (c8_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c8_errCode, (char_T *)hipGetErrorName
                       (c8_errCode), (char_T *)hipGetErrorString(c8_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c8_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc8_sliding_controller_const
  (SFc8_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c8_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c8_nameCaptureInfo = NULL;
  c8_nameCaptureInfo = NULL;
  sf_mex_assign(&c8_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c8_nameCaptureInfo;
}

static void c8_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_b_msg, const char_T *c8_identifier,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_y)
{
  emlrtMsgIdentifier c8_thisId;
  c8_thisId.fIdentifier = const_cast<const char_T *>(c8_identifier);
  c8_thisId.fParent = NULL;
  c8_thisId.bParentIsCell = false;
  c8_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c8_b_msg), &c8_thisId, c8_y);
  sf_mex_destroy(&c8_b_msg);
}

static void c8_b_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_y)
{
  static const char_T *c8_fieldNames[9] = { "Header", "Name", "Name_SL_Info",
    "Position", "Position_SL_Info", "Velocity", "Velocity_SL_Info", "Effort",
    "Effort_SL_Info" };

  emlrtMsgIdentifier c8_thisId;
  c8_thisId.fParent = c8_parentId;
  c8_thisId.bParentIsCell = false;
  sf_mex_check_struct(c8_parentId, c8_u, 9, c8_fieldNames, 0U, NULL);
  c8_thisId.fIdentifier = "Header";
  c8_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c8_u, "Header",
    "Header", 0)), &c8_thisId, &c8_y->Header);
  c8_thisId.fIdentifier = "Name";
  c8_i_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c8_u, "Name",
    "Name", 0)), &c8_thisId, c8_y->Name);
  c8_thisId.fIdentifier = "Name_SL_Info";
  c8_y->Name_SL_Info = c8_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c8_u, "Name_SL_Info", "Name_SL_Info", 0)), &c8_thisId);
  c8_thisId.fIdentifier = "Position";
  c8_j_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c8_u,
    "Position", "Position", 0)), &c8_thisId, c8_y->Position);
  c8_thisId.fIdentifier = "Position_SL_Info";
  c8_y->Position_SL_Info = c8_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c8_u, "Position_SL_Info", "Position_SL_Info", 0)),
    &c8_thisId);
  c8_thisId.fIdentifier = "Velocity";
  c8_j_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c8_u,
    "Velocity", "Velocity", 0)), &c8_thisId, c8_y->Velocity);
  c8_thisId.fIdentifier = "Velocity_SL_Info";
  c8_y->Velocity_SL_Info = c8_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c8_u, "Velocity_SL_Info", "Velocity_SL_Info", 0)),
    &c8_thisId);
  c8_thisId.fIdentifier = "Effort";
  c8_j_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c8_u, "Effort",
    "Effort", 0)), &c8_thisId, c8_y->Effort);
  c8_thisId.fIdentifier = "Effort_SL_Info";
  c8_y->Effort_SL_Info = c8_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c8_u, "Effort_SL_Info", "Effort_SL_Info", 0)), &c8_thisId);
  sf_mex_destroy(&c8_u);
}

static void c8_c_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  c8_SL_Bus_sliding_controller_const_std_msgs_Header *c8_y)
{
  static const char_T *c8_fieldNames[4] = { "Seq", "Stamp", "FrameId",
    "FrameId_SL_Info" };

  emlrtMsgIdentifier c8_thisId;
  c8_thisId.fParent = c8_parentId;
  c8_thisId.bParentIsCell = false;
  sf_mex_check_struct(c8_parentId, c8_u, 4, c8_fieldNames, 0U, NULL);
  c8_thisId.fIdentifier = "Seq";
  c8_y->Seq = c8_d_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c8_u, "Seq", "Seq", 0)), &c8_thisId);
  c8_thisId.fIdentifier = "Stamp";
  c8_y->Stamp = c8_e_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c8_u, "Stamp", "Stamp", 0)), &c8_thisId);
  c8_thisId.fIdentifier = "FrameId";
  c8_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c8_u,
    "FrameId", "FrameId", 0)), &c8_thisId, c8_y->FrameId);
  c8_thisId.fIdentifier = "FrameId_SL_Info";
  c8_y->FrameId_SL_Info = c8_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c8_u, "FrameId_SL_Info", "FrameId_SL_Info", 0)), &c8_thisId);
  sf_mex_destroy(&c8_u);
}

static uint32_T c8_d_emlrt_marshallIn
  (SFc8_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c8_u, const emlrtMsgIdentifier *c8_parentId)
{
  uint32_T c8_b_u;
  uint32_T c8_y;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_u), &c8_b_u, 1, 7, 0U, 0, 0U, 0);
  c8_y = c8_b_u;
  sf_mex_destroy(&c8_u);
  return c8_y;
}

static c8_SL_Bus_sliding_controller_const_ros_time_Time c8_e_emlrt_marshallIn
  (SFc8_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c8_u, const emlrtMsgIdentifier *c8_parentId)
{
  static const char_T *c8_fieldNames[2] = { "Sec", "Nsec" };

  c8_SL_Bus_sliding_controller_const_ros_time_Time c8_y;
  emlrtMsgIdentifier c8_thisId;
  c8_thisId.fParent = c8_parentId;
  c8_thisId.bParentIsCell = false;
  sf_mex_check_struct(c8_parentId, c8_u, 2, c8_fieldNames, 0U, NULL);
  c8_thisId.fIdentifier = "Sec";
  c8_y.Sec = c8_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c8_u, "Sec", "Sec", 0)), &c8_thisId);
  c8_thisId.fIdentifier = "Nsec";
  c8_y.Nsec = c8_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c8_u, "Nsec", "Nsec", 0)), &c8_thisId);
  sf_mex_destroy(&c8_u);
  return c8_y;
}

static real_T c8_f_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId)
{
  real_T c8_d;
  real_T c8_y;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_u), &c8_d, 1, 0, 0U, 0, 0U, 0);
  c8_y = c8_d;
  sf_mex_destroy(&c8_u);
  return c8_y;
}

static void c8_g_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  uint8_T c8_y[128])
{
  int32_T c8_i;
  uint8_T c8_uv[128];
  sf_mex_import(c8_parentId, sf_mex_dup(c8_u), c8_uv, 1, 3, 0U, 1, 0U, 1, 128);
  for (c8_i = 0; c8_i < 128; c8_i++) {
    c8_y[c8_i] = c8_uv[c8_i];
  }

  sf_mex_destroy(&c8_u);
}

static c8_SL_Bus_ROSVariableLengthArrayInfo c8_h_emlrt_marshallIn
  (SFc8_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c8_u, const emlrtMsgIdentifier *c8_parentId)
{
  static const char_T *c8_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c8_SL_Bus_ROSVariableLengthArrayInfo c8_y;
  emlrtMsgIdentifier c8_thisId;
  c8_thisId.fParent = c8_parentId;
  c8_thisId.bParentIsCell = false;
  sf_mex_check_struct(c8_parentId, c8_u, 2, c8_fieldNames, 0U, NULL);
  c8_thisId.fIdentifier = "CurrentLength";
  c8_y.CurrentLength = c8_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c8_u, "CurrentLength", "CurrentLength", 0)), &c8_thisId);
  c8_thisId.fIdentifier = "ReceivedLength";
  c8_y.ReceivedLength = c8_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c8_u, "ReceivedLength", "ReceivedLength", 0)), &c8_thisId);
  sf_mex_destroy(&c8_u);
  return c8_y;
}

static void c8_i_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  c8_SL_Bus_sliding_controller_const_std_msgs_String c8_y[16])
{
  static const char_T *c8_fieldNames[2] = { "Data", "Data_SL_Info" };

  emlrtMsgIdentifier c8_thisId;
  int32_T c8_i;
  uint32_T c8_uv[1];
  c8_uv[0] = 16U;
  c8_thisId.fParent = c8_parentId;
  c8_thisId.bParentIsCell = false;
  sf_mex_check_struct(c8_parentId, c8_u, 2, c8_fieldNames, 1U, c8_uv);
  for (c8_i = 0; c8_i < 16; c8_i++) {
    c8_thisId.fIdentifier = "Data";
    c8_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c8_u, "Data",
      "Data", c8_i)), &c8_thisId, c8_y[c8_i].Data);
    c8_thisId.fIdentifier = "Data_SL_Info";
    c8_y[c8_i].Data_SL_Info = c8_h_emlrt_marshallIn(chartInstance, sf_mex_dup
      (sf_mex_getfield(c8_u, "Data_SL_Info", "Data_SL_Info", c8_i)), &c8_thisId);
  }

  sf_mex_destroy(&c8_u);
}

static void c8_j_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId,
  real_T c8_y[7])
{
  real_T c8_dv[7];
  int32_T c8_i;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_u), c8_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c8_i = 0; c8_i < 7; c8_i++) {
    c8_y[c8_i] = c8_dv[c8_i];
  }

  sf_mex_destroy(&c8_u);
}

static uint8_T c8_k_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_b_is_active_c8_sliding_controller_const,
  const char_T *c8_identifier)
{
  emlrtMsgIdentifier c8_thisId;
  uint8_T c8_y;
  c8_thisId.fIdentifier = const_cast<const char_T *>(c8_identifier);
  c8_thisId.fParent = NULL;
  c8_thisId.bParentIsCell = false;
  c8_y = c8_l_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c8_b_is_active_c8_sliding_controller_const), &c8_thisId);
  sf_mex_destroy(&c8_b_is_active_c8_sliding_controller_const);
  return c8_y;
}

static uint8_T c8_l_emlrt_marshallIn(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c8_u, const emlrtMsgIdentifier *c8_parentId)
{
  uint8_T c8_b_u;
  uint8_T c8_y;
  sf_mex_import(c8_parentId, sf_mex_dup(c8_u), &c8_b_u, 1, 3, 0U, 0, 0U, 0);
  c8_y = c8_b_u;
  sf_mex_destroy(&c8_u);
  return c8_y;
}

static void c8_eML_blk_kernel(SFc8_sliding_controller_constInstanceStruct
  *chartInstance, c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c8_b_blankMsg)
{
  static uint8_T c8_uv5[15] = { 82U, 105U, 103U, 104U, 116U, 72U, 117U, 109U,
    97U, 110U, 65U, 110U, 107U, 108U, 101U };

  static uint8_T c8_uv2[14] = { 76U, 101U, 102U, 116U, 72U, 117U, 109U, 97U,
    110U, 65U, 110U, 107U, 108U, 101U };

  static uint8_T c8_uv4[14] = { 82U, 105U, 103U, 104U, 116U, 72U, 117U, 109U,
    97U, 110U, 75U, 110U, 101U, 101U };

  static uint8_T c8_uv1[13] = { 76U, 101U, 102U, 116U, 72U, 117U, 109U, 97U,
    110U, 75U, 110U, 101U, 101U };

  static uint8_T c8_uv3[13] = { 82U, 105U, 103U, 104U, 116U, 72U, 117U, 109U,
    97U, 110U, 72U, 105U, 112U };

  static uint8_T c8_uv[12] = { 76U, 101U, 102U, 116U, 72U, 117U, 109U, 97U, 110U,
    72U, 105U, 112U };

  hipMemcpy(chartInstance->c8_gpu_blankMsg, c8_b_blankMsg, 2536UL,
             hipMemcpyHostToDevice);
  c8_eML_blk_kernel_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c8_gpu_blankMsg);
  hipMemcpy(*chartInstance->c8_gpu_uv, c8_uv, 12UL, hipMemcpyHostToDevice);
  c8_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c8_gpu_uv, chartInstance->c8_gpu_blankMsg);
  c8_eML_blk_kernel_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c8_gpu_blankMsg);
  hipMemcpy(*chartInstance->c8_gpu_uv1, c8_uv1, 13UL, hipMemcpyHostToDevice);
  c8_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c8_gpu_uv1, chartInstance->c8_gpu_blankMsg);
  c8_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c8_gpu_blankMsg);
  hipMemcpy(*chartInstance->c8_gpu_uv2, c8_uv2, 14UL, hipMemcpyHostToDevice);
  c8_eML_blk_kernel_kernel15<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c8_gpu_uv2, chartInstance->c8_gpu_blankMsg);
  c8_eML_blk_kernel_kernel16<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c8_gpu_blankMsg);
  hipMemcpy(*chartInstance->c8_gpu_uv3, c8_uv3, 13UL, hipMemcpyHostToDevice);
  c8_eML_blk_kernel_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c8_gpu_uv3, chartInstance->c8_gpu_blankMsg);
  c8_eML_blk_kernel_kernel18<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c8_gpu_blankMsg);
  hipMemcpy(*chartInstance->c8_gpu_uv4, c8_uv4, 14UL, hipMemcpyHostToDevice);
  c8_eML_blk_kernel_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c8_gpu_uv4, chartInstance->c8_gpu_blankMsg);
  c8_eML_blk_kernel_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c8_gpu_blankMsg);
  hipMemcpy(*chartInstance->c8_gpu_uv5, c8_uv5, 15UL, hipMemcpyHostToDevice);
  c8_eML_blk_kernel_kernel21<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c8_gpu_uv5, chartInstance->c8_gpu_blankMsg);
  c8_eML_blk_kernel_kernel22<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c8_gpu_blankMsg);
  hipMemcpy(c8_b_blankMsg, chartInstance->c8_gpu_blankMsg, 2536UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(128, 1) void
  c8_sf_gateway_c8_sliding_controller_const_kernel1
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg,
   c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg)
{
  uint64_T c8_threadId;
  int32_T c8_i;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i = (int32_T)c8_threadId;
  if (c8_i < 128) {
    c8_b_msg->Header.FrameId[c8_i] = c8_b_blankMsg->Header.FrameId[c8_i];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c8_sf_gateway_c8_sliding_controller_const_kernel2(const int32_T c8_i1,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg)
{
  uint64_T c8_threadId;
  int32_T c8_i2;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i2 = (int32_T)c8_threadId;
  if (c8_i2 < 128) {
    c8_b_msg->Name[c8_i1].Data[c8_i2] = c8_b_blankMsg->Name[c8_i1].Data[c8_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c8_sf_gateway_c8_sliding_controller_const_kernel3
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg,
   c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg)
{
  uint64_T c8_threadId;
  int32_T c8_i3;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i3 = (int32_T)c8_threadId;
  if (c8_i3 < 7) {
    c8_b_msg->Position[c8_i3] = c8_b_blankMsg->Position[c8_i3];
    c8_b_msg->Velocity[c8_i3] = c8_b_blankMsg->Velocity[c8_i3];
    c8_b_msg->Effort[c8_i3] = c8_b_blankMsg->Effort[c8_i3];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c8_sf_gateway_c8_sliding_controller_const_kernel4
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg,
   c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r)
{
  uint64_T c8_threadId;
  int32_T c8_i4;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i4 = (int32_T)c8_threadId;
  if (c8_i4 < 128) {
    c8_r->Header.FrameId[c8_i4] = c8_b_msg->Header.FrameId[c8_i4];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c8_sf_gateway_c8_sliding_controller_const_kernel5(const int32_T c8_i5,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r)
{
  uint64_T c8_threadId;
  int32_T c8_i6;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i6 = (int32_T)c8_threadId;
  if (c8_i6 < 128) {
    c8_r->Name[c8_i5].Data[c8_i6] = c8_b_msg->Name[c8_i5].Data[c8_i6];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c8_sf_gateway_c8_sliding_controller_const_kernel6
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg,
   c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r)
{
  uint64_T c8_threadId;
  int32_T c8_i7;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i7 = (int32_T)c8_threadId;
  if (c8_i7 < 7) {
    c8_r->Position[c8_i7] = c8_b_msg->Position[c8_i7];
    c8_r->Velocity[c8_i7] = c8_b_msg->Velocity[c8_i7];
    c8_r->Effort[c8_i7] = c8_b_msg->Effort[c8_i7];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c8_sf_gateway_c8_sliding_controller_const_kernel7(const
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg)
{
  uint64_T c8_threadId;
  int32_T c8_i8;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i8 = (int32_T)c8_threadId;
  if (c8_i8 < 128) {
    c8_b_msg->Header.FrameId[c8_i8] = c8_r->Header.FrameId[c8_i8];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c8_sf_gateway_c8_sliding_controller_const_kernel8(const
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r, const int32_T
  c8_i9, c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg)
{
  uint64_T c8_threadId;
  int32_T c8_i10;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i10 = (int32_T)c8_threadId;
  if (c8_i10 < 128) {
    c8_b_msg->Name[c8_i9].Data[c8_i10] = c8_r->Name[c8_i9].Data[c8_i10];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c8_sf_gateway_c8_sliding_controller_const_kernel9(const
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_r,
  c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_msg)
{
  uint64_T c8_threadId;
  int32_T c8_i11;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i11 = (int32_T)c8_threadId;
  if (c8_i11 < 7) {
    c8_b_msg->Position[c8_i11] = c8_r->Position[c8_i11];
    c8_b_msg->Velocity[c8_i11] = c8_r->Velocity[c8_i11];
    c8_b_msg->Effort[c8_i11] = c8_r->Effort[c8_i11];
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel10
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_tmpIdx;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_tmpIdx = (int32_T)c8_threadId;
  if (c8_tmpIdx < 1) {
    c8_b_blankMsg->Name_SL_Info.CurrentLength = 6U;
    c8_b_blankMsg->Effort_SL_Info.CurrentLength = 6U;
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel11(const
  uint8_T c8_uv[12], c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_i;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i = (int32_T)c8_threadId;
  if (c8_i < 12) {
    c8_b_blankMsg->Name[0].Data[c8_i] = c8_uv[c8_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel12
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_tmpIdx;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_tmpIdx = (int32_T)c8_threadId;
  if (c8_tmpIdx < 1) {
    c8_b_blankMsg->Name[0].Data_SL_Info.CurrentLength = 12U;
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel13(const
  uint8_T c8_uv1[13], c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *
  c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_i1;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i1 = (int32_T)c8_threadId;
  if (c8_i1 < 13) {
    c8_b_blankMsg->Name[1].Data[c8_i1] = c8_uv1[c8_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel14
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_tmpIdx;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_tmpIdx = (int32_T)c8_threadId;
  if (c8_tmpIdx < 1) {
    c8_b_blankMsg->Name[1].Data_SL_Info.CurrentLength = 13U;
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel15(const
  uint8_T c8_uv2[14], c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *
  c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_i2;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i2 = (int32_T)c8_threadId;
  if (c8_i2 < 14) {
    c8_b_blankMsg->Name[2].Data[c8_i2] = c8_uv2[c8_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel16
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_tmpIdx;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_tmpIdx = (int32_T)c8_threadId;
  if (c8_tmpIdx < 1) {
    c8_b_blankMsg->Name[2].Data_SL_Info.CurrentLength = 14U;
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel17(const
  uint8_T c8_uv3[13], c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *
  c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_i3;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i3 = (int32_T)c8_threadId;
  if (c8_i3 < 13) {
    c8_b_blankMsg->Name[3].Data[c8_i3] = c8_uv3[c8_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel18
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_tmpIdx;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_tmpIdx = (int32_T)c8_threadId;
  if (c8_tmpIdx < 1) {
    c8_b_blankMsg->Name[3].Data_SL_Info.CurrentLength = 13U;
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel19(const
  uint8_T c8_uv4[14], c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *
  c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_i4;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i4 = (int32_T)c8_threadId;
  if (c8_i4 < 14) {
    c8_b_blankMsg->Name[4].Data[c8_i4] = c8_uv4[c8_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel20
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_tmpIdx;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_tmpIdx = (int32_T)c8_threadId;
  if (c8_tmpIdx < 1) {
    c8_b_blankMsg->Name[4].Data_SL_Info.CurrentLength = 14U;
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel21(const
  uint8_T c8_uv5[15], c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *
  c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_i5;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_i5 = (int32_T)c8_threadId;
  if (c8_i5 < 15) {
    c8_b_blankMsg->Name[5].Data[c8_i5] = c8_uv5[c8_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c8_eML_blk_kernel_kernel22
  (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c8_b_blankMsg)
{
  uint64_T c8_threadId;
  int32_T c8_tmpIdx;
  c8_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c8_tmpIdx = (int32_T)c8_threadId;
  if (c8_tmpIdx < 1) {
    c8_b_blankMsg->Name[5].Data_SL_Info.CurrentLength = 15U;
  }
}

static void init_dsm_address_info(SFc8_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc8_sliding_controller_constInstanceStruct
  *chartInstance)
{
  chartInstance->c8_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c8_blankMsg =
    (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c8_msg =
    (c8_SL_Bus_sliding_controller_const_sensor_msgs_JointState *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c8_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3784877350U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3397629398U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2349802474U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(426083053U);
}

mxArray *sf_c8_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c8_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c8_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c8_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvyqK+yH8zOL4xOSSzLLU+GSL+OKczJTMvPT45Py8kqL8nJzUIhCzuAQW/0AAAEcFHtg="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c8_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sOiu2rNucHOolxgc4QjETf";
}

static void sf_opaque_initialize_c8_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c8_sliding_controller_const(void *chartInstanceVar)
{
  enable_c8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c8_sliding_controller_const(void *chartInstanceVar)
{
  disable_c8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c8_sliding_controller_const(void *chartInstanceVar)
{
  sf_gateway_c8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c8_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c8_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c8_sliding_controller_const(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc8_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c8_sliding_controller_const
      ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc8_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c8_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c8_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc8_sliding_controller_const
    ((SFc8_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c8_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c8_sliding_controller_const
      ((SFc8_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc8_sliding_controller_const
      ((SFc8_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c8_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV0tv00AQdkNBlJcqhDgh0UsFJ4QQh56gbR4iUksKToFbtV1P4iXrXbOPtL1w5cQ/4cB/4cK",
    "Rn8CFAzdmbSdNHduhRFSAsOTYa38zOzPfPBxvob3t4XENz/XrnncBrxfxrHnpcT5bL0yc6fNF70",
    "62fo9CwkY7RJFIe5WHIBE8By25NUyKtujJQhgTPVAgKGJjqUyZNs0iy5kYtKygTp9+GTIa+qG0P",
    "NhEWRJ0BD9CbbE1O6inwRRQ0wIITKik7YctTvpji5U5qIdAB9pGVS5oML6NnVl623LDYg7NQ6Bt",
    "oQ1Bi/Wxbb4hBurmsNRN56n2R0AZxZwRUehtSLQPMQbYwG4c4G/HGnQqD6MhUWYTQjIEvcUGiU4",
    "pIK+TaXyxzwQxUjHCmxGvO8Fp23Y42rMtA+AVAUHbNhWQQSyZMOX8+y30tCnIPocG7Nt+uTYf3l",
    "hH/gsGB6BK49aryyEo0oeOKN00CUjzMGFrnCXTMMMieEHUBkX+NASl2YuZo32CPEEXJcpgkDjZ1",
    "l3FhhjeUm02arvMnFUyNkrJ1rNgibbmEKpYGGtrUVEnnOtSWFfGWzAEnmhtEEOqYanWYpzWLOhK",
    "DLBL7/JqsIIh8RmsLkXACuka5gBJ33mKjeUkklptZFTH5G1sbU2/noa1hQHVIxSKuoAiTAPGLAl",
    "vubaAacc9AtEqk5hXBE4zZBbK0z0rGgdSDTAmFU3k2AXHaCkw0n3kEithV2PRVMEcl7NwlNAQAt",
    "dgGIdtLBvEFsREu9a2gXU3ZOaoAZoqFhewarHqsA01XUIdxbArBkIeiJaSkZ/1+DS8AJhtRAkm+",
    "pvYv9RRC3cvosLNs/ve8Ty78hPzbCSXv96d0LNQoMebuOb3XapV71vDu4XR/J2Qu5rbZzEn53DL",
    "7kZ+vxe+u/RttXH505e3H9bm2f9j7XTz/1q2vjVqtOPCGU7lq8M+mbBrsUD/zQn9y9lad5h9oJ5",
    "a+qQj+WGfPnz2utntZfzMsLeWs3f0fMV1fMyyJD8VbQfZh4lbE5uOa6d/bcLeCzPisZQ9T4+vj+",
    "eTv7Ge57EoXudOxOucq9wTefvr+6+szyef7v9qhv2rOb5Xk7m+R1y3gD26tqc5C7DS96gURknOQ",
    "blbbXL1+av5flo574zl/hY7/8v9ft5/Zn6dtZx3xnLz+nfaOf6n4avmgZfDL//Bfsz7ffW78Z+9",
    "030H3c7Wj8Z/deoh40HBV2/2Gj9he0Vv/4E8/QHYrbUD",
    ""
  };

  static char newstr [1245] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c8_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c8_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(788098816U));
  ssSetChecksum1(S,(4111107944U));
  ssSetChecksum2(S,(3540730917U));
  ssSetChecksum3(S,(950828768U));
}

static void mdlRTW_c8_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c8_sliding_controller_const(SimStruct *S)
{
  SFc8_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc8_sliding_controller_constInstanceStruct *)utMalloc(sizeof
    (SFc8_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc8_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc8_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c8_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c8_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c8_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c8_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c8_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c8_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c8_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c8_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c8_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c8_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c8_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c8_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c8_JITStateAnimation,
    chartInstance->c8_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c8_sliding_controller_const(chartInstance);
}

void c8_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c8_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c8_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c8_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c8_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
