/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c7_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void c7_do_animation_call_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c7_st);
static void sf_gateway_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance);
static void c7_eML_blk_kernel(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, real_T c7_b_q[2], real_T c7_b_y[4]);
static void c7_emlrt_marshallIn(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c7_b_y, const char_T *c7_identifier, real_T
  c7_c_y[4]);
static void c7_b_emlrt_marshallIn(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId,
  real_T c7_b_y[4]);
static uint8_T c7_c_emlrt_marshallIn(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c7_b_is_active_c7_sliding_controller_const,
  const char_T *c7_identifier);
static uint8_T c7_d_emlrt_marshallIn(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId);
static void init_dsm_address_info(SFc7_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc7_sliding_controller_constInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c7_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c7_is_active_c7_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c7_do_animation_call_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c7_b_y = NULL;
  const mxArray *c7_c_y = NULL;
  const mxArray *c7_d_y = NULL;
  const mxArray *c7_st;
  c7_st = NULL;
  c7_b_y = NULL;
  sf_mex_assign(&c7_b_y, sf_mex_createcellmatrix(2, 1), false);
  c7_c_y = NULL;
  sf_mex_assign(&c7_c_y, sf_mex_create("y", *chartInstance->c7_y, 0, 0U, 1U, 0U,
    2, 2, 2), false);
  sf_mex_setcell(c7_b_y, 0, c7_c_y);
  c7_d_y = NULL;
  sf_mex_assign(&c7_d_y, sf_mex_create("y",
    &chartInstance->c7_is_active_c7_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c7_b_y, 1, c7_d_y);
  sf_mex_assign(&c7_st, c7_b_y, false);
  return c7_st;
}

static void set_sim_state_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c7_st)
{
  const mxArray *c7_u;
  c7_u = sf_mex_dup(c7_st);
  c7_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c7_u, 0)), "y",
                      *chartInstance->c7_y);
  chartInstance->c7_is_active_c7_sliding_controller_const =
    c7_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c7_u, 1)),
    "is_active_c7_sliding_controller_const");
  sf_mex_destroy(&c7_u);
  sf_mex_destroy(&c7_st);
}

static void sf_gateway_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c7_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c7_eML_blk_kernel(chartInstance, *chartInstance->c7_q, *chartInstance->c7_y);
  c7_do_animation_call_c7_sliding_controller_const(chartInstance);
}

static void mdl_start_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c7_errCode;
  c7_errCode = hipGetLastError();
  if (c7_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c7_errCode, (char_T *)hipGetErrorName
                       (c7_errCode), (char_T *)hipGetErrorString(c7_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c7_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc7_sliding_controller_const
  (SFc7_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c7_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c7_nameCaptureInfo = NULL;
  c7_nameCaptureInfo = NULL;
  sf_mex_assign(&c7_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c7_nameCaptureInfo;
}

static void c7_eML_blk_kernel(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, real_T c7_b_q[2], real_T c7_b_y[4])
{
  c7_b_y[0] = -0.324 * muDoubleScalarSin(c7_b_q[0]);
  c7_b_y[2] = 0.0;
  c7_b_y[1] = 0.324 * muDoubleScalarCos(c7_b_q[0]);
  c7_b_y[3] = 0.0;
}

static void c7_emlrt_marshallIn(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c7_b_y, const char_T *c7_identifier, real_T
  c7_c_y[4])
{
  emlrtMsgIdentifier c7_thisId;
  c7_thisId.fIdentifier = const_cast<const char_T *>(c7_identifier);
  c7_thisId.fParent = NULL;
  c7_thisId.bParentIsCell = false;
  c7_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c7_b_y), &c7_thisId, c7_c_y);
  sf_mex_destroy(&c7_b_y);
}

static void c7_b_emlrt_marshallIn(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId,
  real_T c7_b_y[4])
{
  real_T c7_dv[4];
  int32_T c7_i;
  sf_mex_import(c7_parentId, sf_mex_dup(c7_u), c7_dv, 1, 0, 0U, 1, 0U, 2, 2, 2);
  for (c7_i = 0; c7_i < 4; c7_i++) {
    c7_b_y[c7_i] = c7_dv[c7_i];
  }

  sf_mex_destroy(&c7_u);
}

static uint8_T c7_c_emlrt_marshallIn(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c7_b_is_active_c7_sliding_controller_const,
  const char_T *c7_identifier)
{
  emlrtMsgIdentifier c7_thisId;
  uint8_T c7_b_y;
  c7_thisId.fIdentifier = const_cast<const char_T *>(c7_identifier);
  c7_thisId.fParent = NULL;
  c7_thisId.bParentIsCell = false;
  c7_b_y = c7_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c7_b_is_active_c7_sliding_controller_const), &c7_thisId);
  sf_mex_destroy(&c7_b_is_active_c7_sliding_controller_const);
  return c7_b_y;
}

static uint8_T c7_d_emlrt_marshallIn(SFc7_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId)
{
  uint8_T c7_b_u;
  uint8_T c7_b_y;
  sf_mex_import(c7_parentId, sf_mex_dup(c7_u), &c7_b_u, 1, 3, 0U, 0, 0U, 0);
  c7_b_y = c7_b_u;
  sf_mex_destroy(&c7_u);
  return c7_b_y;
}

static void init_dsm_address_info(SFc7_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc7_sliding_controller_constInstanceStruct
  *chartInstance)
{
  chartInstance->c7_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c7_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c7_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c7_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(536815311U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1945133011U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2900394615U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(32238838U);
}

mxArray *sf_c7_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c7_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c7_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c7_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cnm8cU5mSmZeenxyfl5JUX5OTmpRSBmcQmSPwF/xh4F"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c7_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sFqlSox9r5ekjZXT2w93GAD";
}

static void sf_opaque_initialize_c7_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c7_sliding_controller_const(void *chartInstanceVar)
{
  enable_c7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c7_sliding_controller_const(void *chartInstanceVar)
{
  disable_c7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c7_sliding_controller_const(void *chartInstanceVar)
{
  sf_gateway_c7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c7_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c7_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c7_sliding_controller_const(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc7_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c7_sliding_controller_const
      ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc7_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c7_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c7_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc7_sliding_controller_const
    ((SFc7_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c7_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c7_sliding_controller_const
      ((SFc7_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc7_sliding_controller_const
      ((SFc7_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c7_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u00AQdqOCQPyoQkhICAkuCI4IhIALtM0PRKS0wikgLtV2PYmXrHfN/iTNC3DjTThw50V",
    "4Ai5ceARmbScNju1QIipAWHLs9X4zOzPf7MzGW2lveXidx3v9guedxOcpvGteep3Ixiszd/p91b",
    "uRjd+jkLDRDlEk0l7lJUgEz0FLbg2Toi16shDGRA8UCIrYWCpTpk2zyHImBi0rqNOnX4aMhn4oL",
    "Q82UZYE24KPUVtszQ7qaTAF1LQAAhMqafthi5P+1GJlRvUQ6EDbqMoFDca3sTNLb1luWMyheQC0",
    "LbQhaLE+tM03xEDdHJS66TzV/gQoo5gzIgq9DYn2IcYAG9iNA/zdtgadysNoSJTZhJAMQXfYINE",
    "pBeR1Mo0T+0wQIxUjvBnxuhOct22Hoz1bMgBeERC0bVMBGcSSCVPOv99CT5uC7HNowL7tl2vz4a",
    "115L9gMAJVGrdeXQ5BkT5si9JFk4A0DxK2plkyDzMsghdEbVDkT0NQmr2YOdonyBN0UaIMBomTb",
    "d1VbIjhLdVmo7bLzEVbxkYp2XoRLNHWHEIVC1NtLSrqhHNdCuvKuAND4InWBjGkGpZqLcZpzYKu",
    "xAC79C7fDVYwJD6D1aUIWCFdwxwgqTvPsLD8iKRWGxnVMXkbnc789DysLQyoHqFQVAUUYRowZkl",
    "4y7UFTDvuEYhWmcS8InCaIYtQnu5Z0RhJNcCYVBSRQxcco6XASPeRS9wJuxo3TRXMcbkIRwkNIX",
    "AFhnHYwm2D2IKYaFfaNnDfDZkZN0BTxeICVi3uOixDTZdQ4xh2xUDIkWgpGflZjU/DC4DZRpRgo",
    "r+J9UuNW7h6ERWun93yDvvZ2Z/oZxO5/PPmjJ6VAj3ezDO/7ula9bo1fFuZ9N8ZuXO5dVZzcg63",
    "hvflD3e/fLqjT/AzTzvvvr7sLLP+x9rR+v/5bHxlUminG2c4l68O+2TGrtUC/Zdm9K9lY916y31",
    "58EDdhcGb16+6t0cP7jzeaKT8LLC3lrN38v2aq/iYZUl+KtoOsoOJGxObtmun//6MvScXxON09j",
    "29vj1aTv7iep7H1QXya/g2zuXtr69/bX05+XT9Vwvsv57j+3rS1/eIqxawR+/tac4C3Ol7VAqjJ",
    "Oeg3Ks2OT9/Nd+PKucds9zfYud/ud/P+8/0r+OW845Zbln/jtrH/zR8VT/wcvi1P9iPZc9Xvxv/",
    "2TvaOehqNn44/atTDxkPCk692TQeYXtFs/9Ann4H9QuzUw==",
    ""
  };

  static char newstr [1249] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c7_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c7_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3862277146U));
  ssSetChecksum1(S,(91435963U));
  ssSetChecksum2(S,(1279986540U));
  ssSetChecksum3(S,(1280830342U));
}

static void mdlRTW_c7_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c7_sliding_controller_const(SimStruct *S)
{
  SFc7_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc7_sliding_controller_constInstanceStruct *)utMalloc(sizeof
    (SFc7_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc7_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc7_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c7_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c7_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c7_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c7_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c7_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c7_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c7_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c7_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c7_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c7_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c7_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c7_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c7_JITStateAnimation,
    chartInstance->c7_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c7_sliding_controller_const(chartInstance);
}

void c7_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c7_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c7_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c7_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c7_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
