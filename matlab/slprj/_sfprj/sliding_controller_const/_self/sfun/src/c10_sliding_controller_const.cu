/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c10_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void c10_do_animation_call_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c10_st);
static void sf_gateway_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);
static void c10_eML_blk_kernel(SFc10_sliding_controller_constInstanceStruct
  *chartInstance, real_T c10_b_q[2], real_T c10_b_qd[2], real_T c10_b_knee[2],
  real_T c10_b_dknee[2], real_T c10_b_ankle[2], real_T c10_b_danke[2]);
static void c10_emlrt_marshallIn(SFc10_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c10_b_ankle, const char_T *c10_identifier,
  real_T c10_y[2]);
static void c10_b_emlrt_marshallIn(SFc10_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId,
  real_T c10_y[2]);
static uint8_T c10_c_emlrt_marshallIn
  (SFc10_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c10_b_is_active_c10_sliding_controller_const, const char_T *c10_identifier);
static uint8_T c10_d_emlrt_marshallIn
  (SFc10_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c10_u, const emlrtMsgIdentifier *c10_parentId);
static void init_dsm_address_info(SFc10_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address
  (SFc10_sliding_controller_constInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c10_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c10_is_active_c10_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c10_do_animation_call_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c10_b_y = NULL;
  const mxArray *c10_c_y = NULL;
  const mxArray *c10_d_y = NULL;
  const mxArray *c10_e_y = NULL;
  const mxArray *c10_f_y = NULL;
  const mxArray *c10_st;
  const mxArray *c10_y = NULL;
  c10_st = NULL;
  c10_y = NULL;
  sf_mex_assign(&c10_y, sf_mex_createcellmatrix(5, 1), false);
  c10_b_y = NULL;
  sf_mex_assign(&c10_b_y, sf_mex_create("y", *chartInstance->c10_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 0, c10_b_y);
  c10_c_y = NULL;
  sf_mex_assign(&c10_c_y, sf_mex_create("y", *chartInstance->c10_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 1, c10_c_y);
  c10_d_y = NULL;
  sf_mex_assign(&c10_d_y, sf_mex_create("y", *chartInstance->c10_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 2, c10_d_y);
  c10_e_y = NULL;
  sf_mex_assign(&c10_e_y, sf_mex_create("y", *chartInstance->c10_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c10_y, 3, c10_e_y);
  c10_f_y = NULL;
  sf_mex_assign(&c10_f_y, sf_mex_create("y",
    &chartInstance->c10_is_active_c10_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c10_y, 4, c10_f_y);
  sf_mex_assign(&c10_st, c10_y, false);
  return c10_st;
}

static void set_sim_state_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c10_st)
{
  const mxArray *c10_u;
  c10_u = sf_mex_dup(c10_st);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 0)),
                       "ankle", *chartInstance->c10_ankle);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 1)),
                       "danke", *chartInstance->c10_danke);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 2)),
                       "dknee", *chartInstance->c10_dknee);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 3)),
                       "knee", *chartInstance->c10_knee);
  chartInstance->c10_is_active_c10_sliding_controller_const =
    c10_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 4)),
    "is_active_c10_sliding_controller_const");
  sf_mex_destroy(&c10_u);
  sf_mex_destroy(&c10_st);
}

static void sf_gateway_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c10_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c10_eML_blk_kernel(chartInstance, *chartInstance->c10_q,
                     *chartInstance->c10_qd, *chartInstance->c10_knee,
                     *chartInstance->c10_dknee, *chartInstance->c10_ankle,
                     *chartInstance->c10_danke);
  c10_do_animation_call_c10_sliding_controller_const(chartInstance);
}

static void mdl_start_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c10_errCode;
  c10_errCode = hipGetLastError();
  if (c10_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c10_errCode, (char_T *)hipGetErrorName
                       (c10_errCode), (char_T *)hipGetErrorString(c10_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c10_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc10_sliding_controller_const
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c10_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c10_nameCaptureInfo = NULL;
  const char_T *c10_data[4] = {
    "789cc553cb4ac34014bd955adc545d09825b372e1cdd886b8b850a2db50f5c88c69b64d20c9d874c52dbbaf113fc0137fe853fe53f98a44dd30443c4623d10ee"
    "1c4ee69e7b0f0c941acd12006c07dffb0bc079052254670576e77503d2c8eaa59c1a6313caa97ba1fe16f8bdceb9a5a44f27fe8c48147471d3568249947e6ffa",
    "4841534ff1276a478ac338ed3141bbcba41532515f92162494c273cda5d6b03b12a05d2f99902f93288f100f39fb960bf2c8229b47f6bfd86ff24bbfb8ff5e81"
    "5fac3b4a8f51dbc6904926d0679667703a587defd8b7923bc74cb1d5c8e434f1fb58d1af97eb97d66f2fef88ab0425127d1725a37ca0b8eda03689857e908631",
    "f688a72d82c2748c31f221d5240888a349becfec58847d8b72dbf9e11e79efa70a5b51fd6cefe33afdee8f0e0ed7e917e3bffcd6f5fe6eda177872f67cc53a1d"
    "5d6f5ef7bb8dfe69ab96ccd12ef0299a0372f85ff7ff0232e066a9", "" };

  c10_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c10_data[0], 1616U, &c10_nameCaptureInfo);
  return c10_nameCaptureInfo;
}

static void c10_eML_blk_kernel(SFc10_sliding_controller_constInstanceStruct
  *chartInstance, real_T c10_b_q[2], real_T c10_b_qd[2], real_T c10_b_knee[2],
  real_T c10_b_dknee[2], real_T c10_b_ankle[2], real_T c10_b_danke[2])
{
  real_T c10_a;
  real_T c10_b_a;
  c10_b_knee[0] = -0.324 * muDoubleScalarCos(c10_b_q[0]);
  c10_b_knee[1] = -0.324 * muDoubleScalarSin(c10_b_q[0]);
  c10_a = -0.324 * c10_b_qd[0];
  c10_b_dknee[0] = c10_a * -muDoubleScalarSin(c10_b_q[0]);
  c10_b_dknee[1] = c10_a * muDoubleScalarCos(c10_b_q[0]);
  c10_b_a = -0.39 * (c10_b_qd[0] + c10_b_qd[1]);
  c10_b_ankle[0] = c10_b_knee[0] - -0.39 * muDoubleScalarSin(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_ankle[1] = c10_b_knee[1] - -0.39 * muDoubleScalarCos(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_danke[0] = c10_b_dknee[0] + c10_b_a * -muDoubleScalarSin(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_danke[1] = c10_b_dknee[1] + c10_b_a * muDoubleScalarCos(c10_b_q[0] +
    c10_b_q[1]);
}

static void c10_emlrt_marshallIn(SFc10_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c10_b_ankle, const char_T *c10_identifier,
  real_T c10_y[2])
{
  emlrtMsgIdentifier c10_thisId;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c10_b_ankle), &c10_thisId,
    c10_y);
  sf_mex_destroy(&c10_b_ankle);
}

static void c10_b_emlrt_marshallIn(SFc10_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId,
  real_T c10_y[2])
{
  real_T c10_dv[2];
  int32_T c10_i;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), c10_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c10_i = 0; c10_i < 2; c10_i++) {
    c10_y[c10_i] = c10_dv[c10_i];
  }

  sf_mex_destroy(&c10_u);
}

static uint8_T c10_c_emlrt_marshallIn
  (SFc10_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c10_b_is_active_c10_sliding_controller_const, const char_T *c10_identifier)
{
  emlrtMsgIdentifier c10_thisId;
  uint8_T c10_y;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_y = c10_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c10_b_is_active_c10_sliding_controller_const), &c10_thisId);
  sf_mex_destroy(&c10_b_is_active_c10_sliding_controller_const);
  return c10_y;
}

static uint8_T c10_d_emlrt_marshallIn
  (SFc10_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c10_u, const emlrtMsgIdentifier *c10_parentId)
{
  uint8_T c10_b_u;
  uint8_T c10_y;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), &c10_b_u, 1, 3, 0U, 0, 0U, 0);
  c10_y = c10_b_u;
  sf_mex_destroy(&c10_u);
  return c10_y;
}

static void init_dsm_address_info(SFc10_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address
  (SFc10_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c10_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c10_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c10_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c10_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c10_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c10_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c10_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c10_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c10_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c10_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c10_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c10_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6HFA4ifWRyfmFySWZYan"
    "2xoEF+ck5mSmZcen5yfV1KUn5OTWgRiFpcg4gkAp34vDQ=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c10_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c10_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c10_sliding_controller_const(void *chartInstanceVar)
{
  enable_c10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c10_sliding_controller_const(void
  *chartInstanceVar)
{
  disable_c10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c10_sliding_controller_const(void
  *chartInstanceVar)
{
  sf_gateway_c10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c10_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c10_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c10_sliding_controller_const
  (void *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc10_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c10_sliding_controller_const
      ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc10_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c10_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c10_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc10_sliding_controller_const
    ((SFc10_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c10_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c10_sliding_controller_const
      ((SFc10_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
    initSimStructsc10_sliding_controller_const
      ((SFc10_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
  }
}

const char* sf_c10_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFAL0EbFEXaSwEXRYH20timJESBHTulnAY91FgvR+JCy112f2ir79C",
    "ce+m9D9BLn6BAniCnPEPQJ+gsScsyRVKR3SROURq0tOQ33843Mzu78lqDXQ+vm3j/esvzruHndb",
    "zbXn6tFePW3J0/73ifFuNf0EjYeJ8oEmuv8RIkhu9AS24Nk2IgRrISxsQIFAiK2EQqU8emWWw5E",
    "5O+FdTx6e8jRqMgkpaH22hLwj3Bp8iWWLOPPF2mgJo+QGgiJe046nMynnmszLEfAZ1oGzdJ0GAC",
    "mzi39K7lhiUceidAB0Ibgh7rM98CQwz45qRWplOqg1OgjBPOiKhUGxEdQIIBNnCQhPh/zxoUVYb",
    "RiCizDRFJQe+wScYpBZQ5mcYXR0wQIxUjvBdz3xku+rbP0Z9dGQJvCAj6tq2ATBLJhKnPf9BHpT",
    "1Bjjh04ciO69kC+Mm65D9mcAyqNm4jX6agyBj2RO2kWUB6J1m2ZlWyCDMshsdEbVHMn4awtnqxc",
    "nRAME8wRIs6GGQiB3qoWIrhrWWz8cBV5rIlY+M82XoZLGPrpdCUhRlbnwqfcK5rYUOZ7EAKPGPt",
    "EkOaYTlrNU5rFg4lBtiVd/1qsIJh4guYL0XIKtOVlgBZ33mIjeU8klptZOxj8XZ3dhZfL8IGwoA",
    "aEQpVXUARpgFjloW3ni1k2uUegeiVydyrAucVsgzl6ZEV3WOpJhiThiZyJsFltBYY6zHmElfCgc",
    "ZF0wRzuVyGo4RGELoGwzjs4rJBbEVMtGttW7juUmamXdBUsaQiqxZXHbahniuoaQIHYiLksegrG",
    "QdFj8/DC4DVRpRgYryN/UtN+zh7VSrcfrbhne1n773CfnZqV/78bI6nVcHjzX2W573RPo9vl+Zt",
    "41+rldttztm9X5qnU7JzuHW8H7z47ZOHH/7x91/PPn86/f3PH5fN31qYv5V9d3bX11bb/28W49u",
    "njXa2cNKFenXY+3N+dSr4P5jjXy/G+lH0lUh/uB9tPSLDeLDx5c/fbKRHfsb3pNPs71rJ39Pnd1",
    "zHxyrL6lPRQVgcTNyY2Hy7dvxfz/l7bUk8bhTP8+vlvcvZ394s23eW6FsvxkRMOJTq9+3p+PjCO",
    "kIUcnV03Lm4jgkeGq6Mjlub5f5QpaNzTkfHcxL+Hf8X47iafT7/kyX+3y3l4W52XjwkbheCQ/rF",
    "xqHmLMQt5JBKYZTkHJT7qs35PF20j65q571hu3fFz//tXn/eX+Vc1L6gXeeS57A3ZXdZfaueD68",
    "avmk/8Er49Sus47Ln9teNf+6tdr7+qBh/O/sJ7UeMhxW/porX+NNoVPX2P1Cn/wAtacYN",
    ""
  };

  static char newstr [1345] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c10_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c10_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(614325834U));
  ssSetChecksum1(S,(4071890510U));
  ssSetChecksum2(S,(2385365703U));
  ssSetChecksum3(S,(1589225593U));
}

static void mdlRTW_c10_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c10_sliding_controller_const(SimStruct *S)
{
  SFc10_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc10_sliding_controller_constInstanceStruct *)utMalloc
    (sizeof(SFc10_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc10_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc10_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c10_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c10_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c10_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c10_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c10_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c10_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c10_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c10_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c10_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c10_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c10_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c10_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c10_JITStateAnimation,
    chartInstance->c10_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c10_sliding_controller_const(chartInstance);
}

void c10_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c10_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c10_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c10_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c10_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
