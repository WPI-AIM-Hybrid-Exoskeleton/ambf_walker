#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c15_sliding_controller_const.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void c15_do_animation_call_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_st);
static void sf_gateway_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);
static void c15_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c15_b_msg, const char_T *c15_identifier,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_y);
static void c15_b_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_y);
static void c15_c_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  c15_SL_Bus_sliding_controller_const_std_msgs_Header *c15_y);
static uint32_T c15_d_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId);
static c15_SL_Bus_sliding_controller_const_ros_time_Time c15_e_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId);
static real_T c15_f_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId);
static void c15_g_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  uint8_T c15_y[128]);
static c15_SL_Bus_ROSVariableLengthArrayInfo c15_h_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId);
static void c15_i_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  c15_SL_Bus_sliding_controller_const_std_msgs_String c15_y[16]);
static void c15_j_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  real_T c15_y[7]);
static uint8_T c15_k_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_b_is_active_c15_sliding_controller_const, const char_T *c15_identifier);
static uint8_T c15_l_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId);
static void c15_eML_blk_kernel(SFc15_sliding_controller_constInstanceStruct
  *chartInstance, c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c15_b_blankMsg);
static __global__ void c15_sf_gateway_c15_sliding_controller_const_kernel1
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg,
   c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg);
static __global__ void c15_sf_gateway_c15_sliding_controller_const_kernel2(const
  int32_T c15_i1, c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c15_b_blankMsg, c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c15_b_msg);
static __global__ void c15_sf_gateway_c15_sliding_controller_const_kernel3
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg,
   c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg);
static __global__ void c15_sf_gateway_c15_sliding_controller_const_kernel4
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg,
   c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r);
static __global__ void c15_sf_gateway_c15_sliding_controller_const_kernel5(const
  int32_T c15_i5, c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c15_b_msg, c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r);
static __global__ void c15_sf_gateway_c15_sliding_controller_const_kernel6
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg,
   c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r);
static __global__ void c15_sf_gateway_c15_sliding_controller_const_kernel7(const
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg);
static __global__ void c15_sf_gateway_c15_sliding_controller_const_kernel8(const
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r, const
  int32_T c15_i9, c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c15_b_msg);
static __global__ void c15_sf_gateway_c15_sliding_controller_const_kernel9(const
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg);
static __global__ void c15_eML_blk_kernel_kernel10
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel11(const uint8_T c15_uv[7],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel12
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel13(const uint8_T c15_uv1[8],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel14
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel15(const uint8_T c15_uv2[9],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel16
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel17(const uint8_T c15_uv3[8],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel18
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel19(const uint8_T c15_uv4[9],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel20
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel21(const uint8_T c15_uv5[10],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel22
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel23(const uint8_T c15_uv6[6],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static __global__ void c15_eML_blk_kernel_kernel24
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg);
static void init_dsm_address_info(SFc15_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address
  (SFc15_sliding_controller_constInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c15_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c15_is_active_c15_sliding_controller_const = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c15_gpu_uv6, 6UL);
  hipMalloc(&chartInstance->c15_gpu_uv5, 10UL);
  hipMalloc(&chartInstance->c15_gpu_uv4, 9UL);
  hipMalloc(&chartInstance->c15_gpu_uv3, 8UL);
  hipMalloc(&chartInstance->c15_gpu_uv2, 9UL);
  hipMalloc(&chartInstance->c15_gpu_uv1, 8UL);
  hipMalloc(&chartInstance->c15_gpu_uv, 7UL);
  hipMalloc(&chartInstance->c15_gpu_blankMsg, 2536UL);
}

static void initialize_params_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c15_do_animation_call_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
  static const char_T *c15_sv[9] = { "Header", "Name", "Name_SL_Info",
    "Position", "Position_SL_Info", "Velocity", "Velocity_SL_Info", "Effort",
    "Effort_SL_Info" };

  static const char_T *c15_sv1[4] = { "Seq", "Stamp", "FrameId",
    "FrameId_SL_Info" };

  static const char_T *c15_sv2[2] = { "Sec", "Nsec" };

  static const char_T *c15_sv3[2] = { "CurrentLength", "ReceivedLength" };

  static const char_T *c15_sv4[2] = { "Data", "Data_SL_Info" };

  c15_SL_Bus_sliding_controller_const_std_msgs_String c15_g_u[16];
  const c15_SL_Bus_sliding_controller_const_std_msgs_String *c15_r;
  const mxArray *c15_ab_y = NULL;
  const mxArray *c15_b_y = NULL;
  const mxArray *c15_bb_y = NULL;
  const mxArray *c15_c_y = NULL;
  const mxArray *c15_cb_y = NULL;
  const mxArray *c15_d_y = NULL;
  const mxArray *c15_db_y = NULL;
  const mxArray *c15_e_y = NULL;
  const mxArray *c15_eb_y = NULL;
  const mxArray *c15_f_y = NULL;
  const mxArray *c15_fb_y = NULL;
  const mxArray *c15_g_y = NULL;
  const mxArray *c15_gb_y = NULL;
  const mxArray *c15_h_y = NULL;
  const mxArray *c15_i_y = NULL;
  const mxArray *c15_j_y = NULL;
  const mxArray *c15_k_y = NULL;
  const mxArray *c15_l_y = NULL;
  const mxArray *c15_m_y = NULL;
  const mxArray *c15_n_y = NULL;
  const mxArray *c15_o_y = NULL;
  const mxArray *c15_p_y = NULL;
  const mxArray *c15_q_y = NULL;
  const mxArray *c15_r_y = NULL;
  const mxArray *c15_s_y = NULL;
  const mxArray *c15_st;
  const mxArray *c15_t_y = NULL;
  const mxArray *c15_u_y = NULL;
  const mxArray *c15_v_y = NULL;
  const mxArray *c15_w_y = NULL;
  const mxArray *c15_x_y = NULL;
  const mxArray *c15_y = NULL;
  const mxArray *c15_y_y = NULL;
  real_T c15_l_u[7];
  real_T c15_b_u;
  real_T c15_c_u;
  real_T c15_u_Stamp_Nsec;
  real_T c15_u_Stamp_Sec;
  int32_T c15_iv[1];
  int32_T c15_b_i;
  int32_T c15_i;
  int32_T c15_i1;
  int32_T c15_i2;
  int32_T c15_i3;
  int32_T c15_i4;
  int32_T c15_i5;
  int32_T c15_i6;
  int32_T c15_i7;
  uint32_T c15_e_u;
  uint32_T c15_f_u;
  uint32_T c15_h_u;
  uint32_T c15_i_u;
  uint32_T c15_j_u;
  uint32_T c15_k_u;
  uint32_T c15_m_u;
  uint32_T c15_n_u;
  uint32_T c15_o_u;
  uint32_T c15_p_u;
  uint32_T c15_q_u;
  uint32_T c15_r_u;
  uint32_T c15_u;
  uint32_T c15_u_CurrentLength;
  uint32_T c15_u_FrameId_SL_Info_CurrentLength;
  uint32_T c15_u_FrameId_SL_Info_ReceivedLength;
  uint32_T c15_u_ReceivedLength;
  uint32_T c15_u_Seq;
  uint8_T c15_d_u[128];
  uint8_T c15_u_FrameId[128];
  c15_st = NULL;
  c15_y = NULL;
  sf_mex_assign(&c15_y, sf_mex_createcellmatrix(2, 1), false);
  c15_b_y = NULL;
  sf_mex_assign(&c15_b_y, sf_mex_createstruct("structure", 9, c15_sv, 2, 1, 1),
                false);
  c15_u_Seq = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[0];
  c15_u_Stamp_Sec = *(real_T *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[8])[0];
  c15_u_Stamp_Nsec = *(real_T *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[8])[8];
  for (c15_i = 0; c15_i < 128; c15_i++) {
    c15_u_FrameId[c15_i] = ((uint8_T *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
      chartInstance->c15_msg)[0])[24])[c15_i];
  }

  c15_u_FrameId_SL_Info_CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[0];
  c15_u_FrameId_SL_Info_ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[4];
  c15_c_y = NULL;
  sf_mex_assign(&c15_c_y, sf_mex_createstruct("structure", 4, c15_sv1, 2, 1, 1),
                false);
  c15_u = c15_u_Seq;
  c15_d_y = NULL;
  sf_mex_assign(&c15_d_y, sf_mex_create("y", &c15_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_c_y, 0, "Seq", c15_d_y, 0);
  c15_e_y = NULL;
  sf_mex_assign(&c15_e_y, sf_mex_createstruct("structure", 2, c15_sv2, 2, 1, 1),
                false);
  c15_b_u = c15_u_Stamp_Sec;
  c15_f_y = NULL;
  sf_mex_assign(&c15_f_y, sf_mex_create("y", &c15_b_u, 0, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_e_y, 0, "Sec", c15_f_y, 0);
  c15_c_u = c15_u_Stamp_Nsec;
  c15_g_y = NULL;
  sf_mex_assign(&c15_g_y, sf_mex_create("y", &c15_c_u, 0, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_e_y, 0, "Nsec", c15_g_y, 1);
  sf_mex_setfieldbynum(c15_c_y, 0, "Stamp", c15_e_y, 1);
  for (c15_i1 = 0; c15_i1 < 128; c15_i1++) {
    c15_d_u[c15_i1] = c15_u_FrameId[c15_i1];
  }

  c15_h_y = NULL;
  sf_mex_assign(&c15_h_y, sf_mex_create("y", c15_d_u, 3, 0U, 1U, 0U, 1, 128),
                false);
  sf_mex_setfieldbynum(c15_c_y, 0, "FrameId", c15_h_y, 2);
  c15_i_y = NULL;
  sf_mex_assign(&c15_i_y, sf_mex_createstruct("structure", 2, c15_sv3, 2, 1, 1),
                false);
  c15_e_u = c15_u_FrameId_SL_Info_CurrentLength;
  c15_j_y = NULL;
  sf_mex_assign(&c15_j_y, sf_mex_create("y", &c15_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_i_y, 0, "CurrentLength", c15_j_y, 0);
  c15_f_u = c15_u_FrameId_SL_Info_ReceivedLength;
  c15_k_y = NULL;
  sf_mex_assign(&c15_k_y, sf_mex_create("y", &c15_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_i_y, 0, "ReceivedLength", c15_k_y, 1);
  sf_mex_setfieldbynum(c15_c_y, 0, "FrameId_SL_Info", c15_i_y, 3);
  sf_mex_setfieldbynum(c15_b_y, 0, "Header", c15_c_y, 0);
  for (c15_i2 = 0; c15_i2 < 16; c15_i2++) {
    for (c15_i3 = 0; c15_i3 < 128; c15_i3++) {
      c15_g_u[c15_i2].Data[c15_i3] = ((uint8_T *)&((char_T *)
        (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
        (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
        chartInstance->c15_msg)[160])[136 * (int32_T)c15_i2])[0])[c15_i3];
    }

    c15_g_u[c15_i2].Data_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
      (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i2])[128])[0];
    c15_g_u[c15_i2].Data_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
      (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i2])[128])[4];
  }

  c15_l_y = NULL;
  c15_iv[0] = 16;
  sf_mex_assign(&c15_l_y, sf_mex_createstructarray("structure", 1, c15_iv, 2,
    c15_sv4), false);
  for (c15_b_i = 0; c15_b_i < 16; c15_b_i++) {
    c15_r = &c15_g_u[c15_b_i];
    for (c15_i4 = 0; c15_i4 < 128; c15_i4++) {
      c15_d_u[c15_i4] = c15_r->Data[c15_i4];
    }

    c15_n_y = NULL;
    sf_mex_assign(&c15_n_y, sf_mex_create("y", c15_d_u, 3, 0U, 1U, 0U, 1, 128),
                  false);
    sf_mex_setfieldbynum(c15_l_y, c15_b_i, "Data", c15_n_y, 0);
    c15_u_CurrentLength = c15_r->Data_SL_Info.CurrentLength;
    c15_u_ReceivedLength = c15_r->Data_SL_Info.ReceivedLength;
    c15_p_y = NULL;
    sf_mex_assign(&c15_p_y, sf_mex_createstruct("structure", 2, c15_sv3, 2, 1, 1),
                  false);
    c15_j_u = c15_u_CurrentLength;
    c15_r_y = NULL;
    sf_mex_assign(&c15_r_y, sf_mex_create("y", &c15_j_u, 7, 0U, 0U, 0U, 0),
                  false);
    sf_mex_setfieldbynum(c15_p_y, 0, "CurrentLength", c15_r_y, 0);
    c15_k_u = c15_u_ReceivedLength;
    c15_s_y = NULL;
    sf_mex_assign(&c15_s_y, sf_mex_create("y", &c15_k_u, 7, 0U, 0U, 0U, 0),
                  false);
    sf_mex_setfieldbynum(c15_p_y, 0, "ReceivedLength", c15_s_y, 1);
    sf_mex_setfieldbynum(c15_l_y, c15_b_i, "Data_SL_Info", c15_p_y, 1);
  }

  sf_mex_setfieldbynum(c15_b_y, 0, "Name", c15_l_y, 1);
  c15_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2336])[0];
  c15_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2336])[4];
  c15_m_y = NULL;
  sf_mex_assign(&c15_m_y, sf_mex_createstruct("structure", 2, c15_sv3, 2, 1, 1),
                false);
  c15_h_u = c15_u_CurrentLength;
  c15_o_y = NULL;
  sf_mex_assign(&c15_o_y, sf_mex_create("y", &c15_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_m_y, 0, "CurrentLength", c15_o_y, 0);
  c15_i_u = c15_u_ReceivedLength;
  c15_q_y = NULL;
  sf_mex_assign(&c15_q_y, sf_mex_create("y", &c15_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_m_y, 0, "ReceivedLength", c15_q_y, 1);
  sf_mex_setfieldbynum(c15_b_y, 0, "Name_SL_Info", c15_m_y, 2);
  for (c15_i5 = 0; c15_i5 < 7; c15_i5++) {
    c15_l_u[c15_i5] = ((real_T *)&((char_T *)chartInstance->c15_msg)[2344])
      [c15_i5];
  }

  c15_t_y = NULL;
  sf_mex_assign(&c15_t_y, sf_mex_create("y", c15_l_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c15_b_y, 0, "Position", c15_t_y, 3);
  c15_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2400])[0];
  c15_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2400])[4];
  c15_u_y = NULL;
  sf_mex_assign(&c15_u_y, sf_mex_createstruct("structure", 2, c15_sv3, 2, 1, 1),
                false);
  c15_m_u = c15_u_CurrentLength;
  c15_v_y = NULL;
  sf_mex_assign(&c15_v_y, sf_mex_create("y", &c15_m_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_u_y, 0, "CurrentLength", c15_v_y, 0);
  c15_n_u = c15_u_ReceivedLength;
  c15_w_y = NULL;
  sf_mex_assign(&c15_w_y, sf_mex_create("y", &c15_n_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_u_y, 0, "ReceivedLength", c15_w_y, 1);
  sf_mex_setfieldbynum(c15_b_y, 0, "Position_SL_Info", c15_u_y, 4);
  for (c15_i6 = 0; c15_i6 < 7; c15_i6++) {
    c15_l_u[c15_i6] = ((real_T *)&((char_T *)chartInstance->c15_msg)[2408])
      [c15_i6];
  }

  c15_x_y = NULL;
  sf_mex_assign(&c15_x_y, sf_mex_create("y", c15_l_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c15_b_y, 0, "Velocity", c15_x_y, 5);
  c15_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2464])[0];
  c15_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2464])[4];
  c15_y_y = NULL;
  sf_mex_assign(&c15_y_y, sf_mex_createstruct("structure", 2, c15_sv3, 2, 1, 1),
                false);
  c15_o_u = c15_u_CurrentLength;
  c15_ab_y = NULL;
  sf_mex_assign(&c15_ab_y, sf_mex_create("y", &c15_o_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_y_y, 0, "CurrentLength", c15_ab_y, 0);
  c15_p_u = c15_u_ReceivedLength;
  c15_bb_y = NULL;
  sf_mex_assign(&c15_bb_y, sf_mex_create("y", &c15_p_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_y_y, 0, "ReceivedLength", c15_bb_y, 1);
  sf_mex_setfieldbynum(c15_b_y, 0, "Velocity_SL_Info", c15_y_y, 6);
  for (c15_i7 = 0; c15_i7 < 7; c15_i7++) {
    c15_l_u[c15_i7] = ((real_T *)&((char_T *)chartInstance->c15_msg)[2472])
      [c15_i7];
  }

  c15_cb_y = NULL;
  sf_mex_assign(&c15_cb_y, sf_mex_create("y", c15_l_u, 0, 0U, 1U, 0U, 1, 7),
                false);
  sf_mex_setfieldbynum(c15_b_y, 0, "Effort", c15_cb_y, 7);
  c15_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2528])[0];
  c15_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2528])[4];
  c15_db_y = NULL;
  sf_mex_assign(&c15_db_y, sf_mex_createstruct("structure", 2, c15_sv3, 2, 1, 1),
                false);
  c15_q_u = c15_u_CurrentLength;
  c15_eb_y = NULL;
  sf_mex_assign(&c15_eb_y, sf_mex_create("y", &c15_q_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_db_y, 0, "CurrentLength", c15_eb_y, 0);
  c15_r_u = c15_u_ReceivedLength;
  c15_fb_y = NULL;
  sf_mex_assign(&c15_fb_y, sf_mex_create("y", &c15_r_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c15_db_y, 0, "ReceivedLength", c15_fb_y, 1);
  sf_mex_setfieldbynum(c15_b_y, 0, "Effort_SL_Info", c15_db_y, 8);
  sf_mex_setcell(c15_y, 0, c15_b_y);
  c15_gb_y = NULL;
  sf_mex_assign(&c15_gb_y, sf_mex_create("y",
    &chartInstance->c15_is_active_c15_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c15_y, 1, c15_gb_y);
  sf_mex_assign(&c15_st, c15_y, false);
  return c15_st;
}

static void set_sim_state_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_st)
{
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState c15_r;
  const mxArray *c15_u;
  int32_T c15_i;
  int32_T c15_i1;
  int32_T c15_i2;
  int32_T c15_i3;
  c15_u = sf_mex_dup(c15_st);
  c15_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c15_u, 0)),
                       "msg", &c15_r);
  *(uint32_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c15_msg)[0])[0] = c15_r.Header.Seq;
  *(real_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c15_msg)[0])[8])[0] =
    c15_r.Header.Stamp.Sec;
  *(real_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c15_msg)[0])[8])[8] =
    c15_r.Header.Stamp.Nsec;
  for (c15_i = 0; c15_i < 128; c15_i++) {
    ((uint8_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header
       *)&((char_T *)chartInstance->c15_msg)[0])[24])[c15_i] =
      c15_r.Header.FrameId[c15_i];
  }

  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[0] =
    c15_r.Header.FrameId_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[4] =
    c15_r.Header.FrameId_SL_Info.ReceivedLength;
  for (c15_i1 = 0; c15_i1 < 16; c15_i1++) {
    for (c15_i2 = 0; c15_i2 < 128; c15_i2++) {
      ((uint8_T *)&((char_T *)
                    (c15_SL_Bus_sliding_controller_const_std_msgs_String *)
                    &((char_T *)
                      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)
                      &((char_T *)chartInstance->c15_msg)[160])[136 * (int32_T)
                    c15_i1])[0])[c15_i2] = c15_r.Name[c15_i1].Data[c15_i2];
    }

    *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i1])[128])[0] =
      c15_r.Name[c15_i1].Data_SL_Info.CurrentLength;
    *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i1])[128])[4] =
      c15_r.Name[c15_i1].Data_SL_Info.ReceivedLength;
  }

  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2336])[0] = c15_r.Name_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2336])[4] = c15_r.Name_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2400])[0] = c15_r.Position_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2400])[4] = c15_r.Position_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2464])[0] = c15_r.Velocity_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2464])[4] = c15_r.Velocity_SL_Info.ReceivedLength;
  for (c15_i3 = 0; c15_i3 < 7; c15_i3++) {
    ((real_T *)&((char_T *)chartInstance->c15_msg)[2344])[c15_i3] =
      c15_r.Position[c15_i3];
    ((real_T *)&((char_T *)chartInstance->c15_msg)[2408])[c15_i3] =
      c15_r.Velocity[c15_i3];
    ((real_T *)&((char_T *)chartInstance->c15_msg)[2472])[c15_i3] =
      c15_r.Effort[c15_i3];
  }

  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2528])[0] = c15_r.Effort_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2528])[4] = c15_r.Effort_SL_Info.ReceivedLength;
  chartInstance->c15_is_active_c15_sliding_controller_const =
    c15_k_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c15_u, 1)),
    "is_active_c15_sliding_controller_const");
  sf_mex_destroy(&c15_u);
  sf_mex_destroy(&c15_st);
}

static void sf_gateway_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState c15_r;
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_gpu_blankMsg;
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_gpu_msg;
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_gpu_r;
  int32_T c15_i1;
  int32_T c15_i5;
  int32_T c15_i9;
  hipMalloc(&c15_gpu_r, 2536UL);
  hipMalloc(&c15_gpu_msg, 2536UL);
  hipMalloc(&c15_b_gpu_blankMsg, 2536UL);
  chartInstance->c15_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  *(uint32_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c15_msg)[0])[0] = *(uint32_T *)
    &((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T
        *)chartInstance->c15_blankMsg)[0])[0];
  *(real_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c15_msg)[0])[8])[0] = *(real_T *)
    &((char_T *)(c15_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
       (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
        chartInstance->c15_blankMsg)[0])[8])[0];
  *(real_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c15_msg)[0])[8])[8] = *(real_T *)
    &((char_T *)(c15_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
       (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
        chartInstance->c15_blankMsg)[0])[8])[8];
  hipMemcpy(c15_b_gpu_blankMsg, chartInstance->c15_blankMsg, 2536UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c15_gpu_msg, chartInstance->c15_msg, 2536UL, hipMemcpyHostToDevice);
  c15_sf_gateway_c15_sliding_controller_const_kernel1<<<dim3(1U, 1U, 1U), dim3
    (128U, 1U, 1U)>>>(c15_b_gpu_blankMsg, c15_gpu_msg);
  hipMemcpy(chartInstance->c15_msg, c15_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c15_blankMsg, c15_b_gpu_blankMsg, 2536UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[0] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_blankMsg)[0])[152])[0];
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[4] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_blankMsg)[0])[152])[4];
  for (c15_i1 = 0; c15_i1 < 16; c15_i1++) {
    hipMemcpy(c15_b_gpu_blankMsg, chartInstance->c15_blankMsg, 2536UL,
               hipMemcpyHostToDevice);
    hipMemcpy(c15_gpu_msg, chartInstance->c15_msg, 2536UL,
               hipMemcpyHostToDevice);
    c15_sf_gateway_c15_sliding_controller_const_kernel2<<<dim3(1U, 1U, 1U), dim3
      (128U, 1U, 1U)>>>(c15_i1, c15_b_gpu_blankMsg, c15_gpu_msg);
    hipMemcpy(chartInstance->c15_msg, c15_gpu_msg, 2536UL,
               hipMemcpyDeviceToHost);
    hipMemcpy(chartInstance->c15_blankMsg, c15_b_gpu_blankMsg, 2536UL,
               hipMemcpyDeviceToHost);
    *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i1])[128])[0] =
      *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)
                     &((char_T *)
                       (c15_SL_Bus_sliding_controller_const_std_msgs_String *)
                       &((char_T *)
                         (c15_SL_Bus_sliding_controller_const_std_msgs_String *)
                         &((char_T *)chartInstance->c15_blankMsg)[160])[136 *
                       (int32_T)c15_i1])[128])[0];
    *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i1])[128])[4] =
      *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)
                     &((char_T *)
                       (c15_SL_Bus_sliding_controller_const_std_msgs_String *)
                       &((char_T *)
                         (c15_SL_Bus_sliding_controller_const_std_msgs_String *)
                         &((char_T *)chartInstance->c15_blankMsg)[160])[136 *
                       (int32_T)c15_i1])[128])[4];
  }

  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2336])[0] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_blankMsg)[2336])[0];
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2336])[4] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_blankMsg)[2336])[4];
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2400])[0] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_blankMsg)[2400])[0];
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2400])[4] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_blankMsg)[2400])[4];
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2464])[0] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_blankMsg)[2464])[0];
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2464])[4] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_blankMsg)[2464])[4];
  hipMemcpy(c15_b_gpu_blankMsg, chartInstance->c15_blankMsg, 2536UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c15_gpu_msg, chartInstance->c15_msg, 2536UL, hipMemcpyHostToDevice);
  c15_sf_gateway_c15_sliding_controller_const_kernel3<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c15_b_gpu_blankMsg, c15_gpu_msg);
  hipMemcpy(chartInstance->c15_msg, c15_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c15_blankMsg, c15_b_gpu_blankMsg, 2536UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2528])[0] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_blankMsg)[2528])[0];
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2528])[4] = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_blankMsg)[2528])[4];
  c15_r.Header.Seq = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[0];
  c15_r.Header.Stamp.Sec = *(real_T *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[8])[0];
  c15_r.Header.Stamp.Nsec = *(real_T *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_ros_time_Time *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[8])[8];
  hipMemcpy(c15_gpu_r, &c15_r, 2536UL, hipMemcpyHostToDevice);
  hipMemcpy(c15_gpu_msg, chartInstance->c15_msg, 2536UL, hipMemcpyHostToDevice);
  c15_sf_gateway_c15_sliding_controller_const_kernel4<<<dim3(1U, 1U, 1U), dim3
    (128U, 1U, 1U)>>>(c15_gpu_msg, c15_gpu_r);
  hipMemcpy(chartInstance->c15_msg, c15_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c15_r, c15_gpu_r, 2536UL, hipMemcpyDeviceToHost);
  c15_r.Header.FrameId_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[0];
  c15_r.Header.FrameId_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[4];
  for (c15_i5 = 0; c15_i5 < 16; c15_i5++) {
    hipMemcpy(c15_gpu_r, &c15_r, 2536UL, hipMemcpyHostToDevice);
    hipMemcpy(c15_gpu_msg, chartInstance->c15_msg, 2536UL,
               hipMemcpyHostToDevice);
    c15_sf_gateway_c15_sliding_controller_const_kernel5<<<dim3(1U, 1U, 1U), dim3
      (128U, 1U, 1U)>>>(c15_i5, c15_gpu_msg, c15_gpu_r);
    hipMemcpy(chartInstance->c15_msg, c15_gpu_msg, 2536UL,
               hipMemcpyDeviceToHost);
    hipMemcpy(&c15_r, c15_gpu_r, 2536UL, hipMemcpyDeviceToHost);
    c15_r.Name[c15_i5].Data_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
      (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i5])[128])[0];
    c15_r.Name[c15_i5].Data_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
      (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i5])[128])[4];
  }

  c15_r.Name_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2336])[0];
  c15_r.Name_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2336])[4];
  c15_r.Position_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2400])[0];
  c15_r.Position_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2400])[4];
  c15_r.Velocity_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2464])[0];
  c15_r.Velocity_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2464])[4];
  hipMemcpy(c15_gpu_r, &c15_r, 2536UL, hipMemcpyHostToDevice);
  hipMemcpy(c15_gpu_msg, chartInstance->c15_msg, 2536UL, hipMemcpyHostToDevice);
  c15_sf_gateway_c15_sliding_controller_const_kernel6<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c15_gpu_msg, c15_gpu_r);
  hipMemcpy(chartInstance->c15_msg, c15_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c15_r, c15_gpu_r, 2536UL, hipMemcpyDeviceToHost);
  c15_r.Effort_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2528])[0];
  c15_r.Effort_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c15_msg)
    [2528])[4];
  c15_eML_blk_kernel(chartInstance, &c15_r);
  *(uint32_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c15_msg)[0])[0] = c15_r.Header.Seq;
  *(real_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c15_msg)[0])[8])[0] =
    c15_r.Header.Stamp.Sec;
  *(real_T *)&((char_T *)(c15_SL_Bus_sliding_controller_const_ros_time_Time *)
               &((char_T *)(c15_SL_Bus_sliding_controller_const_std_msgs_Header *)
                 &((char_T *)chartInstance->c15_msg)[0])[8])[8] =
    c15_r.Header.Stamp.Nsec;
  hipMemcpy(c15_gpu_r, &c15_r, 2536UL, hipMemcpyHostToDevice);
  hipMemcpy(c15_gpu_msg, chartInstance->c15_msg, 2536UL, hipMemcpyHostToDevice);
  c15_sf_gateway_c15_sliding_controller_const_kernel7<<<dim3(1U, 1U, 1U), dim3
    (128U, 1U, 1U)>>>(c15_gpu_r, c15_gpu_msg);
  hipMemcpy(chartInstance->c15_msg, c15_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[0] =
    c15_r.Header.FrameId_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c15_SL_Bus_sliding_controller_const_std_msgs_Header *)&((char_T *)
    chartInstance->c15_msg)[0])[152])[4] =
    c15_r.Header.FrameId_SL_Info.ReceivedLength;
  for (c15_i9 = 0; c15_i9 < 16; c15_i9++) {
    hipMemcpy(c15_gpu_msg, chartInstance->c15_msg, 2536UL,
               hipMemcpyHostToDevice);
    c15_sf_gateway_c15_sliding_controller_const_kernel8<<<dim3(1U, 1U, 1U), dim3
      (128U, 1U, 1U)>>>(c15_gpu_r, c15_i9, c15_gpu_msg);
    hipMemcpy(chartInstance->c15_msg, c15_gpu_msg, 2536UL,
               hipMemcpyDeviceToHost);
    *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i9])[128])[0] =
      c15_r.Name[c15_i9].Data_SL_Info.CurrentLength;
    *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      (c15_SL_Bus_sliding_controller_const_std_msgs_String *)&((char_T *)
      chartInstance->c15_msg)[160])[136 * (int32_T)c15_i9])[128])[4] =
      c15_r.Name[c15_i9].Data_SL_Info.ReceivedLength;
  }

  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2336])[0] = c15_r.Name_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2336])[4] = c15_r.Name_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2400])[0] = c15_r.Position_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2400])[4] = c15_r.Position_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2464])[0] = c15_r.Velocity_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2464])[4] = c15_r.Velocity_SL_Info.ReceivedLength;
  hipMemcpy(c15_gpu_msg, chartInstance->c15_msg, 2536UL, hipMemcpyHostToDevice);
  c15_sf_gateway_c15_sliding_controller_const_kernel9<<<dim3(1U, 1U, 1U), dim3
    (32U, 1U, 1U)>>>(c15_gpu_r, c15_gpu_msg);
  hipMemcpy(chartInstance->c15_msg, c15_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2528])[0] = c15_r.Effort_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c15_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c15_msg)[2528])[4] = c15_r.Effort_SL_Info.ReceivedLength;
  c15_do_animation_call_c15_sliding_controller_const(chartInstance);
  hipFree(c15_b_gpu_blankMsg);
  hipFree(c15_gpu_msg);
  hipFree(c15_gpu_r);
}

static void mdl_start_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c15_errCode;
  hipFree(*chartInstance->c15_gpu_uv6);
  hipFree(*chartInstance->c15_gpu_uv5);
  hipFree(*chartInstance->c15_gpu_uv4);
  hipFree(*chartInstance->c15_gpu_uv3);
  hipFree(*chartInstance->c15_gpu_uv2);
  hipFree(*chartInstance->c15_gpu_uv1);
  hipFree(*chartInstance->c15_gpu_uv);
  hipFree(chartInstance->c15_gpu_blankMsg);
  c15_errCode = hipGetLastError();
  if (c15_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c15_errCode, (char_T *)hipGetErrorName
                       (c15_errCode), (char_T *)hipGetErrorString(c15_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c15_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc15_sliding_controller_const
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c15_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c15_nameCaptureInfo = NULL;
  c15_nameCaptureInfo = NULL;
  sf_mex_assign(&c15_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c15_nameCaptureInfo;
}

static void c15_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c15_b_msg, const char_T *c15_identifier,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_y)
{
  emlrtMsgIdentifier c15_thisId;
  c15_thisId.fIdentifier = const_cast<const char_T *>(c15_identifier);
  c15_thisId.fParent = NULL;
  c15_thisId.bParentIsCell = false;
  c15_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c15_b_msg), &c15_thisId,
    c15_y);
  sf_mex_destroy(&c15_b_msg);
}

static void c15_b_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_y)
{
  static const char_T *c15_fieldNames[9] = { "Header", "Name", "Name_SL_Info",
    "Position", "Position_SL_Info", "Velocity", "Velocity_SL_Info", "Effort",
    "Effort_SL_Info" };

  emlrtMsgIdentifier c15_thisId;
  c15_thisId.fParent = c15_parentId;
  c15_thisId.bParentIsCell = false;
  sf_mex_check_struct(c15_parentId, c15_u, 9, c15_fieldNames, 0U, NULL);
  c15_thisId.fIdentifier = "Header";
  c15_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c15_u,
    "Header", "Header", 0)), &c15_thisId, &c15_y->Header);
  c15_thisId.fIdentifier = "Name";
  c15_i_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c15_u, "Name",
    "Name", 0)), &c15_thisId, c15_y->Name);
  c15_thisId.fIdentifier = "Name_SL_Info";
  c15_y->Name_SL_Info = c15_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c15_u, "Name_SL_Info", "Name_SL_Info", 0)), &c15_thisId);
  c15_thisId.fIdentifier = "Position";
  c15_j_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c15_u,
    "Position", "Position", 0)), &c15_thisId, c15_y->Position);
  c15_thisId.fIdentifier = "Position_SL_Info";
  c15_y->Position_SL_Info = c15_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c15_u, "Position_SL_Info", "Position_SL_Info", 0)),
    &c15_thisId);
  c15_thisId.fIdentifier = "Velocity";
  c15_j_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c15_u,
    "Velocity", "Velocity", 0)), &c15_thisId, c15_y->Velocity);
  c15_thisId.fIdentifier = "Velocity_SL_Info";
  c15_y->Velocity_SL_Info = c15_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c15_u, "Velocity_SL_Info", "Velocity_SL_Info", 0)),
    &c15_thisId);
  c15_thisId.fIdentifier = "Effort";
  c15_j_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c15_u,
    "Effort", "Effort", 0)), &c15_thisId, c15_y->Effort);
  c15_thisId.fIdentifier = "Effort_SL_Info";
  c15_y->Effort_SL_Info = c15_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c15_u, "Effort_SL_Info", "Effort_SL_Info", 0)), &c15_thisId);
  sf_mex_destroy(&c15_u);
}

static void c15_c_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  c15_SL_Bus_sliding_controller_const_std_msgs_Header *c15_y)
{
  static const char_T *c15_fieldNames[4] = { "Seq", "Stamp", "FrameId",
    "FrameId_SL_Info" };

  emlrtMsgIdentifier c15_thisId;
  c15_thisId.fParent = c15_parentId;
  c15_thisId.bParentIsCell = false;
  sf_mex_check_struct(c15_parentId, c15_u, 4, c15_fieldNames, 0U, NULL);
  c15_thisId.fIdentifier = "Seq";
  c15_y->Seq = c15_d_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c15_u, "Seq", "Seq", 0)), &c15_thisId);
  c15_thisId.fIdentifier = "Stamp";
  c15_y->Stamp = c15_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c15_u, "Stamp", "Stamp", 0)), &c15_thisId);
  c15_thisId.fIdentifier = "FrameId";
  c15_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c15_u,
    "FrameId", "FrameId", 0)), &c15_thisId, c15_y->FrameId);
  c15_thisId.fIdentifier = "FrameId_SL_Info";
  c15_y->FrameId_SL_Info = c15_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c15_u, "FrameId_SL_Info", "FrameId_SL_Info", 0)),
    &c15_thisId);
  sf_mex_destroy(&c15_u);
}

static uint32_T c15_d_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId)
{
  uint32_T c15_b_u;
  uint32_T c15_y;
  sf_mex_import(c15_parentId, sf_mex_dup(c15_u), &c15_b_u, 1, 7, 0U, 0, 0U, 0);
  c15_y = c15_b_u;
  sf_mex_destroy(&c15_u);
  return c15_y;
}

static c15_SL_Bus_sliding_controller_const_ros_time_Time c15_e_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId)
{
  static const char_T *c15_fieldNames[2] = { "Sec", "Nsec" };

  c15_SL_Bus_sliding_controller_const_ros_time_Time c15_y;
  emlrtMsgIdentifier c15_thisId;
  c15_thisId.fParent = c15_parentId;
  c15_thisId.bParentIsCell = false;
  sf_mex_check_struct(c15_parentId, c15_u, 2, c15_fieldNames, 0U, NULL);
  c15_thisId.fIdentifier = "Sec";
  c15_y.Sec = c15_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c15_u, "Sec", "Sec", 0)), &c15_thisId);
  c15_thisId.fIdentifier = "Nsec";
  c15_y.Nsec = c15_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c15_u, "Nsec", "Nsec", 0)), &c15_thisId);
  sf_mex_destroy(&c15_u);
  return c15_y;
}

static real_T c15_f_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId)
{
  real_T c15_d;
  real_T c15_y;
  sf_mex_import(c15_parentId, sf_mex_dup(c15_u), &c15_d, 1, 0, 0U, 0, 0U, 0);
  c15_y = c15_d;
  sf_mex_destroy(&c15_u);
  return c15_y;
}

static void c15_g_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  uint8_T c15_y[128])
{
  int32_T c15_i;
  uint8_T c15_uv[128];
  sf_mex_import(c15_parentId, sf_mex_dup(c15_u), c15_uv, 1, 3, 0U, 1, 0U, 1, 128);
  for (c15_i = 0; c15_i < 128; c15_i++) {
    c15_y[c15_i] = c15_uv[c15_i];
  }

  sf_mex_destroy(&c15_u);
}

static c15_SL_Bus_ROSVariableLengthArrayInfo c15_h_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId)
{
  static const char_T *c15_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c15_SL_Bus_ROSVariableLengthArrayInfo c15_y;
  emlrtMsgIdentifier c15_thisId;
  c15_thisId.fParent = c15_parentId;
  c15_thisId.bParentIsCell = false;
  sf_mex_check_struct(c15_parentId, c15_u, 2, c15_fieldNames, 0U, NULL);
  c15_thisId.fIdentifier = "CurrentLength";
  c15_y.CurrentLength = c15_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c15_u, "CurrentLength", "CurrentLength", 0)), &c15_thisId);
  c15_thisId.fIdentifier = "ReceivedLength";
  c15_y.ReceivedLength = c15_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c15_u, "ReceivedLength", "ReceivedLength", 0)), &c15_thisId);
  sf_mex_destroy(&c15_u);
  return c15_y;
}

static void c15_i_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  c15_SL_Bus_sliding_controller_const_std_msgs_String c15_y[16])
{
  static const char_T *c15_fieldNames[2] = { "Data", "Data_SL_Info" };

  emlrtMsgIdentifier c15_thisId;
  int32_T c15_i;
  uint32_T c15_uv[1];
  c15_uv[0] = 16U;
  c15_thisId.fParent = c15_parentId;
  c15_thisId.bParentIsCell = false;
  sf_mex_check_struct(c15_parentId, c15_u, 2, c15_fieldNames, 1U, c15_uv);
  for (c15_i = 0; c15_i < 16; c15_i++) {
    c15_thisId.fIdentifier = "Data";
    c15_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c15_u,
      "Data", "Data", c15_i)), &c15_thisId, c15_y[c15_i].Data);
    c15_thisId.fIdentifier = "Data_SL_Info";
    c15_y[c15_i].Data_SL_Info = c15_h_emlrt_marshallIn(chartInstance, sf_mex_dup
      (sf_mex_getfield(c15_u, "Data_SL_Info", "Data_SL_Info", c15_i)),
      &c15_thisId);
  }

  sf_mex_destroy(&c15_u);
}

static void c15_j_emlrt_marshallIn(SFc15_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c15_u, const emlrtMsgIdentifier *c15_parentId,
  real_T c15_y[7])
{
  real_T c15_dv[7];
  int32_T c15_i;
  sf_mex_import(c15_parentId, sf_mex_dup(c15_u), c15_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c15_i = 0; c15_i < 7; c15_i++) {
    c15_y[c15_i] = c15_dv[c15_i];
  }

  sf_mex_destroy(&c15_u);
}

static uint8_T c15_k_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_b_is_active_c15_sliding_controller_const, const char_T *c15_identifier)
{
  emlrtMsgIdentifier c15_thisId;
  uint8_T c15_y;
  c15_thisId.fIdentifier = const_cast<const char_T *>(c15_identifier);
  c15_thisId.fParent = NULL;
  c15_thisId.bParentIsCell = false;
  c15_y = c15_l_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c15_b_is_active_c15_sliding_controller_const), &c15_thisId);
  sf_mex_destroy(&c15_b_is_active_c15_sliding_controller_const);
  return c15_y;
}

static uint8_T c15_l_emlrt_marshallIn
  (SFc15_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c15_u, const emlrtMsgIdentifier *c15_parentId)
{
  uint8_T c15_b_u;
  uint8_T c15_y;
  sf_mex_import(c15_parentId, sf_mex_dup(c15_u), &c15_b_u, 1, 3, 0U, 0, 0U, 0);
  c15_y = c15_b_u;
  sf_mex_destroy(&c15_u);
  return c15_y;
}

static void c15_eML_blk_kernel(SFc15_sliding_controller_constInstanceStruct
  *chartInstance, c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c15_b_blankMsg)
{
  static uint8_T c15_uv5[10] = { 82U, 105U, 103U, 104U, 116U, 65U, 110U, 107U,
    108U, 101U };

  static uint8_T c15_uv2[9] = { 76U, 101U, 102U, 116U, 65U, 110U, 107U, 108U,
    101U };

  static uint8_T c15_uv4[9] = { 82U, 105U, 103U, 104U, 116U, 75U, 110U, 101U,
    101U };

  static uint8_T c15_uv1[8] = { 76U, 101U, 102U, 116U, 75U, 110U, 101U, 101U };

  static uint8_T c15_uv3[8] = { 82U, 105U, 103U, 104U, 116U, 72U, 105U, 112U };

  static uint8_T c15_uv[7] = { 76U, 101U, 102U, 116U, 72U, 105U, 112U };

  static uint8_T c15_uv6[6] = { 98U, 108U, 97U, 110U, 99U, 107U };

  hipMemcpy(chartInstance->c15_gpu_blankMsg, c15_b_blankMsg, 2536UL,
             hipMemcpyHostToDevice);
  c15_eML_blk_kernel_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c15_gpu_blankMsg);
  hipMemcpy(*chartInstance->c15_gpu_uv, c15_uv, 7UL, hipMemcpyHostToDevice);
  c15_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c15_gpu_uv, chartInstance->c15_gpu_blankMsg);
  c15_eML_blk_kernel_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c15_gpu_blankMsg);
  hipMemcpy(*chartInstance->c15_gpu_uv1, c15_uv1, 8UL, hipMemcpyHostToDevice);
  c15_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c15_gpu_uv1, chartInstance->c15_gpu_blankMsg);
  c15_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c15_gpu_blankMsg);
  hipMemcpy(*chartInstance->c15_gpu_uv2, c15_uv2, 9UL, hipMemcpyHostToDevice);
  c15_eML_blk_kernel_kernel15<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c15_gpu_uv2, chartInstance->c15_gpu_blankMsg);
  c15_eML_blk_kernel_kernel16<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c15_gpu_blankMsg);
  hipMemcpy(*chartInstance->c15_gpu_uv3, c15_uv3, 8UL, hipMemcpyHostToDevice);
  c15_eML_blk_kernel_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c15_gpu_uv3, chartInstance->c15_gpu_blankMsg);
  c15_eML_blk_kernel_kernel18<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c15_gpu_blankMsg);
  hipMemcpy(*chartInstance->c15_gpu_uv4, c15_uv4, 9UL, hipMemcpyHostToDevice);
  c15_eML_blk_kernel_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c15_gpu_uv4, chartInstance->c15_gpu_blankMsg);
  c15_eML_blk_kernel_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c15_gpu_blankMsg);
  hipMemcpy(*chartInstance->c15_gpu_uv5, c15_uv5, 10UL, hipMemcpyHostToDevice);
  c15_eML_blk_kernel_kernel21<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c15_gpu_uv5, chartInstance->c15_gpu_blankMsg);
  c15_eML_blk_kernel_kernel22<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c15_gpu_blankMsg);
  hipMemcpy(*chartInstance->c15_gpu_uv6, c15_uv6, 6UL, hipMemcpyHostToDevice);
  c15_eML_blk_kernel_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c15_gpu_uv6, chartInstance->c15_gpu_blankMsg);
  c15_eML_blk_kernel_kernel24<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c15_gpu_blankMsg);
  hipMemcpy(c15_b_blankMsg, chartInstance->c15_gpu_blankMsg, 2536UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(128, 1) void
  c15_sf_gateway_c15_sliding_controller_const_kernel1
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg,
   c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg)
{
  uint64_T c15_threadId;
  int32_T c15_i;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i = (int32_T)c15_threadId;
  if (c15_i < 128) {
    c15_b_msg->Header.FrameId[c15_i] = c15_b_blankMsg->Header.FrameId[c15_i];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c15_sf_gateway_c15_sliding_controller_const_kernel2(const int32_T c15_i1,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg)
{
  uint64_T c15_threadId;
  int32_T c15_i2;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i2 = (int32_T)c15_threadId;
  if (c15_i2 < 128) {
    c15_b_msg->Name[c15_i1].Data[c15_i2] = c15_b_blankMsg->Name[c15_i1]
      .Data[c15_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c15_sf_gateway_c15_sliding_controller_const_kernel3
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg,
   c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg)
{
  uint64_T c15_threadId;
  int32_T c15_i3;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i3 = (int32_T)c15_threadId;
  if (c15_i3 < 7) {
    c15_b_msg->Position[c15_i3] = c15_b_blankMsg->Position[c15_i3];
    c15_b_msg->Velocity[c15_i3] = c15_b_blankMsg->Velocity[c15_i3];
    c15_b_msg->Effort[c15_i3] = c15_b_blankMsg->Effort[c15_i3];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c15_sf_gateway_c15_sliding_controller_const_kernel4
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg,
   c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r)
{
  uint64_T c15_threadId;
  int32_T c15_i4;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i4 = (int32_T)c15_threadId;
  if (c15_i4 < 128) {
    c15_r->Header.FrameId[c15_i4] = c15_b_msg->Header.FrameId[c15_i4];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c15_sf_gateway_c15_sliding_controller_const_kernel5(const int32_T c15_i5,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r)
{
  uint64_T c15_threadId;
  int32_T c15_i6;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i6 = (int32_T)c15_threadId;
  if (c15_i6 < 128) {
    c15_r->Name[c15_i5].Data[c15_i6] = c15_b_msg->Name[c15_i5].Data[c15_i6];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c15_sf_gateway_c15_sliding_controller_const_kernel6
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg,
   c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r)
{
  uint64_T c15_threadId;
  int32_T c15_i7;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i7 = (int32_T)c15_threadId;
  if (c15_i7 < 7) {
    c15_r->Position[c15_i7] = c15_b_msg->Position[c15_i7];
    c15_r->Velocity[c15_i7] = c15_b_msg->Velocity[c15_i7];
    c15_r->Effort[c15_i7] = c15_b_msg->Effort[c15_i7];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c15_sf_gateway_c15_sliding_controller_const_kernel7(const
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg)
{
  uint64_T c15_threadId;
  int32_T c15_i8;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i8 = (int32_T)c15_threadId;
  if (c15_i8 < 128) {
    c15_b_msg->Header.FrameId[c15_i8] = c15_r->Header.FrameId[c15_i8];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c15_sf_gateway_c15_sliding_controller_const_kernel8(const
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r, const
  int32_T c15_i9, c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState
  *c15_b_msg)
{
  uint64_T c15_threadId;
  int32_T c15_i10;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i10 = (int32_T)c15_threadId;
  if (c15_i10 < 128) {
    c15_b_msg->Name[c15_i9].Data[c15_i10] = c15_r->Name[c15_i9].Data[c15_i10];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c15_sf_gateway_c15_sliding_controller_const_kernel9(const
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_r,
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_msg)
{
  uint64_T c15_threadId;
  int32_T c15_i11;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i11 = (int32_T)c15_threadId;
  if (c15_i11 < 7) {
    c15_b_msg->Position[c15_i11] = c15_r->Position[c15_i11];
    c15_b_msg->Velocity[c15_i11] = c15_r->Velocity[c15_i11];
    c15_b_msg->Effort[c15_i11] = c15_r->Effort[c15_i11];
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel10
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_tmpIdx;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_tmpIdx = (int32_T)c15_threadId;
  if (c15_tmpIdx < 1) {
    c15_b_blankMsg->Name_SL_Info.CurrentLength = 7U;
    c15_b_blankMsg->Velocity_SL_Info.CurrentLength = 7U;
    c15_b_blankMsg->Position_SL_Info.CurrentLength = 7U;
    c15_b_blankMsg->Effort_SL_Info.CurrentLength = 7U;
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel11(
  const uint8_T c15_uv[7],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_i;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i = (int32_T)c15_threadId;
  if (c15_i < 7) {
    c15_b_blankMsg->Name[0].Data[c15_i] = c15_uv[c15_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel12
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_tmpIdx;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_tmpIdx = (int32_T)c15_threadId;
  if (c15_tmpIdx < 1) {
    c15_b_blankMsg->Name[0].Data_SL_Info.CurrentLength = 7U;
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel13(
  const uint8_T c15_uv1[8],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_i1;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i1 = (int32_T)c15_threadId;
  if (c15_i1 < 8) {
    c15_b_blankMsg->Name[1].Data[c15_i1] = c15_uv1[c15_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel14
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_tmpIdx;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_tmpIdx = (int32_T)c15_threadId;
  if (c15_tmpIdx < 1) {
    c15_b_blankMsg->Name[1].Data_SL_Info.CurrentLength = 8U;
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel15(
  const uint8_T c15_uv2[9],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_i2;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i2 = (int32_T)c15_threadId;
  if (c15_i2 < 9) {
    c15_b_blankMsg->Name[2].Data[c15_i2] = c15_uv2[c15_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel16
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_tmpIdx;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_tmpIdx = (int32_T)c15_threadId;
  if (c15_tmpIdx < 1) {
    c15_b_blankMsg->Name[2].Data_SL_Info.CurrentLength = 9U;
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel17(
  const uint8_T c15_uv3[8],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_i3;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i3 = (int32_T)c15_threadId;
  if (c15_i3 < 8) {
    c15_b_blankMsg->Name[3].Data[c15_i3] = c15_uv3[c15_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel18
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_tmpIdx;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_tmpIdx = (int32_T)c15_threadId;
  if (c15_tmpIdx < 1) {
    c15_b_blankMsg->Name[3].Data_SL_Info.CurrentLength = 8U;
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel19(
  const uint8_T c15_uv4[9],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_i4;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i4 = (int32_T)c15_threadId;
  if (c15_i4 < 9) {
    c15_b_blankMsg->Name[4].Data[c15_i4] = c15_uv4[c15_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel20
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_tmpIdx;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_tmpIdx = (int32_T)c15_threadId;
  if (c15_tmpIdx < 1) {
    c15_b_blankMsg->Name[4].Data_SL_Info.CurrentLength = 9U;
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel21(
  const uint8_T c15_uv5[10],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_i5;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i5 = (int32_T)c15_threadId;
  if (c15_i5 < 10) {
    c15_b_blankMsg->Name[5].Data[c15_i5] = c15_uv5[c15_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel22
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_tmpIdx;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_tmpIdx = (int32_T)c15_threadId;
  if (c15_tmpIdx < 1) {
    c15_b_blankMsg->Name[5].Data_SL_Info.CurrentLength = 10U;
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel23(
  const uint8_T c15_uv6[6],
  c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_i6;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_i6 = (int32_T)c15_threadId;
  if (c15_i6 < 6) {
    c15_b_blankMsg->Name[6].Data[c15_i6] = c15_uv6[c15_i6];
  }
}

static __global__ __launch_bounds__(32, 1) void c15_eML_blk_kernel_kernel24
  (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *c15_b_blankMsg)
{
  uint64_T c15_threadId;
  int32_T c15_tmpIdx;
  c15_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c15_tmpIdx = (int32_T)c15_threadId;
  if (c15_tmpIdx < 1) {
    c15_b_blankMsg->Name[6].Data_SL_Info.CurrentLength = 6U;
  }
}

static void init_dsm_address_info(SFc15_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address
  (SFc15_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c15_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c15_blankMsg =
    (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c15_msg =
    (c15_SL_Bus_sliding_controller_const_sensor_msgs_JointState *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c15_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2735176283U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(345749275U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(939477939U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(7884259U);
}

mxArray *sf_c15_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c15_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c15_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c15_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvxqK+yH8zOL4xOSSzLLU+GRD0/jinMyUzLz0+OT8vJKi/Jyc1CIQs7iEAe5PAEN8Hwg="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c15_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "smwuocKm6OMBjJNMYsbyoM";
}

static void sf_opaque_initialize_c15_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c15_sliding_controller_const(void *chartInstanceVar)
{
  enable_c15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c15_sliding_controller_const(void
  *chartInstanceVar)
{
  disable_c15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c15_sliding_controller_const(void
  *chartInstanceVar)
{
  sf_gateway_c15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c15_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c15_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c15_sliding_controller_const
  (void *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc15_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c15_sliding_controller_const
      ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc15_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c15_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c15_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc15_sliding_controller_const
    ((SFc15_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c15_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c15_sliding_controller_const
      ((SFc15_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
    initSimStructsc15_sliding_controller_const
      ((SFc15_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
  }
}

const char* sf_c15_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u00AQdkNBIAGqEOKEBAcE3IADiBO0+ROBhla45edUbdeTeMl61+xP0py4cOVdeAWeAQn",
    "xGDwCs7aTpo7tUCIqQFhynbW/mZ2Zb3623kqn6+F1Ee/1S553Bp9n8a556XU6W6/M3On7Ve9Wtv",
    "6IQsJG20SRSHuVlyARvAAtuTVMio7oyUIYEz1QIChiY6lMmTbNIsuZGLStoE6ffhUyGvqhtDyoo",
    "ywJtgQfo7bYmm3U02QKqGkDBCZU0vbDNif9qcXKjBoh0IG2UZULGoxvY2eW7lpuWMyhdQC0I7Qh",
    "aLE+tM03xEDDHJS66TzV/gQoo5gzIgq9DYn2IcYAG9iNA/y7ZQ06lYfRkChTh5AMQW+yQaJTCsj",
    "rZBo/7DNBjFSM8FbEG05w3rZtjvZ0ZQC8IiBoW10BGcSSCVPOv99GT1uC7HNowr7tl2vz4Z115L",
    "9kMAJVGrdeQw5BkT5sidJNk4C0DhK2plkyDzMsgpdEbVDkT0NQmr2YOdonyBPsoEQZDBInO3pHs",
    "SGGt1SbjTouMxeVjI1SsvUiWKKtNYQqFqba2lQ0COe6FLYj400YAk+0Nokh1bBUazFOaxbsSAyw",
    "S+/yarCCIfEZrCFFwArpGuYASd95jo3lKJJabWTUwORtbm7Of56HdYQB1SMUirqAIkwDxiwJb7m",
    "2gGnHPQLRKpOYVwROM2QRytM9K5ojqQYYk4omcuiCY7QUGOk+comVsKuxaKpgjstFOEpoCIFrMI",
    "xDF8sGsQUx0a61bWDdDZkZN0FTxeICVi1WHbahlkuocQy7YiDkSLSVjPysx6fhBcBsI0ow0a9j/",
    "1LjNu5eRIWbZ3e9w3l2/ifm2UQu/7w9o2elQI8388zve65WvW8Nf61M5u+M3IXcPqs5OYdbw7uv",
    "7I3eh9b7L18fX/q8Ed5ZZv9PtePN/4vZ+uqk0U4LZziXrw77ZMau1QL9V2b0r2VrHY2spM+iB1v",
    "d+tunz7tv9P5YdjN+Fthby9k7eX/ddXzMsiQ/Fe0E2cHErYlNx7XT/3DG3jML4nEue59e3x8vJ3",
    "95Pc9jUbxOHYnXKVe5R/L21/e/vr6cfLr/6wX238zxfTOZ63vEdQvYo/fu72nOAiz1PSqFUZJzU",
    "O6nNkfr81fz/bhy3gnL/S12/pf7/bz/zPw6aTnvhOWW9e+4c/xPw1fNAy+HX/uD/Vj2fPW78d+8",
    "452DrmXrR9N/dRoh40HBqTf7jEfYXtHXfyBPfwArwLTp",
    ""
  };

  static char newstr [1245] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c15_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c15_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(611676775U));
  ssSetChecksum1(S,(2135262310U));
  ssSetChecksum2(S,(306173651U));
  ssSetChecksum3(S,(795361729U));
}

static void mdlRTW_c15_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c15_sliding_controller_const(SimStruct *S)
{
  SFc15_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc15_sliding_controller_constInstanceStruct *)utMalloc
    (sizeof(SFc15_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc15_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc15_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c15_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c15_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c15_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c15_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c15_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c15_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c15_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c15_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c15_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c15_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c15_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c15_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c15_JITStateAnimation,
    chartInstance->c15_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c15_sliding_controller_const(chartInstance);
}

void c15_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c15_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c15_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c15_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c15_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
