/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c12_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void c12_do_animation_call_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c12_st);
static void sf_gateway_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);
static void c12_eML_blk_kernel(SFc12_sliding_controller_constInstanceStruct
  *chartInstance, real_T c12_b_q[2], real_T c12_b_y[4]);
static void c12_emlrt_marshallIn(SFc12_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c12_b_y, const char_T *c12_identifier, real_T
  c12_c_y[4]);
static void c12_b_emlrt_marshallIn(SFc12_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId,
  real_T c12_b_y[4]);
static uint8_T c12_c_emlrt_marshallIn
  (SFc12_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c12_b_is_active_c12_sliding_controller_const, const char_T *c12_identifier);
static uint8_T c12_d_emlrt_marshallIn
  (SFc12_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c12_u, const emlrtMsgIdentifier *c12_parentId);
static void init_dsm_address_info(SFc12_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address
  (SFc12_sliding_controller_constInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c12_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c12_is_active_c12_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c12_do_animation_call_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c12_b_y = NULL;
  const mxArray *c12_c_y = NULL;
  const mxArray *c12_d_y = NULL;
  const mxArray *c12_st;
  c12_st = NULL;
  c12_b_y = NULL;
  sf_mex_assign(&c12_b_y, sf_mex_createcellmatrix(2, 1), false);
  c12_c_y = NULL;
  sf_mex_assign(&c12_c_y, sf_mex_create("y", *chartInstance->c12_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c12_b_y, 0, c12_c_y);
  c12_d_y = NULL;
  sf_mex_assign(&c12_d_y, sf_mex_create("y",
    &chartInstance->c12_is_active_c12_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c12_b_y, 1, c12_d_y);
  sf_mex_assign(&c12_st, c12_b_y, false);
  return c12_st;
}

static void set_sim_state_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c12_st)
{
  const mxArray *c12_u;
  c12_u = sf_mex_dup(c12_st);
  c12_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c12_u, 0)), "y",
                       *chartInstance->c12_y);
  chartInstance->c12_is_active_c12_sliding_controller_const =
    c12_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c12_u, 1)),
    "is_active_c12_sliding_controller_const");
  sf_mex_destroy(&c12_u);
  sf_mex_destroy(&c12_st);
}

static void sf_gateway_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c12_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c12_eML_blk_kernel(chartInstance, *chartInstance->c12_q, *chartInstance->c12_y);
  c12_do_animation_call_c12_sliding_controller_const(chartInstance);
}

static void mdl_start_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c12_errCode;
  c12_errCode = hipGetLastError();
  if (c12_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c12_errCode, (char_T *)hipGetErrorName
                       (c12_errCode), (char_T *)hipGetErrorString(c12_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c12_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc12_sliding_controller_const
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c12_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c12_nameCaptureInfo = NULL;
  c12_nameCaptureInfo = NULL;
  sf_mex_assign(&c12_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c12_nameCaptureInfo;
}

static void c12_eML_blk_kernel(SFc12_sliding_controller_constInstanceStruct
  *chartInstance, real_T c12_b_q[2], real_T c12_b_y[4])
{
  c12_b_y[0] = -0.324 * muDoubleScalarSin(c12_b_q[0]) - 0.39 * muDoubleScalarSin
    (c12_b_q[0] + c12_b_q[1]);
  c12_b_y[2] = 0.324 * muDoubleScalarCos(c12_b_q[0]) + 0.39 * muDoubleScalarSin
    (c12_b_q[0] + c12_b_q[1]);
  c12_b_y[1] = -0.39 * muDoubleScalarSin(c12_b_q[0] + c12_b_q[1]);
  c12_b_y[3] = 0.39 * muDoubleScalarCos(c12_b_q[0] + c12_b_q[1]);
}

static void c12_emlrt_marshallIn(SFc12_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c12_b_y, const char_T *c12_identifier, real_T
  c12_c_y[4])
{
  emlrtMsgIdentifier c12_thisId;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c12_b_y), &c12_thisId,
    c12_c_y);
  sf_mex_destroy(&c12_b_y);
}

static void c12_b_emlrt_marshallIn(SFc12_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c12_u, const emlrtMsgIdentifier *c12_parentId,
  real_T c12_b_y[4])
{
  real_T c12_dv[4];
  int32_T c12_i;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), c12_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c12_i = 0; c12_i < 4; c12_i++) {
    c12_b_y[c12_i] = c12_dv[c12_i];
  }

  sf_mex_destroy(&c12_u);
}

static uint8_T c12_c_emlrt_marshallIn
  (SFc12_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c12_b_is_active_c12_sliding_controller_const, const char_T *c12_identifier)
{
  emlrtMsgIdentifier c12_thisId;
  uint8_T c12_b_y;
  c12_thisId.fIdentifier = const_cast<const char_T *>(c12_identifier);
  c12_thisId.fParent = NULL;
  c12_thisId.bParentIsCell = false;
  c12_b_y = c12_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c12_b_is_active_c12_sliding_controller_const), &c12_thisId);
  sf_mex_destroy(&c12_b_is_active_c12_sliding_controller_const);
  return c12_b_y;
}

static uint8_T c12_d_emlrt_marshallIn
  (SFc12_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c12_u, const emlrtMsgIdentifier *c12_parentId)
{
  uint8_T c12_b_u;
  uint8_T c12_b_y;
  sf_mex_import(c12_parentId, sf_mex_dup(c12_u), &c12_b_u, 1, 3, 0U, 0, 0U, 0);
  c12_b_y = c12_b_u;
  sf_mex_destroy(&c12_u);
  return c12_b_y;
}

static void init_dsm_address_info(SFc12_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address
  (SFc12_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c12_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c12_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c12_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c12_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(552332206U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2608078302U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1441289006U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4066884445U);
}

mxArray *sf_c12_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c12_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c12_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c12_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfj"
    "UU90P4mcXxicklmWWp8cmGRvHFOZkpmXnp8cn5eSVF+Tk5qUUgZnEJwp8Ae/YeMw=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c12_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sjS5Om55RaK9v6MEldWgzUF";
}

static void sf_opaque_initialize_c12_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c12_sliding_controller_const(void *chartInstanceVar)
{
  enable_c12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c12_sliding_controller_const(void
  *chartInstanceVar)
{
  disable_c12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c12_sliding_controller_const(void
  *chartInstanceVar)
{
  sf_gateway_c12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c12_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c12_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c12_sliding_controller_const
  (void *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc12_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c12_sliding_controller_const
      ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc12_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c12_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c12_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc12_sliding_controller_const
    ((SFc12_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c12_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c12_sliding_controller_const
      ((SFc12_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
    initSimStructsc12_sliding_controller_const
      ((SFc12_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
  }
}

const char* sf_c12_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgXXrYM2MIoAvQSoT3aPbQEX7aGJbf3AQqzaMGW7N2O9HIkbLXeZ/aGtvkNOQd+",
    "gxxzyJD333hco0BfoLEnJMkVScYQYaVECFLncb2Zn5pudWXkr3Z6H10O8dz/3vDV8foJ3w8uuj/",
    "LxysydfV/1tvPxSxQSNjomikTaq70EieAEtOTWMCm6YiBLYUwMQIGgiI2lMlXaNIssZ2LUsYI6f",
    "fo8ZDT0Q2l5sI+yJDgSfIzaYmuOUU+LKaCmAxCYUEk7DDucDKcWK3PVDIGOtI3qXNBgfBs7s3TP",
    "csNiDu1roF2hDUGL9Y1tviEGmua60k3nqfYnQBnFnBFR6m1ItA8xBtjAaRzg75E16FQRRkOizD6",
    "EJAF9yEapTimgqJNpnLhkghipGOHtiDed4Lxtxxzt6ckAeE1A0LZ9BWQUSyZMNf9+Bz1tC3LJoQ",
    "WXdlitzYcX1pF/xuAKVGXcBk2ZgCJDOBKVi6YBaV+nbE2zZB5mWARnRO1R5E9DUJm9mDnaJ8gT9",
    "FGiCgapk13dVyzB8FZqs1HXZeaiLWOjjGy9CJZqaydQx8JUW4eKJuFcV8L6Mj6EBHiqtUUMqYdl",
    "WstxWrOgLzHALr2rd4MVDInPYU0pAlZKV1IApHXnJywst5HUaiOjJiZv6/Bwfnoe1hUG1IBQKKs",
    "CijANGLM0vNXaAqYd9whEq0xqXhk4y5BFKE8PrGhdSTXCmNQUkRsXHKOVwEgPkUvcCacaN00dzH",
    "G5CEcJDSFwBYZx6OG2QWxJTLQrbXu47xJmxi3QVLG4hFWLuw7LUNsl1DiGUzES8kp0lIz8vMZn4",
    "QXAbCNKMDHcx/qlxh1cvYwK18++9m762adv0c8mcsXnVzN6Vkr0eDPP4rrrjfp1G/i2Mum/M3Kf",
    "FdZZLcg53AbeD55v/vr35uvx768+/vNg+7cfl1n/TeNu/f9hPn48KbTTjZPM5avDHszYtVqi/4s",
    "Z/Rv5WD/3d46inZ0T8uyH5Ltemwfnw19OOxk/C+xtFOydfN90FR+zLM1PRbtBfjBxY2Kzdu30fz",
    "9j79qCeKzn37Prr6fLyT/aLfK4ukB+A9/Ghbx99/U3d5eTz9b/eYH9WwW+t9K+fkFctYAL+s23F",
    "5qzALf6BZXCKMk5KPeqzW0/3zXf7yrn3bPcv8XO/+XeP+9v07/uW867Z7ll/btrH//Q8HX9wCvg",
    "Nz5gP5Y9X71v/B/e3c5BX+bjJ9O/Os2Q8aDk1JtP4xF2UDb7H8jTfwCUfLPm",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c12_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c12_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2501929482U));
  ssSetChecksum1(S,(2041323763U));
  ssSetChecksum2(S,(3808924622U));
  ssSetChecksum3(S,(1034168136U));
}

static void mdlRTW_c12_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c12_sliding_controller_const(SimStruct *S)
{
  SFc12_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc12_sliding_controller_constInstanceStruct *)utMalloc
    (sizeof(SFc12_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc12_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc12_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c12_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c12_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c12_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c12_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c12_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c12_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c12_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c12_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c12_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c12_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c12_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c12_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c12_JITStateAnimation,
    chartInstance->c12_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c12_sliding_controller_const(chartInstance);
}

void c12_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c12_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c12_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c12_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c12_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
