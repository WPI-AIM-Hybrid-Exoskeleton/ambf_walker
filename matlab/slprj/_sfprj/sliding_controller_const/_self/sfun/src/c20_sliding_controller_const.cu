/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c20_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void c20_do_animation_call_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c20_st);
static void sf_gateway_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);
static void c20_eML_blk_kernel(SFc20_sliding_controller_constInstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2]);
static void c20_emlrt_marshallIn(SFc20_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2]);
static void c20_b_emlrt_marshallIn(SFc20_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2]);
static uint8_T c20_c_emlrt_marshallIn
  (SFc20_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c20_b_is_active_c20_sliding_controller_const, const char_T *c20_identifier);
static uint8_T c20_d_emlrt_marshallIn
  (SFc20_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c20_u, const emlrtMsgIdentifier *c20_parentId);
static void init_dsm_address_info(SFc20_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address
  (SFc20_sliding_controller_constInstanceStruct *chartInstance);

/* Function Definitions */
static void initialize_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c20_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c20_is_active_c20_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c20_do_animation_call_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c20_b_y = NULL;
  const mxArray *c20_c_y = NULL;
  const mxArray *c20_d_y = NULL;
  const mxArray *c20_e_y = NULL;
  const mxArray *c20_f_y = NULL;
  const mxArray *c20_st;
  const mxArray *c20_y = NULL;
  c20_st = NULL;
  c20_y = NULL;
  sf_mex_assign(&c20_y, sf_mex_createcellmatrix(5, 1), false);
  c20_b_y = NULL;
  sf_mex_assign(&c20_b_y, sf_mex_create("y", *chartInstance->c20_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 0, c20_b_y);
  c20_c_y = NULL;
  sf_mex_assign(&c20_c_y, sf_mex_create("y", *chartInstance->c20_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 1, c20_c_y);
  c20_d_y = NULL;
  sf_mex_assign(&c20_d_y, sf_mex_create("y", *chartInstance->c20_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 2, c20_d_y);
  c20_e_y = NULL;
  sf_mex_assign(&c20_e_y, sf_mex_create("y", *chartInstance->c20_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c20_y, 3, c20_e_y);
  c20_f_y = NULL;
  sf_mex_assign(&c20_f_y, sf_mex_create("y",
    &chartInstance->c20_is_active_c20_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c20_y, 4, c20_f_y);
  sf_mex_assign(&c20_st, c20_y, false);
  return c20_st;
}

static void set_sim_state_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c20_st)
{
  const mxArray *c20_u;
  c20_u = sf_mex_dup(c20_st);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 0)),
                       "ankle", *chartInstance->c20_ankle);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 1)),
                       "danke", *chartInstance->c20_danke);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 2)),
                       "dknee", *chartInstance->c20_dknee);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 3)),
                       "knee", *chartInstance->c20_knee);
  chartInstance->c20_is_active_c20_sliding_controller_const =
    c20_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 4)),
    "is_active_c20_sliding_controller_const");
  sf_mex_destroy(&c20_u);
  sf_mex_destroy(&c20_st);
}

static void sf_gateway_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c20_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c20_eML_blk_kernel(chartInstance, *chartInstance->c20_q,
                     *chartInstance->c20_qd, *chartInstance->c20_knee,
                     *chartInstance->c20_dknee, *chartInstance->c20_ankle,
                     *chartInstance->c20_danke);
  c20_do_animation_call_c20_sliding_controller_const(chartInstance);
}

static void mdl_start_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c20_errCode;
  c20_errCode = hipGetLastError();
  if (c20_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c20_errCode, (char_T *)hipGetErrorName
                       (c20_errCode), (char_T *)hipGetErrorString(c20_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c20_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc20_sliding_controller_const
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c20_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c20_nameCaptureInfo = NULL;
  const char_T *c20_data[4] = {
    "789cc553cb4ac34014bd955adc545d09825b372e1cdd886b8b850a2db50f5c88c69b64d20c9d874c52dbbaf113fc1dbfc7853bffc1246d9a26182216eb8170e7"
    "7032f7dc7b60a0d4689600603bf83e5e00ce2b10a13a2bb03baf1b9046562fe5d4189b504edd0bf5f7c0ef75ce2d257d3af16744a2a08b9bb6124ca2f47bd347",
    "0a9a7a8a3f513b521cc6698f09da5d26ad9089fa92b420a1149e6b2eb586dd9100ed7ac9847c9944798478c8d9b75c904716d93cb2ffc57e935ffac5fdf70afc"
    "62dd517a8cda36864c32813eb33c83d3c1ea7bc7be95dc39668aad4626a789dfdb8a7ebd5cbfb47e7b79475c252891e8bb2819e503c56d07b5492cf483348cb1",
    "473c6d1114a6638c910fa92641401c4df27d66c722ec5b94dbce0ff7c87b3f55d88aea677b1fd7e9777f7470b84ebf18ffe5b7aef777d3bec093b3e72bd6e9e8"
    "7af3badf6df44f5bb5648e76814fd11c90c3ffbaff177b596734", "" };

  c20_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c20_data[0], 1616U, &c20_nameCaptureInfo);
  return c20_nameCaptureInfo;
}

static void c20_eML_blk_kernel(SFc20_sliding_controller_constInstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2])
{
  real_T c20_a;
  real_T c20_b_a;
  c20_b_knee[0] = -0.324 * muDoubleScalarCos(c20_b_q[0]);
  c20_b_knee[1] = -0.324 * muDoubleScalarSin(c20_b_q[0]);
  c20_a = -0.324 * c20_b_qd[0];
  c20_b_dknee[0] = c20_a * -muDoubleScalarSin(c20_b_q[0]);
  c20_b_dknee[1] = c20_a * muDoubleScalarCos(c20_b_q[0]);
  c20_b_a = -0.39 * (c20_b_qd[0] + c20_b_qd[1]);
  c20_b_ankle[0] = c20_b_knee[0] - -0.39 * muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_ankle[1] = c20_b_knee[1] - -0.39 * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[0] = c20_b_dknee[0] + c20_b_a * -muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[1] = c20_b_dknee[1] + c20_b_a * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
}

static void c20_emlrt_marshallIn(SFc20_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2])
{
  emlrtMsgIdentifier c20_thisId;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c20_b_ankle), &c20_thisId,
    c20_y);
  sf_mex_destroy(&c20_b_ankle);
}

static void c20_b_emlrt_marshallIn(SFc20_sliding_controller_constInstanceStruct *
  chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2])
{
  real_T c20_dv[2];
  int32_T c20_i;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), c20_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c20_i = 0; c20_i < 2; c20_i++) {
    c20_y[c20_i] = c20_dv[c20_i];
  }

  sf_mex_destroy(&c20_u);
}

static uint8_T c20_c_emlrt_marshallIn
  (SFc20_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c20_b_is_active_c20_sliding_controller_const, const char_T *c20_identifier)
{
  emlrtMsgIdentifier c20_thisId;
  uint8_T c20_y;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_y = c20_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c20_b_is_active_c20_sliding_controller_const), &c20_thisId);
  sf_mex_destroy(&c20_b_is_active_c20_sliding_controller_const);
  return c20_y;
}

static uint8_T c20_d_emlrt_marshallIn
  (SFc20_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c20_u, const emlrtMsgIdentifier *c20_parentId)
{
  uint8_T c20_b_u;
  uint8_T c20_y;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), &c20_b_u, 1, 3, 0U, 0, 0U, 0);
  c20_y = c20_b_u;
  sf_mex_destroy(&c20_u);
  return c20_y;
}

static void init_dsm_address_info(SFc20_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address
  (SFc20_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c20_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c20_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c20_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c20_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c20_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c20_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c20_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c20_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c20_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c20_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3q6HFA4ifWRyfmFySWZYan"
    "2xkEF+ck5mSmZcen5yfV1KUn5OTWgRiFpcg4gkAp6MvDg=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c20_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c20_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c20_sliding_controller_const(void *chartInstanceVar)
{
  enable_c20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c20_sliding_controller_const(void
  *chartInstanceVar)
{
  disable_c20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c20_sliding_controller_const(void
  *chartInstanceVar)
{
  sf_gateway_c20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c20_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c20_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c20_sliding_controller_const
  (void *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc20_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c20_sliding_controller_const
      ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc20_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c20_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c20_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc20_sliding_controller_const
    ((SFc20_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c20_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c20_sliding_controller_const
      ((SFc20_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
    initSimStructsc20_sliding_controller_const
      ((SFc20_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr
       (S));
  }
}

const char* sf_c20_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFAL0EbFEXaSwGjRYH20timJESBHTulnAY91FgvR+JCy112f2ir79C",
    "ce+m9D9BLn6BAniCnPEPQJ+gsScsyRVKR3SROURq0tOQ33843Mzu78lqDXQ+vm3j/esvzruHndb",
    "zbXn6tFePW3J0/73ifFuNf0EjYeJ8oEmuv8RIkhu9AS24Nk2IgRrISxsQIFAiK2EQqU8emWWw5E",
    "5O+FdTx6e8jRqMgkpaH22hLwj3Bp8iWWLOPPF2mgJo+QGgiJe046nMynnmszLEfAZ1oGzdJ0GAC",
    "mzi39K7lhiUceidAB0Ibgh7rM98CQwz45qRWplOqg1OgjBPOiKhUGxEdQIIBNnCQhPh/zxoUVYb",
    "RiCizDRFJQe+wScYpBZQ5mcYXR0wQIxUjvBdz3xku+rbP0Z9dGQJvCAj6tq2ATBLJhKnPf9BHpT",
    "1Bjjh04ciO69kC+Mm65D9mcAyqNm4jX6agyBj2RO2kWUB6J1m2ZlWyCDMshsdEbVHMn4awtnqxc",
    "nRAME8wRIs6GGQiB3qoWIrhrWWz8cBV5rIlY+M82XoZLGPrpdCUhRlbnwqfcK5rYUOZ7EAKPGPt",
    "EkOaYTlrNU5rFg4lBtiVd/1qsIJh4guYL0XIKtOVlgBZ33mIjeU8klptZOxj8XZ3dhZfL8IGwoA",
    "aEQpVXUARpgFjloW3ni1k2uUegeiVydyrAucVsgzl6ZEV3WOpJhiThiZyJsFltBYY6zHmElfCgc",
    "ZF0wRzuVyGo4RGELoGwzjs4rJBbEVMtGttW7juUmamXdBUsaQiqxZXHbahniuoaQIHYiLksegrG",
    "QdFj8/DC4DVRpRgYryN/UtN+zh7VSrcfrbhne1n773CfnZqV/78bI6nVcHjzX2W573RPo9vl+Zt",
    "41+rldttztm9X5qnU7JzuHW8H7z47ZOHH/7x91/PPn86/f3PH5fN31qYv5V9d3bX11bb/28W49u",
    "njXa2cNKFenXY+3N+dSr4P5jjXy/G+lH0lUh/uB9tPSLDeLDx5c/fbKRHfsb3pNPs71rJ39Pnd1",
    "zHxyrL6lPRQVgcTNyY2Hy7dvxfz/l7bUk8bhTP8+vlvcvZ394s23eW6FsvxkRMOJTq9+3p+PjCO",
    "kIUcnV03Lm4jgkeGq6Mjlub5f5QpaNzTkfHcxL+Hf8X47iafT7/kyX+3y3l4W52XjwkbheCQ/rF",
    "xqHmLMQt5JBKYZTkHJT7qs35PF20j65q571hu3fFz//tXn/eX+Vc1L6gXeeS57A3ZXdZfaueD68",
    "avmk/8Er49Sus47Ln9teNf+6tdr7+qBh/O/sJ7UeMhxW/porX+NNoVPX2P1Cn/wA1HsYO",
    ""
  };

  static char newstr [1345] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c20_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c20_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(614325834U));
  ssSetChecksum1(S,(4071890510U));
  ssSetChecksum2(S,(2385365703U));
  ssSetChecksum3(S,(1589225593U));
}

static void mdlRTW_c20_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c20_sliding_controller_const(SimStruct *S)
{
  SFc20_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc20_sliding_controller_constInstanceStruct *)utMalloc
    (sizeof(SFc20_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc20_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc20_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c20_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c20_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c20_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c20_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c20_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c20_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c20_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c20_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c20_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c20_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c20_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c20_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c20_JITStateAnimation,
    chartInstance->c20_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c20_sliding_controller_const(chartInstance);
}

void c20_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c20_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c20_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c20_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c20_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
