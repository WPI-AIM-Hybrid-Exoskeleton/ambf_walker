/* Include files */

#include "sliding_controller_const_sfun.h"
#include "c9_sliding_controller_const.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void initialize_params_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void enable_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void disable_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void c9_do_animation_call_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void ext_mode_exec_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static const mxArray *get_sim_state_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void set_sim_state_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c9_st);
static void sf_gateway_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_start_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_terminate_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void initSimStructsc9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance);
static void c9_eML_blk_kernel(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, real_T c9_b_q[2], real_T c9_b_y[4]);
static void c9_emlrt_marshallIn(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c9_b_y, const char_T *c9_identifier, real_T
  c9_c_y[4]);
static void c9_b_emlrt_marshallIn(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId,
  real_T c9_b_y[4]);
static uint8_T c9_c_emlrt_marshallIn(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c9_b_is_active_c9_sliding_controller_const,
  const char_T *c9_identifier);
static uint8_T c9_d_emlrt_marshallIn(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId);
static void init_dsm_address_info(SFc9_sliding_controller_constInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc9_sliding_controller_constInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c9_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c9_is_active_c9_sliding_controller_const = 0U;
  hipGetLastError();
}

static void initialize_params_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void enable_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c9_do_animation_call_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
  const mxArray *c9_b_y = NULL;
  const mxArray *c9_c_y = NULL;
  const mxArray *c9_d_y = NULL;
  const mxArray *c9_st;
  c9_st = NULL;
  c9_b_y = NULL;
  sf_mex_assign(&c9_b_y, sf_mex_createcellmatrix(2, 1), false);
  c9_c_y = NULL;
  sf_mex_assign(&c9_c_y, sf_mex_create("y", *chartInstance->c9_y, 0, 0U, 1U, 0U,
    2, 2, 2), false);
  sf_mex_setcell(c9_b_y, 0, c9_c_y);
  c9_d_y = NULL;
  sf_mex_assign(&c9_d_y, sf_mex_create("y",
    &chartInstance->c9_is_active_c9_sliding_controller_const, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c9_b_y, 1, c9_d_y);
  sf_mex_assign(&c9_st, c9_b_y, false);
  return c9_st;
}

static void set_sim_state_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance, const mxArray
   *c9_st)
{
  const mxArray *c9_u;
  c9_u = sf_mex_dup(c9_st);
  c9_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c9_u, 0)), "y",
                      *chartInstance->c9_y);
  chartInstance->c9_is_active_c9_sliding_controller_const =
    c9_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c9_u, 1)),
    "is_active_c9_sliding_controller_const");
  sf_mex_destroy(&c9_u);
  sf_mex_destroy(&c9_st);
}

static void sf_gateway_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
  chartInstance->c9_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c9_eML_blk_kernel(chartInstance, *chartInstance->c9_q, *chartInstance->c9_y);
  c9_do_animation_call_c9_sliding_controller_const(chartInstance);
}

static void mdl_start_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void mdl_terminate_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
  hipError_t c9_errCode;
  c9_errCode = hipGetLastError();
  if (c9_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c9_errCode, (char_T *)hipGetErrorName
                       (c9_errCode), (char_T *)hipGetErrorString(c9_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c9_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
}

static void initSimStructsc9_sliding_controller_const
  (SFc9_sliding_controller_constInstanceStruct *chartInstance)
{
}

const mxArray *sf_c9_sliding_controller_const_get_eml_resolved_functions_info()
{
  const mxArray *c9_nameCaptureInfo = NULL;
  c9_nameCaptureInfo = NULL;
  sf_mex_assign(&c9_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c9_nameCaptureInfo;
}

static void c9_eML_blk_kernel(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, real_T c9_b_q[2], real_T c9_b_y[4])
{
  c9_b_y[0] = -0.324 * muDoubleScalarSin(c9_b_q[0]) - 0.39 * muDoubleScalarSin
    (c9_b_q[0] + c9_b_q[1]);
  c9_b_y[2] = 0.324 * muDoubleScalarCos(c9_b_q[0]) + 0.39 * muDoubleScalarSin
    (c9_b_q[0] + c9_b_q[1]);
  c9_b_y[1] = -0.39 * muDoubleScalarSin(c9_b_q[0] + c9_b_q[1]);
  c9_b_y[3] = 0.39 * muDoubleScalarCos(c9_b_q[0] + c9_b_q[1]);
}

static void c9_emlrt_marshallIn(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c9_b_y, const char_T *c9_identifier, real_T
  c9_c_y[4])
{
  emlrtMsgIdentifier c9_thisId;
  c9_thisId.fIdentifier = const_cast<const char_T *>(c9_identifier);
  c9_thisId.fParent = NULL;
  c9_thisId.bParentIsCell = false;
  c9_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c9_b_y), &c9_thisId, c9_c_y);
  sf_mex_destroy(&c9_b_y);
}

static void c9_b_emlrt_marshallIn(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId,
  real_T c9_b_y[4])
{
  real_T c9_dv[4];
  int32_T c9_i;
  sf_mex_import(c9_parentId, sf_mex_dup(c9_u), c9_dv, 1, 0, 0U, 1, 0U, 2, 2, 2);
  for (c9_i = 0; c9_i < 4; c9_i++) {
    c9_b_y[c9_i] = c9_dv[c9_i];
  }

  sf_mex_destroy(&c9_u);
}

static uint8_T c9_c_emlrt_marshallIn(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c9_b_is_active_c9_sliding_controller_const,
  const char_T *c9_identifier)
{
  emlrtMsgIdentifier c9_thisId;
  uint8_T c9_b_y;
  c9_thisId.fIdentifier = const_cast<const char_T *>(c9_identifier);
  c9_thisId.fParent = NULL;
  c9_thisId.bParentIsCell = false;
  c9_b_y = c9_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c9_b_is_active_c9_sliding_controller_const), &c9_thisId);
  sf_mex_destroy(&c9_b_is_active_c9_sliding_controller_const);
  return c9_b_y;
}

static uint8_T c9_d_emlrt_marshallIn(SFc9_sliding_controller_constInstanceStruct
  *chartInstance, const mxArray *c9_u, const emlrtMsgIdentifier *c9_parentId)
{
  uint8_T c9_b_u;
  uint8_T c9_b_y;
  sf_mex_import(c9_parentId, sf_mex_dup(c9_u), &c9_b_u, 1, 3, 0U, 0, 0U, 0);
  c9_b_y = c9_b_u;
  sf_mex_destroy(&c9_u);
  return c9_b_y;
}

static void init_dsm_address_info(SFc9_sliding_controller_constInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc9_sliding_controller_constInstanceStruct
  *chartInstance)
{
  chartInstance->c9_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c9_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c9_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c9_sliding_controller_const_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(552332206U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2608078302U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1441289006U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4066884445U);
}

mxArray *sf_c9_sliding_controller_const_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c9_sliding_controller_const_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c9_sliding_controller_const_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c9_sliding_controller_const(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfl"
    "UU90P4mcXxicklmWWp8cmW8cU5mSmZeenxyfl5JUX5OTmpRSBmcQmSPwGAEB4H"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c9_sliding_controller_const_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sjS5Om55RaK9v6MEldWgzUF";
}

static void sf_opaque_initialize_c9_sliding_controller_const(void
  *chartInstanceVar)
{
  initialize_params_c9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar);
  initialize_c9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c9_sliding_controller_const(void *chartInstanceVar)
{
  enable_c9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c9_sliding_controller_const(void *chartInstanceVar)
{
  disable_c9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c9_sliding_controller_const(void *chartInstanceVar)
{
  sf_gateway_c9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c9_sliding_controller_const
  (SimStruct* S)
{
  return get_sim_state_c9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c9_sliding_controller_const(SimStruct* S,
  const mxArray *st)
{
  set_sim_state_c9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S),
     st);
}

static void sf_opaque_cleanup_runtime_resources_c9_sliding_controller_const(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc9_sliding_controller_constInstanceStruct*)
                    chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller_const_optimization_info();
    }

    mdl_cleanup_runtime_resources_c9_sliding_controller_const
      ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar);
    ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar)->
      ~SFc9_sliding_controller_constInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c9_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_start_c9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c9_sliding_controller_const(void
  *chartInstanceVar)
{
  mdl_terminate_c9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc9_sliding_controller_const
    ((SFc9_sliding_controller_constInstanceStruct*) chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c9_sliding_controller_const(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c9_sliding_controller_const
      ((SFc9_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc9_sliding_controller_const
      ((SFc9_sliding_controller_constInstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c9_sliding_controller_const_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV01v00wQdqPyKUAVQuKCRC8IjlyK4ACU5kNENLTCLXCrtutJvM161+yH2/Af3tMr/sF7fA/",
    "8Es7c+QNI/AFmbScNju1QIipAWHLs9T4zOzPP7MzGW+r2PLyu4L1+1fPO4vM83g0vu87k46WpO/",
    "u+7N3Ox/+gkLDRNlEk0l7tJUgEL0BLbg2Toiv6shTGRB8UCIrYWCpTpU2zyHImhh0rqNOnX4WMh",
    "n4oLQ82UJYEW4KPUFtszTbqaTEF1HQAAhMqaQdhh5PBxGJlDpsh0KG2UZ0LGoxvY2eW7lluWMyh",
    "fQS0K7QhaLE+ts03xEDTHFW66TzV/hgoo5gzIkq9DYn2IcYAG9iNA/zdsgadKsJoSJTZgJAkoDf",
    "ZMNUpBRR1Mo0T+0wQIxUjvB3xphOctW2boz09GQCvCQjatqGADGPJhKnm3++gp21B9jm0YN8Oqr",
    "X58MY68l8yOARVGbd+UyagyAC2ROWiaUDaRylbkyyZhRkWwUuinlDkT0NQmb2YOdonyBPsoEQVD",
    "FInu3pHsQTDW6nNRl2XmfO2jI0ysvU8WKqtnUAdCxNtHSqahHNdCduR8SYkwFOtLWJIPSzTWo7T",
    "mgU7EgPs0rt6N1jBkPgc1pQiYKV0JQVAWneeY2H5FkmtNjJqYvK2Njdnp2dhXWFA9QmFsiqgCNO",
    "AMUvDW60tYNpxj0C0yqTmlYGzDJmH8nTfitahVEOMSU0ROXbBMVoJjPQAucSdsKtx09TBHJfzcJ",
    "TQEAJXYBiHHm4bxJbERLvS9gT3XcLMqAWaKhaXsGpx12EZaruEGsWwK4ZCHoqOkpGf1/gsvACYb",
    "UQJJgYbWL/UqIOrl1Hh+tld77ifXfqOfjaWKz7vTOlZKtHjTT2L615o1K/bwLelcf+dkrtcWGe5",
    "IOdwK3hfPFh992X1/9GHf899enr7v4eLrP++cbL+fyUf3xgX2snGSWby1WGfTtm1XKL/+pT+lXy",
    "sD/y1rWht7QV59iC512vz4NXg7W4n42eOvY2CvePvq67iY5al+aloN8gPJm5MbNaunf77U/aenR",
    "OPC/n37Pr8eDH5a+tFHpfnyK/g26iQtz++/ur6YvLZ+q/n2H+rwPettK/vEVctYI8+2NOcBbjT9",
    "6gURknOQblXbQp+/mi+n1TOO2W538XOv3I/n/fv6V+nLeedstyi/p20j/9q+Lp+4BXwK7+wH4ue",
    "r342/qN3snPQzXz8aPJXpxkyHpScevNpPML2y2b/gDz9CkvHs7o=",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c9_sliding_controller_const(SimStruct *S)
{
  const char* newstr = sf_c9_sliding_controller_const_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2501929482U));
  ssSetChecksum1(S,(2041323763U));
  ssSetChecksum2(S,(3808924622U));
  ssSetChecksum3(S,(1034168136U));
}

static void mdlRTW_c9_sliding_controller_const(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c9_sliding_controller_const(SimStruct *S)
{
  SFc9_sliding_controller_constInstanceStruct *chartInstance;
  chartInstance = (SFc9_sliding_controller_constInstanceStruct *)utMalloc(sizeof
    (SFc9_sliding_controller_constInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc9_sliding_controller_constInstanceStruct));
  chartInstance = new (chartInstance)
    SFc9_sliding_controller_constInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c9_sliding_controller_const;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c9_sliding_controller_const;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c9_sliding_controller_const;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c9_sliding_controller_const;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c9_sliding_controller_const;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c9_sliding_controller_const;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c9_sliding_controller_const;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c9_sliding_controller_const;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c9_sliding_controller_const;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c9_sliding_controller_const;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c9_sliding_controller_const;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c9_sliding_controller_const;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c9_JITStateAnimation,
    chartInstance->c9_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c9_sliding_controller_const(chartInstance);
}

void c9_sliding_controller_const_method_dispatcher(SimStruct *S, int_T method,
  void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c9_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c9_sliding_controller_const(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c9_sliding_controller_const(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c9_sliding_controller_const_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
