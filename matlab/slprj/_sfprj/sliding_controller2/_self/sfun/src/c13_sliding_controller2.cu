/* Include files */

#include "sliding_controller2_sfun.h"
#include "c13_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void enable_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void disable_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void c13_do_animation_call_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance, const mxArray *c13_st);
static void sf_gateway_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance);
static void c13_eML_blk_kernel(SFc13_sliding_controller2InstanceStruct
  *chartInstance, real_T c13_b_q[2], real_T c13_b_y[4]);
static void c13_emlrt_marshallIn(SFc13_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c13_b_y, const char_T *c13_identifier, real_T
  c13_c_y[4]);
static void c13_b_emlrt_marshallIn(SFc13_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId,
  real_T c13_b_y[4]);
static uint8_T c13_c_emlrt_marshallIn(SFc13_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c13_b_is_active_c13_sliding_controller2, const
  char_T *c13_identifier);
static uint8_T c13_d_emlrt_marshallIn(SFc13_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId);
static void init_dsm_address_info(SFc13_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc13_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c13_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c13_is_active_c13_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c13_do_animation_call_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c13_b_y = NULL;
  const mxArray *c13_c_y = NULL;
  const mxArray *c13_d_y = NULL;
  const mxArray *c13_st;
  c13_st = NULL;
  c13_b_y = NULL;
  sf_mex_assign(&c13_b_y, sf_mex_createcellmatrix(2, 1), false);
  c13_c_y = NULL;
  sf_mex_assign(&c13_c_y, sf_mex_create("y", *chartInstance->c13_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c13_b_y, 0, c13_c_y);
  c13_d_y = NULL;
  sf_mex_assign(&c13_d_y, sf_mex_create("y",
    &chartInstance->c13_is_active_c13_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c13_b_y, 1, c13_d_y);
  sf_mex_assign(&c13_st, c13_b_y, false);
  return c13_st;
}

static void set_sim_state_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance, const mxArray *c13_st)
{
  const mxArray *c13_u;
  c13_u = sf_mex_dup(c13_st);
  c13_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c13_u, 0)), "y",
                       *chartInstance->c13_y);
  chartInstance->c13_is_active_c13_sliding_controller2 = c13_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c13_u, 1)),
     "is_active_c13_sliding_controller2");
  sf_mex_destroy(&c13_u);
  sf_mex_destroy(&c13_st);
}

static void sf_gateway_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
  real_T c13_dv1[4];
  real_T c13_dv[2];
  int32_T c13_i;
  int32_T c13_i1;
  chartInstance->c13_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c13_i = 0; c13_i < 2; c13_i++) {
    c13_dv[c13_i] = (*chartInstance->c13_q)[c13_i];
  }

  c13_eML_blk_kernel(chartInstance, c13_dv, c13_dv1);
  for (c13_i1 = 0; c13_i1 < 4; c13_i1++) {
    (*chartInstance->c13_y)[c13_i1] = c13_dv1[c13_i1];
  }

  c13_do_animation_call_c13_sliding_controller2(chartInstance);
}

static void mdl_start_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c13_errCode;
  c13_errCode = hipGetLastError();
  if (c13_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c13_errCode, (char_T *)hipGetErrorName
                       (c13_errCode), (char_T *)hipGetErrorString(c13_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c13_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc13_sliding_controller2
  (SFc13_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c13_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c13_nameCaptureInfo = NULL;
  c13_nameCaptureInfo = NULL;
  sf_mex_assign(&c13_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c13_nameCaptureInfo;
}

static void c13_eML_blk_kernel(SFc13_sliding_controller2InstanceStruct
  *chartInstance, real_T c13_b_q[2], real_T c13_b_y[4])
{
  const mxArray *c13_c_y = NULL;
  c13_b_y[0] = -0.324 * muDoubleScalarCos(c13_b_q[0]) + -0.39 *
    muDoubleScalarCos(c13_b_q[0] + c13_b_q[1]);
  c13_b_y[2] = 0.0;
  c13_b_y[1] = 0.324 * muDoubleScalarSin(c13_b_q[0]) - -0.39 * muDoubleScalarSin
    (c13_b_q[0] + c13_b_q[1]);
  c13_b_y[3] = 0.0;
  sf_mex_printf("%s =\\n", "y");
  c13_c_y = NULL;
  sf_mex_assign(&c13_c_y, sf_mex_create("y", c13_b_y, 0, 0U, 1U, 0U, 2, 2, 2),
                false);
  sf_mex_call(chartInstance->c13_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c13_c_y);
}

static void c13_emlrt_marshallIn(SFc13_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c13_b_y, const char_T *c13_identifier, real_T
  c13_c_y[4])
{
  emlrtMsgIdentifier c13_thisId;
  c13_thisId.fIdentifier = const_cast<const char_T *>(c13_identifier);
  c13_thisId.fParent = NULL;
  c13_thisId.bParentIsCell = false;
  c13_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c13_b_y), &c13_thisId,
    c13_c_y);
  sf_mex_destroy(&c13_b_y);
}

static void c13_b_emlrt_marshallIn(SFc13_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId,
  real_T c13_b_y[4])
{
  real_T c13_dv[4];
  int32_T c13_i;
  sf_mex_import(c13_parentId, sf_mex_dup(c13_u), c13_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c13_i = 0; c13_i < 4; c13_i++) {
    c13_b_y[c13_i] = c13_dv[c13_i];
  }

  sf_mex_destroy(&c13_u);
}

static uint8_T c13_c_emlrt_marshallIn(SFc13_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c13_b_is_active_c13_sliding_controller2, const
  char_T *c13_identifier)
{
  emlrtMsgIdentifier c13_thisId;
  uint8_T c13_b_y;
  c13_thisId.fIdentifier = const_cast<const char_T *>(c13_identifier);
  c13_thisId.fParent = NULL;
  c13_thisId.bParentIsCell = false;
  c13_b_y = c13_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c13_b_is_active_c13_sliding_controller2), &c13_thisId);
  sf_mex_destroy(&c13_b_is_active_c13_sliding_controller2);
  return c13_b_y;
}

static uint8_T c13_d_emlrt_marshallIn(SFc13_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c13_u, const emlrtMsgIdentifier *c13_parentId)
{
  uint8_T c13_b_u;
  uint8_T c13_b_y;
  sf_mex_import(c13_parentId, sf_mex_dup(c13_u), &c13_b_u, 1, 3, 0U, 0, 0U, 0);
  c13_b_y = c13_b_u;
  sf_mex_destroy(&c13_u);
  return c13_b_y;
}

static void init_dsm_address_info(SFc13_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc13_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c13_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c13_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c13_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c13_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(4287055537U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1544704453U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1491530946U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3410447944U);
}

mxArray *sf_c13_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c13_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c13_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c13_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8cmGxvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABbHRvW"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c13_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "snQO6VnpiMl9TNTLTiXAGiG";
}

static void sf_opaque_initialize_c13_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c13_sliding_controller2
    ((SFc13_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c13_sliding_controller2((SFc13_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c13_sliding_controller2(void *chartInstanceVar)
{
  enable_c13_sliding_controller2((SFc13_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c13_sliding_controller2(void *chartInstanceVar)
{
  disable_c13_sliding_controller2((SFc13_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c13_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c13_sliding_controller2((SFc13_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c13_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c13_sliding_controller2
    ((SFc13_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c13_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c13_sliding_controller2((SFc13_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c13_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc13_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c13_sliding_controller2
      ((SFc13_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc13_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc13_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c13_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c13_sliding_controller2((SFc13_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c13_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c13_sliding_controller2((SFc13_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc13_sliding_controller2((SFc13_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c13_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c13_sliding_controller2
      ((SFc13_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc13_sliding_controller2
      ((SFc13_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c13_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV91u00gUdqOCqBZQhZDYC8TSq2UvkBaQENywbf5KpIQWnHa5q6bjk3iU8YyZn7R5CN6Ah9h",
    "9DSSkvdzH2BdYac/YTpp1bIcSUQHCkuuM/Z0z55zv/Ey9tU7Pw+s63ts3PO8yPq/gXfPS61K2Xp",
    "u70/fr3s/Z+i0KCRvtE0Ui7VVegkTwCrTk1jApOmIgC2FMDECBoIiNpTJl2jSLLGdi1LaCOn369",
    "5DR0A+l5UEdZUmwJ/gEtcXW7KOeJlNATRsgMKGSdhi2ORnOLFbmpBECHWkbVbmgwfg2dmbpnuWG",
    "xRxap0A7QhuCFusz23xDDDTMaambzlPtT4EyijkjotDbkGgfYgywgYM4wL971qBTeRgNiTJ1CMk",
    "YdJeNEp1SQF4n0/jhmAlipGKEtyLecIKLtu1ztKcnA+AVAUHb6grIKJZMmHL+/TZ62hLkmEMTju",
    "2wXJsPb6wj/5DBCajSuA0acgyKDGFPlG6aBKR1mrA1y5JFmGERHBK1Q5E/DUFp9mLmaJ8gT9BHi",
    "TIYJE52dF+xMYa3VJuNOi4zl5WMjVKy9TJYoq01hioWZtraVDQI57oU1pdxF8bAE61NYkg1LNVa",
    "jNOaBX2JAXbpXV4NVjAkPoM1pAhYIV3jHCDpOy+wsfwfSa02Mmpg8ja73cXPi7COMKAGhEJRF1C",
    "EacCYJeEt1xYw7bhHIFplEvOKwGmGLEN5emBF80SqEcakoomcueAYLQVGeohcYiUcaCyaKpjjch",
    "mOEhpC4BoM49DDskFsQUy0a207WHdjZiZN0FSxuIBVi1WHbajlEmoSw4EYCXki2kpGftbj0/ACY",
    "LYRJZgY1rF/qUkbdy+iws2zX72zeXb1I+bZVC7/vDenZ61Ajzf3zO+7Uavet4a/1qbzd07uWm6f",
    "9Zycw23iff/Du7/e//jvL5368Z+37/zQX2X/P2rnm//Xs/XtaaOdFc54IV8d9vmcXesF+m/N6d/",
    "M1lq83Ht8KGLW40/7L/rdPnu9s8t2U36W2FvL2Tt9f9d1fMyyJD8V7QTZwcStiU3HtdP/ZM7ey0",
    "visZG9T69/fltN/uZ2nsf1JfKb+GuSy9tP3//u9mry6f6vl9i/leN7K5nrR8R1CziiDx4dac4CL",
    "PUjKoVRknNQDwvq81Pz/bxy3gXLfS12fpf7/Lx/zPy6aDnvguVW9e+8c/xLw1fNAy+H3/yC/Vj1",
    "fPW58X975zsH/ZStn83+1WmEjAcFp97sMx5hB0Vfv4E8/Q/wzrFk",
    ""
  };

  static char newstr [1253] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c13_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c13_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3499937069U));
  ssSetChecksum1(S,(704452810U));
  ssSetChecksum2(S,(2976006729U));
  ssSetChecksum3(S,(1410014748U));
}

static void mdlRTW_c13_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c13_sliding_controller2(SimStruct *S)
{
  SFc13_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc13_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc13_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc13_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc13_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c13_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c13_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c13_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c13_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c13_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c13_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c13_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c13_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c13_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c13_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c13_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c13_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c13_JITStateAnimation,
    chartInstance->c13_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c13_sliding_controller2(chartInstance);
}

void c13_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c13_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c13_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c13_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c13_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
