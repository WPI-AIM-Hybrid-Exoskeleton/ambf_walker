#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c22_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void enable_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void disable_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void c22_do_animation_call_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance, const mxArray *c22_st);
static void sf_gateway_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void c22_eML_blk_kernel(SFc22_sliding_controller2InstanceStruct
  *chartInstance, real_T c22_b_u[6], real_T c22_b_y[7]);
static void c22_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_y, const char_T *c22_identifier, real_T
  c22_c_y[7]);
static void c22_b_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId,
  real_T c22_b_y[7]);
static uint8_T c22_c_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_is_active_c22_sliding_controller2, const
  char_T *c22_identifier);
static uint8_T c22_d_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId);
static __global__ void c22_eML_blk_kernel_kernel1(real_T c22_b_y[7]);
static void init_dsm_address_info(SFc22_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc22_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c22_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c22_is_active_c22_sliding_controller2 = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c22_gpu_y, 56UL);
}

static void initialize_params_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c22_do_animation_call_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c22_b_y = NULL;
  const mxArray *c22_c_y = NULL;
  const mxArray *c22_d_y = NULL;
  const mxArray *c22_st;
  c22_st = NULL;
  c22_b_y = NULL;
  sf_mex_assign(&c22_b_y, sf_mex_createcellmatrix(2, 1), false);
  c22_c_y = NULL;
  sf_mex_assign(&c22_c_y, sf_mex_create("y", *chartInstance->c22_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c22_b_y, 0, c22_c_y);
  c22_d_y = NULL;
  sf_mex_assign(&c22_d_y, sf_mex_create("y",
    &chartInstance->c22_is_active_c22_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c22_b_y, 1, c22_d_y);
  sf_mex_assign(&c22_st, c22_b_y, false);
  return c22_st;
}

static void set_sim_state_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance, const mxArray *c22_st)
{
  const mxArray *c22_b_u;
  c22_b_u = sf_mex_dup(c22_st);
  c22_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c22_b_u, 0)),
                       "y", *chartInstance->c22_y);
  chartInstance->c22_is_active_c22_sliding_controller2 = c22_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c22_b_u, 1)),
     "is_active_c22_sliding_controller2");
  sf_mex_destroy(&c22_b_u);
  sf_mex_destroy(&c22_st);
}

static void sf_gateway_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  real_T c22_dv1[7];
  real_T c22_dv[6];
  int32_T c22_i;
  int32_T c22_i1;
  chartInstance->c22_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c22_i = 0; c22_i < 6; c22_i++) {
    c22_dv[c22_i] = (*chartInstance->c22_u)[c22_i];
  }

  c22_eML_blk_kernel(chartInstance, c22_dv, c22_dv1);
  for (c22_i1 = 0; c22_i1 < 7; c22_i1++) {
    (*chartInstance->c22_y)[c22_i1] = c22_dv1[c22_i1];
  }

  c22_do_animation_call_c22_sliding_controller2(chartInstance);
}

static void mdl_start_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c22_errCode;
  hipFree(*chartInstance->c22_gpu_y);
  c22_errCode = hipGetLastError();
  if (c22_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c22_errCode, (char_T *)hipGetErrorName
                       (c22_errCode), (char_T *)hipGetErrorString(c22_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c22_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c22_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c22_nameCaptureInfo = NULL;
  c22_nameCaptureInfo = NULL;
  sf_mex_assign(&c22_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c22_nameCaptureInfo;
}

static void c22_eML_blk_kernel(SFc22_sliding_controller2InstanceStruct
  *chartInstance, real_T c22_b_u[6], real_T c22_b_y[7])
{
  const mxArray *c22_c_y = NULL;
  c22_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c22_gpu_y);
  hipMemcpy(c22_b_y, *chartInstance->c22_gpu_y, 56UL, hipMemcpyDeviceToHost);
  c22_b_y[0] = c22_b_u[3];
  c22_b_y[1] = 0.0;
  sf_mex_printf("%s =\\n", "reordered");
  c22_c_y = NULL;
  sf_mex_assign(&c22_c_y, sf_mex_create("y", c22_b_y, 0, 0U, 1U, 0U, 2, 1, 7),
                false);
  sf_mex_call(chartInstance->c22_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c22_c_y);
  c22_b_y[2] = c22_b_u[0];
  c22_b_y[3] = c22_b_u[1];
  c22_b_y[4] = c22_b_u[4];
  c22_b_y[5] = c22_b_u[5];
  c22_b_y[6] = c22_b_u[2];
}

static void c22_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_y, const char_T *c22_identifier, real_T
  c22_c_y[7])
{
  emlrtMsgIdentifier c22_thisId;
  c22_thisId.fIdentifier = const_cast<const char_T *>(c22_identifier);
  c22_thisId.fParent = NULL;
  c22_thisId.bParentIsCell = false;
  c22_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c22_b_y), &c22_thisId,
    c22_c_y);
  sf_mex_destroy(&c22_b_y);
}

static void c22_b_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId,
  real_T c22_b_y[7])
{
  real_T c22_dv[7];
  int32_T c22_i;
  sf_mex_import(c22_parentId, sf_mex_dup(c22_b_u), c22_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c22_i = 0; c22_i < 7; c22_i++) {
    c22_b_y[c22_i] = c22_dv[c22_i];
  }

  sf_mex_destroy(&c22_b_u);
}

static uint8_T c22_c_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_is_active_c22_sliding_controller2, const
  char_T *c22_identifier)
{
  emlrtMsgIdentifier c22_thisId;
  uint8_T c22_b_y;
  c22_thisId.fIdentifier = const_cast<const char_T *>(c22_identifier);
  c22_thisId.fParent = NULL;
  c22_thisId.bParentIsCell = false;
  c22_b_y = c22_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c22_b_is_active_c22_sliding_controller2), &c22_thisId);
  sf_mex_destroy(&c22_b_is_active_c22_sliding_controller2);
  return c22_b_y;
}

static uint8_T c22_d_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId)
{
  uint8_T c22_b_y;
  uint8_T c22_c_u;
  sf_mex_import(c22_parentId, sf_mex_dup(c22_b_u), &c22_c_u, 1, 3, 0U, 0, 0U, 0);
  c22_b_y = c22_c_u;
  sf_mex_destroy(&c22_b_u);
  return c22_b_y;
}

static __global__ __launch_bounds__(32, 1) void c22_eML_blk_kernel_kernel1
  (real_T c22_b_y[7])
{
  uint64_T c22_threadId;
  int32_T c22_i;
  c22_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c22_i = (int32_T)c22_threadId;
  if (c22_i < 7) {
    c22_b_y[c22_i] = 0.0;
  }
}

static void init_dsm_address_info(SFc22_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc22_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c22_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c22_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c22_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c22_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1104270009U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4002144469U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1451542329U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1457894201U);
}

mxArray *sf_c22_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c22_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c22_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c22_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8clGRvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABbHhvW"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c22_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "scU9zZ5H1kMhWDkDI28BpgD";
}

static void sf_opaque_initialize_c22_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c22_sliding_controller2
    ((SFc22_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c22_sliding_controller2(void *chartInstanceVar)
{
  enable_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c22_sliding_controller2(void *chartInstanceVar)
{
  disable_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c22_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c22_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c22_sliding_controller2
    ((SFc22_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c22_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c22_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc22_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c22_sliding_controller2
      ((SFc22_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc22_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc22_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c22_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c22_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c22_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c22_sliding_controller2
      ((SFc22_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc22_sliding_controller2
      ((SFc22_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c22_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgU3iIEkMIoGvQRIckqPrQAX6aW1JUqwACs2SjsuejHWy5G40HKX3R/ZyisEuRS",
    "59JBnSJ6hQN6hl75DbwV66ixJySpFUnEEG25RAjS15DezM/PNz9pb6/U9vO7hvf2p593C5228G1",
    "52fZKv1+bu7P269yRfv0IhYeMDokisvdpLkBi+By25NUyKnhjIUhgTA1AgKGITqUyVNs1iy5kYd",
    "a2gTp8+jhiNgkhaHrZQloT7gk9QW2LNAerxmQJqugChiZS0w6jLyXBmsTJn7QjoSNu4zgUNJrCJ",
    "M0v3LTcs4dA5B9oT2hC0WF/YFhhioG3OK910nupgCpRxwhkRpd5GRAeQYIANHCUh/t23Bp0qwmh",
    "ElGlBRMag99go1SkFFHUyjR9OmSBGKkZ4J+ZtJ7ho2wFHe/oyBF4TELStpYCMEsmEqeY/6KKnHU",
    "FOOfhwaofV2gL4yTrynzM4A1UZt0FbjkGRIeyLyk3TgHTOU7ZmWbIIMyyG50TtUORPQ1iZvZg5O",
    "iDIExyiRBUMUid7+lCxMYa3UpuNey4zl5WMjTOy9TJYqq0zhjoWZtq6VLQJ57oSdiiTPRgDT7X6",
    "xJB6WKa1HKc1Cw8lBtild3U1WMGQ+BzWliJkpXSNC4C07zzDxvJPJLXayLiNyevv7S1+XoT1hAE",
    "1IBTKuoAiTAPGLA1vtbaQacc9AtEqk5pXBs4yZBnK0wMr/DOpRhiTmiZy4YJjtBIY6yFyiZVwpL",
    "Fo6mCOy2U4SmgEoWswjEMfywaxJTHRrrXtYN2NmZn4oKliSQmrFqsO21DHJdQkgSMxEvJMdJWMg",
    "7zHZ+EFwGwjSjAxbGH/UpMu7l5GhZtnX3oX8+zOB8yzqVzx+cWcnrUSPd7cs7jvRqN+3wb+WpvO",
    "3zm5u4V91gtyDreJ9/2/Xr7+8/Wbt+9//nrr91/e/7rK/u8al5v/9/L1g2mjnRXOeCFfHXZ3zq7",
    "1Ev2fz+nfzNeaHn3z4set3a9G/ejYH/m95tNWMvQzfpbY2yjYO33/yHV8zLI0PxXthfnBxK2Jzc",
    "a10/90zt5bS+Kxkb/Prj++W03+s+0ij+tL5Dfx16SQtx+//6Pt1eSz/X9YYv/jAt+P07l+Qly3g",
    "BPabJ5ozkIs9RMqhVGSc1DNkvr82Hy/rJx3zXL/Fjv/l7t63j9kfl23nHfNcqv6d9k5ftPwdfPA",
    "K+A3b7Afq56vrhr/m3e5c9DDfP3t7F+ddsR4WHLqzT/jEXZQ9vU/kKd/A13WtA4=",
    ""
  };

  static char newstr [1265] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c22_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c22_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2492201493U));
  ssSetChecksum1(S,(2946340087U));
  ssSetChecksum2(S,(892768965U));
  ssSetChecksum3(S,(3267729885U));
}

static void mdlRTW_c22_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c22_sliding_controller2(SimStruct *S)
{
  SFc22_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc22_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc22_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc22_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc22_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c22_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c22_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c22_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c22_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c22_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c22_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c22_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c22_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c22_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c22_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c22_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c22_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c22_JITStateAnimation,
    chartInstance->c22_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c22_sliding_controller2(chartInstance);
}

void c22_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c22_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c22_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c22_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c22_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
