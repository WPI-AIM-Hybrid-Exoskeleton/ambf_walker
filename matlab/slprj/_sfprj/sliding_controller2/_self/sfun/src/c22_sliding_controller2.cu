/* Include files */

#include "sliding_controller2_sfun.h"
#include "c22_sliding_controller2.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void enable_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void disable_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void c22_do_animation_call_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance, const mxArray *c22_st);
static void sf_gateway_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance);
static void c22_eML_blk_kernel(SFc22_sliding_controller2InstanceStruct
  *chartInstance, real_T c22_b_u[7], real_T c22_b_y[7]);
static void c22_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_y, const char_T *c22_identifier, real_T
  c22_c_y[7]);
static void c22_b_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId,
  real_T c22_b_y[7]);
static uint8_T c22_c_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_is_active_c22_sliding_controller2, const
  char_T *c22_identifier);
static uint8_T c22_d_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId);
static void init_dsm_address_info(SFc22_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc22_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c22_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c22_is_active_c22_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c22_do_animation_call_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c22_b_y = NULL;
  const mxArray *c22_c_y = NULL;
  const mxArray *c22_d_y = NULL;
  const mxArray *c22_st;
  c22_st = NULL;
  c22_b_y = NULL;
  sf_mex_assign(&c22_b_y, sf_mex_createcellmatrix(2, 1), false);
  c22_c_y = NULL;
  sf_mex_assign(&c22_c_y, sf_mex_create("y", *chartInstance->c22_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c22_b_y, 0, c22_c_y);
  c22_d_y = NULL;
  sf_mex_assign(&c22_d_y, sf_mex_create("y",
    &chartInstance->c22_is_active_c22_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c22_b_y, 1, c22_d_y);
  sf_mex_assign(&c22_st, c22_b_y, false);
  return c22_st;
}

static void set_sim_state_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance, const mxArray *c22_st)
{
  const mxArray *c22_b_u;
  c22_b_u = sf_mex_dup(c22_st);
  c22_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c22_b_u, 0)),
                       "y", *chartInstance->c22_y);
  chartInstance->c22_is_active_c22_sliding_controller2 = c22_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c22_b_u, 1)),
     "is_active_c22_sliding_controller2");
  sf_mex_destroy(&c22_b_u);
  sf_mex_destroy(&c22_st);
}

static void sf_gateway_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c22_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c22_eML_blk_kernel(chartInstance, *chartInstance->c22_u, *chartInstance->c22_y);
  c22_do_animation_call_c22_sliding_controller2(chartInstance);
}

static void mdl_start_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c22_errCode;
  c22_errCode = hipGetLastError();
  if (c22_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c22_errCode, (char_T *)hipGetErrorName
                       (c22_errCode), (char_T *)hipGetErrorString(c22_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c22_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc22_sliding_controller2
  (SFc22_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c22_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c22_nameCaptureInfo = NULL;
  c22_nameCaptureInfo = NULL;
  sf_mex_assign(&c22_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c22_nameCaptureInfo;
}

static void c22_eML_blk_kernel(SFc22_sliding_controller2InstanceStruct
  *chartInstance, real_T c22_b_u[7], real_T c22_b_y[7])
{
  c22_b_y[0] = c22_b_u[3];
  c22_b_y[1] = c22_b_u[6];
  c22_b_y[2] = c22_b_u[0];
  c22_b_y[3] = c22_b_u[1];
  c22_b_y[4] = c22_b_u[4];
  c22_b_y[5] = c22_b_u[5];
  c22_b_y[6] = c22_b_u[2];
}

static void c22_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_y, const char_T *c22_identifier, real_T
  c22_c_y[7])
{
  emlrtMsgIdentifier c22_thisId;
  c22_thisId.fIdentifier = const_cast<const char_T *>(c22_identifier);
  c22_thisId.fParent = NULL;
  c22_thisId.bParentIsCell = false;
  c22_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c22_b_y), &c22_thisId,
    c22_c_y);
  sf_mex_destroy(&c22_b_y);
}

static void c22_b_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId,
  real_T c22_b_y[7])
{
  real_T c22_dv[7];
  int32_T c22_i;
  sf_mex_import(c22_parentId, sf_mex_dup(c22_b_u), c22_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c22_i = 0; c22_i < 7; c22_i++) {
    c22_b_y[c22_i] = c22_dv[c22_i];
  }

  sf_mex_destroy(&c22_b_u);
}

static uint8_T c22_c_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_is_active_c22_sliding_controller2, const
  char_T *c22_identifier)
{
  emlrtMsgIdentifier c22_thisId;
  uint8_T c22_b_y;
  c22_thisId.fIdentifier = const_cast<const char_T *>(c22_identifier);
  c22_thisId.fParent = NULL;
  c22_thisId.bParentIsCell = false;
  c22_b_y = c22_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c22_b_is_active_c22_sliding_controller2), &c22_thisId);
  sf_mex_destroy(&c22_b_is_active_c22_sliding_controller2);
  return c22_b_y;
}

static uint8_T c22_d_emlrt_marshallIn(SFc22_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c22_b_u, const emlrtMsgIdentifier *c22_parentId)
{
  uint8_T c22_b_y;
  uint8_T c22_c_u;
  sf_mex_import(c22_parentId, sf_mex_dup(c22_b_u), &c22_c_u, 1, 3, 0U, 0, 0U, 0);
  c22_b_y = c22_c_u;
  sf_mex_destroy(&c22_b_u);
  return c22_b_y;
}

static void init_dsm_address_info(SFc22_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc22_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c22_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c22_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c22_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c22_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3007509169U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1926089682U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2239166282U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3770191902U);
}

mxArray *sf_c22_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c22_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c22_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c22_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8clGRvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABbHhvW"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c22_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sdegTANPF1GH9p3UVy5QBRH";
}

static void sf_opaque_initialize_c22_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c22_sliding_controller2
    ((SFc22_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c22_sliding_controller2(void *chartInstanceVar)
{
  enable_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c22_sliding_controller2(void *chartInstanceVar)
{
  disable_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c22_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c22_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c22_sliding_controller2
    ((SFc22_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c22_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c22_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc22_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c22_sliding_controller2
      ((SFc22_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc22_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc22_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c22_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c22_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc22_sliding_controller2((SFc22_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c22_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c22_sliding_controller2
      ((SFc22_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc22_sliding_controller2
      ((SFc22_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c22_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgXXaIDUMIoAvRSJfSjaW1sHBZJLausvFmDHbijbvQTGejkSF1ruMvsjW+cciz5",
    "FXiCXvEueIsilvRXoLEnJKkVStgUbblECNLXkN7Mz883P2lvq7Hl4reK99aXnreDzc7xrXnp9lq",
    "2Xpu70/bL3bbb+HYWEjQ6IIpH2Ki9BIngJWnJrmBQd0ZOFMCZ6oEBQxMZSmTJtmkWWMzFoW0GdP",
    "n0cMhr6obQ8qKMsCfYFH6G22JoD1NNkCqhpAwQmVNL2wzYn/YnFypw1QqADbaMqFzQY38bOLL1n",
    "uWExh9Y50I7QhqDF+sI23xADDXNe6qbzVPtjoIxizogo9DYk2ocYA2zgMA7w77416FQeRkOiTB1",
    "CMgS9ywaJTikgr5Np/HDKBDFSMcJbEW84wVnbDjjasycD4BUBQdvqCsgglkyYcv79NnraEuSUQx",
    "NObb9cmw+vrSP/iMEZqNK49RpyCIr0YV+UbpoEpHWesDXJklmYYREcEbVNkT8NQWn2YuZonyBP0",
    "EWJMhgkTnZ0V7EhhrdUm406LjPnlYyNUrL1PFiirTWEKhYm2tpUNAjnuhTWlfEuDIEnWpvEkGpY",
    "qrUYpzULuhID7NK7vBqsYEh8BmtIEbBCuoY5QNJ3XmBj+SeSWm1k1MDkbe7uzn6ehXWEAdUjFIq",
    "6gCJMA8YsCW+5toBpxz0C0SqTmFcETjNkHsrTPSuaZ1INMCYVTeTCBcdoKTDSfeQSK+FQY9FUwR",
    "yX83CU0BAC12AYhz0sG8QWxES71raNdTdkZtQETRWLC1i1WHXYhlouoUYxHIqBkGeirWTkZz0+D",
    "S8AZhtRgol+HfuXGrVx9yIq3Dz7wbuYZ/cvMc/Gcvnnd1N6lgr0eFPP/L73atX71vDX0nj+Tsl9",
    "kdtnOSfncGt4//Xg+P1v37/5Y/UtefXxz28eLrL/u9rV5v9qtv563GgnhTOcyVeH3Zmya7lA/1d",
    "T+teytQ6g391+cdD+8fnO0/jx4dHop1/qL3dSfubYW8vZO36/7jo+ZlmSn4p2guxg4tbEpuPa6X",
    "8yZe/KnHjcy96n16efF5N/sJXncXmO/Br+GuXy9vr7r28tJp/u/+sc+zdyfG8kc/2EuG4BJ3Rz8",
    "0RzFmCpn1ApjJKcg9osqM/r5vtV5bxblvu32Pm/3M3zfpn5ddty3i3LLerfVef4XcNXzQMvh1+7",
    "w34ser66afwH72rnoEfZ+tnkX51GyHhQcOrNPuMRtlf09T+Qp38DZAOyDQ==",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c22_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c22_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042383101U));
  ssSetChecksum1(S,(4119015307U));
  ssSetChecksum2(S,(1566679822U));
  ssSetChecksum3(S,(505804526U));
}

static void mdlRTW_c22_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c22_sliding_controller2(SimStruct *S)
{
  SFc22_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc22_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc22_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc22_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc22_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c22_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c22_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c22_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c22_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c22_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c22_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c22_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c22_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c22_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c22_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c22_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c22_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c22_JITStateAnimation,
    chartInstance->c22_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c22_sliding_controller2(chartInstance);
}

void c22_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c22_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c22_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c22_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c22_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
