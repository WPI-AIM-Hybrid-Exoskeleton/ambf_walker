/* Include files */

#include "sliding_controller2_sfun.h"
#include "c11_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void enable_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void disable_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void c11_do_animation_call_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance, const mxArray *c11_st);
static void sf_gateway_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance);
static void c11_eML_blk_kernel(SFc11_sliding_controller2InstanceStruct
  *chartInstance, real_T c11_b_q[2], real_T c11_b_y[4]);
static void c11_emlrt_marshallIn(SFc11_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c11_b_y, const char_T *c11_identifier, real_T
  c11_c_y[4]);
static void c11_b_emlrt_marshallIn(SFc11_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId,
  real_T c11_b_y[4]);
static uint8_T c11_c_emlrt_marshallIn(SFc11_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c11_b_is_active_c11_sliding_controller2, const
  char_T *c11_identifier);
static uint8_T c11_d_emlrt_marshallIn(SFc11_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId);
static void init_dsm_address_info(SFc11_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc11_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c11_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c11_is_active_c11_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c11_do_animation_call_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c11_b_y = NULL;
  const mxArray *c11_c_y = NULL;
  const mxArray *c11_d_y = NULL;
  const mxArray *c11_st;
  c11_st = NULL;
  c11_b_y = NULL;
  sf_mex_assign(&c11_b_y, sf_mex_createcellmatrix(2, 1), false);
  c11_c_y = NULL;
  sf_mex_assign(&c11_c_y, sf_mex_create("y", *chartInstance->c11_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c11_b_y, 0, c11_c_y);
  c11_d_y = NULL;
  sf_mex_assign(&c11_d_y, sf_mex_create("y",
    &chartInstance->c11_is_active_c11_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c11_b_y, 1, c11_d_y);
  sf_mex_assign(&c11_st, c11_b_y, false);
  return c11_st;
}

static void set_sim_state_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance, const mxArray *c11_st)
{
  const mxArray *c11_u;
  c11_u = sf_mex_dup(c11_st);
  c11_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c11_u, 0)), "y",
                       *chartInstance->c11_y);
  chartInstance->c11_is_active_c11_sliding_controller2 = c11_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c11_u, 1)),
     "is_active_c11_sliding_controller2");
  sf_mex_destroy(&c11_u);
  sf_mex_destroy(&c11_st);
}

static void sf_gateway_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c11_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c11_eML_blk_kernel(chartInstance, *chartInstance->c11_q, *chartInstance->c11_y);
  c11_do_animation_call_c11_sliding_controller2(chartInstance);
}

static void mdl_start_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c11_errCode;
  c11_errCode = hipGetLastError();
  if (c11_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c11_errCode, (char_T *)hipGetErrorName
                       (c11_errCode), (char_T *)hipGetErrorString(c11_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c11_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc11_sliding_controller2
  (SFc11_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c11_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c11_nameCaptureInfo = NULL;
  c11_nameCaptureInfo = NULL;
  sf_mex_assign(&c11_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c11_nameCaptureInfo;
}

static void c11_eML_blk_kernel(SFc11_sliding_controller2InstanceStruct
  *chartInstance, real_T c11_b_q[2], real_T c11_b_y[4])
{
  c11_b_y[0] = -0.324 * muDoubleScalarSin(c11_b_q[0]);
  c11_b_y[2] = 0.0;
  c11_b_y[1] = 0.324 * muDoubleScalarCos(c11_b_q[0]);
  c11_b_y[3] = 0.0;
}

static void c11_emlrt_marshallIn(SFc11_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c11_b_y, const char_T *c11_identifier, real_T
  c11_c_y[4])
{
  emlrtMsgIdentifier c11_thisId;
  c11_thisId.fIdentifier = const_cast<const char_T *>(c11_identifier);
  c11_thisId.fParent = NULL;
  c11_thisId.bParentIsCell = false;
  c11_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c11_b_y), &c11_thisId,
    c11_c_y);
  sf_mex_destroy(&c11_b_y);
}

static void c11_b_emlrt_marshallIn(SFc11_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId,
  real_T c11_b_y[4])
{
  real_T c11_dv[4];
  int32_T c11_i;
  sf_mex_import(c11_parentId, sf_mex_dup(c11_u), c11_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c11_i = 0; c11_i < 4; c11_i++) {
    c11_b_y[c11_i] = c11_dv[c11_i];
  }

  sf_mex_destroy(&c11_u);
}

static uint8_T c11_c_emlrt_marshallIn(SFc11_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c11_b_is_active_c11_sliding_controller2, const
  char_T *c11_identifier)
{
  emlrtMsgIdentifier c11_thisId;
  uint8_T c11_b_y;
  c11_thisId.fIdentifier = const_cast<const char_T *>(c11_identifier);
  c11_thisId.fParent = NULL;
  c11_thisId.bParentIsCell = false;
  c11_b_y = c11_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c11_b_is_active_c11_sliding_controller2), &c11_thisId);
  sf_mex_destroy(&c11_b_is_active_c11_sliding_controller2);
  return c11_b_y;
}

static uint8_T c11_d_emlrt_marshallIn(SFc11_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c11_u, const emlrtMsgIdentifier *c11_parentId)
{
  uint8_T c11_b_u;
  uint8_T c11_b_y;
  sf_mex_import(c11_parentId, sf_mex_dup(c11_u), &c11_b_u, 1, 3, 0U, 0, 0U, 0);
  c11_b_y = c11_b_u;
  sf_mex_destroy(&c11_u);
  return c11_b_y;
}

static void init_dsm_address_info(SFc11_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc11_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c11_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c11_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c11_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c11_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(536815311U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1945133011U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2900394615U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(32238838U);
}

mxArray *sf_c11_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c11_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c11_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c11_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8cmGhvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABa1RvU"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c11_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sFqlSox9r5ekjZXT2w93GAD";
}

static void sf_opaque_initialize_c11_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c11_sliding_controller2
    ((SFc11_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c11_sliding_controller2((SFc11_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c11_sliding_controller2(void *chartInstanceVar)
{
  enable_c11_sliding_controller2((SFc11_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c11_sliding_controller2(void *chartInstanceVar)
{
  disable_c11_sliding_controller2((SFc11_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c11_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c11_sliding_controller2((SFc11_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c11_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c11_sliding_controller2
    ((SFc11_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c11_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c11_sliding_controller2((SFc11_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c11_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc11_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c11_sliding_controller2
      ((SFc11_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc11_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc11_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c11_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c11_sliding_controller2((SFc11_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c11_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c11_sliding_controller2((SFc11_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc11_sliding_controller2((SFc11_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c11_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c11_sliding_controller2
      ((SFc11_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc11_sliding_controller2
      ((SFc11_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c11_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV0tv00AQdqOCQDxUISQkhASc4MhDSHCBtnlAREornALiUm3Xk3jJetfsI2n+ADf+CQfu/BF",
    "+ARcu/ARmbScNju1QIipAWHKdtb+ZnZlvHltvpb3l4XUe7/ULnncSn6fwrnnpdSJbr8zc6ftV70",
    "a2fo9CwkY7RJFIe5WXIBE8By25NUyKtujJQhgTPVAgKGJjqUyZNs0iy5kYtKygTp9+GTIa+qG0P",
    "NhEWRJsCz5GbbE1O6inwRRQ0wIITKik7YctTvpTi5UZ1UOgA22jKhc0GN/Gziy9ZblhMYfmAdC2",
    "0IagxfrQNt8QA3VzUOqm81T7E6CMYs6IKPQ2JNqHGANsYDcO8O+2NehUHkZDoswmhGQIusMGiU4",
    "pIK+TafywzwQxUjHCmxGvO8F523Y42rMlA+AVAUHbNhWQQSyZMOX8+y30tCnIPocG7Nt+uTYf3l",
    "pH/gsGI1ClcevV5RAU6cO2KN00CUjzIGFrmiXzMMMieEHUBkX+NASl2YuZo32CPEEXJcpgkDjZ1",
    "l3FhhjeUm02arvMXFQyNkrJ1otgibbmEKpYmGprUVEnnOtSWFfGHRgCT7Q2iCHVsFRrMU5rFnQl",
    "Btild3k1WMGQ+AxWlyJghXQNc4Ck7zzDxvIjklptZFTH5G10OvOf52FtYUD1CIWiLqAI04AxS8J",
    "bri1g2nGPQLTKJOYVgdMMWYTydM+KxkiqAcakookcuuAYLQVGuo9cYiXsaiyaKpjjchGOEhpC4B",
    "oM47CFZYPYgpho19o2sO6GzIwboKlicQGrFqsO21DTJdQ4hl0xEHIkWkpGftbj0/ACYLYRJZjob",
    "2L/UuMW7l5EhZtnt7zDeXb2J+bZRC7/vDmjZ6VAjzfzzO97ula9bw1/rUzm74zcudw+qzk5h1vD",
    "+/KHe18+3dUn+JmnnXdfX3aW2f9j7Wjz/3y2vjJptNPCGc7lq8M+mbFrtUD/pRn9a9lat95yXx4",
    "8UPdg8Ob1q+6d0YO7jzcaKT8L7K3l7J28v+Y6PmZZkp+KtoPsYOLWxKbj2um/P2PvyQXxOJ29T6",
    "9vj5aTv7ie53F1gfwa/hrn8vbX97+2vpx8uv+rBfZfz/F9PZnre8R1C9ijt2/vac4CLPU9KoVRk",
    "nNQdwrq81fz/ahy3jHL/S12/pf7/bz/zPw6bjnvmOWW9e+oc/xPw1fNAy+HX/uD/Vj2fPW78Z+9",
    "o52Drmbrh9N/deoh40HBqTf7jEfYXtHXfyBPvwM8mrEi",
    ""
  };

  static char newstr [1245] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c11_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c11_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3862277146U));
  ssSetChecksum1(S,(91435963U));
  ssSetChecksum2(S,(1279986540U));
  ssSetChecksum3(S,(1280830342U));
}

static void mdlRTW_c11_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c11_sliding_controller2(SimStruct *S)
{
  SFc11_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc11_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc11_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc11_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc11_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c11_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c11_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c11_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c11_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c11_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c11_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c11_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c11_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c11_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c11_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c11_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c11_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c11_JITStateAnimation,
    chartInstance->c11_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c11_sliding_controller2(chartInstance);
}

void c11_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c11_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c11_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c11_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c11_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
