#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c2_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void enable_c2_sliding_controller2(SFc2_sliding_controller2InstanceStruct
  *chartInstance);
static void disable_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void c2_do_animation_call_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance, const mxArray *c2_st);
static void sf_gateway_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance);
static void c2_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_b_msg, const char_T *c2_identifier,
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_y);
static void c2_b_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_y);
static void c2_c_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  uint8_T c2_y[4]);
static c2_SL_Bus_ROSVariableLengthArrayInfo c2_d_emlrt_marshallIn
  (SFc2_sliding_controller2InstanceStruct *chartInstance, const mxArray *c2_u,
   const emlrtMsgIdentifier *c2_parentId);
static uint32_T c2_e_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static void c2_f_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real_T c2_y[7]);
static uint8_T c2_g_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_sliding_controller2, const
  char_T *c2_identifier);
static uint8_T c2_h_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static void c2_eML_blk_kernel(SFc2_sliding_controller2InstanceStruct
  *chartInstance, c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex
  *c2_b_blankMsg);
static __global__ void c2_sf_gateway_c2_sliding_controller2_kernel1
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg);
static __global__ void c2_sf_gateway_c2_sliding_controller2_kernel2
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg);
static __global__ void c2_sf_gateway_c2_sliding_controller2_kernel3
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg,
   c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_r);
static __global__ void c2_sf_gateway_c2_sliding_controller2_kernel4
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg,
   c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_r);
static __global__ void c2_sf_gateway_c2_sliding_controller2_kernel5(const
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_r,
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg);
static __global__ void c2_sf_gateway_c2_sliding_controller2_kernel6(const
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_r,
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg);
static __global__ void c2_eML_blk_kernel_kernel7
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg);
static __global__ void c2_eML_blk_kernel_kernel8(const uint8_T c2_uv[3],
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg);
static __global__ void c2_eML_blk_kernel_kernel9
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg);
static void init_dsm_address_info(SFc2_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc2_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c2_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c2_is_active_c2_sliding_controller2 = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c2_gpu_uv, 3UL);
  hipMalloc(&chartInstance->c2_gpu_blankMsg, 208UL);
}

static void initialize_params_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c2_sliding_controller2(SFc2_sliding_controller2InstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c2_do_animation_call_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
  static const char_T *c2_sv[8] = { "ModelName", "ModelName_SL_Info", "Q",
    "Q_SL_Info", "Qd", "Qd_SL_Info", "Qdd", "Qdd_SL_Info" };

  static const char_T *c2_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c2_b_y = NULL;
  const mxArray *c2_c_y = NULL;
  const mxArray *c2_d_y = NULL;
  const mxArray *c2_e_y = NULL;
  const mxArray *c2_f_y = NULL;
  const mxArray *c2_g_y = NULL;
  const mxArray *c2_h_y = NULL;
  const mxArray *c2_i_y = NULL;
  const mxArray *c2_j_y = NULL;
  const mxArray *c2_k_y = NULL;
  const mxArray *c2_l_y = NULL;
  const mxArray *c2_m_y = NULL;
  const mxArray *c2_n_y = NULL;
  const mxArray *c2_o_y = NULL;
  const mxArray *c2_p_y = NULL;
  const mxArray *c2_q_y = NULL;
  const mxArray *c2_r_y = NULL;
  const mxArray *c2_s_y = NULL;
  const mxArray *c2_st;
  const mxArray *c2_y = NULL;
  real_T c2_d_u[7];
  int32_T c2_i;
  int32_T c2_i1;
  int32_T c2_i2;
  int32_T c2_i3;
  uint32_T c2_b_u;
  uint32_T c2_c_u;
  uint32_T c2_e_u;
  uint32_T c2_f_u;
  uint32_T c2_g_u;
  uint32_T c2_h_u;
  uint32_T c2_i_u;
  uint32_T c2_j_u;
  uint32_T c2_u_CurrentLength;
  uint32_T c2_u_ReceivedLength;
  uint8_T c2_u[4];
  c2_st = NULL;
  c2_y = NULL;
  sf_mex_assign(&c2_y, sf_mex_createcellmatrix(2, 1), false);
  c2_b_y = NULL;
  sf_mex_assign(&c2_b_y, sf_mex_createstruct("structure", 8, c2_sv, 2, 1, 1),
                false);
  for (c2_i = 0; c2_i < 4; c2_i++) {
    c2_u[c2_i] = ((uint8_T *)&((char_T *)chartInstance->c2_msg)[0])[c2_i];
  }

  c2_c_y = NULL;
  sf_mex_assign(&c2_c_y, sf_mex_create("y", c2_u, 3, 0U, 1U, 0U, 1, 4), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "ModelName", c2_c_y, 0);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [4];
  c2_d_y = NULL;
  sf_mex_assign(&c2_d_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_b_u = c2_u_CurrentLength;
  c2_e_y = NULL;
  sf_mex_assign(&c2_e_y, sf_mex_create("y", &c2_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_d_y, 0, "CurrentLength", c2_e_y, 0);
  c2_c_u = c2_u_ReceivedLength;
  c2_f_y = NULL;
  sf_mex_assign(&c2_f_y, sf_mex_create("y", &c2_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_d_y, 0, "ReceivedLength", c2_f_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "ModelName_SL_Info", c2_d_y, 1);
  for (c2_i1 = 0; c2_i1 < 7; c2_i1++) {
    c2_d_u[c2_i1] = ((real_T *)&((char_T *)chartInstance->c2_msg)[16])[c2_i1];
  }

  c2_g_y = NULL;
  sf_mex_assign(&c2_g_y, sf_mex_create("y", c2_d_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "Q", c2_g_y, 2);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[4];
  c2_h_y = NULL;
  sf_mex_assign(&c2_h_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_e_u = c2_u_CurrentLength;
  c2_i_y = NULL;
  sf_mex_assign(&c2_i_y, sf_mex_create("y", &c2_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_h_y, 0, "CurrentLength", c2_i_y, 0);
  c2_f_u = c2_u_ReceivedLength;
  c2_j_y = NULL;
  sf_mex_assign(&c2_j_y, sf_mex_create("y", &c2_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_h_y, 0, "ReceivedLength", c2_j_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "Q_SL_Info", c2_h_y, 3);
  for (c2_i2 = 0; c2_i2 < 7; c2_i2++) {
    c2_d_u[c2_i2] = ((real_T *)&((char_T *)chartInstance->c2_msg)[80])[c2_i2];
  }

  c2_k_y = NULL;
  sf_mex_assign(&c2_k_y, sf_mex_create("y", c2_d_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qd", c2_k_y, 4);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[4];
  c2_l_y = NULL;
  sf_mex_assign(&c2_l_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_g_u = c2_u_CurrentLength;
  c2_m_y = NULL;
  sf_mex_assign(&c2_m_y, sf_mex_create("y", &c2_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_l_y, 0, "CurrentLength", c2_m_y, 0);
  c2_h_u = c2_u_ReceivedLength;
  c2_n_y = NULL;
  sf_mex_assign(&c2_n_y, sf_mex_create("y", &c2_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_l_y, 0, "ReceivedLength", c2_n_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qd_SL_Info", c2_l_y, 5);
  for (c2_i3 = 0; c2_i3 < 7; c2_i3++) {
    c2_d_u[c2_i3] = ((real_T *)&((char_T *)chartInstance->c2_msg)[144])[c2_i3];
  }

  c2_o_y = NULL;
  sf_mex_assign(&c2_o_y, sf_mex_create("y", c2_d_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qdd", c2_o_y, 6);
  c2_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[0];
  c2_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[4];
  c2_p_y = NULL;
  sf_mex_assign(&c2_p_y, sf_mex_createstruct("structure", 2, c2_sv1, 2, 1, 1),
                false);
  c2_i_u = c2_u_CurrentLength;
  c2_q_y = NULL;
  sf_mex_assign(&c2_q_y, sf_mex_create("y", &c2_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_p_y, 0, "CurrentLength", c2_q_y, 0);
  c2_j_u = c2_u_ReceivedLength;
  c2_r_y = NULL;
  sf_mex_assign(&c2_r_y, sf_mex_create("y", &c2_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c2_p_y, 0, "ReceivedLength", c2_r_y, 1);
  sf_mex_setfieldbynum(c2_b_y, 0, "Qdd_SL_Info", c2_p_y, 7);
  sf_mex_setcell(c2_y, 0, c2_b_y);
  c2_s_y = NULL;
  sf_mex_assign(&c2_s_y, sf_mex_create("y",
    &chartInstance->c2_is_active_c2_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c2_y, 1, c2_s_y);
  sf_mex_assign(&c2_st, c2_y, false);
  return c2_st;
}

static void set_sim_state_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance, const mxArray *c2_st)
{
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex c2_r;
  const mxArray *c2_u;
  int32_T c2_i;
  int32_T c2_i1;
  c2_u = sf_mex_dup(c2_st);
  c2_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 0)), "msg",
                      &c2_r);
  for (c2_i = 0; c2_i < 4; c2_i++) {
    ((uint8_T *)&((char_T *)chartInstance->c2_msg)[0])[c2_i] =
      c2_r.ModelName[c2_i];
  }

  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[0] = c2_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[4] = c2_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[0] = c2_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[4] = c2_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[0] = c2_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[4] = c2_r.Qd_SL_Info.ReceivedLength;
  for (c2_i1 = 0; c2_i1 < 7; c2_i1++) {
    ((real_T *)&((char_T *)chartInstance->c2_msg)[16])[c2_i1] = c2_r.Q[c2_i1];
    ((real_T *)&((char_T *)chartInstance->c2_msg)[80])[c2_i1] = c2_r.Qd[c2_i1];
    ((real_T *)&((char_T *)chartInstance->c2_msg)[144])[c2_i1] = c2_r.Qdd[c2_i1];
  }

  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[0] = c2_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[4] = c2_r.Qdd_SL_Info.ReceivedLength;
  chartInstance->c2_is_active_c2_sliding_controller2 = c2_g_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 1)),
     "is_active_c2_sliding_controller2");
  sf_mex_destroy(&c2_u);
  sf_mex_destroy(&c2_st);
}

static void sf_gateway_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex c2_r;
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex
    *c2_b_gpu_blankMsg;
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_gpu_msg;
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_gpu_r;
  hipMalloc(&c2_gpu_r, 208UL);
  hipMalloc(&c2_gpu_msg, 208UL);
  hipMalloc(&c2_b_gpu_blankMsg, 208UL);
  chartInstance->c2_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  hipMemcpy(c2_b_gpu_blankMsg, chartInstance->c2_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller2_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U,
    1U)>>>(c2_b_gpu_blankMsg, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c2_blankMsg, c2_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[8])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[8])[4];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[72])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[72])[4];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[136])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[136])[4];
  hipMemcpy(c2_b_gpu_blankMsg, chartInstance->c2_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller2_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U,
    1U)>>>(c2_b_gpu_blankMsg, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c2_blankMsg, c2_b_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_blankMsg)[200])[4];
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller2_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U,
    1U)>>>(c2_gpu_msg, c2_gpu_r);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c2_r, c2_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c2_r.ModelName_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [0];
  c2_r.ModelName_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)[8])
    [4];
  c2_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[0];
  c2_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [72])[4];
  c2_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[0];
  c2_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [136])[4];
  hipMemcpy(c2_gpu_r, &c2_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller2_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U,
    1U)>>>(c2_gpu_msg, c2_gpu_r);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c2_r, c2_gpu_r, 208UL, hipMemcpyDeviceToHost);
  c2_r.Qdd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[0];
  c2_r.Qdd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c2_msg)
    [200])[4];
  c2_eML_blk_kernel(chartInstance, &c2_r);
  hipMemcpy(c2_gpu_r, &c2_r, 208UL, hipMemcpyHostToDevice);
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller2_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U,
    1U)>>>(c2_gpu_r, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[0] = c2_r.ModelName_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[8])[4] = c2_r.ModelName_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[0] = c2_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[72])[4] = c2_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[0] = c2_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[136])[4] = c2_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c2_gpu_msg, chartInstance->c2_msg, 208UL, hipMemcpyHostToDevice);
  c2_sf_gateway_c2_sliding_controller2_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U,
    1U)>>>(c2_gpu_r, c2_gpu_msg);
  hipMemcpy(chartInstance->c2_msg, c2_gpu_msg, 208UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[0] = c2_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c2_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c2_msg)[200])[4] = c2_r.Qdd_SL_Info.ReceivedLength;
  c2_do_animation_call_c2_sliding_controller2(chartInstance);
  hipFree(c2_b_gpu_blankMsg);
  hipFree(c2_gpu_msg);
  hipFree(c2_gpu_r);
}

static void mdl_start_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c2_errCode;
  hipFree(*chartInstance->c2_gpu_uv);
  hipFree(chartInstance->c2_gpu_blankMsg);
  c2_errCode = hipGetLastError();
  if (c2_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c2_errCode, (char_T *)hipGetErrorName
                       (c2_errCode), (char_T *)hipGetErrorString(c2_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c2_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc2_sliding_controller2
  (SFc2_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c2_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c2_nameCaptureInfo = NULL;
  c2_nameCaptureInfo = NULL;
  sf_mex_assign(&c2_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c2_nameCaptureInfo;
}

static void c2_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_b_msg, const char_T *c2_identifier,
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_y)
{
  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c2_b_msg), &c2_thisId, c2_y);
  sf_mex_destroy(&c2_b_msg);
}

static void c2_b_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_y)
{
  static const char_T *c2_fieldNames[8] = { "ModelName", "ModelName_SL_Info",
    "Q", "Q_SL_Info", "Qd", "Qd_SL_Info", "Qdd", "Qdd_SL_Info" };

  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fParent = c2_parentId;
  c2_thisId.bParentIsCell = false;
  sf_mex_check_struct(c2_parentId, c2_u, 8, c2_fieldNames, 0U, NULL);
  c2_thisId.fIdentifier = "ModelName";
  c2_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u,
    "ModelName", "ModelName", 0)), &c2_thisId, c2_y->ModelName);
  c2_thisId.fIdentifier = "ModelName_SL_Info";
  c2_y->ModelName_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "ModelName_SL_Info", "ModelName_SL_Info", 0)),
    &c2_thisId);
  c2_thisId.fIdentifier = "Q";
  c2_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u, "Q", "Q",
    0)), &c2_thisId, c2_y->Q);
  c2_thisId.fIdentifier = "Q_SL_Info";
  c2_y->Q_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "Q_SL_Info", "Q_SL_Info", 0)), &c2_thisId);
  c2_thisId.fIdentifier = "Qd";
  c2_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u, "Qd",
    "Qd", 0)), &c2_thisId, c2_y->Qd);
  c2_thisId.fIdentifier = "Qd_SL_Info";
  c2_y->Qd_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c2_thisId);
  c2_thisId.fIdentifier = "Qdd";
  c2_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c2_u, "Qdd",
    "Qdd", 0)), &c2_thisId, c2_y->Qdd);
  c2_thisId.fIdentifier = "Qdd_SL_Info";
  c2_y->Qdd_SL_Info = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "Qdd_SL_Info", "Qdd_SL_Info", 0)), &c2_thisId);
  sf_mex_destroy(&c2_u);
}

static void c2_c_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  uint8_T c2_y[4])
{
  int32_T c2_i;
  uint8_T c2_uv[4];
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), c2_uv, 1, 3, 0U, 1, 0U, 1, 4);
  for (c2_i = 0; c2_i < 4; c2_i++) {
    c2_y[c2_i] = c2_uv[c2_i];
  }

  sf_mex_destroy(&c2_u);
}

static c2_SL_Bus_ROSVariableLengthArrayInfo c2_d_emlrt_marshallIn
  (SFc2_sliding_controller2InstanceStruct *chartInstance, const mxArray *c2_u,
   const emlrtMsgIdentifier *c2_parentId)
{
  static const char_T *c2_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c2_SL_Bus_ROSVariableLengthArrayInfo c2_y;
  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fParent = c2_parentId;
  c2_thisId.bParentIsCell = false;
  sf_mex_check_struct(c2_parentId, c2_u, 2, c2_fieldNames, 0U, NULL);
  c2_thisId.fIdentifier = "CurrentLength";
  c2_y.CurrentLength = c2_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "CurrentLength", "CurrentLength", 0)), &c2_thisId);
  c2_thisId.fIdentifier = "ReceivedLength";
  c2_y.ReceivedLength = c2_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c2_u, "ReceivedLength", "ReceivedLength", 0)), &c2_thisId);
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static uint32_T c2_e_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint32_T c2_b_u;
  uint32_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 7, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static void c2_f_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real_T c2_y[7])
{
  real_T c2_dv[7];
  int32_T c2_i;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), c2_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c2_i = 0; c2_i < 7; c2_i++) {
    c2_y[c2_i] = c2_dv[c2_i];
  }

  sf_mex_destroy(&c2_u);
}

static uint8_T c2_g_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_sliding_controller2, const
  char_T *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  uint8_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c2_b_is_active_c2_sliding_controller2), &c2_thisId);
  sf_mex_destroy(&c2_b_is_active_c2_sliding_controller2);
  return c2_y;
}

static uint8_T c2_h_emlrt_marshallIn(SFc2_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint8_T c2_b_u;
  uint8_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 3, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static void c2_eML_blk_kernel(SFc2_sliding_controller2InstanceStruct
  *chartInstance, c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex
  *c2_b_blankMsg)
{
  static uint8_T c2_uv[3] = { 101U, 120U, 111U };

  hipMemcpy(chartInstance->c2_gpu_blankMsg, c2_b_blankMsg, 208UL,
             hipMemcpyHostToDevice);
  c2_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c2_gpu_blankMsg);
  hipMemcpy(*chartInstance->c2_gpu_uv, c2_uv, 3UL, hipMemcpyHostToDevice);
  c2_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c2_gpu_uv, chartInstance->c2_gpu_blankMsg);
  c2_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c2_gpu_blankMsg);
  hipMemcpy(c2_b_blankMsg, chartInstance->c2_gpu_blankMsg, 208UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller2_kernel1
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i = (int32_T)c2_threadId;
  if (c2_i < 4) {
    c2_b_msg->ModelName[c2_i] = c2_b_blankMsg->ModelName[c2_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller2_kernel2
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg,
   c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i1;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i1 = (int32_T)c2_threadId;
  if (c2_i1 < 7) {
    c2_b_msg->Q[c2_i1] = c2_b_blankMsg->Q[c2_i1];
    c2_b_msg->Qd[c2_i1] = c2_b_blankMsg->Qd[c2_i1];
    c2_b_msg->Qdd[c2_i1] = c2_b_blankMsg->Qdd[c2_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller2_kernel3
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg,
   c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_r)
{
  uint64_T c2_threadId;
  int32_T c2_i2;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i2 = (int32_T)c2_threadId;
  if (c2_i2 < 4) {
    c2_r->ModelName[c2_i2] = c2_b_msg->ModelName[c2_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller2_kernel4
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg,
   c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_r)
{
  uint64_T c2_threadId;
  int32_T c2_i3;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i3 = (int32_T)c2_threadId;
  if (c2_i3 < 7) {
    c2_r->Q[c2_i3] = c2_b_msg->Q[c2_i3];
    c2_r->Qd[c2_i3] = c2_b_msg->Qd[c2_i3];
    c2_r->Qdd[c2_i3] = c2_b_msg->Qdd[c2_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller2_kernel5(const
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_r,
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i4;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i4 = (int32_T)c2_threadId;
  if (c2_i4 < 4) {
    c2_b_msg->ModelName[c2_i4] = c2_r->ModelName[c2_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_sf_gateway_c2_sliding_controller2_kernel6(const
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_r,
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_msg)
{
  uint64_T c2_threadId;
  int32_T c2_i5;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i5 = (int32_T)c2_threadId;
  if (c2_i5 < 7) {
    c2_b_msg->Q[c2_i5] = c2_r->Q[c2_i5];
    c2_b_msg->Qd[c2_i5] = c2_r->Qd[c2_i5];
    c2_b_msg->Qdd[c2_i5] = c2_r->Qdd[c2_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel7
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg)
{
  uint64_T c2_threadId;
  int32_T c2_tmpIdx;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_tmpIdx = (int32_T)c2_threadId;
  if (c2_tmpIdx < 1) {
    c2_b_blankMsg->ModelName_SL_Info.CurrentLength = 3U;
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel8(const
  uint8_T c2_uv[3],
  c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg)
{
  uint64_T c2_threadId;
  int32_T c2_i;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_i = (int32_T)c2_threadId;
  if (c2_i < 3) {
    c2_b_blankMsg->ModelName[c2_i] = c2_uv[c2_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c2_eML_blk_kernel_kernel9
  (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *c2_b_blankMsg)
{
  uint64_T c2_threadId;
  int32_T c2_tmpIdx;
  c2_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c2_tmpIdx = (int32_T)c2_threadId;
  if (c2_tmpIdx < 1) {
    c2_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
    c2_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
    c2_b_blankMsg->Qdd_SL_Info.CurrentLength = 7U;
  }
}

static void init_dsm_address_info(SFc2_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc2_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c2_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c2_blankMsg =
    (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c2_msg =
    (c2_SL_Bus_sliding_controller2_RBDLInverseDynamicsReques_b5z6ex *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c2_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(234684748U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1579829110U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2233304279U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(164346871U);
}

mxArray *sf_c2_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c2_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c2_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c2_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiD+wcjAwAakOYCYiQECWKF8RqgYI1ycBS6uAMQllQWpIPHiomTPFCCdl5gL5ie"
    "WVnjmpeWDzbdgQJjPhsV8RiTzOaHiEPDBnjL9Ig4g/QZI+lmw6GdG0i8A5OUWp4PdDQsf8u1XcK"
    "BMP8T+AALuV0BxP4SfWRyfmFySWZYan2wUX5yTmZKZlx6fnJ9XUpSfk5NaZATzHwAm+h1j"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c2_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "slepYeUXowTO3g5Jd5Ug84B";
}

static void sf_opaque_initialize_c2_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c2_sliding_controller2
    ((SFc2_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c2_sliding_controller2((SFc2_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c2_sliding_controller2(void *chartInstanceVar)
{
  enable_c2_sliding_controller2((SFc2_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c2_sliding_controller2(void *chartInstanceVar)
{
  disable_c2_sliding_controller2((SFc2_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c2_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c2_sliding_controller2((SFc2_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c2_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c2_sliding_controller2
    ((SFc2_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c2_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c2_sliding_controller2((SFc2_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c2_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc2_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c2_sliding_controller2
      ((SFc2_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc2_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc2_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c2_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c2_sliding_controller2((SFc2_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c2_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c2_sliding_controller2((SFc2_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc2_sliding_controller2((SFc2_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c2_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c2_sliding_controller2
      ((SFc2_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc2_sliding_controller2((SFc2_sliding_controller2InstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c2_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV01v00gYdtKCQIKqQkhckCgn9rjiQ+K0lHyJoJaWdcLHqZqO38SjjGfMfCTNj+DKkSPiyj/",
    "gN3Daf7F72MMeecd209SxHbpZKna1llx37Od95v1+J16tu+vhtYH3w2uedxGfl/Cue+l1IVvX5u",
    "70/bp3J1u/RSFho32iSKS9ykuQCH4FLbk1TIquGMhCGBMDUCAoYmOpTBmbZpHlTIw6VlDHp1+Gj",
    "IZ+KC0PGihLgj3Bp8gWW7OPPC2mgJoOQGBCJe0w7HAynGmszKQZAh1pG1WZoMH4NnZq6V3LDYs5",
    "tI+AdoU2BDXWJ7r5hhhomqNSM52l2j8GyijmjIhCa0OifYjRwQb6cYB/96xBo/IwGhJlGhCSMeg",
    "dNko4pYA8J9P44ZAJYqRihLcj3nSCi7rtc9RnVwbAKxyCujUUkFEsmTDl8fc7aGlbkEMOLTi0w3",
    "I2H95YF/wXDCagSv02aMoxKDKEPVG6aeKQ9lESrVmWLMIMi+AFUY8pxk9DUJq9mDnaJxgn6KFEG",
    "QwSI7u6p9gY3VvKZqOuy8xlJWOjNNh6GSxha4+hKgoztg4VTcK5LoX1ZLwDY+AJa4sYUg1LWYtx",
    "WrOgJ9HBLr3Lq8EKhoHPYE0pAlYYrnEOkPSdZ9hYTiOp1UZGTUze1s7O4udFWFcYUANCoagLKMI",
    "0oM8S95azBUy72CMQtTKJekXgNEOWoTw9sKI1kWqEPqloIicmuIiWAiM9xFhiJfQ1Fk0VzMVyGY",
    "4SGkLgGgzjsItlg9gCn2jX2h5j3Y2ZmbZAU8XigqharDpsQ22XUNMY+mIk5ER0lIz8rMen7gXAb",
    "CNKMDFsYP9S0w7uXhQKN89+9k7m2ZVvmGfHcvnnT3M8tQIeb+6Z3/dyvXrfOv5Xy+S25+Su5vZZ",
    "z8k53Cbea1/eb7y89vnD8+mfn7Z/f3d7lf0/1s82/zey9c3jRjsrnPFCvjrskzm91gv4b8zxb2Z",
    "rzSF+Df1XctLbuzd88DR40B8+vN9I+P6qVetbz+l7/H7LdXzMsiQ/Fe0G2cHErYlNx3VyHprT9+",
    "ISf1zO3qfXH49Wk7++nY9jkb/WTvlrzVXuqbz9+/tvba8mn+6/v0T/rVy8t5K5fkBct4ADevdAc",
    "xZgpR9QKYySnIO6u/EP1NlZ5bxzlvu36Pm/3PeP+7fMrfOW885ZblX7zjq/fzR81RzwcvjNH9iO",
    "Vc9V3xv/m3e288+tbP3L7CdOM2Q8KDjtZp/x6Doo+vofyNOv71Oz0A==",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c2_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c2_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(245289475U));
  ssSetChecksum1(S,(2797605463U));
  ssSetChecksum2(S,(2985589073U));
  ssSetChecksum3(S,(563539776U));
}

static void mdlRTW_c2_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c2_sliding_controller2(SimStruct *S)
{
  SFc2_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc2_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc2_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc2_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc2_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c2_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c2_sliding_controller2;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c2_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c2_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c2_sliding_controller2;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c2_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c2_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c2_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c2_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c2_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c2_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c2_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c2_JITStateAnimation,
    chartInstance->c2_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c2_sliding_controller2(chartInstance);
}

void c2_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c2_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c2_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c2_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c2_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
