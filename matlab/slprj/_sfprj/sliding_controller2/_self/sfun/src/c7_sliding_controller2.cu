/* Include files */

#include "sliding_controller2_sfun.h"
#include "c7_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void enable_c7_sliding_controller2(SFc7_sliding_controller2InstanceStruct
  *chartInstance);
static void disable_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void c7_do_animation_call_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance, const mxArray *c7_st);
static void sf_gateway_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance);
static void c7_eML_blk_kernel(SFc7_sliding_controller2InstanceStruct
  *chartInstance, real_T c7_b_q[2], real_T c7_b_y[4]);
static void c7_emlrt_marshallIn(SFc7_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c7_b_y, const char_T *c7_identifier, real_T
  c7_c_y[4]);
static void c7_b_emlrt_marshallIn(SFc7_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId,
  real_T c7_b_y[4]);
static uint8_T c7_c_emlrt_marshallIn(SFc7_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c7_b_is_active_c7_sliding_controller2, const
  char_T *c7_identifier);
static uint8_T c7_d_emlrt_marshallIn(SFc7_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId);
static void init_dsm_address_info(SFc7_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc7_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c7_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c7_is_active_c7_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c7_sliding_controller2(SFc7_sliding_controller2InstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c7_do_animation_call_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c7_b_y = NULL;
  const mxArray *c7_c_y = NULL;
  const mxArray *c7_d_y = NULL;
  const mxArray *c7_st;
  c7_st = NULL;
  c7_b_y = NULL;
  sf_mex_assign(&c7_b_y, sf_mex_createcellmatrix(2, 1), false);
  c7_c_y = NULL;
  sf_mex_assign(&c7_c_y, sf_mex_create("y", *chartInstance->c7_y, 0, 0U, 1U, 0U,
    2, 2, 2), false);
  sf_mex_setcell(c7_b_y, 0, c7_c_y);
  c7_d_y = NULL;
  sf_mex_assign(&c7_d_y, sf_mex_create("y",
    &chartInstance->c7_is_active_c7_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c7_b_y, 1, c7_d_y);
  sf_mex_assign(&c7_st, c7_b_y, false);
  return c7_st;
}

static void set_sim_state_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance, const mxArray *c7_st)
{
  const mxArray *c7_u;
  c7_u = sf_mex_dup(c7_st);
  c7_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c7_u, 0)), "y",
                      *chartInstance->c7_y);
  chartInstance->c7_is_active_c7_sliding_controller2 = c7_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c7_u, 1)),
     "is_active_c7_sliding_controller2");
  sf_mex_destroy(&c7_u);
  sf_mex_destroy(&c7_st);
}

static void sf_gateway_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c7_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c7_eML_blk_kernel(chartInstance, *chartInstance->c7_q, *chartInstance->c7_y);
  c7_do_animation_call_c7_sliding_controller2(chartInstance);
}

static void mdl_start_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c7_errCode;
  c7_errCode = hipGetLastError();
  if (c7_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c7_errCode, (char_T *)hipGetErrorName
                       (c7_errCode), (char_T *)hipGetErrorString(c7_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c7_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc7_sliding_controller2
  (SFc7_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c7_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c7_nameCaptureInfo = NULL;
  c7_nameCaptureInfo = NULL;
  sf_mex_assign(&c7_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c7_nameCaptureInfo;
}

static void c7_eML_blk_kernel(SFc7_sliding_controller2InstanceStruct
  *chartInstance, real_T c7_b_q[2], real_T c7_b_y[4])
{
  c7_b_y[0] = -0.324 * muDoubleScalarSin(c7_b_q[0]);
  c7_b_y[2] = 0.0;
  c7_b_y[1] = 0.324 * muDoubleScalarCos(c7_b_q[0]);
  c7_b_y[3] = 0.0;
}

static void c7_emlrt_marshallIn(SFc7_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c7_b_y, const char_T *c7_identifier, real_T
  c7_c_y[4])
{
  emlrtMsgIdentifier c7_thisId;
  c7_thisId.fIdentifier = const_cast<const char_T *>(c7_identifier);
  c7_thisId.fParent = NULL;
  c7_thisId.bParentIsCell = false;
  c7_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c7_b_y), &c7_thisId, c7_c_y);
  sf_mex_destroy(&c7_b_y);
}

static void c7_b_emlrt_marshallIn(SFc7_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId,
  real_T c7_b_y[4])
{
  real_T c7_dv[4];
  int32_T c7_i;
  sf_mex_import(c7_parentId, sf_mex_dup(c7_u), c7_dv, 1, 0, 0U, 1, 0U, 2, 2, 2);
  for (c7_i = 0; c7_i < 4; c7_i++) {
    c7_b_y[c7_i] = c7_dv[c7_i];
  }

  sf_mex_destroy(&c7_u);
}

static uint8_T c7_c_emlrt_marshallIn(SFc7_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c7_b_is_active_c7_sliding_controller2, const
  char_T *c7_identifier)
{
  emlrtMsgIdentifier c7_thisId;
  uint8_T c7_b_y;
  c7_thisId.fIdentifier = const_cast<const char_T *>(c7_identifier);
  c7_thisId.fParent = NULL;
  c7_thisId.bParentIsCell = false;
  c7_b_y = c7_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c7_b_is_active_c7_sliding_controller2), &c7_thisId);
  sf_mex_destroy(&c7_b_is_active_c7_sliding_controller2);
  return c7_b_y;
}

static uint8_T c7_d_emlrt_marshallIn(SFc7_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c7_u, const emlrtMsgIdentifier *c7_parentId)
{
  uint8_T c7_b_u;
  uint8_T c7_b_y;
  sf_mex_import(c7_parentId, sf_mex_dup(c7_u), &c7_b_u, 1, 3, 0U, 0, 0U, 0);
  c7_b_y = c7_b_u;
  sf_mex_destroy(&c7_u);
  return c7_b_y;
}

static void init_dsm_address_info(SFc7_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc7_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c7_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c7_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c7_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c7_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(536815311U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1945133011U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2900394615U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(32238838U);
}

mxArray *sf_c7_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c7_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c7_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c7_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiD+wcjAwAakOYCYiQECWKF8RqgYI1ycBS6uAMQllQWpIPHiomTPFCCdl5gL5ie"
    "WVnjmpeWDzbdgQJjPhsV8RiTzOaHiEPDBnjL9Ig4g/QZI+lkI6BcAsiqh4QILH/LtV3CgTD/E/g"
    "AC7ldAcT+En1kcn5hcklmWGp9sHl+ck5mSmZcen5yfV1KUn5OTWmQE8x8AZwEclg=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c7_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sFqlSox9r5ekjZXT2w93GAD";
}

static void sf_opaque_initialize_c7_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c7_sliding_controller2
    ((SFc7_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c7_sliding_controller2((SFc7_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c7_sliding_controller2(void *chartInstanceVar)
{
  enable_c7_sliding_controller2((SFc7_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c7_sliding_controller2(void *chartInstanceVar)
{
  disable_c7_sliding_controller2((SFc7_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c7_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c7_sliding_controller2((SFc7_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c7_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c7_sliding_controller2
    ((SFc7_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c7_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c7_sliding_controller2((SFc7_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c7_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc7_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c7_sliding_controller2
      ((SFc7_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc7_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc7_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c7_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c7_sliding_controller2((SFc7_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c7_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c7_sliding_controller2((SFc7_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc7_sliding_controller2((SFc7_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c7_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c7_sliding_controller2
      ((SFc7_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc7_sliding_controller2((SFc7_sliding_controller2InstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c7_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u00oUdqKCLgKq6upKV0JIdAdLBELABtrmByJSWuEUEJtqOj6J52Y8Y+YnaV6AHe/Bgjf",
    "gRXgCNmxYsuSM7abBsR1KoAJ0LbnO2N/55vyfqVfrbHt4reJ952/PO4vPv/Cue+l1JlvXZu70/Y",
    "p3NVu/RiFho12iSKS9ykuQCJ6AltwaJkVH9GUhjIk+KBAUsbFUpoxNs8hyJoZtK6jj089CRkM/l",
    "JYHWyhLgh3BJ8gWW7OLPE2mgJo2QGBCJe0gbHMymGqszLgRAh1qG1WZoMH4NnZq6W3LDYs5tA6B",
    "doQ2BDXWx7r5hhhomMNSM52l2j8CyijmjIhCa0OifYjRwQb24gD/7liDRuVhNCTKbEFIRqC7bJh",
    "wSgF5TqbxwwETxEjFCG9FvOEE53Xb5ajPtgyAVzgEddtSQIaxZMKUx99vo6UtQQ44NOHADsrZfH",
    "hpXfCfMhiDKvVbvyFHoMgAdkTppolDWodJtKZZMg8zLIKnRG1SjJ+GoDR7MXO0TzBO0EOJMhgkR",
    "nZ0T7ERureUzUYdl5mLSsZGabD1IljC1hpBVRSmbG0qGoRzXQrrybgLI+AJa5MYUg1LWYtxWrOg",
    "J9HBLr3Lq8EKhoHPYA0pAlYYrlEOkPSdx9hYvkZSq42MGpi8zW53/vM8rCMMqD6hUNQFFGEa0Ge",
    "Je8vZAqZd7BGIWplEvSJwmiGLUJ7uW9EcSzVEn1Q0kWMTXERLgZEeYCyxEvY0Fk0VzMVyEY4SGk",
    "LgGgzjsI1lg9gCn2jX2jax7kbMTJqgqWJxQVQtVh22oZZLqEkMe2Io5Fi0lYz8rMen7gXAbCNKM",
    "DHYwv6lJm3cvSgUbp5d947n2YVvmGdHcvnntRmeWgGPN/PM73uuXr1vHX/VMrmNGbmLuX1WcnIO",
    "t4b3pbe3Pry7qc/w84+6rz4+6y6z/5v6yeb/ara+fNRop4UzmstXh304o9dKAf+/M/xr2Vq3X3J",
    "fHt5Vt2D434vnvRvjuzcfbDYTvs+1an3rOX2P3q+7jo9ZluSnop0gO5i4NbHpuE7OQzP6nl3gj3",
    "PZ+/T6dH85+X828nFcWSC/hr8mubz9/v3XN5aTT/ffXaD/ei7e68lc3yeuW8A+vb2vOQuw0vepF",
    "EZJzkHdWP0BdXZSOe+U5X4XPf+X+/lx/5a5ddpy3inLLWvfSef3r4avmgNeDr/2C9ux7LnqZ+Pf",
    "eyc7/1zJ1vem/+I0QsaDgtNu9hmPrv2ir39Ann4BqgWx1A==",
    ""
  };

  static char newstr [1249] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c7_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c7_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3862277146U));
  ssSetChecksum1(S,(91435963U));
  ssSetChecksum2(S,(1279986540U));
  ssSetChecksum3(S,(1280830342U));
}

static void mdlRTW_c7_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c7_sliding_controller2(SimStruct *S)
{
  SFc7_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc7_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc7_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc7_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc7_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c7_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c7_sliding_controller2;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c7_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c7_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c7_sliding_controller2;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c7_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c7_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c7_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c7_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c7_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c7_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c7_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c7_JITStateAnimation,
    chartInstance->c7_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c7_sliding_controller2(chartInstance);
}

void c7_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c7_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c7_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c7_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c7_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
