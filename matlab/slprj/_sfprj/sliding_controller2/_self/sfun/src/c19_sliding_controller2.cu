/* Include files */

#include "sliding_controller2_sfun.h"
#include "c19_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void enable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void disable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void c19_do_animation_call_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance, const mxArray *c19_st);
static void sf_gateway_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void c19_eML_blk_kernel(SFc19_sliding_controller2InstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2]);
static void c19_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2]);
static void c19_b_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2]);
static uint8_T c19_c_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_is_active_c19_sliding_controller2, const
  char_T *c19_identifier);
static uint8_T c19_d_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId);
static void init_dsm_address_info(SFc19_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc19_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c19_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c19_is_active_c19_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c19_do_animation_call_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c19_b_y = NULL;
  const mxArray *c19_c_y = NULL;
  const mxArray *c19_d_y = NULL;
  const mxArray *c19_e_y = NULL;
  const mxArray *c19_f_y = NULL;
  const mxArray *c19_st;
  const mxArray *c19_y = NULL;
  c19_st = NULL;
  c19_y = NULL;
  sf_mex_assign(&c19_y, sf_mex_createcellmatrix(5, 1), false);
  c19_b_y = NULL;
  sf_mex_assign(&c19_b_y, sf_mex_create("y", *chartInstance->c19_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 0, c19_b_y);
  c19_c_y = NULL;
  sf_mex_assign(&c19_c_y, sf_mex_create("y", *chartInstance->c19_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 1, c19_c_y);
  c19_d_y = NULL;
  sf_mex_assign(&c19_d_y, sf_mex_create("y", *chartInstance->c19_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 2, c19_d_y);
  c19_e_y = NULL;
  sf_mex_assign(&c19_e_y, sf_mex_create("y", *chartInstance->c19_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c19_y, 3, c19_e_y);
  c19_f_y = NULL;
  sf_mex_assign(&c19_f_y, sf_mex_create("y",
    &chartInstance->c19_is_active_c19_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c19_y, 4, c19_f_y);
  sf_mex_assign(&c19_st, c19_y, false);
  return c19_st;
}

static void set_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance, const mxArray *c19_st)
{
  const mxArray *c19_u;
  c19_u = sf_mex_dup(c19_st);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 0)),
                       "ankle", *chartInstance->c19_ankle);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 1)),
                       "danke", *chartInstance->c19_danke);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 2)),
                       "dknee", *chartInstance->c19_dknee);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 3)),
                       "knee", *chartInstance->c19_knee);
  chartInstance->c19_is_active_c19_sliding_controller2 = c19_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 4)),
     "is_active_c19_sliding_controller2");
  sf_mex_destroy(&c19_u);
  sf_mex_destroy(&c19_st);
}

static void sf_gateway_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c19_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c19_eML_blk_kernel(chartInstance, *chartInstance->c19_q,
                     *chartInstance->c19_qd, *chartInstance->c19_knee,
                     *chartInstance->c19_dknee, *chartInstance->c19_ankle,
                     *chartInstance->c19_danke);
  c19_do_animation_call_c19_sliding_controller2(chartInstance);
}

static void mdl_start_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c19_errCode;
  c19_errCode = hipGetLastError();
  if (c19_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c19_errCode, (char_T *)hipGetErrorName
                       (c19_errCode), (char_T *)hipGetErrorString(c19_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c19_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c19_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c19_nameCaptureInfo = NULL;
  const char_T *c19_data[4] = {
    "789cc553cb4ac34014bd915adc545df90b2e04c7baeabe2a88580a0d6e44e224b969c6cc432653dbbaf10fec57f80ffe8e7b3fc2246dfa088688c57a20dc399c"
    "cc3df71e18b02eaf2d00d84d3efb05a055870c8d6981fd59dd82551475aba4e6d886dacabd54ef267e9319f7943438325322a9c0f94d5f0926a934f6f8114163",
    "acf813fa9912308e3613d85b269d94898b25694e52293db743f4a2de40800ee3c5847c996479a4b82fd9b756914711c53c8affe57ea35ffae5fd0f2afc723d50"
    "7a48b5ef444c32410df36287637ffdbd73df7ae91c53c5570397e3c2ef7d4d3fbbd46f55bf3dbf23a11248243521950c795f713fa0da251e35491ace3026b1f6",
    "08156ee00c298f509324204e5df27d66c722ed5b95dbde0ff7287b3f0dd8c9eadbeb11dda4dfc7e1a7b549bf1cffe5b7a9f777753266cd67f61045a7ad76a7e9"
    "bade8d6fce1673742b7caae68012fed7fdbf00b09967db", "" };

  c19_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c19_data[0], 1616U, &c19_nameCaptureInfo);
  return c19_nameCaptureInfo;
}

static void c19_eML_blk_kernel(SFc19_sliding_controller2InstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2])
{
  real_T c19_a;
  real_T c19_b_a;
  c19_b_knee[0] = -0.324 * muDoubleScalarSin(c19_b_q[0]);
  c19_b_knee[1] = -0.324 * muDoubleScalarCos(c19_b_q[0]);
  c19_a = -0.324 * c19_b_qd[0];
  c19_b_dknee[0] = c19_a * muDoubleScalarCos(c19_b_q[0]);
  c19_b_dknee[1] = c19_a * -muDoubleScalarSin(c19_b_q[0]);
  c19_b_a = -0.39 * (c19_b_qd[0] + c19_b_qd[1]);
  c19_b_ankle[0] = c19_b_knee[0] + -0.39 * muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_ankle[1] = c19_b_knee[1] + -0.39 * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[0] = c19_b_dknee[0] + c19_b_a * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[1] = c19_b_dknee[1] + c19_b_a * -muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
}

static void c19_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2])
{
  emlrtMsgIdentifier c19_thisId;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c19_b_ankle), &c19_thisId,
    c19_y);
  sf_mex_destroy(&c19_b_ankle);
}

static void c19_b_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2])
{
  real_T c19_dv[2];
  int32_T c19_i;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), c19_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c19_i = 0; c19_i < 2; c19_i++) {
    c19_y[c19_i] = c19_dv[c19_i];
  }

  sf_mex_destroy(&c19_u);
}

static uint8_T c19_c_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_is_active_c19_sliding_controller2, const
  char_T *c19_identifier)
{
  emlrtMsgIdentifier c19_thisId;
  uint8_T c19_y;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_y = c19_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c19_b_is_active_c19_sliding_controller2), &c19_thisId);
  sf_mex_destroy(&c19_b_is_active_c19_sliding_controller2);
  return c19_y;
}

static uint8_T c19_d_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId)
{
  uint8_T c19_b_u;
  uint8_T c19_y;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), &c19_b_u, 1, 3, 0U, 0, 0U, 0);
  c19_y = c19_b_u;
  sf_mex_destroy(&c19_u);
  return c19_y;
}

static void init_dsm_address_info(SFc19_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc19_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c19_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c19_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c19_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c19_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c19_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c19_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c19_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c19_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c19_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c19_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6LFA4ifWRyfmFySWZYan"
    "2xoGV+ck5mSmZcen5yfV1KUn5OTWmSEZD4IAACHxSy4"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c19_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c19_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c19_sliding_controller2
    ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c19_sliding_controller2(void *chartInstanceVar)
{
  enable_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c19_sliding_controller2(void *chartInstanceVar)
{
  disable_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c19_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c19_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c19_sliding_controller2
    ((SFc19_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c19_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c19_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c19_sliding_controller2
      ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc19_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c19_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c19_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c19_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c19_sliding_controller2
      ((SFc19_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc19_sliding_controller2
      ((SFc19_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c19_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIAm8CFALwGaFCiSo9uiQHNpbVMSIsCunVBOg6KAsVqOxIWWu+z+0HabJyj",
    "Qe9F36XvkGXpJgR6TQ2ZJWpYpkorsJnWL0qDkJb/5dr6Z2dmV1xrsenjdwvvX2553Db+v49328m",
    "utGLfm7vx5x7tfjH9BI2HjfaJIrL3GS5AYnoCW3BomxUCMZSWMiTEoEBSxiVSmjk2z2HImpn0rq",
    "OPT30aMRkEkLQ+30ZaEe4KfIFtizT7ydJkCavoAoYmUtJOoz8lk5rEyR34EdKpt3CRBgwls4tzS",
    "u5YblnDoHQMdCG0IeqzPfAsMMeCb41qZTqkOToEyTjgjolJtRHQACQbYwEES4ueeNSiqDKMRUWY",
    "bIpKC3mHTjFMKKHMyjS9GTBAjFSO8F3PfGS76ts/Rn10ZAm8ICPq2rYBME8mEqc9/0EelPUFGHL",
    "owspN6tgB+sC75TxkcgaqN29iXKSgygT1RO2kWkN5xlq1ZlSzCDIvhKVFbFPOnIaytXqwcHRDME",
    "wzRog4GmciBHiqWYnhr2Ww8cJW5bMnYOE+2XgbL2HopNGVhxtanwiec61rYUCY7kALPWLvEkGZY",
    "zlqN05qFQ4kBduVdvxqsYJj4AuZLEbLKdKUlQNZ3vsHGch5JrTYy9rF4uzs7i68XYQNhQI0Jhao",
    "uoAjTgDHLwlvPFjLtco9A9Mpk7lWB8wpZhvL02IrukVRTjElDEzmT4DJaC4z1BHOJK+FA46Jpgr",
    "lcLsNRQiMIXYNhHHZx2SC2IibatbYtXHcpMydd0FSxpCKrFlcdtqGeK6iTBA7EVMgj0VcyDooen",
    "4cXAKuNKMHEZBv7lzrp4+xVqXD72YZ3tp998Bb72ald+fvBHE+rgseb+y7Pe6N9Ht8uzdvGv1Yr",
    "t9ucs7tZmqdTsnO4dbz/eP3XJ89f/jz66UX4/Pvffn+1bP7Wwvyt7H9nd31ttf3/VjG+c9poZws",
    "nXahXh30051engv/DOf71YqwfR1+I9LtH0dZjMowHG5//+HAjHfkZ37NOs79rJX9Pn991HR+rLK",
    "tPRQdhcTBxY2Lz7drxfznn77Ul8bhRPM+vP7++nP2dzbJ9Z4m+9WJMxJRDqX7/OR0fX1hHiEKuj",
    "o67F9cxxUPDldFxe7PcH6p0dM7p6HhOwt/j/2IcV7PP53+2xP97pTzcy86Lh8TtQnBIP314qDkL",
    "cQs5pFIYJTkH9VlF379oH13VznvPdv8WP/+3e/d5f5tzUfuCdp1LnsPel91l9a16Prxq+Kb9wCv",
    "h16+wjsue2981/oW32vn6o2L81ewntB8xHlb8mipe40+jcdXb/0CdvgGFb8Vv",
    ""
  };

  static char newstr [1337] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c19_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c19_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(636746986U));
  ssSetChecksum1(S,(1653272444U));
  ssSetChecksum2(S,(2086983803U));
  ssSetChecksum3(S,(4223507036U));
}

static void mdlRTW_c19_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c19_sliding_controller2(SimStruct *S)
{
  SFc19_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc19_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc19_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc19_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc19_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c19_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c19_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c19_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c19_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c19_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c19_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c19_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c19_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c19_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c19_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c19_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c19_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c19_JITStateAnimation,
    chartInstance->c19_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c19_sliding_controller2(chartInstance);
}

void c19_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c19_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c19_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c19_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c19_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
