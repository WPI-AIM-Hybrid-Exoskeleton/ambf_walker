/* Include files */

#include "sliding_controller2_sfun.h"
#include "c19_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void enable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void disable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void c19_do_animation_call_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance, const mxArray *c19_st);
static void sf_gateway_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance);
static void c19_eML_blk_kernel(SFc19_sliding_controller2InstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2]);
static void c19_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2]);
static void c19_b_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2]);
static uint8_T c19_c_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_is_active_c19_sliding_controller2, const
  char_T *c19_identifier);
static uint8_T c19_d_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId);
static void init_dsm_address_info(SFc19_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc19_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c19_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c19_is_active_c19_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c19_do_animation_call_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c19_b_y = NULL;
  const mxArray *c19_c_y = NULL;
  const mxArray *c19_d_y = NULL;
  const mxArray *c19_e_y = NULL;
  const mxArray *c19_f_y = NULL;
  const mxArray *c19_st;
  const mxArray *c19_y = NULL;
  c19_st = NULL;
  c19_y = NULL;
  sf_mex_assign(&c19_y, sf_mex_createcellmatrix(5, 1), false);
  c19_b_y = NULL;
  sf_mex_assign(&c19_b_y, sf_mex_create("y", *chartInstance->c19_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 0, c19_b_y);
  c19_c_y = NULL;
  sf_mex_assign(&c19_c_y, sf_mex_create("y", *chartInstance->c19_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 1, c19_c_y);
  c19_d_y = NULL;
  sf_mex_assign(&c19_d_y, sf_mex_create("y", *chartInstance->c19_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c19_y, 2, c19_d_y);
  c19_e_y = NULL;
  sf_mex_assign(&c19_e_y, sf_mex_create("y", *chartInstance->c19_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c19_y, 3, c19_e_y);
  c19_f_y = NULL;
  sf_mex_assign(&c19_f_y, sf_mex_create("y",
    &chartInstance->c19_is_active_c19_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c19_y, 4, c19_f_y);
  sf_mex_assign(&c19_st, c19_y, false);
  return c19_st;
}

static void set_sim_state_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance, const mxArray *c19_st)
{
  const mxArray *c19_u;
  c19_u = sf_mex_dup(c19_st);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 0)),
                       "ankle", *chartInstance->c19_ankle);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 1)),
                       "danke", *chartInstance->c19_danke);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 2)),
                       "dknee", *chartInstance->c19_dknee);
  c19_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 3)),
                       "knee", *chartInstance->c19_knee);
  chartInstance->c19_is_active_c19_sliding_controller2 = c19_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c19_u, 4)),
     "is_active_c19_sliding_controller2");
  sf_mex_destroy(&c19_u);
  sf_mex_destroy(&c19_st);
}

static void sf_gateway_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c19_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c19_eML_blk_kernel(chartInstance, *chartInstance->c19_q,
                     *chartInstance->c19_qd, *chartInstance->c19_knee,
                     *chartInstance->c19_dknee, *chartInstance->c19_ankle,
                     *chartInstance->c19_danke);
  c19_do_animation_call_c19_sliding_controller2(chartInstance);
}

static void mdl_start_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c19_errCode;
  c19_errCode = hipGetLastError();
  if (c19_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c19_errCode, (char_T *)hipGetErrorName
                       (c19_errCode), (char_T *)hipGetErrorString(c19_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c19_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc19_sliding_controller2
  (SFc19_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c19_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c19_nameCaptureInfo = NULL;
  const char_T *c19_data[4] = {
    "789cc553cb4ac34014bd95b6e8a2eacaad7fe08082b8562a58a8545af085a493e4a6193b0f9da4a65df909fe8a1bc1bff067fc00f368da26182216eb8170e770"
    "32f7dc7b60a072d6ae00c066f86d3c031cd521462329b03dad6b90455eaf14d41435a866ee457a2df47b99724b491fc77e42241538bb692bc124957e6ff280a0",
    "d153fc09ed587118c71e13d85d24e71113a70bd28c4452743e71d11a764702b4ebcd27e48b24ce2342bf60df6a491e79e4f3c8ff97fa8d7fe997f6df29f14b75"
    "47e9806adb1832c904f599e5191c07cbef9dfad60be748145b8d4c8e73bff725fd7a857e59fdb679475c259048eabb5432e403c56d876a9358d40fd330028f78",
    "da2254988e11503e444dc2803835c9f799ed89a86f596e5b3fdca3e8fd34603daee4f3b5bf4abf8fb7fbdd55faa5f82fbf55bd3fab134c9a97fb37faa2ed1c1e"
    "5cf1c7eb564b1ecfe7e894f894cd0105fcaffb7f01900e67d7", "" };

  c19_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c19_data[0], 1616U, &c19_nameCaptureInfo);
  return c19_nameCaptureInfo;
}

static void c19_eML_blk_kernel(SFc19_sliding_controller2InstanceStruct
  *chartInstance, real_T c19_b_q[2], real_T c19_b_qd[2], real_T c19_b_knee[2],
  real_T c19_b_dknee[2], real_T c19_b_ankle[2], real_T c19_b_danke[2])
{
  real_T c19_a;
  real_T c19_b_a;
  c19_b_knee[0] = 0.324 * muDoubleScalarCos(c19_b_q[0]);
  c19_b_knee[1] = 0.324 * muDoubleScalarSin(c19_b_q[0]);
  c19_a = 0.324 * c19_b_qd[0];
  c19_b_dknee[0] = c19_a * -muDoubleScalarSin(c19_b_q[0]);
  c19_b_dknee[1] = c19_a * muDoubleScalarCos(c19_b_q[0]);
  c19_b_a = 0.39 * (c19_b_qd[0] + c19_b_qd[1]);
  c19_b_ankle[0] = c19_b_knee[0] + 0.39 * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_ankle[1] = c19_b_knee[1] + 0.39 * muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[0] = c19_b_dknee[0] + c19_b_a * -muDoubleScalarSin(c19_b_q[0] +
    c19_b_q[1]);
  c19_b_danke[1] = c19_b_dknee[1] + c19_b_a * muDoubleScalarCos(c19_b_q[0] +
    c19_b_q[1]);
}

static void c19_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_ankle, const char_T *c19_identifier,
  real_T c19_y[2])
{
  emlrtMsgIdentifier c19_thisId;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c19_b_ankle), &c19_thisId,
    c19_y);
  sf_mex_destroy(&c19_b_ankle);
}

static void c19_b_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId,
  real_T c19_y[2])
{
  real_T c19_dv[2];
  int32_T c19_i;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), c19_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c19_i = 0; c19_i < 2; c19_i++) {
    c19_y[c19_i] = c19_dv[c19_i];
  }

  sf_mex_destroy(&c19_u);
}

static uint8_T c19_c_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_b_is_active_c19_sliding_controller2, const
  char_T *c19_identifier)
{
  emlrtMsgIdentifier c19_thisId;
  uint8_T c19_y;
  c19_thisId.fIdentifier = const_cast<const char_T *>(c19_identifier);
  c19_thisId.fParent = NULL;
  c19_thisId.bParentIsCell = false;
  c19_y = c19_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c19_b_is_active_c19_sliding_controller2), &c19_thisId);
  sf_mex_destroy(&c19_b_is_active_c19_sliding_controller2);
  return c19_y;
}

static uint8_T c19_d_emlrt_marshallIn(SFc19_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c19_u, const emlrtMsgIdentifier *c19_parentId)
{
  uint8_T c19_b_u;
  uint8_T c19_y;
  sf_mex_import(c19_parentId, sf_mex_dup(c19_u), &c19_b_u, 1, 3, 0U, 0, 0U, 0);
  c19_y = c19_b_u;
  sf_mex_destroy(&c19_u);
  return c19_y;
}

static void init_dsm_address_info(SFc19_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc19_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c19_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c19_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c19_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c19_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c19_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c19_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c19_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c19_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c19_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c19_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c19_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6LFA4ifWRyfmFySWZYan"
    "2xoGV+ck5mSmZcen5yfV1KUn5OTWmSEZD4IAACHxSy4"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c19_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c19_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c19_sliding_controller2
    ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c19_sliding_controller2(void *chartInstanceVar)
{
  enable_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c19_sliding_controller2(void *chartInstanceVar)
{
  disable_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c19_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c19_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c19_sliding_controller2
    ((SFc19_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c19_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c19_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c19_sliding_controller2
      ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc19_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc19_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c19_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c19_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc19_sliding_controller2((SFc19_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c19_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c19_sliding_controller2
      ((SFc19_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc19_sliding_controller2
      ((SFc19_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c19_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIDW8CFALwHyc2mObosCTQ+pbUpCCNi1U8ppUKAw1suRuNByl9kf2u475N5",
    "DX6HPkmN66jP02GNnSVqWKZKK7CZ1i9Kg5CW/+Xa+mdnZldcJ9jy81vH++a7n3cLv23h3veJaK8",
    "edubt43vM+Lcev0UjY5IAokmiv9RIkge9AS24NkyIQY1kLY2IMCgRFbCqVaWLTLLGcienQCur49",
    "Pcxo3EYS8ujHbQl0b7gZ8iWWnOAPH2mgJohQGRiJe0kHnIymXmszIkfA51qm7RJ0GBCmzq39J7l",
    "hqUcBqdAA6ENQY/1hW+hIQZ8c9oo0ynV4TlQJilnRNSqjYkOIcUAGzhMI/zctwZFVWE0JsrsQEw",
    "y0LtsmnNKAVVOpvHFMRPESMUIHyTcd4aLvh1w9GdPRsBbAoK+7Sgg01QyYZrzHw5R6UCQYw59OL",
    "aTZrYQXlmX/BcMTkA1xm3sywwUmcC+aJw0D8jgNM/WrEoWYYYl8IKobYr50xA1Vi9Wjg4J5glGa",
    "NEEg1xkoEeKZRjeRjabBK4yly0ZmxTJ1stgOdsgg7YszNiGVPiEc90IG8l0FzLgOWufGNIOK1jr",
    "cVqzaCQxwK68m1eDFQwTX8J8KSJWm66sAsj7zrfYWC4jqdVGJj4Wb393d/H1IiwQBtSYUKjrAoo",
    "wDRizPLzNbBHTLvcIRK9M7l4duKiQZShPj63on0g1xZi0NJELCS6jjcBETzCXuBIONS6aNpjL5T",
    "IcJTSGyDUYxmEPlw1ia2KiXWvbxnWXMXPWB00VS2uyanHVYRsauII6S+FQTIU8EUMlk7Ds8UV4A",
    "bDaiBJMTHawf6mzIc5elwq3n216F/vZR++wn53bVb8fz/F0ani8ue/qvHe6l/Hdyrxd/Ot0Crut",
    "ObuPK/P0KnYOt4H3r8H64zev35L+7788/fpH789l83cW5u/k/zu722ur7f/r5fjeeaOdLZxsoV4",
    "d9tmcX70a/k/m+DfKsX4efymyH57F28/JKAk2v/jpyWZ27Od8L3vt/q5V/D1//sB1fKyyvD4VDa",
    "LyYOLGxBbbteP/as7fW0vicad8Xlx/fHM9+3tbVfveEn0b5ZiIKYdK/f5zOh5dWUeEQm6OjgdX1",
    "zHFQ8ON0XF3q9of6nT0LunoeU7C3+P/YhxXsy/mf7nE/4eVPDzMz4tHxO1CcEQ/e3KkOYtwCzmi",
    "UhglOQf1eU3fv2ofXdXO+8B2/xY//7d7/3l/l3NR94p2vWuewz6U3XX1rXo+vGn4tv3Aq+A3brC",
    "O657b3zf+N2+18/X9cvx09hPajxmPan5Nla/xp9G47u1/oE7/Auy4woQ=",
    ""
  };

  static char newstr [1333] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c19_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c19_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(672025005U));
  ssSetChecksum1(S,(1641516495U));
  ssSetChecksum2(S,(1050467908U));
  ssSetChecksum3(S,(4160773434U));
}

static void mdlRTW_c19_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c19_sliding_controller2(SimStruct *S)
{
  SFc19_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc19_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc19_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc19_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc19_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c19_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c19_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c19_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c19_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c19_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c19_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c19_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c19_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c19_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c19_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c19_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c19_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c19_JITStateAnimation,
    chartInstance->c19_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c19_sliding_controller2(chartInstance);
}

void c19_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c19_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c19_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c19_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c19_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
