/* Include files */

#include "sliding_controller2_sfun.h"
#include "c1_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void enable_c1_sliding_controller2(SFc1_sliding_controller2InstanceStruct
  *chartInstance);
static void disable_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void c1_do_animation_call_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance, const mxArray *c1_st);
static void sf_gateway_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void c1_eML_blk_kernel(SFc1_sliding_controller2InstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2]);
static void c1_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2]);
static void c1_b_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2]);
static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller2, const
  char_T *c1_identifier);
static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static void init_dsm_address_info(SFc1_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc1_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c1_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c1_is_active_c1_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c1_sliding_controller2(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c1_do_animation_call_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c1_b_y = NULL;
  const mxArray *c1_c_y = NULL;
  const mxArray *c1_d_y = NULL;
  const mxArray *c1_e_y = NULL;
  const mxArray *c1_f_y = NULL;
  const mxArray *c1_st;
  const mxArray *c1_y = NULL;
  c1_st = NULL;
  c1_y = NULL;
  sf_mex_assign(&c1_y, sf_mex_createcellmatrix(5, 1), false);
  c1_b_y = NULL;
  sf_mex_assign(&c1_b_y, sf_mex_create("y", *chartInstance->c1_ankle, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 0, c1_b_y);
  c1_c_y = NULL;
  sf_mex_assign(&c1_c_y, sf_mex_create("y", *chartInstance->c1_danke, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 1, c1_c_y);
  c1_d_y = NULL;
  sf_mex_assign(&c1_d_y, sf_mex_create("y", *chartInstance->c1_dknee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 2, c1_d_y);
  c1_e_y = NULL;
  sf_mex_assign(&c1_e_y, sf_mex_create("y", *chartInstance->c1_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 3, c1_e_y);
  c1_f_y = NULL;
  sf_mex_assign(&c1_f_y, sf_mex_create("y",
    &chartInstance->c1_is_active_c1_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c1_y, 4, c1_f_y);
  sf_mex_assign(&c1_st, c1_y, false);
  return c1_st;
}

static void set_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance, const mxArray *c1_st)
{
  const mxArray *c1_u;
  c1_u = sf_mex_dup(c1_st);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 0)),
                      "ankle", *chartInstance->c1_ankle);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 1)),
                      "danke", *chartInstance->c1_danke);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 2)),
                      "dknee", *chartInstance->c1_dknee);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 3)), "knee",
                      *chartInstance->c1_knee);
  chartInstance->c1_is_active_c1_sliding_controller2 = c1_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 4)),
     "is_active_c1_sliding_controller2");
  sf_mex_destroy(&c1_u);
  sf_mex_destroy(&c1_st);
}

static void sf_gateway_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c1_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c1_eML_blk_kernel(chartInstance, *chartInstance->c1_q, *chartInstance->c1_qd, *
                    chartInstance->c1_knee, *chartInstance->c1_dknee,
                    *chartInstance->c1_ankle, *chartInstance->c1_danke);
  c1_do_animation_call_c1_sliding_controller2(chartInstance);
}

static void mdl_start_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c1_errCode;
  c1_errCode = hipGetLastError();
  if (c1_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c1_errCode, (char_T *)hipGetErrorName
                       (c1_errCode), (char_T *)hipGetErrorString(c1_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c1_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c1_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c1_nameCaptureInfo = NULL;
  const char_T *c1_data[4] = {
    "789cc5534d4bc340109d4a2d5eaa9e04c1ab170fae5ec4b3c5428596da0f3c88c649b26996ee876c52db7af1ee41ff8e7fcaff6092364d130c118bf541987dbc"
    "ecbc99070ba546b30400dbc1f7f602705e8108d55981dd79dd8034b27a29a7c6d88472ea5ea8bf067eef736e29e9d3893f2312055ddcb4956012a5df9b3e52d0",
    "d453fc89da91e2304e7b4cd0ee3269854cd497a40509a5f05c73a935ec8e0468d74b26e4cb24ca23c443cebee5823cb2c8e691fd2ff69bfcd22feebf57e017eb"
    "8ed263d4b631649209f499e5199c0e56df3bf6ade4ce31536c3532394dfc3e56f4ebe5faa5f5dbcb3be22a418944df45c9281f286e3ba84d62a11fa4618c3de2",
    "698ba0301d638c7c48350902e26892ef333b1661dfa2dc767eb847defba9c256543fdbfbb84ebffba383c375fac5f82fbf75bdbf9bf6059e9c3d5fb14e47d79b"
    "d7fd6ea37fdaaa2573b40b7c8ae6801cfed7fdbf0052dd6684", "" };

  c1_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c1_data[0], 1616U, &c1_nameCaptureInfo);
  return c1_nameCaptureInfo;
}

static void c1_eML_blk_kernel(SFc1_sliding_controller2InstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2])
{
  real_T c1_a;
  real_T c1_b_a;
  c1_b_knee[0] = -0.324 * muDoubleScalarCos(c1_b_q[0]);
  c1_b_knee[1] = -0.324 * muDoubleScalarSin(c1_b_q[0]);
  c1_a = -0.324 * c1_b_qd[0];
  c1_b_dknee[0] = c1_a * -muDoubleScalarSin(c1_b_q[0]);
  c1_b_dknee[1] = c1_a * muDoubleScalarCos(c1_b_q[0]);
  c1_b_a = -0.39 * (c1_b_qd[0] + c1_b_qd[1]);
  c1_b_ankle[0] = c1_b_knee[0] - -0.39 * muDoubleScalarSin(c1_b_q[0] + c1_b_q[1]);
  c1_b_ankle[1] = c1_b_knee[1] - -0.39 * muDoubleScalarCos(c1_b_q[0] + c1_b_q[1]);
  c1_b_danke[0] = c1_b_dknee[0] + c1_b_a * -muDoubleScalarSin(c1_b_q[0] +
    c1_b_q[1]);
  c1_b_danke[1] = c1_b_dknee[1] + c1_b_a * muDoubleScalarCos(c1_b_q[0] + c1_b_q
    [1]);
}

static void c1_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2])
{
  emlrtMsgIdentifier c1_thisId;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_b_ankle), &c1_thisId, c1_y);
  sf_mex_destroy(&c1_b_ankle);
}

static void c1_b_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2])
{
  real_T c1_dv[2];
  int32_T c1_i;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), c1_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c1_i = 0; c1_i < 2; c1_i++) {
    c1_y[c1_i] = c1_dv[c1_i];
  }

  sf_mex_destroy(&c1_u);
}

static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller2, const
  char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  uint8_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c1_b_is_active_c1_sliding_controller2), &c1_thisId);
  sf_mex_destroy(&c1_b_is_active_c1_sliding_controller2);
  return c1_y;
}

static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  uint8_T c1_b_u;
  uint8_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_b_u, 1, 3, 0U, 0, 0U, 0);
  c1_y = c1_b_u;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static void init_dsm_address_info(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c1_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c1_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c1_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c1_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c1_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c1_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c1_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c1_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c1_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c1_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6DFA4ifWRyfmFySWZYan"
    "2wYX5yTmZKZlx6fnJ9XUpSfk5NaZATzHwD+USxt"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c1_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c1_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c1_sliding_controller2
    ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c1_sliding_controller2(void *chartInstanceVar)
{
  enable_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c1_sliding_controller2(void *chartInstanceVar)
{
  disable_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c1_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c1_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c1_sliding_controller2
    ((SFc1_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c1_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c1_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c1_sliding_controller2
      ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc1_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c1_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c1_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c1_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c1_sliding_controller2
      ((SFc1_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c1_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV91u40QUdqLsipWg6sVK3KygQivBDVIBIcEN29ZJtFm12yxOF4kLqun4JB5lPGPmx214h0W",
    "8wN7zANzwBEg8AVc8A+IJOGO7aerYTtOyS0G4ctKxv/PN+c45c2bitQYHHl4beP9w3/Pu4vcbeL",
    "e9/LpTjFsLd/68471fjF+gkbDxkCgSa6/xEiSGL0FLbg2TYiDGshLGxBgUCIrYRCpTx6ZZbDkT0",
    "74V1PHpryJGoyCSlod7aEvCQ8FnyJZYM0SeLlNATR8gNJGSdhL1OZnMPVbm1I+ATrWNmyRoMIFN",
    "nFv6wHLDEg69M6ADoQ1Bj/WFb4EhBnxzVivTKdXBOVDGCWdEVKqNiA4gwQAbOEpC/Dy0BkWVYTQ",
    "iyuxBRFLQ+2yacUoBZU6m8cUJE8RIxQjvxdx3hsu+DTn6cyBD4A0BQd/2FJBpIpkw9fkP+qi0J8",
    "gJhy6c2Ek9WwDfWpf85wxOQdXGbezLFBSZwKGonTQLSO8sy9a8SpZhhsXwnKhdivnTENZWL1aOD",
    "gjmCUZoUQeDTORAjxRLMby1bDYeuMpctWRsnCdbr4JlbL0UmrIwZ+tT4RPOdS1sJJN9SIFnrF1i",
    "SDMsZ63Gac3CkcQAu/KuXw1WMEx8AfOlCFllutISIOs7T7GxXEZSq42MfSze7v7+8utl2EAYUGN",
    "CoaoLKMI0YMyy8NazhUy73CMQvTKZe1XgvEJWoTw9tqJ7KtUUY9LQRC4kuIzWAmM9wVziSjjSuG",
    "iaYC6Xq3CU0AhC12AYhwNcNoitiIl2rW0X113KzKwLmiqWVGTV4qrDNtRzBTVL4EhMhTwVfSXjo",
    "OjxeXgBsNqIEkxM9rB/qVkfZ69KhdvPtr2L/ezNK+xn53bl7w8WeFoVPN7Cd3nee+3L+HZp3jb+",
    "tVq53c6C3VuleTolO4fbxPvJ7y8fPn3npz9/+fXD72c//vzNqvlbS/O3sv83CtA6+/9GMX5w3mj",
    "nCyddqleHfbzgV6eC/+0F/s1irJ9Fn4r068fR7jMyigfbn3z3+XZ64md8w06zv3dK/p4/33IdH6",
    "ssq09FB2FxMHFjYvPt2vF/tuDv3RXxuFc8z68/Ht3M/sFO2b6zQt9mMSZiyqFUv/+cjveurSNEI",
    "bdHx9b1dUzx0HBrdNzfKfeHKh2dSzo6npPw9/i/HMf17PP5hyv83yrlYSs7Lx4TtwvBMf3oWHMW",
    "4g5yTKUwSnIO6uONK/bvpv65rp33mu3+LX7+b/fq836V81D7mnadG56/XpfdTfWtey68bfimfcA",
    "r4TdvsY6bntdfNf43b71z9bvF+Iv5T2c/Yjys+BVVvMafROOqt/+BOv0LHBTDXQ==",
    ""
  };

  static char newstr [1341] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c1_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c1_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(614325834U));
  ssSetChecksum1(S,(4071890510U));
  ssSetChecksum2(S,(2385365703U));
  ssSetChecksum3(S,(1589225593U));
}

static void mdlRTW_c1_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c1_sliding_controller2(SimStruct *S)
{
  SFc1_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc1_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc1_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc1_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc1_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c1_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c1_sliding_controller2;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c1_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c1_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c1_sliding_controller2;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c1_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c1_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c1_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c1_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c1_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c1_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c1_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c1_JITStateAnimation,
    chartInstance->c1_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c1_sliding_controller2(chartInstance);
}

void c1_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c1_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c1_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c1_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c1_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
