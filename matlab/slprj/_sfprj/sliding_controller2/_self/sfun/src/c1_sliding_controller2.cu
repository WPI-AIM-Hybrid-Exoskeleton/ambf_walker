/* Include files */

#include "sliding_controller2_sfun.h"
#include "c1_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void enable_c1_sliding_controller2(SFc1_sliding_controller2InstanceStruct
  *chartInstance);
static void disable_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void c1_do_animation_call_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance, const mxArray *c1_st);
static void sf_gateway_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void c1_eML_blk_kernel(SFc1_sliding_controller2InstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2]);
static void c1_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2]);
static void c1_b_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2]);
static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller2, const
  char_T *c1_identifier);
static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static void init_dsm_address_info(SFc1_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc1_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c1_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c1_is_active_c1_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c1_sliding_controller2(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c1_do_animation_call_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c1_b_y = NULL;
  const mxArray *c1_c_y = NULL;
  const mxArray *c1_d_y = NULL;
  const mxArray *c1_e_y = NULL;
  const mxArray *c1_f_y = NULL;
  const mxArray *c1_st;
  const mxArray *c1_y = NULL;
  c1_st = NULL;
  c1_y = NULL;
  sf_mex_assign(&c1_y, sf_mex_createcellmatrix(5, 1), false);
  c1_b_y = NULL;
  sf_mex_assign(&c1_b_y, sf_mex_create("y", *chartInstance->c1_ankle, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 0, c1_b_y);
  c1_c_y = NULL;
  sf_mex_assign(&c1_c_y, sf_mex_create("y", *chartInstance->c1_danke, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 1, c1_c_y);
  c1_d_y = NULL;
  sf_mex_assign(&c1_d_y, sf_mex_create("y", *chartInstance->c1_dknee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 2, c1_d_y);
  c1_e_y = NULL;
  sf_mex_assign(&c1_e_y, sf_mex_create("y", *chartInstance->c1_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 3, c1_e_y);
  c1_f_y = NULL;
  sf_mex_assign(&c1_f_y, sf_mex_create("y",
    &chartInstance->c1_is_active_c1_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c1_y, 4, c1_f_y);
  sf_mex_assign(&c1_st, c1_y, false);
  return c1_st;
}

static void set_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance, const mxArray *c1_st)
{
  const mxArray *c1_u;
  c1_u = sf_mex_dup(c1_st);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 0)),
                      "ankle", *chartInstance->c1_ankle);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 1)),
                      "danke", *chartInstance->c1_danke);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 2)),
                      "dknee", *chartInstance->c1_dknee);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 3)), "knee",
                      *chartInstance->c1_knee);
  chartInstance->c1_is_active_c1_sliding_controller2 = c1_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 4)),
     "is_active_c1_sliding_controller2");
  sf_mex_destroy(&c1_u);
  sf_mex_destroy(&c1_st);
}

static void sf_gateway_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c1_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c1_eML_blk_kernel(chartInstance, *chartInstance->c1_q, *chartInstance->c1_qd, *
                    chartInstance->c1_knee, *chartInstance->c1_dknee,
                    *chartInstance->c1_ankle, *chartInstance->c1_danke);
  c1_do_animation_call_c1_sliding_controller2(chartInstance);
}

static void mdl_start_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c1_errCode;
  c1_errCode = hipGetLastError();
  if (c1_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c1_errCode, (char_T *)hipGetErrorName
                       (c1_errCode), (char_T *)hipGetErrorString(c1_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c1_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c1_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c1_nameCaptureInfo = NULL;
  const char_T *c1_data[4] = {
    "789cc553cb4ac34014bd915adc545df90b2e04c7baeabe2a88580a0d6e44e224b969c6cc432653dbbaf10fec57f80ffe8e7b3fc2246dfa088688c57a20dc399c"
    "cc3df71e18b02eaf2d00d84d3efb05a055870c8d6981fd59dd82551475aba4e6d886dacabd54ef267e9319f7943438325322a9c0f94d5f0926a934f6f8114163",
    "acf813fa9912308e3613d85b269d94898b25694e52293db743f4a2de40800ee3c5847c996479a4b82fd9b756914711c53c8affe57ea35ffae5fd0f2afc723d50"
    "7a48b5ef444c32410df36287637ffdbd73df7ae91c53c5570397e3c2ef7d4d3fbbd46f55bf3dbf23a11248243521950c795f713fa0da251e35491ace3026b1f6",
    "08156ee00c298f509324204e5df27d66c722ed5b95dbde0ff7287b3f0dd8c9eadbeb11dda4dfc7e1a7b549bf1cffe5b7a9f777753266cd67f61045a7ad76a7e9"
    "bade8d6fce1673742b7caae68012fed7fdbf00b09967db", "" };

  c1_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c1_data[0], 1616U, &c1_nameCaptureInfo);
  return c1_nameCaptureInfo;
}

static void c1_eML_blk_kernel(SFc1_sliding_controller2InstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2])
{
  real_T c1_a;
  real_T c1_b_a;
  c1_b_knee[0] = -0.324 * muDoubleScalarSin(c1_b_q[0]);
  c1_b_knee[1] = -0.324 * muDoubleScalarCos(c1_b_q[0]);
  c1_a = -0.324 * c1_b_qd[0];
  c1_b_dknee[0] = c1_a * muDoubleScalarCos(c1_b_q[0]);
  c1_b_dknee[1] = c1_a * -muDoubleScalarSin(c1_b_q[0]);
  c1_b_a = -0.39 * (c1_b_qd[0] + c1_b_qd[1]);
  c1_b_ankle[0] = c1_b_knee[0] + -0.39 * muDoubleScalarSin(c1_b_q[0] + c1_b_q[1]);
  c1_b_ankle[1] = c1_b_knee[1] + -0.39 * muDoubleScalarCos(c1_b_q[0] + c1_b_q[1]);
  c1_b_danke[0] = c1_b_dknee[0] + c1_b_a * muDoubleScalarCos(c1_b_q[0] + c1_b_q
    [1]);
  c1_b_danke[1] = c1_b_dknee[1] + c1_b_a * -muDoubleScalarSin(c1_b_q[0] +
    c1_b_q[1]);
}

static void c1_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2])
{
  emlrtMsgIdentifier c1_thisId;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_b_ankle), &c1_thisId, c1_y);
  sf_mex_destroy(&c1_b_ankle);
}

static void c1_b_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2])
{
  real_T c1_dv[2];
  int32_T c1_i;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), c1_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c1_i = 0; c1_i < 2; c1_i++) {
    c1_y[c1_i] = c1_dv[c1_i];
  }

  sf_mex_destroy(&c1_u);
}

static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller2, const
  char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  uint8_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c1_b_is_active_c1_sliding_controller2), &c1_thisId);
  sf_mex_destroy(&c1_b_is_active_c1_sliding_controller2);
  return c1_y;
}

static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  uint8_T c1_b_u;
  uint8_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_b_u, 1, 3, 0U, 0, 0U, 0);
  c1_y = c1_b_u;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static void init_dsm_address_info(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c1_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c1_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c1_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c1_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c1_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c1_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c1_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c1_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c1_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c1_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6DFA4ifWRyfmFySWZYan"
    "2wYX5yTmZKZlx6fnJ9XUpSfk5NaZATzHwD+USxt"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c1_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c1_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c1_sliding_controller2
    ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c1_sliding_controller2(void *chartInstanceVar)
{
  enable_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c1_sliding_controller2(void *chartInstanceVar)
{
  disable_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c1_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c1_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c1_sliding_controller2
    ((SFc1_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c1_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c1_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c1_sliding_controller2
      ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc1_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c1_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c1_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c1_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c1_sliding_controller2
      ((SFc1_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c1_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV91u40QUdqLsipWWqhcrcbMSFdIKLgsICW6grZNoI7XbLE4XCSFVk/FJPMp4xsyP28I+ARL",
    "SXu+77HvsM3ADEpdwwRnbTVPHdpqWXQrClZOO/Z1vznfOmTMTrzU48PDawPvFA8+7i9/v4N328u",
    "tOMW4t3PnzjvdhMf4ZjYSNh0SRWHuNlyAxfA1acmuYFAMxkZUwJiagQFDEJlKZOjbNYsuZmPWto",
    "I5PfxMxGgWRtDzcQ1sSHgp+hmyJNUPk6TIF1PQBQhMpaadRn5Pp3GNlTvwI6EzbuEmCBhPYxLml",
    "Dyw3LOHQOwU6ENoQ9Fhf+BYYYsA3p7UynVIdnANlnHBGRKXaiOgAEgywgaMkxM9Da1BUGUYjosw",
    "eRCQFvc9mGacUUOZkGl+MmSBGKkZ4L+a+M1z2bcjRnwMZAm8ICPq2p4DMEsmEqc9/0EelPUHGHL",
    "owttN6tgC+ty75zxicgKqN28SXKSgyhUNRO2kWkN5plq15lSzDDIvhGVG7FPOnIaytXqwcHRDME",
    "4zQog4GmciBHimWYnhr2Ww8cJW5asnYOE+2XgXL2HopNGVhztanwiec61rYSCb7kALPWLvEkGZY",
    "zlqN05qFI4kBduVdvxqsYJj4AuZLEbLKdKUlQNZ3nmBjuYykVhsZ+1i83f395dfLsIEwoCaEQlU",
    "XUIRpwJhl4a1nC5l2uUcgemUy96rAeYWsQnl6YkX3RKoZxqShiVxIcBmtBcZ6irnElXCkcdE0wV",
    "wuV+EooRGErsEwDge4bBBbERPtWtsurruUmbMuaKpYUpFVi6sO21DPFdRZAkdiJuSJ6CsZB0WPz",
    "8MLgNVGlGBiuof9S531cfaqVLj9bNu72M/uX2E/O7crf3+0wNOq4PEWvsvz3mtfxrdL87bxr9XK",
    "7XYW7N4tzdMp2TncJt6//Pn7o+e//jT+8XX4/LuXr/5YNX9raf5W9v9GAVpn/98oxg/PG+184aR",
    "L9eqwjxf86lTwv7fAv1mM9dPoM5F++zjafUpG8WD70x++2E7HfsY37DT7e6fk7/nzLdfxscqy+l",
    "R0EBYHEzcmNt+uHf/nC/7eXRGPe8Xz/Prtq5vZP9wp23dW6NssxkTMOJTq95/T8cG1dYQo5Pbo2",
    "Lq+jhkeGm6Njgc75f5QpaNzSUfHcxL+Hv+X47iefT7/cIX/W6U8bGXnxWPidiE4ph8fa85C3EGO",
    "qRRGSc5BfbJxxf7d1D/XtfPest2/xc//7d583q9yHmpf065zw/PX27K7qb51z4W3Dd+0D3gl/OY",
    "t1nHT8/qbxr/21jtXv1+Mv5z/dPYjxsOKX1HFa/xJNKl6+x+o078AKfnFFA==",
    ""
  };

  static char newstr [1337] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c1_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c1_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(636746986U));
  ssSetChecksum1(S,(1653272444U));
  ssSetChecksum2(S,(2086983803U));
  ssSetChecksum3(S,(4223507036U));
}

static void mdlRTW_c1_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c1_sliding_controller2(SimStruct *S)
{
  SFc1_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc1_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc1_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc1_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc1_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c1_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c1_sliding_controller2;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c1_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c1_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c1_sliding_controller2;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c1_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c1_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c1_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c1_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c1_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c1_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c1_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c1_JITStateAnimation,
    chartInstance->c1_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c1_sliding_controller2(chartInstance);
}

void c1_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c1_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c1_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c1_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c1_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
