/* Include files */

#include "sliding_controller2_sfun.h"
#include "c1_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void enable_c1_sliding_controller2(SFc1_sliding_controller2InstanceStruct
  *chartInstance);
static void disable_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void c1_do_animation_call_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance, const mxArray *c1_st);
static void sf_gateway_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance);
static void c1_eML_blk_kernel(SFc1_sliding_controller2InstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2]);
static void c1_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2]);
static void c1_b_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2]);
static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller2, const
  char_T *c1_identifier);
static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static void init_dsm_address_info(SFc1_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc1_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c1_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c1_is_active_c1_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c1_sliding_controller2(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c1_do_animation_call_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c1_b_y = NULL;
  const mxArray *c1_c_y = NULL;
  const mxArray *c1_d_y = NULL;
  const mxArray *c1_e_y = NULL;
  const mxArray *c1_f_y = NULL;
  const mxArray *c1_st;
  const mxArray *c1_y = NULL;
  c1_st = NULL;
  c1_y = NULL;
  sf_mex_assign(&c1_y, sf_mex_createcellmatrix(5, 1), false);
  c1_b_y = NULL;
  sf_mex_assign(&c1_b_y, sf_mex_create("y", *chartInstance->c1_ankle, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 0, c1_b_y);
  c1_c_y = NULL;
  sf_mex_assign(&c1_c_y, sf_mex_create("y", *chartInstance->c1_danke, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 1, c1_c_y);
  c1_d_y = NULL;
  sf_mex_assign(&c1_d_y, sf_mex_create("y", *chartInstance->c1_dknee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 2, c1_d_y);
  c1_e_y = NULL;
  sf_mex_assign(&c1_e_y, sf_mex_create("y", *chartInstance->c1_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c1_y, 3, c1_e_y);
  c1_f_y = NULL;
  sf_mex_assign(&c1_f_y, sf_mex_create("y",
    &chartInstance->c1_is_active_c1_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c1_y, 4, c1_f_y);
  sf_mex_assign(&c1_st, c1_y, false);
  return c1_st;
}

static void set_sim_state_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance, const mxArray *c1_st)
{
  const mxArray *c1_u;
  c1_u = sf_mex_dup(c1_st);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 0)),
                      "ankle", *chartInstance->c1_ankle);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 1)),
                      "danke", *chartInstance->c1_danke);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 2)),
                      "dknee", *chartInstance->c1_dknee);
  c1_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 3)), "knee",
                      *chartInstance->c1_knee);
  chartInstance->c1_is_active_c1_sliding_controller2 = c1_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 4)),
     "is_active_c1_sliding_controller2");
  sf_mex_destroy(&c1_u);
  sf_mex_destroy(&c1_st);
}

static void sf_gateway_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c1_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c1_eML_blk_kernel(chartInstance, *chartInstance->c1_q, *chartInstance->c1_qd, *
                    chartInstance->c1_knee, *chartInstance->c1_dknee,
                    *chartInstance->c1_ankle, *chartInstance->c1_danke);
  c1_do_animation_call_c1_sliding_controller2(chartInstance);
}

static void mdl_start_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c1_errCode;
  c1_errCode = hipGetLastError();
  if (c1_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c1_errCode, (char_T *)hipGetErrorName
                       (c1_errCode), (char_T *)hipGetErrorString(c1_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c1_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc1_sliding_controller2
  (SFc1_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c1_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c1_nameCaptureInfo = NULL;
  const char_T *c1_data[4] = {
    "789cc553cb4ac34014bd95b6e8a2eacaad7fe08082b8562a58a8545af085a493e4a6193b0f9da4a65df909fe8a1bc1bff067fc00f368da26182216eb8170e770"
    "32f7dc7b60a072d6ae00c066f86d3c031cd521462329b03dad6b90455eaf14d41435a866ee457a2df47b99724b491fc77e42241538bb692bc124957e6ff280a0",
    "d153fc09ed587118c71e13d85d24e71113a70bd28c4452743e71d11a764702b4ebcd27e48b24ce2342bf60df6a491e79e4f3c8ff97fa8d7fe997f6df29f14b75"
    "47e9806adb1832c904f599e5191c07cbef9dfad60be748145b8d4c8e73bff725fd7a857e59fdb679475c259048eabb5432e403c56d876a9358d40fd330028f78",
    "da2254988e11503e444dc2803835c9f799ed89a86f596e5b3fdca3e8fd34603daee4f3b5bf4abf8fb7fbdd55faa5f82fbf55bd3fab134c9a97fb37faa2ed1c1e"
    "5cf1c7eb564b1ecfe7e894f894cd0105fcaffb7f01900e67d7", "" };

  c1_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c1_data[0], 1616U, &c1_nameCaptureInfo);
  return c1_nameCaptureInfo;
}

static void c1_eML_blk_kernel(SFc1_sliding_controller2InstanceStruct
  *chartInstance, real_T c1_b_q[2], real_T c1_b_qd[2], real_T c1_b_knee[2],
  real_T c1_b_dknee[2], real_T c1_b_ankle[2], real_T c1_b_danke[2])
{
  real_T c1_a;
  real_T c1_b_a;
  c1_b_knee[0] = 0.324 * muDoubleScalarCos(c1_b_q[0]);
  c1_b_knee[1] = 0.324 * muDoubleScalarSin(c1_b_q[0]);
  c1_a = 0.324 * c1_b_qd[0];
  c1_b_dknee[0] = c1_a * -muDoubleScalarSin(c1_b_q[0]);
  c1_b_dknee[1] = c1_a * muDoubleScalarCos(c1_b_q[0]);
  c1_b_a = 0.39 * (c1_b_qd[0] + c1_b_qd[1]);
  c1_b_ankle[0] = c1_b_knee[0] + 0.39 * muDoubleScalarCos(c1_b_q[0] + c1_b_q[1]);
  c1_b_ankle[1] = c1_b_knee[1] + 0.39 * muDoubleScalarSin(c1_b_q[0] + c1_b_q[1]);
  c1_b_danke[0] = c1_b_dknee[0] + c1_b_a * -muDoubleScalarSin(c1_b_q[0] +
    c1_b_q[1]);
  c1_b_danke[1] = c1_b_dknee[1] + c1_b_a * muDoubleScalarCos(c1_b_q[0] + c1_b_q
    [1]);
}

static void c1_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_ankle, const char_T *c1_identifier, real_T
  c1_y[2])
{
  emlrtMsgIdentifier c1_thisId;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_b_ankle), &c1_thisId, c1_y);
  sf_mex_destroy(&c1_b_ankle);
}

static void c1_b_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId,
  real_T c1_y[2])
{
  real_T c1_dv[2];
  int32_T c1_i;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), c1_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c1_i = 0; c1_i < 2; c1_i++) {
    c1_y[c1_i] = c1_dv[c1_i];
  }

  sf_mex_destroy(&c1_u);
}

static uint8_T c1_c_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_b_is_active_c1_sliding_controller2, const
  char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  uint8_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c1_b_is_active_c1_sliding_controller2), &c1_thisId);
  sf_mex_destroy(&c1_b_is_active_c1_sliding_controller2);
  return c1_y;
}

static uint8_T c1_d_emlrt_marshallIn(SFc1_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  uint8_T c1_b_u;
  uint8_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_b_u, 1, 3, 0U, 0, 0U, 0);
  c1_y = c1_b_u;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static void init_dsm_address_info(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc1_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c1_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c1_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c1_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c1_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c1_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c1_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c1_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c1_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c1_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c1_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c1_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiAOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6DFA4ifWRyfmFySWZYan"
    "2wYX5yTmZKZlx6fnJ9XUpSfk5NaZATzHwD+USxt"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c1_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c1_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c1_sliding_controller2
    ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c1_sliding_controller2(void *chartInstanceVar)
{
  enable_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c1_sliding_controller2(void *chartInstanceVar)
{
  disable_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c1_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c1_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c1_sliding_controller2
    ((SFc1_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c1_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c1_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c1_sliding_controller2
      ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc1_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc1_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c1_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c1_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c1_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c1_sliding_controller2
      ((SFc1_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc1_sliding_controller2((SFc1_sliding_controller2InstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c1_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV91u40QUdqLsipUg6sVK3Ky0FTfsZQEhARdLWyfRWmppFqeLhISq6fgkHmU8Y+bHbXmHlXg",
    "AXoFn4RKueAYuueSM7aapYztNyy4F4cpJx/7ON+c758yZidcJDj28+nj/+NjzHuL3O3h3veJ6UI",
    "47S3fxvOd9WI5fo5GwyZgokmiv9RIkga9BS24NkyIQU1kLY2IKCgRFbCqVaWLTLLGcifnICur49",
    "Dcxo3EYS8ujfbQl0ZHgF8iWWjNGngFTQM0IIDKxknYWjziZLTxW5syPgc61TdokaDChTZ1b+tBy",
    "w1IOw3OggdCGoMf6yrfQEAO+OW+U6ZTq8BIok5QzImrVxkSHkGKADRynEX4eWYOiqjAaE2X2ISY",
    "Z6AM2zzmlgCon0/jilAlipGKEDxPuO8NV38Yc/TmUEfCWgKBv+wrIPJVMmOb8hyNUOhTklMMATu",
    "2smS2E761L/isGZ6Aa4zb1ZQaKzOBINE6aB2R4nmdrUSWrMMMSeEXUHsX8aYgaqxcrR4cE8wQTt",
    "GiCQS4y0BPFMgxvI5tNAleZ65aMTYpk63WwnG2YQVsWFmwjKnzCuW6ETWR6ABnwnHVADGmHFaz1",
    "OK1ZNJEYYFfezavBCoaJL2G+FBGrTVdWAeR95ytsLNeR1GojEx+Ld3BwsPp6FRYIA2pKKNR1AUW",
    "YBoxZHt5mtohpl3sEolcmd68OXFTIOpSnp1YMzqSaY0xamsiVBJfRRmCiZ5hLXAnHGhdNG8zlch",
    "2OEhpD5BoM43CIywaxNTHRrrXt4brLmLkYgKaKpTVZtbjqsA0NXUFdpHAs5kKeiZGSSVj2+CK8A",
    "FhtRAkmZvvYv9TFCGevS4Xbz3a8q/3s3RvsZ5d21e9nSzydGh5v6bs676PudXy3Mm8X/zqdwm53",
    "ye69yjy9ip3DbeH9c9B/9svrX8ng95+ef/Gd9+e6+Tsr83fy//slaJP9v1+On1w22sXCyVbq1WF",
    "fLPnVq+F/f4l/qxzrl/GnIvv2Rbz3kkySYOeTHz7fyU79nG/ca/f3QcXfy+fbruNjleX1qWgQlQ",
    "cTNya22K4d/2dL/j5cE49H5fPi+uPLu9k/2a3a99bo2yrHRMw5VOr3n9Pxwa11RCjk/ujYvr2OO",
    "R4a7o2Ox7vV/lCno3dNR89zEv4e/1fjuJl9Mf94jf/blTxs5+fFE+J2ITihH51oziLcQU6oFEZJ",
    "zkF93L9h/27rn5vaeW/Z7t/i5/92bz7vNzkPdW9p17vj+ett2d1V36bnwvuGb9sHvAp+6x7ruOt",
    "5/U3jf/M2O1c/LcfPFz+d/ZjxqOZXVPkafxJN697+B+r0L6iawik=",
    ""
  };

  static char newstr [1329] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c1_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c1_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(672025005U));
  ssSetChecksum1(S,(1641516495U));
  ssSetChecksum2(S,(1050467908U));
  ssSetChecksum3(S,(4160773434U));
}

static void mdlRTW_c1_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c1_sliding_controller2(SimStruct *S)
{
  SFc1_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc1_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc1_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc1_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc1_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c1_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c1_sliding_controller2;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c1_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c1_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c1_sliding_controller2;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c1_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c1_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c1_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c1_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c1_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c1_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c1_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c1_JITStateAnimation,
    chartInstance->c1_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c1_sliding_controller2(chartInstance);
}

void c1_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c1_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c1_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c1_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c1_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
