/* Include files */

#include "sliding_controller2_sfun.h"
#include "c10_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void enable_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void disable_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void c10_do_animation_call_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance, const mxArray *c10_st);
static void sf_gateway_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void c10_eML_blk_kernel(SFc10_sliding_controller2InstanceStruct
  *chartInstance, real_T c10_b_q[2], real_T c10_b_qd[2], real_T c10_b_knee[2],
  real_T c10_b_dknee[2], real_T c10_b_ankle[2], real_T c10_b_danke[2]);
static void c10_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_b_ankle, const char_T *c10_identifier,
  real_T c10_y[2]);
static void c10_b_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId,
  real_T c10_y[2]);
static uint8_T c10_c_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_b_is_active_c10_sliding_controller2, const
  char_T *c10_identifier);
static uint8_T c10_d_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId);
static void init_dsm_address_info(SFc10_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc10_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c10_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c10_is_active_c10_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c10_do_animation_call_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c10_b_y = NULL;
  const mxArray *c10_c_y = NULL;
  const mxArray *c10_d_y = NULL;
  const mxArray *c10_e_y = NULL;
  const mxArray *c10_f_y = NULL;
  const mxArray *c10_st;
  const mxArray *c10_y = NULL;
  c10_st = NULL;
  c10_y = NULL;
  sf_mex_assign(&c10_y, sf_mex_createcellmatrix(5, 1), false);
  c10_b_y = NULL;
  sf_mex_assign(&c10_b_y, sf_mex_create("y", *chartInstance->c10_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 0, c10_b_y);
  c10_c_y = NULL;
  sf_mex_assign(&c10_c_y, sf_mex_create("y", *chartInstance->c10_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 1, c10_c_y);
  c10_d_y = NULL;
  sf_mex_assign(&c10_d_y, sf_mex_create("y", *chartInstance->c10_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 2, c10_d_y);
  c10_e_y = NULL;
  sf_mex_assign(&c10_e_y, sf_mex_create("y", *chartInstance->c10_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c10_y, 3, c10_e_y);
  c10_f_y = NULL;
  sf_mex_assign(&c10_f_y, sf_mex_create("y",
    &chartInstance->c10_is_active_c10_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c10_y, 4, c10_f_y);
  sf_mex_assign(&c10_st, c10_y, false);
  return c10_st;
}

static void set_sim_state_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance, const mxArray *c10_st)
{
  const mxArray *c10_u;
  c10_u = sf_mex_dup(c10_st);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 0)),
                       "ankle", *chartInstance->c10_ankle);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 1)),
                       "danke", *chartInstance->c10_danke);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 2)),
                       "dknee", *chartInstance->c10_dknee);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 3)),
                       "knee", *chartInstance->c10_knee);
  chartInstance->c10_is_active_c10_sliding_controller2 = c10_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 4)),
     "is_active_c10_sliding_controller2");
  sf_mex_destroy(&c10_u);
  sf_mex_destroy(&c10_st);
}

static void sf_gateway_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c10_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c10_eML_blk_kernel(chartInstance, *chartInstance->c10_q,
                     *chartInstance->c10_qd, *chartInstance->c10_knee,
                     *chartInstance->c10_dknee, *chartInstance->c10_ankle,
                     *chartInstance->c10_danke);
  c10_do_animation_call_c10_sliding_controller2(chartInstance);
}

static void mdl_start_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c10_errCode;
  c10_errCode = hipGetLastError();
  if (c10_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c10_errCode, (char_T *)hipGetErrorName
                       (c10_errCode), (char_T *)hipGetErrorString(c10_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c10_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c10_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c10_nameCaptureInfo = NULL;
  const char_T *c10_data[4] = {
    "789cc553cb4ac34014bd915adc545df90b2e04c7baeabe2a88580a0d6e44e224b969c6cc432653dbbaf10fec57f80ffe8e7b3fc2246dfa088688c57a20dc399c"
    "cc3df71e18b02eaf2d00d84d3efb05a055870c8d6981fd59dd82551475aba4e6d886dacabd54ef267e9319f7943438325322a9c0f94d5f0926a934f6f8114163",
    "acf813fa9912308e3613d85b269d94898b25694e52293db743f4a2de40800ee3c5847c996479a4b82fd9b756914711c53c8affe57ea35ffae5fd0f2afc723d50"
    "7a48b5ef444c32410df36287637ffdbd73df7ae91c53c5570397e3c2ef7d4d3fbbd46f55bf3dbf23a11248243521950c795f713fa0da251e35491ace3026b1f6",
    "08156ee00c298f509324204e5df27d66c722ed5b95dbde0ff7287b3f0dd8c9eadbeb11dda4dfc7e1a7b549bf1cffe5b7a9f777753266cd67f61045a7ad76a7e9"
    "bade8d6fce1673742b7caae68012fed7fdbf00b09967db", "" };

  c10_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c10_data[0], 1616U, &c10_nameCaptureInfo);
  return c10_nameCaptureInfo;
}

static void c10_eML_blk_kernel(SFc10_sliding_controller2InstanceStruct
  *chartInstance, real_T c10_b_q[2], real_T c10_b_qd[2], real_T c10_b_knee[2],
  real_T c10_b_dknee[2], real_T c10_b_ankle[2], real_T c10_b_danke[2])
{
  real_T c10_a;
  real_T c10_b_a;
  c10_b_knee[0] = -0.324 * muDoubleScalarSin(c10_b_q[0]);
  c10_b_knee[1] = -0.324 * muDoubleScalarCos(c10_b_q[0]);
  c10_a = -0.324 * c10_b_qd[0];
  c10_b_dknee[0] = c10_a * muDoubleScalarCos(c10_b_q[0]);
  c10_b_dknee[1] = c10_a * -muDoubleScalarSin(c10_b_q[0]);
  c10_b_a = -0.39 * (c10_b_qd[0] + c10_b_qd[1]);
  c10_b_ankle[0] = c10_b_knee[0] + -0.39 * muDoubleScalarSin(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_ankle[1] = c10_b_knee[1] + -0.39 * muDoubleScalarCos(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_danke[0] = c10_b_dknee[0] + c10_b_a * muDoubleScalarCos(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_danke[1] = c10_b_dknee[1] + c10_b_a * -muDoubleScalarSin(c10_b_q[0] +
    c10_b_q[1]);
}

static void c10_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_b_ankle, const char_T *c10_identifier,
  real_T c10_y[2])
{
  emlrtMsgIdentifier c10_thisId;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c10_b_ankle), &c10_thisId,
    c10_y);
  sf_mex_destroy(&c10_b_ankle);
}

static void c10_b_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId,
  real_T c10_y[2])
{
  real_T c10_dv[2];
  int32_T c10_i;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), c10_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c10_i = 0; c10_i < 2; c10_i++) {
    c10_y[c10_i] = c10_dv[c10_i];
  }

  sf_mex_destroy(&c10_u);
}

static uint8_T c10_c_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_b_is_active_c10_sliding_controller2, const
  char_T *c10_identifier)
{
  emlrtMsgIdentifier c10_thisId;
  uint8_T c10_y;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_y = c10_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c10_b_is_active_c10_sliding_controller2), &c10_thisId);
  sf_mex_destroy(&c10_b_is_active_c10_sliding_controller2);
  return c10_y;
}

static uint8_T c10_d_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId)
{
  uint8_T c10_b_u;
  uint8_T c10_y;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), &c10_b_u, 1, 3, 0U, 0, 0U, 0);
  c10_y = c10_b_u;
  sf_mex_destroy(&c10_u);
  return c10_y;
}

static void init_dsm_address_info(SFc10_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc10_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c10_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c10_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c10_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c10_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c10_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c10_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c10_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c10_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c10_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c10_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c10_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c10_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6LFA4ifWRyfmFySWZYan"
    "2xoEF+ck5mSmZcen5yfV1KUn5OTWmSEZD4IAACGgSyv"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c10_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c10_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c10_sliding_controller2
    ((SFc10_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c10_sliding_controller2(void *chartInstanceVar)
{
  enable_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c10_sliding_controller2(void *chartInstanceVar)
{
  disable_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c10_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c10_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c10_sliding_controller2
    ((SFc10_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c10_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c10_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc10_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c10_sliding_controller2
      ((SFc10_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc10_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc10_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c10_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c10_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c10_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c10_sliding_controller2
      ((SFc10_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc10_sliding_controller2
      ((SFc10_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c10_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFALwHyAxTt0W1QIL20tikJEWDXTiinQVDAWC1H4kLLXWZ/aLvNExT",
    "ovei79D3yDL20QI/NobMkLcsUSUV2krpFaVDykt98O9/M7OzKaw12Pbxu4v3zLc+7ht/X8W57+b",
    "VWjFtzd/68431SjH9CI2HjfaJIrL3GS5AYnoCW3BomxUCMZSWMiTEoEBSxiVSmjk2z2HImpn0rq",
    "OPT30aMRkEkLQ+30ZaEe4KfIFtizT7ydJkCavoAoYmUtJOoz8lk5rEyR34EdKpt3CRBgwls4tzS",
    "u5YblnDoHQMdCG0IeqzPfAsMMeCb41qZTqkOToEyTjgjolJtRHQACQbYwEES4ueeNSiqDKMRUWY",
    "bIpKC3mHTjFMKKHMyjS9GTBAjFSO8F3PfGS76ts/Rn10ZAm8ICPq2rYBME8mEqc9/0EelPUFGHL",
    "owspN6tgBeWJf8pwyOQNXGbezLFBSZwJ6onTQLSO84y9asShZhhsXwlKgtivnTENZWL1aODgjmC",
    "YZoUQeDTORADxVLMby1bDYeuMpctmRsnCdbL4NlbL0UmrIwY+tT4RPOdS1sKJMdSIFnrF1iSDMs",
    "Z63Gac3CocQAu/KuXw1WMEx8AfOlCFllutISIOs732BjOY+kVhsZ+1i83Z2dxdeLsIEwoMaEQlU",
    "XUIRpwJhl4a1nC5l2uUcgemUy96rAeYUsQ3l6bEX3SKopxqShiZxJcBmtBcZ6grnElXCgcdE0wV",
    "wul+EooRGErsEwDru4bBBbERPtWtsWrruUmZMuaKpYUpFVi6sO21DPFdRJAgdiKuSR6CsZB0WPz",
    "8MLgNVGlGBiso39S530cfaqVLj9bMM7288+eIP97NSu/P3pHE+rgseb+y7Pe6N9Ht8uzdvGv1Yr",
    "t9ucs/uwNE+nZOdw63j/9vrPj1/+/uPoh1fhy+9++fWvZfO3FuZvZf87u+trq+3/N4vx7dNGO1s",
    "46UK9OuyjOb86FfwfzfGvF2P9OPpCpM8fRVuPyTAebDz4/suNdORnfM86zf6ulfw9fX7XdXyssq",
    "w+FR2ExcHEjYnNt2vH/3DO32tL4nGjeJ5ff3x9Ofvbm2X7zhJ968WYiCmHUv3+czruX1hHiEKuj",
    "o67F9cxxUPDldFxa7PcH6p0dM7p6HhOwtvxfzGOq9nn8z9b4v+9Uh7uZefFQ+J2ITikn20cas5C",
    "3EIOqRRGSc5BfV7R9y/aR1e1896z3b/Fz//t3n3e3+Rc1L6gXeeS57D3ZXdZfaueD68avmk/8Er",
    "49Sus47Ln9neNf+Wtdr6+U4y/mv2E9iPGw4pfU8Vr/Gk0rnr7H6jTvwFAG8Vm",
    ""
  };

  static char newstr [1337] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c10_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c10_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(636746986U));
  ssSetChecksum1(S,(1653272444U));
  ssSetChecksum2(S,(2086983803U));
  ssSetChecksum3(S,(4223507036U));
}

static void mdlRTW_c10_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c10_sliding_controller2(SimStruct *S)
{
  SFc10_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc10_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc10_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc10_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc10_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c10_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c10_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c10_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c10_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c10_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c10_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c10_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c10_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c10_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c10_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c10_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c10_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c10_JITStateAnimation,
    chartInstance->c10_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c10_sliding_controller2(chartInstance);
}

void c10_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c10_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c10_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c10_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c10_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
