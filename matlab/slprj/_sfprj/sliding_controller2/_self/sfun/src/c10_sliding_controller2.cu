/* Include files */

#include "sliding_controller2_sfun.h"
#include "c10_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void enable_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void disable_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void c10_do_animation_call_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance, const mxArray *c10_st);
static void sf_gateway_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance);
static void c10_eML_blk_kernel(SFc10_sliding_controller2InstanceStruct
  *chartInstance, real_T c10_b_q[2], real_T c10_b_qd[2], real_T c10_b_knee[2],
  real_T c10_b_dknee[2], real_T c10_b_ankle[2], real_T c10_b_danke[2]);
static void c10_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_b_ankle, const char_T *c10_identifier,
  real_T c10_y[2]);
static void c10_b_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId,
  real_T c10_y[2]);
static uint8_T c10_c_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_b_is_active_c10_sliding_controller2, const
  char_T *c10_identifier);
static uint8_T c10_d_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId);
static void init_dsm_address_info(SFc10_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc10_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c10_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c10_is_active_c10_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c10_do_animation_call_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c10_b_y = NULL;
  const mxArray *c10_c_y = NULL;
  const mxArray *c10_d_y = NULL;
  const mxArray *c10_e_y = NULL;
  const mxArray *c10_f_y = NULL;
  const mxArray *c10_st;
  const mxArray *c10_y = NULL;
  c10_st = NULL;
  c10_y = NULL;
  sf_mex_assign(&c10_y, sf_mex_createcellmatrix(5, 1), false);
  c10_b_y = NULL;
  sf_mex_assign(&c10_b_y, sf_mex_create("y", *chartInstance->c10_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 0, c10_b_y);
  c10_c_y = NULL;
  sf_mex_assign(&c10_c_y, sf_mex_create("y", *chartInstance->c10_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 1, c10_c_y);
  c10_d_y = NULL;
  sf_mex_assign(&c10_d_y, sf_mex_create("y", *chartInstance->c10_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c10_y, 2, c10_d_y);
  c10_e_y = NULL;
  sf_mex_assign(&c10_e_y, sf_mex_create("y", *chartInstance->c10_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c10_y, 3, c10_e_y);
  c10_f_y = NULL;
  sf_mex_assign(&c10_f_y, sf_mex_create("y",
    &chartInstance->c10_is_active_c10_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c10_y, 4, c10_f_y);
  sf_mex_assign(&c10_st, c10_y, false);
  return c10_st;
}

static void set_sim_state_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance, const mxArray *c10_st)
{
  const mxArray *c10_u;
  c10_u = sf_mex_dup(c10_st);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 0)),
                       "ankle", *chartInstance->c10_ankle);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 1)),
                       "danke", *chartInstance->c10_danke);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 2)),
                       "dknee", *chartInstance->c10_dknee);
  c10_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 3)),
                       "knee", *chartInstance->c10_knee);
  chartInstance->c10_is_active_c10_sliding_controller2 = c10_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c10_u, 4)),
     "is_active_c10_sliding_controller2");
  sf_mex_destroy(&c10_u);
  sf_mex_destroy(&c10_st);
}

static void sf_gateway_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c10_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c10_eML_blk_kernel(chartInstance, *chartInstance->c10_q,
                     *chartInstance->c10_qd, *chartInstance->c10_knee,
                     *chartInstance->c10_dknee, *chartInstance->c10_ankle,
                     *chartInstance->c10_danke);
  c10_do_animation_call_c10_sliding_controller2(chartInstance);
}

static void mdl_start_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c10_errCode;
  c10_errCode = hipGetLastError();
  if (c10_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c10_errCode, (char_T *)hipGetErrorName
                       (c10_errCode), (char_T *)hipGetErrorString(c10_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c10_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc10_sliding_controller2
  (SFc10_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c10_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c10_nameCaptureInfo = NULL;
  const char_T *c10_data[4] = {
    "789cc5534d4bc340109d4a2d5eaa9e04c1ab170fae5ec4b3c5428596da0f3c88c649b26996ee876c52db7af1ee41ff8e7fcaff6092364d130c118bf541987dbc"
    "ecbc99070ba546b30400dbc1f7f602705e8108d55981dd79dd8034b27a29a7c6d88472ea5ea8bf067eef736e29e9d3893f2312055ddcb4956012a5df9b3e52d0",
    "d453fc89da91e2304e7b4cd0ee3269854cd497a40509a5f05c73a935ec8e0468d74b26e4cb24ca23c443cebee5823cb2c8e691fd2ff69bfcd22feebf57e017eb"
    "8ed263d4b631649209f499e5199c0e56df3bf6ade4ce31536c3532394dfc3e56f4ebe5faa5f5dbcb3be22a418944df45c9281f286e3ba84d62a11fa4618c3de2",
    "698ba0301d638c7c48350902e26892ef333b1661dfa2dc767eb847defba9c256543fdbfbb84ebffba383c375fac5f82fbf75bdbf9bf6059e9c3d5fb14e47d79b"
    "d7fd6ea37fdaaa2573b40b7c8ae6801cfed7fdbf0052dd6684", "" };

  c10_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c10_data[0], 1616U, &c10_nameCaptureInfo);
  return c10_nameCaptureInfo;
}

static void c10_eML_blk_kernel(SFc10_sliding_controller2InstanceStruct
  *chartInstance, real_T c10_b_q[2], real_T c10_b_qd[2], real_T c10_b_knee[2],
  real_T c10_b_dknee[2], real_T c10_b_ankle[2], real_T c10_b_danke[2])
{
  real_T c10_a;
  real_T c10_b_a;
  c10_b_knee[0] = -0.324 * muDoubleScalarCos(c10_b_q[0]);
  c10_b_knee[1] = -0.324 * muDoubleScalarSin(c10_b_q[0]);
  c10_a = -0.324 * c10_b_qd[0];
  c10_b_dknee[0] = c10_a * -muDoubleScalarSin(c10_b_q[0]);
  c10_b_dknee[1] = c10_a * muDoubleScalarCos(c10_b_q[0]);
  c10_b_a = -0.39 * (c10_b_qd[0] + c10_b_qd[1]);
  c10_b_ankle[0] = c10_b_knee[0] - -0.39 * muDoubleScalarSin(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_ankle[1] = c10_b_knee[1] - -0.39 * muDoubleScalarCos(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_danke[0] = c10_b_dknee[0] + c10_b_a * -muDoubleScalarSin(c10_b_q[0] +
    c10_b_q[1]);
  c10_b_danke[1] = c10_b_dknee[1] + c10_b_a * muDoubleScalarCos(c10_b_q[0] +
    c10_b_q[1]);
}

static void c10_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_b_ankle, const char_T *c10_identifier,
  real_T c10_y[2])
{
  emlrtMsgIdentifier c10_thisId;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c10_b_ankle), &c10_thisId,
    c10_y);
  sf_mex_destroy(&c10_b_ankle);
}

static void c10_b_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId,
  real_T c10_y[2])
{
  real_T c10_dv[2];
  int32_T c10_i;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), c10_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c10_i = 0; c10_i < 2; c10_i++) {
    c10_y[c10_i] = c10_dv[c10_i];
  }

  sf_mex_destroy(&c10_u);
}

static uint8_T c10_c_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_b_is_active_c10_sliding_controller2, const
  char_T *c10_identifier)
{
  emlrtMsgIdentifier c10_thisId;
  uint8_T c10_y;
  c10_thisId.fIdentifier = const_cast<const char_T *>(c10_identifier);
  c10_thisId.fParent = NULL;
  c10_thisId.bParentIsCell = false;
  c10_y = c10_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c10_b_is_active_c10_sliding_controller2), &c10_thisId);
  sf_mex_destroy(&c10_b_is_active_c10_sliding_controller2);
  return c10_y;
}

static uint8_T c10_d_emlrt_marshallIn(SFc10_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c10_u, const emlrtMsgIdentifier *c10_parentId)
{
  uint8_T c10_b_u;
  uint8_T c10_y;
  sf_mex_import(c10_parentId, sf_mex_dup(c10_u), &c10_b_u, 1, 3, 0U, 0, 0U, 0);
  c10_y = c10_b_u;
  sf_mex_destroy(&c10_u);
  return c10_y;
}

static void init_dsm_address_info(SFc10_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc10_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c10_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c10_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c10_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c10_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c10_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c10_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c10_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c10_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c10_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c10_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c10_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c10_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6LFA4ifWRyfmFySWZYan"
    "2xoEF+ck5mSmZcen5yfV1KUn5OTWmSEZD4IAACGgSyv"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c10_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c10_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c10_sliding_controller2
    ((SFc10_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c10_sliding_controller2(void *chartInstanceVar)
{
  enable_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c10_sliding_controller2(void *chartInstanceVar)
{
  disable_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c10_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c10_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c10_sliding_controller2
    ((SFc10_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c10_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c10_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc10_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c10_sliding_controller2
      ((SFc10_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc10_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc10_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c10_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c10_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc10_sliding_controller2((SFc10_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c10_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c10_sliding_controller2
      ((SFc10_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc10_sliding_controller2
      ((SFc10_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c10_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFAL0GbFAXaSwE3RYHm0tqmJESFHTulnAQ91FgvR+JCy112f2ir79C",
    "ee8k9D9BLn6BAniCnPkPRJ+gsScsyRVKR3aRuERqUvOQ33843Mzu78lqDXQ+vm3j/csvzruH3db",
    "zbXn6tFePW3J0/73gfF+Of0EjYeJ8oEmuv8RIkhm9BS24Nk2IgRrISxsQIFAiK2EQqU8emWWw5E",
    "5O+FdTx6ScRo1EQScvDbbQl4Z7gU2RLrNlHni5TQE0fIDSRknYc9TkZzzxW5tiPgE60jZskaDCB",
    "TZxbetdywxIOvROgA6ENQY/1mW+BIQZ8c1Ir0ynVwSlQxglnRFSqjYgOIMEAGzhIQvzcswZFlWE",
    "0IspsQ0RS0DtsknFKAWVOpvHFERPESMUI78Xcd4aLvu1z9GdXhsAbAoK+bSsgk0QyYerzH/RRaU",
    "+QIw5dOLLjerYAfrAu+Y8ZHIOqjdvIlykoMoY9UTtpFpDeSZatWZUswgyL4TFRWxTzpyGsrV6sH",
    "B0QzBMM0aIOBpnIgR4qlmJ4a9lsPHCVuWzJ2DhPtl4Gy9h6KTRlYcbWp8InnOta2FAmO5ACz1i7",
    "xJBmWM5ajdOahUOJAXblXb8arGCY+ALmSxGyynSlJUDWdx5iYzmPpFYbGftYvN2dncXXi7CBMKB",
    "GhEJVF1CEacCYZeGtZwuZdrlHIHplMveqwHmFLEN5emRF91iqCcakoYmcSXAZrQXGeoy5xJVwoH",
    "HRNMFcLpfhKKERhK7BMA67uGwQWxET7VrbFq67lJlpFzRVLKnIqsVVh22o5wpqmsCBmAh5LPpKx",
    "kHR4/PwAmC1ESWYGG9j/1LTPs5elQq3n214Z/vZO6+wn53alb8/meNpVfB4c9/leW+0z+PbpXnb",
    "+Ndq5Xabc3bvlubplOwcbh3vb/549tHD93/96/cXn/48ff7b98vmby3M38r+d3bX11bb/28W49u",
    "njXa2cNKFenXYB3N+dSr435vjXy/G+lH0hUi/exBtPSLDeLDx+Y/3N9IjP+N72mn2d63k7+nzO6",
    "7jY5Vl9anoICwOJm5MbL5dO/4v5/y9tiQeN4rn+fXn15ezv71Ztu8s0bdejImYcCjV77+n48ML6",
    "whRyNXRcefiOiZ4aLgyOm5tlvtDlY7OOR0dz0n4Z/xfjONq9vn8T5f4f7eUh7vZefGQuF0IDuln",
    "G4easxC3kEMqhVGSc1D3Kvr+RfvoqnbeG7b7r/j51u715/1VzkXtC9p1LnkOe1N2l9W36vnwquG",
    "b9gOvhF+/wjoue25/3fiX3mrn6w+K8Vezn9B+xHhY8WuqeI0/jUZVb/8Hdfo3JH7Drw==",
    ""
  };

  static char newstr [1345] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c10_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c10_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(614325834U));
  ssSetChecksum1(S,(4071890510U));
  ssSetChecksum2(S,(2385365703U));
  ssSetChecksum3(S,(1589225593U));
}

static void mdlRTW_c10_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c10_sliding_controller2(SimStruct *S)
{
  SFc10_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc10_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc10_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc10_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc10_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c10_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c10_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c10_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c10_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c10_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c10_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c10_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c10_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c10_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c10_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c10_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c10_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c10_JITStateAnimation,
    chartInstance->c10_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c10_sliding_controller2(chartInstance);
}

void c10_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c10_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c10_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c10_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c10_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
