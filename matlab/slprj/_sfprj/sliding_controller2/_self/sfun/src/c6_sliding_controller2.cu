#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c6_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void enable_c6_sliding_controller2(SFc6_sliding_controller2InstanceStruct
  *chartInstance);
static void disable_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void c6_do_animation_call_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance, const mxArray *c6_st);
static void sf_gateway_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance);
static void c6_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_b_msg, const char_T *c6_identifier,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_y);
static void c6_b_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_y);
static void c6_c_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  c6_SL_Bus_sliding_controller2_std_msgs_Header *c6_y);
static uint32_T c6_d_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId);
static c6_SL_Bus_sliding_controller2_ros_time_Time c6_e_emlrt_marshallIn
  (SFc6_sliding_controller2InstanceStruct *chartInstance, const mxArray *c6_u,
   const emlrtMsgIdentifier *c6_parentId);
static real_T c6_f_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId);
static void c6_g_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  uint8_T c6_y[128]);
static c6_SL_Bus_ROSVariableLengthArrayInfo c6_h_emlrt_marshallIn
  (SFc6_sliding_controller2InstanceStruct *chartInstance, const mxArray *c6_u,
   const emlrtMsgIdentifier *c6_parentId);
static void c6_i_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  c6_SL_Bus_sliding_controller2_std_msgs_String c6_y[16]);
static void c6_j_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  real_T c6_y[7]);
static uint8_T c6_k_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_b_is_active_c6_sliding_controller2, const
  char_T *c6_identifier);
static uint8_T c6_l_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId);
static void c6_eML_blk_kernel(SFc6_sliding_controller2InstanceStruct
  *chartInstance, c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg);
static __global__ void c6_sf_gateway_c6_sliding_controller2_kernel1
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg,
   c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg);
static __global__ void c6_sf_gateway_c6_sliding_controller2_kernel2(const
  int32_T c6_i1, c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg, c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg);
static __global__ void c6_sf_gateway_c6_sliding_controller2_kernel3
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg,
   c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg);
static __global__ void c6_sf_gateway_c6_sliding_controller2_kernel4
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg,
   c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r);
static __global__ void c6_sf_gateway_c6_sliding_controller2_kernel5(const
  int32_T c6_i5, c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r);
static __global__ void c6_sf_gateway_c6_sliding_controller2_kernel6
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg,
   c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r);
static __global__ void c6_sf_gateway_c6_sliding_controller2_kernel7(const
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg);
static __global__ void c6_sf_gateway_c6_sliding_controller2_kernel8(const
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r, const int32_T
  c6_i9, c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg);
static __global__ void c6_sf_gateway_c6_sliding_controller2_kernel9(const
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg);
static __global__ void c6_eML_blk_kernel_kernel10
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel11(const uint8_T c6_uv[10],
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel12
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel13(const uint8_T c6_uv1[11],
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel14
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel15(const uint8_T c6_uv2[12],
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel16
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel17(const uint8_T c6_uv3[11],
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel18
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel19(const uint8_T c6_uv4[12],
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel20
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel21(const uint8_T c6_uv5[13],
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel22
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel23(const uint8_T c6_uv6[8],
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static __global__ void c6_eML_blk_kernel_kernel24
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg);
static void init_dsm_address_info(SFc6_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc6_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c6_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c6_is_active_c6_sliding_controller2 = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c6_gpu_uv6, 8UL);
  hipMalloc(&chartInstance->c6_gpu_uv5, 13UL);
  hipMalloc(&chartInstance->c6_gpu_uv4, 12UL);
  hipMalloc(&chartInstance->c6_gpu_uv3, 11UL);
  hipMalloc(&chartInstance->c6_gpu_uv2, 12UL);
  hipMalloc(&chartInstance->c6_gpu_uv1, 11UL);
  hipMalloc(&chartInstance->c6_gpu_uv, 10UL);
  hipMalloc(&chartInstance->c6_gpu_blankMsg, 2536UL);
}

static void initialize_params_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c6_sliding_controller2(SFc6_sliding_controller2InstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c6_do_animation_call_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
  static const char_T *c6_sv[9] = { "Header", "Name", "Name_SL_Info", "Position",
    "Position_SL_Info", "Velocity", "Velocity_SL_Info", "Effort",
    "Effort_SL_Info" };

  static const char_T *c6_sv1[4] = { "Seq", "Stamp", "FrameId",
    "FrameId_SL_Info" };

  static const char_T *c6_sv2[2] = { "Sec", "Nsec" };

  static const char_T *c6_sv3[2] = { "CurrentLength", "ReceivedLength" };

  static const char_T *c6_sv4[2] = { "Data", "Data_SL_Info" };

  c6_SL_Bus_sliding_controller2_std_msgs_String c6_g_u[16];
  const c6_SL_Bus_sliding_controller2_std_msgs_String *c6_r;
  const mxArray *c6_ab_y = NULL;
  const mxArray *c6_b_y = NULL;
  const mxArray *c6_bb_y = NULL;
  const mxArray *c6_c_y = NULL;
  const mxArray *c6_cb_y = NULL;
  const mxArray *c6_d_y = NULL;
  const mxArray *c6_db_y = NULL;
  const mxArray *c6_e_y = NULL;
  const mxArray *c6_eb_y = NULL;
  const mxArray *c6_f_y = NULL;
  const mxArray *c6_fb_y = NULL;
  const mxArray *c6_g_y = NULL;
  const mxArray *c6_gb_y = NULL;
  const mxArray *c6_h_y = NULL;
  const mxArray *c6_i_y = NULL;
  const mxArray *c6_j_y = NULL;
  const mxArray *c6_k_y = NULL;
  const mxArray *c6_l_y = NULL;
  const mxArray *c6_m_y = NULL;
  const mxArray *c6_n_y = NULL;
  const mxArray *c6_o_y = NULL;
  const mxArray *c6_p_y = NULL;
  const mxArray *c6_q_y = NULL;
  const mxArray *c6_r_y = NULL;
  const mxArray *c6_s_y = NULL;
  const mxArray *c6_st;
  const mxArray *c6_t_y = NULL;
  const mxArray *c6_u_y = NULL;
  const mxArray *c6_v_y = NULL;
  const mxArray *c6_w_y = NULL;
  const mxArray *c6_x_y = NULL;
  const mxArray *c6_y = NULL;
  const mxArray *c6_y_y = NULL;
  real_T c6_l_u[7];
  real_T c6_b_u;
  real_T c6_c_u;
  real_T c6_u_Stamp_Nsec;
  real_T c6_u_Stamp_Sec;
  int32_T c6_iv[1];
  int32_T c6_b_i;
  int32_T c6_i;
  int32_T c6_i1;
  int32_T c6_i2;
  int32_T c6_i3;
  int32_T c6_i4;
  int32_T c6_i5;
  int32_T c6_i6;
  int32_T c6_i7;
  uint32_T c6_e_u;
  uint32_T c6_f_u;
  uint32_T c6_h_u;
  uint32_T c6_i_u;
  uint32_T c6_j_u;
  uint32_T c6_k_u;
  uint32_T c6_m_u;
  uint32_T c6_n_u;
  uint32_T c6_o_u;
  uint32_T c6_p_u;
  uint32_T c6_q_u;
  uint32_T c6_r_u;
  uint32_T c6_u;
  uint32_T c6_u_CurrentLength;
  uint32_T c6_u_FrameId_SL_Info_CurrentLength;
  uint32_T c6_u_FrameId_SL_Info_ReceivedLength;
  uint32_T c6_u_ReceivedLength;
  uint32_T c6_u_Seq;
  uint8_T c6_d_u[128];
  uint8_T c6_u_FrameId[128];
  c6_st = NULL;
  c6_y = NULL;
  sf_mex_assign(&c6_y, sf_mex_createcellmatrix(2, 1), false);
  c6_b_y = NULL;
  sf_mex_assign(&c6_b_y, sf_mex_createstruct("structure", 9, c6_sv, 2, 1, 1),
                false);
  c6_u_Seq = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[0];
  c6_u_Stamp_Sec = *(real_T *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_ros_time_Time *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[8])[0];
  c6_u_Stamp_Nsec = *(real_T *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_ros_time_Time *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[8])[8];
  for (c6_i = 0; c6_i < 128; c6_i++) {
    c6_u_FrameId[c6_i] = ((uint8_T *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
      chartInstance->c6_msg)[0])[24])[c6_i];
  }

  c6_u_FrameId_SL_Info_CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[0];
  c6_u_FrameId_SL_Info_ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[4];
  c6_c_y = NULL;
  sf_mex_assign(&c6_c_y, sf_mex_createstruct("structure", 4, c6_sv1, 2, 1, 1),
                false);
  c6_u = c6_u_Seq;
  c6_d_y = NULL;
  sf_mex_assign(&c6_d_y, sf_mex_create("y", &c6_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_c_y, 0, "Seq", c6_d_y, 0);
  c6_e_y = NULL;
  sf_mex_assign(&c6_e_y, sf_mex_createstruct("structure", 2, c6_sv2, 2, 1, 1),
                false);
  c6_b_u = c6_u_Stamp_Sec;
  c6_f_y = NULL;
  sf_mex_assign(&c6_f_y, sf_mex_create("y", &c6_b_u, 0, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_e_y, 0, "Sec", c6_f_y, 0);
  c6_c_u = c6_u_Stamp_Nsec;
  c6_g_y = NULL;
  sf_mex_assign(&c6_g_y, sf_mex_create("y", &c6_c_u, 0, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_e_y, 0, "Nsec", c6_g_y, 1);
  sf_mex_setfieldbynum(c6_c_y, 0, "Stamp", c6_e_y, 1);
  for (c6_i1 = 0; c6_i1 < 128; c6_i1++) {
    c6_d_u[c6_i1] = c6_u_FrameId[c6_i1];
  }

  c6_h_y = NULL;
  sf_mex_assign(&c6_h_y, sf_mex_create("y", c6_d_u, 3, 0U, 1U, 0U, 1, 128),
                false);
  sf_mex_setfieldbynum(c6_c_y, 0, "FrameId", c6_h_y, 2);
  c6_i_y = NULL;
  sf_mex_assign(&c6_i_y, sf_mex_createstruct("structure", 2, c6_sv3, 2, 1, 1),
                false);
  c6_e_u = c6_u_FrameId_SL_Info_CurrentLength;
  c6_j_y = NULL;
  sf_mex_assign(&c6_j_y, sf_mex_create("y", &c6_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_i_y, 0, "CurrentLength", c6_j_y, 0);
  c6_f_u = c6_u_FrameId_SL_Info_ReceivedLength;
  c6_k_y = NULL;
  sf_mex_assign(&c6_k_y, sf_mex_create("y", &c6_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_i_y, 0, "ReceivedLength", c6_k_y, 1);
  sf_mex_setfieldbynum(c6_c_y, 0, "FrameId_SL_Info", c6_i_y, 3);
  sf_mex_setfieldbynum(c6_b_y, 0, "Header", c6_c_y, 0);
  for (c6_i2 = 0; c6_i2 < 16; c6_i2++) {
    for (c6_i3 = 0; c6_i3 < 128; c6_i3++) {
      c6_g_u[c6_i2].Data[c6_i3] = ((uint8_T *)&((char_T *)
        (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
        (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
        chartInstance->c6_msg)[160])[136 * (int32_T)c6_i2])[0])[c6_i3];
    }

    c6_g_u[c6_i2].Data_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
      (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i2])[128])[0];
    c6_g_u[c6_i2].Data_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
      (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i2])[128])[4];
  }

  c6_l_y = NULL;
  c6_iv[0] = 16;
  sf_mex_assign(&c6_l_y, sf_mex_createstructarray("structure", 1, c6_iv, 2,
    c6_sv4), false);
  for (c6_b_i = 0; c6_b_i < 16; c6_b_i++) {
    c6_r = &c6_g_u[c6_b_i];
    for (c6_i4 = 0; c6_i4 < 128; c6_i4++) {
      c6_d_u[c6_i4] = c6_r->Data[c6_i4];
    }

    c6_n_y = NULL;
    sf_mex_assign(&c6_n_y, sf_mex_create("y", c6_d_u, 3, 0U, 1U, 0U, 1, 128),
                  false);
    sf_mex_setfieldbynum(c6_l_y, c6_b_i, "Data", c6_n_y, 0);
    c6_u_CurrentLength = c6_r->Data_SL_Info.CurrentLength;
    c6_u_ReceivedLength = c6_r->Data_SL_Info.ReceivedLength;
    c6_p_y = NULL;
    sf_mex_assign(&c6_p_y, sf_mex_createstruct("structure", 2, c6_sv3, 2, 1, 1),
                  false);
    c6_j_u = c6_u_CurrentLength;
    c6_r_y = NULL;
    sf_mex_assign(&c6_r_y, sf_mex_create("y", &c6_j_u, 7, 0U, 0U, 0U, 0), false);
    sf_mex_setfieldbynum(c6_p_y, 0, "CurrentLength", c6_r_y, 0);
    c6_k_u = c6_u_ReceivedLength;
    c6_s_y = NULL;
    sf_mex_assign(&c6_s_y, sf_mex_create("y", &c6_k_u, 7, 0U, 0U, 0U, 0), false);
    sf_mex_setfieldbynum(c6_p_y, 0, "ReceivedLength", c6_s_y, 1);
    sf_mex_setfieldbynum(c6_l_y, c6_b_i, "Data_SL_Info", c6_p_y, 1);
  }

  sf_mex_setfieldbynum(c6_b_y, 0, "Name", c6_l_y, 1);
  c6_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2336])[0];
  c6_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2336])[4];
  c6_m_y = NULL;
  sf_mex_assign(&c6_m_y, sf_mex_createstruct("structure", 2, c6_sv3, 2, 1, 1),
                false);
  c6_h_u = c6_u_CurrentLength;
  c6_o_y = NULL;
  sf_mex_assign(&c6_o_y, sf_mex_create("y", &c6_h_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_m_y, 0, "CurrentLength", c6_o_y, 0);
  c6_i_u = c6_u_ReceivedLength;
  c6_q_y = NULL;
  sf_mex_assign(&c6_q_y, sf_mex_create("y", &c6_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_m_y, 0, "ReceivedLength", c6_q_y, 1);
  sf_mex_setfieldbynum(c6_b_y, 0, "Name_SL_Info", c6_m_y, 2);
  for (c6_i5 = 0; c6_i5 < 7; c6_i5++) {
    c6_l_u[c6_i5] = ((real_T *)&((char_T *)chartInstance->c6_msg)[2344])[c6_i5];
  }

  c6_t_y = NULL;
  sf_mex_assign(&c6_t_y, sf_mex_create("y", c6_l_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c6_b_y, 0, "Position", c6_t_y, 3);
  c6_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2400])[0];
  c6_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2400])[4];
  c6_u_y = NULL;
  sf_mex_assign(&c6_u_y, sf_mex_createstruct("structure", 2, c6_sv3, 2, 1, 1),
                false);
  c6_m_u = c6_u_CurrentLength;
  c6_v_y = NULL;
  sf_mex_assign(&c6_v_y, sf_mex_create("y", &c6_m_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_u_y, 0, "CurrentLength", c6_v_y, 0);
  c6_n_u = c6_u_ReceivedLength;
  c6_w_y = NULL;
  sf_mex_assign(&c6_w_y, sf_mex_create("y", &c6_n_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_u_y, 0, "ReceivedLength", c6_w_y, 1);
  sf_mex_setfieldbynum(c6_b_y, 0, "Position_SL_Info", c6_u_y, 4);
  for (c6_i6 = 0; c6_i6 < 7; c6_i6++) {
    c6_l_u[c6_i6] = ((real_T *)&((char_T *)chartInstance->c6_msg)[2408])[c6_i6];
  }

  c6_x_y = NULL;
  sf_mex_assign(&c6_x_y, sf_mex_create("y", c6_l_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c6_b_y, 0, "Velocity", c6_x_y, 5);
  c6_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2464])[0];
  c6_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2464])[4];
  c6_y_y = NULL;
  sf_mex_assign(&c6_y_y, sf_mex_createstruct("structure", 2, c6_sv3, 2, 1, 1),
                false);
  c6_o_u = c6_u_CurrentLength;
  c6_ab_y = NULL;
  sf_mex_assign(&c6_ab_y, sf_mex_create("y", &c6_o_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_y_y, 0, "CurrentLength", c6_ab_y, 0);
  c6_p_u = c6_u_ReceivedLength;
  c6_bb_y = NULL;
  sf_mex_assign(&c6_bb_y, sf_mex_create("y", &c6_p_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_y_y, 0, "ReceivedLength", c6_bb_y, 1);
  sf_mex_setfieldbynum(c6_b_y, 0, "Velocity_SL_Info", c6_y_y, 6);
  for (c6_i7 = 0; c6_i7 < 7; c6_i7++) {
    c6_l_u[c6_i7] = ((real_T *)&((char_T *)chartInstance->c6_msg)[2472])[c6_i7];
  }

  c6_cb_y = NULL;
  sf_mex_assign(&c6_cb_y, sf_mex_create("y", c6_l_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c6_b_y, 0, "Effort", c6_cb_y, 7);
  c6_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2528])[0];
  c6_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2528])[4];
  c6_db_y = NULL;
  sf_mex_assign(&c6_db_y, sf_mex_createstruct("structure", 2, c6_sv3, 2, 1, 1),
                false);
  c6_q_u = c6_u_CurrentLength;
  c6_eb_y = NULL;
  sf_mex_assign(&c6_eb_y, sf_mex_create("y", &c6_q_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_db_y, 0, "CurrentLength", c6_eb_y, 0);
  c6_r_u = c6_u_ReceivedLength;
  c6_fb_y = NULL;
  sf_mex_assign(&c6_fb_y, sf_mex_create("y", &c6_r_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c6_db_y, 0, "ReceivedLength", c6_fb_y, 1);
  sf_mex_setfieldbynum(c6_b_y, 0, "Effort_SL_Info", c6_db_y, 8);
  sf_mex_setcell(c6_y, 0, c6_b_y);
  c6_gb_y = NULL;
  sf_mex_assign(&c6_gb_y, sf_mex_create("y",
    &chartInstance->c6_is_active_c6_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c6_y, 1, c6_gb_y);
  sf_mex_assign(&c6_st, c6_y, false);
  return c6_st;
}

static void set_sim_state_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance, const mxArray *c6_st)
{
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState c6_r;
  const mxArray *c6_u;
  int32_T c6_i;
  int32_T c6_i1;
  int32_T c6_i2;
  int32_T c6_i3;
  c6_u = sf_mex_dup(c6_st);
  c6_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c6_u, 0)), "msg",
                      &c6_r);
  *(uint32_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                 &((char_T *)chartInstance->c6_msg)[0])[0] = c6_r.Header.Seq;
  *(real_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_ros_time_Time *)
               &((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                 &((char_T *)chartInstance->c6_msg)[0])[8])[0] =
    c6_r.Header.Stamp.Sec;
  *(real_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_ros_time_Time *)
               &((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                 &((char_T *)chartInstance->c6_msg)[0])[8])[8] =
    c6_r.Header.Stamp.Nsec;
  for (c6_i = 0; c6_i < 128; c6_i++) {
    ((uint8_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                  &((char_T *)chartInstance->c6_msg)[0])[24])[c6_i] =
      c6_r.Header.FrameId[c6_i];
  }

  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[0] =
    c6_r.Header.FrameId_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[4] =
    c6_r.Header.FrameId_SL_Info.ReceivedLength;
  for (c6_i1 = 0; c6_i1 < 16; c6_i1++) {
    for (c6_i2 = 0; c6_i2 < 128; c6_i2++) {
      ((uint8_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_String *)
                    &((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_String *)
                      &((char_T *)chartInstance->c6_msg)[160])[136 * (int32_T)
                    c6_i1])[0])[c6_i2] = c6_r.Name[c6_i1].Data[c6_i2];
    }

    *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i1])[128])[0] =
      c6_r.Name[c6_i1].Data_SL_Info.CurrentLength;
    *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i1])[128])[4] =
      c6_r.Name[c6_i1].Data_SL_Info.ReceivedLength;
  }

  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2336])[0] = c6_r.Name_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2336])[4] = c6_r.Name_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2400])[0] = c6_r.Position_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2400])[4] = c6_r.Position_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2464])[0] = c6_r.Velocity_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2464])[4] = c6_r.Velocity_SL_Info.ReceivedLength;
  for (c6_i3 = 0; c6_i3 < 7; c6_i3++) {
    ((real_T *)&((char_T *)chartInstance->c6_msg)[2344])[c6_i3] =
      c6_r.Position[c6_i3];
    ((real_T *)&((char_T *)chartInstance->c6_msg)[2408])[c6_i3] =
      c6_r.Velocity[c6_i3];
    ((real_T *)&((char_T *)chartInstance->c6_msg)[2472])[c6_i3] =
      c6_r.Effort[c6_i3];
  }

  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2528])[0] = c6_r.Effort_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2528])[4] = c6_r.Effort_SL_Info.ReceivedLength;
  chartInstance->c6_is_active_c6_sliding_controller2 = c6_k_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c6_u, 1)),
     "is_active_c6_sliding_controller2");
  sf_mex_destroy(&c6_u);
  sf_mex_destroy(&c6_st);
}

static void sf_gateway_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState c6_r;
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_gpu_blankMsg;
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_gpu_msg;
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_gpu_r;
  int32_T c6_i1;
  int32_T c6_i5;
  int32_T c6_i9;
  hipMalloc(&c6_gpu_r, 2536UL);
  hipMalloc(&c6_gpu_msg, 2536UL);
  hipMalloc(&c6_b_gpu_blankMsg, 2536UL);
  chartInstance->c6_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  *(uint32_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                 &((char_T *)chartInstance->c6_msg)[0])[0] = *(uint32_T *)
    &((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
       chartInstance->c6_blankMsg)[0])[0];
  *(real_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_ros_time_Time *)
               &((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                 &((char_T *)chartInstance->c6_msg)[0])[8])[0] = *(real_T *)
    &((char_T *)(c6_SL_Bus_sliding_controller2_ros_time_Time *)&((char_T *)
       (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
        chartInstance->c6_blankMsg)[0])[8])[0];
  *(real_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_ros_time_Time *)
               &((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                 &((char_T *)chartInstance->c6_msg)[0])[8])[8] = *(real_T *)
    &((char_T *)(c6_SL_Bus_sliding_controller2_ros_time_Time *)&((char_T *)
       (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
        chartInstance->c6_blankMsg)[0])[8])[8];
  hipMemcpy(c6_b_gpu_blankMsg, chartInstance->c6_blankMsg, 2536UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c6_gpu_msg, chartInstance->c6_msg, 2536UL, hipMemcpyHostToDevice);
  c6_sf_gateway_c6_sliding_controller2_kernel1<<<dim3(1U, 1U, 1U), dim3(128U, 1U,
    1U)>>>(c6_b_gpu_blankMsg, c6_gpu_msg);
  hipMemcpy(chartInstance->c6_msg, c6_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c6_blankMsg, c6_b_gpu_blankMsg, 2536UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[0] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_blankMsg)[0])[152])[0];
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[4] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_blankMsg)[0])[152])[4];
  for (c6_i1 = 0; c6_i1 < 16; c6_i1++) {
    hipMemcpy(c6_b_gpu_blankMsg, chartInstance->c6_blankMsg, 2536UL,
               hipMemcpyHostToDevice);
    hipMemcpy(c6_gpu_msg, chartInstance->c6_msg, 2536UL, hipMemcpyHostToDevice);
    c6_sf_gateway_c6_sliding_controller2_kernel2<<<dim3(1U, 1U, 1U), dim3(128U,
      1U, 1U)>>>(c6_i1, c6_b_gpu_blankMsg, c6_gpu_msg);
    hipMemcpy(chartInstance->c6_msg, c6_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
    hipMemcpy(chartInstance->c6_blankMsg, c6_b_gpu_blankMsg, 2536UL,
               hipMemcpyDeviceToHost);
    *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i1])[128])[0] = *(uint32_T *)
      &((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
         (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
          (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
           chartInstance->c6_blankMsg)[160])[136 * (int32_T)c6_i1])[128])[0];
    *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i1])[128])[4] = *(uint32_T *)
      &((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
         (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
          (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
           chartInstance->c6_blankMsg)[160])[136 * (int32_T)c6_i1])[128])[4];
  }

  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2336])[0] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_blankMsg)[2336])[0];
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2336])[4] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_blankMsg)[2336])[4];
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2400])[0] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_blankMsg)[2400])[0];
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2400])[4] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_blankMsg)[2400])[4];
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2464])[0] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_blankMsg)[2464])[0];
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2464])[4] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_blankMsg)[2464])[4];
  hipMemcpy(c6_b_gpu_blankMsg, chartInstance->c6_blankMsg, 2536UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c6_gpu_msg, chartInstance->c6_msg, 2536UL, hipMemcpyHostToDevice);
  c6_sf_gateway_c6_sliding_controller2_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U,
    1U)>>>(c6_b_gpu_blankMsg, c6_gpu_msg);
  hipMemcpy(chartInstance->c6_msg, c6_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c6_blankMsg, c6_b_gpu_blankMsg, 2536UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2528])[0] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_blankMsg)[2528])[0];
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2528])[4] = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_blankMsg)[2528])[4];
  c6_r.Header.Seq = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[0];
  c6_r.Header.Stamp.Sec = *(real_T *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_ros_time_Time *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[8])[0];
  c6_r.Header.Stamp.Nsec = *(real_T *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_ros_time_Time *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[8])[8];
  hipMemcpy(c6_gpu_r, &c6_r, 2536UL, hipMemcpyHostToDevice);
  hipMemcpy(c6_gpu_msg, chartInstance->c6_msg, 2536UL, hipMemcpyHostToDevice);
  c6_sf_gateway_c6_sliding_controller2_kernel4<<<dim3(1U, 1U, 1U), dim3(128U, 1U,
    1U)>>>(c6_gpu_msg, c6_gpu_r);
  hipMemcpy(chartInstance->c6_msg, c6_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c6_r, c6_gpu_r, 2536UL, hipMemcpyDeviceToHost);
  c6_r.Header.FrameId_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[0];
  c6_r.Header.FrameId_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[4];
  for (c6_i5 = 0; c6_i5 < 16; c6_i5++) {
    hipMemcpy(c6_gpu_r, &c6_r, 2536UL, hipMemcpyHostToDevice);
    hipMemcpy(c6_gpu_msg, chartInstance->c6_msg, 2536UL, hipMemcpyHostToDevice);
    c6_sf_gateway_c6_sliding_controller2_kernel5<<<dim3(1U, 1U, 1U), dim3(128U,
      1U, 1U)>>>(c6_i5, c6_gpu_msg, c6_gpu_r);
    hipMemcpy(chartInstance->c6_msg, c6_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
    hipMemcpy(&c6_r, c6_gpu_r, 2536UL, hipMemcpyDeviceToHost);
    c6_r.Name[c6_i5].Data_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
      (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i5])[128])[0];
    c6_r.Name[c6_i5].Data_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
      (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i5])[128])[4];
  }

  c6_r.Name_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2336])[0];
  c6_r.Name_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2336])[4];
  c6_r.Position_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2400])[0];
  c6_r.Position_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2400])[4];
  c6_r.Velocity_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2464])[0];
  c6_r.Velocity_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2464])[4];
  hipMemcpy(c6_gpu_r, &c6_r, 2536UL, hipMemcpyHostToDevice);
  hipMemcpy(c6_gpu_msg, chartInstance->c6_msg, 2536UL, hipMemcpyHostToDevice);
  c6_sf_gateway_c6_sliding_controller2_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U,
    1U)>>>(c6_gpu_msg, c6_gpu_r);
  hipMemcpy(chartInstance->c6_msg, c6_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c6_r, c6_gpu_r, 2536UL, hipMemcpyDeviceToHost);
  c6_r.Effort_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2528])[0];
  c6_r.Effort_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c6_msg)
    [2528])[4];
  c6_eML_blk_kernel(chartInstance, &c6_r);
  *(uint32_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                 &((char_T *)chartInstance->c6_msg)[0])[0] = c6_r.Header.Seq;
  *(real_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_ros_time_Time *)
               &((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                 &((char_T *)chartInstance->c6_msg)[0])[8])[0] =
    c6_r.Header.Stamp.Sec;
  *(real_T *)&((char_T *)(c6_SL_Bus_sliding_controller2_ros_time_Time *)
               &((char_T *)(c6_SL_Bus_sliding_controller2_std_msgs_Header *)
                 &((char_T *)chartInstance->c6_msg)[0])[8])[8] =
    c6_r.Header.Stamp.Nsec;
  hipMemcpy(c6_gpu_r, &c6_r, 2536UL, hipMemcpyHostToDevice);
  hipMemcpy(c6_gpu_msg, chartInstance->c6_msg, 2536UL, hipMemcpyHostToDevice);
  c6_sf_gateway_c6_sliding_controller2_kernel7<<<dim3(1U, 1U, 1U), dim3(128U, 1U,
    1U)>>>(c6_gpu_r, c6_gpu_msg);
  hipMemcpy(chartInstance->c6_msg, c6_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[0] =
    c6_r.Header.FrameId_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    (c6_SL_Bus_sliding_controller2_std_msgs_Header *)&((char_T *)
    chartInstance->c6_msg)[0])[152])[4] =
    c6_r.Header.FrameId_SL_Info.ReceivedLength;
  for (c6_i9 = 0; c6_i9 < 16; c6_i9++) {
    hipMemcpy(c6_gpu_msg, chartInstance->c6_msg, 2536UL, hipMemcpyHostToDevice);
    c6_sf_gateway_c6_sliding_controller2_kernel8<<<dim3(1U, 1U, 1U), dim3(128U,
      1U, 1U)>>>(c6_gpu_r, c6_i9, c6_gpu_msg);
    hipMemcpy(chartInstance->c6_msg, c6_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
    *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i9])[128])[0] =
      c6_r.Name[c6_i9].Data_SL_Info.CurrentLength;
    *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      (c6_SL_Bus_sliding_controller2_std_msgs_String *)&((char_T *)
      chartInstance->c6_msg)[160])[136 * (int32_T)c6_i9])[128])[4] =
      c6_r.Name[c6_i9].Data_SL_Info.ReceivedLength;
  }

  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2336])[0] = c6_r.Name_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2336])[4] = c6_r.Name_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2400])[0] = c6_r.Position_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2400])[4] = c6_r.Position_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2464])[0] = c6_r.Velocity_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2464])[4] = c6_r.Velocity_SL_Info.ReceivedLength;
  hipMemcpy(c6_gpu_msg, chartInstance->c6_msg, 2536UL, hipMemcpyHostToDevice);
  c6_sf_gateway_c6_sliding_controller2_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U,
    1U)>>>(c6_gpu_r, c6_gpu_msg);
  hipMemcpy(chartInstance->c6_msg, c6_gpu_msg, 2536UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2528])[0] = c6_r.Effort_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c6_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c6_msg)[2528])[4] = c6_r.Effort_SL_Info.ReceivedLength;
  c6_do_animation_call_c6_sliding_controller2(chartInstance);
  hipFree(c6_b_gpu_blankMsg);
  hipFree(c6_gpu_msg);
  hipFree(c6_gpu_r);
}

static void mdl_start_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c6_errCode;
  hipFree(*chartInstance->c6_gpu_uv6);
  hipFree(*chartInstance->c6_gpu_uv5);
  hipFree(*chartInstance->c6_gpu_uv4);
  hipFree(*chartInstance->c6_gpu_uv3);
  hipFree(*chartInstance->c6_gpu_uv2);
  hipFree(*chartInstance->c6_gpu_uv1);
  hipFree(*chartInstance->c6_gpu_uv);
  hipFree(chartInstance->c6_gpu_blankMsg);
  c6_errCode = hipGetLastError();
  if (c6_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c6_errCode, (char_T *)hipGetErrorName
                       (c6_errCode), (char_T *)hipGetErrorString(c6_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c6_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc6_sliding_controller2
  (SFc6_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c6_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c6_nameCaptureInfo = NULL;
  c6_nameCaptureInfo = NULL;
  sf_mex_assign(&c6_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c6_nameCaptureInfo;
}

static void c6_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_b_msg, const char_T *c6_identifier,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_y)
{
  emlrtMsgIdentifier c6_thisId;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c6_b_msg), &c6_thisId, c6_y);
  sf_mex_destroy(&c6_b_msg);
}

static void c6_b_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_y)
{
  static const char_T *c6_fieldNames[9] = { "Header", "Name", "Name_SL_Info",
    "Position", "Position_SL_Info", "Velocity", "Velocity_SL_Info", "Effort",
    "Effort_SL_Info" };

  emlrtMsgIdentifier c6_thisId;
  c6_thisId.fParent = c6_parentId;
  c6_thisId.bParentIsCell = false;
  sf_mex_check_struct(c6_parentId, c6_u, 9, c6_fieldNames, 0U, NULL);
  c6_thisId.fIdentifier = "Header";
  c6_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c6_u, "Header",
    "Header", 0)), &c6_thisId, &c6_y->Header);
  c6_thisId.fIdentifier = "Name";
  c6_i_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c6_u, "Name",
    "Name", 0)), &c6_thisId, c6_y->Name);
  c6_thisId.fIdentifier = "Name_SL_Info";
  c6_y->Name_SL_Info = c6_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c6_u, "Name_SL_Info", "Name_SL_Info", 0)), &c6_thisId);
  c6_thisId.fIdentifier = "Position";
  c6_j_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c6_u,
    "Position", "Position", 0)), &c6_thisId, c6_y->Position);
  c6_thisId.fIdentifier = "Position_SL_Info";
  c6_y->Position_SL_Info = c6_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c6_u, "Position_SL_Info", "Position_SL_Info", 0)),
    &c6_thisId);
  c6_thisId.fIdentifier = "Velocity";
  c6_j_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c6_u,
    "Velocity", "Velocity", 0)), &c6_thisId, c6_y->Velocity);
  c6_thisId.fIdentifier = "Velocity_SL_Info";
  c6_y->Velocity_SL_Info = c6_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c6_u, "Velocity_SL_Info", "Velocity_SL_Info", 0)),
    &c6_thisId);
  c6_thisId.fIdentifier = "Effort";
  c6_j_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c6_u, "Effort",
    "Effort", 0)), &c6_thisId, c6_y->Effort);
  c6_thisId.fIdentifier = "Effort_SL_Info";
  c6_y->Effort_SL_Info = c6_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c6_u, "Effort_SL_Info", "Effort_SL_Info", 0)), &c6_thisId);
  sf_mex_destroy(&c6_u);
}

static void c6_c_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  c6_SL_Bus_sliding_controller2_std_msgs_Header *c6_y)
{
  static const char_T *c6_fieldNames[4] = { "Seq", "Stamp", "FrameId",
    "FrameId_SL_Info" };

  emlrtMsgIdentifier c6_thisId;
  c6_thisId.fParent = c6_parentId;
  c6_thisId.bParentIsCell = false;
  sf_mex_check_struct(c6_parentId, c6_u, 4, c6_fieldNames, 0U, NULL);
  c6_thisId.fIdentifier = "Seq";
  c6_y->Seq = c6_d_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c6_u, "Seq", "Seq", 0)), &c6_thisId);
  c6_thisId.fIdentifier = "Stamp";
  c6_y->Stamp = c6_e_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c6_u, "Stamp", "Stamp", 0)), &c6_thisId);
  c6_thisId.fIdentifier = "FrameId";
  c6_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c6_u,
    "FrameId", "FrameId", 0)), &c6_thisId, c6_y->FrameId);
  c6_thisId.fIdentifier = "FrameId_SL_Info";
  c6_y->FrameId_SL_Info = c6_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c6_u, "FrameId_SL_Info", "FrameId_SL_Info", 0)), &c6_thisId);
  sf_mex_destroy(&c6_u);
}

static uint32_T c6_d_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId)
{
  uint32_T c6_b_u;
  uint32_T c6_y;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_u), &c6_b_u, 1, 7, 0U, 0, 0U, 0);
  c6_y = c6_b_u;
  sf_mex_destroy(&c6_u);
  return c6_y;
}

static c6_SL_Bus_sliding_controller2_ros_time_Time c6_e_emlrt_marshallIn
  (SFc6_sliding_controller2InstanceStruct *chartInstance, const mxArray *c6_u,
   const emlrtMsgIdentifier *c6_parentId)
{
  static const char_T *c6_fieldNames[2] = { "Sec", "Nsec" };

  c6_SL_Bus_sliding_controller2_ros_time_Time c6_y;
  emlrtMsgIdentifier c6_thisId;
  c6_thisId.fParent = c6_parentId;
  c6_thisId.bParentIsCell = false;
  sf_mex_check_struct(c6_parentId, c6_u, 2, c6_fieldNames, 0U, NULL);
  c6_thisId.fIdentifier = "Sec";
  c6_y.Sec = c6_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c6_u, "Sec", "Sec", 0)), &c6_thisId);
  c6_thisId.fIdentifier = "Nsec";
  c6_y.Nsec = c6_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield
    (c6_u, "Nsec", "Nsec", 0)), &c6_thisId);
  sf_mex_destroy(&c6_u);
  return c6_y;
}

static real_T c6_f_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId)
{
  real_T c6_d;
  real_T c6_y;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_u), &c6_d, 1, 0, 0U, 0, 0U, 0);
  c6_y = c6_d;
  sf_mex_destroy(&c6_u);
  return c6_y;
}

static void c6_g_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  uint8_T c6_y[128])
{
  int32_T c6_i;
  uint8_T c6_uv[128];
  sf_mex_import(c6_parentId, sf_mex_dup(c6_u), c6_uv, 1, 3, 0U, 1, 0U, 1, 128);
  for (c6_i = 0; c6_i < 128; c6_i++) {
    c6_y[c6_i] = c6_uv[c6_i];
  }

  sf_mex_destroy(&c6_u);
}

static c6_SL_Bus_ROSVariableLengthArrayInfo c6_h_emlrt_marshallIn
  (SFc6_sliding_controller2InstanceStruct *chartInstance, const mxArray *c6_u,
   const emlrtMsgIdentifier *c6_parentId)
{
  static const char_T *c6_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c6_SL_Bus_ROSVariableLengthArrayInfo c6_y;
  emlrtMsgIdentifier c6_thisId;
  c6_thisId.fParent = c6_parentId;
  c6_thisId.bParentIsCell = false;
  sf_mex_check_struct(c6_parentId, c6_u, 2, c6_fieldNames, 0U, NULL);
  c6_thisId.fIdentifier = "CurrentLength";
  c6_y.CurrentLength = c6_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c6_u, "CurrentLength", "CurrentLength", 0)), &c6_thisId);
  c6_thisId.fIdentifier = "ReceivedLength";
  c6_y.ReceivedLength = c6_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c6_u, "ReceivedLength", "ReceivedLength", 0)), &c6_thisId);
  sf_mex_destroy(&c6_u);
  return c6_y;
}

static void c6_i_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  c6_SL_Bus_sliding_controller2_std_msgs_String c6_y[16])
{
  static const char_T *c6_fieldNames[2] = { "Data", "Data_SL_Info" };

  emlrtMsgIdentifier c6_thisId;
  int32_T c6_i;
  uint32_T c6_uv[1];
  c6_uv[0] = 16U;
  c6_thisId.fParent = c6_parentId;
  c6_thisId.bParentIsCell = false;
  sf_mex_check_struct(c6_parentId, c6_u, 2, c6_fieldNames, 1U, c6_uv);
  for (c6_i = 0; c6_i < 16; c6_i++) {
    c6_thisId.fIdentifier = "Data";
    c6_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c6_u, "Data",
      "Data", c6_i)), &c6_thisId, c6_y[c6_i].Data);
    c6_thisId.fIdentifier = "Data_SL_Info";
    c6_y[c6_i].Data_SL_Info = c6_h_emlrt_marshallIn(chartInstance, sf_mex_dup
      (sf_mex_getfield(c6_u, "Data_SL_Info", "Data_SL_Info", c6_i)), &c6_thisId);
  }

  sf_mex_destroy(&c6_u);
}

static void c6_j_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId,
  real_T c6_y[7])
{
  real_T c6_dv[7];
  int32_T c6_i;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_u), c6_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c6_i = 0; c6_i < 7; c6_i++) {
    c6_y[c6_i] = c6_dv[c6_i];
  }

  sf_mex_destroy(&c6_u);
}

static uint8_T c6_k_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_b_is_active_c6_sliding_controller2, const
  char_T *c6_identifier)
{
  emlrtMsgIdentifier c6_thisId;
  uint8_T c6_y;
  c6_thisId.fIdentifier = const_cast<const char_T *>(c6_identifier);
  c6_thisId.fParent = NULL;
  c6_thisId.bParentIsCell = false;
  c6_y = c6_l_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c6_b_is_active_c6_sliding_controller2), &c6_thisId);
  sf_mex_destroy(&c6_b_is_active_c6_sliding_controller2);
  return c6_y;
}

static uint8_T c6_l_emlrt_marshallIn(SFc6_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c6_u, const emlrtMsgIdentifier *c6_parentId)
{
  uint8_T c6_b_u;
  uint8_T c6_y;
  sf_mex_import(c6_parentId, sf_mex_dup(c6_u), &c6_b_u, 1, 3, 0U, 0, 0U, 0);
  c6_y = c6_b_u;
  sf_mex_destroy(&c6_u);
  return c6_y;
}

static void c6_eML_blk_kernel(SFc6_sliding_controller2InstanceStruct
  *chartInstance, c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg)
{
  static uint8_T c6_uv5[13] = { 82U, 105U, 103U, 104U, 116U, 69U, 120U, 111U,
    65U, 110U, 107U, 108U, 101U };

  static uint8_T c6_uv2[12] = { 76U, 101U, 102U, 116U, 69U, 120U, 111U, 65U,
    110U, 107U, 108U, 101U };

  static uint8_T c6_uv4[12] = { 82U, 105U, 103U, 104U, 116U, 69U, 120U, 111U,
    75U, 110U, 101U, 101U };

  static uint8_T c6_uv1[11] = { 76U, 101U, 102U, 116U, 69U, 120U, 111U, 75U,
    110U, 101U, 101U };

  static uint8_T c6_uv3[11] = { 82U, 105U, 103U, 104U, 116U, 69U, 120U, 111U,
    72U, 105U, 112U };

  static uint8_T c6_uv[10] = { 76U, 101U, 102U, 116U, 69U, 120U, 111U, 72U, 105U,
    112U };

  static uint8_T c6_uv6[8] = { 67U, 114U, 117U, 116U, 99U, 104U, 101U, 115U };

  hipMemcpy(chartInstance->c6_gpu_blankMsg, c6_b_blankMsg, 2536UL,
             hipMemcpyHostToDevice);
  c6_eML_blk_kernel_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c6_gpu_blankMsg);
  hipMemcpy(*chartInstance->c6_gpu_uv, c6_uv, 10UL, hipMemcpyHostToDevice);
  c6_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c6_gpu_uv, chartInstance->c6_gpu_blankMsg);
  c6_eML_blk_kernel_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c6_gpu_blankMsg);
  hipMemcpy(*chartInstance->c6_gpu_uv1, c6_uv1, 11UL, hipMemcpyHostToDevice);
  c6_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c6_gpu_uv1, chartInstance->c6_gpu_blankMsg);
  c6_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c6_gpu_blankMsg);
  hipMemcpy(*chartInstance->c6_gpu_uv2, c6_uv2, 12UL, hipMemcpyHostToDevice);
  c6_eML_blk_kernel_kernel15<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c6_gpu_uv2, chartInstance->c6_gpu_blankMsg);
  c6_eML_blk_kernel_kernel16<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c6_gpu_blankMsg);
  hipMemcpy(*chartInstance->c6_gpu_uv3, c6_uv3, 11UL, hipMemcpyHostToDevice);
  c6_eML_blk_kernel_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c6_gpu_uv3, chartInstance->c6_gpu_blankMsg);
  c6_eML_blk_kernel_kernel18<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c6_gpu_blankMsg);
  hipMemcpy(*chartInstance->c6_gpu_uv4, c6_uv4, 12UL, hipMemcpyHostToDevice);
  c6_eML_blk_kernel_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c6_gpu_uv4, chartInstance->c6_gpu_blankMsg);
  c6_eML_blk_kernel_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c6_gpu_blankMsg);
  hipMemcpy(*chartInstance->c6_gpu_uv5, c6_uv5, 13UL, hipMemcpyHostToDevice);
  c6_eML_blk_kernel_kernel21<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c6_gpu_uv5, chartInstance->c6_gpu_blankMsg);
  c6_eML_blk_kernel_kernel22<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c6_gpu_blankMsg);
  hipMemcpy(*chartInstance->c6_gpu_uv6, c6_uv6, 8UL, hipMemcpyHostToDevice);
  c6_eML_blk_kernel_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c6_gpu_uv6, chartInstance->c6_gpu_blankMsg);
  c6_eML_blk_kernel_kernel24<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (chartInstance->c6_gpu_blankMsg);
  hipMemcpy(c6_b_blankMsg, chartInstance->c6_gpu_blankMsg, 2536UL,
             hipMemcpyDeviceToHost);
}

static __global__ __launch_bounds__(128, 1) void
  c6_sf_gateway_c6_sliding_controller2_kernel1
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg,
   c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg)
{
  uint64_T c6_threadId;
  int32_T c6_i;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i = (int32_T)c6_threadId;
  if (c6_i < 128) {
    c6_b_msg->Header.FrameId[c6_i] = c6_b_blankMsg->Header.FrameId[c6_i];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c6_sf_gateway_c6_sliding_controller2_kernel2(const int32_T c6_i1,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg)
{
  uint64_T c6_threadId;
  int32_T c6_i2;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i2 = (int32_T)c6_threadId;
  if (c6_i2 < 128) {
    c6_b_msg->Name[c6_i1].Data[c6_i2] = c6_b_blankMsg->Name[c6_i1].Data[c6_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c6_sf_gateway_c6_sliding_controller2_kernel3
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg,
   c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg)
{
  uint64_T c6_threadId;
  int32_T c6_i3;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i3 = (int32_T)c6_threadId;
  if (c6_i3 < 7) {
    c6_b_msg->Position[c6_i3] = c6_b_blankMsg->Position[c6_i3];
    c6_b_msg->Velocity[c6_i3] = c6_b_blankMsg->Velocity[c6_i3];
    c6_b_msg->Effort[c6_i3] = c6_b_blankMsg->Effort[c6_i3];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c6_sf_gateway_c6_sliding_controller2_kernel4
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg,
   c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r)
{
  uint64_T c6_threadId;
  int32_T c6_i4;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i4 = (int32_T)c6_threadId;
  if (c6_i4 < 128) {
    c6_r->Header.FrameId[c6_i4] = c6_b_msg->Header.FrameId[c6_i4];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c6_sf_gateway_c6_sliding_controller2_kernel5(const int32_T c6_i5,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r)
{
  uint64_T c6_threadId;
  int32_T c6_i6;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i6 = (int32_T)c6_threadId;
  if (c6_i6 < 128) {
    c6_r->Name[c6_i5].Data[c6_i6] = c6_b_msg->Name[c6_i5].Data[c6_i6];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c6_sf_gateway_c6_sliding_controller2_kernel6
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg,
   c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r)
{
  uint64_T c6_threadId;
  int32_T c6_i7;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i7 = (int32_T)c6_threadId;
  if (c6_i7 < 7) {
    c6_r->Position[c6_i7] = c6_b_msg->Position[c6_i7];
    c6_r->Velocity[c6_i7] = c6_b_msg->Velocity[c6_i7];
    c6_r->Effort[c6_i7] = c6_b_msg->Effort[c6_i7];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c6_sf_gateway_c6_sliding_controller2_kernel7(const
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg)
{
  uint64_T c6_threadId;
  int32_T c6_i8;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i8 = (int32_T)c6_threadId;
  if (c6_i8 < 128) {
    c6_b_msg->Header.FrameId[c6_i8] = c6_r->Header.FrameId[c6_i8];
  }
}

static __global__ __launch_bounds__(128, 1) void
  c6_sf_gateway_c6_sliding_controller2_kernel8(const
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r, const int32_T
  c6_i9, c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg)
{
  uint64_T c6_threadId;
  int32_T c6_i10;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i10 = (int32_T)c6_threadId;
  if (c6_i10 < 128) {
    c6_b_msg->Name[c6_i9].Data[c6_i10] = c6_r->Name[c6_i9].Data[c6_i10];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c6_sf_gateway_c6_sliding_controller2_kernel9(const
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_r,
  c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_msg)
{
  uint64_T c6_threadId;
  int32_T c6_i11;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i11 = (int32_T)c6_threadId;
  if (c6_i11 < 7) {
    c6_b_msg->Position[c6_i11] = c6_r->Position[c6_i11];
    c6_b_msg->Velocity[c6_i11] = c6_r->Velocity[c6_i11];
    c6_b_msg->Effort[c6_i11] = c6_r->Effort[c6_i11];
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel10
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_tmpIdx;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_tmpIdx = (int32_T)c6_threadId;
  if (c6_tmpIdx < 1) {
    c6_b_blankMsg->Name_SL_Info.CurrentLength = 7U;
    c6_b_blankMsg->Effort_SL_Info.CurrentLength = 7U;
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel11(const
  uint8_T c6_uv[10], c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_i;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i = (int32_T)c6_threadId;
  if (c6_i < 10) {
    c6_b_blankMsg->Name[0].Data[c6_i] = c6_uv[c6_i];
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel12
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_tmpIdx;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_tmpIdx = (int32_T)c6_threadId;
  if (c6_tmpIdx < 1) {
    c6_b_blankMsg->Name[0].Data_SL_Info.CurrentLength = 10U;
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel13(const
  uint8_T c6_uv1[11], c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_i1;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i1 = (int32_T)c6_threadId;
  if (c6_i1 < 11) {
    c6_b_blankMsg->Name[1].Data[c6_i1] = c6_uv1[c6_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel14
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_tmpIdx;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_tmpIdx = (int32_T)c6_threadId;
  if (c6_tmpIdx < 1) {
    c6_b_blankMsg->Name[1].Data_SL_Info.CurrentLength = 11U;
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel15(const
  uint8_T c6_uv2[12], c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_i2;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i2 = (int32_T)c6_threadId;
  if (c6_i2 < 12) {
    c6_b_blankMsg->Name[2].Data[c6_i2] = c6_uv2[c6_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel16
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_tmpIdx;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_tmpIdx = (int32_T)c6_threadId;
  if (c6_tmpIdx < 1) {
    c6_b_blankMsg->Name[2].Data_SL_Info.CurrentLength = 12U;
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel17(const
  uint8_T c6_uv3[11], c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_i3;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i3 = (int32_T)c6_threadId;
  if (c6_i3 < 11) {
    c6_b_blankMsg->Name[3].Data[c6_i3] = c6_uv3[c6_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel18
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_tmpIdx;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_tmpIdx = (int32_T)c6_threadId;
  if (c6_tmpIdx < 1) {
    c6_b_blankMsg->Name[3].Data_SL_Info.CurrentLength = 11U;
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel19(const
  uint8_T c6_uv4[12], c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_i4;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i4 = (int32_T)c6_threadId;
  if (c6_i4 < 12) {
    c6_b_blankMsg->Name[4].Data[c6_i4] = c6_uv4[c6_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel20
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_tmpIdx;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_tmpIdx = (int32_T)c6_threadId;
  if (c6_tmpIdx < 1) {
    c6_b_blankMsg->Name[4].Data_SL_Info.CurrentLength = 12U;
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel21(const
  uint8_T c6_uv5[13], c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_i5;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i5 = (int32_T)c6_threadId;
  if (c6_i5 < 13) {
    c6_b_blankMsg->Name[5].Data[c6_i5] = c6_uv5[c6_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel22
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_tmpIdx;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_tmpIdx = (int32_T)c6_threadId;
  if (c6_tmpIdx < 1) {
    c6_b_blankMsg->Name[5].Data_SL_Info.CurrentLength = 13U;
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel23(const
  uint8_T c6_uv6[8], c6_SL_Bus_sliding_controller2_sensor_msgs_JointState
  *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_i6;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_i6 = (int32_T)c6_threadId;
  if (c6_i6 < 8) {
    c6_b_blankMsg->Name[6].Data[c6_i6] = c6_uv6[c6_i6];
  }
}

static __global__ __launch_bounds__(32, 1) void c6_eML_blk_kernel_kernel24
  (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *c6_b_blankMsg)
{
  uint64_T c6_threadId;
  int32_T c6_tmpIdx;
  c6_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c6_tmpIdx = (int32_T)c6_threadId;
  if (c6_tmpIdx < 1) {
    c6_b_blankMsg->Name[6].Data_SL_Info.CurrentLength = 8U;
  }
}

static void init_dsm_address_info(SFc6_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc6_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c6_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c6_blankMsg =
    (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c6_msg = (c6_SL_Bus_sliding_controller2_sensor_msgs_JointState *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c6_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(936871815U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3255440910U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2911217935U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2629295146U);
}

mxArray *sf_c6_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c6_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c6_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c6_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiD+wcjAwAakOYCYiQECWKF8RqgYI1ycBS6uAMQllQWpIPHiomTPFCCdl5gL5ie"
    "WVnjmpeWDzbdgQJjPhsV8RiTzOaHiEPDBnjL9Ig4g/QZI+lmw6GdG0i8A5OUWp4PdDQsf8u1XcK"
    "BMP8T+AALuV0BxP4SfWRyfmFySWZYan2wWX5yTmZKZlx6fnJ9XUpSfk5NaZATzHwAnbh1n"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c6_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "s8raH7go15DoxtI0f9CCC6G";
}

static void sf_opaque_initialize_c6_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c6_sliding_controller2
    ((SFc6_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c6_sliding_controller2((SFc6_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c6_sliding_controller2(void *chartInstanceVar)
{
  enable_c6_sliding_controller2((SFc6_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c6_sliding_controller2(void *chartInstanceVar)
{
  disable_c6_sliding_controller2((SFc6_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c6_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c6_sliding_controller2((SFc6_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c6_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c6_sliding_controller2
    ((SFc6_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c6_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c6_sliding_controller2((SFc6_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c6_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc6_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c6_sliding_controller2
      ((SFc6_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc6_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc6_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c6_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c6_sliding_controller2((SFc6_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c6_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c6_sliding_controller2((SFc6_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc6_sliding_controller2((SFc6_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c6_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c6_sliding_controller2
      ((SFc6_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc6_sliding_controller2((SFc6_sliding_controller2InstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c6_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u00AQdkJBIEFVISQuSPQGR0CiwAVKnQQitbTCLRyr7XoSr7LeNfuTtneQuHDlGRAvg7g",
    "BT8GRI7O2m6aO7VACFSAsue7a33w7/7PxGt01D695vO9e9Lwz+DyLd9PLrtP5ujF2Z+/nvGv5+g",
    "0KCRtvEEVi7dVegsTwFLTk1jApuqInS2FM9ECBoIhNpDJVbJrFljMx6FhBHZ9+HjEaBZG0PFxBW",
    "RKuC76PbIk1G8jTYgqo6QCEJlLS9qMOJ/2Rxsrs+hHQgbZxnQkaTGATp5Zes9ywhEN7D2hXaENQ",
    "Y32oW2CIAd/sVZrpLNXBAVDGCWdElFobER1Agg42sJWE+HfdGjSqCKMRUWYFIjIEvcoGKacUUOR",
    "kGj/sMEGMVIzwdsx9Jzip2wZHfdZkCLzGIajbigIySCQTpjr+QQctbQuyw6EFO7ZfzRbAC+uC/4",
    "zBLqhKv/V8OQRF+rAuKjdNHdLeS6M1ypJJmGExPCPqIcX4aQgrsxczRwcE4wSbKFEFg9TIrt5Ub",
    "IjurWSzcddl5rSSsXEWbD0NlrK1h1AXhRFbhwqfcK4rYZsyWYUh8JS1RQyph2Ws5TitWbgp0cEu",
    "vaurwQqGgc9hvhQhKw3XsABI+84TbCxHkdRqI2Mfk7e1ujr5eRLWFQZUj1Ao6wKKMA3os9S91Ww",
    "h0y72CEStTKpeGTjLkGkoT/esaO1KNUCf1DSRQxNcRCuBse5jLLEStjQWTR3MxXIajhIaQegaDO",
    "OwhmWD2BKfaNfaHmLdDZnZb4GmiiUlUbVYddiG2i6h9hPYEgMhd0VHyTjIe3zmXgDMNqIEE/0V7",
    "F9qv4O7l4XCzbMb3uE8O/8D8+xArvi8PsbTKOHxxp7Ffc816/dt4n+NXG55TO5CYZ+5gpzDLeD9",
    "8fP7lx+XLr969/bL7S8fWq9n2f9d83jzfz5fXzlotKPCGU7kq8M+HtNrroT/8hj/Qr7WdxV5fKc",
    "vb95uyT3TvdG75/v+0qOU71ujXt9mQd+D94uu42OWpfmpaDfMDyZuTWw2rtPz0Ji+Z6b441z+Pr",
    "u+PphN/tJyMY5l/jp1xF+nXOUeyduf339xeTb5bP+NKfovFuK9mM71beK6BWzTpW3NWYiVvk2lM",
    "EpyDurW/C+os+PKeScs97fo+V/u98f9R+bWSct5Jyw3q33Hnd9/Gr5uDngF/MIfbMes56rfjf/k",
    "He/8czVf3x/9xPEjxsOS027+GY+uvbKv/0Cefgd7CLTe",
    ""
  };

  static char newstr [1245] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c6_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c6_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2225789393U));
  ssSetChecksum1(S,(2232891089U));
  ssSetChecksum2(S,(903517864U));
  ssSetChecksum3(S,(2286211034U));
}

static void mdlRTW_c6_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c6_sliding_controller2(SimStruct *S)
{
  SFc6_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc6_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc6_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc6_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc6_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c6_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c6_sliding_controller2;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c6_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c6_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c6_sliding_controller2;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c6_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c6_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c6_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c6_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c6_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c6_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c6_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c6_JITStateAnimation,
    chartInstance->c6_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c6_sliding_controller2(chartInstance);
}

void c6_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void
  *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c6_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c6_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c6_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c6_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
