/* Include files */

#include "sliding_controller2_sfun.h"
#include "c20_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void enable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void disable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void c20_do_animation_call_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance, const mxArray *c20_st);
static void sf_gateway_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void c20_eML_blk_kernel(SFc20_sliding_controller2InstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2]);
static void c20_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2]);
static void c20_b_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2]);
static uint8_T c20_c_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_is_active_c20_sliding_controller2, const
  char_T *c20_identifier);
static uint8_T c20_d_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId);
static void init_dsm_address_info(SFc20_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc20_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c20_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c20_is_active_c20_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c20_do_animation_call_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c20_b_y = NULL;
  const mxArray *c20_c_y = NULL;
  const mxArray *c20_d_y = NULL;
  const mxArray *c20_e_y = NULL;
  const mxArray *c20_f_y = NULL;
  const mxArray *c20_st;
  const mxArray *c20_y = NULL;
  c20_st = NULL;
  c20_y = NULL;
  sf_mex_assign(&c20_y, sf_mex_createcellmatrix(5, 1), false);
  c20_b_y = NULL;
  sf_mex_assign(&c20_b_y, sf_mex_create("y", *chartInstance->c20_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 0, c20_b_y);
  c20_c_y = NULL;
  sf_mex_assign(&c20_c_y, sf_mex_create("y", *chartInstance->c20_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 1, c20_c_y);
  c20_d_y = NULL;
  sf_mex_assign(&c20_d_y, sf_mex_create("y", *chartInstance->c20_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 2, c20_d_y);
  c20_e_y = NULL;
  sf_mex_assign(&c20_e_y, sf_mex_create("y", *chartInstance->c20_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c20_y, 3, c20_e_y);
  c20_f_y = NULL;
  sf_mex_assign(&c20_f_y, sf_mex_create("y",
    &chartInstance->c20_is_active_c20_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c20_y, 4, c20_f_y);
  sf_mex_assign(&c20_st, c20_y, false);
  return c20_st;
}

static void set_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance, const mxArray *c20_st)
{
  const mxArray *c20_u;
  c20_u = sf_mex_dup(c20_st);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 0)),
                       "ankle", *chartInstance->c20_ankle);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 1)),
                       "danke", *chartInstance->c20_danke);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 2)),
                       "dknee", *chartInstance->c20_dknee);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 3)),
                       "knee", *chartInstance->c20_knee);
  chartInstance->c20_is_active_c20_sliding_controller2 = c20_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 4)),
     "is_active_c20_sliding_controller2");
  sf_mex_destroy(&c20_u);
  sf_mex_destroy(&c20_st);
}

static void sf_gateway_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c20_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c20_eML_blk_kernel(chartInstance, *chartInstance->c20_q,
                     *chartInstance->c20_qd, *chartInstance->c20_knee,
                     *chartInstance->c20_dknee, *chartInstance->c20_ankle,
                     *chartInstance->c20_danke);
  c20_do_animation_call_c20_sliding_controller2(chartInstance);
}

static void mdl_start_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c20_errCode;
  c20_errCode = hipGetLastError();
  if (c20_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c20_errCode, (char_T *)hipGetErrorName
                       (c20_errCode), (char_T *)hipGetErrorString(c20_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c20_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c20_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c20_nameCaptureInfo = NULL;
  const char_T *c20_data[4] = {
    "789cc553cb4ac34014bd915adc545df90b2e04c7baeabe2a88580a0d6e44e224b969c6cc432653dbbaf10fec57f80ffe8e7b3fc2246dfa088688c57a20dc399c"
    "cc3df71e18b02eaf2d00d84d3efb05a055870c8d6981fd59dd82551475aba4e6d886dacabd54ef267e9319f7943438325322a9c0f94d5f0926a934f6f8114163",
    "acf813fa9912308e3613d85b269d94898b25694e52293db743f4a2de40800ee3c5847c996479a4b82fd9b756914711c53c8affe57ea35ffae5fd0f2afc723d50"
    "7a48b5ef444c32410df36287637ffdbd73df7ae91c53c5570397e3c2ef7d4d3fbbd46f55bf3dbf23a11248243521950c795f713fa0da251e35491ace3026b1f6",
    "08156ee00c298f509324204e5df27d66c722ed5b95dbde0ff7287b3f0dd8c9eadbeb11dda4dfc7e1a7b549bf1cffe5b7a9f777753266cd67f61045a7ad76a7e9"
    "bade8d6fce1673742b7caae68012fed7fdbf00b09967db", "" };

  c20_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c20_data[0], 1616U, &c20_nameCaptureInfo);
  return c20_nameCaptureInfo;
}

static void c20_eML_blk_kernel(SFc20_sliding_controller2InstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2])
{
  real_T c20_a;
  real_T c20_b_a;
  c20_b_knee[0] = -0.324 * muDoubleScalarSin(c20_b_q[0]);
  c20_b_knee[1] = -0.324 * muDoubleScalarCos(c20_b_q[0]);
  c20_a = -0.324 * c20_b_qd[0];
  c20_b_dknee[0] = c20_a * muDoubleScalarCos(c20_b_q[0]);
  c20_b_dknee[1] = c20_a * -muDoubleScalarSin(c20_b_q[0]);
  c20_b_a = -0.39 * (c20_b_qd[0] + c20_b_qd[1]);
  c20_b_ankle[0] = c20_b_knee[0] + -0.39 * muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_ankle[1] = c20_b_knee[1] + -0.39 * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[0] = c20_b_dknee[0] + c20_b_a * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[1] = c20_b_dknee[1] + c20_b_a * -muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
}

static void c20_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2])
{
  emlrtMsgIdentifier c20_thisId;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c20_b_ankle), &c20_thisId,
    c20_y);
  sf_mex_destroy(&c20_b_ankle);
}

static void c20_b_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2])
{
  real_T c20_dv[2];
  int32_T c20_i;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), c20_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c20_i = 0; c20_i < 2; c20_i++) {
    c20_y[c20_i] = c20_dv[c20_i];
  }

  sf_mex_destroy(&c20_u);
}

static uint8_T c20_c_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_is_active_c20_sliding_controller2, const
  char_T *c20_identifier)
{
  emlrtMsgIdentifier c20_thisId;
  uint8_T c20_y;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_y = c20_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c20_b_is_active_c20_sliding_controller2), &c20_thisId);
  sf_mex_destroy(&c20_b_is_active_c20_sliding_controller2);
  return c20_y;
}

static uint8_T c20_d_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId)
{
  uint8_T c20_b_u;
  uint8_T c20_y;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), &c20_b_u, 1, 3, 0U, 0, 0U, 0);
  c20_y = c20_b_u;
  sf_mex_destroy(&c20_u);
  return c20_y;
}

static void init_dsm_address_info(SFc20_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc20_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c20_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c20_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c20_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c20_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c20_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c20_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c20_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c20_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c20_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c20_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6LFA4ifWRyfmFySWZYan"
    "2xkEF+ck5mSmZcen5yfV1KUn5OTWmSEZD4IAACGpiyw"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c20_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c20_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c20_sliding_controller2
    ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c20_sliding_controller2(void *chartInstanceVar)
{
  enable_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c20_sliding_controller2(void *chartInstanceVar)
{
  disable_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c20_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c20_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c20_sliding_controller2
    ((SFc20_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c20_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c20_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c20_sliding_controller2
      ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc20_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c20_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c20_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c20_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c20_sliding_controller2
      ((SFc20_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc20_sliding_controller2
      ((SFc20_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c20_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIA28CFALwGSFAjao5EiQHtpbFMSIsCunVJOgyKAsVqOxIWWu+z+0HabJyi",
    "Qe9F36XvkGXJJgR7bQ2dJWpYpkorsJnWL0KDkJb/5dr6Z2dmV1xrsenjdxPuXW553Db+v49328m",
    "utGLfm7vx5x/usGL9EI2HjfaJIrL3GS5AYvgUtuTVMioEYy0oYE2NQIChiE6lMHZtmseVMTPtWU",
    "Menv4sYjYJIWh5uoy0J9wQ/QbbEmn3k6TIF1PQBQhMpaSdRn5PJzGNljvwI6FTbuEmCBhPYxLml",
    "dy03LOHQOwY6ENoQ9Fif+RYYYsA3x7UynVIdnAJlnHBGRKXaiOgAEgywgYMkxM89a1BUGUYjosw",
    "2RCQFvcOmGacUUOZkGl+MmCBGKkZ4L+a+M1z0bZ+jP7syBN4QEPRtWwGZJpIJU5//oI9Ke4KMOH",
    "RhZCf1bAH8YF3ynzI4AlUbt7EvU1BkAnuidtIsIL3jLFuzKlmEGRbDU6K2KOZPQ1hbvVg5OiCYJ",
    "xiiRR0MMpEDPVQsxfDWstl44Cpz2ZKxcZ5svQyWsfVSaMrCjK1PhU8417WwoUx2IAWesXaJIc2w",
    "nLUapzULhxID7Mq7fjVYwTDxBcyXImSV6UpLgKzvfION5TySWm1k7GPxdnd2Fl8vwgbCgBoTClV",
    "dQBGmAWOWhbeeLWTa5R6B6JXJ3KsC5xWyDOXpsRXdI6mmGJOGJnImwWW0FhjrCeYSV8KBxkXTBH",
    "O5XIajhEYQugbDOOziskFsRUy0a21buO5SZk66oKliSUVWLa46bEM9V1AnCRyIqZBHoq9kHBQ9P",
    "g8vAFYbUYKJyTb2L3XSx9mrUuH2sw3vbD/76C32s1O78vfnczytCh5v7rs87432eXy7NG8b/1qt",
    "3G5zzu7j0jydkp3DreP9+q8/7r948/Pop1fhi+e//vbnsvlbC/O3sv+d3fW11fb/m8X49mmjnS2",
    "cdKFeHfbxnF+dCv5P5vjXi7F+Ej0U6fePo60nZBgPNr748auNdORnfM86zf6ulfw9fX7XdXyssq",
    "w+FR2ExcHEjYnNt2vH/+Wcv9eWxONG8Ty/fn90Ofvbm2X7zhJ968WYiCmHUv3+ezo+vbCOEIVcH",
    "R13L65jioeGK6Pj1ma5P1Tp6JzT0fGchH/G/8U4rmafz/9sif/3Snm4l50XD4nbheCQPtg41JyF",
    "uIUcUimMkpyDelDR9y/aR1e1896z3X/Fzw927z7vb3Mual/QrnPJc9j7srusvlXPh1cN37QfeCX",
    "8+hXWcdlz+7vGv/JWO1/fKcZfz35C+xHjYcWvqeI1/jQaV739H9Tp30fQxWc=",
    ""
  };

  static char newstr [1337] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c20_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c20_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(636746986U));
  ssSetChecksum1(S,(1653272444U));
  ssSetChecksum2(S,(2086983803U));
  ssSetChecksum3(S,(4223507036U));
}

static void mdlRTW_c20_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c20_sliding_controller2(SimStruct *S)
{
  SFc20_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc20_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc20_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc20_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc20_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c20_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c20_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c20_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c20_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c20_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c20_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c20_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c20_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c20_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c20_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c20_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c20_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c20_JITStateAnimation,
    chartInstance->c20_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c20_sliding_controller2(chartInstance);
}

void c20_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c20_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c20_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c20_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c20_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
