/* Include files */

#include "sliding_controller2_sfun.h"
#include "c20_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void enable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void disable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void c20_do_animation_call_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance, const mxArray *c20_st);
static void sf_gateway_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance);
static void c20_eML_blk_kernel(SFc20_sliding_controller2InstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2]);
static void c20_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2]);
static void c20_b_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2]);
static uint8_T c20_c_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_is_active_c20_sliding_controller2, const
  char_T *c20_identifier);
static uint8_T c20_d_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId);
static void init_dsm_address_info(SFc20_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc20_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c20_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c20_is_active_c20_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c20_do_animation_call_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c20_b_y = NULL;
  const mxArray *c20_c_y = NULL;
  const mxArray *c20_d_y = NULL;
  const mxArray *c20_e_y = NULL;
  const mxArray *c20_f_y = NULL;
  const mxArray *c20_st;
  const mxArray *c20_y = NULL;
  c20_st = NULL;
  c20_y = NULL;
  sf_mex_assign(&c20_y, sf_mex_createcellmatrix(5, 1), false);
  c20_b_y = NULL;
  sf_mex_assign(&c20_b_y, sf_mex_create("y", *chartInstance->c20_ankle, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 0, c20_b_y);
  c20_c_y = NULL;
  sf_mex_assign(&c20_c_y, sf_mex_create("y", *chartInstance->c20_danke, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 1, c20_c_y);
  c20_d_y = NULL;
  sf_mex_assign(&c20_d_y, sf_mex_create("y", *chartInstance->c20_dknee, 0, 0U,
    1U, 0U, 1, 2), false);
  sf_mex_setcell(c20_y, 2, c20_d_y);
  c20_e_y = NULL;
  sf_mex_assign(&c20_e_y, sf_mex_create("y", *chartInstance->c20_knee, 0, 0U, 1U,
    0U, 1, 2), false);
  sf_mex_setcell(c20_y, 3, c20_e_y);
  c20_f_y = NULL;
  sf_mex_assign(&c20_f_y, sf_mex_create("y",
    &chartInstance->c20_is_active_c20_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c20_y, 4, c20_f_y);
  sf_mex_assign(&c20_st, c20_y, false);
  return c20_st;
}

static void set_sim_state_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance, const mxArray *c20_st)
{
  const mxArray *c20_u;
  c20_u = sf_mex_dup(c20_st);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 0)),
                       "ankle", *chartInstance->c20_ankle);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 1)),
                       "danke", *chartInstance->c20_danke);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 2)),
                       "dknee", *chartInstance->c20_dknee);
  c20_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 3)),
                       "knee", *chartInstance->c20_knee);
  chartInstance->c20_is_active_c20_sliding_controller2 = c20_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c20_u, 4)),
     "is_active_c20_sliding_controller2");
  sf_mex_destroy(&c20_u);
  sf_mex_destroy(&c20_st);
}

static void sf_gateway_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c20_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c20_eML_blk_kernel(chartInstance, *chartInstance->c20_q,
                     *chartInstance->c20_qd, *chartInstance->c20_knee,
                     *chartInstance->c20_dknee, *chartInstance->c20_ankle,
                     *chartInstance->c20_danke);
  c20_do_animation_call_c20_sliding_controller2(chartInstance);
}

static void mdl_start_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c20_errCode;
  c20_errCode = hipGetLastError();
  if (c20_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c20_errCode, (char_T *)hipGetErrorName
                       (c20_errCode), (char_T *)hipGetErrorString(c20_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c20_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc20_sliding_controller2
  (SFc20_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c20_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c20_nameCaptureInfo = NULL;
  const char_T *c20_data[4] = {
    "789cc5534d4bc340109d4a2d5eaa9e04c1ab170fae5ec4b3c5428596da0f3c88c649b26996ee876c52db7af1ee41ff8e7fcaff6092364d130c118bf541987dbc"
    "ecbc99070ba546b30400dbc1f7f602705e8108d55981dd79dd8034b27a29a7c6d88472ea5ea8bf067eef736e29e9d3893f2312055ddcb4956012a5df9b3e52d0",
    "d453fc89da91e2304e7b4cd0ee3269854cd497a40509a5f05c73a935ec8e0468d74b26e4cb24ca23c443cebee5823cb2c8e691fd2ff69bfcd22feebf57e017eb"
    "8ed263d4b631649209f499e5199c0e56df3bf6ade4ce31536c3532394dfc3e56f4ebe5faa5f5dbcb3be22a418944df45c9281f286e3ba84d62a11fa4618c3de2",
    "698ba0301d638c7c48350902e26892ef333b1661dfa2dc767eb847defba9c256543fdbfbb84ebffba383c375fac5f82fbf75bdbf9bf6059e9c3d5fb14e47d79b"
    "d7fd6ea37fdaaa2573b40b7c8ae6801cfed7fdbf0052dd6684", "" };

  c20_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c20_data[0], 1616U, &c20_nameCaptureInfo);
  return c20_nameCaptureInfo;
}

static void c20_eML_blk_kernel(SFc20_sliding_controller2InstanceStruct
  *chartInstance, real_T c20_b_q[2], real_T c20_b_qd[2], real_T c20_b_knee[2],
  real_T c20_b_dknee[2], real_T c20_b_ankle[2], real_T c20_b_danke[2])
{
  real_T c20_a;
  real_T c20_b_a;
  c20_b_knee[0] = -0.324 * muDoubleScalarCos(c20_b_q[0]);
  c20_b_knee[1] = -0.324 * muDoubleScalarSin(c20_b_q[0]);
  c20_a = -0.324 * c20_b_qd[0];
  c20_b_dknee[0] = c20_a * -muDoubleScalarSin(c20_b_q[0]);
  c20_b_dknee[1] = c20_a * muDoubleScalarCos(c20_b_q[0]);
  c20_b_a = -0.39 * (c20_b_qd[0] + c20_b_qd[1]);
  c20_b_ankle[0] = c20_b_knee[0] - -0.39 * muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_ankle[1] = c20_b_knee[1] - -0.39 * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[0] = c20_b_dknee[0] + c20_b_a * -muDoubleScalarSin(c20_b_q[0] +
    c20_b_q[1]);
  c20_b_danke[1] = c20_b_dknee[1] + c20_b_a * muDoubleScalarCos(c20_b_q[0] +
    c20_b_q[1]);
}

static void c20_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_ankle, const char_T *c20_identifier,
  real_T c20_y[2])
{
  emlrtMsgIdentifier c20_thisId;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c20_b_ankle), &c20_thisId,
    c20_y);
  sf_mex_destroy(&c20_b_ankle);
}

static void c20_b_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId,
  real_T c20_y[2])
{
  real_T c20_dv[2];
  int32_T c20_i;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), c20_dv, 1, 0, 0U, 1, 0U, 1, 2);
  for (c20_i = 0; c20_i < 2; c20_i++) {
    c20_y[c20_i] = c20_dv[c20_i];
  }

  sf_mex_destroy(&c20_u);
}

static uint8_T c20_c_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_b_is_active_c20_sliding_controller2, const
  char_T *c20_identifier)
{
  emlrtMsgIdentifier c20_thisId;
  uint8_T c20_y;
  c20_thisId.fIdentifier = const_cast<const char_T *>(c20_identifier);
  c20_thisId.fParent = NULL;
  c20_thisId.bParentIsCell = false;
  c20_y = c20_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c20_b_is_active_c20_sliding_controller2), &c20_thisId);
  sf_mex_destroy(&c20_b_is_active_c20_sliding_controller2);
  return c20_y;
}

static uint8_T c20_d_emlrt_marshallIn(SFc20_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c20_u, const emlrtMsgIdentifier *c20_parentId)
{
  uint8_T c20_b_u;
  uint8_T c20_y;
  sf_mex_import(c20_parentId, sf_mex_dup(c20_u), &c20_b_u, 1, 3, 0U, 0, 0U, 0);
  c20_y = c20_b_u;
  sf_mex_destroy(&c20_u);
  return c20_y;
}

static void init_dsm_address_info(SFc20_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc20_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c20_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c20_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c20_knee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_dknee = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c20_qd = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c20_ankle = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
  chartInstance->c20_danke = (real_T (*)[2])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 4);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c20_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1880680695U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1680452377U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(3479659674U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3127653308U);
}

mxArray *sf_c20_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c20_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c20_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c20_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiCOYGFgYAPSHEDMxAABrFA+I5TNCBdngYsrAHFJZUEqSLy4KNkzBUjnJeaC+Ym"
    "lFZ55aflg8y0YEOazYTGfEcl8Tqg4BHywp0y/jAO6fhYC/hOA8hPzsnNSoeEDC6eB84cS2f5IAX"
    "pk8PhDgXx/ZOelDh5/iID9YUDAHywo/mBhAHmBOu7HDEfS9EPzPQH3K6LFA4ifWRyfmFySWZYan"
    "2xkEF+ck5mSmZcen5yfV1KUn5OTWmSEZD4IAACGpiyw"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c20_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQh5nvZHhAQaTmI03z90vbC";
}

static void sf_opaque_initialize_c20_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c20_sliding_controller2
    ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c20_sliding_controller2(void *chartInstanceVar)
{
  enable_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c20_sliding_controller2(void *chartInstanceVar)
{
  disable_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c20_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c20_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c20_sliding_controller2
    ((SFc20_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c20_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c20_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c20_sliding_controller2
      ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc20_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc20_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c20_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c20_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc20_sliding_controller2((SFc20_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c20_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c20_sliding_controller2
      ((SFc20_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc20_sliding_controller2
      ((SFc20_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c20_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV81u20YQpgQ5SIAm8CFAL0GbFAHaSwEjRYH20tqmJESFHTulnAY91FgvR+JCy112f2ir79C",
    "ee+m9D9BLniBAnyCnPEPRJ+gsScsyRVKR3SROURqUvOQ33843Mzu78lqDXQ+vW3j/etvzruH3db",
    "zbXn6tFePW3J0/73gfF+Of0UjYeJ8oEmuv8RIkhm9BS24Nk2IgRrISxsQIFAiK2EQqU8emWWw5E",
    "5O+FdTx6e8iRqMgkpaH22hLwj3Bp8iWWLOPPF2mgJo+QGgiJe046nMynnmszLEfAZ1oGzdJ0GAC",
    "mzi39K7lhiUceidAB0Ibgh7rM98CQwz45qRWplOqg1OgjBPOiKhUGxEdQIIBNnCQhPi5Zw2KKsN",
    "oRJTZhoikoHfYJOOUAsqcTOOLIyaIkYoR3ou57wwXfdvn6M+uDIE3BAR921ZAJolkwtTnP+ij0p",
    "4gRxy6cGTH9WwB/Ghd8p8wOAZVG7eRL1NQZAx7onbSLCC9kyxbsypZhBkWwxOitijmT0NYW71YO",
    "TogmCcYokUdDDKRAz1ULMXw1rLZeOAqc9mSsXGebL0MlrH1UmjKwoytT4VPONe1sKFMdiAFnrF2",
    "iSHNsJy1Gqc1C4cSA+zKu341WMEw8QXMlyJklelKS4Cs7zzCxnIeSa02MvaxeLs7O4uvF2EDYUC",
    "NCIWqLqAI04Axy8JbzxYy7XKPQPTKZO5VgfMKWYby9MiK7rFUE4xJQxM5k+AyWguM9RhziSvhQO",
    "OiaYK5XC7DUUIjCF2DYRx2cdkgtiIm2rW2LVx3KTPTLmiqWFKRVYurDttQzxXUNIEDMRHyWPSVj",
    "IOix+fhBcBqI0owMd7G/qWmfZy9KhVuP9vwzvaz915hPzu1K39/MsfTquDx5r7L895on8e3S/O2",
    "8a/Vyu025+xulubplOwcbh3vb17+dv/RB3/8/fzPT3+Z/v7sh2Xztxbmb2X/O7vra6vt/7eK8Z3",
    "TRjtbOOlCvTrswzm/OhX878/xrxdj/Tj6XKTfP4y2HpNhPNj47KcvN9IjP+N72mn2d63k7+nzu6",
    "7jY5Vl9anoICwOJm5MbL5dO/4v5vy9tiQeN4rn+fXX15ezv7NZtu8s0bdejImYcCjV79vT8dGFd",
    "YQo5OrouHtxHRM8NFwZHbc3y/2hSkfnnI6O5yT8O/4vxnE1+3z+p0v8v1fKw73svHhI3C4Eh/TB",
    "xqHmLMQt5JBKYZTkHNSDir5/0T66qp33hu3eFT//t3v9eX+Vc1H7gnadS57D3pTdZfWtej68avi",
    "m/cAr4devsI7LnttfN/6Ft9r5+sNi/NXsJ7QfMR5W/JoqXuNPo1HV2/9Anf4DLDPDsA==",
    ""
  };

  static char newstr [1345] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c20_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c20_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(614325834U));
  ssSetChecksum1(S,(4071890510U));
  ssSetChecksum2(S,(2385365703U));
  ssSetChecksum3(S,(1589225593U));
}

static void mdlRTW_c20_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c20_sliding_controller2(SimStruct *S)
{
  SFc20_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc20_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc20_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc20_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc20_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c20_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c20_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c20_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c20_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c20_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c20_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c20_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c20_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c20_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c20_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c20_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c20_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c20_JITStateAnimation,
    chartInstance->c20_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c20_sliding_controller2(chartInstance);
}

void c20_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c20_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c20_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c20_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c20_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
