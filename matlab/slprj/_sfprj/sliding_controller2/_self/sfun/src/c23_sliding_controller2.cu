/* Include files */

#include "sliding_controller2_sfun.h"
#include "c23_sliding_controller2.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void enable_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void disable_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void c23_do_animation_call_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance, const mxArray *c23_st);
static void sf_gateway_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance);
static void c23_eML_blk_kernel(SFc23_sliding_controller2InstanceStruct
  *chartInstance, real_T c23_b_u[7], real_T c23_b_y[7]);
static void c23_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_y, const char_T *c23_identifier, real_T
  c23_c_y[7]);
static void c23_b_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_u, const emlrtMsgIdentifier *c23_parentId,
  real_T c23_b_y[7]);
static uint8_T c23_c_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_is_active_c23_sliding_controller2, const
  char_T *c23_identifier);
static uint8_T c23_d_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_u, const emlrtMsgIdentifier *c23_parentId);
static void init_dsm_address_info(SFc23_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc23_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c23_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c23_is_active_c23_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c23_do_animation_call_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c23_b_y = NULL;
  const mxArray *c23_c_y = NULL;
  const mxArray *c23_d_y = NULL;
  const mxArray *c23_st;
  c23_st = NULL;
  c23_b_y = NULL;
  sf_mex_assign(&c23_b_y, sf_mex_createcellmatrix(2, 1), false);
  c23_c_y = NULL;
  sf_mex_assign(&c23_c_y, sf_mex_create("y", *chartInstance->c23_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c23_b_y, 0, c23_c_y);
  c23_d_y = NULL;
  sf_mex_assign(&c23_d_y, sf_mex_create("y",
    &chartInstance->c23_is_active_c23_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c23_b_y, 1, c23_d_y);
  sf_mex_assign(&c23_st, c23_b_y, false);
  return c23_st;
}

static void set_sim_state_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance, const mxArray *c23_st)
{
  const mxArray *c23_b_u;
  c23_b_u = sf_mex_dup(c23_st);
  c23_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c23_b_u, 0)),
                       "y", *chartInstance->c23_y);
  chartInstance->c23_is_active_c23_sliding_controller2 = c23_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c23_b_u, 1)),
     "is_active_c23_sliding_controller2");
  sf_mex_destroy(&c23_b_u);
  sf_mex_destroy(&c23_st);
}

static void sf_gateway_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c23_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c23_eML_blk_kernel(chartInstance, *chartInstance->c23_u, *chartInstance->c23_y);
  c23_do_animation_call_c23_sliding_controller2(chartInstance);
}

static void mdl_start_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c23_errCode;
  c23_errCode = hipGetLastError();
  if (c23_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c23_errCode, (char_T *)hipGetErrorName
                       (c23_errCode), (char_T *)hipGetErrorString(c23_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c23_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc23_sliding_controller2
  (SFc23_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c23_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c23_nameCaptureInfo = NULL;
  c23_nameCaptureInfo = NULL;
  sf_mex_assign(&c23_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c23_nameCaptureInfo;
}

static void c23_eML_blk_kernel(SFc23_sliding_controller2InstanceStruct
  *chartInstance, real_T c23_b_u[7], real_T c23_b_y[7])
{
  c23_b_y[0] = c23_b_u[3];
  c23_b_y[1] = c23_b_u[6];
  c23_b_y[2] = c23_b_u[0];
  c23_b_y[3] = c23_b_u[1];
  c23_b_y[4] = c23_b_u[4];
  c23_b_y[5] = c23_b_u[5];
  c23_b_y[6] = c23_b_u[2];
}

static void c23_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_y, const char_T *c23_identifier, real_T
  c23_c_y[7])
{
  emlrtMsgIdentifier c23_thisId;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c23_b_y), &c23_thisId,
    c23_c_y);
  sf_mex_destroy(&c23_b_y);
}

static void c23_b_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_u, const emlrtMsgIdentifier *c23_parentId,
  real_T c23_b_y[7])
{
  real_T c23_dv[7];
  int32_T c23_i;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_b_u), c23_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c23_i = 0; c23_i < 7; c23_i++) {
    c23_b_y[c23_i] = c23_dv[c23_i];
  }

  sf_mex_destroy(&c23_b_u);
}

static uint8_T c23_c_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_is_active_c23_sliding_controller2, const
  char_T *c23_identifier)
{
  emlrtMsgIdentifier c23_thisId;
  uint8_T c23_b_y;
  c23_thisId.fIdentifier = const_cast<const char_T *>(c23_identifier);
  c23_thisId.fParent = NULL;
  c23_thisId.bParentIsCell = false;
  c23_b_y = c23_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c23_b_is_active_c23_sliding_controller2), &c23_thisId);
  sf_mex_destroy(&c23_b_is_active_c23_sliding_controller2);
  return c23_b_y;
}

static uint8_T c23_d_emlrt_marshallIn(SFc23_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c23_b_u, const emlrtMsgIdentifier *c23_parentId)
{
  uint8_T c23_b_y;
  uint8_T c23_c_u;
  sf_mex_import(c23_parentId, sf_mex_dup(c23_b_u), &c23_c_u, 1, 3, 0U, 0, 0U, 0);
  c23_b_y = c23_c_u;
  sf_mex_destroy(&c23_b_u);
  return c23_b_y;
}

static void init_dsm_address_info(SFc23_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc23_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c23_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c23_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c23_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c23_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3007509169U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1926089682U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2239166282U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3770191902U);
}

mxArray *sf_c23_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c23_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c23_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c23_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8clGxvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABbQhvX"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c23_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sdegTANPF1GH9p3UVy5QBRH";
}

static void sf_opaque_initialize_c23_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c23_sliding_controller2
    ((SFc23_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c23_sliding_controller2(void *chartInstanceVar)
{
  enable_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c23_sliding_controller2(void *chartInstanceVar)
{
  disable_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c23_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c23_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c23_sliding_controller2
    ((SFc23_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c23_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c23_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc23_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c23_sliding_controller2
      ((SFc23_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc23_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc23_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c23_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c23_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc23_sliding_controller2((SFc23_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c23_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c23_sliding_controller2
      ((SFc23_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc23_sliding_controller2
      ((SFc23_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c23_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u1EgQdkYhIhJEEULigthwQHDbBYQEl91k/jYjJSQbTwIXFHXaNePWtLu9/TPJnDmueAp",
    "egAvvsk+x4rJ7W2mrbc9k1mN7CCMiQFhyPG1/VV1VX/10vKXOrofXGt6bNzxvBZ9X8a556XUlWy",
    "9N3en7Ze9+tn6DQsJG+0SRSHuVlyARHICW3BomRUf0ZCGMiR4oEBSxsVSmTJtmkeVMDNpWUKdPv",
    "wgZDf1QWh7UUZYEe4KPUFtszT7qaTIF1LQBAhMqafthm5P+xGJlThsh0IG2UZULGoxvY2eW3rXc",
    "sJhD6wxoR2hD0GJ9bptviIGGOSt103mq/TFQRjFnRBR6GxLtQ4wBNnAYB/h3zxp0Kg+jIVGmDiE",
    "Zgt5hg0SnFJDXyTR+OGGCGKkY4a2IN5zgrG37HO3ZlQHwioCgbXUFZBBLJkw5/34bPW0JcsKhCS",
    "e2X67Nh9+tI/+IwSmo0rj1GnIIivRhT5RumgSkdZawNcmSWZhhERwRtUWRPw1BafZi5mifIE/QR",
    "YkyGCROdnRXsSGGt1SbjTouM+eVjI1SsvU8WKKtNYQqFiba2lQ0COe6FNaV8Q4MgSdam8SQaliq",
    "tRinNQu6EgPs0ru8GqxgSHwGa0gRsEK6hjlA0neeY2P5P5JabWTUwORt7uzMfp6FdYQB1SMUirq",
    "AIkwDxiwJb7m2gGnHPQLRKpOYVwROM2QeytM9K5qnUg0wJhVN5NwFx2gpMNJ95BIr4VBj0VTBHJ",
    "fzcJTQEALXYBiHXSwbxBbERLvWtoV1N2Rm1ARNFYsLWLVYddiGWi6hRjEcioGQp6KtZORnPT4NL",
    "wBmG1GCiX4d+5catXH3IircPPvJO59n1z5ino3l8s8HU3qWCvR4U8/8vqu16n1r+GtpPH+n5K7n",
    "9lnOyTncOt7/3nzx/o8fX/+99pa8+uufe3cW2f9d7WLzfy1b3x432knhDGfy1WG3p+xaLtB/a0r",
    "/erbWAfS7W8/32w9/3X4WPz48Gj35rX6wnfIzx95azt7x+w3X8THLkvxUtBNkBxO3JjYd107/0y",
    "l7V+bEYzV7n14ffllM/uZmnsflOfLr+GuUy9tP339jczH5dP+Xc+y/m+P7bjLXj4nrFnBMHz0+1",
    "pwFWOrHVAqjJOegHhXU56fm+0XlvEuW+1rs/C73+Xn/mPl12XLeJcst6t9F5/iXhq+aB14Ov/4F",
    "+7Ho+epz4//0LnYO+iFb/zz5V6cRMh4UnHqzz3iE7RV9/Qby9D9rt7IO",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c23_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c23_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042383101U));
  ssSetChecksum1(S,(4119015307U));
  ssSetChecksum2(S,(1566679822U));
  ssSetChecksum3(S,(505804526U));
}

static void mdlRTW_c23_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c23_sliding_controller2(SimStruct *S)
{
  SFc23_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc23_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc23_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc23_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc23_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c23_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c23_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c23_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c23_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c23_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c23_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c23_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c23_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c23_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c23_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c23_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c23_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c23_JITStateAnimation,
    chartInstance->c23_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c23_sliding_controller2(chartInstance);
}

void c23_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c23_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c23_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c23_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c23_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
