#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c17_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#include "MWGpuFunctionImplUtility.hpp"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include "MWCUSOLVERUtils.hpp"
#include "hip/hip_math_constants.h"

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void enable_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void disable_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void c17_do_animation_call_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance, const mxArray *c17_st);
static void sf_gateway_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance);
static void c17_eML_blk_kernel(SFc17_sliding_controller2InstanceStruct
  *chartInstance, real_T c17_b_Tid[7], real_T c17_b_Th[7], real_T c17_b_qd[7],
  real_T c17_b_K[49], real_T c17_b_B[49]);
static void c17_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_b_B, const char_T *c17_identifier, real_T
  c17_y[49]);
static void c17_b_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId,
  real_T c17_y[49]);
static uint8_T c17_c_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_b_is_active_c17_sliding_controller2, const
  char_T *c17_identifier);
static uint8_T c17_d_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId);
static __global__ void c17_eML_blk_kernel_kernel1(const real_T c17_b_Tid[7],
  const real_T c17_b_qd[7], const real_T c17_b_Th[7], ptrdiff_t c17_jpvt_t[7],
  real_T c17_A[7], real_T c17_x[7], real_T c17_b_x[7]);
static __global__ void c17_eML_blk_kernel_kernel2(const ptrdiff_t c17_jpvt_t[7],
  int32_T c17_jpvt[7]);
static __global__ void c17_eML_blk_kernel_kernel3(int32_T c17_jpvt[7], real_T
  c17_A[7]);
static __global__ void c17_eML_blk_kernel_kernel4(real_T c17_change[49]);
static __global__ void c17_eML_blk_kernel_kernel5(const real_T c17_b_Th[7],
  real_T c17_b_B[7]);
static __global__ void c17_eML_blk_kernel_kernel6(real_T c17_b_B[7]);
static __global__ void c17_eML_blk_kernel_kernel7(const int32_T c17_jpvt[7],
  const int32_T c17_rankA, real_T c17_A[7], real_T c17_b_B[7], real_T
  c17_change[49]);
static __global__ void c17_eML_blk_kernel_kernel8(const real_T c17_change[49],
  real_T c17_Y[49]);
static __global__ void c17_eML_blk_kernel_kernel9(const real_T c17_Y[49], real_T
  c17_b_K[49], real_T c17_change[49]);
static __global__ void c17_eML_blk_kernel_kernel10(real_T c17_b_K[49]);
static __global__ void c17_eML_blk_kernel_kernel11(real_T c17_b_B[49]);
static __global__ void c17_eML_blk_kernel_kernel12(const int8_T c17_Kp0[49],
  const real_T c17_change[49], const int8_T c17_Kn0[49], const real_T c17_x[7],
  const real_T c17_b_x[7], real_T c17_b_K[49], real_T c17_b_B[49]);
static void init_dsm_address_info(SFc17_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc17_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c17_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c17_is_active_c17_sliding_controller2 = 0U;
  cusolverEnsureInitialization();
  hipGetLastError();
  hipMalloc(&chartInstance->c17_gpu_Kn0, 49UL);
  hipMalloc(&chartInstance->c17_gpu_Kp0, 49UL);
  hipMalloc(&chartInstance->c17_gpu_B, 392UL);
  hipMalloc(&chartInstance->c17_gpu_K, 392UL);
  hipMalloc(&chartInstance->c17_gpu_Y, 392UL);
  hipMalloc(&chartInstance->c17_gpu_info_t, 4UL);
  hipMalloc(&chartInstance->c17_gpu_tau, 8UL);
  hipMalloc(&chartInstance->c17_b_gpu_B, 56UL);
  hipMalloc(&chartInstance->c17_gpu_change, 392UL);
  hipMalloc(&chartInstance->c17_gpu_jpvt, 28UL);
  hipMalloc(&chartInstance->c17_gpu_x, 56UL);
  hipMalloc(&chartInstance->c17_b_gpu_x, 56UL);
  hipMalloc(&chartInstance->c17_gpu_A, 56UL);
  hipMalloc(&chartInstance->c17_gpu_jpvt_t, 56UL);
  hipMalloc(&chartInstance->c17_gpu_Th, 56UL);
  hipMalloc(&chartInstance->c17_gpu_qd, 56UL);
  hipMalloc(&chartInstance->c17_gpu_Tid, 56UL);
}

static void initialize_params_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c17_do_animation_call_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c17_b_y = NULL;
  const mxArray *c17_c_y = NULL;
  const mxArray *c17_d_y = NULL;
  const mxArray *c17_st;
  const mxArray *c17_y = NULL;
  c17_st = NULL;
  c17_y = NULL;
  sf_mex_assign(&c17_y, sf_mex_createcellmatrix(3, 1), false);
  c17_b_y = NULL;
  sf_mex_assign(&c17_b_y, sf_mex_create("y", *chartInstance->c17_B, 0, 0U, 1U,
    0U, 2, 7, 7), false);
  sf_mex_setcell(c17_y, 0, c17_b_y);
  c17_c_y = NULL;
  sf_mex_assign(&c17_c_y, sf_mex_create("y", *chartInstance->c17_K, 0, 0U, 1U,
    0U, 2, 7, 7), false);
  sf_mex_setcell(c17_y, 1, c17_c_y);
  c17_d_y = NULL;
  sf_mex_assign(&c17_d_y, sf_mex_create("y",
    &chartInstance->c17_is_active_c17_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c17_y, 2, c17_d_y);
  sf_mex_assign(&c17_st, c17_y, false);
  return c17_st;
}

static void set_sim_state_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance, const mxArray *c17_st)
{
  const mxArray *c17_u;
  c17_u = sf_mex_dup(c17_st);
  c17_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 0)), "B",
                       *chartInstance->c17_B);
  c17_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 1)), "K",
                       *chartInstance->c17_K);
  chartInstance->c17_is_active_c17_sliding_controller2 = c17_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c17_u, 2)),
     "is_active_c17_sliding_controller2");
  sf_mex_destroy(&c17_u);
  sf_mex_destroy(&c17_st);
}

static void sf_gateway_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  real_T c17_dv3[49];
  real_T c17_dv4[49];
  real_T c17_dv[7];
  real_T c17_dv1[7];
  real_T c17_dv2[7];
  int32_T c17_i;
  int32_T c17_i1;
  int32_T c17_i2;
  int32_T c17_i3;
  int32_T c17_i4;
  chartInstance->c17_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c17_i = 0; c17_i < 7; c17_i++) {
    c17_dv[c17_i] = (*chartInstance->c17_Tid)[c17_i];
  }

  for (c17_i1 = 0; c17_i1 < 7; c17_i1++) {
    c17_dv1[c17_i1] = (*chartInstance->c17_Th)[c17_i1];
  }

  for (c17_i2 = 0; c17_i2 < 7; c17_i2++) {
    c17_dv2[c17_i2] = (*chartInstance->c17_qd)[c17_i2];
  }

  c17_eML_blk_kernel(chartInstance, c17_dv, c17_dv1, c17_dv2, c17_dv3, c17_dv4);
  for (c17_i3 = 0; c17_i3 < 49; c17_i3++) {
    (*chartInstance->c17_B)[c17_i3] = c17_dv4[c17_i3];
  }

  for (c17_i4 = 0; c17_i4 < 49; c17_i4++) {
    (*chartInstance->c17_K)[c17_i4] = c17_dv3[c17_i4];
  }

  c17_do_animation_call_c17_sliding_controller2(chartInstance);
}

static void mdl_start_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c17_errCode;
  hipFree(*chartInstance->c17_gpu_Kn0);
  hipFree(*chartInstance->c17_gpu_Kp0);
  hipFree(*chartInstance->c17_gpu_B);
  hipFree(*chartInstance->c17_gpu_K);
  hipFree(*chartInstance->c17_gpu_Y);
  hipFree(chartInstance->c17_gpu_info_t);
  hipFree(chartInstance->c17_gpu_tau);
  hipFree(*chartInstance->c17_b_gpu_B);
  hipFree(*chartInstance->c17_gpu_change);
  hipFree(*chartInstance->c17_gpu_jpvt);
  hipFree(*chartInstance->c17_gpu_x);
  hipFree(*chartInstance->c17_b_gpu_x);
  hipFree(*chartInstance->c17_gpu_A);
  hipFree(*chartInstance->c17_gpu_jpvt_t);
  hipFree(*chartInstance->c17_gpu_Th);
  hipFree(*chartInstance->c17_gpu_qd);
  hipFree(*chartInstance->c17_gpu_Tid);
  c17_errCode = hipGetLastError();
  if (c17_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c17_errCode, (char_T *)hipGetErrorName
                       (c17_errCode), (char_T *)hipGetErrorString(c17_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c17_fEmlrtCtx);
  }

  cusolverDestroyWorkspace();
  cusolverEnsureDestruction();
}

static void mdl_setup_runtime_resources_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc17_sliding_controller2
  (SFc17_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c17_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c17_nameCaptureInfo = NULL;
  c17_nameCaptureInfo = NULL;
  sf_mex_assign(&c17_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c17_nameCaptureInfo;
}

static void c17_eML_blk_kernel(SFc17_sliding_controller2InstanceStruct
  *chartInstance, real_T c17_b_Tid[7], real_T c17_b_Th[7], real_T c17_b_qd[7],
  real_T c17_b_K[49], real_T c17_b_B[49])
{
  static int8_T c17_Kn0[49] = { 50, 0, 0, 0, 0, 0, 0, 0, 50, 0, 0, 0, 0, 0, 0, 0,
    50, 0, 0, 0, 0, 0, 0, 0, 50, 0, 0, 0, 0, 0, 0, 0, 50, 0, 0, 0, 0, 0, 0, 0,
    50, 0, 0, 0, 0, 0, 0, 0, 50 };

  static int8_T c17_Kp0[49] = { 75, 0, 0, 0, 0, 0, 0, 0, 75, 0, 0, 0, 0, 0, 0, 0,
    75, 0, 0, 0, 0, 0, 0, 0, 75, 0, 0, 0, 0, 0, 0, 0, 75, 0, 0, 0, 0, 0, 0, 0,
    75, 0, 0, 0, 0, 0, 0, 0, 75 };

  ptrdiff_t c17_jpvt_t[7];
  ptrdiff_t c17_info_t;
  real_T c17_A[7];
  real_T c17_tau;
  real_T c17_tol;
  int32_T c17_b_info_t;
  int32_T c17_rankA;
  boolean_T c17_A_dirtyOnCpu;
  boolean_T c17_A_dirtyOnGpu;
  hipMemcpy(*chartInstance->c17_gpu_Tid, c17_b_Tid, 56UL,
             hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_qd, c17_b_qd, 56UL, hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_Th, c17_b_Th, 56UL, hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Tid, *chartInstance->c17_gpu_qd,
     *chartInstance->c17_gpu_Th, *chartInstance->c17_gpu_jpvt_t,
     *chartInstance->c17_gpu_A, *chartInstance->c17_b_gpu_x,
     *chartInstance->c17_gpu_x);
  hipMemcpy(c17_A, *chartInstance->c17_gpu_A, 56UL, hipMemcpyDeviceToHost);
  hipMemcpy(c17_jpvt_t, *chartInstance->c17_gpu_jpvt_t, 56UL,
             hipMemcpyDeviceToHost);
  c17_info_t = LAPACKE_dgeqp3(102, (ptrdiff_t)1, (ptrdiff_t)7, &c17_A[0],
    (ptrdiff_t)1, &c17_jpvt_t[0], &c17_tau);
  c17_A_dirtyOnGpu = false;
  c17_A_dirtyOnCpu = true;
  if ((int32_T)c17_info_t != 0) {
    c17_tau = rtNaN;
    c17_eML_blk_kernel_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_gpu_jpvt, *chartInstance->c17_gpu_A);
    c17_A_dirtyOnCpu = false;
    c17_A_dirtyOnGpu = true;
  } else {
    hipMemcpy(*chartInstance->c17_gpu_jpvt_t, c17_jpvt_t, 56UL,
               hipMemcpyHostToDevice);
    c17_eML_blk_kernel_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_gpu_jpvt_t, *chartInstance->c17_gpu_jpvt);
  }

  c17_rankA = 0;
  if (c17_A_dirtyOnGpu) {
    hipMemcpy(c17_A, *chartInstance->c17_gpu_A, 56UL, hipMemcpyDeviceToHost);
  }

  c17_tol = 1.5543122344752192E-14 * muDoubleScalarAbs(c17_A[0]);
  if (!(muDoubleScalarAbs(c17_A[0]) <= c17_tol)) {
    c17_rankA = 1;
  }

  c17_eML_blk_kernel_kernel4<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Th, *chartInstance->c17_b_gpu_B);
  c17_b_info_t = 0;
  if (c17_A_dirtyOnCpu) {
    hipMemcpy(*chartInstance->c17_gpu_A, c17_A, 56UL, hipMemcpyHostToDevice);
  }

  hipMemcpy(chartInstance->c17_gpu_tau, &c17_tau, 8UL, hipMemcpyHostToDevice);
  hipsolverDnDormqr_bufferSize(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT,
    HIPBLAS_OP_T, 1, 7, 1, (double *)&(*chartInstance->c17_gpu_A)[0], 1, (double *)
    chartInstance->c17_gpu_tau, (double *)&(*chartInstance->c17_b_gpu_B)[0], 1,
    getCuSolverWorkspaceReq());
  setCuSolverWorkspaceTypeSize(8);
  cusolverInitWorkspace();
  hipMemcpy(chartInstance->c17_gpu_info_t, &c17_b_info_t, 4UL,
             hipMemcpyHostToDevice);
  hipsolverDnDormqr(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, 1,
                   7, 1, (double *)&(*chartInstance->c17_gpu_A)[0], 1, (double *)
                   chartInstance->c17_gpu_tau, (double *)
                   &(*chartInstance->c17_b_gpu_B)[0], 1, (real_T *)
                   getCuSolverWorkspaceBuff(), *getCuSolverWorkspaceReq(),
                   chartInstance->c17_gpu_info_t);
  hipMemcpy(&c17_b_info_t, chartInstance->c17_gpu_info_t, 4UL,
             hipMemcpyDeviceToHost);
  if (c17_b_info_t < 0) {
    c17_eML_blk_kernel_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c17_b_gpu_B);
  }

  c17_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_jpvt, c17_rankA, *chartInstance->c17_gpu_A,
     *chartInstance->c17_b_gpu_B, *chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_change, *chartInstance->c17_gpu_Y);
  c17_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Y, *chartInstance->c17_gpu_K,
     *chartInstance->c17_gpu_change);
  c17_eML_blk_kernel_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_K);
  c17_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_B);
  hipMemcpy(*chartInstance->c17_gpu_Kp0, c17_Kp0, 49UL, hipMemcpyHostToDevice);
  hipMemcpy(*chartInstance->c17_gpu_Kn0, c17_Kn0, 49UL, hipMemcpyHostToDevice);
  c17_eML_blk_kernel_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c17_gpu_Kp0, *chartInstance->c17_gpu_change,
     *chartInstance->c17_gpu_Kn0, *chartInstance->c17_b_gpu_x,
     *chartInstance->c17_gpu_x, *chartInstance->c17_gpu_K,
     *chartInstance->c17_gpu_B);
  hipMemcpy(c17_b_K, *chartInstance->c17_gpu_K, 392UL, hipMemcpyDeviceToHost);
  hipMemcpy(c17_b_B, *chartInstance->c17_gpu_B, 392UL, hipMemcpyDeviceToHost);
}

static void c17_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_b_B, const char_T *c17_identifier, real_T
  c17_y[49])
{
  emlrtMsgIdentifier c17_thisId;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c17_b_B), &c17_thisId, c17_y);
  sf_mex_destroy(&c17_b_B);
}

static void c17_b_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId,
  real_T c17_y[49])
{
  real_T c17_dv[49];
  int32_T c17_i;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), c17_dv, 1, 0, 0U, 1, 0U, 2, 7,
                7);
  for (c17_i = 0; c17_i < 49; c17_i++) {
    c17_y[c17_i] = c17_dv[c17_i];
  }

  sf_mex_destroy(&c17_u);
}

static uint8_T c17_c_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_b_is_active_c17_sliding_controller2, const
  char_T *c17_identifier)
{
  emlrtMsgIdentifier c17_thisId;
  uint8_T c17_y;
  c17_thisId.fIdentifier = const_cast<const char_T *>(c17_identifier);
  c17_thisId.fParent = NULL;
  c17_thisId.bParentIsCell = false;
  c17_y = c17_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c17_b_is_active_c17_sliding_controller2), &c17_thisId);
  sf_mex_destroy(&c17_b_is_active_c17_sliding_controller2);
  return c17_y;
}

static uint8_T c17_d_emlrt_marshallIn(SFc17_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c17_u, const emlrtMsgIdentifier *c17_parentId)
{
  uint8_T c17_b_u;
  uint8_T c17_y;
  sf_mex_import(c17_parentId, sf_mex_dup(c17_u), &c17_b_u, 1, 3, 0U, 0, 0U, 0);
  c17_y = c17_b_u;
  sf_mex_destroy(&c17_u);
  return c17_y;
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel1(const
  real_T c17_b_Tid[7], const real_T c17_b_qd[7], const real_T c17_b_Th[7],
  ptrdiff_t c17_jpvt_t[7], real_T c17_A[7], real_T c17_x[7], real_T c17_b_x[7])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    c17_b_x[c17_k] = c17_b_Th[c17_k];
    c17_b_x[c17_k] = mwGpuSign(c17_b_x[c17_k]);
    c17_x[c17_k] = c17_b_qd[c17_k];
    c17_x[c17_k] = mwGpuSign(c17_x[c17_k]);
    c17_A[c17_k] = c17_b_Tid[c17_k];
    c17_jpvt_t[c17_k] = (ptrdiff_t)0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel2(const
  ptrdiff_t c17_jpvt_t[7], int32_T c17_jpvt[7])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    c17_jpvt[c17_k] = (int32_T)c17_jpvt_t[c17_k];
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel3
  (int32_T c17_jpvt[7], real_T c17_A[7])
{
  uint64_T c17_threadId;
  int32_T c17_j;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_j = (int32_T)c17_threadId;
  if (c17_j < 7) {
    c17_A[c17_j] = HIP_NAN;
    c17_jpvt[c17_j] = c17_j + 1;
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel4
  (real_T c17_change[49])
{
  uint64_T c17_threadId;
  int32_T c17_i;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i = (int32_T)c17_threadId;
  if (c17_i < 49) {
    c17_change[c17_i] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel5(const
  real_T c17_b_Th[7], real_T c17_b_B[7])
{
  uint64_T c17_threadId;
  int32_T c17_i1;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i1 = (int32_T)c17_threadId;
  if (c17_i1 < 7) {
    c17_b_B[c17_i1] = c17_b_Th[c17_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel6
  (real_T c17_b_B[7])
{
  uint64_T c17_threadId;
  int32_T c17_i2;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i2 = (int32_T)c17_threadId;
  if (c17_i2 < 7) {
    c17_b_B[c17_i2] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel7(const
  int32_T c17_jpvt[7], const int32_T c17_rankA, real_T c17_A[7], real_T c17_b_B
  [7], real_T c17_change[49])
{
  uint64_T c17_threadId;
  int32_T c17_i;
  int32_T c17_j;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    for (c17_i = 0; c17_i < c17_rankA; c17_i++) {
      c17_change[(c17_jpvt[0] + 7 * c17_k) - 1] = c17_b_B[c17_k];
    }

    for (c17_j = 0; c17_j < c17_rankA; c17_j++) {
      c17_change[(c17_jpvt[c17_rankA - 1] + 7 * c17_k) - 1] /= c17_A[(c17_rankA
        + c17_rankA) - 2];
    }
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel8(const
  real_T c17_change[49], real_T c17_Y[49])
{
  uint64_T c17_threadId;
  int32_T c17_i3;
  int32_T c17_i4;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i4 = (int32_T)(c17_threadId % 7UL);
  c17_i3 = (int32_T)((c17_threadId - (uint64_T)c17_i4) / 7UL);
  if ((c17_i3 < 7) && (c17_i4 < 7)) {
    c17_Y[c17_i4 + 7 * c17_i3] = c17_change[c17_i3 + 7 * c17_i4];
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel9(const
  real_T c17_Y[49], real_T c17_b_K[49], real_T c17_change[49])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 49) {
    c17_change[c17_k] = fabs(c17_Y[c17_k]);
    c17_b_K[c17_k] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel10
  (real_T c17_b_K[49])
{
  uint64_T c17_threadId;
  int32_T c17_k;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_k = (int32_T)c17_threadId;
  if (c17_k < 7) {
    c17_b_K[c17_k + 7 * c17_k] = 1.0;
  }
}

static __global__ __launch_bounds__(64, 1) void c17_eML_blk_kernel_kernel11
  (real_T c17_b_B[49])
{
  uint64_T c17_threadId;
  int32_T c17_i5;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_i5 = (int32_T)c17_threadId;
  if (c17_i5 < 49) {
    c17_b_B[c17_i5] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c17_eML_blk_kernel_kernel12(
  const int8_T c17_Kp0[49], const real_T c17_change[49], const int8_T c17_Kn0[49],
  const real_T c17_x[7], const real_T c17_b_x[7], real_T c17_b_K[49], real_T
  c17_b_B[49])
{
  uint64_T c17_threadId;
  int32_T c17_ii;
  c17_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c17_ii = (int32_T)c17_threadId;
  if (c17_ii < 7) {
    c17_b_B[c17_ii + 7 * c17_ii] = 1.0;
    if (c17_b_x[c17_ii] == c17_x[c17_ii]) {
      c17_b_K[c17_ii + 7 * c17_ii] = (real_T)c17_Kp0[c17_ii + 7 * c17_ii] + 3.0 *
        c17_change[c17_ii];
      c17_b_B[c17_ii + 7 * c17_ii] = (real_T)c17_Kn0[c17_ii + 7 * c17_ii] + 0.5 *
        c17_change[c17_ii];
    } else {
      c17_b_K[c17_ii + 7 * c17_ii] = (real_T)c17_Kn0[c17_ii + 7 * c17_ii] - 3.0 *
        c17_change[c17_ii];
      c17_b_B[c17_ii + 7 * c17_ii] = (real_T)c17_Kn0[c17_ii + 7 * c17_ii] - 0.5 *
        c17_change[c17_ii];
    }
  }
}

static void init_dsm_address_info(SFc17_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc17_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c17_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c17_Tid = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c17_K = (real_T (*)[49])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_Th = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c17_qd = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c17_B = (real_T (*)[49])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c17_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1104077548U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(657016577U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2266812003U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1878717813U);
}

mxArray *sf_c17_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,1);
  mxSetCell(mxcell3p, 0, mxCreateString("coder.internal.lapack.LAPACKApi"));
  return(mxcell3p);
}

mxArray *sf_c17_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("late");
  mxArray *hiddenFallbackReason = mxCreateString("ir_function_calls");
  mxArray *incompatibleSymbol = mxCreateString("__gpu_kernel");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c17_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c17_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiA+wMTAwAakOYCYiQECWKF8RiBmhtIQcRa4uAIQl1QWpILEi4uSPVOAdF5iLpi"
    "fWFrhmZeWDzbfggFhPhsW8xmRzOeEikPAB3vK9Cs5gPQbIOlnIaBfAMhygoYLLHwGzv0iZLnfm2"
    "ruV3CgTD/E/ggC7ldEcT+En1kcn5hcklmWGp9saB5fnJOZkpmXHp+cn1dSlJ+Tk1pkhGQ+CAAA7"
    "zcftQ=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c17_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sQrNOskEAq2IwyZVoPCM93G";
}

static void sf_opaque_initialize_c17_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c17_sliding_controller2
    ((SFc17_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c17_sliding_controller2(void *chartInstanceVar)
{
  enable_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c17_sliding_controller2(void *chartInstanceVar)
{
  disable_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c17_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c17_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c17_sliding_controller2
    ((SFc17_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c17_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c17_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc17_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c17_sliding_controller2
      ((SFc17_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc17_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc17_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c17_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c17_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc17_sliding_controller2((SFc17_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c17_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c17_sliding_controller2
      ((SFc17_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc17_sliding_controller2
      ((SFc17_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c17_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [19] = {
    "eNrtV01u20YUpmQ3aIA2MJAA2RSI0027TNNFkk1ii5JSoVbslLbbZGOMh0/iQMMZZn4k6xAFuu4",
    "JuijQAxQouukdCnTTI+QIeUPSskKRVGzltygBihzO9755/zPyGr2+h9cV93PV8y7h42O8m152fZ",
    "SPG3N39n3d+yIf/4hCwsZ7RJFYe7WXIDF8B1pya5gUPTGQpTAmBqBAUMQmUpkqNs1iy5kYda2gj",
    "k9/HzEaBZG0PGyhLAl3BZ8iW2LNHvK0mQJqugChiZS0w6jLyXCmsTITPwI60jauM0GDCWzi1NJ9",
    "yw1LOHROgPaENgQ11me6BYYY8M1JpZnOUh2cAmWccEZEqbUR0QEk6GADB0mIv7vWoFFFGI2IMi2",
    "IyBj0DhulnFJAkZNpnDhmghipGOGdmPtOcFG3PY769GUIvMYhqFtLARklkglTHf+gi5Z2BDnm0I",
    "ZjO6xmC+CZdcE/ZDABVem3gS/HoMgQdkXloqlDOidptGZZsggzLIZDorYpxk9DWJm9mDk6IBgn2",
    "EeJKhikRvb0vmJjdG8lm417LjOXlYyNs2DrZbCUrTOGuijM2LpU+IRzXQnbl8kOjIGnrG1iSD0s",
    "Yy3Hac3CfYkOduldXQ1WMAx8DvOlCFlpuMYFQNp3HmFjeRlJrTYy9jF52zs7i9OLsJ4woAaEQlk",
    "XUIRpQJ+l7q1mC5l2sUcgamVS9crAWYYsQ3l6YEV7ItUIfVLTRM5McBGtBMZ6iLHESjjQWDR1MB",
    "fLZThKaAShazCMQx/LBrElPtGutW1j3Y2ZmbZBU8WSkqharDpsQx2XUNMEDsRIyInoKhkHeY/P3",
    "AuA2UaUYGLYwv6lpl1cvSwUbj+75Z3tZ5+8wn422wcLzy/neBolPN7cs7ju5ebL+LXCuk380mg0",
    "UrmtOblPC+usF+QcbgPvzb/++eMn/e+vd37/rfsz++XJsvUbC+s30ncnl6ydb/+/ko8/O220s8I",
    "ZL+Srw34zp9d6Cf/1Of6NfKwfq0e7etTZfna7N5k+PZR7fv/e1w9Tvj+b9fquFfQ9/b7pOj5mWZ",
    "qfivbC/GDixsRm27Xjvzun76Ul/ricf8+u5w9Wk/98qxjH9SXyG/jWKuTtu9P/2oX0//a16b+5t",
    "Zp8tv4PS/S/WcjXm+m55Ii4bgdH9Ks7R5qzEFvVEZXCKMk5qNsl/eWi9XpeOe8ty30oev4v9+bj",
    "/ir779oF5Zor7vdvS25V+857Dnnf8HX7gVfAb7zHdqx6PnzT+L+9853jbuTj+7O/an7EeFhyas+",
    "n8Qg+KJv9D+TpC59btxQ=",
    ""
  };

  static char newstr [1297] = "";
  newstr[0] = '\0';
  for (i = 0; i < 19; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c17_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c17_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3202073888U));
  ssSetChecksum1(S,(2900390800U));
  ssSetChecksum2(S,(1185921847U));
  ssSetChecksum3(S,(1504209306U));
}

static void mdlRTW_c17_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c17_sliding_controller2(SimStruct *S)
{
  SFc17_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc17_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc17_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc17_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc17_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c17_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c17_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c17_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c17_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c17_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c17_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c17_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c17_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c17_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c17_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c17_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c17_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c17_JITStateAnimation,
    chartInstance->c17_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c17_sliding_controller2(chartInstance);
}

void c17_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c17_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c17_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c17_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c17_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
