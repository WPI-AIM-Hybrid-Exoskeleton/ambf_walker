#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c21_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void enable_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void disable_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void c21_do_animation_call_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance, const mxArray *c21_st);
static void sf_gateway_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void c21_eML_blk_kernel(SFc21_sliding_controller2InstanceStruct
  *chartInstance, real_T c21_b_u[6], real_T c21_b_y[7]);
static void c21_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_y, const char_T *c21_identifier, real_T
  c21_c_y[7]);
static void c21_b_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId,
  real_T c21_b_y[7]);
static uint8_T c21_c_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_is_active_c21_sliding_controller2, const
  char_T *c21_identifier);
static uint8_T c21_d_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId);
static __global__ void c21_eML_blk_kernel_kernel1(real_T c21_b_y[7]);
static void init_dsm_address_info(SFc21_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc21_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c21_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c21_is_active_c21_sliding_controller2 = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c21_gpu_y, 56UL);
}

static void initialize_params_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c21_do_animation_call_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c21_b_y = NULL;
  const mxArray *c21_c_y = NULL;
  const mxArray *c21_d_y = NULL;
  const mxArray *c21_st;
  c21_st = NULL;
  c21_b_y = NULL;
  sf_mex_assign(&c21_b_y, sf_mex_createcellmatrix(2, 1), false);
  c21_c_y = NULL;
  sf_mex_assign(&c21_c_y, sf_mex_create("y", *chartInstance->c21_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c21_b_y, 0, c21_c_y);
  c21_d_y = NULL;
  sf_mex_assign(&c21_d_y, sf_mex_create("y",
    &chartInstance->c21_is_active_c21_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c21_b_y, 1, c21_d_y);
  sf_mex_assign(&c21_st, c21_b_y, false);
  return c21_st;
}

static void set_sim_state_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance, const mxArray *c21_st)
{
  const mxArray *c21_b_u;
  c21_b_u = sf_mex_dup(c21_st);
  c21_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c21_b_u, 0)),
                       "y", *chartInstance->c21_y);
  chartInstance->c21_is_active_c21_sliding_controller2 = c21_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c21_b_u, 1)),
     "is_active_c21_sliding_controller2");
  sf_mex_destroy(&c21_b_u);
  sf_mex_destroy(&c21_st);
}

static void sf_gateway_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  real_T c21_dv1[7];
  real_T c21_dv[6];
  int32_T c21_i;
  int32_T c21_i1;
  chartInstance->c21_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c21_i = 0; c21_i < 6; c21_i++) {
    c21_dv[c21_i] = (*chartInstance->c21_u)[c21_i];
  }

  c21_eML_blk_kernel(chartInstance, c21_dv, c21_dv1);
  for (c21_i1 = 0; c21_i1 < 7; c21_i1++) {
    (*chartInstance->c21_y)[c21_i1] = c21_dv1[c21_i1];
  }

  c21_do_animation_call_c21_sliding_controller2(chartInstance);
}

static void mdl_start_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c21_errCode;
  hipFree(*chartInstance->c21_gpu_y);
  c21_errCode = hipGetLastError();
  if (c21_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c21_errCode, (char_T *)hipGetErrorName
                       (c21_errCode), (char_T *)hipGetErrorString(c21_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c21_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c21_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c21_nameCaptureInfo = NULL;
  c21_nameCaptureInfo = NULL;
  sf_mex_assign(&c21_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c21_nameCaptureInfo;
}

static void c21_eML_blk_kernel(SFc21_sliding_controller2InstanceStruct
  *chartInstance, real_T c21_b_u[6], real_T c21_b_y[7])
{
  const mxArray *c21_c_y = NULL;
  c21_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c21_gpu_y);
  hipMemcpy(c21_b_y, *chartInstance->c21_gpu_y, 56UL, hipMemcpyDeviceToHost);
  c21_b_y[0] = c21_b_u[3];
  c21_b_y[1] = 0.0;
  sf_mex_printf("%s =\\n", "reordered");
  c21_c_y = NULL;
  sf_mex_assign(&c21_c_y, sf_mex_create("y", c21_b_y, 0, 0U, 1U, 0U, 2, 1, 7),
                false);
  sf_mex_call(chartInstance->c21_fEmlrtCtx, NULL, "disp", 0U, 1U, 14, c21_c_y);
  c21_b_y[2] = c21_b_u[0];
  c21_b_y[3] = c21_b_u[1];
  c21_b_y[4] = c21_b_u[4];
  c21_b_y[5] = c21_b_u[5];
  c21_b_y[6] = c21_b_u[2];
}

static void c21_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_y, const char_T *c21_identifier, real_T
  c21_c_y[7])
{
  emlrtMsgIdentifier c21_thisId;
  c21_thisId.fIdentifier = const_cast<const char_T *>(c21_identifier);
  c21_thisId.fParent = NULL;
  c21_thisId.bParentIsCell = false;
  c21_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c21_b_y), &c21_thisId,
    c21_c_y);
  sf_mex_destroy(&c21_b_y);
}

static void c21_b_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId,
  real_T c21_b_y[7])
{
  real_T c21_dv[7];
  int32_T c21_i;
  sf_mex_import(c21_parentId, sf_mex_dup(c21_b_u), c21_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c21_i = 0; c21_i < 7; c21_i++) {
    c21_b_y[c21_i] = c21_dv[c21_i];
  }

  sf_mex_destroy(&c21_b_u);
}

static uint8_T c21_c_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_is_active_c21_sliding_controller2, const
  char_T *c21_identifier)
{
  emlrtMsgIdentifier c21_thisId;
  uint8_T c21_b_y;
  c21_thisId.fIdentifier = const_cast<const char_T *>(c21_identifier);
  c21_thisId.fParent = NULL;
  c21_thisId.bParentIsCell = false;
  c21_b_y = c21_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c21_b_is_active_c21_sliding_controller2), &c21_thisId);
  sf_mex_destroy(&c21_b_is_active_c21_sliding_controller2);
  return c21_b_y;
}

static uint8_T c21_d_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId)
{
  uint8_T c21_b_y;
  uint8_T c21_c_u;
  sf_mex_import(c21_parentId, sf_mex_dup(c21_b_u), &c21_c_u, 1, 3, 0U, 0, 0U, 0);
  c21_b_y = c21_c_u;
  sf_mex_destroy(&c21_b_u);
  return c21_b_y;
}

static __global__ __launch_bounds__(32, 1) void c21_eML_blk_kernel_kernel1
  (real_T c21_b_y[7])
{
  uint64_T c21_threadId;
  int32_T c21_i;
  c21_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c21_i = (int32_T)c21_threadId;
  if (c21_i < 7) {
    c21_b_y[c21_i] = 0.0;
  }
}

static void init_dsm_address_info(SFc21_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc21_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c21_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c21_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c21_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c21_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(1104270009U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4002144469U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1451542329U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(1457894201U);
}

mxArray *sf_c21_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c21_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c21_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c21_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8clGhvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABa+hvV"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c21_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "scU9zZ5H1kMhWDkDI28BpgD";
}

static void sf_opaque_initialize_c21_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c21_sliding_controller2
    ((SFc21_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c21_sliding_controller2(void *chartInstanceVar)
{
  enable_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c21_sliding_controller2(void *chartInstanceVar)
{
  disable_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c21_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c21_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c21_sliding_controller2
    ((SFc21_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c21_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c21_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc21_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c21_sliding_controller2
      ((SFc21_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc21_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc21_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c21_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c21_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c21_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c21_sliding_controller2
      ((SFc21_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc21_sliding_controller2
      ((SFc21_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c21_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV09v00gUd6OCFglQtWLFBQk4wREqsYLLbkmciEgNrXALaC/VdPwSjzKeMfMnbfgKiAviwoH",
    "PAJ8Bie/Ahe/ADYkTb2wnDY7tUCIqdrWWXGfs33vz3vu9P1Nvpdvz8DqP98bvnncan7/h3fCy61",
    "S+Xpm5s/er3rV8/RyFhI23iSKx9movQWJ4AFpya5gUXdGXpTAm+qBAUMQmUpkqbZrFljMx7FhBn",
    "T79KGI0CiJpedhEWRJuCT5GbYk126jHZwqo6QCEJlLSDqIOJ4OpxcoctCKgQ23jOhc0mMAmzizd",
    "s9ywhEP7EGhXaEPQYn1kW2CIgZY5rHTTeaqDCVDGCWdElHobER1AggE2sJuE+HfLGnSqCKMRUaY",
    "JERmB3mTDVKcUUNTJNH7YZ4IYqRjh7Zi3nOC8bdsc7enJEHhNQNC2pgIyTCQTppr/oIOetgXZ5+",
    "DDvh1UawvgiXXkP2RwAKoybv2WHIEiA9gSlZumAWkfpmxNs2QeZlgMD4m6S5E/DWFl9mLm6IAgT",
    "7CDElUwSJ3s6h3FRhjeSm027rrMXFQyNs7I1otgqbb2COpYmGrrUNEinOtK2I5MNmEEPNXqE0Pq",
    "YZnWcpzWLNyRGGCX3tXVYAVD4nNYS4qQldI1KgDSvnMfG8u3SGq1kXELk9ff3Jz/PA/rCgOqTyi",
    "UdQFFmAaMWRream0h0457BKJVJjWvDJxlyCKUp/tW+AdSDTEmNU3kyAXHaCUw1gPkEithV2PR1M",
    "Ecl4twlNAIQtdgGIcelg1iS2KiXWu7i3U3Ymbsg6aKJSWsWqw6bENtl1DjBHbFUMgD0VEyDvIen",
    "4UXALONKMHEoIn9S407uHsZFW6e3fCO5tnZ75hnE7ni8/qMnpUSPd7Ms7jvmUb9vg38tTKZvzNy",
    "5wr7rBbkHG4N7z++PHv5+eXrN+9f/Hnr46v375bZ/23jePP/fL6+NGm008IZzeWrw96bsWu1RP/",
    "FGf1r+VrT3TtP/7l17+awFz3yh353/XYzGfgZPwvsbRTsnby/4jo+Zlman4p2w/xg4tbEZuPa6b",
    "89Y+/pBfE4k7/Prk9/Lyd/YaPI4+oC+TX8NS7k7Y/vf2VjOfls/8cL7L9a4PtqOtf3iOsWsEfXb",
    "+5pzkIs9T0qhVGSc1DrJfX5o/l+XDnvhOX+LXb+L/fzef+e+XXSct4Jyy3r33Hn+K+Gr5sHXgG/",
    "9gv7sez56mfjP3jHOwddztd/Tf/VaUWMhyWn3vwzHmH7ZV//A3n6FVYitA0=",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c21_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c21_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2492201493U));
  ssSetChecksum1(S,(2946340087U));
  ssSetChecksum2(S,(892768965U));
  ssSetChecksum3(S,(3267729885U));
}

static void mdlRTW_c21_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c21_sliding_controller2(SimStruct *S)
{
  SFc21_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc21_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc21_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc21_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc21_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c21_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c21_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c21_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c21_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c21_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c21_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c21_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c21_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c21_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c21_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c21_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c21_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c21_JITStateAnimation,
    chartInstance->c21_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c21_sliding_controller2(chartInstance);
}

void c21_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c21_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c21_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c21_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c21_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
