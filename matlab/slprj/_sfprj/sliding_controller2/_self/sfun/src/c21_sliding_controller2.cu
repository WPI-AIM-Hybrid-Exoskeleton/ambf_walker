/* Include files */

#include "sliding_controller2_sfun.h"
#include "c21_sliding_controller2.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void enable_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void disable_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void c21_do_animation_call_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance, const mxArray *c21_st);
static void sf_gateway_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance);
static void c21_eML_blk_kernel(SFc21_sliding_controller2InstanceStruct
  *chartInstance, real_T c21_b_u[7], real_T c21_b_y[7]);
static void c21_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_y, const char_T *c21_identifier, real_T
  c21_c_y[7]);
static void c21_b_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId,
  real_T c21_b_y[7]);
static uint8_T c21_c_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_is_active_c21_sliding_controller2, const
  char_T *c21_identifier);
static uint8_T c21_d_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId);
static void init_dsm_address_info(SFc21_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc21_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c21_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c21_is_active_c21_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c21_do_animation_call_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c21_b_y = NULL;
  const mxArray *c21_c_y = NULL;
  const mxArray *c21_d_y = NULL;
  const mxArray *c21_st;
  c21_st = NULL;
  c21_b_y = NULL;
  sf_mex_assign(&c21_b_y, sf_mex_createcellmatrix(2, 1), false);
  c21_c_y = NULL;
  sf_mex_assign(&c21_c_y, sf_mex_create("y", *chartInstance->c21_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c21_b_y, 0, c21_c_y);
  c21_d_y = NULL;
  sf_mex_assign(&c21_d_y, sf_mex_create("y",
    &chartInstance->c21_is_active_c21_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c21_b_y, 1, c21_d_y);
  sf_mex_assign(&c21_st, c21_b_y, false);
  return c21_st;
}

static void set_sim_state_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance, const mxArray *c21_st)
{
  const mxArray *c21_b_u;
  c21_b_u = sf_mex_dup(c21_st);
  c21_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c21_b_u, 0)),
                       "y", *chartInstance->c21_y);
  chartInstance->c21_is_active_c21_sliding_controller2 = c21_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c21_b_u, 1)),
     "is_active_c21_sliding_controller2");
  sf_mex_destroy(&c21_b_u);
  sf_mex_destroy(&c21_st);
}

static void sf_gateway_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c21_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c21_eML_blk_kernel(chartInstance, *chartInstance->c21_u, *chartInstance->c21_y);
  c21_do_animation_call_c21_sliding_controller2(chartInstance);
}

static void mdl_start_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c21_errCode;
  c21_errCode = hipGetLastError();
  if (c21_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c21_errCode, (char_T *)hipGetErrorName
                       (c21_errCode), (char_T *)hipGetErrorString(c21_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c21_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc21_sliding_controller2
  (SFc21_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c21_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c21_nameCaptureInfo = NULL;
  c21_nameCaptureInfo = NULL;
  sf_mex_assign(&c21_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c21_nameCaptureInfo;
}

static void c21_eML_blk_kernel(SFc21_sliding_controller2InstanceStruct
  *chartInstance, real_T c21_b_u[7], real_T c21_b_y[7])
{
  c21_b_y[0] = c21_b_u[3];
  c21_b_y[1] = c21_b_u[6];
  c21_b_y[2] = c21_b_u[0];
  c21_b_y[3] = c21_b_u[1];
  c21_b_y[4] = c21_b_u[4];
  c21_b_y[5] = c21_b_u[5];
  c21_b_y[6] = c21_b_u[2];
}

static void c21_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_y, const char_T *c21_identifier, real_T
  c21_c_y[7])
{
  emlrtMsgIdentifier c21_thisId;
  c21_thisId.fIdentifier = const_cast<const char_T *>(c21_identifier);
  c21_thisId.fParent = NULL;
  c21_thisId.bParentIsCell = false;
  c21_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c21_b_y), &c21_thisId,
    c21_c_y);
  sf_mex_destroy(&c21_b_y);
}

static void c21_b_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId,
  real_T c21_b_y[7])
{
  real_T c21_dv[7];
  int32_T c21_i;
  sf_mex_import(c21_parentId, sf_mex_dup(c21_b_u), c21_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c21_i = 0; c21_i < 7; c21_i++) {
    c21_b_y[c21_i] = c21_dv[c21_i];
  }

  sf_mex_destroy(&c21_b_u);
}

static uint8_T c21_c_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_is_active_c21_sliding_controller2, const
  char_T *c21_identifier)
{
  emlrtMsgIdentifier c21_thisId;
  uint8_T c21_b_y;
  c21_thisId.fIdentifier = const_cast<const char_T *>(c21_identifier);
  c21_thisId.fParent = NULL;
  c21_thisId.bParentIsCell = false;
  c21_b_y = c21_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c21_b_is_active_c21_sliding_controller2), &c21_thisId);
  sf_mex_destroy(&c21_b_is_active_c21_sliding_controller2);
  return c21_b_y;
}

static uint8_T c21_d_emlrt_marshallIn(SFc21_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c21_b_u, const emlrtMsgIdentifier *c21_parentId)
{
  uint8_T c21_b_y;
  uint8_T c21_c_u;
  sf_mex_import(c21_parentId, sf_mex_dup(c21_b_u), &c21_c_u, 1, 3, 0U, 0, 0U, 0);
  c21_b_y = c21_c_u;
  sf_mex_destroy(&c21_b_u);
  return c21_b_y;
}

static void init_dsm_address_info(SFc21_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc21_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c21_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c21_u = (real_T (*)[7])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c21_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c21_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3007509169U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(1926089682U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2239166282U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(3770191902U);
}

mxArray *sf_c21_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c21_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c21_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c21_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8clGhvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABa+hvV"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c21_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sdegTANPF1GH9p3UVy5QBRH";
}

static void sf_opaque_initialize_c21_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c21_sliding_controller2
    ((SFc21_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c21_sliding_controller2(void *chartInstanceVar)
{
  enable_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c21_sliding_controller2(void *chartInstanceVar)
{
  disable_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c21_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c21_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c21_sliding_controller2
    ((SFc21_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c21_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c21_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc21_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c21_sliding_controller2
      ((SFc21_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc21_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc21_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c21_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c21_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc21_sliding_controller2((SFc21_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c21_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c21_sliding_controller2
      ((SFc21_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc21_sliding_controller2
      ((SFc21_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c21_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV0tv20YQpgU3aIDUMIoAvRRNfCjaW5sUBZJLYusVC5AfDWWnl8JYL0fiQstdZh+ydc4x6K/",
    "oH+il/yW/IugluQXILEnJKkVSkQUbaRACNLXkN7Mz881j7a119jy8NvDe/trzbuDzS7xrXnp9ka",
    "3XZu70/br3Q7b+E4WEjQ6JIpH2Ki9BIngKWnJrmBQd0ZeFMCb6oEBQxMZSmTJtmkWWMzFsW0GdP",
    "v0sZDT0Q2l5UEdZEhwIPkZtsTWHqKfJFFDTBghMqKQdhG1OBlOLlTlrhECH2kZVLmgwvo2dWXrP",
    "csNiDq1zoB2hDUGL9YVtviEGGua81E3nqfYnQBnFnBFR6G1ItA8xBtjAURzg3wNr0Kk8jIZEmTq",
    "EZAS6y4aJTikgr5Np/HDKBDFSMcJbEW84wXnbDjnasycD4BUBQdvqCsgwlkyYcv79NnraEuSUQx",
    "NO7aBcmw/PrSP/mMEZqNK49RtyBIoM4ECUbpoEpHWesDXNknmYYREcE7VDkT8NQWn2YuZonyBP0",
    "EOJMhgkTnZ0T7ERhrdUm406LjMXlYyNUrL1IliirTWCKham2tpUNAjnuhTWk3EXRsATrU1iSDUs",
    "1VqM05oFPYkBduldXg1WMCQ+gzWkCFghXaMcIOk7+9hY/oukVhsZNTB5m93u/Od5WEcYUH1Coag",
    "LKMI0YMyS8JZrC5h23CMQrTKJeUXgNEMWoTzdt6J5JtUQY1LRRC5ccIyWAiM9QC6xEo40Fk0VzH",
    "G5CEcJDSFwDYZx2MOyQWxBTLRrbTtYdyNmxk3QVLG4gFWLVYdtqOUSahzDkRgKeSbaSkZ+1uPT8",
    "AJgthElmBjUsX+pcRt3L6LCzbOfvYt5dusD5tlELv/8cUbPWoEeb+aZ3/dmrXrfGv5am8zfGbmv",
    "cvus5+QcbhPvd7ef/fPypxdvNv4if7x++/13q+z/d225+b+Rrb+dNNpp4Yzm8tVhd2fsWi/Q/82",
    "M/s1srQMY9Hb2D9v3nuw+jH85Oh7/+lv96W7KzwJ7azl7J+/vuo6PWZbkp6KdIDuYuDWx6bh2+h",
    "/M2HtjQTxuZu/T69/Hq8nf3s7zuL5AfhN/jXN5e/n9726vJp/u//sC+7dyfG8lc/2EuG4BJ/T+v",
    "RPNWYClfkKlMEpyDup+QX1eNt+XlfOuWe7/Yudnuavn/UPm13XLedcst6p/y87xjw1fNQ+8HH7z",
    "I/Zj1fPVVeNfecudg+5k60fTf3UaIeNBwak3+4xH2H7R108gT98DXE+yDA==",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c21_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c21_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3042383101U));
  ssSetChecksum1(S,(4119015307U));
  ssSetChecksum2(S,(1566679822U));
  ssSetChecksum3(S,(505804526U));
}

static void mdlRTW_c21_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c21_sliding_controller2(SimStruct *S)
{
  SFc21_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc21_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc21_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc21_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc21_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c21_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c21_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c21_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c21_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c21_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c21_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c21_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c21_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c21_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c21_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c21_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c21_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c21_JITStateAnimation,
    chartInstance->c21_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c21_sliding_controller2(chartInstance);
}

void c21_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c21_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c21_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c21_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c21_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
