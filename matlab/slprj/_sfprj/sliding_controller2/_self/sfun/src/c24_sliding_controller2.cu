/* Include files */

#include "sliding_controller2_sfun.h"
#include "c24_sliding_controller2.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void enable_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void disable_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void c24_do_animation_call_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance, const mxArray *c24_st);
static void sf_gateway_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance);
static void c24_eML_blk_kernel(SFc24_sliding_controller2InstanceStruct
  *chartInstance, real_T c24_b_u[6], real_T c24_b_y[7]);
static void c24_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_y, const char_T *c24_identifier, real_T
  c24_c_y[7]);
static void c24_b_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId,
  real_T c24_b_y[7]);
static uint8_T c24_c_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_is_active_c24_sliding_controller2, const
  char_T *c24_identifier);
static uint8_T c24_d_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId);
static void init_dsm_address_info(SFc24_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc24_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c24_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c24_is_active_c24_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c24_do_animation_call_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c24_b_y = NULL;
  const mxArray *c24_c_y = NULL;
  const mxArray *c24_d_y = NULL;
  const mxArray *c24_st;
  c24_st = NULL;
  c24_b_y = NULL;
  sf_mex_assign(&c24_b_y, sf_mex_createcellmatrix(2, 1), false);
  c24_c_y = NULL;
  sf_mex_assign(&c24_c_y, sf_mex_create("y", *chartInstance->c24_y, 0, 0U, 1U,
    0U, 2, 1, 7), false);
  sf_mex_setcell(c24_b_y, 0, c24_c_y);
  c24_d_y = NULL;
  sf_mex_assign(&c24_d_y, sf_mex_create("y",
    &chartInstance->c24_is_active_c24_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c24_b_y, 1, c24_d_y);
  sf_mex_assign(&c24_st, c24_b_y, false);
  return c24_st;
}

static void set_sim_state_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance, const mxArray *c24_st)
{
  const mxArray *c24_b_u;
  c24_b_u = sf_mex_dup(c24_st);
  c24_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c24_b_u, 0)),
                       "y", *chartInstance->c24_y);
  chartInstance->c24_is_active_c24_sliding_controller2 = c24_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c24_b_u, 1)),
     "is_active_c24_sliding_controller2");
  sf_mex_destroy(&c24_b_u);
  sf_mex_destroy(&c24_st);
}

static void sf_gateway_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c24_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c24_eML_blk_kernel(chartInstance, *chartInstance->c24_u, *chartInstance->c24_y);
  c24_do_animation_call_c24_sliding_controller2(chartInstance);
}

static void mdl_start_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c24_errCode;
  c24_errCode = hipGetLastError();
  if (c24_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c24_errCode, (char_T *)hipGetErrorName
                       (c24_errCode), (char_T *)hipGetErrorString(c24_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c24_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc24_sliding_controller2
  (SFc24_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c24_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c24_nameCaptureInfo = NULL;
  c24_nameCaptureInfo = NULL;
  sf_mex_assign(&c24_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c24_nameCaptureInfo;
}

static void c24_eML_blk_kernel(SFc24_sliding_controller2InstanceStruct
  *chartInstance, real_T c24_b_u[6], real_T c24_b_y[7])
{
  c24_b_y[0] = c24_b_u[3];
  c24_b_y[1] = c24_b_u[0];
  c24_b_y[2] = c24_b_u[1];
  c24_b_y[3] = c24_b_u[2];
  c24_b_y[4] = c24_b_u[4];
  c24_b_y[5] = c24_b_u[5];
  c24_b_y[6] = 0.0;
}

static void c24_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_y, const char_T *c24_identifier, real_T
  c24_c_y[7])
{
  emlrtMsgIdentifier c24_thisId;
  c24_thisId.fIdentifier = const_cast<const char_T *>(c24_identifier);
  c24_thisId.fParent = NULL;
  c24_thisId.bParentIsCell = false;
  c24_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c24_b_y), &c24_thisId,
    c24_c_y);
  sf_mex_destroy(&c24_b_y);
}

static void c24_b_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId,
  real_T c24_b_y[7])
{
  real_T c24_dv[7];
  int32_T c24_i;
  sf_mex_import(c24_parentId, sf_mex_dup(c24_b_u), c24_dv, 1, 0, 0U, 1, 0U, 2, 1,
                7);
  for (c24_i = 0; c24_i < 7; c24_i++) {
    c24_b_y[c24_i] = c24_dv[c24_i];
  }

  sf_mex_destroy(&c24_b_u);
}

static uint8_T c24_c_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_is_active_c24_sliding_controller2, const
  char_T *c24_identifier)
{
  emlrtMsgIdentifier c24_thisId;
  uint8_T c24_b_y;
  c24_thisId.fIdentifier = const_cast<const char_T *>(c24_identifier);
  c24_thisId.fParent = NULL;
  c24_thisId.bParentIsCell = false;
  c24_b_y = c24_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c24_b_is_active_c24_sliding_controller2), &c24_thisId);
  sf_mex_destroy(&c24_b_is_active_c24_sliding_controller2);
  return c24_b_y;
}

static uint8_T c24_d_emlrt_marshallIn(SFc24_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c24_b_u, const emlrtMsgIdentifier *c24_parentId)
{
  uint8_T c24_b_y;
  uint8_T c24_c_u;
  sf_mex_import(c24_parentId, sf_mex_dup(c24_b_u), &c24_c_u, 1, 3, 0U, 0, 0U, 0);
  c24_b_y = c24_c_u;
  sf_mex_destroy(&c24_b_u);
  return c24_b_y;
}

static void init_dsm_address_info(SFc24_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc24_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c24_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c24_u = (real_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c24_y = (real_T (*)[7])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c24_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3080704608U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2208426765U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2131008065U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2283259860U);
}

mxArray *sf_c24_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c24_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c24_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c24_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8clGJvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABbZhvY"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c24_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sApjR2e37QLsrAPnkhirTIF";
}

static void sf_opaque_initialize_c24_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c24_sliding_controller2
    ((SFc24_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c24_sliding_controller2(void *chartInstanceVar)
{
  enable_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c24_sliding_controller2(void *chartInstanceVar)
{
  disable_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c24_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c24_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c24_sliding_controller2
    ((SFc24_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c24_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c24_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc24_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c24_sliding_controller2
      ((SFc24_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc24_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc24_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c24_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c24_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc24_sliding_controller2((SFc24_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c24_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c24_sliding_controller2
      ((SFc24_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc24_sliding_controller2
      ((SFc24_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c24_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u1EgQdkYhu5EWFKGV9rLSwmkRl0UBCS5AkvnRjjQh2XjCj4QUddo1495pd3v7Z5J5gD1",
    "y57SvsHkHXoJX4IDEgQeg2vZMBo/tSRgRAcKS42n7q+qq+uqn4y21tz28ruC9cdXzVvD5I941L7",
    "0uZeulqTt9v+z9nq1foJCw0S5RJNJe5SVIBHugJbeGSdEWPVkIY6IHCgRFbCyVKdOmWWQ5E4OWF",
    "dTp009CRkM/lJYHWyhLgh3BR6gttmYX9TSYAmpaAIEJlbT9sMVJf2KxMkf1EOhA26jKBQ3Gt7Ez",
    "S29bbljMoXkMtC20IWixPrXNN8RA3RyXuuk81f4YKKOYMyIKvQ2J9iHGABvYjwP8u2MNOpWH0ZA",
    "oswUhGYLusEGiUwrI62QaPxwyQYxUjPBmxOtOcNa2XY72bMsAeEVA0LYtBWQQSyZMOf9+Cz1tCn",
    "LIoQGHtl+uzYd/rCP/MYMjUKVx69XlEBTpw44o3TQJSPM4YWuSJbMwwyJ4TNQmRf40BKXZi5mjf",
    "YI8QRclymCQONnWXcWGGN5SbTZqu8ycVzI2SsnW82CJtuYQqliYaGtRUSec61JYV8YdGAJPtDaI",
    "IdWwVGsxTmsWdCUG2KV3eTVYwZD4DFaXImCFdA1zgKTvPMLG8jGSWm1kVMfkbXQ6s59nYW1hQPU",
    "IhaIuoAjTgDFLwluuLWDacY9AtMok5hWB0wyZh/J0z4rGkVQDjElFEzl1wTFaCox0H7nEStjXWD",
    "RVMMflPBwlNITANRjGYRvLBrEFMdGutW1i3Q2ZGTVAU8XiAlYtVh22oaZLqFEM+2Ig5JFoKRn5W",
    "Y9PwwuA2UaUYKK/hf1LjVq4exEVbp7d8k7n2U9nmGdjufzzxpSepQI93tQzv+9qrXrfGv5aGs/f",
    "KbnLuX2Wc3IOt4b3fzfvv3r+5uTuDy+fvX3/7x//L7L/Se188/9Ktv513GgnhTOcyVeH/XPKruU",
    "C/b9M6V/L1noz/ntvHW7f/auj1eauGIRMddutlJ859tZy9o7fX3MdH7MsyU9F20F2MHFrYtNx7f",
    "Tfm7J3ZU48VrP36fXu4WLyP2/keVyeI7+Gv0a5vP30/a9tLCaf7v90jv3Xc3xfT+b6AXHdAg7o+",
    "p0DzVmApX5ApTBKcg5qvaA+PzXfzyvnXbDc12Lnd7nPz/tZ5tdFy3kXLLeof+ed418avmoeeDn8",
    "2hfsx6Lnq8+Nf+2d7xz0W7Z+MPlXpx4yHhScerPPeITtFX39BvL0A1G+sw8=",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c24_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c24_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3275565726U));
  ssSetChecksum1(S,(934340956U));
  ssSetChecksum2(S,(3982071815U));
  ssSetChecksum3(S,(2939127284U));
}

static void mdlRTW_c24_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c24_sliding_controller2(SimStruct *S)
{
  SFc24_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc24_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc24_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc24_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc24_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c24_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c24_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c24_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c24_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c24_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c24_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c24_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c24_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c24_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c24_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c24_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c24_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c24_JITStateAnimation,
    chartInstance->c24_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c24_sliding_controller2(chartInstance);
}

void c24_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c24_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c24_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c24_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c24_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
