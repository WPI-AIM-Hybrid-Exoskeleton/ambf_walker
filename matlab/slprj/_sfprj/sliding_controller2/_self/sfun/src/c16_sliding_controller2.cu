#include "hip/hip_runtime.h"
/* Include files */

#include "sliding_controller2_sfun.h"
#include "c16_sliding_controller2.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void enable_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void disable_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void c16_do_animation_call_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance, const mxArray *c16_st);
static void sf_gateway_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance);
static void c16_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_b_msg, const char_T *c16_identifier,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_y);
static void c16_b_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_y);
static void c16_c_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  real_T c16_y[7]);
static c16_SL_Bus_ROSVariableLengthArrayInfo c16_d_emlrt_marshallIn
  (SFc16_sliding_controller2InstanceStruct *chartInstance, const mxArray *c16_u,
   const emlrtMsgIdentifier *c16_parentId);
static uint32_T c16_e_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId);
static void c16_f_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  uint8_T c16_y[4]);
static uint8_T c16_g_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_b_is_active_c16_sliding_controller2, const
  char_T *c16_identifier);
static uint8_T c16_h_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId);
static void c16_eML_blk_kernel(SFc16_sliding_controller2InstanceStruct
  *chartInstance, c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *
  c16_b_blankMsg);
static __global__ void c16_sf_gateway_c16_sliding_controller2_kernel1
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller2_kernel2
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller2_kernel3
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller2_kernel4
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r);
static __global__ void c16_sf_gateway_c16_sliding_controller2_kernel5
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r);
static __global__ void c16_sf_gateway_c16_sliding_controller2_kernel6(const
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller2_kernel7(const
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg);
static __global__ void c16_sf_gateway_c16_sliding_controller2_kernel8(const
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg);
static void init_dsm_address_info(SFc16_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc16_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c16_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c16_is_active_c16_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c16_do_animation_call_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
  static const char_T *c16_sv[10] = { "Q", "Q_SL_Info", "Qd", "Qd_SL_Info",
    "Qdd", "Qdd_SL_Info", "Controller", "Controller_SL_Info", "Other",
    "Other_SL_Info" };

  static const char_T *c16_sv1[2] = { "CurrentLength", "ReceivedLength" };

  const mxArray *c16_b_y = NULL;
  const mxArray *c16_c_y = NULL;
  const mxArray *c16_d_y = NULL;
  const mxArray *c16_e_y = NULL;
  const mxArray *c16_f_y = NULL;
  const mxArray *c16_g_y = NULL;
  const mxArray *c16_h_y = NULL;
  const mxArray *c16_i_y = NULL;
  const mxArray *c16_j_y = NULL;
  const mxArray *c16_k_y = NULL;
  const mxArray *c16_l_y = NULL;
  const mxArray *c16_m_y = NULL;
  const mxArray *c16_n_y = NULL;
  const mxArray *c16_o_y = NULL;
  const mxArray *c16_p_y = NULL;
  const mxArray *c16_q_y = NULL;
  const mxArray *c16_r_y = NULL;
  const mxArray *c16_s_y = NULL;
  const mxArray *c16_st;
  const mxArray *c16_t_y = NULL;
  const mxArray *c16_u_y = NULL;
  const mxArray *c16_v_y = NULL;
  const mxArray *c16_w_y = NULL;
  const mxArray *c16_y = NULL;
  real_T c16_u[7];
  int32_T c16_i;
  int32_T c16_i1;
  int32_T c16_i2;
  int32_T c16_i3;
  int32_T c16_i4;
  uint32_T c16_b_u;
  uint32_T c16_c_u;
  uint32_T c16_d_u;
  uint32_T c16_e_u;
  uint32_T c16_f_u;
  uint32_T c16_g_u;
  uint32_T c16_i_u;
  uint32_T c16_j_u;
  uint32_T c16_k_u;
  uint32_T c16_l_u;
  uint32_T c16_u_CurrentLength;
  uint32_T c16_u_ReceivedLength;
  uint8_T c16_h_u[4];
  c16_st = NULL;
  c16_y = NULL;
  sf_mex_assign(&c16_y, sf_mex_createcellmatrix(2, 1), false);
  c16_b_y = NULL;
  sf_mex_assign(&c16_b_y, sf_mex_createstruct("structure", 10, c16_sv, 2, 1, 1),
                false);
  for (c16_i = 0; c16_i < 7; c16_i++) {
    c16_u[c16_i] = ((real_T *)&((char_T *)chartInstance->c16_msg)[0])[c16_i];
  }

  c16_c_y = NULL;
  sf_mex_assign(&c16_c_y, sf_mex_create("y", c16_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Q", c16_c_y, 0);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [56])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [56])[4];
  c16_d_y = NULL;
  sf_mex_assign(&c16_d_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_b_u = c16_u_CurrentLength;
  c16_e_y = NULL;
  sf_mex_assign(&c16_e_y, sf_mex_create("y", &c16_b_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_d_y, 0, "CurrentLength", c16_e_y, 0);
  c16_c_u = c16_u_ReceivedLength;
  c16_f_y = NULL;
  sf_mex_assign(&c16_f_y, sf_mex_create("y", &c16_c_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_d_y, 0, "ReceivedLength", c16_f_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Q_SL_Info", c16_d_y, 1);
  for (c16_i1 = 0; c16_i1 < 7; c16_i1++) {
    c16_u[c16_i1] = ((real_T *)&((char_T *)chartInstance->c16_msg)[64])[c16_i1];
  }

  c16_g_y = NULL;
  sf_mex_assign(&c16_g_y, sf_mex_create("y", c16_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qd", c16_g_y, 2);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [120])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [120])[4];
  c16_h_y = NULL;
  sf_mex_assign(&c16_h_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_d_u = c16_u_CurrentLength;
  c16_i_y = NULL;
  sf_mex_assign(&c16_i_y, sf_mex_create("y", &c16_d_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_h_y, 0, "CurrentLength", c16_i_y, 0);
  c16_e_u = c16_u_ReceivedLength;
  c16_j_y = NULL;
  sf_mex_assign(&c16_j_y, sf_mex_create("y", &c16_e_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_h_y, 0, "ReceivedLength", c16_j_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qd_SL_Info", c16_h_y, 3);
  for (c16_i2 = 0; c16_i2 < 7; c16_i2++) {
    c16_u[c16_i2] = ((real_T *)&((char_T *)chartInstance->c16_msg)[128])[c16_i2];
  }

  c16_k_y = NULL;
  sf_mex_assign(&c16_k_y, sf_mex_create("y", c16_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qdd", c16_k_y, 4);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [184])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [184])[4];
  c16_l_y = NULL;
  sf_mex_assign(&c16_l_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_f_u = c16_u_CurrentLength;
  c16_m_y = NULL;
  sf_mex_assign(&c16_m_y, sf_mex_create("y", &c16_f_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_l_y, 0, "CurrentLength", c16_m_y, 0);
  c16_g_u = c16_u_ReceivedLength;
  c16_n_y = NULL;
  sf_mex_assign(&c16_n_y, sf_mex_create("y", &c16_g_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_l_y, 0, "ReceivedLength", c16_n_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Qdd_SL_Info", c16_l_y, 5);
  for (c16_i3 = 0; c16_i3 < 4; c16_i3++) {
    c16_h_u[c16_i3] = ((uint8_T *)&((char_T *)chartInstance->c16_msg)[192])
      [c16_i3];
  }

  c16_o_y = NULL;
  sf_mex_assign(&c16_o_y, sf_mex_create("y", c16_h_u, 3, 0U, 1U, 0U, 1, 4),
                false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Controller", c16_o_y, 6);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[4];
  c16_p_y = NULL;
  sf_mex_assign(&c16_p_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_i_u = c16_u_CurrentLength;
  c16_q_y = NULL;
  sf_mex_assign(&c16_q_y, sf_mex_create("y", &c16_i_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_p_y, 0, "CurrentLength", c16_q_y, 0);
  c16_j_u = c16_u_ReceivedLength;
  c16_r_y = NULL;
  sf_mex_assign(&c16_r_y, sf_mex_create("y", &c16_j_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_p_y, 0, "ReceivedLength", c16_r_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Controller_SL_Info", c16_p_y, 7);
  for (c16_i4 = 0; c16_i4 < 7; c16_i4++) {
    c16_u[c16_i4] = ((real_T *)&((char_T *)chartInstance->c16_msg)[208])[c16_i4];
  }

  c16_s_y = NULL;
  sf_mex_assign(&c16_s_y, sf_mex_create("y", c16_u, 0, 0U, 1U, 0U, 1, 7), false);
  sf_mex_setfieldbynum(c16_b_y, 0, "Other", c16_s_y, 8);
  c16_u_CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [264])[0];
  c16_u_ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [264])[4];
  c16_t_y = NULL;
  sf_mex_assign(&c16_t_y, sf_mex_createstruct("structure", 2, c16_sv1, 2, 1, 1),
                false);
  c16_k_u = c16_u_CurrentLength;
  c16_u_y = NULL;
  sf_mex_assign(&c16_u_y, sf_mex_create("y", &c16_k_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_t_y, 0, "CurrentLength", c16_u_y, 0);
  c16_l_u = c16_u_ReceivedLength;
  c16_v_y = NULL;
  sf_mex_assign(&c16_v_y, sf_mex_create("y", &c16_l_u, 7, 0U, 0U, 0U, 0), false);
  sf_mex_setfieldbynum(c16_t_y, 0, "ReceivedLength", c16_v_y, 1);
  sf_mex_setfieldbynum(c16_b_y, 0, "Other_SL_Info", c16_t_y, 9);
  sf_mex_setcell(c16_y, 0, c16_b_y);
  c16_w_y = NULL;
  sf_mex_assign(&c16_w_y, sf_mex_create("y",
    &chartInstance->c16_is_active_c16_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c16_y, 1, c16_w_y);
  sf_mex_assign(&c16_st, c16_y, false);
  return c16_st;
}

static void set_sim_state_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance, const mxArray *c16_st)
{
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz c16_r;
  const mxArray *c16_u;
  int32_T c16_i;
  int32_T c16_i1;
  int32_T c16_i2;
  c16_u = sf_mex_dup(c16_st);
  c16_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c16_u, 0)),
                       "msg", &c16_r);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[0] = c16_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[4] = c16_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[0] = c16_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[4] = c16_r.Qd_SL_Info.ReceivedLength;
  for (c16_i = 0; c16_i < 7; c16_i++) {
    ((real_T *)&((char_T *)chartInstance->c16_msg)[0])[c16_i] = c16_r.Q[c16_i];
    ((real_T *)&((char_T *)chartInstance->c16_msg)[64])[c16_i] = c16_r.Qd[c16_i];
    ((real_T *)&((char_T *)chartInstance->c16_msg)[128])[c16_i] =
      c16_r.Qdd[c16_i];
  }

  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[0] = c16_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[4] = c16_r.Qdd_SL_Info.ReceivedLength;
  for (c16_i1 = 0; c16_i1 < 4; c16_i1++) {
    ((uint8_T *)&((char_T *)chartInstance->c16_msg)[192])[c16_i1] =
      c16_r.Controller[c16_i1];
  }

  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[0] = c16_r.Controller_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[4] = c16_r.Controller_SL_Info.ReceivedLength;
  for (c16_i2 = 0; c16_i2 < 7; c16_i2++) {
    ((real_T *)&((char_T *)chartInstance->c16_msg)[208])[c16_i2] =
      c16_r.Other[c16_i2];
  }

  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[0] = c16_r.Other_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[4] = c16_r.Other_SL_Info.ReceivedLength;
  chartInstance->c16_is_active_c16_sliding_controller2 = c16_g_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c16_u, 1)),
     "is_active_c16_sliding_controller2");
  sf_mex_destroy(&c16_u);
  sf_mex_destroy(&c16_st);
}

static void sf_gateway_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz c16_r;
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz
    *c16_gpu_blankMsg;
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_gpu_msg;
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_gpu_r;
  hipMalloc(&c16_gpu_r, 272UL);
  hipMalloc(&c16_gpu_msg, 272UL);
  hipMalloc(&c16_gpu_blankMsg, 272UL);
  chartInstance->c16_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[56])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[56])[4];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[120])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[120])[4];
  hipMemcpy(c16_gpu_blankMsg, chartInstance->c16_blankMsg, 272UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller2_kernel1<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c16_gpu_blankMsg, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c16_blankMsg, c16_gpu_blankMsg, 272UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[184])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[184])[4];
  hipMemcpy(c16_gpu_blankMsg, chartInstance->c16_blankMsg, 272UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller2_kernel2<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c16_gpu_blankMsg, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c16_blankMsg, c16_gpu_blankMsg, 272UL,
             hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[200])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[200])[4];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[0] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[264])[0];
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[4] = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_blankMsg)[264])[4];
  c16_r.Q_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [56])[0];
  c16_r.Q_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [56])[4];
  c16_r.Qd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [120])[0];
  c16_r.Qd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [120])[4];
  hipMemcpy(c16_gpu_r, &c16_r, 272UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_blankMsg, chartInstance->c16_blankMsg, 272UL,
             hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller2_kernel3<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c16_gpu_r, c16_gpu_blankMsg, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(chartInstance->c16_blankMsg, c16_gpu_blankMsg, 272UL,
             hipMemcpyDeviceToHost);
  hipMemcpy(&c16_r, c16_gpu_r, 272UL, hipMemcpyDeviceToHost);
  c16_r.Qdd_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [184])[0];
  c16_r.Qdd_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [184])[4];
  hipMemcpy(c16_gpu_r, &c16_r, 272UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller2_kernel4<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c16_gpu_msg, c16_gpu_r);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c16_r, c16_gpu_r, 272UL, hipMemcpyDeviceToHost);
  c16_r.Controller_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[0];
  c16_r.Controller_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [200])[4];
  hipMemcpy(c16_gpu_r, &c16_r, 272UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller2_kernel5<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c16_gpu_msg, c16_gpu_r);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  hipMemcpy(&c16_r, c16_gpu_r, 272UL, hipMemcpyDeviceToHost);
  c16_r.Other_SL_Info.CurrentLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [264])[0];
  c16_r.Other_SL_Info.ReceivedLength = *(uint32_T *)&((char_T *)
    (c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)chartInstance->c16_msg)
    [264])[4];
  c16_eML_blk_kernel(chartInstance, &c16_r);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[0] = c16_r.Q_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[56])[4] = c16_r.Q_SL_Info.ReceivedLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[0] = c16_r.Qd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[120])[4] = c16_r.Qd_SL_Info.ReceivedLength;
  hipMemcpy(c16_gpu_r, &c16_r, 272UL, hipMemcpyHostToDevice);
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller2_kernel6<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c16_gpu_r, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[0] = c16_r.Qdd_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[184])[4] = c16_r.Qdd_SL_Info.ReceivedLength;
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller2_kernel7<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c16_gpu_r, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[0] = c16_r.Controller_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[200])[4] = c16_r.Controller_SL_Info.ReceivedLength;
  hipMemcpy(c16_gpu_msg, chartInstance->c16_msg, 272UL, hipMemcpyHostToDevice);
  c16_sf_gateway_c16_sliding_controller2_kernel8<<<dim3(1U, 1U, 1U), dim3(32U,
    1U, 1U)>>>(c16_gpu_r, c16_gpu_msg);
  hipMemcpy(chartInstance->c16_msg, c16_gpu_msg, 272UL, hipMemcpyDeviceToHost);
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[0] = c16_r.Other_SL_Info.CurrentLength;
  *(uint32_T *)&((char_T *)(c16_SL_Bus_ROSVariableLengthArrayInfo *)&((char_T *)
    chartInstance->c16_msg)[264])[4] = c16_r.Other_SL_Info.ReceivedLength;
  c16_do_animation_call_c16_sliding_controller2(chartInstance);
  hipFree(c16_gpu_blankMsg);
  hipFree(c16_gpu_msg);
  hipFree(c16_gpu_r);
}

static void mdl_start_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c16_errCode;
  c16_errCode = hipGetLastError();
  if (c16_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c16_errCode, (char_T *)hipGetErrorName
                       (c16_errCode), (char_T *)hipGetErrorString(c16_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c16_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc16_sliding_controller2
  (SFc16_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c16_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c16_nameCaptureInfo = NULL;
  c16_nameCaptureInfo = NULL;
  sf_mex_assign(&c16_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c16_nameCaptureInfo;
}

static void c16_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_b_msg, const char_T *c16_identifier,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_y)
{
  emlrtMsgIdentifier c16_thisId;
  c16_thisId.fIdentifier = const_cast<const char_T *>(c16_identifier);
  c16_thisId.fParent = NULL;
  c16_thisId.bParentIsCell = false;
  c16_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c16_b_msg), &c16_thisId,
    c16_y);
  sf_mex_destroy(&c16_b_msg);
}

static void c16_b_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_y)
{
  static const char_T *c16_fieldNames[10] = { "Q", "Q_SL_Info", "Qd",
    "Qd_SL_Info", "Qdd", "Qdd_SL_Info", "Controller", "Controller_SL_Info",
    "Other", "Other_SL_Info" };

  emlrtMsgIdentifier c16_thisId;
  c16_thisId.fParent = c16_parentId;
  c16_thisId.bParentIsCell = false;
  sf_mex_check_struct(c16_parentId, c16_u, 10, c16_fieldNames, 0U, NULL);
  c16_thisId.fIdentifier = "Q";
  c16_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u, "Q",
    "Q", 0)), &c16_thisId, c16_y->Q);
  c16_thisId.fIdentifier = "Q_SL_Info";
  c16_y->Q_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Q_SL_Info", "Q_SL_Info", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "Qd";
  c16_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u, "Qd",
    "Qd", 0)), &c16_thisId, c16_y->Qd);
  c16_thisId.fIdentifier = "Qd_SL_Info";
  c16_y->Qd_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Qd_SL_Info", "Qd_SL_Info", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "Qdd";
  c16_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u, "Qdd",
    "Qdd", 0)), &c16_thisId, c16_y->Qdd);
  c16_thisId.fIdentifier = "Qdd_SL_Info";
  c16_y->Qdd_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Qdd_SL_Info", "Qdd_SL_Info", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "Controller";
  c16_f_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u,
    "Controller", "Controller", 0)), &c16_thisId, c16_y->Controller);
  c16_thisId.fIdentifier = "Controller_SL_Info";
  c16_y->Controller_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Controller_SL_Info", "Controller_SL_Info", 0)),
    &c16_thisId);
  c16_thisId.fIdentifier = "Other";
  c16_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getfield(c16_u,
    "Other", "Other", 0)), &c16_thisId, c16_y->Other);
  c16_thisId.fIdentifier = "Other_SL_Info";
  c16_y->Other_SL_Info = c16_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "Other_SL_Info", "Other_SL_Info", 0)), &c16_thisId);
  sf_mex_destroy(&c16_u);
}

static void c16_c_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  real_T c16_y[7])
{
  real_T c16_dv[7];
  int32_T c16_i;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), c16_dv, 1, 0, 0U, 1, 0U, 1, 7);
  for (c16_i = 0; c16_i < 7; c16_i++) {
    c16_y[c16_i] = c16_dv[c16_i];
  }

  sf_mex_destroy(&c16_u);
}

static c16_SL_Bus_ROSVariableLengthArrayInfo c16_d_emlrt_marshallIn
  (SFc16_sliding_controller2InstanceStruct *chartInstance, const mxArray *c16_u,
   const emlrtMsgIdentifier *c16_parentId)
{
  static const char_T *c16_fieldNames[2] = { "CurrentLength", "ReceivedLength" };

  c16_SL_Bus_ROSVariableLengthArrayInfo c16_y;
  emlrtMsgIdentifier c16_thisId;
  c16_thisId.fParent = c16_parentId;
  c16_thisId.bParentIsCell = false;
  sf_mex_check_struct(c16_parentId, c16_u, 2, c16_fieldNames, 0U, NULL);
  c16_thisId.fIdentifier = "CurrentLength";
  c16_y.CurrentLength = c16_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "CurrentLength", "CurrentLength", 0)), &c16_thisId);
  c16_thisId.fIdentifier = "ReceivedLength";
  c16_y.ReceivedLength = c16_e_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getfield(c16_u, "ReceivedLength", "ReceivedLength", 0)), &c16_thisId);
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static uint32_T c16_e_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  uint32_T c16_b_u;
  uint32_T c16_y;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), &c16_b_u, 1, 7, 0U, 0, 0U, 0);
  c16_y = c16_b_u;
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static void c16_f_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId,
  uint8_T c16_y[4])
{
  int32_T c16_i;
  uint8_T c16_uv[4];
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), c16_uv, 1, 3, 0U, 1, 0U, 1, 4);
  for (c16_i = 0; c16_i < 4; c16_i++) {
    c16_y[c16_i] = c16_uv[c16_i];
  }

  sf_mex_destroy(&c16_u);
}

static uint8_T c16_g_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_b_is_active_c16_sliding_controller2, const
  char_T *c16_identifier)
{
  emlrtMsgIdentifier c16_thisId;
  uint8_T c16_y;
  c16_thisId.fIdentifier = const_cast<const char_T *>(c16_identifier);
  c16_thisId.fParent = NULL;
  c16_thisId.bParentIsCell = false;
  c16_y = c16_h_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c16_b_is_active_c16_sliding_controller2), &c16_thisId);
  sf_mex_destroy(&c16_b_is_active_c16_sliding_controller2);
  return c16_y;
}

static uint8_T c16_h_emlrt_marshallIn(SFc16_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c16_u, const emlrtMsgIdentifier *c16_parentId)
{
  uint8_T c16_b_u;
  uint8_T c16_y;
  sf_mex_import(c16_parentId, sf_mex_dup(c16_u), &c16_b_u, 1, 3, 0U, 0, 0U, 0);
  c16_y = c16_b_u;
  sf_mex_destroy(&c16_u);
  return c16_y;
}

static void c16_eML_blk_kernel(SFc16_sliding_controller2InstanceStruct
  *chartInstance, c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *
  c16_b_blankMsg)
{
  c16_b_blankMsg->Q_SL_Info.CurrentLength = 7U;
  c16_b_blankMsg->Qd_SL_Info.CurrentLength = 7U;
  c16_b_blankMsg->Qdd_SL_Info.CurrentLength = 7U;
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller2_kernel1
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i = (int32_T)c16_threadId;
  if (c16_i < 7) {
    c16_b_msg->Q[c16_i] = c16_b_blankMsg->Q[c16_i];
    c16_b_msg->Qd[c16_i] = c16_b_blankMsg->Qd[c16_i];
    c16_b_msg->Qdd[c16_i] = c16_b_blankMsg->Qdd[c16_i];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller2_kernel2
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i1;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i1 = (int32_T)c16_threadId;
  if (c16_i1 < 4) {
    c16_b_msg->Controller[c16_i1] = c16_b_blankMsg->Controller[c16_i1];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller2_kernel3
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_blankMsg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i2;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i2 = (int32_T)c16_threadId;
  if (c16_i2 < 7) {
    c16_b_msg->Other[c16_i2] = c16_b_blankMsg->Other[c16_i2];
    c16_r->Q[c16_i2] = c16_b_msg->Q[c16_i2];
    c16_r->Qd[c16_i2] = c16_b_msg->Qd[c16_i2];
    c16_r->Qdd[c16_i2] = c16_b_msg->Qdd[c16_i2];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller2_kernel4
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r)
{
  uint64_T c16_threadId;
  int32_T c16_i3;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i3 = (int32_T)c16_threadId;
  if (c16_i3 < 4) {
    c16_r->Controller[c16_i3] = c16_b_msg->Controller[c16_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller2_kernel5
  (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg,
   c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r)
{
  uint64_T c16_threadId;
  int32_T c16_i4;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i4 = (int32_T)c16_threadId;
  if (c16_i4 < 7) {
    c16_r->Other[c16_i4] = c16_b_msg->Other[c16_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller2_kernel6(const
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i5;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i5 = (int32_T)c16_threadId;
  if (c16_i5 < 7) {
    c16_b_msg->Q[c16_i5] = c16_r->Q[c16_i5];
    c16_b_msg->Qd[c16_i5] = c16_r->Qd[c16_i5];
    c16_b_msg->Qdd[c16_i5] = c16_r->Qdd[c16_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller2_kernel7(const
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i6;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i6 = (int32_T)c16_threadId;
  if (c16_i6 < 4) {
    c16_b_msg->Controller[c16_i6] = c16_r->Controller[c16_i6];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c16_sf_gateway_c16_sliding_controller2_kernel8(const
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_r,
  c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *c16_b_msg)
{
  uint64_T c16_threadId;
  int32_T c16_i7;
  c16_threadId = (uint64_T)mwGetGlobalThreadIndexInXDimension();
  c16_i7 = (int32_T)c16_threadId;
  if (c16_i7 < 7) {
    c16_b_msg->Other[c16_i7] = c16_r->Other[c16_i7];
  }
}

static void init_dsm_address_info(SFc16_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc16_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c16_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c16_blankMsg =
    (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *)
    ssGetInputPortSignal_wrapper(chartInstance->S, 0);
  chartInstance->c16_msg =
    (c16_SL_Bus_sliding_controller2_DesiredJointsCmdRequest_9kimzz *)
    ssGetOutputPortSignal_wrapper(chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c16_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(4060299274U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(855152936U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1233590466U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2400911185U);
}

mxArray *sf_c16_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c16_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c16_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c16_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WfBop8ZSb8AkJdbnA4JF2j4kG+/ggNl+i"
    "H2RxBwvyKK+yH8zOL4xOSSzLLU+GRDs/jinMyUzLz0+OT8vJKi/Jyc1CIjJPNBAAAioxyr"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c16_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sj0i2VOwELrqvUaOaT8uA6C";
}

static void sf_opaque_initialize_c16_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c16_sliding_controller2
    ((SFc16_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c16_sliding_controller2((SFc16_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c16_sliding_controller2(void *chartInstanceVar)
{
  enable_c16_sliding_controller2((SFc16_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c16_sliding_controller2(void *chartInstanceVar)
{
  disable_c16_sliding_controller2((SFc16_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c16_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c16_sliding_controller2((SFc16_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c16_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c16_sliding_controller2
    ((SFc16_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c16_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c16_sliding_controller2((SFc16_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c16_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc16_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c16_sliding_controller2
      ((SFc16_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc16_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc16_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c16_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c16_sliding_controller2((SFc16_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c16_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c16_sliding_controller2((SFc16_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc16_sliding_controller2((SFc16_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c16_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c16_sliding_controller2
      ((SFc16_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc16_sliding_controller2
      ((SFc16_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c16_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQphU3aIAmMIoCvRRocmpyS3MI0ktrm5IQAXJkhLLTm7FejsStlrv0/lD2OUCBHHr",
    "IE+QW5JgXyHvkDXrtI2SWpGSZIqk4Qoy0KAGaXvKb2Zn55mflbfT2PLxu4b39reddx+fXeLe8/P",
    "qqWG8s3Pn7Te+nYv0XCgkb7xNFYu01XoLE8BS05NYwKXpiJCthTIxAgaCITaQyddo0iy1nYtK1g",
    "jp9+lnEaBRE0vJwF2VJOBD8DLUl1uyjnjZTQE0XIDSRknYcdTkZzy1WZupHQCfaxk0uaDCBTZxZ",
    "es9ywxIOnVOgPaENQYv1uW2BIQZ8c1rrpvNUBzOgjBPOiKj0NiI6gAQDbOAgCfHvwBp0qgyjEVF",
    "mFyKSgu6zSaZTCijrZBo/HDNBjFSM8E7MfSe4bNs+R3v2ZAi8ISBo264CMkkkE6ae/6CLnnYEOe",
    "bQhmM7rtcWwIl15B8ymIKqjdvIlykoMoaBqN00C0jnNGNrniXLMMNiOCRqhyJ/GsLa7MXM0QFBn",
    "mCIEnUwyJzs6aFiKYa3VpuNey4zV5WMjXOy9SpYpq2TQhMLc21dKnzCua6FDWXShxR4prVNDGmG",
    "5VqrcVqzcCgxwC6966vBCobEFzBfipBV0pWWAFnfeYKN5SKSWm1k7GPytvv95c/LsJ4woEaEQlU",
    "XUIRpwJhl4a3XFjLtuEcgWmUy86rAeYasQnl6ZEV7KtUEY9LQRM5dcIzWAmM9Ri6xEg40Fk0TzH",
    "G5CkcJjSB0DYZx2MOyQWxFTLRrbTtYdykzZ23QVLGkglWLVYdtqOMS6iyBAzERciq6SsZB0ePz8",
    "AJgthElmBjvYv9SZ13cvYoKN8/ue+fz7JuPmGczufLz7oKejQo93sKzvO+NVvO+LfxvYzZ/F+Ru",
    "lvbZLMk53Bbeb17ee3Ez+vPk9bu/X/zy/NW9dfZ/27rc/L9VrH+YNdp54aRL+eqwjxfs2qzQ//2",
    "C/q1irf+4zx4cDqadvjpJD8iADB/ZnYd+zs8Ke1sle2fvb7uOj1mW5aeivbA4mLg1sfm4dvofLd",
    "h7fUU8bhTv8+uf39aT/267zGNVvK5diNc1V7kX8vbT97+9vZ58vv/vK+y/U+L7TjbXj4jrFnBEf",
    "354pDkLsdSPqBRGSc5BPaioz0/N98vKeVcs92+x83+5z8/7x8yvq5bzrlhuXf8uO8e/NHzTPPBK",
    "+K0v2I91z1efG//eu9w56Mdi/ev8p44fMR5WnHqLz3iEHVV9/Q/k6Qems7M8",
    ""
  };

  static char newstr [1261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c16_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c16_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2301202856U));
  ssSetChecksum1(S,(1904633869U));
  ssSetChecksum2(S,(2313665958U));
  ssSetChecksum3(S,(698319673U));
}

static void mdlRTW_c16_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c16_sliding_controller2(SimStruct *S)
{
  SFc16_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc16_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc16_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc16_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc16_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c16_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c16_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c16_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c16_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c16_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c16_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c16_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c16_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c16_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c16_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c16_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c16_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c16_JITStateAnimation,
    chartInstance->c16_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c16_sliding_controller2(chartInstance);
}

void c16_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c16_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c16_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c16_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c16_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
