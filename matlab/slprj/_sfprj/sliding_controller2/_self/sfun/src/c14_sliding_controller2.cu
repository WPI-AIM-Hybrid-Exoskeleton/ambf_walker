/* Include files */

#include "sliding_controller2_sfun.h"
#include "c14_sliding_controller2.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void initialize_params_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void enable_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void disable_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void c14_do_animation_call_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void ext_mode_exec_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static const mxArray *get_sim_state_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void set_sim_state_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance, const mxArray *c14_st);
static void sf_gateway_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void mdl_start_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void mdl_terminate_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void mdl_setup_runtime_resources_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void initSimStructsc14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance);
static void c14_eML_blk_kernel(SFc14_sliding_controller2InstanceStruct
  *chartInstance, real_T c14_b_q[2], real_T c14_b_y[4]);
static void c14_emlrt_marshallIn(SFc14_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c14_b_y, const char_T *c14_identifier, real_T
  c14_c_y[4]);
static void c14_b_emlrt_marshallIn(SFc14_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId,
  real_T c14_b_y[4]);
static uint8_T c14_c_emlrt_marshallIn(SFc14_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c14_b_is_active_c14_sliding_controller2, const
  char_T *c14_identifier);
static uint8_T c14_d_emlrt_marshallIn(SFc14_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId);
static void init_dsm_address_info(SFc14_sliding_controller2InstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc14_sliding_controller2InstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c14_fEmlrtCtx, (const char_T *)
    "distrib_computing_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c14_is_active_c14_sliding_controller2 = 0U;
  hipGetLastError();
}

static void initialize_params_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
}

static void enable_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c14_do_animation_call_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
}

static const mxArray *get_sim_state_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
  const mxArray *c14_b_y = NULL;
  const mxArray *c14_c_y = NULL;
  const mxArray *c14_d_y = NULL;
  const mxArray *c14_st;
  c14_st = NULL;
  c14_b_y = NULL;
  sf_mex_assign(&c14_b_y, sf_mex_createcellmatrix(2, 1), false);
  c14_c_y = NULL;
  sf_mex_assign(&c14_c_y, sf_mex_create("y", *chartInstance->c14_y, 0, 0U, 1U,
    0U, 2, 2, 2), false);
  sf_mex_setcell(c14_b_y, 0, c14_c_y);
  c14_d_y = NULL;
  sf_mex_assign(&c14_d_y, sf_mex_create("y",
    &chartInstance->c14_is_active_c14_sliding_controller2, 3, 0U, 0U, 0U, 0),
                false);
  sf_mex_setcell(c14_b_y, 1, c14_d_y);
  sf_mex_assign(&c14_st, c14_b_y, false);
  return c14_st;
}

static void set_sim_state_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance, const mxArray *c14_st)
{
  const mxArray *c14_u;
  c14_u = sf_mex_dup(c14_st);
  c14_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c14_u, 0)), "y",
                       *chartInstance->c14_y);
  chartInstance->c14_is_active_c14_sliding_controller2 = c14_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c14_u, 1)),
     "is_active_c14_sliding_controller2");
  sf_mex_destroy(&c14_u);
  sf_mex_destroy(&c14_st);
}

static void sf_gateway_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
  chartInstance->c14_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  c14_eML_blk_kernel(chartInstance, *chartInstance->c14_q, *chartInstance->c14_y);
  c14_do_animation_call_c14_sliding_controller2(chartInstance);
}

static void mdl_start_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
}

static void mdl_terminate_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
  hipError_t c14_errCode;
  c14_errCode = hipGetLastError();
  if (c14_errCode != hipSuccess) {
    emlrtThinCUDAError((uint32_T)c14_errCode, (char_T *)hipGetErrorName
                       (c14_errCode), (char_T *)hipGetErrorString(c14_errCode),
                       (char_T *)"SimGPUErrorChecks",
                       chartInstance->c14_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
}

static void initSimStructsc14_sliding_controller2
  (SFc14_sliding_controller2InstanceStruct *chartInstance)
{
}

const mxArray *sf_c14_sliding_controller2_get_eml_resolved_functions_info()
{
  const mxArray *c14_nameCaptureInfo = NULL;
  c14_nameCaptureInfo = NULL;
  sf_mex_assign(&c14_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c14_nameCaptureInfo;
}

static void c14_eML_blk_kernel(SFc14_sliding_controller2InstanceStruct
  *chartInstance, real_T c14_b_q[2], real_T c14_b_y[4])
{
  c14_b_y[0] = -0.324 * muDoubleScalarSin(c14_b_q[0]) - 0.39 * muDoubleScalarSin
    (c14_b_q[0] + c14_b_q[1]);
  c14_b_y[2] = 0.324 * muDoubleScalarCos(c14_b_q[0]) + 0.39 * muDoubleScalarSin
    (c14_b_q[0] + c14_b_q[1]);
  c14_b_y[1] = -0.39 * muDoubleScalarSin(c14_b_q[0] + c14_b_q[1]);
  c14_b_y[3] = 0.39 * muDoubleScalarCos(c14_b_q[0] + c14_b_q[1]);
}

static void c14_emlrt_marshallIn(SFc14_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c14_b_y, const char_T *c14_identifier, real_T
  c14_c_y[4])
{
  emlrtMsgIdentifier c14_thisId;
  c14_thisId.fIdentifier = const_cast<const char_T *>(c14_identifier);
  c14_thisId.fParent = NULL;
  c14_thisId.bParentIsCell = false;
  c14_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c14_b_y), &c14_thisId,
    c14_c_y);
  sf_mex_destroy(&c14_b_y);
}

static void c14_b_emlrt_marshallIn(SFc14_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId,
  real_T c14_b_y[4])
{
  real_T c14_dv[4];
  int32_T c14_i;
  sf_mex_import(c14_parentId, sf_mex_dup(c14_u), c14_dv, 1, 0, 0U, 1, 0U, 2, 2,
                2);
  for (c14_i = 0; c14_i < 4; c14_i++) {
    c14_b_y[c14_i] = c14_dv[c14_i];
  }

  sf_mex_destroy(&c14_u);
}

static uint8_T c14_c_emlrt_marshallIn(SFc14_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c14_b_is_active_c14_sliding_controller2, const
  char_T *c14_identifier)
{
  emlrtMsgIdentifier c14_thisId;
  uint8_T c14_b_y;
  c14_thisId.fIdentifier = const_cast<const char_T *>(c14_identifier);
  c14_thisId.fParent = NULL;
  c14_thisId.bParentIsCell = false;
  c14_b_y = c14_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c14_b_is_active_c14_sliding_controller2), &c14_thisId);
  sf_mex_destroy(&c14_b_is_active_c14_sliding_controller2);
  return c14_b_y;
}

static uint8_T c14_d_emlrt_marshallIn(SFc14_sliding_controller2InstanceStruct
  *chartInstance, const mxArray *c14_u, const emlrtMsgIdentifier *c14_parentId)
{
  uint8_T c14_b_u;
  uint8_T c14_b_y;
  sf_mex_import(c14_parentId, sf_mex_dup(c14_u), &c14_b_u, 1, 3, 0U, 0, 0U, 0);
  c14_b_y = c14_b_u;
  sf_mex_destroy(&c14_u);
  return c14_b_y;
}

static void init_dsm_address_info(SFc14_sliding_controller2InstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc14_sliding_controller2InstanceStruct
  *chartInstance)
{
  chartInstance->c14_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c14_q = (real_T (*)[2])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c14_y = (real_T (*)[4])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c14_sliding_controller2_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(552332206U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2608078302U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1441289006U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(4066884445U);
}

mxArray *sf_c14_sliding_controller2_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c14_sliding_controller2_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c14_sliding_controller2_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c14_sliding_controller2(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0iziA9Bsg6WchoF8AyKqEhQuUJt9+BQfK9EPsjyDgfk"
    "UU90P4mcXxicklmWWp8cmGJvHFOZkpmXnp8cn5eSVF+Tk5qUVGSOaDAABbQRvX"
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c14_sliding_controller2_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sjS5Om55RaK9v6MEldWgzUF";
}

static void sf_opaque_initialize_c14_sliding_controller2(void *chartInstanceVar)
{
  initialize_params_c14_sliding_controller2
    ((SFc14_sliding_controller2InstanceStruct*) chartInstanceVar);
  initialize_c14_sliding_controller2((SFc14_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c14_sliding_controller2(void *chartInstanceVar)
{
  enable_c14_sliding_controller2((SFc14_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_disable_c14_sliding_controller2(void *chartInstanceVar)
{
  disable_c14_sliding_controller2((SFc14_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_gateway_c14_sliding_controller2(void *chartInstanceVar)
{
  sf_gateway_c14_sliding_controller2((SFc14_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c14_sliding_controller2(SimStruct*
  S)
{
  return get_sim_state_c14_sliding_controller2
    ((SFc14_sliding_controller2InstanceStruct *)sf_get_chart_instance_ptr(S));/* raw sim ctx */
}

static void sf_opaque_set_sim_state_c14_sliding_controller2(SimStruct* S, const
  mxArray *st)
{
  set_sim_state_c14_sliding_controller2((SFc14_sliding_controller2InstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c14_sliding_controller2(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc14_sliding_controller2InstanceStruct*) chartInstanceVar
      )->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_sliding_controller2_optimization_info();
    }

    mdl_cleanup_runtime_resources_c14_sliding_controller2
      ((SFc14_sliding_controller2InstanceStruct*) chartInstanceVar);
    ((SFc14_sliding_controller2InstanceStruct*) chartInstanceVar)->
      ~SFc14_sliding_controller2InstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c14_sliding_controller2(void *chartInstanceVar)
{
  mdl_start_c14_sliding_controller2((SFc14_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c14_sliding_controller2(void
  *chartInstanceVar)
{
  mdl_terminate_c14_sliding_controller2((SFc14_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc14_sliding_controller2((SFc14_sliding_controller2InstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c14_sliding_controller2(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c14_sliding_controller2
      ((SFc14_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
    initSimStructsc14_sliding_controller2
      ((SFc14_sliding_controller2InstanceStruct*)sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c14_sliding_controller2_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [18] = {
    "eNrtV81u20YQpgXXSYrWMIICvRSoc0qPbVAHySGtbf0gQq3aMP3Tm7BejsS1lrvM/tBW36Gnom/",
    "QYw99kp577wsU6At0lqRkhSIp24KNpCgBmlrym9mZ+eZn7a10ex5e63hvP/a8NXw+xLvhZdcH+X",
    "pl5s7er3pP8/VPKCRsdEAUibRXewkSwSFoya1hUnTFQJbCmBiAAkERG0tlqrRpFlnOxKhjBXX69",
    "GnIaOiH0vJgF2VJsC/4GLXF1hygnhZTQE0HIDChknYYdjgZTi1W5qIZAh1pG9W5oMH4NnZm6Z7l",
    "hsUc2pdAu0IbghbrK9t8Qww0zWWlm85T7U+AMoo5I6LU25BoH2IMsIHjOMC/+9agU0UYDYkyuxC",
    "SBPQeG6U6pYCiTqbxwxkTxEjFCG9HvOkE52074GhPTwbAawKCtu0qIKNYMmGq+fc76GlbkDMOLT",
    "izw2ptPryxjvwTBhegKuM2aMoEFBnCvqjcNA1I+zJla5ol8zDDIjghaocifxqCyuzFzNE+QZ7gC",
    "CWqYJA62dVHiiUY3kptNuq6zFxUMjbKyNaLYKm2dgJ1LEy1dahoEs51JexIxnuQAE+1togh9bBM",
    "azlOaxYcSQywS+/qarCCIfE5rClFwErpSgqAtO98j43lbSS12sioicnb2tub/zwP6woDakAolHU",
    "BRZgGjFka3mptAdOOewSiVSY1rwycZcgilKcHVrQupBphTGqayJULjtFKYKSHyCVWwrHGoqmDOS",
    "4X4SihIQSuwTAOPSwbxJbERLvWtoN1lzAzboGmisUlrFqsOmxDbZdQ4xiOxUjIC9FRMvLzHp+FF",
    "wCzjSjBxHAX+5cad3D3MircPPvSu5pnH11jnk3kis8vZvSslOjxZp7FfR816vdt4K+Vyfydkfu4",
    "sM9qQc7hNvD+8Hzzl382fxv/8fODv14//fXVMvv/3rjZ/F/P159NGu20cJK5fHXY1zN2rZbo/3R",
    "G/0a+1uf+1n60tXVIvnuZPO+1eXA6/PG4k/GzwN5Gwd7J+03X8THL0vxUtBvkBxO3JjYb107/ix",
    "l71xbE41H+Prv+/nY5+U+2izyuLpDfwF/jQt7efv/N7eXks/1/WGD/kwLfT9K53ieuW0CffvV1X",
    "3MWYKn3qRRGSc5BPSupz9vm+03lvHuWe1/s/F/u7nm/zvy6bznvnuWW9e+mc/xdw9fNA6+A33iH",
    "/Vj2fHXX+D+9m52DPs/X30z/1WmGjAclp978Mx5hB2Vf/wN5+i+a+bGK",
    ""
  };

  static char newstr [1257] = "";
  newstr[0] = '\0';
  for (i = 0; i < 18; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c14_sliding_controller2(SimStruct *S)
{
  const char* newstr = sf_c14_sliding_controller2_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2501929482U));
  ssSetChecksum1(S,(2041323763U));
  ssSetChecksum2(S,(3808924622U));
  ssSetChecksum3(S,(1034168136U));
}

static void mdlRTW_c14_sliding_controller2(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c14_sliding_controller2(SimStruct *S)
{
  SFc14_sliding_controller2InstanceStruct *chartInstance;
  chartInstance = (SFc14_sliding_controller2InstanceStruct *)utMalloc(sizeof
    (SFc14_sliding_controller2InstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc14_sliding_controller2InstanceStruct));
  chartInstance = new (chartInstance) SFc14_sliding_controller2InstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway =
    sf_opaque_gateway_c14_sliding_controller2;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c14_sliding_controller2;
  chartInstance->chartInfo.mdlStart =
    sf_opaque_mdl_start_c14_sliding_controller2;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c14_sliding_controller2;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c14_sliding_controller2;
  chartInstance->chartInfo.enableChart =
    sf_opaque_enable_c14_sliding_controller2;
  chartInstance->chartInfo.disableChart =
    sf_opaque_disable_c14_sliding_controller2;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c14_sliding_controller2;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c14_sliding_controller2;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c14_sliding_controller2;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c14_sliding_controller2;
  chartInstance->chartInfo.mdlSetWorkWidths =
    mdlSetWorkWidths_c14_sliding_controller2;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c14_JITStateAnimation,
    chartInstance->c14_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c14_sliding_controller2(chartInstance);
}

void c14_sliding_controller2_method_dispatcher(SimStruct *S, int_T method, void *
  data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c14_sliding_controller2(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c14_sliding_controller2(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c14_sliding_controller2(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c14_sliding_controller2_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
